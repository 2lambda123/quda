#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <quda_internal.h>
#include <color_spinor_field.h>
#include <blas_quda.h>

#include <test_util.h>
#include <face_quda.h>

// include because of nasty globals used in the tests
#include <dslash_util.h>

// Wilson, clover-improved Wilson, and twisted mass are supported.
extern QudaDslashType dslash_type;
extern bool tune;
extern int device;
extern int xdim;
extern int ydim;
extern int zdim;
extern int tdim;
extern int gridsize_from_cmdline[];
extern int niter;

extern bool tune;

extern void usage(char** );

#if (__COMPUTE_CAPABILITY__ >= 200)
const int Nkernels = 32;
#else // exclude Heavy Quark Norm if on Tesla architecture
const int Nkernels = 31;
#endif

using namespace quda;

ColorSpinorField *xH, *yH, *zH, *wH, *vH, *hH, *lH;
ColorSpinorField *xD, *yD, *zD, *wD, *vD, *hD, *lD;
int Nspin;

void setPrec(ColorSpinorParam &param, const QudaPrecision precision)
{
  param.precision = precision;
  if (Nspin == 1 || precision == QUDA_DOUBLE_PRECISION) {
    param.fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
  } else {
    param.fieldOrder = QUDA_FLOAT4_FIELD_ORDER;
  }
}

void
display_test_info()
{
  printfQuda("running the following test:\n");
    
  printfQuda("S_dimension T_dimension Nspin\n");
  printfQuda("%d/%d/%d        %d      %d\n", xdim, ydim, zdim, tdim, Nspin);     

  printfQuda("Grid partition info:     X  Y  Z  T\n"); 
  printfQuda("                         %d  %d  %d  %d\n", 
	     dimPartitioned(0),
	     dimPartitioned(1),
	     dimPartitioned(2),
	     dimPartitioned(3)); 
  
  return;  
}

void initFields(int prec)
{
  // precisions used for the source field in the copyCuda() benchmark
  QudaPrecision high_aux_prec;
  QudaPrecision low_aux_prec;

  ColorSpinorParam param;
  param.nColor = 3;
  // set spin according to the type of dslash
  Nspin = (dslash_type == QUDA_ASQTAD_DSLASH) ? 1 : 4;
  param.nSpin = Nspin;
  param.nDim = 4; // number of spacetime dimensions

  param.pad = 0; // padding must be zero for cpu fields
  param.siteSubset = QUDA_PARITY_SITE_SUBSET;
  if (param.siteSubset == QUDA_PARITY_SITE_SUBSET) param.x[0] = xdim/2;
  else param.x[0] = xdim;
  param.x[1] = ydim;
  param.x[2] = zdim;
  param.x[3] = tdim;

  param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
  param.gammaBasis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;
  param.precision = QUDA_DOUBLE_PRECISION;
  param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;

  param.create = QUDA_ZERO_FIELD_CREATE;

  vH = new cpuColorSpinorField(param);
  wH = new cpuColorSpinorField(param);
  xH = new cpuColorSpinorField(param);
  yH = new cpuColorSpinorField(param);
  zH = new cpuColorSpinorField(param);
  hH = new cpuColorSpinorField(param);
  lH = new cpuColorSpinorField(param);

  static_cast<cpuColorSpinorField*>(vH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(wH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(xH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(yH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(zH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(hH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(lH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);

  // Now set the parameters for the cuda fields
  //param.pad = xdim*ydim*zdim/2;
  
  if (param.nSpin == 4) param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
  param.create = QUDA_ZERO_FIELD_CREATE;

  switch(prec) {
  case 0:
    setPrec(param, QUDA_HALF_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_SINGLE_PRECISION;
    break;
  case 1:
    setPrec(param, QUDA_SINGLE_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  case 2:
    setPrec(param, QUDA_DOUBLE_PRECISION);
    high_aux_prec = QUDA_SINGLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  }

  checkCudaError();

  vD = new cudaColorSpinorField(param);
  wD = new cudaColorSpinorField(param);
  xD = new cudaColorSpinorField(param);
  yD = new cudaColorSpinorField(param);
  zD = new cudaColorSpinorField(param);

  setPrec(param, high_aux_prec);
  hD = new cudaColorSpinorField(param);

  setPrec(param, low_aux_prec);
  lD = new cudaColorSpinorField(param);

  // check for successful allocation
  checkCudaError();

  *vD = *vH;
  *wD = *wH;
  *xD = *xH;
  *yD = *yH;
  *zD = *zH;
  *hD = *hH;
  *lD = *lH;
}


void freeFields()
{

  // release memory
  delete vD;
  delete wD;
  delete xD;
  delete yD;
  delete zD;
  delete hD;
  delete lD;

  // release memory
  delete vH;
  delete wH;
  delete xH;
  delete yH;
  delete zH;
  delete hH;
  delete lH;
}


double benchmark(int kernel, const int niter) {

  double a, b, c;
  quda::Complex a2, b2, c2;

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  for (int i=0; i < niter; ++i) {

    switch (kernel) {

    case 0:
      blas::copy(*yD, *hD);
      break;

    case 1:
      blas::copy(*yD, *lD);
      break;
      
    case 2:
      blas::axpby(a, *xD, b, *yD);
      break;

    case 3:
      blas::xpy(*xD, *yD);
      break;

    case 4:
      blas::axpy(a, *xD, *yD);
      break;

    case 5:
      blas::xpay(*xD, a, *yD);
      break;

    case 6:
      blas::mxpy(*xD, *yD);
      break;

    case 7:
      blas::ax(a, *xD);
      break;

    case 8:
      blas::caxpy(a2, *xD, *yD);
      break;

    case 9:
      blas::caxpby(a2, *xD, b2, *yD);
      break;

    case 10:
      blas::cxpaypbz(*xD, a2, *yD, b2, *zD);
      break;

    case 11:
      blas::axpyBzpcx(a, *xD, *yD, b, *zD, c);
      break;

    case 12:
      blas::axpyZpbx(a, *xD, *yD, *zD, b);
      break;

    case 13:
      blas::caxpbypzYmbw(a2, *xD, b2, *yD, *zD, *wD);
      break;
      
    case 14:
      blas::cabxpyAx(a, b2, *xD, *yD);
      break;

    case 15:
      blas::caxpbypz(a2, *xD, b2, *yD, *zD);
      break;

    case 16:
      blas::caxpbypczpw(a2, *xD, b2, *yD, c2, *zD, *wD);
      break;

    case 17:
      blas::caxpyXmaz(a2, *xD, *yD, *zD);
      break;

      // double
    case 18:
      blas::norm2(*xD);
      break;

    case 19:
      blas::reDotProduct(*xD, *yD);
      break;

    case 20:
      blas::axpyNorm(a, *xD, *yD);
      break;

    case 21:
      blas::xmyNorm(*xD, *yD);
      break;
      
    case 22:
      blas::caxpyNorm(a2, *xD, *yD);
      break;

    case 23:
      blas::caxpyXmazNormX(a2, *xD, *yD, *zD);
      break;

    case 24:
      blas::cabxpyAxNorm(a, b2, *xD, *yD);
      break;

    // double2
    case 25:
      blas::cDotProduct(*xD, *yD);
      break;

    case 26:
      blas::xpaycDotzy(*xD, a, *yD, *zD);
      break;
      
    case 27:
      blas::caxpyDotzy(a2, *xD, *yD, *zD);
      break;

    // double3
    case 28:
      blas::cDotProductNormA(*xD, *yD);
      break;

    case 29:
      blas::cDotProductNormB(*xD, *yD);
      break;

    case 30:
      blas::caxpbypzYmbwcDotProductUYNormY(a2, *xD, b2, *yD, *zD, *wD, *vD);
      break;

    case 31:
      blas::HeavyQuarkResidualNorm(*xD, *yD);
      break;

    default:
      errorQuda("Undefined blas kernel %d\n", kernel);
    }
  }
  
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float runTime;
  hipEventElapsedTime(&runTime, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);

  double secs = runTime / 1000;
  return secs;
}

#define ERROR(a) fabs(blas::norm2(*a##D) - blas::norm2(*a##H)) / blas::norm2(*a##H)

double test(int kernel) {

  double a = 1.5, b = 2.5, c = 3.5;
  quda::Complex a2(a, b), b2(b, -c), c2(a+b, c*a);
  double error = 0;

  switch (kernel) {

  case 0:
    *hD = *hH;
    blas::copy(*yD, *hD);
    blas::copy(*yH, *hH);
    error = ERROR(y);
    break;

  case 1:
    *lD = *lH;
    blas::copy(*yD, *lD);
    blas::copy(*yH, *lH);
    error = ERROR(y);
    break;
      
  case 2:
    *xD = *xH;
    *yD = *yH;
    blas::axpby(a, *xD, b, *yD);
    blas::axpby(a, *xH, b, *yH);
    error = ERROR(y);
    break;

  case 3:
    *xD = *xH;
    *yD = *yH;
    blas::xpy(*xD, *yD);
    blas::xpy(*xH, *yH);
    error = ERROR(y);
    break;

  case 4:
    *xD = *xH;
    *yD = *yH;
    blas::axpy(a, *xD, *yD);
    blas::axpy(a, *xH, *yH);
    error = ERROR(y);
    break;

  case 5:
    *xD = *xH;
    *yD = *yH;
    blas::xpay(*xD, a, *yD);
    blas::xpay(*xH, a, *yH);
    error = ERROR(y);
    break;

  case 6:
    *xD = *xH;
    *yD = *yH;
    blas::mxpy(*xD, *yD);
    blas::mxpy(*xH, *yH);
    error = ERROR(y);
    break;

  case 7:
    *xD = *xH;
    blas::ax(a, *xD);
    blas::ax(a, *xH);
    error = ERROR(x);
    break;

  case 8:
    *xD = *xH;
    *yD = *yH;
    blas::caxpy(a2, *xD, *yD);
    blas::caxpy(a2, *xH, *yH);
    error = ERROR(y);
    break;

  case 9:
    *xD = *xH;
    *yD = *yH;
    blas::caxpby(a2, *xD, b2, *yD);
    blas::caxpby(a2, *xH, b2, *yH);
    error = ERROR(y);
    break;

  case 10:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::cxpaypbz(*xD, a2, *yD, b2, *zD);
    blas::cxpaypbz(*xH, a2, *yH, b2, *zH);
    error = ERROR(z);
    break;

  case 11:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::axpyBzpcx(a, *xD, *yD, b, *zD, c);
    blas::axpyBzpcx(a, *xH, *yH, b, *zH, c);
    error = ERROR(x) + ERROR(y);
    break;

  case 12:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::axpyZpbx(a, *xD, *yD, *zD, b);
    blas::axpyZpbx(a, *xH, *yH, *zH, b);
    error = ERROR(x) + ERROR(y);
    break;

  case 13:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    blas::caxpbypzYmbw(a2, *xD, b2, *yD, *zD, *wD);
    blas::caxpbypzYmbw(a2, *xH, b2, *yH, *zH, *wH);
    error = ERROR(z) + ERROR(y);
    break;
      
  case 14:
    *xD = *xH;
    *yD = *yH;
    blas::cabxpyAx(a, b2, *xD, *yD);
    blas::cabxpyAx(a, b2, *xH, *yH);
    error = ERROR(y) + ERROR(x);
    break;

  case 15:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpbypz(a2, *xD, b2, *yD, *zD);
      blas::caxpbypz(a2, *xH, b2, *yH, *zH);
      error = ERROR(z); }
    break;
    
  case 16:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    {blas::caxpbypczpw(a2, *xD, b2, *yD, c2, *zD, *wD);
      blas::caxpbypczpw(a2, *xH, b2, *yH, c2, *zH, *wH);
      error = ERROR(w); }
    break;

  case 17:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpyXmaz(a, *xD, *yD, *zD);
     blas::caxpyXmaz(a, *xH, *yH, *zH);
     error = ERROR(y) + ERROR(x);}
    break;

    // double
  case 18:
    *xD = *xH;
    error = fabs(blas::norm2(*xD) - blas::norm2(*xH)) / blas::norm2(*xH);
    break;
    
  case 19:
    *xD = *xH;
    *yD = *yH;
    error = fabs(blas::reDotProduct(*xD, *yD) - blas::reDotProduct(*xH, *yH)) / fabs(blas::reDotProduct(*xH, *yH));
    break;

  case 20:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::axpyNorm(a, *xD, *yD);
    double h = blas::axpyNorm(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 21:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::xmyNorm(*xD, *yD);
    double h = blas::xmyNorm(*xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;
    
  case 22:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::caxpyNorm(a, *xD, *yD);
    double h = blas::caxpyNorm(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 23:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {double d = blas::caxpyXmazNormX(a, *xD, *yD, *zD);
      double h = blas::caxpyXmazNormX(a, *xH, *yH, *zH);
      error = ERROR(y) + ERROR(x) + fabs(d-h)/fabs(h);}
    break;

  case 24:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::cabxpyAxNorm(a, b2, *xD, *yD);
      double h = blas::cabxpyAxNorm(a, b2, *xH, *yH);
      error = ERROR(x) + ERROR(y) + fabs(d-h)/fabs(h);}
    break;

    // double2
  case 25:
    *xD = *xH;
    *yD = *yH;
    error = abs(blas::cDotProduct(*xD, *yD) - blas::cDotProduct(*xH, *yH)) / abs(blas::cDotProduct(*xH, *yH));
    break;
    
  case 26:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { quda::Complex d = blas::xpaycDotzy(*xD, a, *yD, *zD);
      quda::Complex h = blas::xpaycDotzy(*xH, a, *yH, *zH);
      error =  fabs(blas::norm2(*yD) - blas::norm2(*yH)) / blas::norm2(*yH) + abs(d-h)/abs(h);
    }
    break;
    
  case 27:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {quda::Complex d = blas::caxpyDotzy(a, *xD, *yD, *zD);
      quda::Complex h = blas::caxpyDotzy(a, *xH, *yH, *zH);
    error = ERROR(y) + abs(d-h)/abs(h);}
    break;

    // double3
  case 28:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::cDotProductNormA(*xD, *yD);
      double3 h = blas::cDotProductNormA(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 29:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::cDotProductNormB(*xD, *yD);
      double3 h = blas::cDotProductNormB(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 30:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    *vD = *vH;
    { double3 d = blas::caxpbypzYmbwcDotProductUYNormY(a2, *xD, b2, *yD, *zD, *wD, *vD);
      double3 h = blas::caxpbypzYmbwcDotProductUYNormY(a2, *xH, b2, *yH, *zH, *wH, *vH);
      error = ERROR(z) + ERROR(y) + fabs(d.x - h.x) / fabs(h.x) + 
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 31:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::HeavyQuarkResidualNorm(*xD, *yD);
      double3 h = blas::HeavyQuarkResidualNorm(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + 
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  default:
    errorQuda("Undefined blas kernel %d\n", kernel);
  }

  return error;
}

int main(int argc, char** argv)
{
  for (int i = 1; i < argc; i++){
    if(process_command_line_option(argc, argv, &i) == 0){
      continue;
    } 
    printfQuda("ERROR: Invalid option:%s\n", argv[i]);
    usage(argv);
  }

  setSpinorSiteSize(24);
  initComms(argc, argv, gridsize_from_cmdline);
  display_test_info();
  initQuda(device);

  char *names[] = {
    "copyHS",
    "copyLS",
    "axpby",
    "xpy",
    "axpy",
    "xpay",
    "mxpy",
    "ax",
    "caxpy",
    "caxpby",
    "cxpaypbz",
    "axpyBzpcx",
    "axpyZpbx",
    "caxpbypzYmbw",
    "cabxpyAx",
    "caxpbypz",
    "caxpbypczpw",
    "caxpyXmaz",
    "norm",
    "reDotProduct",
    "axpyNorm",
    "xmyNorm",
    "caxpyNorm",
    "caxpyXmazNormX",
    "cabxpyAxNorm",
    "cDotProduct",
    "xpaycDotzy",
    "caxpyDotzy",
    "cDotProductNormA",
    "cDotProductNormB",
    "caxpbypzYmbwcDotProductWYNormY",
    "HeavyQuarkResidualNorm"
  };

  char *prec_str[] = {"half", "single", "double"};
  
  // Only benchmark double precision if supported
#if (__COMPUTE_CAPABILITY__ >= 130)
  int Nprec = 3;
#else
  int Nprec = 2;
#endif

  // enable the tuning
  quda::blas::setTuning(tune ? QUDA_TUNE_YES : QUDA_TUNE_NO, QUDA_SILENT);

  for (int prec = 0; prec < Nprec; prec++) {

    printfQuda("\nBenchmarking %s precision with %d iterations...\n\n", prec_str[prec], niter);
    initFields(prec);

    for (int kernel = 0; kernel < Nkernels; kernel++) {
      // only benchmark "high precision" copyCuda() if double is supported
      if ((Nprec < 3) && (kernel == 0)) continue;

      // do the initial tune
      benchmark(kernel, 1);
    
      // now rerun with more iterations to get accurate speed measurements
      quda::blas::flops = 0;
      quda::blas::bytes = 0;
      
      double secs = benchmark(kernel, niter);
      
      double gflops = (quda::blas::flops*1e-9)/(secs);
      double gbytes = quda::blas::bytes/(secs*1e9);
    
      printfQuda("%-31s: Gflop/s = %6.1f, GB/s = %6.1f\n", names[kernel], gflops, gbytes);
    }
    freeFields();
  }

  // clear the error state
  hipGetLastError();

  // lastly check for correctness
  for (int prec = 0; prec < Nprec; prec++) {
    printfQuda("\nTesting %s precision...\n\n", prec_str[prec]);
    initFields(prec);
    
    for (int kernel = 0; kernel < Nkernels; kernel++) {
      // only benchmark "high precision" copyCuda() if double is supported
      if ((Nprec < 3) && (kernel == 0)) continue;
      double error = test(kernel);
      printfQuda("%-35s error = %e, \n", names[kernel], error);
    }
    freeFields();
  }

  endQuda();

  finalizeComms();
}
