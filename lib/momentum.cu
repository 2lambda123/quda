#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <cub_helper.cuh>

namespace quda {

#ifdef GPU_GAUGE_TOOLS

  template <typename Mom>
  struct MomActionArg : public ReduceArg<double> {
    int threads; // number of active threads required
    Mom mom;
    int X[4]; // grid dimensions
    
    MomActionArg(const Mom &mom, const GaugeField &meta)
      : ReduceArg<double>(), mom(mom) {
      threads = meta.VolumeCB();
      for(int dir=0; dir<4; ++dir) X[dir] = meta.X()[dir];
    }
  };

  template<int blockSize, typename Float, typename Mom>
  __global__ void computeMomAction(MomActionArg<Mom> arg){
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int parity = threadIdx.y;
    double action = 0.0;
    
    if(x < arg.threads) {  
      // loop over direction
      for (int mu=0; mu<4; mu++) {
	Float v[10];
	arg.mom.load(v, x, mu, parity);

	double local_sum = 0.0;
	for (int j=0; j<6; j++) local_sum += v[j]*v[j];
	for (int j=6; j<9; j++) local_sum += 0.5*v[j]*v[j];
	local_sum -= 4.0;
	action += local_sum;
      }
    }
    
    // perform final inter-block reduction and write out result
    reduce2d<blockSize,2>(arg, action);
  }

  template<typename Float, typename Mom>
    class MomAction : Tunable {
      MomActionArg<Mom> arg;
      const QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.threads; }

      public:
      MomAction(MomActionArg<Mom> &arg)
        : arg(arg), location(QUDA_CUDA_FIELD_LOCATION) {}
      ~MomAction () { }

      bool advanceBlockDim(TuneParam &param) const {
      	bool rtn = Tunable::advanceBlockDim(param);
	param.block.y = 2;
	return rtn;
      }

      void initTuneParam(TuneParam &param) const {
	Tunable::initTuneParam(param);
	param.block.y = 2;
      }

      void apply(const hipStream_t &stream){
        if(location == QUDA_CUDA_FIELD_LOCATION){
          arg.result_h[0] = 0.0;
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	  LAUNCH_KERNEL(computeMomAction, tp, stream, arg, Float, Mom);
        } else {
          errorQuda("CPU not supported yet\n");
        }
      }

      TuneKey tuneKey() const {
        std::stringstream vol, aux;
        vol << arg.X[0] << "x" << arg.X[1] << "x" << arg.X[2] << "x" << arg.X[3];
	aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
        return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
      }

      std::string paramString(const TuneParam &param) const {
        std::stringstream ps;
        ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
        ps << "shared=" << param.shared_bytes;
        return ps.str();
      }

      void preTune(){}
      void postTune(){}
      long long flops() const { return 4*2*arg.threads*23; }
      long long bytes() const { return 4*2*arg.threads*arg.mom.Bytes(); } 
    }; 

  template<typename Float, typename Mom>
  void momAction(const Mom mom, const GaugeField& meta, double &action) {
    MomActionArg<Mom> arg(mom, meta);
    MomAction<Float,Mom> momAction(arg);

    momAction.apply(0);
    hipDeviceSynchronize();

    comm_allreduce((double*)arg.result_h);
    action = arg.result_h[0];
  }
  
  template<typename Float>
  double momAction(const GaugeField& mom) {
    double action = 0.0;
    
    if (mom.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
      if (mom.Reconstruct() == QUDA_RECONSTRUCT_10) {
	momAction<Float>(FloatNOrder<Float,10,2,10>(mom), mom, action);
      } else {
	errorQuda("Reconstruction type %d not supported", mom.Reconstruct());
      }
    } else {
      errorQuda("Gauge Field order %d not supported", mom.Order());
    }
    
    return action;
  }
#endif
  
  double computeMomAction(const GaugeField& mom) {
    double action = 0.0;
#ifdef GPU_GAUGE_TOOLS
    if (mom.Precision() == QUDA_DOUBLE_PRECISION) {
      action = momAction<double>(mom);
    } else if(mom.Precision() == QUDA_SINGLE_PRECISION) {
      action = momAction<float>(mom);
    } else {
      errorQuda("Precision %d not supported", mom.Precision());
    }
#else
    errorQuda("%s not build", __func__);
#endif
    return action;
  }
}
