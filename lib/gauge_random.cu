#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <atomic.cuh>
#include <cub_helper.cuh>
#include <index_helper.cuh>
#include <random_quda.h>

namespace quda {

#ifdef GPU_GAUGE_TOOLS

  template <typename Gauge>
  struct GaugeRandomArg {
    int threads; // number of active threads required
    int E[4]; // extended grid dimensions
    int X[4]; // true grid dimensions
    int border[4]; 
    Gauge dataDs;
    RNG rngstate;
    
    GaugeRandomArg(const Gauge &dataDs, const GaugeField &data, RNG &rngstate)
      : dataDs(dataDs), rngstate(rngstate)
    {
      int R = 0;
      for (int dir=0; dir<4; ++dir){
	border[dir] = data.R()[dir];
	E[dir] = data.X()[dir];
	X[dir] = data.X()[dir] - border[dir]*2;
	R += border[dir];
      }
      threads = X[0]*X[1]*X[2]*X[3]/2;
    }
  };


  template<typename Float>
  __device__ __host__  Matrix<complex<Float>,3> genRandomSU3(cuRNGState &localState){
       Matrix<complex<Float>, 3> ret;
       for (int i = 0; i < 3; i++)
	   for (int j = 0; j < 3; j++){
	       //ret(i,j) = 0.0;
	       ret(i,j) = complex<Float>( (Float)(Random<Float>(localState) - 0.5), (Float)(Random<Float>(localState) - 0.5) );
	   }
       return ret;
  }


  template<typename Float, typename Gauge>
  __global__ void computeGenRandom(GaugeRandomArg<Gauge> arg){
    typedef Matrix<complex<Float>,3> Link;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int parity = threadIdx.y + blockIdx.y*blockDim.y;

    typedef Matrix<complex<Float>,3> Link;


    if(idx < arg.threads) {
	int x[4];
	getCoords(x, idx, arg.X, parity);
	for (int dr=0; dr<4; ++dr) x[dr] += arg.border[dr]; // extended grid coordinates

	int dx[4] = {0, 0, 0, 0};
	for(int mu = 0; mu < 4; mu++){
	    cuRNGState localState = arg.rngstate.State()[idx + parity*arg.threads];

	    Link U = genRandomSU3<Float>(localState);

	    arg.rngstate.State()[ idx + parity*arg.threads ] = localState;
	    arg.dataDs(mu, linkIndexShift(x,dx,arg.X), parity) = U;
	}

    }
  }

  template<typename Float, typename Gauge>
    class GaugeRandom : TunableVectorY {
      GaugeRandomArg<Gauge> arg;
      GaugeField &gf;

      private:
      unsigned int minThreads() const { return arg.threads; }
      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.

      public:
      GaugeRandom(GaugeRandomArg<Gauge> &arg, GaugeField &gf)
        : TunableVectorY(2), arg(arg), gf(gf){}
      ~GaugeRandom () { }

      void apply(const hipStream_t &stream){
        if(gf.Location() == QUDA_CUDA_FIELD_LOCATION){
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

          computeGenRandom<Float><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
	  hipDeviceSynchronize();
        } else {
          errorQuda("Randomize GaugeFields on CPU not supported yet\n");
        }
      }

      TuneKey tuneKey() const {
        std::stringstream vol, aux;
        vol << arg.X[0] << "x" << arg.X[1] << "x" << arg.X[2] << "x" << arg.X[3];
	aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
        return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
      }

      long long flops() const { return 6ll*2*arg.threads*(3*198+3); }
      long long bytes() const { return 6ll*4*2*arg.threads*arg.dataDs.Bytes(); } 


      void preTune(){
	  //gf.backup();
	  arg.rngstate.backup();
      }
      void postTune(){
	  //gf.restore();
	  arg.rngstate.restore();
      }

    }; 

  template<typename Float, typename Gauge>
  void genRandom(const Gauge dataDs, GaugeField& data, RNG &rngstate) {
      GaugeRandomArg<Gauge> arg(dataDs, data, rngstate);
      GaugeRandom<Float,Gauge> gaugeRandom(arg, data);
      gaugeRandom.apply(0);

    }



  template<typename Float>
  void gaugeRandom(GaugeField &dataDs, RNG &rngstate) {

      if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type Gauge;
	  genRandom<Float>(Gauge(dataDs), dataDs, rngstate);
      }else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type Gauge;
	  genRandom<Float>(Gauge(dataDs), dataDs, rngstate);
      }else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type Gauge;
	  genRandom<Float>(Gauge(dataDs), dataDs, rngstate);
      }else{
	  errorQuda("Reconstruction type %d of origin gauge field not supported", dataDs.Reconstruct());
      }

  }

#endif

  void gaugeRandom(GaugeField &dataDs, RNG &rngstate) {

#ifdef GPU_GAUGE_TOOLS

      if(dataDs.Precision() == QUDA_HALF_PRECISION){
	  errorQuda("Half precision not supported\n");
      }

      if (!dataDs.isNative())
	  errorQuda("Order %d with %d reconstruct not supported", dataDs.Order(), dataDs.Reconstruct());

      if (dataDs.Precision() == QUDA_SINGLE_PRECISION){
	  gaugeRandom<float>(dataDs, rngstate);
      } else if(dataDs.Precision() == QUDA_DOUBLE_PRECISION) {
	  gaugeRandom<double>(dataDs, rngstate);
      } else {
	  errorQuda("Precision %d not supported", dataDs.Precision());
      }
      return;
#else
      errorQuda("Gauge tools are not build");
#endif
  }

}
