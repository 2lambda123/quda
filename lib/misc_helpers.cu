#include "hip/hip_runtime.h"

#include <misc_helpers.h>
#define gaugeSiteSize 18
#define BLOCKSIZE 64

/* This function converts format in CPU form 
   into forms in GPU so as to enable coalesce access
   The function only converts half(even or odd) of the links
   Therefore the entire link conversion need to call this 
   function twice
   
   Without loss of generarity, the parity is assume to be even.
   The actual data format in cpu is following
   [a0a1 .... a17] [a18a19 .....a35] ...[b0b1 ... b17] ...
     X links                             Y links         T,Z links
   
   The GPU format of data looks like the following
   [a0a1][a18a19]  ....[pad][a2a3][a20a21]..... [b0b1][b18b19]....
    X links                                      Y links      T,Z links
   
*/

template<typename FloatN, typename Float>
__global__ void
do_link_format_cpu_to_gpu(FloatN* dst, Float* src,
			  int reconstruct,
			  int bytes, int Vh, int pad, int Vsh)
{
  int tid = blockIdx.x * blockDim.x +  threadIdx.x;
  int thread0_tid = blockIdx.x * blockDim.x;
  __shared__ FloatN buf[gaugeSiteSize/2*BLOCKSIZE];
  
  int dir;
  int j;
  
  for(dir = 0; dir < 4; dir++){
#ifdef MULTI_GPU
      FloatN* src_start = (FloatN*)( src + dir*gaugeSiteSize*(Vh+2*Vsh) + thread0_tid*gaugeSiteSize);   
#else
      FloatN* src_start = (FloatN*)( src + dir*gaugeSiteSize*(Vh) + thread0_tid*gaugeSiteSize);   
#endif
      for(j=0; j < gaugeSiteSize/2; j++){
	  buf[j*blockDim.x + threadIdx.x] =  src_start[j*blockDim.x + threadIdx.x];
      }
      __syncthreads();
      
      FloatN* dst_start = (FloatN*)(dst+dir*gaugeSiteSize/2*(Vh+pad));
      for(j=0; j < gaugeSiteSize/2; j++){
	  dst_start[tid + j*(Vh+pad)] = buf[gaugeSiteSize/2*threadIdx.x + j];
      }
      __syncthreads();
      
  }//dir
}



// we require the cpu precisision and gpu precision are the same
void 
link_format_cpu_to_gpu(void* dst, void* src, 
		       int reconstruct, int bytes, int Vh, int pad, int Vsh, 
		       QudaPrecision prec)
{
  dim3 blockDim(BLOCKSIZE);
#ifdef MULTI_GPU  
  dim3 gridDim((Vh+2*Vsh)/blockDim.x);
#else
  dim3 gridDim(Vh/blockDim.x);
#endif
  //(Vh+2*Vsh) must be multipl of BLOCKSIZE or the kernel does not work
    //because the intermediae GPU data has stride=Vh+2*Vsh and the extra two
    //Vsh is occupied by the back and forward neighbor
    if ((Vh+2*Vsh) % blockDim.x != 0){
	printf("ERROR: Vh(%d) is not multiple of blocksize(%d), exitting\n", Vh, blockDim.x);
	exit(1);
    }
    
    
    switch (prec){
    case QUDA_DOUBLE_PRECISION:
      do_link_format_cpu_to_gpu<<<gridDim, blockDim>>>((double2*)dst, (double*)src, reconstruct, bytes, Vh, pad, Vsh);
      break;
      
    case QUDA_SINGLE_PRECISION:
      if(reconstruct == QUDA_RECONSTRUCT_NO){
	do_link_format_cpu_to_gpu<<<gridDim, blockDim>>>((float2*)dst, (float*)src, reconstruct, bytes, Vh, pad, Vsh);   
      }else if (reconstruct == QUDA_RECONSTRUCT_12){
	//not working yet
	//do_link_format_cpu_to_gpu<<<gridDim, blockDim>>>((float4*)dst, (float*)src, reconstruct, bytes, Vh, pad, Vsh);   
	
      }
      break;
      
    default:
      printf("ERROR: half precision not support in %s\n", __FUNCTION__);
      exit(1);
    }
    
    /*
    if (cuda_prec == QUDA_DOUBLE_PRECISION){
      do_link_format_cpu_to_gpu<<<gridDim, blockDim>>>((double2*)dst, (double*)src, reconstruct, bytes, Vh, pad, Vsh);
    }else if( cuda_prec == QUDA_SINGLE_PRECISION){
      do_link_format_cpu_to_gpu<<<gridDim, blockDim>>>((float2*)dst, (float*)src, reconstruct, bytes, Vh, pad, Vsh);      
    }else{
      printf("ERROR: half precision is not supported in %s\n", __FUNCTION__);
      exit(1);
    }
    */

    return;
    
}


/*
 * src format: the normal link format in GPU that has stride size @stride
 *	       the src is stored with 9 double2
 * dst format: an array of links where x,y,z,t links with the same node id is stored next to each other
 *             This format is used in destination in fatlink computation in cpu
 *    Without loss of generarity, the parity is assume to be even.
 * The actual data format in cpu is following
 *    [a0a1][a18a19]  ....[pad][a2a3][a20a21]..... [b0b1][b18b19]....
 *    X links                                      Y links      T,Z links
 * The temporary data store in GPU shared memory and the CPU format of data are the following
 *    [a0a1 .... a17] [b0b1 .....b17] [c0c1 .....c17] [d0d1 .....d17] [a18a19....a35] ....
 *    |<------------------------site 0 ---------------------------->|<----- site 2 ----->
 *
 *
 * In loading phase the indices for all threads in the first block is the following (assume block size is 64)
 * (half warp works on one direction)
 * threadIdx.x	pos_idx		mydir
 * 0		0		0
 * 1		1		0
 * 2		2		0
 * 3		3		0			
 * 4		4		0		
 * 5		5		0
 * 6		6		0
 * 7		7		0
 * 8		8		0
 * 9		9		0
 * 10		10		0
 * 11		11		0
 * 12		12		0
 * 13		13		0
 * 14		14		0
 * 15		15		0
 * 16		0		1
 * 17		1		1
 * 18	       	2		1
 * 19		3		1
 * 20		4		1
 * 21		5		1
 * 22		6		1
 * 23		7		1
 * 24		8		1
 * 25		9		1
 * 26		10		1
 * 27		11		1
 * 28		12		1
 * 29		13		1
 * 30		14		1
 * 31		15		1
 * 32		0		2
 * 33		1		2
 * 34		2		2
 * 35		3		2
 * 36		4		2
 * 37		5		2
 * 38		6		2
 * 39		7		2
 * 40		8		2
 * 41		9		2
 * 42		10		2
 * 43		11		2
 * 44		12		2
 * 45		13		2
 * 46		14		2
 * 47		15		2
 * 48		0		3
 * 49		1		3
 * 50		2		3
 * 51		3		3
 * 52		4		3
 * 53		5		3
 * 54		6		3
 * 55		7		3
 * 56		8		3
 * 57		9		3
 * 58		10		3
 * 59		11		3
 * 60		12		3
 * 61		13		3
 * 62		14		3
 * 63		15		3
 *
 */

template<typename FloatN>
__global__ void
do_link_format_gpu_to_cpu(FloatN* dst, FloatN* src,
			  int bytes, int Vh, int stride)
{
  __shared__ FloatN buf[gaugeSiteSize/2*BLOCKSIZE];
  
  int j;
  
  int block_idx = blockIdx.x*blockDim.x/4;
  int local_idx = 16*(threadIdx.x/64) + threadIdx.x%16;
  int pos_idx = blockIdx.x * blockDim.x/4 + 16*(threadIdx.x/64) + threadIdx.x%16;
  int mydir = (threadIdx.x >> 4)% 4;
  for(j=0; j < 9; j++){
    buf[local_idx*4*9+mydir*9+j] = src[pos_idx + mydir*9*stride + j*stride];
  }
  __syncthreads();
  
  for(j=0; j < 9; j++){
    dst[block_idx*9*4 + j*blockDim.x + threadIdx.x ] = buf[j*blockDim.x + threadIdx.x];    
  }  
  
}

void 
link_format_gpu_to_cpu(void* dst, void* src, 
		       int bytes, int Vh, int stride, QudaPrecision prec)
{
  
  dim3 blockDim(BLOCKSIZE);
  dim3 gridDim(4*Vh/blockDim.x); //every 4 threads process one site's x,y,z,t links
  //4*Vh must be multipl of BLOCKSIZE or the kernel does not work
  if ((4*Vh) % blockDim.x != 0){
    printf("ERROR: Vh(%d) is not multiple of blocksize(%d), exitting\n", Vh, blockDim.x);
    exit(1);
  }
  if(prec == QUDA_DOUBLE_PRECISION){
    do_link_format_gpu_to_cpu<<<gridDim, blockDim>>>((double2*)dst, (double2*)src, bytes, Vh, stride);
  }else if(prec == QUDA_SINGLE_PRECISION){
    do_link_format_gpu_to_cpu<<<gridDim, blockDim>>>((float2*)dst, (float2*)src,  bytes, Vh, stride);
  }else{
    printf("ERROR: half precision is not supported in %s\n",__FUNCTION__);
    exit(1);
  }
  
}



#define READ_ST_SPINOR(spinor, idx, mystride)           \
  Float2 I0 = spinor[idx + 0*mystride];                 \
  Float2 I1 = spinor[idx + 1*mystride];                 \
  Float2 I2 = spinor[idx + 2*mystride];

#define WRITE_ST_SPINOR(spinor, idx, mystride)  \
  spinor[idx + 0*mystride] = I0;                        \
  spinor[idx + 1*mystride] = I1;                        \
  spinor[idx + 2*mystride] = I2;


template<int dir, int whichway, typename Float2>
__global__ void
staggeredCollectGhostSpinorKernel(Float2* in, const int oddBit,
                                  Float2* nbr_spinor_gpu)
{
#if 1
  int sid = blockIdx.x*blockDim.x + threadIdx.x;
  int z1 = FAST_INT_DIVIDE(sid, X1h);
  int x1h = sid - z1*X1h;
  int z2 = FAST_INT_DIVIDE(z1, X2);
  int x2 = z1 - z2*X2;
  int x4 = FAST_INT_DIVIDE(z2, X3);
  int x3 = z2 - x4*X3;
  int x1odd = (x2 + x3 + x4 + oddBit) & 1;
  int x1 = 2*x1h + x1odd;
  int X = 2*sid + x1odd;

  READ_ST_SPINOR(in, sid, sp_stride);
  int ghost_face_idx;

  if ( dir == 0 && whichway == QUDA_BACKWARDS){
    if (x1 < 3){
      ghost_face_idx = (x1*X4*X3*X2 + x4*(X3*X2)+x3*X2 +x2)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X4*X3*X2/2);
    }
  }

  if ( dir == 0 && whichway == QUDA_FORWARDS){
    if (x1 >= X1 - 3){
      ghost_face_idx = ((x1-X1+3)*X4*X3*X2 + x4*(X3*X2)+x3*X2 +x2)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X4*X3*X2/2);
    }
  }

  if ( dir == 1 && whichway == QUDA_BACKWARDS){
    if (x2 < 3){
      ghost_face_idx = (x2*X4*X3*X1 + x4*X3*X1+x3*X1+x1)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X4*X3*X1/2);
    }
  }

  if ( dir == 1 && whichway == QUDA_FORWARDS){
    if (x2 >= X2 - 3){
      ghost_face_idx = ((x2-X2+3)*X4*X3*X1+ x4*X3*X1+x3*X1+x1)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X4*X3*X1/2);
    }
  }

  if ( dir == 2 && whichway == QUDA_BACKWARDS){
    if (x3 < 3){
      ghost_face_idx = (x3*X4*X2*X1 + x4*X2*X1+x2*X1+x1)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X4*X2*X1/2);
    }
  }

  if ( dir == 2 && whichway == QUDA_FORWARDS){
    if (x3 >= X3 - 3){
      ghost_face_idx = ((x3-X3+3)*X4*X2*X1 + x4*X2*X1 + x2*X1 + x1)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X4*X2*X1/2);
    }
  }

  if ( dir == 3 && whichway == QUDA_BACKWARDS){
    if (x4 < 3){
      ghost_face_idx = (x4*X3*X2*X1 + x3*X2*X1+x2*X1+x1)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X3*X2*X1/2);
    }
  }

  if ( dir == 3 && whichway == QUDA_FORWARDS){
    if (x4 >= X4 - 3){
      ghost_face_idx = ((x4-X4+3)*X3*X2*X1 + x3*X2*X1+x2*X1+x1)>>1;
      WRITE_ST_SPINOR(nbr_spinor_gpu, ghost_face_idx, 3*X3*X2*X1/2);
    }
  }
#endif

}


//@dir can be 0, 1, 2, 3 (X,Y,Z,T directions)
//@whichway can be QUDA_FORWARDS, QUDA_BACKWORDS
void
collectGhostSpinor(void *in, const void *inNorm,
                   void* ghost_spinor_gpu,		   
		   int dir, int whichway,
                   const int parity, cudaColorSpinorField* inSpinor)
{
  
  dim3 gridDim(inSpinor->Volume()/BLOCKSIZE, 1, 1);
  dim3 blockDim(BLOCKSIZE, 1, 1);
    
  if (inSpinor->Precision() == QUDA_DOUBLE_PRECISION){
    switch(dir){
    case 0:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<0, QUDA_BACKWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu);
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<0, QUDA_FORWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu);
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;

    case 1:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<1, QUDA_BACKWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu); CUERR;
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<1, QUDA_FORWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu); CUERR;
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;
      
    case 2:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<2, QUDA_BACKWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu); CUERR;
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<2, QUDA_FORWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu); CUERR;
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;
      
    case 3:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<3, QUDA_BACKWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu); CUERR;
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<3, QUDA_FORWARDS><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu); CUERR;
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;

      
    }
    //staggeredCollectGhostSpinorKernel<dir, whichway><<<gridDim, blockDim>>>((double2*)in, parity, (double2*)ghost_spinor_gpu);
  }else if(inSpinor->Precision() == QUDA_SINGLE_PRECISION){
    //staggeredCollectGhostSpinorKernel<dir, whichway><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu);
    switch(dir){
    case 0:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<0, QUDA_BACKWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu);
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<0, QUDA_FORWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu);
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;

    case 1:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<1, QUDA_BACKWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu); CUERR;
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<1, QUDA_FORWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu); CUERR;
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;

    case 2:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<2, QUDA_BACKWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu); CUERR;
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<2, QUDA_FORWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu); CUERR;
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;

    case 3:
      switch(whichway){
      case QUDA_BACKWARDS:
	staggeredCollectGhostSpinorKernel<3, QUDA_BACKWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu); CUERR;
	break;
      case QUDA_FORWARDS:
	staggeredCollectGhostSpinorKernel<3, QUDA_FORWARDS><<<gridDim, blockDim>>>((float2*)in, parity, (float2*)ghost_spinor_gpu); CUERR;
	break;
      default:
	errorQuda("Invalid whichway");
	break;
      }
      break;
      
    }

  }else{
    printf("ERROR: half precision not implemented yet for %s\n", __FUNCTION__);
    exit(1);
  }
  hipDeviceSynchronize();
  CUERR;
}





#undef gaugeSiteSize 
#undef BLOCKSIZE 
