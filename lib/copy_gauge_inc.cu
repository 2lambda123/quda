#include <gauge_field_order.h>
#include <copy_gauge_helper.cuh>

namespace quda {
  
  template <typename FloatOut, typename FloatIn, int length, typename InOrder>
  void copyGauge(const InOrder &inOrder, GaugeField &out, QudaFieldLocation location, 
		 FloatOut *Out, FloatOut **outGhost, int type) {
    int faceVolumeCB[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) faceVolumeCB[i] = out.SurfaceCB(i) * out.Nface(); 
    if (out.isNative()) {
      if (out.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(FloatOut)==typeid(short) && out.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  copyGauge<short,FloatIn,length>
	    (FloatNOrder<short,length,2,19>(out, (short*)Out, (short**)outGhost), inOrder,
	     out.Volume(), faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
	} else {
	  typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_NO>::type G;
	  copyGauge<FloatOut,FloatIn,length>
	    (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	     out.Ndim(), out.Geometry(), out, location, type);
	}
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_12>::type G;
	copyGauge<FloatOut,FloatIn,length>
	  (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_8>::type G;
	copyGauge<FloatOut,FloatIn,length> 
	  (G(out,Out,outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
#if defined(GPU_STAGGERED_DIRAC) && __COMPUTE_CAPABILITY__ >= 200
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_13>::type G;
        copyGauge<FloatOut,FloatIn,length>
	  (G(out, Out, outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(),  out.Geometry(), out, location, type);
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_9>::type G;
        copyGauge<FloatOut,FloatIn,length>
	  (G(out, Out, outGhost), inOrder, out.Volume(), faceVolumeCB,
	   out.Ndim(), out.Geometry(), out, location, type);
#endif
      } else {
	errorQuda("Reconstruction %d and order %d not supported", out.Reconstruct(), out.Order());
      }
    } else if (out.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(QDPOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(), 
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(QDPJITOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(CPSOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(MILCOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(BQCDOrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      copyGauge<FloatOut,FloatIn,length>
	(TIFROrder<FloatOut,length>(out, Out, outGhost), inOrder, out.Volume(),
	 faceVolumeCB, out.Ndim(), out.Geometry(), out, location, type);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", out.Order());
    }

  }

  template <typename FloatOut, typename FloatIn, int length>
    void copyGauge(GaugeField &out, const GaugeField &in, QudaFieldLocation location, 
		   FloatOut *Out, FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

    // reconstruction only supported on FloatN fields currently
    if (in.isNative()) {      
      if (in.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	if (typeid(FloatIn)==typeid(short) && in.LinkType() == QUDA_ASQTAD_FAT_LINKS) {
	  copyGauge<FloatOut,short,length> (FloatNOrder<short,length,2,19>
					    (in,(short*)In,(short**)inGhost),
					    out, location, Out, outGhost, type);
	} else {
	  typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_NO>::type G;
	  copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
	}
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_12) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_12>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_8) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_8>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
#if defined(GPU_STAGGERED_DIRAC) && __COMPUTE_CAPABILITY__ >= 200
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_13) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_13>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_9) {
	typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_9>::type G;
	copyGauge<FloatOut,FloatIn,length> (G(in,In,inGhost), out, location, Out, outGhost, type);
#endif
      } else {
	errorQuda("Reconstruction %d and order %d not supported", in.Reconstruct(), in.Order());
      }
    } else if (in.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(QDPOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(QDPJITOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(CPSOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(MILCOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(BQCDOrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      copyGauge<FloatOut,FloatIn,length>(TIFROrder<FloatIn,length>(in, In, inGhost), 
					 out, location, Out, outGhost, type);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", in.Order());
    }

  }

  void checkMomOrder(const GaugeField &u);

  template <typename FloatOut, typename FloatIn>
  void copyGauge(GaugeField &out, const GaugeField &in, QudaFieldLocation location, FloatOut *Out, 
		 FloatIn *In, FloatOut **outGhost, FloatIn **inGhost, int type) {

    if (in.Ncolor() != 3 && out.Ncolor() != 3) {
      errorQuda("Unsupported number of colors; out.Nc=%d, in.Nc=%d", out.Ncolor(), in.Ncolor());
    }
    
#if __COMPUTE_CAPABILITY__ < 200
    if (in.Reconstruct() == QUDA_RECONSTRUCT_13 || in.Reconstruct() == QUDA_RECONSTRUCT_9 ||
	out.Reconstruct() == QUDA_RECONSTRUCT_13 || out.Reconstruct() == QUDA_RECONSTRUCT_9)
      errorQuda("Reconstruct 9/13 not supported on pre-Fermi architecture");
#endif

    if (in.LinkType() != QUDA_ASQTAD_MOM_LINKS && out.LinkType() != QUDA_ASQTAD_MOM_LINKS) {
      // we are doing gauge field packing
      copyGauge<FloatOut,FloatIn,18>(out, in, location, Out, In, outGhost, inGhost, type);
    } else {
      if (location != QUDA_CPU_FIELD_LOCATION) errorQuda("Location %d not supported", location);
      if (out.Geometry() != QUDA_VECTOR_GEOMETRY) errorQuda("Unsupported geometry %d", out.Geometry());

      checkMomOrder(in);
      checkMomOrder(out);
    
      int faceVolumeCB[QUDA_MAX_DIM];
      for (int d=0; d<in.Ndim(); d++) faceVolumeCB[d] = in.SurfaceCB(d) * in.Nface();

      // momentum only currently supported on MILC (10), TIFR (18) and Float2 (10) fields currently
	if (out.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    CopyGaugeArg<FloatNOrder<FloatOut,10,2,10>, FloatNOrder<FloatIn,10,2,10> >
	      arg(FloatNOrder<FloatOut,10,2,10>(out, Out), 
		  FloatNOrder<FloatIn,10,2,10>(in, In), in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {
#ifdef BUILD_MILC_INTERFACE
	    CopyGaugeArg<FloatNOrder<FloatOut,10,2,10>, MILCOrder<FloatIn,10> >
	      arg(FloatNOrder<FloatOut,10,2,10>(out, Out), MILCOrder<FloatIn,10>(in, In), 
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
#else
	    errorQuda("MILC interface has not been built\n");
#endif
	    
	  } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {
#ifdef BUILD_TIFR_INTERFACE
	    CopyGaugeArg<FloatNOrder<FloatOut,18,2,11>, TIFROrder<FloatIn,18> >
	      arg(FloatNOrder<FloatOut,18,2,11>(out, Out), TIFROrder<FloatIn,18>(in, In), 
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,18>(arg);
#else
	    errorQuda("TIFR interface has not been built\n");
#endif
	    
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
	} else if (out.Order() == QUDA_MILC_GAUGE_ORDER) {
#ifdef BUILD_MILC_INTERFACE
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    CopyGaugeArg<MILCOrder<FloatOut,10>, FloatNOrder<FloatIn,10,2,10> >
	      arg(MILCOrder<FloatOut,10>(out, Out), FloatNOrder<FloatIn,10,2,10>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {
	    CopyGaugeArg<MILCOrder<FloatOut,10>, MILCOrder<FloatIn,10> >
	      arg(MILCOrder<FloatOut,10>(out, Out), MILCOrder<FloatIn,10>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
#else
	  errorQuda("MILC interface has not been built\n");
#endif
	  
	} else if (out.Order() == QUDA_TIFR_GAUGE_ORDER) {
#ifdef BUILD_TIFR_INTERFACE
	  if (in.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
	    // FIX ME - 11 is a misnomer to avoid confusion in template instantiation
	    CopyGaugeArg<TIFROrder<FloatOut,18>, FloatNOrder<FloatIn,18,2,11> >
	      arg(TIFROrder<FloatOut,18>(out, Out), FloatNOrder<FloatIn,18,2,11>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,18>(arg);
	  } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {
	    CopyGaugeArg<TIFROrder<FloatOut,18>, TIFROrder<FloatIn,18> >
	      arg(TIFROrder<FloatOut,18>(out, Out), TIFROrder<FloatIn,18>(in, In),
		  in.Volume(), faceVolumeCB, in.Ndim(), in.Geometry());
	    copyGauge<FloatOut,FloatIn,10>(arg);
	  } else {
	    errorQuda("Gauge field orders %d not supported", in.Order());
	  }
#else
	  errorQuda("TIFR interface has not been built\n");
#endif
	} else {
	  errorQuda("Gauge field orders %d not supported", out.Order());
	}
    }
  }


} // namespace quda
