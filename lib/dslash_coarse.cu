#include "hip/hip_runtime.h"
#include <multigrid.h>
#include <transfer.h>
#include <gauge_field_order.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>

namespace quda {

#ifdef GPU_MULTIGRID

  template <typename Float, typename F, typename G>
  struct CoarseDslashArg {
    F out;
    const F inA;
    const F inB;
    const G Y;
    const G X;
    Float kappa;
    int parity; // only use this for single parity fields
    int nFace;  // hard code to 1 for now
    int dim[4];   // full lattice dimensions
    int commDim[4]; // whether a given dimension is partitioned or not

    CoarseDslashArg(F &out, const F &inA, const F &inB, const G &Y, const G &X, Float kappa, int parity)
      : out(out), inA(inA), inB(inB), Y(Y), X(X), kappa(kappa), parity(parity), nFace(1) { 
      for (int i=0; i<4; i++) {
	dim[i] = inA.X(i);
	commDim[i] = comm_dim_partitioned(i);
      }
      dim[0] = (inA.Nparity() == 1) ? 2 * dim[0] : dim[0];
    }
  };

  /**
     Applies the coarse dslash on a given parity and checkerboard site index

     @param out The result -2 * kappa * Dslash in
     @param Y The coarse gauge field
     @param kappa Kappa value
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  __device__ __host__ inline void dslash(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity) {
    const int their_spinor_parity = (arg.inA.Nparity() == 2) ? (parity+1)&1 : 0;

    int coord[nDim];
    getCoords(coord, x_cb, arg.dim, parity);

    for(int d = 0; d < nDim; d++) { //Ndim
      //Forward link - compute fwd offset for spinor fetch
      {
	/*if (coord[d] + arg.nFace >= arg.in.X(d) && arg.commDim[d]) {
	// load from ghost
	} else {
	linkIndexP1(coord, arg.X(), d);
	}*/
	int fwd_idx = linkIndexP1(coord, arg.dim, d);

	complex<Float> in[Ns*Nc];
	for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++)
	  in[s*Nc+c] = arg.inA(their_spinor_parity, fwd_idx, s, c);

	for(int s_row = 0; s_row < Ns; s_row++) { //Spin row
	  for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
	      Float sign = (s_row == s_col) ? 1.0 : -1.0;
	      for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		out[s_row*Nc+c_row] += sign*arg.Y(d, parity, x_cb, s_row, s_col, c_row, c_col)
		  * in[s_col*Nc+c_col]; //arg.inA(their_spinor_parity, fwd_idx, s_col, c_col);
	      } //Color column
	    } //Spin column
	  } //Color row
	} //Spin row
      }

      //Backward link - compute back offset for spinor and gauge fetch
      {
	/*if (coord[d] - arg.nFace < 0) {
	// load from ghost
	} else {
	linkIndexM1(coord, arg.X(), d);
	}*/
	int back_idx = linkIndexM1(coord, arg.dim, d);

	complex<Float> in[Ns*Nc];
	for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++)
	  in[s*Nc+c] = arg.inA(their_spinor_parity, back_idx, s, c);

	for(int s_row = 0; s_row < Ns; s_row++) { //Spin row
	  for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
	      for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		out[s_row*Nc+c_row] += conj(arg.Y(d,(parity+1)&1, back_idx, s_col, s_row, c_col, c_row))
		  * in[s_col*Nc+c_col]; //arg.inA(their_spinor_parity, back_idx, s_col, c_col);
	      } //Color column
	    } //Spin column
	  } //Color row
	} //Spin row
      } //nDim
    }

    // apply kappa
    for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++) out[s*Nc+c] *= -(Float)2.0*arg.kappa;
  }

  /**
     Applies the coarse clover matrix on a given parity and
     checkerboard site index

     @param out The result out += X * in
     @param X The coarse clover field
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int Ns, int Nc>
  __device__ __host__ inline void clover(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity) {
    const int spinor_parity = (arg.inB.Nparity() == 2) ? parity : 0;

    complex<Float> in[Ns*Nc];
    for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++)
      in[s*Nc+c] = arg.inB(spinor_parity, x_cb, s, c);

    // apply clover term
    for(int s = 0; s < Ns; s++) { //Spin out
      for(int c = 0; c < Nc; c++) { //Color out
	//This term is now incorporated into the matrix X.
	//out(parity,x_cb,s,c) += in(parity,x_cb,s,c);
	for(int s_col = 0; s_col < Ns; s_col++) { //Spin in
	  for(int c_col = 0; c_col < Nc; c_col++) { //Color in
	    //Factor of 2*kappa now incorporated in X
	    //out(parity,x_cb,s,c) -= 2*kappa*X(0, parity, x_cb, s, s_col, c, c_col)*in(parity,x_cb,s_col,c_col);
	    out[s*Nc+c] += arg.X(0, parity, x_cb, s, s_col, c, c_col)*in[s_col*Nc+c_col];
	  } //Color in
	} //Spin in
      } //Color out
    } //Spin out
  }

  // CPU kernel for applying the coarse Dslash to a vector:
  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  void coarseDslash(CoarseDslashArg<Float,F,G> arg) {

    //#pragma omp parallel for 
    for (int parity= 0; parity < arg.inA.Nparity(); parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.inA.Nparity() == 2) ? parity : arg.parity;

      for(int x_cb = 0; x_cb < arg.inA.VolumeCB(); x_cb++) { //Volume
	complex <Float> out[Ns*Nc];// = { };
	for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++) out[s*Nc+c] = 0.0;
	dslash<Float,F,G,nDim,Ns,Nc>(out, arg, x_cb, parity);
	clover<Float,F,G,Ns,Nc>(out, arg, x_cb, parity);

	const int my_spinor_parity = (arg.inA.Nparity() == 2) ? parity : 0;
	for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++)
	  arg.out(my_spinor_parity, x_cb, s, c) = out[s*Nc+c];
      }//VolumeCB
    } // parity
    
  }

  // FIXME need to instrument parity for gauge fields to be set from
  // parity parity while also set to ignore parity for single parity
  // quark fields

  // GPU Kernel for applying the coarse Dslash to a vector:
  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  __global__ void coarseDslashKernel(CoarseDslashArg<Float,F,G> arg) {

    int x_cb = blockDim.x*blockIdx.x + threadIdx.x;
    if (x_cb >= arg.inA.VolumeCB()) return;

    // for full fields then set parity from y thread index else use arg setting
    int parity = (blockDim.y == 2) ? threadIdx.y : arg.parity;

    complex<Float> out[Ns*Nc];// = { };
    for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++) out[s*Nc+c] = 0.0;

    dslash<Float,F,G,nDim,Ns,Nc>(out, arg, x_cb, parity);
    clover<Float,F,G,Ns,Nc>(out, arg, x_cb, parity);

    const int my_spinor_parity = (blockDim.y == 2) ? parity : 0;
    for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++) 
      arg.out(my_spinor_parity, x_cb, s, c) = out[s*Nc+c];
  }

  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  class CoarseDslash : public Tunable {

  protected:
    CoarseDslashArg<Float,F,G> &arg;
    const ColorSpinorField &meta;

    long long flops() const { return 0; }
    long long bytes() const
    {
      return arg.out.Bytes() + 8*arg.inA.Bytes() + arg.inB.Bytes() + arg.inA.Nparity()*(8*arg.Y.Bytes() + arg.X.Bytes());
    }
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.inA.VolumeCB(); }

    bool advanceTuneParam(TuneParam &param) const 
    {
      bool rtn = Tunable::advanceTuneParam(param);
      param.block.y = arg.inA.Nparity();
      return rtn;
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.block.y = arg.inA.Nparity();
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.block.y = arg.inA.Nparity();
    }


  public:
    CoarseDslash(CoarseDslashArg<Float,F,G> &arg, const ColorSpinorField &meta)
      : arg(arg), meta(meta) { }
    virtual ~CoarseDslash() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	coarseDslash<Float,F,G,nDim,Ns,Nc>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	coarseDslashKernel<Float,F,G,nDim,Ns,Nc> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString());
    }

  };


  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor, int coarseSpin>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,  const GaugeField &Y, const GaugeField &X,
		   double kappa, int parity) {
    typedef typename colorspinor::FieldOrderCB<Float,coarseSpin,coarseColor,1,csOrder> F;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> G;
    F outAccessor(const_cast<ColorSpinorField&>(out));
    F inAccessorA(const_cast<ColorSpinorField&>(inA));
    F inAccessorB(const_cast<ColorSpinorField&>(inB));
    G yAccessor(const_cast<GaugeField&>(Y));
    G xAccessor(const_cast<GaugeField&>(X));
    CoarseDslashArg<Float,F,G> arg(outAccessor, inAccessorA, inAccessorB, yAccessor, xAccessor, (Float)kappa, parity);
    CoarseDslash<Float,F,G,4,coarseSpin,coarseColor> dslash(arg, inA);
    dslash.apply(0);
  }

  // template on the number of coarse colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity) {
    if (inA.Nspin() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",inA.Nspin());

    if (inA.Ncolor() == 2) { 
      ApplyCoarse<Float,csOrder,gOrder,2,2>(out, inA, inB, Y, X, kappa, parity);
    } else if (inA.Ncolor() == 24) { 
      ApplyCoarse<Float,csOrder,gOrder,24,2>(out, inA, inB, Y, X, kappa, parity);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder fOrder>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity) {

    if (Y.FieldOrder() != X.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (Y.FieldOrder() == QUDA_FLOAT2_GAUGE_ORDER) {
      ApplyCoarse<Float,fOrder,QUDA_FLOAT2_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, parity);
    } else if (Y.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      ApplyCoarse<Float,fOrder,QUDA_QDP_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, parity);
    } else {
      errorQuda("Unsupported field order %d\n", Y.FieldOrder());
    }
  }

  template <typename Float>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity) {
    if (inA.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      ApplyCoarse<Float,QUDA_FLOAT2_FIELD_ORDER>(out, inA, inB, Y, X, kappa, parity);
    } else if (inA.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      ApplyCoarse<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(out, inA, inB, Y, X, kappa, parity);
    } else {
      errorQuda("Unsupported field order %d\n", inA.FieldOrder());
    }
  }

#endif // GPU_MULTIGRID

  //Apply the coarse Dirac matrix to a coarse grid vector
  //out(x) = M*in = X*in - 2*kappa*\sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  //Note factor of 2*kappa compensates for the factor of 1/2 already
  //absorbed into the Y matrices.
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, int parity) {
#ifdef GPU_MULTIGRID
    if (Y.Precision() != inA.Precision() || Y.Precision() != inB.Precision() ||
	X.Precision() != Y.Precision() || Y.Precision() != out.Precision())
      errorQuda("Unsupported precision mix");

    if (inA.V() == out.V()) errorQuda("Aliasing pointers");
    if (out.Precision() != inA.Precision() ||
	Y.Precision() != inA.Precision() ||
	X.Precision() != inA.Precision()) 
      errorQuda("Precision mismatch out=%d inA=%d inB=%dY=%d X=%d", 
		out.Precision(), inA.Precision(), inB.Precision(), Y.Precision(), X.Precision());

    // check all locations match
    Location(out, inA, inB, Y, X);

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyCoarse<double>(out, inA, inB, Y, X, kappa, parity);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyCoarse<float>(out, inA, inB, Y, X, kappa, parity);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
#else
    errorQuda("Multigrid has not been built");
#endif
  }//ApplyCoarse

} // namespace quda
