#include "hip/hip_runtime.h"
#include <multigrid.h>
#include <transfer.h>
#include <gauge_field_order.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>

namespace quda {

#ifdef GPU_MULTIGRID

  template <typename Float, typename F, typename G>
  struct CoarseDslashArg {
    F out;
    const F inA;
    const F inB;
    const G Y;
    const G X;
    Float kappa;
    int parity; // only use this for single parity fields
    int nParity; // number of parities we're working on
    int volumeCB;
    int dim[5];   // full lattice dimensions
    int commDim[4]; // whether a given dimension is partitioned or not
    int nFace;  // hard code to 1 for now

    bool staggered_coarse_dslash;//staggered coarse dslash has more sparse structure.

    CoarseDslashArg(F &out, const F &inA, const F &inB, const G &Y, const G &X,
		    Float kappa, int parity, const ColorSpinorField &meta, bool is_staggered)
      : out(out), inA(inA), inB(inB), Y(Y), X(X), kappa(kappa), parity(parity),
	nParity(meta.SiteSubset()), volumeCB(meta.VolumeCB()), nFace(1), staggered_coarse_dslash(is_staggered) {
      for (int i=0; i<4; i++) {
	dim[i] = meta.X(i);
	commDim[i] = comm_dim_partitioned(i);
      }
      dim[0] = (nParity == 1) ? 2 * dim[0] : dim[0];
      dim[4] = 1; // ghost index expects a fifth dimension
    }
  };

  /**
     Applies the coarse dslash on a given parity and checkerboard site index

     @param out The result -2 * kappa * Dslash in
     @param Y The coarse gauge field
     @param kappa Kappa value
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc>
  __device__ __host__ inline void dslash(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity, int s_row, int color_block) {
    const int their_spinor_parity = (arg.nParity == 2) ? (parity+1)&1 : 0;

    int coord[5];
    getCoords(coord, x_cb, arg.dim, parity);
    coord[4] = 0;

    for(int d = 0; d < nDim; d++) { //Ndim
      //Forward link - compute fwd offset for spinor fetch
      {
	const int fwd_idx = linkIndexP1(coord, arg.dim, d);
	if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	  int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);

	  for(int color_local = 0; color_local < Mc; color_local++) { //Color row
	    int c_row = color_block + color_local; // global color index
	    int row = s_row*Nc + c_row;
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
	      Float sign = (s_row == s_col) ? 1.0 : -1.0;
	      for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		int col = s_col*Nc + c_col;
		out[color_local] += sign*(arg.Y(d, parity, x_cb, row, col)) * arg.inA.Ghost(d, 1, their_spinor_parity, ghost_idx, s_col, c_col);
	      }
	    }
	  }
	} else {

	  for(int color_local = 0; color_local < Mc; color_local++) { //Color row
	    int c_row = color_block + color_local; // global color index
	    int row = s_row*Nc + c_row;
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
	      Float sign = (s_row == s_col) ? 1.0 : -1.0;
	      for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		int col = s_col*Nc + c_col;
		out[color_local] += sign*(arg.Y(d, parity, x_cb, row, col)) * arg.inA(their_spinor_parity, fwd_idx, s_col, c_col);
	      }
	    }
	  }
	}

      }

      //Backward link - compute back offset for spinor and gauge fetch
      {
	const int back_idx = linkIndexM1(coord, arg.dim, d);
	const int gauge_idx = back_idx;
	if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	  const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
	  for (int color_local=0; color_local<Mc; color_local++) {
	    int c_row = color_block + color_local;
	    int row = s_row*Nc + c_row;
	    for (int s_col=0; s_col<Ns; s_col++)
	      for (int c_col=0; c_col<Nc; c_col++) {
		int col = s_col*Nc + c_col;
		out[color_local] += conj(arg.Y.Ghost(d, (parity+1)&1, ghost_idx, col, row)) * arg.inA.Ghost(d, 0, their_spinor_parity, ghost_idx, s_col, c_col);
	      }
	  }
	} else {
	  for(int color_local = 0; color_local < Mc; color_local++) {
	    int c_row = color_block + color_local;
	    int row = s_row*Nc + c_row;
	    for(int s_col = 0; s_col < Ns; s_col++)
	      for(int c_col = 0; c_col < Nc; c_col++) {
		int col = s_col*Nc + c_col;
		out[color_local] += conj(arg.Y(d, (parity+1)&1, gauge_idx, col, row)) * arg.inA(their_spinor_parity, back_idx, s_col, c_col);
	      }
	  }

	}

      } //nDim
    }

    // apply kappa
    for (int color_local=0; color_local<Mc; color_local++) out[color_local] *= -(Float)2.0*arg.kappa;
  }


  /**
     A.S.: staggered coarse dslash has more sparse structure
     Applies the coarse dslash on a given parity and checkerboard site index

     @param out The result -2 * kappa * Dslash in
     @param Y The coarse gauge field
     @param kappa Kappa value
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc>
  __device__ __host__ inline void ks_dslash(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity, int s_row, int color_block) {
    const int their_spinor_parity = (arg.nParity == 2) ? (parity+1)&1 : 0;

    int coord[nDim] = {0};
    getCoords(coord, x_cb, arg.dim, parity);
    const int s_col = (1 - s_row);//s_col = 1 if s_row = 0, and  s_col = 0 if s_row = 1.

    for(int d = 0; d < nDim; d++) { //Ndim
      //Forward link - compute fwd offset for spinor fetch
      {
	const int fwd_idx = linkIndexP1(coord, arg.dim, d);

        for(int color_local = 0; color_local < Mc; color_local++) { //Color row
          int c_row = color_block + color_local; // global color index
          int row = s_row*Nc + c_row;
	  for(int c_col = 0; c_col < Nc; c_col++){ //Color column
             int col = s_col*Nc + c_col;
	     out[color_local] -= (arg.Y(d, parity, x_cb, s_row, s_col, row, col)) * arg.inA(their_spinor_parity, fwd_idx, s_col, c_col);
          }
        }
      }
      //Backward link - compute back offset for spinor and gauge fetch
      {
	const int back_idx = linkIndexM1(coord, arg.dim, d);
	const int gauge_idx = back_idx;
        for(int color_local = 0; color_local < Mc; color_local++) {
          int c_row = color_block + color_local;
          int row = s_row*Nc + c_row;
	  for(int c_col = 0; c_col < Nc; c_col++){
             int col = s_col*Nc + c_col;
	     out[color_local] += conj(arg.Y(d, (parity+1)&1, gauge_idx, s_col, s_row, col, row)) * arg.inA(their_spinor_parity, back_idx, s_col, c_col);
          }  
        }
      } 
    } //nDim
  }


  /**
     Applies the coarse clover matrix on a given parity and
     checkerboard site index

     @param out The result out += X * in
     @param X The coarse clover field
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int Ns, int Nc, int Mc>
  __device__ __host__ inline void clover(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity, int s, int color_block) {
    const int spinor_parity = (arg.nParity == 2) ? parity : 0;

    // M is number of colors per thread
    for(int color_local = 0; color_local < Mc; color_local++) {//Color out
      int c = color_block + color_local; // global color index
      int row = s*Nc + c;
      for(int s_col = 0; s_col < Ns; s_col++) //Spin in
	for(int c_col = 0; c_col < Nc; c_col++) { //Color in
	  //Factor of 2*kappa and diagonal addition now incorporated in X
	  int col = s_col*Nc + c_col;
	  out[color_local] += arg.X(0, parity, x_cb, row, col) * arg.inB(spinor_parity, x_cb, s_col, c_col);
	}
    }

  }

  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc>
  __device__ __host__ inline void coarseDslash(CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity, int s, int color_block)
  {
    complex <Float> out[Mc];
    for (int c=0; c<Mc; c++) out[c] = 0.0;
    if(!arg.staggered_coarse_dslash)
      dslash<Float,F,G,nDim,Ns,Nc,Mc>(out, arg, x_cb, parity, s, color_block);
    else
      ks_dslash<Float,F,G,nDim,Ns,Nc,Mc>(out, arg, x_cb, parity, s, color_block);
    clover<Float,F,G,Ns,Nc,Mc>(out, arg, x_cb, parity, s, color_block);

    const int my_spinor_parity = (arg.nParity == 2) ? parity : 0;
    for (int color_local=0; color_local<Mc; color_local++) {
      int c = color_block + color_local; // global color index
      arg.out(my_spinor_parity, x_cb, s, c) = out[color_local];
    }
  }

  // CPU kernel for applying the coarse Dslash to a vector
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc>
  void coarseDslash(CoarseDslashArg<Float,F,G> arg)
  {
    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      //#pragma omp parallel for
      for(int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { //Volume
	for (int s=0; s<2; s++) {
	  for (int color_block=0; color_block<Nc; color_block+=Mc) { // Mc=Nc means all colors in a thread
	    coarseDslash<Float,F,G,nDim,Ns,Nc,Mc>(arg, x_cb, parity, s, color_block);
	  }
	}
      }//VolumeCB
    } // parity
    
  }

  // GPU Kernel for applying the coarse Dslash to a vector
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc>
  __global__ void coarseDslashKernel(CoarseDslashArg<Float,F,G> arg)
  {
    int x_cb = blockDim.x*blockIdx.x + threadIdx.x;
    if (x_cb >= arg.volumeCB) return;

    // for full fields then set parity from y thread index else use arg setting
    int parity = blockDim.y*blockIdx.y + threadIdx.y;
    int sM = blockDim.z*blockIdx.z + threadIdx.z;
    int s = sM / (Nc/Mc);
    int color_block = (sM % (Nc/Mc)) * Mc;

    coarseDslash<Float,F,G,nDim,Ns,Nc,Mc>(arg, x_cb, parity, s, color_block);
  }

  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc, int Mc>
  class CoarseDslash : public Tunable {

  protected:
    CoarseDslashArg<Float,F,G> &arg;
    const ColorSpinorField &meta;

    long long flops() const
    {
      return ((2*nDim+1)*(8*Ns*Nc*Ns*Nc)-2*Ns*Nc)*arg.nParity*arg.volumeCB;
    }
    long long bytes() const
    {
      return arg.out.Bytes() + 8*arg.inA.Bytes() + arg.inB.Bytes() + arg.nParity*(8*arg.Y.Bytes() + arg.X.Bytes());
    }
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.volumeCB; }

    bool advanceBlockDim(TuneParam &param) const
    {
      dim3 block = param.block;
      dim3 grid = param.grid;
      bool ret = Tunable::advanceBlockDim(param);
      param.block.y = block.y; param.block.z = block.z;
      param.grid.y = grid.y; param.grid.z = grid.z;

      if (ret) { // we advanced the block.x so we're done
	return true;
      } else { // block.x (spacetime) was reset

	if (param.block.y == 1 && arg.nParity == 2) { // advance parity
	  param.block.y = arg.nParity;
	  param.grid.y = 1;
	  return true;
	} else {
	  // reset parity
	  param.block.y = 1;
	  param.grid.y = arg.nParity;

	  // let's try to advance spin/block-color
	  while(param.block.z <= 2* (Nc/Mc)) {
	    param.block.z++;
	    if ( (2*(Nc/Mc)) % param.block.z == 0) {
	      param.grid.z = (2 * (Nc/Mc)) / param.block.z;
	      break;
	    }
	  }

	  // we can advance spin/block-color since this is valid
	  if (param.block.z <= 2 * (Nc/Mc)) { //
	    return true;
	  } else { // we have run off the end so let's reset
	    param.block.z = 1;
	    param.grid.z = 2 * (Nc/Mc);
	    return false;
	  }

	}
      }
    }

    bool advanceTuneParam(TuneParam &param) const 
    {
      bool rtn = Tunable::advanceTuneParam(param);
      return rtn;
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.block.y = 1;
      param.grid.y = arg.nParity;
      param.block.z = 1;
      param.grid.z = 2*(Nc/Mc);
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.block.y = 1;
      param.grid.y = arg.nParity;
      param.block.z = 1;
      param.grid.z = 2*(Nc/Mc);
    }


  public:
    CoarseDslash(CoarseDslashArg<Float,F,G> &arg, const ColorSpinorField &meta)
      : arg(arg), meta(meta) {
      strcpy(aux, meta.AuxString());
#ifdef MULTI_GPU
      char comm[5];
      comm[0] = (arg.commDim[0] ? '1' : '0');
      comm[1] = (arg.commDim[1] ? '1' : '0');
      comm[2] = (arg.commDim[2] ? '1' : '0');
      comm[3] = (arg.commDim[3] ? '1' : '0');
      comm[4] = '\0';
      strcat(aux,",comm=");
      strcat(aux,comm);
#endif
    }
    virtual ~CoarseDslash() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	coarseDslash<Float,F,G,nDim,Ns,Nc,Mc>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	coarseDslashKernel<Float,F,G,nDim,Ns,Nc,Mc> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), aux);
    }

  };


  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor, int coarseSpin, QudaFieldLocation location>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,  const GaugeField &Y, const GaugeField &X,
		   double kappa, bool is_staggered, int parity) {
    typedef typename colorspinor::FieldOrderCB<Float,coarseSpin,coarseColor,1,csOrder> F;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> G;

    F outAccessor(const_cast<ColorSpinorField&>(out));
    F inAccessorA(const_cast<ColorSpinorField&>(inA));
    F inAccessorB(const_cast<ColorSpinorField&>(inB));
    G yAccessor(const_cast<GaugeField&>(Y));
    G xAccessor(const_cast<GaugeField&>(X));
    CoarseDslashArg<Float,F,G> arg(outAccessor, inAccessorA, inAccessorB, yAccessor, xAccessor, (Float)kappa, parity, inA, is_staggered);

    const int colors_per_thread = 2;
    CoarseDslash<Float,F,G,4,coarseSpin,coarseColor,colors_per_thread> dslash(arg, inA);
    dslash.apply(0);
  }

  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor, int coarseSpin>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,  const GaugeField &Y, const GaugeField &X,
		   double kappa, bool is_staggered, int parity) {
    if (inA.Location() == QUDA_CUDA_FIELD_LOCATION) {
      ApplyCoarse<Float,csOrder,gOrder,coarseColor,coarseSpin,QUDA_CUDA_FIELD_LOCATION>(out, inA, inB, Y, X, kappa,is_staggered, parity);
    } else {
      ApplyCoarse<Float,csOrder,gOrder,coarseColor,coarseSpin,QUDA_CPU_FIELD_LOCATION>(out, inA, inB, Y, X, kappa,is_staggered, parity);
    }
  }

  // template on the number of coarse colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, bool is_staggered, int parity) {
    if (inA.Nspin() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",inA.Nspin());

    if (inA.Ncolor() == 2) {
      ApplyCoarse<Float,csOrder,gOrder,2,2>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.Ncolor() == 4) {
      ApplyCoarse<Float,csOrder,gOrder,4,2>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.Ncolor() == 8) {
      ApplyCoarse<Float,csOrder,gOrder,8,2>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.Ncolor() == 12) {
      ApplyCoarse<Float,csOrder,gOrder,12,2>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.Ncolor() == 16) {
      ApplyCoarse<Float,csOrder,gOrder,16,2>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.Ncolor() == 20) {
      ApplyCoarse<Float,csOrder,gOrder,20,2>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.Ncolor() == 24) {
      ApplyCoarse<Float,csOrder,gOrder,24,2>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  template <typename Float>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, bool is_staggered, int parity) {

    if (Y.FieldOrder() != X.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (inA.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (inA.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER && Y.FieldOrder() == QUDA_FLOAT2_GAUGE_ORDER) {
      ApplyCoarse<Float,QUDA_FLOAT2_FIELD_ORDER, QUDA_FLOAT2_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER && Y.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      ApplyCoarse<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,QUDA_QDP_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else {
      errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", inA.FieldOrder(), Y.FieldOrder());
    }
  }

#endif // GPU_MULTIGRID

  //Apply the coarse Dirac matrix to a coarse grid vector
  //out(x) = M*in = X*in - 2*kappa*\sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  //Note factor of 2*kappa compensates for the factor of 1/2 already
  //absorbed into the Y matrices.
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, bool is_staggered, int parity) {
#ifdef GPU_MULTIGRID
    if (Y.Precision() != inA.Precision() || Y.Precision() != inB.Precision() ||
	X.Precision() != Y.Precision() || Y.Precision() != out.Precision())
      errorQuda("Unsupported precision mix");

    if (inA.V() == out.V()) errorQuda("Aliasing pointers");
    if (out.Precision() != inA.Precision() ||
	Y.Precision() != inA.Precision() ||
	X.Precision() != inA.Precision()) 
      errorQuda("Precision mismatch out=%d inA=%d inB=%dY=%d X=%d", 
		out.Precision(), inA.Precision(), inB.Precision(), Y.Precision(), X.Precision());

    // check all locations match
    Location(out, inA, inB, Y, X);

    int dummy = 0; // ignored
    inA.exchangeGhost((QudaParity)(1-parity), dummy);

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyCoarse<double>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyCoarse<float>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
#else
    errorQuda("Multigrid has not been built");
#endif
  }//ApplyCoarse

} // namespace quda
