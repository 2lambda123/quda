#include "hip/hip_runtime.h"
#include <multigrid.h>
#include <transfer.h>
#include <gauge_field_order.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>

#define LEGACY_SPINOR
#define LEGACY_GAUGE

namespace quda {

#ifdef GPU_MULTIGRID

  template <typename Float, typename F, typename G>
  struct CoarseDslashArg {
    F out;
    const F inA;
    const F inB;
    const G Y;
    const G X;
    Float kappa;
    int parity; // only use this for single parity fields
    int nParity; // number of parities we're working on
    int volumeCB;
    int dim[5];   // full lattice dimensions
    int commDim[4]; // whether a given dimension is partitioned or not
    int nFace;  // hard code to 1 for now

    bool staggered_coarse_dslash;//staggered coarse dslash has more sparse structure.

    CoarseDslashArg(F &out, const F &inA, const F &inB, const G &Y, const G &X,
		    Float kappa, int parity, const ColorSpinorField &meta, bool is_staggered)
      : out(out), inA(inA), inB(inB), Y(Y), X(X), kappa(kappa), parity(parity),
	nParity(meta.SiteSubset()), volumeCB(meta.VolumeCB()), nFace(1), staggered_coarse_dslash(is_staggered) {
      for (int i=0; i<4; i++) {
	dim[i] = meta.X(i);
	commDim[i] = comm_dim_partitioned(i);
      }
      dim[0] = (nParity == 1) ? 2 * dim[0] : dim[0];
      dim[4] = 1; // ghost index expects a fifth dimension
    }
  };

  /**
     Applies the coarse dslash on a given parity and checkerboard site index

     @param out The result -2 * kappa * Dslash in
     @param Y The coarse gauge field
     @param kappa Kappa value
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  __device__ __host__ inline void dslash(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity) {
    const int their_spinor_parity = (arg.nParity == 2) ? (parity+1)&1 : 0;

    int coord[5];
    getCoords(coord, x_cb, arg.dim, parity);
    coord[4] = 0;

    for(int d = 0; d < nDim; d++) { //Ndim
      //Forward link - compute fwd offset for spinor fetch
      {
	complex<Float> in[Ns*Nc];
	complex<Float> Y[Ns*Nc][Ns*Nc];
	int fwd_idx = linkIndexP1(coord, arg.dim, d);
#ifdef LEGACY_SPINOR
	if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	  int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);
	  for (int s=0; s<Ns; s++)
	    for (int c=0; c<Nc; c++)
	      in[s*Nc+c] = arg.inA.Ghost(d, 1, their_spinor_parity, ghost_idx, s, c);
	} else {
	  for (int s=0; s<Ns; s++)
	    for (int c=0; c<Nc; c++)
	      in[s*Nc+c] = arg.inA(their_spinor_parity, fwd_idx, s, c);
	}

#else
	if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	  int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);
	  arg.inA.loadGhost(reinterpret_cast<Float*>(in), ghost_idx, d, 1, their_spinor_parity);
	} else {
	  arg.inA.load(reinterpret_cast<Float*>(in), fwd_idx, their_spinor_parity);
	}
#endif // LEGACY_SPINOR

#ifdef LEGACY_GAUGE
	for (int s_row=0; s_row<Ns; s_row++)
	  for (int c_row=0; c_row<Nc; c_row++)
	    for (int s_col=0; s_col<Ns; s_col++)
	      for (int c_col=0; c_col<Nc; c_col++)
		Y[s_row*Nc+c_row][s_col*Nc+c_col] = arg.Y(d, parity, x_cb, s_row, s_col, c_row, c_col);
#else
	arg.Y.load(reinterpret_cast<Float*>(Y), x_cb, d, parity);
#endif // LEGACY_GAUGE

	for(int s_row = 0; s_row < Ns; s_row++) { //Spin row
	  for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
	      Float sign = (s_row == s_col) ? 1.0 : -1.0;
	      for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		//out[s_row*Nc+c_row] += sign*arg.Y(d, parity, x_cb, s_row, s_col, c_row, c_col)
		//* in[s_col*Nc+c_col]; //arg.inA(their_spinor_parity, fwd_idx, s_col, c_col);
		out[s_row*Nc+c_row] += sign*(Y[s_row*Nc+c_row][s_col*Nc+c_col]) * in[s_col*Nc+c_col];
	      } //Color column
	    } //Spin column
	  } //Color row
	} //Spin row
      }

      //Backward link - compute back offset for spinor and gauge fetch
      {
	complex<Float> in[Ns*Nc];
	complex<Float> Y[Ns*Nc][Ns*Nc];
	int gauge_idx;
	int back_idx = linkIndexM1(coord, arg.dim, d);
#ifdef LEGACY_SPINOR
	if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	  int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
	  for (int s=0; s<Ns; s++)
	    for (int c=0; c<Nc; c++)
	      in[s*Nc+c] = arg.inA.Ghost(d, 0, their_spinor_parity, ghost_idx, s, c);
	} else {
	  for (int s=0; s<Ns; s++)
	    for (int c=0; c<Nc; c++)
	      in[s*Nc+c] = arg.inA(their_spinor_parity, back_idx, s, c);
	}
#else
	if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	  int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
	  arg.inA.loadGhost(reinterpret_cast<Float*>(in), ghost_idx, d, 0, their_spinor_parity);
	} else {
	  arg.inA.load(reinterpret_cast<Float*>(in), back_idx, their_spinor_parity);
	}
#endif // LEGACY_SPINOR

	gauge_idx = back_idx;
#ifdef LEGACY_GAUGE
	if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	  int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
	  for (int s_row=0; s_row<Ns; s_row++)
	    for (int c_row=0; c_row<Nc; c_row++)
	      for (int s_col=0; s_col<Ns; s_col++)
		for (int c_col=0; c_col<Nc; c_col++) {
		  Y[s_row*Nc+c_row][s_col*Nc+c_col] = arg.Y.Ghost(d, (parity+1)&1, ghost_idx, s_row, s_col, c_row, c_col);
		}

	} else {
	  for (int s_row=0; s_row<Ns; s_row++)
	    for (int c_row=0; c_row<Nc; c_row++)
	      for (int s_col=0; s_col<Ns; s_col++)
		for (int c_col=0; c_col<Nc; c_col++)
		  Y[s_row*Nc+c_row][s_col*Nc+c_col] = arg.Y(d, (parity+1)&1, gauge_idx, s_row, s_col, c_row, c_col);
	}
#else
	if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	  // load from ghost
	  int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
	  arg.Y.loadGhost(reinterpret_cast<Float*>(Y), ghost_idx, d, (parity+1)&1);
        } else {
	  arg.Y.load(reinterpret_cast<Float*>(Y), gauge_idx, d, (parity+1)&1);
	}
#endif // LEGACY_GAUGE

	for(int s_row = 0; s_row < Ns; s_row++) { //Spin row
	  for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
	      for(int c_col = 0; c_col < Nc; c_col++) { //Color column
	//out[s_row*Nc+c_row] += conj(arg.Y(d,(parity+1)&1, gauge_idx, s_col, s_row, c_col, c_row))
	//	  * in[s_col*Nc+c_col]; //arg.inA(their_spinor_parity, back_idx, s_col, c_col);
		out[s_row*Nc+c_row] += conj(Y[s_col*Nc+c_col][s_row*Nc+c_row]) * in[s_col*Nc+c_col];
	      } //Color column
	    } //Spin column
	  } //Color row
	} //Spin row
      } //nDim
    }

    // apply kappa
    for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++) out[s*Nc+c] *= -(Float)2.0*arg.kappa;
  }


  /**
     A.S.: staggered coarse dslash has more sparse structure
     Applies the coarse dslash on a given parity and checkerboard site index

     @param out The result -2 * kappa * Dslash in
     @param Y The coarse gauge field
     @param kappa Kappa value
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  __device__ __host__ inline void ks_dslash(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity) {
    const int their_spinor_parity = (arg.nParity == 2) ? (parity+1)&1 : 0;

    int coord[nDim];
    getCoords(coord, x_cb, arg.dim, parity);

    for(int d = 0; d < nDim; d++) { //Ndim
      //Forward link - compute fwd offset for spinor fetch
      {
	complex<Float> in[Ns*Nc];
	complex<Float> Y[Ns*Nc][Ns*Nc];
#ifdef LEGACY_SPINOR
	int fwd_idx = linkIndexP1(coord, arg.dim, d);
	for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++)
	  in[s*Nc+c] = arg.inA(their_spinor_parity, fwd_idx, s, c);
#else
	int fwd_idx = linkIndexP1(coord, arg.dim, d);
	arg.inA.load(reinterpret_cast<Float*>(in), fwd_idx, their_spinor_parity);
#endif // LEGACY_SPINOR

#ifdef LEGACY_GAUGE
	for (int s_row=0; s_row<Ns; s_row++)
	  for (int c_row=0; c_row<Nc; c_row++)
	    for (int s_col=0; s_col<Ns; s_col++)
	      for (int c_col=0; c_col<Nc; c_col++)
		Y[s_row*Nc+c_row][s_col*Nc+c_col] = arg.Y(d, (parity+1)&1, x_cb, s_row, s_col, c_row, c_col);
#else
	//if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	// load from ghost
	//} else {
	arg.Y.load(reinterpret_cast<Float*>(Y), x_cb, d, (parity+1)&1);
	  //}
#endif // LEGACY_GAUGE

        for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	  for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		out[0*Nc+c_row] -= Y[0*Nc+c_row][1*Nc+c_col] * in[1*Nc+c_col]; //arg.inA(their_spinor_parity, fwd_idx, s_col, c_col);
		out[1*Nc+c_row] -= Y[1*Nc+c_row][0*Nc+c_col] * in[0*Nc+c_col]; //arg.inA(their_spinor_parity, fwd_idx, s_col, c_col);
	  } //Color column
	} //Color row
      }

      //Backward link - compute back offset for spinor and gauge fetch
      {
	complex<Float> in[Ns*Nc];
	complex<Float> Y[Ns*Nc][Ns*Nc];
	int gauge_idx;
	int back_idx = linkIndexM1(coord, arg.dim, d);
#ifdef LEGACY_SPINOR
	for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++)
	  in[s*Nc+c] = arg.inA(their_spinor_parity, back_idx, s, c);
#else
	arg.inA.load(reinterpret_cast<Float*>(in), back_idx, their_spinor_parity);
#endif // LEGACY_SPINOR

	gauge_idx = back_idx;
#ifdef LEGACY_GAUGE
	for (int s_row=0; s_row<Ns; s_row++)
	  for (int c_row=0; c_row<Nc; c_row++)
	    for (int s_col=0; s_col<Ns; s_col++)
	      for (int c_col=0; c_col<Nc; c_col++)
		Y[s_row*Nc+c_row][s_col*Nc+c_col] = arg.Y(d, (parity+1)&1, gauge_idx, s_row, s_col, c_row, c_col);
#else
	//if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	// load from ghost
        //} else {
	arg.Y.load(reinterpret_cast<Float*>(Y), gauge_idx, d, (parity+1)&1);
	  //}
#endif // LEGACY_GAUGE

        for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	  for(int c_col = 0; c_col < Nc; c_col++) { //Color column
	      out[0*Nc+c_row] += conj(Y[1*Nc+c_col][0*Nc+c_row]) * in[1*Nc+c_col]; //arg.inA(their_spinor_parity, back_idx, s_col, c_col);
              out[1*Nc+c_row] += conj(Y[0*Nc+c_col][1*Nc+c_row]) * in[0*Nc+c_col]; //arg.inA(their_spinor_parity, back_idx, s_col, c_col);
	  } //Color column
	} //Color row
      } //nDim
    }
  }


  /**
     Applies the coarse clover matrix on a given parity and
     checkerboard site index

     @param out The result out += X * in
     @param X The coarse clover field
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, typename F, typename G, int Ns, int Nc>
  __device__ __host__ inline void clover(complex<Float> out[], CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity) {
    const int spinor_parity = (arg.nParity == 2) ? parity : 0;

    complex<Float> in[Ns*Nc];
    complex<Float> X[Ns*Nc][Ns*Nc];
#ifdef LEGACY_SPINOR
    for (int s=0; s<Ns; s++)
      for (int c=0; c<Nc; c++)
	in[s*Nc+c] = arg.inB(spinor_parity, x_cb, s, c);
#else
    arg.inB.load(reinterpret_cast<Float*>(in), x_cb, spinor_parity);
#endif // LEGACY_SPINOR

#ifdef LEGACY_GAUGE
    for (int s_row=0; s_row<Ns; s_row++)
      for (int c_row=0; c_row<Nc; c_row++)
	for (int s_col=0; s_col<Ns; s_col++)
	  for (int c_col=0; c_col<Nc; c_col++)
	    X[s_row*Nc+c_row][s_col*Nc+c_col] = arg.X(0, parity, x_cb, s_row, s_col, c_row, c_col);
#else
    arg.X.load(reinterpret_cast<Float*>(X), x_cb, 0, parity);
#endif

    // apply clover term
    for(int s = 0; s < Ns; s++) { //Spin out
      for(int c = 0; c < Nc; c++) { //Color out
	//This term is now incorporated into the matrix X.
	//out(parity,x_cb,s,c) += in(parity,x_cb,s,c);
	for(int s_col = 0; s_col < Ns; s_col++) { //Spin in
	  for(int c_col = 0; c_col < Nc; c_col++) { //Color in
	    //Factor of 2*kappa now incorporated in X
	    out[s*Nc+c] += X[s*Nc+c][s_col*Nc+c_col] *in[s_col*Nc+c_col];
	  } //Color in
	} //Spin in
      } //Color out
    } //Spin out
  }

  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  __device__ __host__ inline void coarseDslash(CoarseDslashArg<Float,F,G> &arg, int x_cb, int parity)
  {
    complex <Float> out[Ns*Nc];
    for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++) out[s*Nc+c] = 0.0;

    if(!arg.staggered_coarse_dslash) 
      dslash<Float,F,G,nDim,Ns,Nc>(out, arg, x_cb, parity);
    else
      ks_dslash<Float,F,G,nDim,Ns,Nc>(out, arg, x_cb, parity);

    clover<Float,F,G,Ns,Nc>(out, arg, x_cb, parity);

    const int my_spinor_parity = (arg.nParity == 2) ? parity : 0;
#ifdef LEGACY_SPINOR
    for (int s=0; s<Ns; s++)
      for (int c=0; c<Nc; c++)
	arg.out(my_spinor_parity, x_cb, s, c) = out[s*Nc+c];
#else
    arg.out.save(reinterpret_cast<Float*>(out), x_cb, my_spinor_parity);
#endif
  }

  // CPU kernel for applying the coarse Dslash to a vector
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  void coarseDslash(CoarseDslashArg<Float,F,G> arg)
  {
    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for(int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { //Volume
        coarseDslash<Float,F,G,nDim,Ns,Nc>(arg, x_cb, parity);
      }//VolumeCB
    } // parity
    
  }

  // GPU Kernel for applying the coarse Dslash to a vector
  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  __global__ void coarseDslashKernel(CoarseDslashArg<Float,F,G> arg)
  {
    int x_cb = blockDim.x*blockIdx.x + threadIdx.x;
    if (x_cb >= arg.volumeCB) return;

    // for full fields then set parity from y thread index else use arg setting
    int parity = (blockDim.y == 2) ? threadIdx.y : arg.parity;

    coarseDslash<Float,F,G,nDim,Ns,Nc>(arg, x_cb, parity);
  }

  template <typename Float, typename F, typename G, int nDim, int Ns, int Nc>
  class CoarseDslash : public Tunable {

  protected:
    CoarseDslashArg<Float,F,G> &arg;
    const ColorSpinorField &meta;

    long long flops() const { return 0; }
    long long bytes() const
    {
      return arg.out.Bytes() + 8*arg.inA.Bytes() + arg.inB.Bytes() + arg.nParity*(8*arg.Y.Bytes() + arg.X.Bytes());
    }
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.volumeCB; }

    bool advanceTuneParam(TuneParam &param) const 
    {
      bool rtn = Tunable::advanceTuneParam(param);
      param.block.y = arg.nParity;
      return rtn;
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.block.y = arg.nParity;
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.block.y = arg.nParity;
    }


  public:
    CoarseDslash(CoarseDslashArg<Float,F,G> &arg, const ColorSpinorField &meta)
      : arg(arg), meta(meta) {
      strcpy(aux, meta.AuxString());
#ifdef MULTI_GPU
      char comm[5];
      comm[0] = (arg.commDim[0] ? '1' : '0');
      comm[1] = (arg.commDim[1] ? '1' : '0');
      comm[2] = (arg.commDim[2] ? '1' : '0');
      comm[3] = (arg.commDim[3] ? '1' : '0');
      comm[4] = '\0';
      strcat(aux,",comm=");
      strcat(aux,comm);
#endif
    }
    virtual ~CoarseDslash() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	coarseDslash<Float,F,G,nDim,Ns,Nc>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	coarseDslashKernel<Float,F,G,nDim,Ns,Nc> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), aux);
    }

  };


  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor, int coarseSpin, QudaFieldLocation location>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,  const GaugeField &Y, const GaugeField &X,
		   double kappa, bool is_staggered, int parity) {
#ifdef LEGACY_SPINOR
    typedef typename colorspinor::FieldOrderCB<Float,coarseSpin,coarseColor,1,csOrder> F;
#else
    typedef typename colorspinor_order_mapper<Float,csOrder,coarseSpin,coarseColor>::type F;
#endif // LEGACY_SPINOR

#ifdef LEGACY_GAUGE
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> G;
#else
    typedef typename gauge_order_mapper<Float,gOrder,coarseSpin*coarseColor>:: type G;
#endif // LEGACY_GAUGE

    F outAccessor(const_cast<ColorSpinorField&>(out));
    F inAccessorA(const_cast<ColorSpinorField&>(inA));
    F inAccessorB(const_cast<ColorSpinorField&>(inB));
    G yAccessor(const_cast<GaugeField&>(Y));
    G xAccessor(const_cast<GaugeField&>(X));
    CoarseDslashArg<Float,F,G> arg(outAccessor, inAccessorA, inAccessorB, yAccessor, xAccessor, (Float)kappa, parity, inA, is_staggered);
    CoarseDslash<Float,F,G,4,coarseSpin,coarseColor> dslash(arg, inA);
    dslash.apply(0);
  }

  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor, int coarseSpin>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,  const GaugeField &Y, const GaugeField &X,
		   double kappa, bool is_staggered, int parity) {
    if (inA.Location() == QUDA_CUDA_FIELD_LOCATION) {
      ApplyCoarse<Float,csOrder,gOrder,coarseColor,coarseSpin,QUDA_CUDA_FIELD_LOCATION>(out, inA, inB, Y, X, kappa,is_staggered, parity);
    } else {
      ApplyCoarse<Float,csOrder,gOrder,coarseColor,coarseSpin,QUDA_CPU_FIELD_LOCATION>(out, inA, inB, Y, X, kappa,is_staggered, parity);
    }
  }

  // template on the number of coarse colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, bool is_staggered, int parity) {
    if (inA.Nspin() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",inA.Nspin());

    if (inA.Ncolor() == 2) { 
      ApplyCoarse<Float,csOrder,gOrder,2,2>(out, inA, inB, Y, X, kappa,is_staggered, parity);
    } else if (inA.Ncolor() == 24) { 
      ApplyCoarse<Float,csOrder,gOrder,24,2>(out, inA, inB, Y, X, kappa,is_staggered, parity);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  template <typename Float>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, bool is_staggered, int parity) {

    if (Y.FieldOrder() != X.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (inA.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (inA.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER && Y.FieldOrder() == QUDA_FLOAT2_GAUGE_ORDER) {
      ApplyCoarse<Float,QUDA_FLOAT2_FIELD_ORDER, QUDA_FLOAT2_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (inA.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER && Y.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      ApplyCoarse<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,QUDA_QDP_GAUGE_ORDER>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else {
      errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", inA.FieldOrder(), Y.FieldOrder());
    }
  }

#endif // GPU_MULTIGRID

  //Apply the coarse Dirac matrix to a coarse grid vector
  //out(x) = M*in = X*in - 2*kappa*\sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  //Note factor of 2*kappa compensates for the factor of 1/2 already
  //absorbed into the Y matrices.
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
		   const GaugeField &Y, const GaugeField &X, double kappa, bool is_staggered, int parity) {
#ifdef GPU_MULTIGRID
    if (Y.Precision() != inA.Precision() || Y.Precision() != inB.Precision() ||
	X.Precision() != Y.Precision() || Y.Precision() != out.Precision())
      errorQuda("Unsupported precision mix");

    if (inA.V() == out.V()) errorQuda("Aliasing pointers");
    if (out.Precision() != inA.Precision() ||
	Y.Precision() != inA.Precision() ||
	X.Precision() != inA.Precision()) 
      errorQuda("Precision mismatch out=%d inA=%d inB=%dY=%d X=%d", 
		out.Precision(), inA.Precision(), inB.Precision(), Y.Precision(), X.Precision());

    // check all locations match
    Location(out, inA, inB, Y, X);

    int dummy = 0; // ignored
    inA.exchangeGhost((QudaParity)(1-parity), dummy);

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyCoarse<double>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyCoarse<float>(out, inA, inB, Y, X, kappa, is_staggered, parity);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
#else
    errorQuda("Multigrid has not been built");
#endif
  }//ApplyCoarse

} // namespace quda
