#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <atomic.cuh>
#include <cub_helper.cuh>
#include <index_helper.cuh>

namespace quda {

#ifdef GPU_GAUGE_TOOLS

  template <typename Gauge>
  struct GaugePlaqArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4]; 
#endif
    Gauge dataOr;
    
    double2 *partial;
    double2 *plaq;
    double2 *plaq_h;

    GaugePlaqArg(const Gauge &dataOr, const GaugeField &data)
      : dataOr(dataOr), 
	partial(static_cast<double2*>(getDeviceReduceBuffer())),
	plaq(static_cast<double2*>(getMappedHostReduceBuffer())),
	plaq_h(static_cast<double2*>(getHostReduceBuffer())) 
    {

#ifdef MULTI_GPU
        for(int dir=0; dir<4; ++dir){
          border[dir] = 2;
	  X[dir] = data.X()[dir] - border[dir]*2;
        }
#else
        for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir];
#endif
	threads = X[0]*X[1]*X[2]*X[3]/2;
    }
  };

  template<int blockSize, typename Float, typename Gauge>
    __global__ void computePlaq(GaugePlaqArg<Gauge> arg){
      int idx = threadIdx.x + blockIdx.x*blockDim.x;
      int parity = threadIdx.y;

      double2 plaq = make_double2(0.0,0.0);

      if(idx < arg.threads) {
        typedef typename ComplexTypeId<Float>::Type Cmplx;
        int X[4]; 
        for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

        int x[4];
        getCoords(x, idx, X, parity);
#ifdef MULTI_GPU
        for(int dr=0; dr<4; ++dr) {
          x[dr] += arg.border[dr];
          X[dr] += 2*arg.border[dr];
        }
#endif

        int dx[4] = {0, 0, 0, 0};
        for (int mu = 0; mu < 3; mu++) {
          for (int nu = (mu+1); nu < 3; nu++) {
            Matrix<Cmplx,3> U1, U2, U3, U4, tmpM;

            arg.dataOr.load((Float*)(U1.data),linkIndexShift(x,dx,X), mu, parity);
	    dx[mu]++;
            arg.dataOr.load((Float*)(U2.data),linkIndexShift(x,dx,X), nu, 1-parity);
            dx[mu]--;
            dx[nu]++;
            arg.dataOr.load((Float*)(U3.data),linkIndexShift(x,dx,X), mu, 1-parity);
	    dx[nu]--;
            arg.dataOr.load((Float*)(U4.data),linkIndexShift(x,dx,X), nu, parity);

	    tmpM = U1 * U2;
	    tmpM = tmpM * conj(U3);
	    tmpM = tmpM * conj(U4);

	    plaq.x += getTrace(tmpM).x;
          }

          Matrix<Cmplx,3> U1, U2, U3, U4, tmpM;

          arg.dataOr.load((Float*)(U1.data),linkIndexShift(x,dx,X), mu, parity);
          dx[mu]++;
          arg.dataOr.load((Float*)(U2.data),linkIndexShift(x,dx,X), 3, 1-parity);
          dx[mu]--;
          dx[3]++;
          arg.dataOr.load((Float*)(U3.data),linkIndexShift(x,dx,X), mu, 1-parity);
          dx[3]--;
          arg.dataOr.load((Float*)(U4.data),linkIndexShift(x,dx,X), 3, parity);

          tmpM = U1 * U2;
          tmpM = tmpM * conj(U3);
          tmpM = tmpM * conj(U4);

          plaq.y += getTrace(tmpM).x;
        }
      }

      // perform final inter-block reduction and write out result
      reduce2d<blockSize,2>(arg.plaq, arg.partial, plaq);
  }

  template<typename Float, typename Gauge>
    class GaugePlaq : Tunable {
      GaugePlaqArg<Gauge> arg;
      const QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.threads; }

      public:
      GaugePlaq(GaugePlaqArg<Gauge> &arg, QudaFieldLocation location)
        : arg(arg), location(location) {}
      ~GaugePlaq () { }

      bool advanceBlockDim(TuneParam &param) const {
      	bool rtn = Tunable::advanceBlockDim(param);
	param.block.y = 2;
	return rtn;
      }

      void initTuneParam(TuneParam &param) const {
	Tunable::initTuneParam(param);
	param.block.y = 2;
      }

      void apply(const hipStream_t &stream){
        if(location == QUDA_CUDA_FIELD_LOCATION){
          arg.plaq_h[0] = make_double2(0.,0.);
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

	  LAUNCH_KERNEL(computePlaq, tp, stream, arg, Float, Gauge);
	  hipDeviceSynchronize();
        } else {
          errorQuda("CPU not supported yet\n");
        }
      }

      TuneKey tuneKey() const {
        std::stringstream vol, aux;
        vol << arg.X[0] << "x";
        vol << arg.X[1] << "x";
        vol << arg.X[2] << "x";
	vol << arg.X[3];
	aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
        return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
      }

      std::string paramString(const TuneParam &param) const {
        std::stringstream ps;
        ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
        ps << "shared=" << param.shared_bytes;
        return ps.str();
      }

      void preTune(){}
      void postTune(){}
      long long flops() const { return 6ll*2*arg.threads*(3*198+3); }
      long long bytes() const { return 6ll*4*2*arg.threads*arg.dataOr.Bytes(); } 

    }; 

  template<typename Float, typename Gauge>
    void plaquette(const Gauge dataOr, const GaugeField& data, QudaFieldLocation location, double2 &plq) {
      GaugePlaqArg<Gauge> arg(dataOr, data);
      GaugePlaq<Float,Gauge> gaugePlaq(arg, location);
      gaugePlaq.apply(0);

      comm_allreduce_array((double*) arg.plaq_h, 2);
      arg.plaq_h[0].x /= 9.*(2*arg.threads*comm_size());
      arg.plaq_h[0].y /= 9.*(2*arg.threads*comm_size());

      plq.x = arg.plaq_h[0].x;
      plq.y = arg.plaq_h[0].y;
    }

  template<typename Float>
    double2 plaquette(const GaugeField& data, QudaFieldLocation location) {
      double2 res;
      if (!data.isNative()) errorQuda("Plaquette computation only supported on native ordered fields");

      if(data.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type Gauge;
	plaquette<Float>(Gauge(data), data, location, res);
      } else if(data.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type Gauge;
	plaquette<Float>(Gauge(data), data, location, res);
      } else if(data.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type Gauge;
	plaquette<Float>(Gauge(data), data, location, res);
      } else {
	errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
      }

      return res;
    }
#endif

  double3 plaquette(const GaugeField& data, QudaFieldLocation location) {

#ifdef GPU_GAUGE_TOOLS
    double2 plq;
    if(data.Precision() == QUDA_HALF_PRECISION) {
      errorQuda("Half precision not supported\n");
    }
    if (data.Precision() == QUDA_SINGLE_PRECISION) {
      plq = plaquette<float> (data, location);
    } else if(data.Precision() == QUDA_DOUBLE_PRECISION) {
      plq = plaquette<double>(data, location);
    } else {
      errorQuda("Precision %d not supported", data.Precision());
    }
    double3 plaq = make_double3(0.5*(plq.x + plq.y), plq.x, plq.y);
#else
    errorQuda("Gauge tools are not build");
    double3 plaq = make_double3(0., 0., 0.);
#endif
    return plaq;
  }
}
