#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <instantiate.h>
#include <tunable_reduction.h>
#include <kernels/gauge_plaq.cuh>

namespace quda {

  template<typename Float, int nColor, QudaReconstructType recon>
  class GaugePlaq : TunableReduction2D<Plaquette> {
    const GaugeField &u;
    double2 &plq;

  public:
    GaugePlaq(const GaugeField &u, double2 &plq) :
      TunableReduction2D(u),
      u(u),
      plq(plq)
    {
      apply(0);
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      GaugePlaqArg<Float, nColor, recon> arg(u);
      launch(tp, stream, arg);
      arg.complete(plq);
      if (!activeTuning()) {
        comm_allreduce_array((double*)&plq, 2);
        for (int i = 0; i < 2; i++) ((double*)&plq)[i] /= 9.*2*arg.threads*comm_size();
      }
    }

    long long flops() const
    {
      auto Nc = u.Ncolor();
      return 6ll*u.Volume()*(3 * (8 * Nc * Nc * Nc - 2 * Nc * Nc) + Nc);
    }
    long long bytes() const { return u.Bytes(); }
  };

  double3 plaquette(const GaugeField &U)
  {
    double2 plq;
    instantiate<GaugePlaq>(U, plq);
    double3 plaq = make_double3(0.5*(plq.x + plq.y), plq.x, plq.y);
    return plaq;
  }

} // namespace quda
