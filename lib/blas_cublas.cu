#include "hip/hip_runtime.h"
#ifdef CUBLAS_LIB
#include <hipblas.h>
#endif
#include <blas_cublas.h>
#include <malloc_quda.h>
#include <quda.h>
#include <complex.h>

#define FMULS_GETRF(m_, n_) ( ((m_) < (n_))				\
			      ? (0.5 * (m_) * ((m_) * ((n_) - (1./3.) * (m_) - 1. ) + (n_)) + (2. / 3.) * (m_)) \
			      : (0.5 * (n_) * ((n_) * ((m_) - (1./3.) * (n_) - 1. ) + (m_)) + (2. / 3.) * (n_)) )
#define FADDS_GETRF(m_, n_) ( ((m_) < (n_)) \
			      ? (0.5 * (m_) * ((m_) * ((n_) - (1./3.) * (m_)      ) - (n_)) + (1. / 6.) * (m_)) \
			      : (0.5 * (n_) * ((n_) * ((m_) - (1./3.) * (n_)      ) - (m_)) + (1. / 6.) * (n_)) )

#define FMULS_GETRI(n_) ( (n_) * ((5. / 6.) + (n_) * ((2. / 3.) * (n_) + 0.5)) )
#define FADDS_GETRI(n_) ( (n_) * ((5. / 6.) + (n_) * ((2. / 3.) * (n_) - 1.5)) )

#define FLOPS_CGETRF(m_, n_) (6. * FMULS_GETRF((double)(m_), (double)(n_)) + 2.0 * FADDS_GETRF((double)(m_), (double)(n_)) )
#define FLOPS_CGETRI(n_) (6. * FMULS_GETRI((double)(n_)) + 2.0 * FADDS_GETRI((double)(n_)) )

namespace quda {
  
  namespace cublas { 

#ifdef CUBLAS_LIB
    static hipblasHandle_t handle;
#endif
    
    void init() {
#ifdef CUBLAS_LIB
      hipblasStatus_t error = hipblasCreate(&handle);
      if (error != HIPBLAS_STATUS_SUCCESS) errorQuda("hipblasCreate failed with error %d", error);
#endif
    }
    
    void destroy() {
#ifdef CUBLAS_LIB
      hipblasStatus_t error = hipblasDestroy(handle);
      if (error != HIPBLAS_STATUS_SUCCESS) errorQuda("\nError indestroying cublas context, error code = %d\n", error);
#endif
    }

    // mini kernel to set the array of pointers needed for batched invert
    template<typename T>
    __global__ void set_pointer_invert(T **output_array_a, T *input_a, T **output_array_b, T *input_b, int batch_offset)
    {
      output_array_a[blockIdx.x] = input_a + blockIdx.x * batch_offset;
      output_array_b[blockIdx.x] = input_b + blockIdx.x * batch_offset;
    }

    // mini kernel to set the array of pointers needed for batched gemm
    template<typename T>
    __global__ void set_pointer_gemm(T **output_array_a, T *input_a, int batch_offset_a, T **output_array_b, T *input_b, int batch_offset_b, T **output_array_c, T *input_c, int batch_offset_c)
    {
      output_array_a[blockIdx.x] = input_a + blockIdx.x * batch_offset_a;
      output_array_b[blockIdx.x] = input_b + blockIdx.x * batch_offset_b;
      output_array_c[blockIdx.x] = input_c + blockIdx.x * batch_offset_c;
    }

    
    // FIXME do this in pipelined fashion to reduce memory overhead.
    long long BatchInvertMatrix(void *Ainv, void* A, const int n, const uint64_t batch, QudaPrecision prec, QudaFieldLocation location)
    {
      long long flops = 0;
#ifdef CUBLAS_LIB
      timeval start, stop;
      gettimeofday(&start, NULL);

      size_t size = 2*n*n*prec*batch;
      void *A_d = location == QUDA_CUDA_FIELD_LOCATION ? A : pool_device_malloc(size);
      void *Ainv_d = location == QUDA_CUDA_FIELD_LOCATION ? Ainv : pool_device_malloc(size);
      if (location == QUDA_CPU_FIELD_LOCATION) qudaMemcpy(A_d, A, size, hipMemcpyHostToDevice);
      
      int *dipiv = static_cast<int*>(pool_device_malloc(batch*n*sizeof(int)));
      int *dinfo_array = static_cast<int*>(pool_device_malloc(batch*sizeof(int)));
      int *info_array = static_cast<int*>(pool_pinned_malloc(batch*sizeof(int)));
      memset(info_array, '0', batch*sizeof(int)); // silence memcheck warnings

      if (prec == QUDA_SINGLE_PRECISION) {
	typedef hipFloatComplex C;
	C **A_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));
	C **Ainv_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));

	set_pointer_invert<C><<<batch,1>>>(A_array, (C*)A_d, Ainv_array, (C*)Ainv_d, n*n);

	hipblasStatus_t error = hipblasCgetrfBatched(handle, n, A_array, n, dipiv, dinfo_array, batch);
	flops += batch*FLOPS_CGETRF(n,n);

	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in LU decomposition (hipblasCgetrfBatched), error code = %d\n", error);

	qudaMemcpy(info_array, dinfo_array, batch*sizeof(int), hipMemcpyDeviceToHost);
	for (uint64_t i=0; i<batch; i++) {
	  if (info_array[i] < 0) {
	    errorQuda("%lu argument had an illegal value or another error occured, such as memory allocation failed", i);
	  } else if (info_array[i] > 0) {
	    errorQuda("%lu factorization completed but the factor U is exactly singular", i);
	  }
	}
    
	error = hipblasCgetriBatched(handle, n, (const C**)A_array, n, dipiv, Ainv_array, n, dinfo_array, batch);
	flops += batch*FLOPS_CGETRI(n);

	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in matrix inversion (hipblasCgetriBatched), error code = %d\n", error);

	qudaMemcpy(info_array, dinfo_array, batch*sizeof(int), hipMemcpyDeviceToHost);

	for (uint64_t i=0; i<batch; i++) {
	  if (info_array[i] < 0) {
	    errorQuda("%lu argument had an illegal value or another error occured, such as memory allocation failed", i);
	  } else if (info_array[i] > 0) {
	    errorQuda("%lu factorization completed but the factor U is exactly singular", i);
	  }
	}

	pool_device_free(Ainv_array);
	pool_device_free(A_array);

      } else {
	errorQuda("%s not implemented for precision=%d", __func__, prec);
      }

      if (location == QUDA_CPU_FIELD_LOCATION) {
	qudaMemcpy(Ainv, Ainv_d, size, hipMemcpyDeviceToHost);
	pool_device_free(Ainv_d);
	pool_device_free(A_d);
      }

      pool_device_free(dipiv);
      pool_device_free(dinfo_array);
      pool_pinned_free(info_array);

      qudaDeviceSynchronize();
      gettimeofday(&stop, NULL);
      long ds = stop.tv_sec - start.tv_sec;
      long dus = stop.tv_usec - start.tv_usec;
      double time = ds + 0.000001*dus;

      if (getVerbosity() >= QUDA_VERBOSE)
	printfQuda("Batched matrix inversion completed in %f seconds with GFLOPS = %f\n", time, 1e-9 * flops / time);
#endif // CUBLAS_LIB

      return flops;
    }

    long long BatchGEMM(void *A_data, void* B_data, void* C_data, QudaCublasParam cublas_param, QudaFieldLocation location)
    {
      long long flops = 0;
#ifdef CUBLAS_LIB
      timeval start, stop;
      gettimeofday(&start, NULL);

      const uint64_t batch = cublas_param.batch_count;
      uint64_t data_size = (cublas_param.data_type == QUDA_CUBLAS_DATATYPE_S ||
			    cublas_param.data_type == QUDA_CUBLAS_DATATYPE_C) ? 4 : 8;

      if(cublas_param.data_type == QUDA_CUBLAS_DATATYPE_C ||
	 cublas_param.data_type == QUDA_CUBLAS_DATATYPE_Z) {
	data_size *= 2;
      }
      
      // Number of data in one batch of the array
      unsigned int A_size = cublas_param.m * cublas_param.k; //A_mk
      unsigned int B_size = cublas_param.k * cublas_param.n; //B_kn
      unsigned int C_size = cublas_param.m * cublas_param.n; //C_mn
      
      // Data size of the entire array 
      size_t sizeAarr = A_size * data_size * batch;
      size_t sizeBarr = B_size * data_size * batch;
      size_t sizeCarr = C_size * data_size * batch;

      // If already on the device, just use the given pointer. If the data is on
      // the host, allocate device memory and transfer
      void *A_d = location == QUDA_CUDA_FIELD_LOCATION ? A_data : pool_device_malloc(sizeAarr);
      void *B_d = location == QUDA_CUDA_FIELD_LOCATION ? B_data : pool_device_malloc(sizeBarr);
      void *C_d = location == QUDA_CUDA_FIELD_LOCATION ? C_data : pool_device_malloc(sizeCarr);
      if (location == QUDA_CPU_FIELD_LOCATION) {
	qudaMemcpy(A_d, A_data, sizeAarr, hipMemcpyHostToDevice);
	qudaMemcpy(B_d, B_data, sizeBarr, hipMemcpyHostToDevice);
	qudaMemcpy(C_d, C_data, sizeCarr, hipMemcpyHostToDevice);
      }

      hipblasOperation_t trans_a = HIPBLAS_OP_N;
      switch(cublas_param.trans_a) {
      case QUDA_CUBLAS_OP_N: trans_a = HIPBLAS_OP_N; break;
      case QUDA_CUBLAS_OP_T: trans_a = HIPBLAS_OP_T; break;
      case QUDA_CUBLAS_OP_C: trans_a = HIPBLAS_OP_C; break;
      default : errorQuda("Unknown QUDA_CUBLAS_OP type %d\n", cublas_param.trans_a);
      }
      
      hipblasOperation_t trans_b = HIPBLAS_OP_N;
      switch(cublas_param.trans_b) {
      case QUDA_CUBLAS_OP_N: trans_b = HIPBLAS_OP_N; break;
      case QUDA_CUBLAS_OP_T: trans_b = HIPBLAS_OP_T; break;
      case QUDA_CUBLAS_OP_C: trans_b = HIPBLAS_OP_C; break;
      default : errorQuda("Unknown QUDA_CUBLAS_OP type %d\n", cublas_param.trans_b);
      }
            
      if (cublas_param.data_type == QUDA_CUBLAS_DATATYPE_Z) {
	
	typedef hipDoubleComplex Z ;
	
	const Z alpha = make_double2((double)creal(cublas_param.alpha),
				     (double)cimag(cublas_param.alpha));
	
	const Z beta  = make_double2((double)creal(cublas_param.beta),
				     (double)cimag(cublas_param.beta));

	hipblasStatus_t error;
	if(batch > 1) {
	  Z **A_ptr_array = static_cast<Z**>(pool_device_malloc(batch*sizeof(Z*)));
	  Z **B_ptr_array = static_cast<Z**>(pool_device_malloc(batch*sizeof(Z*)));
	  Z **C_ptr_array = static_cast<Z**>(pool_device_malloc(batch*sizeof(Z*)));
	  
	  set_pointer_gemm<Z><<<batch,1>>>(A_ptr_array, (Z*)A_d, A_size, B_ptr_array, (Z*)B_d, B_size, C_ptr_array, (Z*)C_d, C_size);	
	  error = hipblasZgemmBatched(handle, trans_a, trans_b, cublas_param.m,
				     cublas_param.n, cublas_param.k, &alpha,
				     A_ptr_array + cublas_param.a_offset, cublas_param.lda,
				     B_ptr_array + cublas_param.b_offset, cublas_param.ldb, &beta,
				     C_ptr_array + cublas_param.c_offset, cublas_param.ldc, batch);
	  
	  pool_device_free(A_ptr_array);
	  pool_device_free(B_ptr_array);
	  pool_device_free(C_ptr_array);
	  
	} else {
	  error = hipblasZgemm(handle, trans_a, trans_b, cublas_param.m,
			      cublas_param.n, cublas_param.k, &alpha,
			      (Z*)A_d + cublas_param.a_offset, cublas_param.lda,
			      (Z*)B_d + cublas_param.b_offset, cublas_param.ldb, &beta,
			      (Z*)C_d + cublas_param.c_offset, cublas_param.ldc);
	}
	
	//flops += batch*FLOPS_CGETRF(n,n);
	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in cuBLASZGEMMBatched), error code = %d\n", error);
	
      } else if (cublas_param.data_type == QUDA_CUBLAS_DATATYPE_C) {
	
	typedef hipComplex C;
	
	const C alpha = make_float2((float)creal(cublas_param.alpha),
				    (float)cimag(cublas_param.alpha));
	
	const C beta  = make_float2((float)creal(cublas_param.beta),
				    (float)cimag(cublas_param.beta));

	hipblasStatus_t error;
	if(batch > 1) {
	  C **A_ptr_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));
	  C **B_ptr_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));
	  C **C_ptr_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));
	  
	  set_pointer_gemm<C><<<batch,1>>>(A_ptr_array, (C*)A_d, A_size, B_ptr_array, (C*)B_d, B_size, C_ptr_array, (C*)C_d, C_size);
	  
	  error = hipblasCgemmBatched(handle, trans_a, trans_b, cublas_param.m,
				     cublas_param.n, cublas_param.k, &alpha,
				     A_ptr_array + cublas_param.a_offset, cublas_param.lda,
				     B_ptr_array + cublas_param.b_offset, cublas_param.ldb, &beta,
				     C_ptr_array + cublas_param.c_offset, cublas_param.ldc, batch);

	  pool_device_free(A_ptr_array);
	  pool_device_free(B_ptr_array);
	  pool_device_free(C_ptr_array);
	  
	} else {
	  error = hipblasCgemm(handle, trans_a, trans_b, cublas_param.m,
			      cublas_param.n, cublas_param.k, &alpha,
			      (C*)A_d + cublas_param.a_offset, cublas_param.lda,
			      (C*)B_d + cublas_param.b_offset, cublas_param.ldb, &beta,
			      (C*)C_d + cublas_param.c_offset, cublas_param.ldc);
	  
	}
	
	//flops += batch*FLOPS_CGETRF(n,n);
	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in cuBLASCGEMMBatched), error code = %d\n", error);
		
      } else if (cublas_param.data_type == QUDA_CUBLAS_DATATYPE_D) {
	
	typedef double D;

	const D alpha = (D)creal(cublas_param.alpha);	
	const D beta  = (D)creal(cublas_param.beta);
	
	hipblasStatus_t error;
	if(batch > 1) {
	  D **A_ptr_array = static_cast<D**>(pool_device_malloc(batch*sizeof(D*)));
	  D **B_ptr_array = static_cast<D**>(pool_device_malloc(batch*sizeof(D*)));
	  D **C_ptr_array = static_cast<D**>(pool_device_malloc(batch*sizeof(D*)));
	  
	  set_pointer_gemm<D><<<batch,1>>>(A_ptr_array, (D*)A_d, A_size, B_ptr_array, (D*)B_d, B_size, C_ptr_array, (D*)C_d, C_size);
	  
	  error = hipblasDgemmBatched(handle, trans_a, trans_b, cublas_param.m,
				     cublas_param.n, cublas_param.k, &alpha,
				     A_ptr_array + cublas_param.a_offset, cublas_param.lda,
				     B_ptr_array + cublas_param.b_offset, cublas_param.ldb, &beta,
				     C_ptr_array + cublas_param.c_offset, cublas_param.ldc, batch);

	  pool_device_free(A_ptr_array);
	  pool_device_free(B_ptr_array);
	  pool_device_free(C_ptr_array);
	  
	} else {
	  error = hipblasDgemm(handle, trans_a, trans_b, cublas_param.m,
			      cublas_param.n, cublas_param.k, &alpha,
			      (D*)A_d + cublas_param.a_offset, cublas_param.lda,
			      (D*)B_d + cublas_param.b_offset, cublas_param.ldb, &beta,
			      (D*)C_d + cublas_param.c_offset, cublas_param.ldc);
	  
	}
	
	//flops += batch*FLOPS_CGETRF(n,n);
	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in cuBLASDGEMMBatched), error code = %d\n", error);
	
      } else if (cublas_param.data_type == QUDA_CUBLAS_DATATYPE_S) {
	
	typedef float S;

	const S alpha = (S)creal(cublas_param.alpha);	
	const S beta  = (S)creal(cublas_param.beta);

	hipblasStatus_t error;
	if(batch > 1) {
	  S **A_ptr_array = static_cast<S**>(pool_device_malloc(batch*sizeof(S*)));
	  S **B_ptr_array = static_cast<S**>(pool_device_malloc(batch*sizeof(S*)));
	  S **C_ptr_array = static_cast<S**>(pool_device_malloc(batch*sizeof(S*)));
	  
	  set_pointer_gemm<S><<<batch,1>>>(A_ptr_array, (S*)A_d, A_size, B_ptr_array, (S*)B_d, B_size, C_ptr_array, (S*)C_d, C_size);
	  
	  error = hipblasSgemmBatched(handle, trans_a, trans_b, cublas_param.m,
				     cublas_param.n, cublas_param.k, &alpha,
				     A_ptr_array + cublas_param.a_offset, cublas_param.lda,
				     B_ptr_array + cublas_param.b_offset, cublas_param.ldb, &beta,
				     C_ptr_array + cublas_param.c_offset, cublas_param.ldc, batch);

	  pool_device_free(A_ptr_array);
	  pool_device_free(B_ptr_array);
	  pool_device_free(C_ptr_array);
	  
	} else {
	  error = hipblasSgemm(handle, trans_a, trans_b, cublas_param.m,
			      cublas_param.n, cublas_param.k, &alpha,
			      (S*)A_d + cublas_param.a_offset, cublas_param.lda,
			      (S*)B_d + cublas_param.b_offset, cublas_param.ldb, &beta,
			      (S*)C_d + cublas_param.c_offset, cublas_param.ldc);
	  
	}
	
	//flops += batch*FLOPS_CGETRF(n,n);
	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in cuBLASSGEMMBatched), error code = %d\n", error);	
	
      } else {
	errorQuda("cublasGEMM type %d not implemented\n", cublas_param.data_type);  	
      }

      if (location == QUDA_CPU_FIELD_LOCATION) {
	qudaMemcpy(C_data, C_d, sizeCarr, hipMemcpyDeviceToHost);
	pool_device_free(A_d);
	pool_device_free(B_d);
	pool_device_free(C_d);
      }

      qudaDeviceSynchronize();
      gettimeofday(&stop, NULL);
      long ds = stop.tv_sec - start.tv_sec;
      long dus = stop.tv_usec - start.tv_usec;
      double time = ds + 0.000001*dus;
      if (getVerbosity() >= QUDA_VERBOSE)
	printfQuda("Batched matrix GEMM completed in %f seconds with GFLOPS = %f\n", time, 1e-9 * flops / time);
      
#endif // CUBLAS_LIB
      
      return flops;
    }
    
  } // namespace cublas

} // namespace quda
