#include <gauge_field_order.h>
#include <extract_gauge_ghost_helper.cuh>

namespace quda {

  using namespace gauge;

  /** This is the template driver for extractGhost */
  template <typename Float, int Nc>
    void extractGhostMG(const GaugeField &u, Float **Ghost) {

    const int length = 2*Nc*Nc;

    QudaFieldLocation location = 
      (typeid(u)==typeid(cudaGaugeField)) ? QUDA_CUDA_FIELD_LOCATION : QUDA_CPU_FIELD_LOCATION;

    if (u.isNative()) {
      if (u.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type G;
	extractGhost<Float,length>(G(u, 0, Ghost), u, location);
      }
    } else if (u.Order() == QUDA_QDP_GAUGE_ORDER) {
      
#ifdef BUILD_QDP_INTERFACE
      extractGhost<Float,length>(QDPOrder<Float,length>(u, 0, Ghost), u, location);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", u.Order());
    }
  }


  /** This is the template driver for extractGhost */
  template <typename Float>
    void extractGhostMG(const GaugeField &u, Float **Ghost) {

    if (u.Reconstruct() != QUDA_RECONSTRUCT_NO) 
      errorQuda("Reconstruct %d not supported", u.Reconstruct());

    if (u.LinkType() != QUDA_COARSE_LINKS)
      errorQuda("Link type %d not supported", u.LinkType());

    if (u.Ncolor() == 4) {
      extractGhostMG<Float, 4>(u, Ghost);
    } else if (u.Ncolor() == 32) {
      extractGhostMG<Float, 32>(u, Ghost);
    } else if (u.Ncolor() == 48) {
      extractGhostMG<Float, 48>(u, Ghost);
    } else if (u.Ncolor() == 96) {
      extractGhostMG<Float, 96>(u, Ghost);
    } else {
      errorQuda("Ncolor = %d not supported", u.Ncolor());
    }
  }

  void extractGaugeGhostMG(const GaugeField &u, void **ghost) {

    if (u.Precision() == QUDA_DOUBLE_PRECISION) {
      extractGhostMG(u, (double**)ghost);
    } else if (u.Precision() == QUDA_SINGLE_PRECISION) {
      extractGhostMG(u, (float**)ghost);
    } else {
      errorQuda("Unknown precision type %d", u.Precision());
    }
  }

} // namespace quda
