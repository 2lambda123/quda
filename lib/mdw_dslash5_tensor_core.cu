#include "hip/hip_runtime.h"
#include <mdw_dslash5_tensor_core.cuh>

namespace quda {

#if defined (GPU_DOMAIN_WALL_DIRAC) && (__COMPUTE_CAPABILITY__ >= 700)
  /**
    @brief Structure containing zMobius / Zolotarev coefficients

    FIXME
    - fix flops counters
    - use kappa notation and not b/c for consistency with other codes and sanity
  */
  template <typename real>
    struct coeff_5 {
      complex<real> a[QUDA_MAX_DWF_LS]; // xpay coefficients
      complex<real> b[QUDA_MAX_DWF_LS];
      complex<real> c[QUDA_MAX_DWF_LS];
    };

  constexpr int size = 4096;
  static __constant__ char mobius_d[size]; // constant buffer used for Mobius coefficients for GPU kernel
  static char mobius_h[size];              // constant buffer used for Mobius coefficients for CPU kernel

  /**
    @brief Helper function for grabbing the constant struct, whether
    we are on the GPU or CPU.
   */
  template <typename real>
    inline __device__ __host__ const coeff_5<real>* coeff() {
#ifdef __CUDA_ARCH__
      return reinterpret_cast<const coeff_5<real>*>(mobius_d);
#else
      return reinterpret_cast<const coeff_5<real>*>(mobius_h);
#endif
    }

  template <typename real, Dslash5Type, typename Arg> struct coeff_type {
    typedef real type;
    const Arg &arg;
    __device__ __host__ coeff_type(const Arg &arg) : arg(arg) { }
    __device__ __host__ real a(int s) { return arg.a; }
    __device__ __host__ real b(int s) { return arg.b; }
    __device__ __host__ real c(int s) { return arg.c; }
  };

  template <typename real, typename Arg> struct coeff_type<real,M5_INV_ZMOBIUS,Arg> {
    typedef complex<real> type;
    __device__ __host__ coeff_type(const Arg &arg) { }
    __device__ __host__ complex<real> a(int s) { return coeff<real>()->a[s]; }
    __device__ __host__ complex<real> b(int s) { return coeff<real>()->b[s]; }
    __device__ __host__ complex<real> c(int s) { return coeff<real>()->c[s]; }
  };
  
  /**
    @brief Parameter structure for applying the Dslash
  */
  template<int Ls_>
  struct Dslash5TensorCoreArg {
    typedef typename colorspinor_mapper<short, 4, 3>::type F;
    typedef typename mapper<short>::type real;

    F out;                  // output vector field
    const F in;             // input vector field
    const F x;              // auxiliary input vector field
    const int nParity;      // number of parities we're working on
    const int volume_cb;    // checkerboarded volume
    const int volume_4d_cb; // 4-d checkerboarded volume
    const int_fastdiv Ls;   // length of 5th dimension

    const real m_f;         // fermion mass parameter
    const real m_5;         // Wilson mass shift

    const bool dagger;      // dagger
    const bool xpay;        // whether we are doing xpay or not

    real b;                 // real constant Mobius coefficient
    real c;                 // real constant Mobius coefficient
    real a;                 // real xpay coefficient

    Dslash5Type type;

    Dslash5TensorCoreArg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
        double m_f, double m_5, const Complex *b_5_, const Complex *c_5_, double a_, bool dagger, Dslash5Type type)
      : out(out), in(in), x(x), nParity(in.SiteSubset()),
      volume_cb(in.VolumeCB()), volume_4d_cb(volume_cb/Ls_), Ls(Ls_),
      m_f(m_f), m_5(m_5), a(a_), dagger(dagger), xpay(a_ == 0.0 ? false : true), type(type)
    {
      if(in.Nspin() != 4){
        errorQuda("nSpin = %d not support", in.Nspin());
      }
      
      if (!in.isNative() || !out.isNative()) errorQuda("Unsupported field order out=%d in=%d\n", out.FieldOrder(), in.FieldOrder());

      if (sizeof(coeff_5<real>) > size) errorQuda("Coefficient buffer too large at %lu bytes\n", sizeof(coeff_5<real>));
      coeff_5<real> *coeff = reinterpret_cast<coeff_5<real>*>(&mobius_h);
      auto *a_5 =  coeff->a;
      auto *b_5 =  coeff->b;
      auto *c_5 =  coeff->c;

      switch(type){
        case M5_INV_MOBIUS:
          b = -(c_5_[0].real() * (4.0 + m_5) - 1.0) / (b_5_[0].real() * (4.0 + m_5) + 1.0);
          c = 0.5 / ( 1.0 + std::pow(b,(int)Ls) * m_f );
          a *= pow(0.5 / (b_5_[0].real() * (m_5 + 4.0) + 1.0), 2);
          break;
        default:
          errorQuda("Unknown Dslash5Type %d", type);
      }

      hipMemcpyToSymbolAsync(HIP_SYMBOL(mobius_d), mobius_h, sizeof(coeff_5<real>), 0, hipMemcpyHostToDevice, streams[Nstream-1]);

    }
  };

// The following two are the actual kernels. Since there is no "static_if" to use two version 
// are implemented explicitly.
// TODO: Maybe someone smart people could have a better idea? Or c++49 will have a "static_if"?

  /**
    @brief Tensor core kernel for applying the M5inv operator: reload version
  */
  template<int block_dim_x, int Ls_, bool dagger, bool xpay, class Arg>
  __global__ void dslash5inv_tensor_core_reload(Arg arg)
  {
    float scale;

    TensorCoreSharedMemory<half2> shared_memory_data;
    
    constexpr int M = 4*Ls_;
    constexpr int N = 6*block_dim_x;
    
    constexpr int sm_m_pad_size = 0;
    constexpr int sm_n_pad_size = 16;
    
    constexpr int N_sm = N + sm_n_pad_size;
    constexpr int M_sm = M + sm_m_pad_size;
    
    half2* sm_b = shared_memory_data;
    half*  sm_c = (half*)sm_b;
    half*  sm_a = sm_c+M*N_sm;

    { // Construct matrix A
      construct_matrix_a_m5inv<block_dim_x, Ls_, M_sm, dagger, Arg>(arg, sm_a);
    } // Construct matrix A
    
    __syncthreads();
   
    bool idle = false;
    int s4_base = blockIdx.x*blockDim.x; // base.
    int s4, sid;
  
    while(s4_base < arg.volume_4d_cb){
      
      s4 = s4_base + threadIdx.x;
      sid = threadIdx.y*arg.volume_4d_cb + s4;
      
      if (s4 >= arg.volume_4d_cb){
        idle = true;
      }
    
      if(!idle){
        scale = load_matrix_b_tex<N_sm, Arg>(arg, sm_b, sid);
      }
      
      __syncthreads();
    
      { // wmma.h
        wmma_gemm_reload<block_dim_x, Ls_, M, N, M_sm, N_sm>(sm_a, sm_c, sm_c);        
      } // wmma.h
      
      __syncthreads();
    
      if(!idle){
         store_matrix_c<N_sm, Arg>(arg, sm_b, sid, scale);
      }
    
      s4_base += gridDim.x*blockDim.x;
    
    } // while
  }
  
  /**
    @brief Tensor core kernel for applying the M5inv operator: preload version
  */
  template<int block_dim_x, int Ls_, bool dagger, bool xpay, class Arg>
  __global__ void dslash5inv_tensor_core_preload(Arg arg)
  {
    float scale;

    TensorCoreSharedMemory<half2> shared_memory_data;
    
    constexpr int M = 4*Ls_;
    constexpr int N = 6*block_dim_x;
    
    constexpr int sm_m_pad_size = 0;
    constexpr int sm_n_pad_size = 16;
    
    constexpr int N_sm = N + sm_n_pad_size;
    constexpr int M_sm = M + sm_m_pad_size;
    
    half2* sm_b = shared_memory_data;
    half*  sm_c = (half*)sm_b;
    half*  sm_a = sm_c+M*N_sm;

    { // Construct matrix A
      construct_matrix_a_m5inv<block_dim_x, Ls_, M_sm, dagger, Arg>(arg, sm_a);
    } // Construct matrix A
    
    __syncthreads();
   
    bool idle = false;
    int s4_base = blockIdx.x*blockDim.x; // base.
    int s4, sid;
 
    constexpr int WMMA_M = 16;
    constexpr int WMMA_N = 16;
    constexpr int WMMA_K = 16;
    
    constexpr int tm_dim = M/WMMA_M;
    constexpr int tn_dim = N/WMMA_N;
    
    constexpr int total_warp = block_dim_x*Ls_/32;
    const int this_warp = (threadIdx.y*block_dim_x+threadIdx.x)/32;
    
    constexpr int total_tile = tm_dim*tn_dim;
    
    constexpr int warp_cycle = total_tile/total_warp;
    const int warp_m = this_warp*warp_cycle/tn_dim;
     
    typedef typename nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major> a_type;
    a_type a_frag[tm_dim];
    #pragma unroll
    for( int k = 0; k < tm_dim; k++ ){
      const int a_row = warp_m*WMMA_M;
      const int a_col = k*WMMA_K;
      // Load Matrix
      nvcuda::wmma::load_matrix_sync(a_frag[k], sm_a+a_row+a_col*M_sm, M_sm);
    } 
  
    while(s4_base < arg.volume_4d_cb){
      
      s4 = s4_base + threadIdx.x;
      sid = threadIdx.y*arg.volume_4d_cb + s4;
      
      if (s4 >= arg.volume_4d_cb){
        idle = true;
      }
    
      if(!idle){
        scale = load_matrix_b_tex<N_sm, Arg>(arg, sm_b, sid);
      }
      
      __syncthreads();
    
      { // wmma.h
        wmma_gemm_preload<block_dim_x, Ls_, M, N, M_sm, N_sm>(a_frag, sm_c, sm_c);        
      } // wmma.h
      
      __syncthreads();
    
      if(!idle){
        store_matrix_c<N_sm, Arg>(arg, sm_b, sid, scale);
      }
    
      s4_base += gridDim.x*blockDim.x;
    
    } // while
  }
 
  template<int Ls_, class Arg>
  class Dslash5TensorCore : public TunableVectorYZ {

    protected:
      Arg &arg;
      const ColorSpinorField &meta;
      static constexpr bool shared = true; // whether to use shared memory cache blocking for M5inv

      /** Whether to use variable or fixed coefficient algorithm.  Must be true if using ZMOBIUS */
      static constexpr bool var_inverse = true;

      long long flops() const {
        long long Ls = Ls_;
        long long bulk = (Ls-2)*(meta.Volume()/Ls);
        long long wall = 2*meta.Volume()/Ls;
        long long n = meta.Ncolor() * meta.Nspin();

        long long flops_ = 0;
        switch (arg.type) {
          case M5_INV_MOBIUS: // FIXME flops
            //flops_ = ((2 + 8 * n) * Ls + (arg.xpay ? 4ll : 0)) * meta.Volume();
            flops_ = (144 * Ls + (arg.xpay ? 4ll : 0)) * meta.Volume();
            break;
          default:
            errorQuda("Unknown Dslash5Type %d", arg.type);
        }

        return flops_;
      }

      long long bytes() const {
        // long long Ls = meta.X(4);
        switch (arg.type) {
          case M5_INV_MOBIUS:
            return arg.out.Bytes() + arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
          default: 
            errorQuda("Unknown Dslash5Type %d", arg.type);
        }
        return 0ll;
      }

      virtual bool tuneGridDim() const { return true; }
      virtual bool tuneAuxDim() const { return true; }
      virtual bool tuneSharedBytes() const { return true; }
      unsigned int minThreads() const { return arg.volume_4d_cb; }
  
      unsigned int shared_bytes_per_block(int x, int y) const { 
        // (Ls*4) by (Ls*4), (Ls*4) by (volume_4d*6 + 16)
        return ( (y*4)*(y*4+0)+(y*4)*(x*6+16) )*2; // 4*4*2 TODO: fix this!
      }
   
      virtual bool advanceBlockDim(TuneParam &param) const
      {
        if( param.block.x < max_block_size() ){
          param.block.x += step_block_size();
          param.shared_bytes = shared_bytes_per_block(param.block.x, param.block.y); 
          return true;
        }else{
          return false;
        }
      }
      
      virtual bool advanceGridDim(TuneParam &param) const
      {
        const unsigned int max_blocks = maxGridSize();
        const int step = deviceProp.multiProcessorCount;
        param.grid.x += step;
        if (param.grid.x > max_blocks) {
          return false;
        } else {
          param.block.x = min_block_size();
          param.shared_bytes = shared_bytes_per_block(param.block.x, param.block.y); 
          return true;
        }
      }
      
      virtual bool advanceAux(TuneParam &param) const
      {
        if (param.aux.x < 1) {
          param.aux.x++;
          // We have updated the "aux" so reset all other parameters. 
          param.grid.x = minGridSize();
          param.block.x = min_block_size();
          param.shared_bytes = shared_bytes_per_block(param.block.x, param.block.y); 
          return true;
        } else {
          param.aux.x = 0;
          return false;
        }
      }

      virtual unsigned int maxGridSize() const { return 32*deviceProp.multiProcessorCount; }
      virtual unsigned int minGridSize() const { return deviceProp.multiProcessorCount; }
      unsigned int min_block_size() const { return  8; }
      unsigned int max_block_size() const { return 64; }
      unsigned int step_block_size() const { return  8; }

      // overloaded to return max dynamic shared memory if doing shared-memory inverse
      unsigned int maxSharedBytesPerBlock() const {
        if (shared && (arg.type == M5_INV_DWF || arg.type == M5_INV_MOBIUS || arg.type == M5_INV_ZMOBIUS) ) {
          return maxDynamicSharedBytesPerBlock();
        } else {
          return TunableVectorYZ::maxSharedBytesPerBlock();
        }
      }

    public:
      Dslash5TensorCore(Arg &arg, const ColorSpinorField &meta)
        : TunableVectorYZ(arg.Ls, arg.nParity), arg(arg), meta(meta)
      {
        strcpy(aux, meta.AuxString());
        if (arg.dagger) strcat(aux, ",Dagger");
        if (arg.xpay) strcat(aux,",xpay");
        switch (arg.type) {
          case M5_INV_MOBIUS:
            strcat(aux, ",m5inv_mobius_tensor_core");
            break;
          default: 
            errorQuda("Unknown Dslash5Type %d", arg.type);
        }
      }
      virtual ~Dslash5TensorCore() { }

      template<typename T>
      inline void launch(T *f, const TuneParam &tp, Arg &arg, const hipStream_t &stream) {
        // static bool init = false;
        if ( shared ) {
          // if inverse kernel uses shared memory then maximize total shared memory pool
          setMaxDynamicSharedBytesPerBlock(f);
          // set_shared_memory_on_volta((const void*)f, "Some Function");
          // init = true;
        }
        void *args[] = { &arg };
        qudaLaunchKernel((const void *)f, tp.grid, tp.block, args, tp.shared_bytes, stream);
      }

      void apply(const hipStream_t &stream) {
        // By its name we ONLY have a GPU version
        // TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        TuneParam tp = tuneLaunch(*this, getTuning(), QUDA_DEBUG_VERBOSE);
        if(tp.aux.x == 0){ // preload, NO reload
          switch(arg.type){
            case M5_INV_MOBIUS:
              switch(tp.block.x){
                case  8:
                  if (arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload< 8, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload< 8, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{          
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload< 8, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload< 8, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 16:
                  if (arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<16, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<16, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{          
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<16, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<16, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 24:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<24, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<24, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<24, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<24, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 32:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<32, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<32, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<32, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<32, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 40:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<40, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<40, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<40, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<40, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 48:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<48, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<48, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<48, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<48, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 56:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<56, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<56, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<56, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<56, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 64:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<64, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<64, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_preload<64, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_preload<64, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                default:
                  errorQuda("NOT valid blockDim.x(=%d)\n", tp.block.x);
              }
              break;
            default: 
              errorQuda("Unknown Dslash5Type %d", arg.type);
          }
        }else{ // tp.aux.x
          // RELOAD
          switch(arg.type){
            case M5_INV_MOBIUS:
              switch(tp.block.x){
                case  8:
                  if (arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload< 8, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload< 8, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{          
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload< 8, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload< 8, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 16:
                  if (arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<16, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<16, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{          
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<16, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<16, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 24:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<24, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<24, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<24, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<24, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 32:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<32, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<32, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<32, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<32, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 40:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<40, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<40, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<40, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<40, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 48:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<48, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<48, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<48, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<48, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 56:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<56, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<56, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<56, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<56, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                case 64:
                  if(arg.xpay){ 
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<64, Ls_, true, true, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<64, Ls_,false, true, Arg>, tp, arg, stream) ;
                  }else{
                    arg.dagger ?
                      launch(dslash5inv_tensor_core_reload<64, Ls_, true,false, Arg>, tp, arg, stream) :
                      launch(dslash5inv_tensor_core_reload<64, Ls_,false,false, Arg>, tp, arg, stream) ;
                  }
                  break;
                default:
                  errorQuda("NOT valid blockDim.x(=%d)\n", tp.block.x);
              }
              break;
            default: 
              errorQuda("Unknown Dslash5Type %d", arg.type);
          }
        } // tp.aux.x
      }

      void initTuneParam(TuneParam &param) const {
        TunableVectorYZ::initTuneParam(param);
        param.block = dim3(min_block_size(), arg.Ls, 1); // Ls must be contained in the block
        param.grid = dim3(minGridSize(), 1, 1);
        param.shared_bytes = shared_bytes_per_block(param.block.x, param.block.y); 
        param.aux.x = 0;
      }

      void defaultTuneParam(TuneParam &param) const {
        initTuneParam(param);
      }

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };

#endif // defined (GPU_DOMAIN_WALL_DIRAC) && (__COMPUTE_CAPABILITY__ >= 700)
  
  // Apply the 5th dimension dslash operator to a colorspinor field
  // out = Dslash5 * in
  
  void apply_dslash5_tensor_core(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
      double m_f, double m_5, const Complex* b_5, const Complex* c_5, double a, bool dagger, Dslash5Type type)
  {
#if defined (GPU_DOMAIN_WALL_DIRAC) && (__COMPUTE_CAPABILITY__ >= 700)
    if (in.DWFPCtype() != QUDA_4D_PC) errorQuda("ONLY 4D preconditioned fields are supported");
    checkLocation(out, in);     // check all locations match
  
    if( checkPrecision(out, in) == QUDA_HALF_PRECISION && in.Ncolor() == 3){
      // switch for Ls
      switch(in.X(4)){
        case  8:
          {
            Dslash5TensorCoreArg< 8> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
            Dslash5TensorCore<8, Dslash5TensorCoreArg<8> > dslash(arg, in);
            dslash.apply(streams[Nstream-1]);
          }
        break;
        case 12:
          {
            Dslash5TensorCoreArg<12> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
            Dslash5TensorCore<12, Dslash5TensorCoreArg<12> > dslash(arg, in);
            dslash.apply(streams[Nstream-1]);
          }
        break;
        case 16:
          {
            Dslash5TensorCoreArg<16> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
            Dslash5TensorCore<16, Dslash5TensorCoreArg<16> > dslash(arg, in);
            dslash.apply(streams[Nstream-1]);
          }
        break;
        case 20:
          {
            Dslash5TensorCoreArg<20> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
            Dslash5TensorCore<20, Dslash5TensorCoreArg<20> > dslash(arg, in);
            dslash.apply(streams[Nstream-1]);
          }
        break;
        case 24:
          {
            Dslash5TensorCoreArg<24> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
            Dslash5TensorCore<24, Dslash5TensorCoreArg<24> > dslash(arg, in);
            dslash.apply(streams[Nstream-1]);
          }
        break;
        default: 
          errorQuda("Ls = %d is NOT supported.\n", in.X(4));
      }
    }else{
      errorQuda("Tensor core implemtation ONLY supports HALF precision and n_color = 3.\n");
    }
#else
    errorQuda("Domain wall dslash WITH tensor cores has not been built");
#endif
  }

} // namespace quda

