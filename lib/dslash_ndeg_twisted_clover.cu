#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <clover_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_ndeg_twisted_clover.cuh>

/**
   This is the gauged non-degenerate twisted-clover operator acting on a 
   quark doublet.
*/

namespace quda
{

  template <typename Arg> class NdegTwistedClover : public Dslash<nDegTwistedClover, Arg>
    {
      using Dslash = Dslash<nDegTwistedClover, Arg>;
      using Dslash::arg;
      using Dslash::in;
      
    public:
    NdegTwistedClover(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) : Dslash(arg, out, in)
        {
          TunableVectorYZ::resizeVector(2, arg.nParity);
        }
      
      void apply(const qudaStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        Dslash::setParam(tp);
        if (arg.xpay)
          Dslash::template instantiate<packShmem, true>(tp, stream);
        else
          errorQuda("Non-degenerate twisted-clover operator only defined for xpay=true");
      }
      
      long long flops() const
      {
        int clover_flops = 504;
        long long flops = Dslash::flops();
        switch (arg.kernel_type) {
        case INTERIOR_KERNEL:
        case KERNEL_POLICY:
          // b and c multiply (= 2 * 48 * in.Volume())
          flops += 2 * in.Ncolor() * 4 * 4 * in.Volume(); // complex * Nc * Ns * fma * vol
          flops += clover_flops * in.Volume();
          break;
        default: break; // twisted-mass flops are in the interior kernel
        }
        return flops;
      }
      long long bytes() const
      {
        int clover_bytes = 72 * in.Precision() + (isFixed<typename Arg::Float>::value ? 2 * sizeof(float) : 0);
        
        long long bytes = Dslash::bytes();
        switch (arg.kernel_type) {
        case INTERIOR_KERNEL:
        case KERNEL_POLICY: bytes += clover_bytes * in.Volume(); break;
        default: break;
        }
        
        return bytes;
      }
    };
  
  template <typename Float, int nColor, QudaReconstructType recon> struct NdegTwistedCloverApply {
    
    inline NdegTwistedCloverApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
                                  const CloverField &C, double a,
                                  double b, double c, const ColorSpinorField &x, int parity, bool dagger,
                                  const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      NdegTwistedCloverArg<Float, nColor, nDim, recon> arg(out, in, U, C, a, b, c, x, parity, dagger, comm_override);
      NdegTwistedClover<decltype(arg)> twisted(arg, out, in);
      // why not in.VolumeCB() and in. GhostFaceCB() ??
      dslash::DslashPolicyTune<decltype(twisted)> policy(
        twisted, const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)),
        in.getDslashConstant().volume_4d_cb, in.getDslashConstant().ghostFaceCB, profile);
      policy.apply(0);
    }
  };
  
  void ApplyNdegTwistedClover(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const CloverField &C,
                              double a, double b,
                              double c, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override,
                              TimeProfile &profile)
  {
#ifdef GPU_NDEG_TWISTED_CLOVER_DIRAC
    instantiate<NdegTwistedCloverApply>(out, in, U, C, a, b, c, x, parity, dagger, comm_override, profile);
#else
    errorQuda("Non-degenerate twisted-clover dslash has not been built");
#endif // GPU_NDEG_TWISTED_CLOVER_DIRAC
  }
  
} // namespace quda
