#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <dslash_helper.cuh>
#include <index_helper.cuh>
#include <stencil.h>
#include <color_spinor.h>
#include <worker.h>
#include <tune_quda.h>

namespace quda {
#include <dslash_events.cuh>
#include <dslash_policy.cuh>
}

/**
   This is a staggered Dirac operator
*/

namespace quda {


//MWTODO: This should be merged in the generic ghostFaceIndex function

 /**
     Compute the checkerboarded index into the ghost field
     corresponding to full (local) site index x[]
     @param x local site
     @param X local lattice dimensions
     @param dim dimension
     @param depth of ghost
  */
  template <int dir, typename I>
  __device__ __host__ inline int mwghostFaceIndex(const int x[], const I X[], int dim, int nFace) {
    int index = 0;
    switch(dim) {
    case 0:
      switch(dir) {
      case 0:
 index = ((x[0]+nFace-1)*X[4]*X[3]*X[2]*X[1] + x[4]*X[3]*X[2]*X[1] + x[3]*(X[2]*X[1])+x[2]*X[1] + x[1])>>1;
  break;
      case 1:
  index = ((x[0]-X[0]+nFace)*X[4]*X[3]*X[2]*X[1] + x[4]*X[3]*X[2]*X[1] + x[3]*(X[2]*X[1]) + x[2]*X[1] + x[1])>>1;
  break;
      }
      break;
    case 1:
      switch(dir) {
      case 0:
  index = ((x[1]+nFace-1)*X[4]*X[3]*X[2]*X[0] + x[4]*X[3]*X[2]*X[0] + x[3]*X[2]*X[0]+x[2]*X[0]+x[0])>>1;
  break;
      case 1:
  index = ((x[1]-X[1]+nFace)*X[4]*X[3]*X[2]*X[0] +x[4]*X[3]*X[2]*X[0]+ x[3]*X[2]*X[0] + x[2]*X[0] + x[0])>>1;
  break;
      }
      break;
    case 2:
      switch(dir) {
      case 0:
  index = ((x[2]+nFace-1)*X[4]*X[3]*X[1]*X[0] + x[4]*X[3]*X[1]*X[0] + x[3]*X[1]*X[0]+x[1]*X[0]+x[0])>>1;
  break;
      case 1:
  index = ((x[2]-X[2]+nFace)*X[4]*X[3]*X[1]*X[0] + x[4]*X[3]*X[1]*X[0] + x[3]*X[1]*X[0] + x[1]*X[0] + x[0])>>1;
  break;
      }
      break;
    case 3:
      switch(dir) {
      case 0:
  index = ((x[3]+nFace-1)*X[4]*X[2]*X[1]*X[0] + x[4]*X[2]*X[1]*X[0] + x[2]*X[1]*X[0]+x[1]*X[0]+x[0])>>1;
  break;
      case 1:
  index  = ((x[3]-X[3]+nFace)*X[4]*X[2]*X[1]*X[0] + x[4]*X[2]*X[1]*X[0] + x[2]*X[1]*X[0]+x[1]*X[0] + x[0])>>1;
  break;
      }
      break;
    }
    return index;
  }

  /**
     @brief Parameter structure for driving the Staggered Dslash operator
   */
  template <typename Float, int nColor, QudaReconstructType reconstruct_u_, QudaReconstructType reconstruct_l_, bool improved_>
  struct StaggeredArg : DslashArg<Float> {
    static constexpr int nSpin = 1;
    static constexpr bool spin_project = false;
    static constexpr bool spinor_direct_load = false; // false means texture load
    using F = typename colorspinor_mapper<Float,nSpin,nColor,spin_project,spinor_direct_load>::type;

    static constexpr QudaReconstructType reconstruct_u = reconstruct_u_;
    static constexpr QudaReconstructType reconstruct_l = reconstruct_l_;
    //TODO: recon 9/13 seems to break with gauge_direct_load = false
    static constexpr bool gauge_direct_load = true; // false means texture load
    static constexpr QudaGhostExchange ghost = QUDA_GHOST_EXCHANGE_PAD;
    using GU = typename gauge_mapper<Float,reconstruct_u,18,QUDA_STAGGERED_PHASE_MILC,gauge_direct_load,ghost>::type;
    using GL = typename gauge_mapper<Float,reconstruct_l,18,QUDA_STAGGERED_PHASE_NO,gauge_direct_load,ghost>::type;

    F out;                // output vector field
    const F in;           // input vector field
    const F x;            // input vector when doing xpay
    const GU U;            // the gauge field
    const GL L;            // the long gauge field

    const Float a;
    static constexpr bool improved = improved_;
    StaggeredArg(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const GaugeField &L,
        Float a, const ColorSpinorField &x, int parity, bool dagger,  const int *comm_override)
      : DslashArg<Float>(in, U, 0.0, parity, dagger, a == 0.0 ? false : true, improved_ ? 3 : 1, comm_override), out(out), in(in, improved_ ? 3 :1 ), U(U), L(L), x(x), a(a) 
    {
      if (!out.isNative() || !x.isNative() || !in.isNative() || !U.isNative())
        errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", in.FieldOrder(), U.FieldOrder());
    }
  };


/**
     Applies the off-diagonal part of the Laplace operator

     @param[out] out The out result field
     @param[in] U The gauge field
     @param[in] kappa Kappa value
     @param[in] in The input field
     @param[in] parity The site parity
     @param[in] x_cb The checkerboarded site index
   */
  template <typename Float, int nDim, int nColor, int nParity, bool dagger, KernelType kernel_type, typename Arg, typename Vector>
     __device__ __host__ inline void applyStaggered(Vector &out, Arg &arg, int coord[nDim], int x_cb,
      int parity, int idx, int thread_dim, bool &active) {
      typedef typename mapper<Float>::type real;
      typedef Matrix<complex<real>,nColor> Link;
      const int their_spinor_parity = (arg.nParity == 2) ? 1-parity : 0;


#ifdef XONLY
#pragma unroll
    for (int d = 0; d<1; d++) {// loop over dimension{
#else
#pragma unroll
    for (int d = 0; d<4; d++) {// loop over dimension{
#endif 

      //Forward gather - compute fwd offset for vector fetch
#ifndef XONLY2
      
      // standard - forward direction
      {
        const bool ghost = (coord[d] + 1 >= arg.dim[d]) && isActive<kernel_type>(active, thread_dim, d, coord, arg);
        if ( doHalo<kernel_type>(d) && ghost) {
          const int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, 1);
          const Link U = arg.U(d, x_cb, parity);
          const Vector in = arg.in.Ghost(d, 1, ghost_idx, their_spinor_parity);
          out += U * in;

          // printf("in %f %f %f %f %f %f\n",in.data[0].real(),in.data[0].imag(),in.data[1].real(),in.data[1].imag(),out.data[2].real(),out.data[2].imag());
        }
        else if ( doBulk<kernel_type>() && !ghost ) {
          const int fwd_idx = linkIndexP1(coord, arg.dim, d);
          const Link U = arg.U(d, x_cb, parity);
          const Vector in = arg.in(fwd_idx, their_spinor_parity);
          out += U * in;
           // printf("in %f %f %f %f %f %f\n",in.data[0].real(),in.data[0].imag(),in.data[1].real(),in.data[1].imag(),out.data[2].real(),out.data[2].imag());

        }
      }


      // improved - forward direction
      if(arg.improved){
        const bool ghost = (coord[d] + 3 >= arg.dim[d]) && isActive<kernel_type>(active, thread_dim, d, coord, arg);
        const int fwd3_idx = linkIndexP3(coord, arg.dim, d);
        if ( doHalo<kernel_type>(d) && ghost) {
          const int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);
          const Link L = arg.L(d, x_cb, parity);
          const Vector in = arg.in.Ghost(d, 1, ghost_idx, their_spinor_parity);
          out += L * in;
        // printf("Halo (%i %i %i %i), idx %i x_cb %i ghost %i nbr_idx1 %i \t in %f %f %f %f %f %f\n",coord[0],coord[1],coord[2],coord[3], idx, x_cb, ghost_idx, nbr_idx3,in.data[1].real(),in.data[1].imag(),in2.data[1].real(),in2.data[1].imag(),out.data[1].real(),out.data[1].imag());
        } 
        else if ( doBulk<kernel_type>() && !ghost ) {
          const Link L = arg.L(d, x_cb, parity);
          const Vector in = arg.in(fwd3_idx, their_spinor_parity);
          out += L * in;
        }  
      }
#endif
// #endif
#ifndef XONLY2
      {
      //Backward gather - compute back offset for spinor and gauge fetch

        const bool ghost = (coord[d] - 1 < 0) && isActive<kernel_type>(active, thread_dim, d, coord, arg);

        if ( doHalo<kernel_type>(d) && ghost) {
        // MW - check indexing into GhostFace here
          const int ghost_idx2 = mwghostFaceIndex<0>(coord, arg.dim, d, 1);
          const int ghost_idx = arg.improved ? mwghostFaceIndex<0>(coord, arg.dim, d, 3) : ghost_idx2; 
          const Link U = arg.U.Ghost(d, ghost_idx2, 1-parity);
          const Vector in = arg.in.Ghost(d, 0, ghost_idx, their_spinor_parity);
          out -= conj(U) * in;
        }
        else  if ( doBulk<kernel_type>() && !ghost ) {
          const int back_idx = linkIndexM1(coord, arg.dim, d);
          const int gauge_idx = back_idx;
          const Link U = arg.U(d, gauge_idx, 1-parity);
          const Vector in = arg.in(back_idx, their_spinor_parity);
          out -= conj(U) * in;
        }
      }
#endif
      // #ifndef XONLY
      if(arg.improved){
        //Backward gather - compute back offset for spinor and gauge fetch
        
        const bool ghost = (coord[d] - 3 < 0) && isActive<kernel_type>(active, thread_dim, d, coord, arg);

        if ( doHalo<kernel_type>(d) && ghost) {
          const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
          const Link L = arg.L.Ghost(d, ghost_idx, 1-parity);
          const Vector in = arg.in.Ghost(d, 0, ghost_idx, their_spinor_parity);
          out -= conj(L) * in;
        }
        else if ( doBulk<kernel_type>() && !ghost ) {
          const int back3_idx = linkIndexM3(coord, arg.dim, d);
          const int gauge_idx = back3_idx;
          const Link L = arg.L(d, gauge_idx, 1-parity);
          const Vector in = arg.in(back3_idx, their_spinor_parity);
          out -= conj(L) * in;
        }
      }
// #endif
    } //nDim

  }

  //out(x) = M*in = (-D + m) * in(x-mu)
 template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  __device__ __host__ inline void staggered(Arg &arg, int idx, int parity)
  {
    using real = typename mapper<Float>::type;
    using Vector = ColorSpinor<real,nColor,1>;

    bool active = kernel_type == EXTERIOR_KERNEL_ALL ? false : true; // is thread active (non-trival for fused kernel only)
    int thread_dim; // which dimension is thread working on (fused kernel only)
    int coord[nDim];
    int x_cb = arg. improved ? getCoords<nDim,QUDA_4D_PC,kernel_type, Arg, 3>(coord, arg, idx, parity, thread_dim) : getCoords<nDim,QUDA_4D_PC,kernel_type, Arg, 1>(coord, arg, idx, parity, thread_dim);
    // coord[4] = 0;
    //MWTODO -> coord[4]
    
    const int my_spinor_parity = nParity == 2 ? parity : 0;

    Vector out;

    applyStaggered<Float,nDim,nColor,nParity,dagger,kernel_type>(out, arg, coord, x_cb, parity, idx, thread_dim, active);

// printf("NEW1 Out cb %i %i \t %f %f %f %f %f %f\n",x_cb, my_spinor_parity, out.data[0].real(),out.data[0].imag(),out.data[1].real(),out.data[1].imag(),out.data[2].real(),out.data[2].imag());
   
    //MWTODO: clean up here
    // if (xpay) {
    //   Vector x = arg.x(x_cb, my_spinor_parity);
    //   out = arg.a * x -out ;
    // }
    if (dagger){
      out = real(-1)*out;
    }


    if (xpay && kernel_type == INTERIOR_KERNEL) {
      Vector x = arg.x(x_cb, my_spinor_parity);
      out = arg.a * x -out ;
    } else if (kernel_type != INTERIOR_KERNEL ) {
      Vector x = arg.out(x_cb, my_spinor_parity);
      out = x +  ( xpay ? real(-1)*out : out ); //MWTODO: verify
      //MWTODO - aadd xpay
    }
   // printf("NEW2 Out cb %i %i \t %f %f %f %f %f %f\n",x_cb, my_spinor_parity, out.data[0].real(),out.data[0].imag(),out.data[1].real(),out.data[1].imag(),out.data[2].real(),out.data[2].imag());
    if (kernel_type != EXTERIOR_KERNEL_ALL || active) arg.out(x_cb, my_spinor_parity) = out;
  }



 // GPU Kernel for applying the staggered operator to a vector
  template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  __global__ void staggeredGPU(Arg arg)
   {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    if (x_cb >= arg.threads) return;

    // for full fields set parity from y thread index else use arg setting
    int parity = nParity == 2 ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;

    switch(parity) {
    case 0: staggered<Float,nDim,nColor,nParity,dagger,xpay,kernel_type>(arg, x_cb, 0); break;
    case 1: staggered<Float,nDim,nColor,nParity,dagger,xpay,kernel_type>(arg, x_cb, 1); break;
    // dslashStaggered<Float,nDim,nColor>(arg, x_cb, parity);
  }
}


  template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  struct StaggeredLaunch {
    template <typename Dslash>
    inline static void launch(Dslash &dslash, TuneParam &tp, Arg &arg, const hipStream_t &stream) {
      dslash.launch(staggeredGPU<Float,nDim,nColor,nParity,dagger,xpay,kernel_type,Arg>, tp, arg, stream);
    }
  };


  template <typename Float, int nDim, int nColor, typename Arg>
  class Staggered : public Dslash<Float> {

  protected:
    Arg &arg;
    const ColorSpinorField &in;

//TODO: fix flop / byte count?

/*
    long long flops() const
    {
      return (2*nDim*(8*nColor*nColor)-2*nColor + (arg.xpay ? 2*2*nColor : 0) )*arg.nParity*(long long)in.VolumeCB();
    }
    long long bytes() const
    {
      return arg.out.Bytes() + 2*nDim*arg.in.Bytes() + arg.nParity*2*nDim*arg.U.Bytes()*in.VolumeCB() +
  (arg.xpay ? arg.x.Bytes() : 0);
    }

    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }
*/
  public:
    Staggered(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in)
    : Dslash<Float>(arg, out, in), arg(arg), in(in) {  }

    virtual ~Staggered() { }

    void apply(const hipStream_t &stream) {
      if (in.Location() == QUDA_CPU_FIELD_LOCATION) {
        // dslashStaggeredCPU<Float,nDim,nColor>(arg);
        errorQuda("Staggered Dslash not implemented on CPU");
      } else {
#if 0
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        dslashStaggeredGPU<Float,nDim,nColor> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
# else
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        Dslash<Float>::setParam(arg);
        if (arg.xpay) Dslash<Float>::template instantiate<StaggeredLaunch,nDim,nColor, true>(tp, arg, stream);
        else          Dslash<Float>::template instantiate<StaggeredLaunch,nDim,nColor,false>(tp, arg, stream);
#endif
      }
    }

  TuneKey tuneKey() const { return TuneKey(in.VolString(), typeid(*this).name(), Dslash<Float>::aux[arg.kernel_type]); }
};


  template <typename Float, int nColor, QudaReconstructType recon_u, QudaReconstructType recon_l, bool improved>
  void ApplyDslashStaggered(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const GaugeField &L,
                            double a, const ColorSpinorField &x, int parity, bool dagger,
                            const int *comm_override, TimeProfile &profile) 
  {
    constexpr int nDim = 4; //MWTODO: this probably should be 5 for mrhs Dslash
    StaggeredArg<Float,nColor,recon_u,recon_l,improved> arg(out, in, U, L, a, x, parity, dagger, comm_override); 
    Staggered<Float,nDim,nColor,decltype(arg) > staggered(arg, out, in);

    DslashPolicyTune<decltype(staggered)> policy(staggered, const_cast<cudaColorSpinorField*>(static_cast<const cudaColorSpinorField*>(&in)),
                                              in.VolumeCB(), in.GhostFaceCB(), profile);
    //TODO: launch policy
    //staggered.apply(0);
    policy.apply(0);

    checkCudaError();
  }

  // template on the gauge reconstruction
  template <typename Float, int nColor>
    void ApplyDslashStaggered(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const GaugeField &L,
                            double a, const ColorSpinorField &x, int parity, bool dagger, bool improved,
                            const int *comm_override, TimeProfile &profile) 
  {
    if(improved){
    if (L.Reconstruct()== QUDA_RECONSTRUCT_NO) {
      ApplyDslashStaggered<Float,nColor,QUDA_RECONSTRUCT_NO, QUDA_RECONSTRUCT_NO, true>(out, in, U, L, a, x, parity, dagger, comm_override, profile);
    } else if (L.Reconstruct()== QUDA_RECONSTRUCT_13) {
      ApplyDslashStaggered<Float,nColor,QUDA_RECONSTRUCT_NO, QUDA_RECONSTRUCT_13, true >(out, in, U, L, a, x, parity, dagger, comm_override, profile);
    } else if (L.Reconstruct()== QUDA_RECONSTRUCT_9) {
      ApplyDslashStaggered<Float,nColor,QUDA_RECONSTRUCT_NO, QUDA_RECONSTRUCT_9, true>(out, in, U, L, a, x, parity, dagger, comm_override, profile);
    } else {
      errorQuda("Unsupported reconstruct type %d\n", U.Reconstruct());
    }
  }
  else{
      if (U.Reconstruct()== QUDA_RECONSTRUCT_NO) {
      ApplyDslashStaggered<Float,nColor,QUDA_RECONSTRUCT_NO, QUDA_RECONSTRUCT_NO,false>(out, in, U, L, a, x, parity, dagger, comm_override, profile);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_12) {
      ApplyDslashStaggered<Float,nColor,QUDA_RECONSTRUCT_12, QUDA_RECONSTRUCT_NO,false>(out, in, U, L, a, x, parity, dagger, comm_override, profile);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_8) {
      errorQuda("Recon 8 not implemented for standard staggered.\n");
       // ApplyDslashStaggered<Float,nColor,QUDA_RECONSTRUCT_8, QUDA_RECONSTRUCT_NO, false>(out, in, U, L, a, x, parity);
    } else {
      errorQuda("Unsupported reconstruct type %d\n", U.Reconstruct());
    }  
  }
  }

  // template on the number of colors
  template <typename Float>
  void ApplyDslashStaggered(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const GaugeField &L,
                            double a, const ColorSpinorField &x, int parity, bool dagger, bool improved,
                            const int *comm_override, TimeProfile &profile) 
  {
    if (in.Ncolor() == 3) {
      ApplyDslashStaggered<Float,3>(out, in, U, L, a, x, parity, dagger, improved, comm_override, profile);
    } else {
      errorQuda("Unsupported number of colors %d\n", U.Ncolor());
    }
  }



  void ApplyDslashStaggered(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const GaugeField &L,
                            double a, const ColorSpinorField &x, int parity, bool dagger, bool improved,
                            const int *comm_override, TimeProfile &profile)      
  {
    
#ifdef GPU_STAGGERED_DIRAC
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());
    
    // check all precisions match
    checkPrecision(out, in, U, L);

    // check all locations match
    checkLocation(out, in, U, L);

    // const int nFace = 1;
    // in.exchangeGhost((QudaParity)(1-parity), nFace, 0); // last parameter is dummy

    if (dslash::aux_worker) dslash::aux_worker->apply(0);
    if (U.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyDslashStaggered<double>(out, in, U, L, a, x, parity, dagger, improved, comm_override, profile);
    } else if (U.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyDslashStaggered<float>(out, in, U, L, a, x, parity, dagger, improved, comm_override, profile);
    } 
    else if (U.Precision() == QUDA_HALF_PRECISION) {
      ApplyDslashStaggered<short>(out, in, U, L, a, x, parity, dagger, improved, comm_override, profile);

    } 
    else {
      errorQuda("Unsupported precision %d\n", U.Precision());
    }  
#else
    errorQuda("Staggered dslash has not been built");
#endif
}

} // namespace quda
