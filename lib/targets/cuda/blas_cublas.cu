
#include <hip/hip_runtime.h>
#ifdef NATIVE_BLAS_LIB
#include <blas_lapack.h>
#include <cublas_v2.h>
#include <malloc_quda.h>

namespace quda {

  namespace blas_lapack { 

      
    static cublasHandle_t handle;

    void init() {
      cublasStatus_t error = cublasCreate(&handle);
      if (error != CUBLAS_STATUS_SUCCESS) errorQuda("cublasCreate failed with error %d", error);
    }

    void destroy() {
      cublasStatus_t error = cublasDestroy(handle);
      if (error != CUBLAS_STATUS_SUCCESS) errorQuda("\nError indestroying cublas context, error code = %d\n", error);
    }

    // mini kernel to set the array of pointers needed for batched cublas
    template<typename T>
    __global__ void set_pointer(T **output_array_a, T *input_a, T **output_array_b, T *input_b, int batch_offset)
    {
      output_array_a[blockIdx.x] = input_a + blockIdx.x * batch_offset;
      output_array_b[blockIdx.x] = input_b + blockIdx.x * batch_offset;
    }

    // FIXME do this in pipelined fashion to reduce memory overhead.
    long long BatchInvertMatrix(void *Ainv, void* A, const int n, const uint64_t batch, QudaPrecision prec, QudaFieldLocation location)
    {
      if (getVerbosity() >= QUDA_SUMMARIZE)
	printfQuda("BatchInvertMatrixNATIVE: Nc = %d, batch = %lu\n", n, batch);
      long long flops = 0;      
      timeval start, stop;
      gettimeofday(&start, NULL);

      size_t size = 2*n*n*prec*batch;
      void *A_d = location == QUDA_CUDA_FIELD_LOCATION ? A : pool_device_malloc(size);
      void *Ainv_d = location == QUDA_CUDA_FIELD_LOCATION ? Ainv : pool_device_malloc(size);
      if (location == QUDA_CPU_FIELD_LOCATION) qudaMemcpy(A_d, A, size, cudaMemcpyHostToDevice);

      int *dipiv = static_cast<int*>(pool_device_malloc(batch*n*sizeof(int)));
      int *dinfo_array = static_cast<int*>(pool_device_malloc(batch*sizeof(int)));
      int *info_array = static_cast<int*>(pool_pinned_malloc(batch*sizeof(int)));
      memset(info_array, '0', batch*sizeof(int)); // silence memcheck warnings

      if (prec == QUDA_SINGLE_PRECISION) {
	typedef cuFloatComplex C;
	C **A_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));
	C **Ainv_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));

	set_pointer<C><<<batch,1>>>(A_array, (C*)A_d, Ainv_array, (C*)Ainv_d, n*n);

	cublasStatus_t error = cublasCgetrfBatched(handle, n, A_array, n, dipiv, dinfo_array, batch);
	flops += batch*FLOPS_CGETRF(n,n);

	if (error != CUBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in LU decomposition (cublasCgetrfBatched), error code = %d\n", error);

	qudaMemcpy(info_array, dinfo_array, batch*sizeof(int), cudaMemcpyDeviceToHost);
	for (uint64_t i=0; i<batch; i++) {
	  if (info_array[i] < 0) {
	    errorQuda("%lu argument had an illegal value or another error occured, such as memory allocation failed", i);
	  } else if (info_array[i] > 0) {
	    errorQuda("%lu factorization completed but the factor U is exactly singular", i);
	  }
	}
    
	error = cublasCgetriBatched(handle, n, (const C**)A_array, n, dipiv, Ainv_array, n, dinfo_array, batch);
	flops += batch*FLOPS_CGETRI(n);

	if (error != CUBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in matrix inversion (cublasCgetriBatched), error code = %d\n", error);

	qudaMemcpy(info_array, dinfo_array, batch*sizeof(int), cudaMemcpyDeviceToHost);

	for (uint64_t i=0; i<batch; i++) {
	  if (info_array[i] < 0) {
	    errorQuda("%lu argument had an illegal value or another error occured, such as memory allocation failed", i);
	  } else if (info_array[i] > 0) {
	    errorQuda("%lu factorization completed but the factor U is exactly singular", i);
	  }
	}

	pool_device_free(Ainv_array);
	pool_device_free(A_array);

      } else {
	errorQuda("%s not implemented for precision=%d", __func__, prec);
      }

      if (location == QUDA_CPU_FIELD_LOCATION) {
	qudaMemcpy(Ainv, Ainv_d, size, cudaMemcpyDeviceToHost);
	pool_device_free(Ainv_d);
	pool_device_free(A_d);
      }

      pool_device_free(dipiv);
      pool_device_free(dinfo_array);
      pool_pinned_free(info_array);

      qudaDeviceSynchronize();
      gettimeofday(&stop, NULL);
      long ds = stop.tv_sec - start.tv_sec;
      long dus = stop.tv_usec - start.tv_usec;
      double time = ds + 0.000001*dus;

      if (getVerbosity() >= QUDA_VERBOSE)
	printfQuda("Batched matrix inversion completed in %f seconds with GFLOPS = %f\n", time, 1e-9 * flops / time);
      
      return flops;
    }
  } // namespace blas_lapack
} // namespace quda
#endif //NATIVE_BLAS_LIB
