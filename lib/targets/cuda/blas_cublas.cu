#include "hip/hip_runtime.h"
#include <blas_lapack.h>
#ifdef NATIVE_LAPACK_LIB
#include <hipblas.h>
#include <malloc_quda.h>
#endif

#include <Eigen/LU>
using namespace Eigen;

#define LOCAL_DEBUG

namespace quda {

  namespace native_lapack { 

    
#ifdef NATIVE_LAPACK_LIB
    static hipblasHandle_t handle;
#endif
    static bool cublas_init = false;
      
    void init() {
      if(!cublas_init) {
#ifdef NATIVE_LAPACK_LIB
	hipblasStatus_t error = hipblasCreate(&handle);
	if (error != HIPBLAS_STATUS_SUCCESS) errorQuda("hipblasCreate failed with error %d", error);
	cublas_init = true;
#endif
      }
    }
    
    void destroy() {
      if(cublas_init) {
#ifdef NATIVE_LAPACK_LIB
	hipblasStatus_t error = hipblasDestroy(handle);
	if (error != HIPBLAS_STATUS_SUCCESS) errorQuda("\nError indestroying cublas context, error code = %d\n", error);
	cublas_init = false;
#endif
      }
    }

    // mini kernel to set the array of pointers needed for batched cublas
    template<typename T>
    __global__ void set_pointer(T **output_array_a, T *input_a, T **output_array_b, T *input_b, int batch_offset)
    {
      output_array_a[blockIdx.x] = input_a + blockIdx.x * batch_offset;
      output_array_b[blockIdx.x] = input_b + blockIdx.x * batch_offset;
    }

    template <typename EigenMatrix, typename Float>
    __host__ void checkEigen(std::complex<Float> *A_h, std::complex<Float> *Ainv_h, int n, uint64_t batch)
    {
      
      EigenMatrix A = EigenMatrix::Zero(n, n);
      EigenMatrix Ainv = EigenMatrix::Zero(n, n);
      for (int j = 0; j < n; j++) {
        for (int k = 0; k < n; k++) {
	  A(k, j) = A_h[batch * n * n + j * n + k];
	  Ainv(k, j) = Ainv_h[batch * n * n + j * n + k];
	}
      }
      
      // Check result:
      EigenMatrix unit = EigenMatrix::Identity(n,n);
      EigenMatrix prod = A * Ainv;
      Float L2norm = ((prod - unit).norm()/(n*n));
      printfQuda("cuBLAS: Norm of (A * Ainv - I) batch %lu = %e\n", batch, L2norm);
    }    
    
    // FIXME do this in pipelined fashion to reduce memory overhead.
    long long BatchInvertMatrix(void *Ainv, void* A, const int n, const uint64_t batch, QudaPrecision prec, QudaFieldLocation location)
    {
#ifdef NATIVE_LAPACK_LIB
      if (getVerbosity() >= QUDA_SUMMARIZE)
	printfQuda("BatchInvertMatrixNATIVE: Nc = %d, batch = %lu\n", n, batch);
      long long flops = 0;      
      timeval start, stop;
      gettimeofday(&start, NULL);

      size_t size = 2*n*n*prec*batch;
      void *A_d = location == QUDA_CUDA_FIELD_LOCATION ? A : pool_device_malloc(size);
      void *Ainv_d = location == QUDA_CUDA_FIELD_LOCATION ? Ainv : pool_device_malloc(size);
      if (location == QUDA_CPU_FIELD_LOCATION) qudaMemcpy(A_d, A, size, hipMemcpyHostToDevice);

#ifdef LOCAL_DEBUG
      // Debug code: Copy original A matrix to host
      std::complex<float> *A_h = (location == QUDA_CUDA_FIELD_LOCATION ? static_cast<std::complex<float>*>(pool_pinned_malloc(size)) : static_cast<std::complex<float>*>(A_d));
      if (location == QUDA_CUDA_FIELD_LOCATION) qudaMemcpy((void*)A_h, A_d, size, hipMemcpyDeviceToHost);      
#endif
      
      int *dipiv = static_cast<int*>(pool_device_malloc(batch*n*sizeof(int)));
      int *dinfo_array = static_cast<int*>(pool_device_malloc(batch*sizeof(int)));
      int *info_array = static_cast<int*>(pool_pinned_malloc(batch*sizeof(int)));
      memset(info_array, '0', batch*sizeof(int)); // silence memcheck warnings

      if (prec == QUDA_SINGLE_PRECISION) {
	typedef hipFloatComplex C;
	C **A_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));
	C **Ainv_array = static_cast<C**>(pool_device_malloc(batch*sizeof(C*)));

	set_pointer<C><<<batch,1>>>(A_array, (C*)A_d, Ainv_array, (C*)Ainv_d, n*n);

	hipblasStatus_t error = hipblasCgetrfBatched(handle, n, A_array, n, dipiv, dinfo_array, batch);
	flops += batch*FLOPS_CGETRF(n,n);

	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in LU decomposition (hipblasCgetrfBatched), error code = %d\n", error);

	qudaMemcpy(info_array, dinfo_array, batch*sizeof(int), hipMemcpyDeviceToHost);
	for (uint64_t i=0; i<batch; i++) {
	  if (info_array[i] < 0) {
	    errorQuda("%lu argument had an illegal value or another error occured, such as memory allocation failed", i);
	  } else if (info_array[i] > 0) {
	    errorQuda("%lu factorization completed but the factor U is exactly singular", i);
	  }
	}
    
	error = hipblasCgetriBatched(handle, n, (const C**)A_array, n, dipiv, Ainv_array, n, dinfo_array, batch);
	flops += batch*FLOPS_CGETRI(n);

	if (error != HIPBLAS_STATUS_SUCCESS)
	  errorQuda("\nError in matrix inversion (hipblasCgetriBatched), error code = %d\n", error);

	qudaMemcpy(info_array, dinfo_array, batch*sizeof(int), hipMemcpyDeviceToHost);

	for (uint64_t i=0; i<batch; i++) {
	  if (info_array[i] < 0) {
	    errorQuda("%lu argument had an illegal value or another error occured, such as memory allocation failed", i);
	  } else if (info_array[i] > 0) {
	    errorQuda("%lu factorization completed but the factor U is exactly singular", i);
	  }
	}

#ifdef LOCAL_DEBUG
	// Debug code: Copy computed Ainv to host
	std::complex<float> *Ainv_h = static_cast<std::complex<float>*>(pool_pinned_malloc(size));       
	qudaMemcpy((void*)Ainv_h, Ainv_d, size, hipMemcpyDeviceToHost);
	
        for (uint64_t i = 0; i < batch; i++) { checkEigen<MatrixXcf, float>(A_h, Ainv_h, n, i); }
	pool_pinned_free(Ainv_h);
	pool_pinned_free(A_h);
#endif
      } else {
	errorQuda("%s not implemented for precision=%d", __func__, prec);
      }
      
      if (location == QUDA_CPU_FIELD_LOCATION) {
	qudaMemcpy(Ainv, Ainv_d, size, hipMemcpyDeviceToHost);
	pool_device_free(Ainv_d);
	pool_device_free(A_d);
      }

      pool_device_free(dipiv);
      pool_device_free(dinfo_array);
      pool_pinned_free(info_array);

      qudaDeviceSynchronize();
      gettimeofday(&stop, NULL);
      long ds = stop.tv_sec - start.tv_sec;
      long dus = stop.tv_usec - start.tv_usec;
      double time = ds + 0.000001*dus;

      if (getVerbosity() >= QUDA_SUMMARIZE)
	printfQuda("Batched matrix inversion completed in %f seconds with GFLOPS = %f\n", time, 1e-9 * flops / time);
      
      return flops;
#else
      errorQuda("Native BLAS not built. Please build and use native BLAS or use generic BLAS");
      return 0; // Stops a compiler warning
#endif
    }
  } // namespace native_lapack
} // namespace quda

