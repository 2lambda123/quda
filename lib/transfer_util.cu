#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <typeinfo>
#include <vector>
#include <assert.h>

namespace quda {

  using namespace quda::colorspinor;

//ok for staggered: nSpin = 1 will work as well. Accessors do allow this case as well.

  // copy the null-space vectors into the V-field
  template <int nSpin, int nColor, int nVec, class V, class B>
  void fill(V &out, const B &in, int v) {
    for (int parity=0; parity<out.Nparity(); parity++) {
      for (int x_cb=0; x_cb<out.VolumeCB(); x_cb++) {
	for (int s=0; s<nSpin; s++) {
	  for (int c=0; c<nColor; c++) {
	    out(parity, x_cb, s, c, v) = in(parity, x_cb, s, c);
	  }
	}
      }
    }
  }

  template <typename Float, int nSpin, int nColor, int nVec, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B) {
    FieldOrderCB<Float,nSpin,nColor,nVec,order> vOrder(const_cast<ColorSpinorField&>(V));
    for (int v=0; v<nVec; v++) {
      FieldOrderCB<Float,nSpin,nColor,1,order> bOrder(const_cast<ColorSpinorField&>(*B[v]));
      fill<nSpin,nColor,nVec>(vOrder, bOrder, v);
    }
  }

//for staggered: this does not include factor 2 due to parity decomposition!

  template <typename Float, int nSpin, int nColor, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (Nvec == 2) {
      FillV<Float,nSpin,nColor,2,order>(V,B);
    } else if (Nvec == 4) {
      FillV<Float,nSpin,nColor,4,order>(V,B);
    } else if (Nvec == 8) {
      FillV<Float,nSpin,nColor,8,order>(V,B);
    } else if (Nvec == 12) {
      FillV<Float,nSpin,nColor,12,order>(V,B);
    } else if (Nvec == 16) {
      FillV<Float,nSpin,nColor,16,order>(V,B);
    } else if (Nvec == 20) {
      FillV<Float,nSpin,nColor,20,order>(V,B);
    } else if (Nvec == 24) {
      FillV<Float,nSpin,nColor,24,order>(V,B);
    } else if (Nvec == 48) {
      FillV<Float,nSpin,nColor,48,order>(V,B);
    } else {
      errorQuda("Unsupported Nvec %d", Nvec);
    }
  }

//ok for 2-cycle multigrid, must be extended for more complicated version.

  template <typename Float, int nSpin, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (B[0]->Ncolor()*Nvec != V.Ncolor()) errorQuda("Something wrong here");

    if (B[0]->Ncolor() == 2) {
      FillV<Float,nSpin,2,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 3) {
      FillV<Float,nSpin,3,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 8) {
      FillV<Float,nSpin,8,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 16) {
      FillV<Float,nSpin,16,order>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 24) {
      FillV<Float,nSpin,24,order>(V,B,Nvec);
    } else {
      errorQuda("Unsupported nColor %d", B[0]->Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder order>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.Nspin() == 4) {
      FillV<Float,4,order>(V,B,Nvec);
    } else if (V.Nspin() == 2) {
      FillV<Float,2,order>(V,B,Nvec);
#ifdef GPU_STAGGERED_DIRAC
    } else if (V.Nspin() == 1) {
      FillV<Float,1,order>(V,B,Nvec);
#endif
    } else {
      errorQuda("Unsupported nSpin %d", V.Nspin());
    }
  }

  template <typename Float>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      FillV<Float,QUDA_FLOAT2_FIELD_ORDER>(V,B,Nvec);
    } else if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      FillV<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(V,B,Nvec);
    } else {
      errorQuda("Unsupported field type %d", V.FieldOrder());
    }
  }

  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.Precision() == QUDA_DOUBLE_PRECISION) {
      FillV<double>(V,B,Nvec);
    } else if (V.Precision() == QUDA_SINGLE_PRECISION) {
      FillV<float>(V,B,Nvec);
    } else {
      errorQuda("Unsupported precision %d", V.Precision());
    }
  }

  // Creates a block-ordered version of a ColorSpinorField
  // N.B.: Only works for the V field, as we need to block spin.
  template <bool toBlock, int nVec, class Complex, class FieldOrder>
  void blockOrderV(Complex *out, FieldOrder &in,
		   const int *geo_map, const int *geo_bs, int spin_bs,
		   const cpuColorSpinorField &V) {
    //printfQuda("in.Ncolor = %d\n", in.Ncolor());
    int nSpin_coarse = in.Nspin() / spin_bs; // this is number of chiral blocks

    //Compute the size of each block
    int geoBlockSize = 1;
    for (int d=0; d<in.Ndim(); d++) geoBlockSize *= geo_bs[d];
    int blockSize = geoBlockSize * in.Ncolor() * spin_bs; // blockSize includes internal dof

    int x[QUDA_MAX_DIM]; // global coordinates
    int y[QUDA_MAX_DIM]; // local coordinates within a block (full site ordering)

    int checkLength = in.Volume() * in.Ncolor() * in.Nspin() * in.Nvec();
    int *check = new int[checkLength];
    int count = 0;

    // Run through the fine grid and do the block ordering
    for (int parity = 0; parity<in.Nparity(); parity++) {
      for (int x_cb=0; x_cb<in.VolumeCB(); x_cb++) {
	int i = parity*in.VolumeCB() + x_cb;

	// Get fine grid coordinates
	V.LatticeIndex(x, i);
	
	//Compute the geometric offset within a block 
	// (x fastest direction, t is slowest direction, non-parity ordered)
	int blockOffset = 0;
	for (int d=in.Ndim()-1; d>=0; d--) {
	  y[d] = x[d]%geo_bs[d];
	  blockOffset *= geo_bs[d];
	  blockOffset += y[d];
	}
	
	//Take the block-ordered offset from the coarse grid offset (geo_map) 
	int offset = geo_map[i]*nSpin_coarse*nVec*geoBlockSize*in.Ncolor()*spin_bs;
	
	for (int v=0; v<in.Nvec(); v++) {
	  for (int s=0; s<in.Nspin(); s++) {
	    for (int c=0; c<in.Ncolor(); c++) {
	      
	      int chirality = s / spin_bs; // chirality is the coarse spin
	      int blockSpin = s % spin_bs; // the remaining spin dof left in each block
	      
	      int index = offset +                                              // geo block
		chirality * nVec * geoBlockSize * spin_bs * in.Ncolor() + // chiral block
	                       v * geoBlockSize * spin_bs * in.Ncolor() + // vector
	                            blockOffset * spin_bs * in.Ncolor() + // local geometry
	                                          blockSpin*in.Ncolor() + // block spin
	                                                                   c;   // color

	      if (toBlock) out[index] = in(parity, x_cb, s, c, v); // going to block order
	      else in(parity, x_cb, s, c, v) = out[index]; // coming from block order
	    
	      check[count++] = index;
	    }
	  }
	}
      }

      //printf("blockOrderV done %d / %d\n", i, in.Volume());
    }
    
    if (count != checkLength) {
      errorQuda("Number of elements packed %d does not match expected value %d nvec=%d nspin=%d ncolor=%d", 
		count, checkLength, in.Nvec(), in.Nspin(), in.Ncolor());
    }

    /*
    // need non-quadratic check
    for (int i=0; i<checkLength; i++) {
      for (int j=0; j<i; j++) {
      if (check[i] == check[j]) errorQuda("Collision detected in block ordering\n");
      }
    }
    */
    delete []check;
  }


  // Creates a block-ordered version of a ColorSpinorField, with parity blocking (for staggered fields)
  // N.B.: same as above but parity are separated.
  template <bool toBlock, int nVec, class Complex, class FieldOrder>
  void blockCBOrderV(Complex *out, FieldOrder &in,
		     const int *geo_map, const int *geo_bs, int spin_bs,
		     const cpuColorSpinorField &V) {
    //Compute the size of each block
    int geoBlockSize = 1;
    for (int d=0; d<in.Ndim(); d++) geoBlockSize *= geo_bs[d];
    int blockSize = geoBlockSize * in.Ncolor(); // blockSize includes internal dof

    int x[QUDA_MAX_DIM]; // global coordinates
    int y[QUDA_MAX_DIM]; // local coordinates within a block (full site ordering)

    int checkLength = in.Volume() * in.Ncolor() * in.Nvec();
    int *check = new int[checkLength];
    int count = 0;

    // Run through the fine grid and do the block ordering
    for (int parity = 0; parity<in.Nparity(); parity++) {
      for (int x_cb=0; x_cb<in.VolumeCB(); x_cb++) {
	int i = parity*in.VolumeCB() + x_cb;

	// Get fine grid coordinates
	V.LatticeIndex(x, i);

	//Compute the geometric offset within a block 
	// (x fastest direction, t is slowest direction, non-parity ordered)
	int blockOffset = 0;
	for (int d=in.Ndim()-1; d>=0; d--) {
	  y[d] = x[d]%geo_bs[d];
	  blockOffset *= geo_bs[d];
	  blockOffset += y[d];
	}

	//Take the block-ordered offset from the coarse grid offset (geo_map) 
	//A.S.: geo_map introduced for the full site ordering, so ok to use it for the offset
	int offset = geo_map[i]*nVec*geoBlockSize*in.Ncolor();

	const int s = 0;

	for (int v=0; v<in.Nvec(); v++) {
	  for (int c=0; c<in.Ncolor(); c++) {

	    int chirality = (x[0]+x[1]+x[2]+x[3])%2; // chirality is the fine-grid parity flag

	    int index = offset +                                // geo block
	      chirality * nVec * geoBlockSize * in.Ncolor() + // chiral block
	                     v * geoBlockSize * in.Ncolor() + // vector
	                          blockOffset * in.Ncolor() + // local geometry
	                                                       c;   // color

	    if (toBlock) out[index] = in(parity, x_cb, s, c, v); // going to block order
	    else in(parity, x_cb, s, c, v) = out[index]; // coming from block order

	    check[count++] = index;
	  }
	}

	//printf("blockOrderV done %d / %d\n", i, in.Volume());
      } // x_cb
    } // parity

    if (count != checkLength) {
      errorQuda("Number of elements packed %d does not match expected value %d nvec=%d ncolor=%d", 
		count, checkLength, in.Nvec(), in.Ncolor());
    }

    delete []check;
  }




  // Orthogonalise the nc vectors v[] of length n
  // this assumes the ordering v[(b * Nvec + v) * blocksize + i]

  template <typename sumFloat, typename Float, int N>
  void blockGramSchmidt(complex<Float> *v, int nBlocks, int blockSize) {
    
    for (int b=0; b<nBlocks; b++) {
      for (int jc=0; jc<N; jc++) {
      
	for (int ic=0; ic<jc; ic++) {
	  // Calculate dot product.
	  complex<Float> dot = 0.0;
	  for (int i=0; i<blockSize; i++) 
	    dot += conj(v[(b*N+ic)*blockSize+i]) * v[(b*N+jc)*blockSize+i];
	  
	  // Subtract the blocks to orthogonalise
	  for (int i=0; i<blockSize; i++) 
	    v[(b*N+jc)*blockSize+i] -= dot * v[(b*N+ic)*blockSize+i];
	}
	
	// Normalize the block
	// nrm2 is pure real, but need to use Complex because of template.
	complex<sumFloat> nrm2 = 0.0;
	for (int i=0; i<blockSize; i++) nrm2 += norm(v[(b*N+jc)*blockSize+i]);
	sumFloat scale = nrm2.real() > 0.0 ? 1.0/sqrt(nrm2.real()) : 0.0;
	for (int i=0; i<blockSize; i++) v[(b*N+jc)*blockSize+i] *= scale;
      }


      /*      
      for (int jc=0; jc<N; jc++) {
        complex<sumFloat> nrm2 = 0.0;
        for(int i=0; i<blockSize; i++) nrm2 += norm(v[(b*N+jc)*blockSize+i]);
	//printfQuda("block = %d jc = %d nrm2 = %f\n", b, jc, nrm2.real());
      }
      */

      //printf("blockGramSchmidt done %d / %d\n", b, nBlocks);
    }

  }

  template<typename Float, int nSpin, int nColor, int nVec, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, const int *geo_bs, const int *geo_map, int spin_bs) {
    complex<Float> *Vblock = new complex<Float>[V.Volume()*V.Nspin()*V.Ncolor()];

    FieldOrderCB<Float,nSpin,nColor,nVec,order> vOrder(const_cast<ColorSpinorField&>(V));

    int geo_blocksize = 1;
    for (int d = 0; d < V.Ndim(); d++) geo_blocksize *= geo_bs[d];

    int blocksize = geo_blocksize * vOrder.Ncolor() * spin_bs; 
    int chiralBlocks = (V.Nspin() == 1) ? 2 : vOrder.Nspin() / spin_bs; //always 2 for staggered. 
    int numblocks = (V.Volume()/geo_blocksize) * chiralBlocks;
    
    if(V.Nspin() != 1){//FIXME : this is not good, think about a separate parameter to distinguish staggered stuff!
      printfQuda("Block Orthogonalizing %d blocks of %d length and width %d\n", numblocks, blocksize, nVec);
    
      blockOrderV<true,nVec>(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);
      blockGramSchmidt<double,Float,nVec>(Vblock, numblocks, blocksize);  
      blockOrderV<false,nVec>(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);    
    }
    else{
      blocksize /= chiralBlocks; //for staggered chiral block size is a parity block size   

      printfQuda("Block Orthogonalizing %d blocks of %d length and width %d\n", numblocks, blocksize, nVec);

      blockCBOrderV<true,nVec>(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);
      blockGramSchmidt<double,Float,nVec>(Vblock, numblocks, blocksize);  
      blockCBOrderV<false,nVec>(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);    
   
    }
    delete []Vblock;
  }


  template<typename Float, int nSpin, int nColor, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, const int *geo_bs, const int *geo_map, int spin_bs) {
    if (Nvec == 2) {
      BlockOrthogonalize<Float,nSpin,nColor,2,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 4) {
      BlockOrthogonalize<Float,nSpin,nColor,4,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 8) {
      BlockOrthogonalize<Float,nSpin,nColor,8,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 12) {
      BlockOrthogonalize<Float,nSpin,nColor,12,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 16) {
      BlockOrthogonalize<Float,nSpin,nColor,16,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 20) {
      BlockOrthogonalize<Float,nSpin,nColor,20,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 24) {
      BlockOrthogonalize<Float,nSpin,nColor,24,order>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 48) {
      BlockOrthogonalize<Float,nSpin,nColor,48,order>(V, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported nVec %d\n", Nvec);
    }
  }

  template<typename Float, int nSpin, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Ncolor()/Nvec == 3) {
      BlockOrthogonalize<Float,nSpin,3,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 2) {
      BlockOrthogonalize<Float,nSpin,2,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 8) {
      BlockOrthogonalize<Float,nSpin,8,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 16) {
      BlockOrthogonalize<Float,nSpin,16,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 24) {
      BlockOrthogonalize<Float,nSpin,24,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if (V.Ncolor()/Nvec == 48) {
      BlockOrthogonalize<Float,nSpin,48,order>(V, Nvec, geo_bs, geo_map, spin_bs); //for staggered, even-odd blocking presumed
    }  
    else {
      errorQuda("Unsupported nColor %d\n", V.Ncolor()/Nvec);
    }
  }

  template<typename Float, QudaFieldOrder order>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Nspin() == 4) {
      BlockOrthogonalize<Float,4,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    }
    else if(V.Nspin() ==2) {
      BlockOrthogonalize<Float,2,order>(V, Nvec, geo_bs, geo_map, spin_bs);
    } 
    else if (V.Nspin() == 1) {
      BlockOrthogonalize<Float,1,order>(V, Nvec, geo_bs, geo_map, 1);
    }
    else {
      errorQuda("Unsupported nSpin %d\n", V.Nspin());
    }
  }

  template<typename Float>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
  if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      BlockOrthogonalize<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported field order %d\n", V.FieldOrder());
    }
  }

  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Precision() == QUDA_DOUBLE_PRECISION) {
      BlockOrthogonalize<double>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if (V.Precision() == QUDA_SINGLE_PRECISION) {
      BlockOrthogonalize<float>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported precision %d\n", V.Precision());
    }
  }

} // namespace quda
