#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <dslash_quda.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <dslash_quda.h>
#include <inline_ptx.h>

namespace quda {

#ifdef GPU_DOMAIN_WALL_DIRAC

  /**
     @brief Structure containing zMobius / Zolotarev coefficients
  */
  template <typename real>
  struct coeff_5 {
    complex<real> a[QUDA_MAX_DWF_LS]; // xpay coefficients
    complex<real> b[QUDA_MAX_DWF_LS];
    complex<real> c[QUDA_MAX_DWF_LS];
  };

  constexpr int size = 4096;
  static __constant__ char mobius_d[size]; // constant buffer used for Mobius coefficients for GPU kernel
  static char mobius_h[size];              // constant buffer used for Mobius coefficients for CPU kernel

  /**
     @brief Parameter structure for applying the Dslash
   */
  template <typename Float, int nColor>
  struct Dslash5Arg {
    typedef typename colorspinor_mapper<Float,4,nColor>::type F;
    typedef typename mapper<Float>::type real;

    F out;                  // output vector field
    const F in;             // input vector field
    const F x;              // auxiliary input vector field
    const int nParity;      // number of parities we're working on
    const int volume_cb;    // checkerboarded volume
    const int volume_4d_cb; // 4-d checkerboarded volume
    const int_fastdiv Ls;   // length of 5th dimension

    const real m_f;         // fermion mass parameter
    const real m_5;         // Wilson mass shift

    const bool dagger;      // dagger
    const bool xpay;        // whether we are doing xpay or not

    real b;                 // real constant Mobius coefficient
    real c;                 // real constant Mobius coefficient
    real a;                 // real xpay coefficient

    Dslash5Type type;

    Dslash5Arg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
               double m_f, double m_5, const Complex *b_5_, const Complex *c_5_,
               double a, bool dagger, Dslash5Type type)
      : out(out), in(in), x(x), nParity(in.SiteSubset()),
	volume_cb(in.VolumeCB()), volume_4d_cb(volume_cb/in.X(4)), Ls(in.X(4)),
	m_f(m_f), m_5(m_5), a(a), dagger(dagger), xpay(a == 0.0 /*in.V() == x.V()*/ ? false : true), type(type)
    {
      if (in.Nspin() != 4) errorQuda("nSpin = %d not support", in.Nspin());
      if (!in.isNative() || !out.isNative()) errorQuda("Unsupported field order out=%d in=%d\n", out.FieldOrder(), in.FieldOrder());

      if (sizeof(coeff_5<real>) > size) errorQuda("Coefficient buffer too large at %lu bytes\n", sizeof(coeff_5<real>));
      coeff_5<real> *coeff = reinterpret_cast<coeff_5<real>*>(&mobius_h);
      auto *a_5 =  coeff->a;
      auto *b_5 =  coeff->b;
      auto *c_5 =  coeff->c;

      switch(type) {
      case DSLASH5_DWF:
	break;
      case DSLASH5_MOBIUS_PRE:
	for (int s=0; s<Ls; s++) {
	  b_5[s] = b_5_[s];
	  c_5[s] = 0.5*c_5_[s];

	  // xpay
	  a_5[s] = 0.5/(b_5_[s]*(m_5+4.0) + 1.0);
	  a_5[s] *= a_5[s] * static_cast<real>(a);
        }
	break;
      case DSLASH5_MOBIUS:
	for (int s=0; s<Ls; s++) {
	  b_5[s] = 1.0;
	  c_5[s] = 0.5 * (c_5_[s] * (m_5 + 4.0) - 1.0) / (b_5_[s] * (m_5 + 4.0) + 1.0);

	  // axpy
	  a_5[s] = 0.5 / (b_5_[s] * (m_5 + 4.0) + 1.0);
	  a_5[s] *= a_5[s] * static_cast<real>(a);
	}
	break;
      case M5_INV_DWF:
        b = 2.0 * (0.5/(5.0 + m_5)); // 2  * kappa_5
        c = 0.5 / ( 1.0 + std::pow(b,(int)Ls) * m_f );
        break;
      case M5_INV_MOBIUS:
        {
          complex<double> k = 1.0;
          for (int s=0; s<Ls; s++) {
            b_5[s] = -(c_5_[s] * (4.0 + m_5) - 1.0) / (b_5_[s] * (4.0 + m_5) + 1.0);
            k *= b_5[s];
          }

          for (int s=0; s<Ls; s++) {
            c_5[s] = 0.5 / ( 1.0 + k * m_f );

            a_5[s] = 0.5 * (b_5_[s] * (m_5 + 4.0) + 1.0);
            a_5[s]*= a_5[s] * static_cast<real>(a);
          }
        }
        break;
      default:
	errorQuda("Unknown Dslash5Type %d", type);
      }

      hipMemcpyToSymbolAsync(HIP_SYMBOL(mobius_d), mobius_h, sizeof(coeff_5<real>), 0, hipMemcpyHostToDevice, streams[Nstream-1]);

    }
  };

  template <typename real>
  inline __device__ __host__ const coeff_5<real>* coeff() {
#ifdef __CUDA_ARCH__
    return reinterpret_cast<const coeff_5<real>*>(mobius_d);
#else
    return reinterpret_cast<const coeff_5<real>*>(mobius_h);
#endif
  }

  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  __device__ __host__ inline void dslash5(Arg &arg, int parity, int x_cb, int s) {
    typedef typename mapper<Float>::type real;
    typedef ColorSpinor<real,nColor,4> Vector;

    Vector out;

    { // forwards direction
      const int fwd_idx = ((s + 1) % arg.Ls) * arg.volume_4d_cb + x_cb;
      const Vector in = arg.in(fwd_idx, parity);
      constexpr int proj_dir = dagger ? +1 : -1;
      if (s == arg.Ls-1) {
	out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
      } else {
	out += in.project(4, proj_dir).reconstruct(4, proj_dir);
      }
    }

    { // backwards direction
      const int back_idx = ((s + arg.Ls - 1) % arg.Ls) * arg.volume_4d_cb + x_cb;
      const Vector in = arg.in(back_idx, parity);
      constexpr int proj_dir = dagger ? -1 : +1;
      if (s == 0) {
	out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
      } else {
	out += in.project(4, proj_dir).reconstruct(4, proj_dir);
      }
    }

    if (type == DSLASH5_DWF && xpay) {
      Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
      out = x + arg.a*out;
    } else if (type == DSLASH5_MOBIUS_PRE) {
      Vector diagonal = arg.in(s*arg.volume_4d_cb + x_cb, parity);
      auto *z = coeff<real>();
      out = z->c[s] * out + z->b[s] * diagonal;

      if (xpay) {
	Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
	out = x + z->a[s] * out;
      }
    } else if (type == DSLASH5_MOBIUS) {
      Vector diagonal = arg.in(s*arg.volume_4d_cb + x_cb, parity);
      auto *z = coeff<real>();
      out = z->c[s] * out + diagonal;

      if (xpay) { // really axpy
	Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
	out = z->a[s] * x + out;
      }
    }

    arg.out(s*arg.volume_4d_cb + x_cb, parity) = out;
  }

  // CPU kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  void dslash5CPU(Arg &arg)
  {
    for (int parity= 0; parity < arg.nParity; parity++) {
      for (int s=0; s < arg.Ls; s++) {
	for (int x_cb = 0; x_cb < arg.volume_4d_cb; x_cb++) { // 4-d volume
	  dslash5<Float,nColor,dagger,xpay,type>(arg, parity, x_cb, s);
	}  // 4-d volumeCB
      } // ls
    } // parity

  }

  // GPU Kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  __global__ void dslash5GPU(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int s = blockIdx.y*blockDim.y + threadIdx.y;
    int parity = blockIdx.z*blockDim.z + threadIdx.z;

    if (x_cb >= arg.volume_4d_cb) return;
    if (s >= arg.Ls) return;
    if (parity >= arg.nParity) return;

    dslash5<Float,nColor,dagger,xpay,type>(arg, parity, x_cb, s);
  }

  /*
    @brief Fast power function that works for negative "a" argument
    @param a argument we want to raise to some power
    @param b power that we want to raise a to
    @return pow(a,b)
  */
  template<typename real>
  __device__ __host__ inline real __fast_pow(real a, int b) {
#ifdef __CUDA_ARCH__
    if (sizeof(real) == sizeof(double)) {
      return pow(a, b);
    } else {
      float sign = signbit(a) ? -1.0f : 1.0f;
      float power = __powf(fabsf(a), b);
      return b&1 ? sign * power : power;
    }
#else
    return std::pow(a, b);
#endif
  }

  /**
     @brief Class which wraps around a shared memory cache for a Vector type,
     where each thread in the thread block stores a unique
     Vector in the cache which any other thread can access.
   */
  template <typename real, typename Vector>
  class VectorCache {

    /**
       @brief This is the handle to the shared memory
       @return Shared memory pointer
     */
    __device__ inline real* cache() {
      extern __shared__ int cache_[];
      return reinterpret_cast<real*>(cache_);
    }

  public:

    /**
       @brief Save the vector into the 3-d shared memory cache.
       Implicitly store the vector at coordinates given by threadIdx.
       @param[in] a The vector to store in the shared memory cache
     */
    __device__ inline void save(const Vector &a) {
#pragma unroll
      for (int i=0; i<2*a.n; i++) {
        int j = ((i*blockDim.z + threadIdx.z)*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x;
        cache()[j] = *(reinterpret_cast<const real*>(a.data) + i);
      }
      __syncthreads();
    }

    /**
       @brief Load a vector from the shared memory cache
       @param[in] x The x index to use
       @param[in] y The y index to use
       @param[in] z The z index to use
       @return The Vector at coordinates (x,y,z)
     */
    __device__ inline Vector load(int x, int y, int z) {
      Vector a;
#pragma unroll
      for (int i=0; i<2*a.n; i++) {
        int j = ((i*blockDim.z + z)*blockDim.y + y)*blockDim.x + x;
        *(reinterpret_cast<real*>(a.data) + i) = cache()[j];
      }
      return a;
    }

  };

  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, bool shared, typename Arg>
  __device__ __host__ inline void dslash5inv(Arg &arg, int parity, int x_cb, int s_) {
    constexpr int nSpin = 4;
    typedef typename mapper<Float>::type real;
    typedef ColorSpinor<real,nColor,nSpin> Vector;

    Vector out;
    auto *z = coeff<real>();
    const auto k = type == M5_INV_DWF ? arg.b : z->b[s_].real();
    const auto inv_d_n = type == M5_INV_DWF ? arg.c : z->c[s_].real();

    // if using shared-memory caching then load spinor field for my site into cache
    VectorCache<real,Vector> cache;
    if (shared) cache.save(arg.in(s_*arg.volume_4d_cb + x_cb, parity));

    for (int s=0; s<arg.Ls; s++) {

      Vector in = shared ? cache.load(threadIdx.x, s, parity) : arg.in(s*arg.volume_4d_cb + x_cb, parity);

      {
        int exp = s_ < s ? arg.Ls-s+s_ : s_-s;
        real factorR = inv_d_n * __fast_pow(k,exp) * ( s_ < s ? -arg.m_f : static_cast<real>(1.0) );
        constexpr int proj_dir = dagger ? -1 : +1;
        out += factorR * (in.project(4, proj_dir)).reconstruct(4, proj_dir);
      }

      {
        int exp = s_ > s ? arg.Ls-s_+s : s-s_;
        real factorL = inv_d_n * __fast_pow(k,exp) * ( s_ > s ? -arg.m_f : static_cast<real>(1.0));
        constexpr int proj_dir = dagger ? +1 : -1;
        out += factorL * (in.project(4, proj_dir)).reconstruct(4, proj_dir);
      }

    }

    if (xpay) {
      Vector x = arg.x(s_*arg.volume_4d_cb + x_cb, parity);
      if (type == M5_INV_DWF) {
        out = x + arg.a*out;
      } else if (type == M5_INV_MOBIUS) {
        auto *z = coeff<real>();
        out = x + z->a[s_] * out;
      }
    }

    arg.out(s_*arg.volume_4d_cb + x_cb, parity) = out;
  }

  // CPU kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  void dslash5invCPU(Arg &arg)
  {
    constexpr bool shared = false; // shared memory doesn't apply here
    for (int parity= 0; parity < arg.nParity; parity++) {
      for (int s=0; s < arg.Ls; s++) {
	for (int x_cb = 0; x_cb < arg.volume_4d_cb; x_cb++) { // 4-d volume
	  dslash5inv<Float,nColor,dagger,xpay,type,shared>(arg, parity, x_cb, s);
	}  // 4-d volumeCB
      } // ls
    } // parity

  }

  // GPU Kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, bool shared, typename Arg>
  __global__ void dslash5invGPU(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int s = blockIdx.y*blockDim.y + threadIdx.y;
    int parity = blockIdx.z*blockDim.z + threadIdx.z;

    if (x_cb >= arg.volume_4d_cb) return;
    if (s >= arg.Ls) return;
    if (parity >= arg.nParity) return;

    dslash5inv<Float,nColor,dagger,xpay,type,shared>(arg, parity, x_cb, s);
  }

  template <typename Float, int nColor, typename Arg>
  class Dslash5 : public TunableVectorYZ {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;
    static constexpr bool shared = true; // whether to use shared memory cache blocking for M5inv

    long long flops() const {
      long long Ls = meta.X(4);
      long long bulk = (Ls-2)*(meta.Volume()/Ls);
      long long wall = 2*meta.Volume()/Ls;
      int n = meta.Ncolor() * meta.Nspin();
      bool zMobius = true; // set to true when we have complexity

      long long flops_ = 0;
      switch (arg.type) {
      case DSLASH5_DWF:
        flops_ = n * (8ll*bulk + 10ll*wall + (arg.xpay ? 4ll * meta.Volume() : 0) );
        break;
      case DSLASH5_MOBIUS_PRE:
        flops_ = n * (8ll*bulk + 10ll*wall + (zMobius ? 14ll : 6ll) * meta.Volume() +
                      (arg.xpay ? (zMobius ? 8ll : 4ll) * meta.Volume() : 0) );
        break;
      case DSLASH5_MOBIUS:
        flops_ = n * (8ll*bulk + 10ll*wall + (zMobius ? 8ll : 4ll) * meta.Volume() +
                      (arg.xpay ? (zMobius ? 8ll : 4ll) * meta.Volume() : 0) );
        break;
      case M5_INV_DWF:
        flops_ = (n * n * Ls + (arg.xpay ? 4 : 0)) * meta.Volume();
        break;
      case M5_INV_MOBIUS:
        flops_ = (n * n * Ls + (arg.xpay ? 4 : 0)) * meta.Volume();
        break;
      default:
	errorQuda("Unknown Dslash5Type %d", arg.type);
      }

      return flops_;
    }

    long long bytes() const {
      long long Ls = meta.X(4);
      switch (arg.type) {
      case DSLASH5_DWF:        return arg.out.Bytes() + 2*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case DSLASH5_MOBIUS_PRE: return arg.out.Bytes() + 3*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case DSLASH5_MOBIUS:     return arg.out.Bytes() + 3*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case M5_INV_DWF:         return arg.out.Bytes() + Ls*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case M5_INV_MOBIUS:      return arg.out.Bytes() + Ls*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      default: errorQuda("Unknown Dslash5Type %d", arg.type);
      }
      return 0ll;
    }

    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volume_4d_cb; }
    int blockStep() const { return 4; }
    int blockMin() const { return 4; }
    unsigned int sharedBytesPerThread() const {
      if (shared && arg.type == M5_INV_DWF || arg.type == M5_INV_MOBIUS) {
        return 2*4*nColor*sizeof(typename mapper<Float>::type);
      } else {
        return 0;
      }
    }

  public:
    Dslash5(Arg &arg, const ColorSpinorField &meta)
      : TunableVectorYZ(arg.Ls, arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
      if (arg.dagger) strcat(aux, ",Dagger");
      if (arg.xpay) strcat(aux,",xpay");
      switch (arg.type) {
      case DSLASH5_DWF:        strcat(aux, ",DSLASH5_DWF"); break;
      case DSLASH5_MOBIUS_PRE: strcat(aux, ",DSLASH5_MOBIUS_PRE"); break;
      case DSLASH5_MOBIUS:     strcat(aux, ",DSLASH5_MOBIUS"); break;
      case M5_INV_DWF:         strcat(aux, ",M5_INV_DWF"); break;
      case M5_INV_MOBIUS:      strcat(aux, ",M5_INV_MOBIUS"); break;
      default: errorQuda("Unknown Dslash5Type %d", arg.type);
      }
    }
    virtual ~Dslash5() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	if (arg.type == DSLASH5_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true,true,DSLASH5_DWF>(arg) :
			  dslash5CPU<Float,nColor,false,true,DSLASH5_DWF>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_DWF>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_DWF>(arg);
	} else if (arg.type == DSLASH5_MOBIUS_PRE) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true, true,DSLASH5_MOBIUS_PRE>(arg) :
			  dslash5CPU<Float,nColor,false, true,DSLASH5_MOBIUS_PRE>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_MOBIUS_PRE>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_MOBIUS_PRE>(arg);
	} else if (arg.type == DSLASH5_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true, true,DSLASH5_MOBIUS>(arg) :
			  dslash5CPU<Float,nColor,false, true,DSLASH5_MOBIUS>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_MOBIUS>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_MOBIUS>(arg);
	} else if (arg.type == M5_INV_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5invCPU<Float,nColor, true, true,M5_INV_DWF>(arg) :
			  dslash5invCPU<Float,nColor,false, true,M5_INV_DWF>(arg);
	  else          arg.dagger ?
			  dslash5invCPU<Float,nColor, true,false,M5_INV_DWF>(arg) :
			  dslash5invCPU<Float,nColor,false,false,M5_INV_DWF>(arg);
	} else if (arg.type == M5_INV_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5invCPU<Float,nColor, true, true,M5_INV_MOBIUS>(arg) :
			  dslash5invCPU<Float,nColor,false, true,M5_INV_MOBIUS>(arg);
	  else          arg.dagger ?
			  dslash5invCPU<Float,nColor, true,false,M5_INV_MOBIUS>(arg) :
			  dslash5invCPU<Float,nColor,false,false,M5_INV_MOBIUS>(arg);
	}
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	if (arg.type == DSLASH5_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == DSLASH5_MOBIUS_PRE) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == DSLASH5_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == M5_INV_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5invGPU<Float,nColor, true, true,M5_INV_DWF,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5invGPU<Float,nColor,false, true,M5_INV_DWF,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5invGPU<Float,nColor, true,false,M5_INV_DWF,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5invGPU<Float,nColor,false,false,M5_INV_DWF,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == M5_INV_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5invGPU<Float,nColor, true, true,M5_INV_MOBIUS,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5invGPU<Float,nColor,false, true,M5_INV_MOBIUS,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5invGPU<Float,nColor, true,false,M5_INV_MOBIUS,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5invGPU<Float,nColor,false,false,M5_INV_MOBIUS,shared> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	}
      }
    }

    void initTuneParam(TuneParam &param) const {
      TunableVectorYZ::initTuneParam(param);
      if ( shared && (arg.type == M5_INV_DWF || arg.type == M5_INV_MOBIUS) ) {
        param.block.y = arg.Ls; // Ls must be contained in the block
        param.grid.y = 1;
        param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z;
      }
    }

    void defaultTuneParam(TuneParam &param) const {
      TunableVectorYZ::defaultTuneParam(param);
      if ( shared && (arg.type == M5_INV_DWF || arg.type == M5_INV_MOBIUS) ) {
        param.block.y = arg.Ls; // Ls must be contained in the block
        param.grid.y = 1;
        param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z;
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };


  template <typename Float, int nColor>
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
    Dslash5Arg<Float,nColor> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
    Dslash5<Float,nColor,Dslash5Arg<Float,nColor> > dslash(arg, in);
    dslash.apply(streams[Nstream-1]);
  }

  // template on the number of colors
  template <typename Float>
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
    switch(in.Ncolor()) {
    case 3: ApplyDslash5<Float,3>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    default: errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

#endif

  //Apply the 5th dimension dslash operator to a colorspinor field
  //out = Dslash5*in
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    if (in.DWFPCtype() != QUDA_4D_PC) errorQuda("Only 4-d preconditioned fields are supported");
    checkLocation(out, in);     // check all locations match

    switch(checkPrecision(out,in)) {
    case QUDA_DOUBLE_PRECISION: ApplyDslash5<double>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    case QUDA_SINGLE_PRECISION: ApplyDslash5<float> (out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    case QUDA_HALF_PRECISION:   ApplyDslash5<short> (out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    default: errorQuda("Unsupported precision %d\n", in.Precision());
    }
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }

} // namespace quda

