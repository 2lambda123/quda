#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <dslash_quda.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <dslash_quda.h>
#include <inline_ptx.h>

namespace quda {

#ifdef GPU_DOMAIN_WALL_DIRAC

  template <typename real>
  struct coeff_5 {
    // zMobius / Zolotarev coefficients
    complex<real> a[QUDA_MAX_DWF_LS];
    complex<real> b[QUDA_MAX_DWF_LS];
    complex<real> c[QUDA_MAX_DWF_LS];
  };

  static __constant__ coeff_5<double> mobius_d_d;
  static __constant__ coeff_5<float> mobius_s_d;

  static coeff_5<double> mobius_d_h;
  static coeff_5<float> mobius_s_h;

  /**
     @brief Parameter structure for applying the Dslash
   */
  template <typename Float, int nColor>
  struct Dslash5Arg {
    typedef typename colorspinor_mapper<Float,4,nColor>::type F;
    typedef typename mapper<Float>::type real;

    F out;                  // output vector field
    const F in;             // input vector field
    const F x;              // auxiliary input vector field
    const int nParity;      // number of parities we're working on
    const int volume_cb;    // checkerboarded volume
    const int volume_4d_cb; // 4-d checkerboarded volume
    const int_fastdiv Ls;   // length of 5th dimension

    const real m_f;         // fermion mass parameter
    const real m_5;         // Wilson mass shift

    const bool dagger;      // dagger
    const bool xpay;        // whether we are doing xpay or not

    real b;                 // real constant Mobius coefficient
    real c;                 // real constant Mobius coefficient
    real a;                 // real xpay coefficient

    Dslash5Type type;

    Dslash5Arg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
               double m_f, double m_5, const Complex *b_5_, const Complex *c_5_,
               double a, bool dagger, Dslash5Type type)
      : out(out), in(in), x(x), nParity(in.SiteSubset()),
	volume_cb(in.VolumeCB()), volume_4d_cb(volume_cb/in.X(4)), Ls(in.X(4)),
	m_f(m_f), m_5(m_5), a(a), dagger(dagger), xpay(in.V() == x.V() ? false: true), type(type)
    {
      if (in.Nspin() != 4) errorQuda("nSpin = %d not support", in.Nspin());
      if (!in.isNative() || !out.isNative()) errorQuda("Unsupported field order out=%d in=%d\n", out.FieldOrder(), in.FieldOrder());

      coeff_5<real> *coeff = in.Precision() == QUDA_DOUBLE_PRECISION ? reinterpret_cast<coeff_5<real>*>(&mobius_d_h) :
        reinterpret_cast<coeff_5<real>*>(&mobius_s_h);
      auto *a_5 =  coeff->a;
      auto *b_5 =  coeff->b;
      auto *c_5 =  coeff->c;

      switch(type) {
      case DSLASH5_DWF:
	break;
      case DSLASH5_MOBIUS_PRE:
	for (int s=0; s<Ls; s++) {
	  b_5[s] = b_5_[s].real();
	  c_5[s] = 0.5*c_5_[s].real();

	  // xpay
	  a_5[s] = (0.5/(b_5_[s]*(m_5+4.0) + 1.0)).real();
	  a_5[s] *= a_5[s] * static_cast<real>(a);
        }
	break;
      case DSLASH5_MOBIUS:
	for (int s=0; s<Ls; s++) {
	  b_5[s] = 1.0;
	  c_5[s] = (0.5 * (c_5_[s] * (m_5 + 4.0) - 1.0) / (b_5_[s] * (m_5 + 4.0) + 1.0)).real();

	  // axpy
	  a_5[s] = (0.5 / (b_5_[s] * (m_5 + 4.0) + 1.0)).real();
	  a_5[s] *= a_5[s] * static_cast<real>(a);
	}
	break;
      default:
	errorQuda("Unknown Dslash5Type %d", type);
      }

      if (type == DSLASH5_MOBIUS || type == DSLASH5_MOBIUS_PRE) {
        switch (in.Precision()) {
        case QUDA_DOUBLE_PRECISION:
          hipMemcpyToSymbolAsync(HIP_SYMBOL(mobius_d_d), coeff, sizeof(coeff_5<real>), 0, hipMemcpyHostToDevice, streams[Nstream-1]);
          break;
        case QUDA_SINGLE_PRECISION:
        case QUDA_HALF_PRECISION:
          hipMemcpyToSymbolAsync(HIP_SYMBOL(mobius_s_d), coeff, sizeof(coeff_5<real>), 0, hipMemcpyHostToDevice, streams[Nstream-1]);
          break;
        default:
          errorQuda("Unsupported precision %d\n", in.Precision());
        }
      }

      b = b_5[0].real();
      c = c_5[0].real();
    }
  };

  template <typename real> inline __device__ __host__ complex<real> a_5(int s) {
#ifdef __CUDA_ARCH__
    return mobius_d_d.a[s];
#else
    return mobius_d_h.a[s];
#endif
  }

  template <> inline __device__ __host__ complex<float> a_5<float>(int s) {
#ifdef __CUDA_ARCH__
    return mobius_s_d.a[s];
#else
    return mobius_s_h.a[s];
#endif
  }

  template <typename real> inline __device__ __host__ complex<real> b_5(int s) {
#ifdef __CUDA_ARCH__
    return mobius_d_d.b[s];
#else
    return mobius_d_h.b[s];
#endif
  }

  template <> inline __device__ __host__ complex<float> b_5<float>(int s) {
#ifdef __CUDA_ARCH__
    return mobius_s_d.b[s];
#else
    return mobius_s_h.b[s];
#endif
  }

  template <typename real> inline __device__ __host__ complex<real> c_5(int s) {
#ifdef __CUDA_ARCH__
    return mobius_d_d.c[s];
#else
    return mobius_d_h.c[s];
#endif
  }

  template <> inline __device__ __host__ complex<float> c_5<float>(int s) {
#ifdef __CUDA_ARCH__
    return mobius_s_d.c[s];
#else
    return mobius_s_h.c[s];
#endif
  }

  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  __device__ __host__ inline void dslash5(const Arg &arg, int parity, int x_cb, int s) {
    typedef typename mapper<Float>::type real;
    typedef ColorSpinor<real,nColor,4> Vector;

    Vector out;

    { // forwards direction
      const int fwd_idx = ((s + 1) % arg.Ls) * arg.volume_4d_cb + x_cb;
      const Vector in = arg.in(fwd_idx, parity);
      constexpr int proj_dir = dagger ? +1 : -1;
      if (s == arg.Ls-1) {
	out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
      } else {
	out += in.project(4, proj_dir).reconstruct(4, proj_dir);
      }
    }

    { // backwards direction
      const int back_idx = ((s + arg.Ls - 1) % arg.Ls) * arg.volume_4d_cb + x_cb;
      const Vector in = arg.in(back_idx, parity);
      constexpr int proj_dir = dagger ? -1 : +1;
      if (s == 0) {
	out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
      } else {
	out += in.project(4, proj_dir).reconstruct(4, proj_dir);
      }
    }

    if (type == DSLASH5_DWF && xpay) {
      Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
      out = x + arg.a*out;
    } else if (type == DSLASH5_MOBIUS_PRE) {
      Vector diagonal = arg.in(s*arg.volume_4d_cb + x_cb, parity);
      out = c_5<real>(s) * out + b_5<real>(s) * diagonal;

      if (xpay) {
	Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
	out = x + a_5<real>(s)*out;
      }
    } else if (type == DSLASH5_MOBIUS) {
      Vector diagonal = arg.in(s*arg.volume_4d_cb + x_cb, parity);
      out = c_5<real>(s) * out + diagonal;

      if (xpay) { // really axpy
	Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
	out = a_5<real>(s)*x + out;
      }
    }

    ((Arg*)&arg)->out(s*arg.volume_4d_cb + x_cb, parity) = out;
  }

  // CPU kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  void dslash5CPU(Arg &arg)
  {
    for (int parity= 0; parity < arg.nParity; parity++) {
      for (int s=0; s < arg.Ls; s++) {
	for (int x_cb = 0; x_cb < arg.volume_4d_cb; x_cb++) { // 4-d volume
	  dslash5<Float,nColor,dagger,xpay,type>(arg, parity, x_cb, s);
	}  // 4-d volumeCB
      } // ls
    } // parity

  }

  // GPU Kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  __global__ void dslash5GPU(const Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int s = blockIdx.y*blockDim.y + threadIdx.y;
    int parity = blockIdx.z*blockDim.z + threadIdx.z;

    if (x_cb >= arg.volume_4d_cb) return;
    if (s >= arg.Ls) return;
    if (parity >= arg.nParity) return;

    dslash5<Float,nColor,dagger,xpay,type>(arg, parity, x_cb, s);
  }

  template <typename Float, int nColor, typename Arg>
  class Dslash5 : public TunableVectorYZ {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const {
      long long Ls = meta.X(4);
      long long bulk = (Ls-2)*(meta.Volume()/Ls);
      long long wall = 2*meta.Volume()/Ls;
      int n = meta.Ncolor() * meta.Nspin();
      bool zMobius = true; // set to true when we have complexity

      long long flops_ = 0;
      switch (arg.type) {
      case DSLASH5_DWF:
        flops_ = n * (8ll*bulk + 10ll*wall + (arg.xpay ? 4ll * meta.Volume() : 0) );
        break;
      case DSLASH5_MOBIUS_PRE:
        flops_ = n * (8ll*bulk + 10ll*wall + (zMobius ? 14ll : 6ll) * meta.Volume() +
                      (arg.xpay ? (zMobius ? 8ll : 4ll) * meta.Volume() : 0) );
        break;
      case DSLASH5_MOBIUS:
        flops_ = n * (8ll*bulk + 10ll*wall + (zMobius ? 8ll : 4ll) * meta.Volume() +
                      (arg.xpay ? (zMobius ? 8ll : 4ll) * meta.Volume() : 0) );
        break;
      default:
	errorQuda("Unknown Dslash5Type %d", arg.type);
      }

      return flops_;
    }

    long long bytes() const {
      switch (arg.type) {
      case DSLASH5_DWF:        return arg.out.Bytes() + 2*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case DSLASH5_MOBIUS_PRE: return arg.out.Bytes() + 3*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case DSLASH5_MOBIUS:     return arg.out.Bytes() + 3*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      default: errorQuda("Unknown Dslash5Type %d", arg.type);
      }
      return 0ll;
    }

    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volume_4d_cb; }
    int blockStep() const { return 8; }
    int blockMin() const { return 8; }

  public:
    Dslash5(Arg &arg, const ColorSpinorField &meta)
      : TunableVectorYZ(arg.Ls, arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
      if (arg.dagger) strcat(aux, ",Dagger");
      if (arg.xpay) strcat(aux,",xpay");
      strcat(aux, arg.type == DSLASH5_DWF ? ",DSLASH5_DWF" :
             arg.type == DSLASH5_MOBIUS_PRE ? ",DSLASH5_MOBIUS_PRE" : ",DSLASH5_MOBIUS");
    }
    virtual ~Dslash5() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	if (arg.type == DSLASH5_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true,true,DSLASH5_DWF>(arg) :
			  dslash5CPU<Float,nColor,false,true,DSLASH5_DWF>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_DWF>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_DWF>(arg);
	} else if (arg.type == DSLASH5_MOBIUS_PRE) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true, true,DSLASH5_MOBIUS_PRE>(arg) :
			  dslash5CPU<Float,nColor,false, true,DSLASH5_MOBIUS_PRE>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_MOBIUS_PRE>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_MOBIUS_PRE>(arg);
	} else if (arg.type == DSLASH5_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true, true,DSLASH5_MOBIUS>(arg) :
			  dslash5CPU<Float,nColor,false, true,DSLASH5_MOBIUS>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_MOBIUS>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_MOBIUS>(arg);
	}
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	if (arg.type == DSLASH5_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == DSLASH5_MOBIUS_PRE) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == DSLASH5_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	}
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };


  template <typename Float, int nColor>
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
    Dslash5Arg<Float,nColor> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
    Dslash5<Float,nColor,Dslash5Arg<Float,nColor> > dslash(arg, in);
    dslash.apply(streams[Nstream-1]);
  }

  // template on the number of colors
  template <typename Float>
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
    switch(in.Ncolor()) {
    case 3: ApplyDslash5<Float,3>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    default: errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

#endif

  //Apply the 5th dimension dslash operator to a colorspinor field
  //out = Dslash5*in
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    if (in.DWFPCtype() != QUDA_4D_PC) errorQuda("Only 4-d preconditioned fields are supported");
    checkLocation(out, in);     // check all locations match

    switch(checkPrecision(out,in)) {
    case QUDA_DOUBLE_PRECISION: ApplyDslash5<double>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    case QUDA_SINGLE_PRECISION: ApplyDslash5<float> (out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    case QUDA_HALF_PRECISION:   ApplyDslash5<short> (out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    default: errorQuda("Unsupported precision %d\n", in.Precision());
    }
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }

} // namespace quda

