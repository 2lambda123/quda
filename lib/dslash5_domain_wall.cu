#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <dslash_quda.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <dslash_quda.h>
#include <inline_ptx.h>

namespace quda {

#ifdef GPU_DOMAIN_WALL_DIRAC

  /**
     @brief Parameter structure for applying the Dslash
   */
  template <typename Float, int nColor>
  struct Dslash5Arg {
    typedef typename colorspinor_mapper<Float,4,nColor>::type F;
    typedef typename mapper<Float>::type real;

    F out;                  // output vector field
    const F in;             // input vector field
    const F x;              // auxiliary input vector field
    const int nParity;      // number of parities we're working on
    const int volume_cb;    // checkerboarded volume
    const int volume_4d_cb; // 4-d checkerboarded volume
    const int_fastdiv Ls;   // length of 5th dimension

    const real m_f;         // fermion mass parameter
    const real m_5;         // Wilson mass shift

    const bool dagger;      // dagger
    const bool xpay;        // whether we are doing xpay or not

    // zMobius / Zolotarev coefficients
    complex<real> b_5[QUDA_MAX_DWF_LS];
    complex<real> c_5[QUDA_MAX_DWF_LS];

    // real constant Mobius coefficient
    double b;
    double c;

    // xpay coefficients
    real a;
    complex<real> a_5[QUDA_MAX_DWF_LS];

    Dslash5Type type;

    Dslash5Arg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
               double m_f, double m_5, const Complex *b_5_, const Complex *c_5_,
               double a, bool dagger, Dslash5Type type)
      : out(out), in(in), x(x), nParity(in.SiteSubset()),
	volume_cb(in.VolumeCB()), volume_4d_cb(volume_cb/in.X(4)), Ls(in.X(4)),
	m_f(m_f), m_5(m_5), a(a), dagger(dagger), xpay(in.V() == x.V() ? false: true), type(type)
    {
      if (in.Nspin() != 4) errorQuda("nSpin = %d not support", in.Nspin());
      if (!in.isNative() || !out.isNative()) errorQuda("Unsupported field order out=%d in=%d\n", out.FieldOrder(), in.FieldOrder());

      switch(type) {
      case DSLASH5_DWF:
	// xpay
	for (int s=0; s<Ls; s++) {
	  a_5[s] = a;
	}
	break;
      case DSLASH5_MOBIUS_PRE:
	for (int s=0; s<Ls; s++) {
	  b_5[s] = b_5_[s].real();
	  c_5[s] = 0.5*c_5_[s].real();

	  // xpay
	  a_5[s] = (0.5/(b_5_[s]*(m_5+4.0) + 1.0)).real();
	  a_5[s] *= a_5[s] * static_cast<real>(a);
        }
	break;
      case DSLASH5_MOBIUS:
	for (int s=0; s<Ls; s++) {
	  b_5[s] = 1.0;
	  c_5[s] = (0.5 * (c_5_[s] * (m_5 + 4.0) - 1.0) / (b_5_[s] * (m_5 + 4.0) + 1.0)).real();

	  // axpy
	  a_5[s] = (0.5 / (b_5_[s] * (m_5 + 4.0) + 1.0)).real();
	  a_5[s] *= a_5[s] * static_cast<real>(a);
	}
	break;
      default:
	errorQuda("Unknown Dslash5Type %d", type);
      }
      b = b_5[0].real();
      c = c_5[0].real();
    }
  };

  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  __device__ __host__ inline void dslash5(Arg &arg, int parity, int x_cb, int s) {
    typedef typename mapper<Float>::type real;
    typedef ColorSpinor<real,nColor,4> Vector;

    Vector out;

    { // forwards direction
      const int fwd_idx = ((s + 1) % arg.Ls) * arg.volume_4d_cb + x_cb;
      const Vector in = arg.in(fwd_idx, parity);
      constexpr int proj_dir = dagger ? +1 : -1;
      if (s == arg.Ls-1) {
	out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
      } else {
	out += in.project(4, proj_dir).reconstruct(4, proj_dir);
      }
    }

    { // backwards direction
      const int back_idx = ((s + arg.Ls - 1) % arg.Ls) * arg.volume_4d_cb + x_cb;
      const Vector in = arg.in(back_idx, parity);
      constexpr int proj_dir = dagger ? -1 : +1;
      if (s == 0) {
	out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
      } else {
	out += in.project(4, proj_dir).reconstruct(4, proj_dir);
      }
    }

    if (type == DSLASH5_DWF && xpay) {
      Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
      out = x + arg.a*out;
    } else if (type == DSLASH5_MOBIUS_PRE) {
      Vector diagonal = arg.in(s*arg.volume_4d_cb + x_cb, parity);
      const complex<real> b = arg.b; // arg.b_5[s]
      const complex<real> c = arg.c; // arg.c_5[s]
      out = c * out + b * diagonal;

      if (xpay) {
	Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
        complex<real> a = arg.a; // arg.a_5[s]
	out = x + a*out;
      }
    } else if (type == DSLASH5_MOBIUS) {
      Vector diagonal = arg.in(s*arg.volume_4d_cb + x_cb, parity);
      const complex<real> c = arg.c; // arg.c_5[s]
      out = c * out + diagonal;

      if (xpay) { // really axpy
	Vector x = arg.x(s*arg.volume_4d_cb + x_cb, parity);
        complex<real> a = arg.a; // arg.a_5[s]
	out = a*x + out;
      }
    }

    arg.out(s*arg.volume_4d_cb + x_cb, parity) = out;
  }

  // CPU kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  void dslash5CPU(Arg &arg)
  {
    for (int parity= 0; parity < arg.nParity; parity++) {
      for (int s=0; s < arg.Ls; s++) {
	for (int x_cb = 0; x_cb < arg.volume_4d_cb; x_cb++) { // 4-d volume
	  dslash5<Float,nColor,dagger,xpay,type>(arg, parity, x_cb, s);
	}  // 4-d volumeCB
      } // ls
    } // parity

  }

  // GPU Kernel for applying the dslash operator
  template <typename Float, int nColor, bool dagger, bool xpay, Dslash5Type type, typename Arg>
  __global__ void dslash5GPU(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int s = blockIdx.y*blockDim.y + threadIdx.y;
    int parity = blockIdx.z*blockDim.z + threadIdx.z;

    if (x_cb >= arg.volume_4d_cb) return;
    if (s >= arg.Ls) return;
    if (parity >= arg.nParity) return;

    dslash5<Float,nColor,dagger,xpay,type>(arg, parity, x_cb, s);
  }

  template <typename Float, int nColor, typename Arg>
  class Dslash5 : public TunableVectorYZ {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const {
      long long Ls = meta.X(4);
      long long bulk = (Ls-2)*(meta.Volume()/Ls);
      long long wall = 2*meta.Volume()/Ls;
      int n = meta.Ncolor() * meta.Nspin();
      bool zMobius = false; // set to true when we have complexity

      long long flops_ = 0;
      switch (arg.type) {
      case DSLASH5_DWF:
        flops_ = n * (8ll*bulk + 10ll*wall + (arg.xpay ? 4ll * meta.Volume() : 0) );
        break;
      case DSLASH5_MOBIUS_PRE:
        flops_ = n * (8ll*bulk + 10ll*wall + (zMobius ? 14ll : 6ll) * meta.Volume() +
                      (arg.xpay ? (zMobius ? 8ll : 4ll) * meta.Volume() : 0) );
        break;
      case DSLASH5_MOBIUS:
        flops_ = n * (8ll*bulk + 10ll*wall + (zMobius ? 8ll : 4ll) * meta.Volume() +
                      (arg.xpay ? (zMobius ? 8ll : 4ll) * meta.Volume() : 0) );
        break;
      default:
	errorQuda("Unknown Dslash5Type %d", arg.type);
      }

      return flops_;
    }

    long long bytes() const {
      switch (arg.type) {
      case DSLASH5_DWF:        return arg.out.Bytes() + 2*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case DSLASH5_MOBIUS_PRE: return arg.out.Bytes() + 3*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      case DSLASH5_MOBIUS:     return arg.out.Bytes() + 3*arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
      default: errorQuda("Unknown Dslash5Type %d", arg.type);
      }
      return 0ll;
    }

    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volume_4d_cb; }
    int blockStep() const { return 8; }
    int blockMin() const { return 8; }

  public:
    Dslash5(Arg &arg, const ColorSpinorField &meta)
      : TunableVectorYZ(arg.Ls, arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
      if (arg.dagger) strcat(aux, ",Dagger");
      if (arg.xpay) strcat(aux,",xpay");
      strcat(aux, arg.type == DSLASH5_DWF ? ",DSLASH5_DWF" :
             arg.type == DSLASH5_MOBIUS_PRE ? ",DSLASH5_MOBIUS_PRE" : ",DSLASH5_MOBIUS");
    }
    virtual ~Dslash5() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	if (arg.type == DSLASH5_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true,true,DSLASH5_DWF>(arg) :
			  dslash5CPU<Float,nColor,false,true,DSLASH5_DWF>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_DWF>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_DWF>(arg);
	} else if (arg.type == DSLASH5_MOBIUS_PRE) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true, true,DSLASH5_MOBIUS_PRE>(arg) :
			  dslash5CPU<Float,nColor,false, true,DSLASH5_MOBIUS_PRE>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_MOBIUS_PRE>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_MOBIUS_PRE>(arg);
	} else if (arg.type == DSLASH5_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5CPU<Float,nColor, true, true,DSLASH5_MOBIUS>(arg) :
			  dslash5CPU<Float,nColor,false, true,DSLASH5_MOBIUS>(arg);
	  else          arg.dagger ?
			  dslash5CPU<Float,nColor, true,false,DSLASH5_MOBIUS>(arg) :
			  dslash5CPU<Float,nColor,false,false,DSLASH5_MOBIUS>(arg);
	}
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	if (arg.type == DSLASH5_DWF) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_DWF> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == DSLASH5_MOBIUS_PRE) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_MOBIUS_PRE> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	} else if (arg.type == DSLASH5_MOBIUS) {
	  if (arg.xpay) arg.dagger ?
			  dslash5GPU<Float,nColor, true, true,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false, true,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	  else          arg.dagger ?
			  dslash5GPU<Float,nColor, true,false,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg) :
			  dslash5GPU<Float,nColor,false,false,DSLASH5_MOBIUS> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	}
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };


  template <typename Float, int nColor>
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
    Dslash5Arg<Float,nColor> arg(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
    Dslash5<Float,nColor,Dslash5Arg<Float,nColor> > dslash(arg, in);
    dslash.apply(streams[Nstream-1]);
  }

  // template on the number of colors
  template <typename Float>
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
    switch(in.Ncolor()) {
    case 3: ApplyDslash5<Float,3>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    default: errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

#endif

  //Apply the 5th dimension dslash operator to a colorspinor field
  //out = Dslash5*in
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x,
		    double m_f, double m_5, const Complex *b_5, const Complex *c_5,
		    double a, bool dagger, Dslash5Type type)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    if (in.DWFPCtype() != QUDA_4D_PC) errorQuda("Only 4-d preconditioned fields are supported");
    checkLocation(out, in);     // check all locations match

    switch(checkPrecision(out,in)) {
    case QUDA_DOUBLE_PRECISION: ApplyDslash5<double>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    case QUDA_SINGLE_PRECISION: ApplyDslash5<float> (out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    case QUDA_HALF_PRECISION:   ApplyDslash5<short> (out, in, x, m_f, m_5, b_5, c_5, a, dagger, type); break;
    default: errorQuda("Unsupported precision %d\n", in.Precision());
    }
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }

} // namespace quda

