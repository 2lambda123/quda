#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <typeinfo>

#include <color_spinor_field.h>
#include <blas_quda.h>

#include <string.h>
#include <iostream>
#include <misc_helpers.h>
#include <face_quda.h>
#include <dslash_quda.h>

#ifdef DEVICE_PACK
#define REORDER_LOCATION QUDA_CUDA_FIELD_LOCATION
#else
#define REORDER_LOCATION QUDA_CPU_FIELD_LOCATION
#endif

int zeroCopy = 0;

namespace quda {

  int cudaColorSpinorField::bufferIndex = 0;
  int cudaColorSpinorField::initGhostFaceBuffer = 0;
  void* cudaColorSpinorField::ghostFaceBuffer[2]; //gpu memory
  void* cudaColorSpinorField::fwdGhostFaceBuffer[2][QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  void* cudaColorSpinorField::backGhostFaceBuffer[2][QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  int cudaColorSpinorField::fwdGhostBufferOffset[2][QUDA_MAX_DIM];
  int cudaColorSpinorField::backGhostBufferOffset[2][QUDA_MAX_DIM];
  size_t cudaColorSpinorField::ghostFaceBytes = 0;

#ifdef P2P_COMMS
  MsgHandle* cudaColorSpinorField::mh_send_p2p_fwd[QUDA_MAX_DIM];
  MsgHandle* cudaColorSpinorField::mh_recv_p2p_fwd[QUDA_MAX_DIM];
  MsgHandle* cudaColorSpinorField::mh_send_p2p_back[QUDA_MAX_DIM];
  MsgHandle* cudaColorSpinorField::mh_recv_p2p_back[QUDA_MAX_DIM];
//  bool cudaColorSpinorField::initIPCDslashComms = false;

  // Need events to coordinate IPC memory copies
  hipEvent_t cudaColorSpinorField::ipcCopyEvent[2][QUDA_MAX_DIM];
  hipEvent_t cudaColorSpinorField::ipcRemoteCopyEvent[2][QUDA_MAX_DIM];

  // Pointers to ghost buffers on the local process and remote processes
  hipIpcMemHandle_t cudaColorSpinorField::ipcLocalGhostBufferHandle[2][2][QUDA_MAX_DIM];
  hipIpcMemHandle_t cudaColorSpinorField::ipcRemoteGhostBufferHandle[2][2][QUDA_MAX_DIM];

  void* cudaColorSpinorField::fwdGhostFaceSrcBuffer[2][QUDA_MAX_DIM];
  void* cudaColorSpinorField::backGhostFaceSrcBuffer[2][QUDA_MAX_DIM];


  hipIpcEventHandle_t cudaColorSpinorField::ipcLocalEventHandle[2][QUDA_MAX_DIM];
  hipIpcEventHandle_t cudaColorSpinorField::ipcRemoteEventHandle[2][QUDA_MAX_DIM];

#endif // P2P_COMMS
  /*cudaColorSpinorField::cudaColorSpinorField() : 
    ColorSpinorField(), v(0), norm(0), alloc(false), init(false) {

    }*/

#ifdef P2P_COMMS	

  void cudaColorSpinorField::createIPCDslashComms(){

    if(initIPCDslashComms) return;

    if(!initComms) errorQuda("Can only be called after create comms\n");

    comm_dslash_peer2peer_init();

    checkCudaError();
    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      for(int dir=0; dir<2; ++dir){
        MsgHandle* sendHandle = NULL;
	MsgHandle* receiveHandle = NULL;
	int disp = (dir == 1) ? +1 : -1;

	// first set up receive
	if(comm_dslash_peer2peer_enabled(1-dir,dim)){
          receiveHandle = comm_declare_receive_relative(&ipcRemoteGhostDestHandle[1-dir][dim],
							dim, 
							-disp,
							sizeof(ipcRemoteGhostDestHandle[1-dir][dim]));
	}
        // now send
	if(comm_dslash_peer2peer_enabled(dir,dim)) {
	  hipIpcGetMemHandle(&ipcLocalGhostDestHandle[dir][dim], ghost_field);
	  sendHandle = comm_declare_send_relative(&ipcLocalGhostDestHandle[dir][dim],
						  dim,
				                  disp,
						  sizeof(ipcLocalGhostDestHandle[dir][dim]));
	}
	if(receiveHandle) comm_start(receiveHandle);
  	if(sendHandle) comm_start(sendHandle);
	  
	if(receiveHandle) comm_wait(receiveHandle);
	if(sendHandle) comm_wait(sendHandle);

	if(sendHandle) comm_free(sendHandle);
	if(receiveHandle) comm_free(receiveHandle);
      }
    }

    checkCudaError();
    // open the remote memory handles 
    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      const int num_dir = (comm_dim(dim) == 2) ? 1 : 2;
      for(int dir=0; dir<num_dir; ++dir){
	if(!comm_dslash_peer2peer_enabled(dir,dim)) continue;
	void** ghostDest = (dir==0) ? (&backGhostSendDest[dim]) 
			 : &(fwdGhostSendDest[dim]);
	hipIpcOpenMemHandle(ghostDest, ipcRemoteGhostDestHandle[dir][dim],
		hipIpcMemLazyEnablePeerAccess);
      }
      if(num_dir == 1){
        fwdGhostSendDest[dim] = backGhostSendDest[dim];
      }
    }
   
    checkCudaError();
 
    // Note that the events can and probably should be static. 
    // We don't want a proliferation of events, I don't think
    // Also note that no b index is necessary here 
    // Now communicate the event handles
    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      for(int dir=0; dir<2; ++dir){
        MsgHandle* sendHandle = NULL;
	MsgHandle* receiveHandle = NULL;
	int disp = (dir == 1) ? +1 : -1;
		
	checkCudaError();
	// first set up receive
	if(comm_dslash_peer2peer_enabled(1-dir,dim)){
          receiveHandle = comm_declare_receive_relative(&ipcRemoteEventHandle[1-dir][dim],
							  dim, 
							 -disp,
							  sizeof(ipcRemoteEventHandle[1-dir][dim]));
	}

	checkCudaError();
          // now send
	if(comm_dslash_peer2peer_enabled(dir,dim)) {
	  hipEventCreate(&ipcCopyEvent[dir][dim], hipEventDisableTiming | hipEventInterprocess); 
	  hipIpcGetEventHandle(&(ipcLocalEventHandle[dir][dim]), ipcCopyEvent[dir][dim]);


	  sendHandle = comm_declare_send_relative(&ipcLocalEventHandle[dir][dim],
						  dim,
				                  disp,
						  sizeof(ipcLocalEventHandle[dir][dim]));
	}
	if(receiveHandle) comm_start(receiveHandle);
  	if(sendHandle) comm_start(sendHandle);
	  
	if(receiveHandle) comm_wait(receiveHandle);
	if(sendHandle) comm_wait(sendHandle);

	if(sendHandle) comm_free(sendHandle);
	if(receiveHandle) comm_free(receiveHandle);
      }
    }
    checkCudaError();
    // the b index is completely superfluous here since the buffers aren't static
    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      for(int dir=0; dir<2; ++dir){
	if(!comm_dslash_peer2peer_enabled(dir,dim)) continue;
	hipIpcOpenEventHandle(&(ipcRemoteCopyEvent[dir][dim]), ipcRemoteEventHandle[dir][dim]);
      }
    }

    // Create message handles for IPC synchronization
    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      if(comm_dslash_peer2peer_enabled(1,dim)){
        int dummy;
	// send to processor in forward direction 
	mh_send_p2p_fwd[dim] = comm_declare_send_relative(&dummy,dim,+1,sizeof(int));
	// receive from processor in forward directin
	mh_recv_p2p_fwd[dim] = comm_declare_receive_relative(&dummy, dim, +1, sizeof(int));
      }

      if(comm_dslash_peer2peer_enabled(0,dim)){
        int dummy;
	// send to processor in backward direction
	mh_send_p2p_back[dim] = comm_declare_send_relative(&dummy,dim,-1,sizeof(int));
	// receive from processor in backward direction
	mh_recv_p2p_back[dim] = comm_declare_receive_relative(&dummy,dim,-1,sizeof(int));
      }
    }
    checkCudaError();

    initIPCDslashComms = true;
  }

/*
  void cudaColorSpinorField::createIPCDslashComms(){

    if(!initIPCTimeComms){
      if(commDimPartitioned(3) && !getKernelPackT()){
	
        if(!initComms) errorQuda("Can only be called after create comms\n");

        comm_dslash_peer2peer_init();
	const int num_dir = (comm_dim(3) == 2) ? 1 : 2;

	// first set up receive
	for(int dir=0; dir<2; ++dir){
	  MsgHandle* sendHandle = NULL;
	  MsgHandle* receiveHandle = NULL;
	  MsgHandle* sendNormHandle = NULL;
	  MsgHandle* receiveNormHandle = NULL;
	  int disp = (dir==1) ? +1 : -1;

	  // first set up receive 
	  if(comm_dslash_peer2peer_enabled(1-dir,3)){
	    receiveHandle = comm_declare_receive_relative(&ipcRemoteFieldHandle[1-dir],
							  3, 
						          -disp, 
							  sizeof(ipcRemoteFieldHandle[1-dir]));
	    if(precision == QUDA_HALF_PRECISION){
	      receiveNormHandle = comm_declare_receive_relative(&ipcRemoteNormHandle[1-dir],
							   3,
						          -disp,
							   sizeof(ipcRemoteNormHandle[1-dir]));
	    }
	  } 


	  // Now for send 
	  if(comm_dslash_peer2peer_enabled(dir,3)){
	    hipIpcGetMemHandle(&ipcLocalFieldHandle[dir],v);
	    sendHandle = comm_declare_send_relative(&ipcLocalFieldHandle[dir],
						    3,
						    disp,
						    sizeof(ipcLocalFieldHandle[dir]));
	    if(precision == QUDA_HALF_PRECISION){

	   	hipIpcGetMemHandle(&ipcLocalNormHandle[dir],norm);
	    	sendNormHandle = comm_declare_send_relative(&ipcLocalNormHandle[dir],
						    				3,
						    				disp,
										sizeof(ipcLocalNormHandle[dir]));

	    }
	  }

	  if(receiveHandle) comm_start(receiveHandle);
	  if(sendHandle) comm_start(sendHandle);

	  if(receiveHandle) comm_wait(receiveHandle);
	  if(sendHandle) comm_wait(sendHandle);

	  if(sendHandle) comm_free(sendHandle);
	  if(receiveHandle) comm_free(receiveHandle);

	  if(precision == QUDA_HALF_PRECISION){
	    if(receiveNormHandle) comm_start(receiveNormHandle);
	    if(sendNormHandle) comm_start(sendNormHandle);

	    if(receiveNormHandle) comm_wait(receiveNormHandle);
	    if(sendNormHandle) comm_wait(sendNormHandle);

	    if(sendNormHandle) comm_free(sendNormHandle);
	    if(receiveNormHandle) comm_free(receiveNormHandle);
	  }
 	} // loop over dir	
	// Next, need to open the exported mem handles

	for(int dir=0; dir<num_dir; ++dir){	
	  if(!comm_dslash_peer2peer_enabled(dir,3)) continue;
	  void** remoteFieldSrcBuffer = (dir==0) ? &fwdFieldSrcBuffer : &backFieldSrcBuffer;
	  hipIpcOpenMemHandle(remoteFieldSrcBuffer, 
			       ipcRemoteFieldHandle[dir], 
			       hipIpcMemLazyEnablePeerAccess);

	  if(precision == QUDA_HALF_PRECISION){
	    void** remoteNormSrcBuffer = (dir==0) ? &fwdNormSrcBuffer : &backNormSrcBuffer;
	    hipIpcOpenMemHandle(remoteNormSrcBuffer, 
			       ipcRemoteNormHandle[dir], 
			       hipIpcMemLazyEnablePeerAccess);
	  }
	}
	if(num_dir == 1){
	  backFieldSrcBuffer = fwdFieldSrcBuffer;
	  backNormSrcBuffer = fwdNormSrcBuffer;
	}
        initIPCTimeComms = true;
      }
    } // communication in time



  
    if(initIPCDslashComms) return;

    if(!initComms) errorQuda("Can only be called after create comms\n");

    comm_dslash_peer2peer_init();


    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      if(dim == 3 && !getKernelPackT()) continue;
      const int num_dir = (comm_dim(dim) == 2) ? 1 : 2;
      for(int dir=0; dir<2; ++dir){
        for(int b=0; b<2; ++b){
	  MsgHandle* sendHandle = NULL;
	  MsgHandle* receiveHandle = NULL;
	  int disp = (dir==1) ? +1 : -1;

          // first set up receive
	  if(comm_dslash_peer2peer_enabled(1-dir,dim)){
	    receiveHandle = comm_declare_receive_relative(&ipcRemoteGhostBufferHandle[b][1-dir][dim],
						          dim,
							  -disp,
							  sizeof(ipcRemoteGhostBufferHandle[b][1-dir][dim]));
	  }	  
          // Now for send
	  if(comm_dslash_peer2peer_enabled(dir,dim)){
	    //void* ghost_buffer = (dir==0) ? backGhostFaceBuffer[b][dim] : fwdGhostFaceBuffer[b][dim];
	    void* ghost_buffer = ghostFaceBuffer[b];


            hipIpcGetMemHandle(&ipcLocalGhostBufferHandle[b][dir][dim], ghost_buffer);
	    sendHandle = comm_declare_send_relative(&ipcLocalGhostBufferHandle[b][dir][dim],
						    dim,
						    disp,
						    sizeof(ipcLocalGhostBufferHandle[b][dir][dim]));
	  }


	  if(receiveHandle) comm_start(receiveHandle);
	  if(sendHandle) comm_start(sendHandle);

          if(receiveHandle) comm_wait(receiveHandle);
	  if(sendHandle) comm_wait(sendHandle);

	  if(sendHandle) comm_free(sendHandle);
	  if(receiveHandle) comm_free(receiveHandle);	
	
        } // loop over b
      } // loop over dir (0,1)
    } // loop over dim



    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      if(dim == 3 && !getKernelPackT()) continue;
      const int num_dir = (comm_dim(dim) == 2) ? 1 : 2;
      for(int dir=0; dir<num_dir; ++dir){
        if(!comm_dslash_peer2peer_enabled(dir,dim)) continue;
	for(int b=0; b<2; ++b){
	  void** remoteGhostSrcBuffer = (dir==0) ? &(fwdGhostFaceSrcBuffer[b][dim]) 
					: &(backGhostFaceSrcBuffer[b][dim]);

	  hipError_t result = hipIpcOpenMemHandle(remoteGhostSrcBuffer, ipcRemoteGhostBufferHandle[b][dir][dim], 
						     hipIpcMemLazyEnablePeerAccess);
	}
      }
      if(num_dir == 1){
	for(int b=0; b<2; ++b){ 
	  backGhostFaceSrcBuffer[b][dim] = fwdGhostFaceSrcBuffer[b][dim];
        }
      }
    }

    // Create local events for asynchronous copies from the peer process
    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      for(int dir=0; dir<2; ++dir){
        if(!comm_dslash_peer2peer_enabled(dir,dim)) continue;
	for(int b=0; b<2; ++b){
	  hipEventCreate(&ipcCopyEvent[b][dir][dim]);
	//  hipEventCreate(&ipcCopyEvent[b][dir][dim]);
	}
      }
    }


    // Create message handles for IPC synchronization
    for(int dim=0; dim<4; ++dim){
      if(!commDimPartitioned(dim)) continue;
      if(comm_dslash_peer2peer_enabled(1,dim)){
	int dummy = dim;
	for(int b=0; b<2; ++b){
	  // send to processor in forward direction
	  mh_send_p2p_fwd[b][dim] = comm_declare_send_relative(&dummy,dim,+1,sizeof(int));
	  // receive from processor in forward direction
	  mh_recv_p2p_fwd[b][dim] = comm_declare_receive_relative(&dummy,dim,+1,sizeof(int)); 

	}
      }

      if(comm_dslash_peer2peer_enabled(0,dim)){
	int dummy;
	for(int b=0; b<2; ++b){
	  // send to processor in backward direction
	  mh_send_p2p_back[b][dim] = comm_declare_send_relative(&dummy,dim,-1,sizeof(int));
	  // receive from processor in backward direction
	  mh_recv_p2p_back[b][dim] = comm_declare_receive_relative(&dummy,dim,-1,sizeof(int));
	}
      }
    }


    initIPCDslashComms = true;
  }
*/
#endif // P2P_COMMS



  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorParam &param) : 
    ColorSpinorField(param), alloc(false), init(true), texInit(false), 
    initComms(false), bufferMessageHandler(0), nFaceComms(0) {

#ifdef P2P_COMMS
    initIPCDslashComms = false;
#endif

    // this must come before create
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      v = param.v;
      norm = param.norm;
    }

    create(param.create);

    if  (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // dp nothing
    } else if (param.create == QUDA_COPY_FIELD_CREATE){
      errorQuda("not implemented");
    }
  }

  cudaColorSpinorField::cudaColorSpinorField(const cudaColorSpinorField &src) : 
    ColorSpinorField(src), alloc(false), init(true), texInit(false), 
    initComms(false), bufferMessageHandler(0), nFaceComms(0) {
#ifdef P2P_COMM
    initIPCDslashComms = false;
#endif
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  // creates a copy of src, any differences defined in param
  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src, 
					     const ColorSpinorParam &param) :
    ColorSpinorField(src), alloc(false), init(true), texInit(false), 
    initComms(false), bufferMessageHandler(0), nFaceComms(0) {  
#ifdef P2P_COMMS
    initIPCDslashComms = false;
#endif
    // can only overide if we are not using a reference or parity special case
    if (param.create != QUDA_REFERENCE_FIELD_CREATE || 
	(param.create == QUDA_REFERENCE_FIELD_CREATE && 
	 src.SiteSubset() == QUDA_FULL_SITE_SUBSET && 
	 param.siteSubset == QUDA_PARITY_SITE_SUBSET && 
	 typeid(src) == typeid(cudaColorSpinorField) ) || 
         (param.create == QUDA_REFERENCE_FIELD_CREATE && param.eigv_dim > 0)) {
      reset(param);
    } else {
      errorQuda("Undefined behaviour"); // else silent bug possible?
    }

    // This must be set before create is called
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      if (typeid(src) == typeid(cudaColorSpinorField)) {
	v = (void*)src.V();
	norm = (void*)src.Norm();
      } else {
	errorQuda("Cannot reference a non-cuda field");
      }

      if (this->EigvDim() > 0) 
      {//setup eigenvector form the set
         if(eigv_dim != this->EigvDim()) errorQuda("\nEigenvector set does not match..\n") ;//for debug only.
         if(eigv_id > -1)
         {
           //printfQuda("\nSetting pointers for vector id %d\n", eigv_id); //for debug only.
           v    = (void*)((char*)v + eigv_id*bytes);         
           norm = (void*)((char*)norm + eigv_id*norm_bytes);         
         }
       //do nothing for the eigenvector subset...
      }
    }

    create(param.create);

    if (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      copySpinorField(src);
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else {
      errorQuda("CreateType %d not implemented", param.create);
    }

  }

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src) 
    : ColorSpinorField(src), alloc(false), init(true), texInit(false), 
      initComms(false), bufferMessageHandler(0), nFaceComms(0) {

#ifdef P2P_COMMS
    initIPCDslashComms = false;
#endif
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  ColorSpinorField& cudaColorSpinorField::operator=(const ColorSpinorField &src) {
    if (typeid(src) == typeid(cudaColorSpinorField)) {
      *this = (dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cpuColorSpinorField)) {
      *this = (dynamic_cast<const cpuColorSpinorField&>(src));
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cudaColorSpinorField &src) {
    if (&src != this) {
      // keep current attributes unless unset
      if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
	destroy();
	destroyComms(); // not sure if this necessary
	ColorSpinorField::operator=(src);
	create(QUDA_COPY_FIELD_CREATE);
      }
      copySpinorField(src);
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cpuColorSpinorField &src) {
    // keep current attributes unless unset
    if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
      destroy();
      ColorSpinorField::operator=(src);
      create(QUDA_COPY_FIELD_CREATE);
    }
    loadSpinorField(src);
    return *this;
  }

  cudaColorSpinorField::~cudaColorSpinorField() {
    destroyComms();
    destroy();
  }

  bool cudaColorSpinorField::isNative() const {

    if (precision == QUDA_DOUBLE_PRECISION) {
      if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
    } else if (precision == QUDA_SINGLE_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    } else if (precision == QUDA_HALF_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    }

    return false;
  }

  void cudaColorSpinorField::create(const QudaFieldCreate create) {

    if (siteSubset == QUDA_FULL_SITE_SUBSET && siteOrder != QUDA_EVEN_ODD_SITE_ORDER) {
      errorQuda("Subset not implemented");
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      v = device_malloc(bytes);
      if(ghost_bytes) ghost_field = device_malloc(ghost_bytes);
      if (precision == QUDA_HALF_PRECISION) {
	norm = device_malloc(norm_bytes);
      }
      alloc = true;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {

      printfQuda("QUDA_FULL_SITE_SUBSET\n");

      if(eigv_dim != 0) errorQuda("Eigenvectors must be parity fields!");
      // create the associated even and odd subsets
      ColorSpinorParam param;
      param.siteSubset = QUDA_PARITY_SITE_SUBSET;
      param.nDim = nDim;
      memcpy(param.x, x, nDim*sizeof(int));
      param.x[0] /= 2; // set single parity dimensions
      param.create = QUDA_REFERENCE_FIELD_CREATE;
      param.v = v;
      param.norm = norm;
      even = new cudaColorSpinorField(*this, param);
      odd = new cudaColorSpinorField(*this, param);

      // need this hackery for the moment (need to locate the odd pointers half way into the full field)
      (dynamic_cast<cudaColorSpinorField*>(odd))->v = (void*)((char*)v + bytes/2);
      if (precision == QUDA_HALF_PRECISION) 
	(dynamic_cast<cudaColorSpinorField*>(odd))->norm = (void*)((char*)norm + norm_bytes/2);

      for(int i=0; i<nDim; ++i){
        if(commDimPartitioned(i)){
          (dynamic_cast<cudaColorSpinorField*>(odd))->ghost[i] =
	    static_cast<char*>((dynamic_cast<cudaColorSpinorField*>(odd))->ghost[i]) + bytes/2;
          if(precision == QUDA_HALF_PRECISION)
	    (dynamic_cast<cudaColorSpinorField*>(odd))->ghostNorm[i] =
	      static_cast<char*>((dynamic_cast<cudaColorSpinorField*>(odd))->ghostNorm[i]) + norm_bytes/2;
        }
      }

#ifdef USE_TEXTURE_OBJECTS
      dynamic_cast<cudaColorSpinorField*>(even)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(even)->createTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->createTexObject();
#endif
    }
    else{//siteSubset == QUDA_PARITY_SITE_SUBSET

      //! setup an object for selected eigenvector (the 1st one as a default):
      if ((eigv_dim > 0) && (create != QUDA_REFERENCE_FIELD_CREATE) && (eigv_id == -1)) 
      {
         //if(bytes > 1811939328) warningQuda("\nCUDA API probably won't be able to create texture object for the eigenvector set... Object size is : %u bytes\n", bytes);
         if (getVerbosity() == QUDA_DEBUG_VERBOSE) printfQuda("\nEigenvector set constructor...\n");
         // create the associated even and odd subsets
         ColorSpinorParam param;
         param.siteSubset = QUDA_PARITY_SITE_SUBSET;
         param.nDim = nDim;
         memcpy(param.x, x, nDim*sizeof(int));
         param.create = QUDA_REFERENCE_FIELD_CREATE;
         param.v = v;
         param.norm = norm;
         param.eigv_dim  = eigv_dim;
         //reserve eigvector set
         eigenvectors.reserve(eigv_dim);
         //setup volume, [real_]length and stride for a single eigenvector
         for(int id = 0; id < eigv_dim; id++)
         {
            param.eigv_id = id;
            eigenvectors.push_back(new cudaColorSpinorField(*this, param));

#ifdef USE_TEXTURE_OBJECTS //(a lot of texture objects...)
            dynamic_cast<cudaColorSpinorField*>(eigenvectors[id])->destroyTexObject();
            dynamic_cast<cudaColorSpinorField*>(eigenvectors[id])->createTexObject();
#endif
         }
      }
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
	zeroPad();
      } else {
	(dynamic_cast<cudaColorSpinorField*>(even))->zeroPad();
	(dynamic_cast<cudaColorSpinorField*>(odd))->zeroPad();
      }
    }

#ifdef USE_TEXTURE_OBJECTS
    if((eigv_dim == 0) || (eigv_dim > 0 && eigv_id > -1))
       createTexObject();
#endif

    // initialize the ghost pointers 
    if(siteSubset == QUDA_PARITY_SITE_SUBSET) {
      for(int i=0; i<nDim; ++i){
        if(commDimPartitioned(i)){
          //ghost[i] = (char*)v + (stride*nColor*nSpin*2 + ghostOffset[i])*precision;
          ghost[i] = (char*)ghost_field + ghostOffset[i][0]*precision;
          if(precision == QUDA_HALF_PRECISION)
            //ghostNorm[i] = (char*)norm + (stride + ghostNormOffset[i])*QUDA_SINGLE_PRECISION;
            ghostNorm[i] = (char*)ghost_field + ghostNormOffset[i][0]*QUDA_SINGLE_PRECISION;
        }
      }
    }
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaColorSpinorField::createTexObject() {

    if (isNative()) {
      if (texInit) errorQuda("Already bound textures");
      
      // create the texture for the field components
      
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2
      
      // staggered fields in half and single are always two component
      if (nSpin == 1 && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	desc.x = 8*precision;
	desc.y = 8*precision;
	desc.z = 0;
	desc.w = 0;
      } else { // all others are four component
	desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
      }
      
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = v;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = bytes;
      
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;
      
      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

      // create the ghost texture object
     if(ghost_bytes){

        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeLinear;
        resDesc.res.linear.devPtr = ghost_field;
        resDesc.res.linear.desc = desc;
        resDesc.res.linear.sizeInBytes = ghost_bytes;

        hipCreateTextureObject(&ghostTex, &resDesc, &texDesc, NULL);
      }
      // create the texture for the norm components
      if (precision == QUDA_HALF_PRECISION) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	desc.f = hipChannelFormatKindFloat;
	desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = norm;
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = norm_bytes;
	
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	
	hipCreateTextureObject(&texNorm, &resDesc, &texDesc, NULL);

        // Assign ghostTexNorm
        if(ghost_bytes){ 
          hipResourceDesc resDesc;
          memset(&resDesc, 0, sizeof(resDesc));
          resDesc.resType = hipResourceTypeLinear;
          resDesc.res.linear.devPtr = ghost_field;
          resDesc.res.linear.desc = desc;
          resDesc.res.linear.sizeInBytes = ghost_bytes;                    
      

	  hipCreateTextureObject(&ghostTexNorm, &resDesc, &texDesc, NULL);
        }
      }
      
      texInit = true;
    }
  }

  void cudaColorSpinorField::destroyTexObject() {
    if (isNative() && texInit) {
      hipDestroyTextureObject(tex);
      if(ghost_bytes)hipDestroyTextureObject(ghostTex);
      if (precision == QUDA_HALF_PRECISION){ 
        hipDestroyTextureObject(texNorm);
        if(ghost_bytes)hipDestroyTextureObject(ghostTexNorm);
      }
      texInit = false;
    }
  }
#endif

  void cudaColorSpinorField::destroy() {
    if (alloc) {
      device_free(v);
      if(ghost_bytes) device_free(ghost_field);
      if (precision == QUDA_HALF_PRECISION) device_free(norm);
      if (siteSubset == QUDA_FULL_SITE_SUBSET) {
	delete even;
	delete odd;
      }
      else{
        //! for deflated solvers:
        if (eigv_dim > 0) 
        {
          std::vector<ColorSpinorField*>::iterator vec;
          for(vec = eigenvectors.begin(); vec != eigenvectors.end(); vec++) delete *vec;
        } 
      }
      alloc = false;
    }

#ifdef USE_TEXTURE_OBJECTS
    if((eigv_dim == 0) || (eigv_dim > 0 && eigv_id > -1))
       destroyTexObject();
#endif

  }

  cudaColorSpinorField& cudaColorSpinorField::Even() const { 
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(even)); 
    }

    errorQuda("Cannot return even subset of %d subset", siteSubset);
    exit(-1);
  }

  cudaColorSpinorField& cudaColorSpinorField::Odd() const {
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(odd)); 
    }

    errorQuda("Cannot return odd subset of %d subset", siteSubset);
    exit(-1);
  }

  // cuda's floating point format, IEEE-754, represents the floating point
  // zero as 4 zero bytes
  void cudaColorSpinorField::zero() {
    hipMemsetAsync(v, 0, bytes, streams[Nstream-1]);
    if (precision == QUDA_HALF_PRECISION) hipMemsetAsync(norm, 0, norm_bytes, streams[Nstream-1]);
  }


  void cudaColorSpinorField::zeroPad() {
    size_t pad_bytes = (stride - volume) * precision * fieldOrder;
    int Npad = nColor * nSpin * 2 / fieldOrder;

    if (eigv_dim > 0 && eigv_id == -1){//we consider the whole eigenvector set:
      Npad      *= eigv_dim;
      pad_bytes /= eigv_dim;
    }

    size_t pitch = ((eigv_dim == 0 || eigv_id != -1) ? stride : eigv_stride)*fieldOrder*precision;
    char   *dst  = (char*)v + ((eigv_dim == 0 || eigv_id != -1) ? volume : eigv_volume)*fieldOrder*precision;
    if(pad_bytes) hipMemset2D(dst, pitch, 0, pad_bytes, Npad);

    //for (int i=0; i<Npad; i++) {
    //  if (pad_bytes) hipMemset((char*)v + (volume + i*stride)*fieldOrder*precision, 0, pad_bytes);
    //}
  }

  void cudaColorSpinorField::copy(const cudaColorSpinorField &src) {
    checkField(*this, src);
    copyCuda(*this, src);
  }

  void cudaColorSpinorField::copySpinorField(const ColorSpinorField &src) {
    
    // src is on the device and is native
    if (typeid(src) == typeid(cudaColorSpinorField) && 
	isNative() && dynamic_cast<const cudaColorSpinorField &>(src).isNative()) {
      copy(dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else if (typeid(src) == typeid(cpuColorSpinorField)) { // src is on the host
      loadSpinorField(src);
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
  } 

  void cudaColorSpinorField::loadSpinorField(const ColorSpinorField &src) {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(src) == typeid(cpuColorSpinorField)) {
      for(int b=0; b<2; ++b){
        resizeBufferPinned(bytes + norm_bytes, b);
        memset(bufferPinned[b], 0, bytes+norm_bytes); // FIXME (temporary?) bug fix for padding
      }
      copyGenericColorSpinor(*this, src, QUDA_CPU_FIELD_LOCATION, 
			     bufferPinned[bufferIndex], 0, (char*)bufferPinned[bufferIndex]+bytes, 0);

      hipMemcpy(v, bufferPinned[bufferIndex], bytes, hipMemcpyHostToDevice);
      hipMemcpy(norm, (char*)bufferPinned[bufferIndex]+bytes, norm_bytes, hipMemcpyHostToDevice);
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *Src, *srcNorm;
      if (!zeroCopy) {
	resizeBufferDevice(src.Bytes()+src.NormBytes());
	Src = bufferDevice;
	srcNorm = (char*)bufferDevice + src.Bytes();	
	hipMemcpy(Src, src.V(), src.Bytes(), hipMemcpyHostToDevice);
	hipMemcpy(srcNorm, src.Norm(), src.NormBytes(), hipMemcpyHostToDevice);
      } else {
	for(int b=0; b<2; ++b){
	 resizeBufferPinned(src.Bytes()+src.NormBytes(), b);
	}
	memcpy(bufferPinned[bufferIndex], src.V(), src.Bytes());
	memcpy((char*)bufferPinned[bufferIndex]+src.Bytes(), src.Norm(), src.NormBytes());

	hipHostGetDevicePointer(&Src, bufferPinned[bufferIndex], 0);
	srcNorm = (void*)((char*)Src + src.Bytes());
      }

      hipMemset(v, 0, bytes); // FIXME (temporary?) bug fix for padding
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION, 0, Src, 0, srcNorm);
    }

    return;
  }


  void cudaColorSpinorField::saveSpinorField(ColorSpinorField &dest) const {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(dest) == typeid(cpuColorSpinorField)) {
      for(int b=0; b<2; ++b) resizeBufferPinned(bytes+norm_bytes,b);
      hipMemcpy(bufferPinned[bufferIndex], v, bytes, hipMemcpyDeviceToHost);
      hipMemcpy((char*)bufferPinned[bufferIndex]+bytes, norm, norm_bytes, hipMemcpyDeviceToHost);

      copyGenericColorSpinor(dest, *this, QUDA_CPU_FIELD_LOCATION, 
			     0, bufferPinned[bufferIndex], 0, (char*)bufferPinned[bufferIndex]+bytes);
    } else if (typeid(dest) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *dst, *dstNorm;
      if (!zeroCopy) {
	resizeBufferDevice(dest.Bytes()+dest.NormBytes());
	dst = bufferDevice;
	dstNorm = (char*)bufferDevice+dest.Bytes();
      } else {
	for(int b=0; b<2; ++b) resizeBufferPinned(dest.Bytes()+dest.NormBytes(),b);
	hipHostGetDevicePointer(&dst, bufferPinned[bufferIndex], 0);
	dstNorm = (char*)dst+dest.Bytes();
      }
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION, dst, v, dstNorm, 0);

      if (!zeroCopy) {
	hipMemcpy(dest.V(), dst, dest.Bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(dest.Norm(), dstNorm, dest.NormBytes(), hipMemcpyDeviceToHost);
      } else {
	memcpy(dest.V(), bufferPinned[bufferIndex], dest.Bytes());
	memcpy(dest.Norm(), (char*)bufferPinned[bufferIndex]+dest.Bytes(), dest.NormBytes());
      }
    }

    return;
  }

  void cudaColorSpinorField::allocateGhostBuffer(int nFace) {
    int Nint = nColor * nSpin * 2; // number of internal degrees of freedom
    if (nSpin == 4) Nint /= 2; // spin projection for Wilson

    // compute size of buffer required
    size_t faceBytes = 0;
    for (int i=0; i<4; i++) {
      if(!commDimPartitioned(i)) continue;
      faceBytes += 2*nFace*ghostFace[i]*Nint*precision;
      ghost_face_bytes[i] = nFace*ghostFace[i]*Nint*precision;
      // add extra space for the norms for half precision
      if (precision == QUDA_HALF_PRECISION){ 
        faceBytes += 2*nFace*ghostFace[i]*sizeof(float);
        ghost_face_bytes[i] += nFace*ghostFace[i]*sizeof(float);
      }
    }

    // only allocate if not already allocated or buffer required is bigger than previously
    if(initGhostFaceBuffer == 0 || faceBytes > ghostFaceBytes){    

      if (initGhostFaceBuffer){
        for(int b=0; b<2; ++b) device_free(ghostFaceBuffer[b]); 
      }

      if (faceBytes > 0) {
	for(int b=0; b<2; ++b) ghostFaceBuffer[b] = device_malloc(faceBytes);
	initGhostFaceBuffer = 1;
	ghostFaceBytes = faceBytes;
      }

    }

    size_t offset = 0;
    for (int i=0; i<4; i++) {
      if(!commDimPartitioned(i)) continue;
    
      for(int b=0; b<2; ++b){
	backGhostFaceBuffer[b][i] = (void*)(((char*)ghostFaceBuffer[b]) + offset);
#ifdef P2P_COMMS
	backGhostBufferOffset[b][i] = offset; 
#endif
      }
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
      
      for(int b=0; b<2; ++b){ 
	fwdGhostFaceBuffer[b][i] = (void*)(((char*)ghostFaceBuffer[b]) + offset);
#ifdef P2P_COMMS
	fwdGhostBufferOffset[b][i] = offset;
#endif
      }
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
    }   
    
  }


  void cudaColorSpinorField::freeGhostBuffer(void)
  {
    if (!initGhostFaceBuffer) return;
  
    for(int b=0; b<2; ++b) device_free(ghostFaceBuffer[b]); 

    for(int i=0;i < 4; i++){
      if(!commDimPartitioned(i)) continue;
      for(int b=0; b<2; ++b){
        backGhostFaceBuffer[b][i] = NULL;
        fwdGhostFaceBuffer[b][i] = NULL;
      }
    }
    initGhostFaceBuffer = 0;  
  }

  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(const int nFace, const QudaParity parity, 
                                       const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream, 
				       void *buffer, double a, double b) 
  {
#ifdef MULTI_GPU
    int face_num;
    if(dir == QUDA_BACKWARDS){
      face_num = 0;
    }else if(dir == QUDA_FORWARDS){
      face_num = 1;
    }else{
      face_num = 2;
    }
    void *packBuffer = buffer ? buffer : ghostFaceBuffer[bufferIndex];
    packFace(packBuffer, *this, nFace, dagger, parity, dim, face_num, *stream, a, b); 
#else
    errorQuda("packGhost not built on single-GPU build");
#endif

  }
 
  // send the ghost zone to the host
  void cudaColorSpinorField::sendGhost(void *ghost_spinor, const int nFace, const int dim, 
				       const QudaDirection dir, const int dagger, 
				       hipStream_t *stream) {

#ifdef MULTI_GPU
    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom
    
    if (dim !=3 || getKernelPackT() || getTwistPack()) { // use kernels to pack into contiguous buffers then a single hipMemcpy

      size_t bytes = nFace*Nint*ghostFace[dim]*precision;

      if (precision == QUDA_HALF_PRECISION) bytes += nFace*ghostFace[dim]*sizeof(float);

      void* gpu_buf = 
	(dir == QUDA_BACKWARDS) ? this->backGhostFaceBuffer[bufferIndex][dim] : this->fwdGhostFaceBuffer[bufferIndex][dim];

      hipMemcpyAsync(ghost_spinor, gpu_buf, bytes, hipMemcpyDeviceToHost, *stream); 
    } else if(this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET){ // do multiple cudaMemcpys

      int Npad = Nint / Nvec; // number Nvec buffers we have
      int Nt_minus1_offset = (volume - nFace*ghostFace[3]); // N_t -1 = Vh-Vsh
      int offset = 0;
      if (nSpin == 1) {
	offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset;
      } else if (nSpin == 4) {    
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back  
	if (dir == QUDA_FORWARDS) upper = !upper;
	int lower_spin_offset = Npad*stride;
	if (upper) offset = (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
	else offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
      }
    
      // QUDA Memcpy NPad's worth. 
      //  -- Dest will point to the right beginning PAD. 
      //  -- Each Pad has size Nvec*Vsh Floats. 
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + offset*Nvec*precision;
      size_t len = nFace*ghostFace[3]*Nvec*precision;     
      size_t spitch = stride*Nvec*precision;
      hipMemcpy2DAsync(dst, len, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
	int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + nFace*Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
	hipMemcpyAsync(dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToHost, *stream); 
      }
    }else{
      int flavorVolume = volume / 2;
      int flavorTFace  = ghostFace[3] / 2;
      int Npad = Nint / Nvec; // number Nvec buffers we have
      int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
      int flavor2_Nt_minus1_offset = (volume - flavorTFace);
      int flavor1_offset = 0;
      int flavor2_offset = 0;
      // !dagger: send lower components backwards, send upper components forwards
      // dagger: send upper components backwards, send lower components forwards
      bool upper = dagger ? true : false; // Fwd is !Back
      if (dir == QUDA_FORWARDS) upper = !upper;
      int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
      if (upper){
        flavor1_offset = (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }else{
        flavor1_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }

      // QUDA Memcpy NPad's worth.
      //  -- Dest will point to the right beginning PAD.
      //  -- Each Pad has size Nvec*Vsh Floats.
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + flavor1_offset*Nvec*precision;
      size_t len = flavorTFace*Nvec*precision;
      size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
      size_t dpitch = 2*len;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);
      dst = (char*)ghost_spinor+len;
      src = (char*)v + flavor2_offset*Nvec*precision;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
        int Nt_minus1_offset = (flavorVolume - flavorTFace);
        int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
        size_t dpitch = flavorTFace*sizeof(float);
        size_t spitch = flavorVolume*sizeof(float);
	hipMemcpy2DAsync(dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToHost, *stream);
      }
    }
#else
    errorQuda("sendGhost not built on single-GPU build");
#endif

  }



  void cudaColorSpinorField::unpackGhost(const void* ghost_spinor, const int nFace, 
					 const int dim, const QudaDirection dir, 
					 const int dagger, hipStream_t* stream) 
  {

    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint*precision;
    const void *src = ghost_spinor;
  
    int ghost_offset = (dir == QUDA_BACKWARDS) ? ghostOffset[dim][0] : ghostOffset[dim][1];
    void *ghost_dst = (char*)ghost_field + precision*ghost_offset;

    if(precision == QUDA_HALF_PRECISION) len += nFace*ghostFace[dim]*sizeof(float);

    hipMemcpyAsync(ghost_dst, src, len, hipMemcpyHostToDevice, *stream);
  }




   // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhostExtended(const int nFace, const int R[], const QudaParity parity,
                                       const int dim, const QudaDirection dir,
                                       const int dagger, hipStream_t *stream,
                                       void *buffer)
  {
#ifdef MULTI_GPU
    int face_num;
    if(dir == QUDA_BACKWARDS){
      face_num = 0;
    }else if(dir == QUDA_FORWARDS){
      face_num = 1;
    }else{
      face_num = 2;
    }
    void *packBuffer = buffer ? buffer : ghostFaceBuffer[bufferIndex];
    packFaceExtended(packBuffer, *this, nFace, R, dagger, parity, dim, face_num, *stream);
#else
    errorQuda("packGhostExtended not built on single-GPU build");
#endif

  }


  

  // copy data from host buffer into boundary region of device field
  void cudaColorSpinorField::unpackGhostExtended(const void* ghost_spinor, const int nFace, const QudaParity parity,
                                                 const int dim, const QudaDirection dir, 
                                                 const int dagger, hipStream_t* stream)
  {

     
     
    // First call the regular unpackGhost routine to copy data into the `usual' ghost-zone region 
    // of the data array 
    unpackGhost(ghost_spinor, nFace, dim, dir, dagger, stream);

    // Next step is to copy data from the ghost zone back to the interior region
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim][0];
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

#ifdef MULTI_GPU
    const int face_num = 2;
    const bool unpack = true;
    const int R[4] = {0,0,0,0};
    packFaceExtended(ghostFaceBuffer[bufferIndex], *this, nFace, R, dagger, parity, dim, face_num, *stream, unpack); 
#else
    errorQuda("unpackGhostExtended not built on single-GPU build");
#endif
  }



  hipStream_t *stream;

  void cudaColorSpinorField::createComms(int nFace) {


    if(bufferMessageHandler != bufferPinnedResizeCount) destroyComms();

    if (!initComms || nFaceComms != nFace) {

      // if we are requesting a new number of faces destroy and start over
      if(nFace != nFaceComms) destroyComms();

      if (siteSubset != QUDA_PARITY_SITE_SUBSET) 
	errorQuda("Only supports single parity fields");

#ifdef GPU_COMMS
      bool comms = false;
      for (int i=0; i<nDimComms; i++) if (commDimPartitioned(i)) comms = true;
#endif

      if (nFace > maxNface) 
	errorQuda("Requested number of faces %d in communicator is greater than supported %d",
		  nFace, maxNface);

      // faceBytes is the sum of all face sizes 
      size_t faceBytes = 0;
      
      // nbytes is the size in bytes of each face
      size_t nbytes[QUDA_MAX_DIM];
      
      // The number of degrees of freedom per site for the given
      // field.  Currently assumes spin projection of a Wilson-like
      // field (so half the number of degrees of freedom).
      int Ndof = (2 * nSpin * nColor) / (nSpin==4 ? 2 : 1);

      for (int i=0; i<nDimComms; i++) {
	nbytes[i] = maxNface*surfaceCB[i]*Ndof*precision;
	if (precision == QUDA_HALF_PRECISION) nbytes[i] += maxNface*surfaceCB[i]*sizeof(float);
	if (!commDimPartitioned(i)) continue;
	faceBytes += 2*nbytes[i];
      }
      
#ifndef GPU_COMMS
      // use static pinned memory for face buffers
      for(int b=0; b<2; ++b){
        resizeBufferPinned(2*faceBytes, b); // oversizes for GPU_COMMS case

        my_face[b] = bufferPinned[b];
        from_face[b] = static_cast<char*>(bufferPinned[b]) + faceBytes;
      }

      // assign pointers for each face - it's ok to alias for different Nface parameters
      size_t offset = 0;
#endif
      for (int i=0; i<nDimComms; i++) {
	if (!commDimPartitioned(i)) continue;
	
#ifdef GPU_COMMS
	for(int b=0; b<2; ++b){
	  my_back_face[b][i] = backGhostFaceBuffer[b][i];
	  from_back_face[b][i] = ghost[i];
	
	  if(precision == QUDA_HALF_PRECISION){
	    my_back_norm_face[b][i]  = static_cast<char*>(backGhostFaceBuffer[b][i]) + nFace*ghostFace[i]*Ndof*precision;
	    from_back_norm_face[b][i] = ghostNorm[i];
	  }
	} // loop over b

#else
        for(int b=0; b<2; ++b){
	  my_back_face[b][i] = static_cast<char*>(my_face[b]) + offset;
	  from_back_face[b][i] = static_cast<char*>(from_face[b]) + offset;
	}
	offset += nbytes[i];
#endif
	
#ifdef GPU_COMMS
	for(int b=0; b<2; ++b){
	  my_fwd_face[b][i] = fwdGhostFaceBuffer[b][i];	
	  //from_fwd_face[b][i] = ghost[i] + nFace*ghostFace[i]*Ndof*precision;
	  from_fwd_face[b][i] = ghost_field + ghostOffset[i][1]*precision;

	  if(precision == QUDA_HALF_PRECISION){
	    my_fwd_norm_face[b][i] = static_cast<char*>(fwdGhostFaceBuffer[b][i]) + nFace*ghostFace[i]*Ndof*precision;
	   // from_fwd_norm_face[b][i] = static_cast<char*>(ghostNorm[i]) + nFace*ghostFace[i]*sizeof(float);
            from_fwd_norm_face[b][i] = static_cast<char*>(ghost_field) + ghostNormOffset[i][1]*sizeof(float);
	  }
	} // loop over b
#else
	for(int b=0; b<2; ++b){
	  my_fwd_face[b][i] = static_cast<char*>(my_face[b]) + offset;
	  from_fwd_face[b][i] = static_cast<char*>(from_face[b]) + offset;
	}
	offset += nbytes[i];
#endif

      }

      // create a different message handler for each direction and Nface
      for(int b=0; b<2; ++b){
        mh_send_fwd[b] = new MsgHandle**[maxNface];
        mh_send_back[b] = new MsgHandle**[maxNface];
        mh_recv_fwd[b] = new MsgHandle**[maxNface];
        mh_recv_back[b] = new MsgHandle**[maxNface];
#ifdef GPU_COMMS
        if(precision == QUDA_HALF_PRECISION){
      	  mh_send_norm_fwd[b]  = new MsgHandle**[maxNface];
      	  mh_send_norm_back[b] = new MsgHandle**[maxNface];
     	  mh_recv_norm_fwd[b]  = new MsgHandle**[maxNface];
      	  mh_recv_norm_back[b] = new MsgHandle**[maxNface]; 
        }
#endif
      } // loop over b
      for (int j=0; j<maxNface; j++) {
	for(int b=0; b<2; ++b){
	  mh_send_fwd[b][j] = new MsgHandle*[2*nDimComms];
	  mh_send_back[b][j] = new MsgHandle*[2*nDimComms];
	  mh_recv_fwd[b][j] = new MsgHandle*[nDimComms];
	  mh_recv_back[b][j] = new MsgHandle*[nDimComms];
		
#ifdef GPU_COMMS
	  if(precision == QUDA_HALF_PRECISION){
	    mh_send_norm_fwd[b][j] = new MsgHandle*[2*nDimComms];
	    mh_send_norm_back[b][j] = new MsgHandle*[2*nDimComms];
	    mh_recv_norm_fwd[b][j] = new MsgHandle*[nDimComms];
	    mh_recv_norm_back[b][j] = new MsgHandle*[nDimComms];
	  }
#endif	
	} // loop over b


	for (int i=0; i<nDimComms; i++) {
	  if (!commDimPartitioned(i)) continue;
#ifdef GPU_COMMS
	  size_t nbytes_Nface = surfaceCB[i]*Ndof*precision*(j+1);
	  size_t nbytes_Nface_norm = surfaceCB[i]*(j+1)*sizeof(float);
	  if (i != 3 || getKernelPackT() || getTwistPack()) {
#else 
	    size_t nbytes_Nface = (nbytes[i] / maxNface) * (j+1);
#endif
	    for(int b=0; b<2; ++b){
	      mh_send_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_fwd_face[b][i], i, +1, nbytes_Nface) : NULL;
	      mh_send_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_back_face[b][i], i, -1, nbytes_Nface) : NULL;
	      mh_send_fwd[b][j][2*i+1] = mh_send_fwd[b][j][2*i]; // alias pointers
	      mh_send_back[b][j][2*i+1] = mh_send_back[b][j][2*i]; // alias pointers
	    }
#ifdef GPU_COMMS

	    if(precision == QUDA_HALF_PRECISION){
	      for(int b=0; b<2; ++b){
		mh_send_norm_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_fwd_norm_face[b][i], i, +1, nbytes_Nface_norm) : NULL;
		mh_send_norm_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(my_back_norm_face[b][i], i, -1, nbytes_Nface_norm) : NULL;
		mh_send_norm_fwd[b][j][2*i+1] = mh_send_norm_fwd[b][j][2*i];
		mh_send_norm_back[b][j][2*i+1] = mh_send_norm_back[b][j][2*i]; 	
	      }
	    }

	  } else if (this->TwistFlavor() == QUDA_TWIST_NONDEG_DOUBLET) {
	    errorQuda("GPU_COMMS for non-degenerate doublet only supported with time-dimension kernel packing enabled.");
	  } else {
	    /* 
	       use a strided communicator, here we can't really use
	       the previously declared my_fwd_face and my_back_face
	       pointers since they don't really map 1-to-1 so let's
	       just compute the required base pointers and pass these
	       directly into the communicator construction
	    */
	    
	    int Nblocks = Ndof / Nvec(); // number of Nvec buffers we have
	    // start of last time slice chunk we are sending forwards
	    int endOffset = (volume - (j+1)*ghostFace[i]); 

	    size_t offset[4];
	    void *base[4];
	    if (nSpin == 1) { // staggered is invariant with dagger
	      offset[2*0 + 0] = 0;
	      offset[2*1 + 0] = endOffset;
	      offset[2*0 + 1] = offset[2*0 + 0];
	      offset[2*1 + 1] = offset[2*1 + 0];
	    } else if (nSpin == 4) {    
	      // !dagger: send last components backwards, send first components forwards
	      offset[2*0 + 0] = Nblocks*stride;
	      offset[2*1 + 0] = endOffset;
	      //  dagger: send first components backwards, send last components forwards
	      offset[2*0 + 1] = 0;
	      offset[2*1 + 1] = Nblocks*stride + endOffset;
	    } else {
	      errorQuda("Unsupported number of spin components");
	    }

	    for (int k=0; k<4; k++) {
	      base[k] = static_cast<char*>(v) + offset[k]*Nvec()*precision; // total offset in bytes
	    }

	    size_t blksize  = (j+1)*ghostFace[i]*Nvec()*precision; // (j+1) is number of faces
	    size_t Stride = stride*Nvec()*precision;

	    if (blksize * Nblocks != nbytes_Nface) 
	      errorQuda("Total strided message size does not match expected size");

	    //printf("%d strided sends with Nface=%d Nblocks=%d blksize=%d Stride=%d\n", i, j+1, Nblocks, blksize, Stride);

            for(int b=0; b<2; ++b){
	      // only allocate a communicator for the present face (this needs cleaned up)
	      mh_send_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[2], i, +1, blksize, Nblocks, Stride) : NULL;
	      mh_send_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[0], i, -1, blksize, Nblocks, Stride) : NULL;
	      if (nSpin ==4) { // dagger communicators
	        mh_send_fwd[b][j][2*i+1] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[3], i, +1, blksize, Nblocks, Stride) : NULL;
	        mh_send_back[b][j][2*i+1] = (j+1 == nFace) ? comm_declare_strided_send_relative(base[1], i, -1, blksize, Nblocks, Stride) : NULL;
	      } else {
	        mh_send_fwd[b][j][2*i+1] = mh_send_fwd[b][j][2*i+0];
	        mh_send_back[b][j][2*i+1] = mh_send_back[b][j][2*i+0];
	      }

            } // loop over b

          
	    if(precision == QUDA_HALF_PRECISION){
	      int Nt_minus1_offset = (volume - nFace*ghostFace[3]); // The space-time coordinate of the start of the last time slice
	      void *norm_fwd = static_cast<float*>(norm) + Nt_minus1_offset;
	      void *norm_back = norm; // the first time slice has zero offset
	      for(int b=0; b<2; ++b){
		mh_send_norm_fwd[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(norm_fwd, i, +1, surfaceCB[i]*(j+1)*sizeof(float)) : NULL;
		mh_send_norm_back[b][j][2*i+0] = (j+1 == nFace) ? comm_declare_send_relative(norm_back, i, -1, surfaceCB[i]*(j+1)*sizeof(float)) : NULL;
		mh_send_norm_fwd[b][j][2*i+1] = mh_send_norm_fwd[b][j][2*i];
		mh_send_norm_back[b][j][2*i+1] = mh_send_norm_back[b][j][2*i];  
	      }
	    }

	  }

	  if(precision == QUDA_HALF_PRECISION){
            for(int b=0; b<2; ++b){
	      mh_recv_norm_fwd[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_fwd_norm_face[b][i], i, +1, nbytes_Nface_norm) : NULL;
	      mh_recv_norm_back[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_back_norm_face[b][i], i, -1, nbytes_Nface_norm) : NULL;
            }
	  }
#endif // GPU_COMMS

	  for(int b=0; b<2; ++b){
	    mh_recv_fwd[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_fwd_face[b][i], i, +1, nbytes_Nface) : NULL;
	    mh_recv_back[b][j][i] = (j+1 == nFace) ? comm_declare_receive_relative(from_back_face[b][i], i, -1, nbytes_Nface) : NULL;
	  }
	 


	} // loop over dimension
      }
     
      bufferMessageHandler = bufferPinnedResizeCount;
      initComms = true;
      nFaceComms = nFace;
    }
    checkCudaError();
#ifdef P2P_COMMS
    createIPCDslashComms();
#endif
    checkCudaError();
  }
   
#ifdef P2P_COMMS
  void cudaColorSpinorField::destroyIPCDslashComms() {


    if(initIPCTimeComms){
      if(commDimPartitioned(3) && !getKernelPackT()){ 

	if(comm_dslash_peer2peer_enabled(1,3)){
//	  hipIpcCloseMemHandle(backFieldSrcBuffer);
//	  if(precision == QUDA_HALF_PRECISION) hipIpcCloseMemHandle(backNormSrcBuffer);
	}

	if(comm_dslash_peer2peer_enabled(0,3)){
	  const int num_dir = (comm_dim(3) == 2) ? 1 : 2;
		
           if(num_dir == 2){
 //            hipIpcCloseMemHandle(fwdFieldSrcBuffer);
 //	     if(precision == QUDA_HALF_PRECISION) hipIpcCloseMemHandle(fwdNormSrcBuffer);
	   }
	}
        initIPCTimeComms = false;
      }
    }


    if(!initIPCDslashComms) return;

    for(int dim=0; dim<4; ++dim){

      if(!commDimPartitioned(dim)) continue;
    
      if(comm_dslash_peer2peer_enabled(1,dim)) {
	comm_free(mh_send_p2p_fwd[dim]);
	comm_free(mh_recv_p2p_fwd[dim]);

//	if(dim != 3 || getKernelPackT()) hipIpcCloseMemHandle(backGhostFaceSrcBuffer[dim]);
	hipEventDestroy(ipcCopyEvent[1][dim]);
      }


	
      if(comm_dslash_peer2peer_enabled(0,dim)) {
        const int num_dir = (comm_dim(dim) == 2) ? 1 : 2;
	comm_free(mh_send_p2p_back[dim]);
	comm_free(mh_recv_p2p_back[dim]);

	if(dim != 3 || getKernelPackT()){
//	  if(num_dir == 2) hipIpcCloseMemHandle(fwdGhostFaceSrcBuffer[dim]);
	} 
	hipEventDestroy(ipcCopyEvent[0][dim]);
      }    
    } // iterate over dim

    checkCudaError();
    initIPCDslashComms = false;
  }
#endif
 
  void cudaColorSpinorField::destroyComms() {
    if (initComms) {
#ifdef P2P_COMMS
      destroyIPCDslashComms();
      checkCudaError();
#endif
      for(int b=0; b<2; ++b){
      for (int j=0; j<maxNface; j++) {
	for (int i=0; i<nDimComms; i++) {
	  if (commDimPartitioned(i)) {
	    if (mh_recv_fwd[b][j][i]) comm_free(mh_recv_fwd[b][j][i]);
	    if (mh_recv_fwd[b][j][i]) comm_free(mh_recv_back[b][j][i]);
	    if (mh_send_fwd[b][j][2*i]) comm_free(mh_send_fwd[b][j][2*i]);
	    if (mh_send_back[b][j][2*i]) comm_free(mh_send_back[b][j][2*i]);
	    // only in a special case are these not aliasing pointers
#ifdef GPU_COMMS
	    if(precision == QUDA_HALF_PRECISION){
	      if (mh_recv_norm_fwd[b][j][i]) comm_free(mh_recv_norm_fwd[b][j][i]);
	      if (mh_recv_norm_back[b][j][i]) comm_free(mh_recv_norm_back[b][j][i]);
	      if (mh_send_norm_fwd[b][j][2*i]) comm_free(mh_send_norm_fwd[b][j][2*i]);
	      if (mh_send_norm_back[b][j][2*i]) comm_free(mh_send_norm_back[b][j][2*i]);
	    }

	    if (i == 3 && !getKernelPackT() && nSpin == 4) {
	      if (mh_send_fwd[b][j][2*i+1]) comm_free(mh_send_fwd[b][j][2*i+1]);
	      if (mh_send_back[b][j][2*i+1]) comm_free(mh_send_back[b][j][2*i+1]);
	    }
#endif // GPU_COMMS

	  }
	}
	delete []mh_recv_fwd[b][j];
	delete []mh_recv_back[b][j];
	delete []mh_send_fwd[b][j];
	delete []mh_send_back[b][j];
#ifdef GPU_COMMS
	if(precision == QUDA_HALF_PRECISION){
	  delete []mh_recv_norm_fwd[b][j];
	  delete []mh_recv_norm_back[b][j];
	  delete []mh_send_norm_fwd[b][j];
	  delete []mh_send_norm_back[b][j];
	}
#endif
      }    
      delete []mh_recv_fwd[b];
      delete []mh_recv_back[b];
      delete []mh_send_fwd[b];
      delete []mh_send_back[b];
      
      for (int i=0; i<nDimComms; i++) {
	my_fwd_face[b][i] = NULL;
	my_back_face[b][i] = NULL;
	from_fwd_face[b][i] = NULL;
	from_back_face[b][i] = NULL;      
      }
#ifdef GPU_COMMS
      if(precision == QUDA_HALF_PRECISION){
	delete []mh_recv_norm_fwd[b];
	delete []mh_recv_norm_back[b];
	delete []mh_send_norm_fwd[b];
	delete []mh_send_norm_back[b];
      }
	
      for(int i=0; i<nDimComms; i++){
	my_fwd_norm_face[b][i] = NULL;
	my_back_norm_face[b][i] = NULL;
	from_fwd_norm_face[b][i] = NULL;
	from_back_norm_face[b][i] = NULL;
      }
#endif 

      } // loop over b

      initComms = false;
      checkCudaError();
    }
  }

  void cudaColorSpinorField::streamInit(hipStream_t *stream_p){
    stream = stream_p;
  }


  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, hipStream_t *stream_p, 
				  bool zeroCopyPack, double a, double b) {


    allocateGhostBuffer(nFace);   // allocate the ghost buffer if not yet allocated  
    createComms(nFace); // must call this first

    stream = stream_p;
    
    const int dim=-1; // pack all partitioned dimensions
 
    if (zeroCopyPack) {
      void *my_face_d;
      hipHostGetDevicePointer(&my_face_d, my_face[bufferIndex], 0); // set the matching device pointer
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d, a, b);
    } else {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger,  &stream[Nstream-1], 0, a, b);
    }
  }

  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, int stream_idx, 
				  bool zeroCopyPack, double a, double b) {
    allocateGhostBuffer(nFace);   // allocate the ghost buffer if not yet allocated  
    createComms(nFace); // must call this first

    const int dim=-1; // pack all partitioned dimensions
 
    if (zeroCopyPack) {
      void *my_face_d;
      hipHostGetDevicePointer(&my_face_d, my_face[bufferIndex], 0); // set the matching device pointer
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[stream_idx], my_face_d, a, b);
    } else {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger,  &stream[stream_idx], 0, a, b);
    }
  }

  void cudaColorSpinorField::packExtended(const int nFace, const int R[], const int parity, 
                                          const int dagger, const int dim,
                                          hipStream_t *stream_p, const bool zeroCopyPack){

    allocateGhostBuffer(nFace); // allocate the ghost buffer if not yet allocated
    createComms(nFace); // must call this first

    stream = stream_p;
 
    void *my_face_d = NULL;
    if(zeroCopyPack){ 
      hipHostGetDevicePointer(&my_face_d, my_face[bufferIndex], 0);
      packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d);
    }else{
      packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[Nstream-1], my_face_d);
    }
  }
                                                      


  void cudaColorSpinorField::gather(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;

    // If stream_p != 0, use pack_stream, else use the stream array
    hipStream_t *pack_stream = (stream_p) ? stream_p : stream+dir;

    if(dir%2 == 0){
      // backwards copy to host
#ifdef P2P_COMMS
      if(comm_dslash_peer2peer_enabled(0,dim)) return;
#endif
      sendGhost(my_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, pack_stream);
    } else {
      // forwards copy to host
#ifdef P2P_COMMS
      if(comm_dslash_peer2peer_enabled(1,dim)) return;
#endif
      sendGhost(my_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, pack_stream);
    }
  }


  void cudaColorSpinorField::recvStart(int nFace, int dir, int dagger, hipStream_t* stream_p) {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;

    if (dir%2 == 0) { // sending backwards
#ifdef P2P_COMMS
      if(comm_dslash_peer2peer_enabled(1,dim)){
	// receive from the processor in the +1 direction
	comm_start(mh_recv_p2p_fwd[dim]);
      } else 
#endif
      {
        // Prepost receive
        comm_start(mh_recv_fwd[bufferIndex][nFace-1][dim]);
      }
    } else { //sending forwards
      // Prepost receive
#ifdef P2P_COMMS
      if (comm_dslash_peer2peer_enabled(0,dim)) {
	comm_start(mh_recv_p2p_back[dim]);
      } else
#endif
      {
        comm_start(mh_recv_back[bufferIndex][nFace-1][dim]);
      }
    }
  }


  void cudaColorSpinorField::sendStart(int nFace, int dir, int dagger, hipStream_t* stream_p) {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;


    if(dir%2 == 0){
#ifdef P2P_COMMS
	if(!comm_dslash_peer2peer_enabled(0,dim))
	{
#endif
	  comm_start(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
#ifdef P2P_COMMS
	} else {
	   hipStream_t *copy_stream = (stream_p) ? stream_p : stream + dir;	
	   // start the copy
	   // all goes here
	   void* ghost_dst = (void*)((char*)(backGhostSendDest[dim]) + precision*ghostOffset[dim][1]);
	   if(dim != 3 || getKernelPackT()) {
	    hipMemcpyAsync(ghost_dst,
			  backGhostFaceBuffer[bufferIndex][dim], // Change this??
			  ghost_face_bytes[dim],
			  hipMemcpyDeviceToDevice,	
			  *copy_stream); // copy to forward processor
	    hipDeviceSynchronize(); 
	    } else {
	     int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
	     int Nint = (nColor * nSpin * 2)/(nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom
	     int Npad = Nint/Nvec;
	     int offset = 0;
	     if(nSpin == 1){
	       offset = 0;
	     } else if (nSpin == 4){
		// !dagger: send lower components backwards, send upper components forwards
		// dagger: send upper components backwardsm send lower components forwards
		bool upper = dagger ? true : false;
		int lower_spin_offset = Npad*stride;
		
		offset = upper ? 0 : lower_spin_offset;
	     }

	     void* src = (char*)v + offset*Nvec*precision;
	     size_t len = nFace*ghostFace[3]*Nvec*precision;
	     size_t spitch = stride*Nvec*precision;
	     hipMemcpy2DAsync(ghost_dst, len, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	     if(precision == QUDA_HALF_PRECISION) {
	       int norm_offset = 0;
	       void *ghost_norm_dst = backGhostSendDest[dim] + QUDA_SINGLE_PRECISION*ghostNormOffset[dim][1];
	       void* src = norm;
	       hipMemcpyAsync(ghost_norm_dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToDevice, *copy_stream);
	     }
	   }
	   // record the event 
	   hipEventRecord(ipcCopyEvent[0][dim], *copy_stream);
	   // send to the propcessor in the -1 direction
	   comm_start(mh_send_p2p_back[dim]);
	}
#endif
      } else { // sending forwards
#ifdef P2P_COMMS
	if(!comm_dslash_peer2peer_enabled(1,dim)) {
#endif
	  comm_start(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
#ifdef P2P_COMMS
      } else {
	  hipStream_t *copy_stream = (stream_p) ? stream_p : stream + dir;

	  // start the copy 
	  void* ghost_dst = (void*)((char*)(fwdGhostSendDest[dim]) + precision*ghostOffset[dim][0]);
	
          if(dim != 3 || getKernelPackT()){
	    hipMemcpyAsync(ghost_dst,
			  fwdGhostFaceBuffer[bufferIndex][dim],
			  ghost_face_bytes[dim],	  
			  hipMemcpyDeviceToDevice,	
			  *copy_stream); // copy to forward processor
	    hipDeviceSynchronize(); 
	  } else {
	    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
	    int Nint = (nColor * nSpin *2)/(nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom
	    int Npad = Nint / Nvec;
	    int Nt_minus1_offset = (volume - nFace*ghostFace[3]);
	    int offset;
	    if(nSpin == 1){
	      offset = Nt_minus1_offset;
	    } else if(nSpin == 4){
	      bool upper = dagger ? true : false;
	      upper = !upper;
	      int lower_spin_offset = Npad*stride;
	      offset = (upper) ? Nt_minus1_offset : lower_spin_offset + Nt_minus1_offset;
	    }
	    void *src = (char*)v + offset*Nvec*precision;
	    size_t len = nFace*ghostFace[3]*Nvec*precision;
	    size_t spitch = stride*Nvec*precision;
	    hipMemcpy2DAsync(ghost_dst, len, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	    if(precision == QUDA_HALF_PRECISION) {
	      int norm_offset = Nt_minus1_offset*sizeof(float);
	      void* ghost_norm_dst = fwdGhostSendDest[dim] + QUDA_SINGLE_PRECISION*ghostNormOffset[dim][0];
	      void *src = (char*)norm + norm_offset;
	      hipMemcpyAsync(ghost_norm_dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToDevice, *copy_stream);
	    }
	  }

	  hipEventRecord(ipcCopyEvent[1][dim], *copy_stream);
	  // send to the processor in the +1 direction
	  comm_start(mh_send_p2p_fwd[dim]);
      }
#endif
  }
}
/*
  void cudaColorSpinorField::sendStart(int nFace, int dir, int dagger, hipStream_t* stream_p) {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;

    checkCudaError();
	
    if(dir%2 == 0){ // sending backwards
#ifdef P2P_COMMS
      if(!comm_dslash_peer2peer_enabled(0,dim))
      {
#endif
        comm_start(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
#ifdef P2P_COMMS
      } else {
	// send to the processor in the -1 direction
        checkCudaError();
	comm_start(mh_send_p2p_back[bufferIndex][dim]);
        checkCudaError();
      } 

      if(comm_dslash_peer2peer_enabled(1,dim)) {
      	hipStream_t *copy_stream = (stream_p) ? stream_p : stream + dir;
        // begin the copy from the forward processor
	void *ghost_dst = ghost_field + precision*ghostOffset[dim][1];
	
	// Synchronize with the forward processor
	comm_wait(mh_recv_p2p_fwd[bufferIndex][dim]);

	if(dim != 3 || getKernelPackT()){
	  hipMemcpyAsync(ghost_dst, 
		(void*)((char*)(backGhostFaceSrcBuffer[bufferIndex][dim]) 
		+ backGhostBufferOffset[bufferIndex][dim]),
		ghost_face_bytes[dim],
		hipMemcpyDeviceToDevice,
		*copy_stream); // copy from forward processor
	} else {
	  int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
	  int Nint = (nColor * nSpin * 2)/(nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom
	  int Npad = Nint / Nvec;
	  int offset = 0;
	  if(nSpin == 1){
	    offset = 0;
	  } else if (nSpin == 4){
	    // !dagger: send lower components backwards, send upper components forwards
	    // dagger: send upper components backwards, send lower components forwards
	    bool upper = dagger ? true : false;
	    int lower_spin_offset = Npad*stride;	

	    
	    offset = upper ? 0 : lower_spin_offset;
	 }

	  void* src = (char*)backFieldSrcBuffer + offset*Nvec*precision;
	  size_t len = nFace*ghostFace[3]*Nvec*precision;
	  size_t spitch = stride*Nvec*precision;
	  hipMemcpy2DAsync(ghost_dst, len, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	  if(precision == QUDA_HALF_PRECISION) {
	    int norm_offset = 0;
	    void *ghost_norm_dst = ghost_field + QUDA_SINGLE_PRECISION*ghostNormOffset[dim][1];
	    void *src = backNormSrcBuffer;
	    hipMemcpyAsync(ghost_norm_dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToDevice, *copy_stream); 
	  }
	}
	hipEventRecord(ipcCopyEvent[bufferIndex][1][dim], *copy_stream);
      }
#endif
    } else { // sending forwards 
#ifdef P2P_COMMS
      if(!comm_dslash_peer2peer_enabled(1,dim)) {
#endif
	comm_start(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
#ifdef P2P_COMMS
      } else {
      // send a message to the processor in the forward direction
	comm_start(mh_send_p2p_fwd[bufferIndex][dim]);
      }


      if(comm_dslash_peer2peer_enabled(0,dim)) {
      	hipStream_t *copy_stream = (stream_p) ? stream_p : stream + dir;
	// copy from backward processor
	void *ghost_dst = ghost_field + precision*ghostOffset[dim][0];

	comm_wait(mh_recv_p2p_back[bufferIndex][dim]);

	if(dim != 3 || getKernelPackT()) {
	  hipMemcpyAsync(ghost_dst, 
		(void*)((char*)(fwdGhostFaceSrcBuffer[bufferIndex][dim]) 
		+ fwdGhostBufferOffset[bufferIndex][dim]),
		ghost_face_bytes[dim],
		hipMemcpyDeviceToDevice,
		*copy_stream); // copy from backward processor
	} else {
	  int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
	  int Nint = (nColor * nSpin * 2)/(nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom
	  int Npad = Nint / Nvec;
	  int Nt_minus1_offset = (volume - nFace*ghostFace[3]);
	  int offset;
	  if(nSpin == 1){
	    offset = Nt_minus1_offset;
	  } else if(nSpin == 4){
	    bool upper = dagger ? true : false; 
	    upper = !upper;
	    int lower_spin_offset = Npad*stride;
	    offset = (upper) ? Nt_minus1_offset : lower_spin_offset + Nt_minus1_offset;
	  }

	  void *src = (char*)fwdFieldSrcBuffer + offset*Nvec*precision;
	  size_t len = nFace*ghostFace[3]*Nvec*precision;
	  size_t spitch = stride*Nvec*precision;
	  hipMemcpy2DAsync(ghost_dst, len, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);  

	  if(precision == QUDA_HALF_PRECISION) {
	    int norm_offset = Nt_minus1_offset*sizeof(float);
	    void *ghost_norm_dst = ghost_field + QUDA_SINGLE_PRECISION*ghostNormOffset[dim][0];
	    void *src = fwdNormSrcBuffer + norm_offset;
	    hipMemcpyAsync(ghost_norm_dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToDevice, *copy_stream); 
	  }
	}
	hipEventRecord(ipcCopyEvent[bufferIndex][0][dim],*copy_stream);
      }
#endif
    }
    checkCudaError();
}
*/

  void cudaColorSpinorField::commsStart(int nFace, int dir, int dagger, hipStream_t* stream_p) {
    int dim = dir/2;
     
    if(!commDimPartitioned(dim)) return;

	
    if(dir%2 == 0){ // sending backwards
#ifdef P2P_COMMS
      if(!comm_dslash_peer2peer_enabled(1,dim))
#endif
      {
        comm_start(mh_recv_fwd[bufferIndex][nFace-1][dim]);
      }
#ifdef P2P_COMMS
      if(!comm_dslash_peer2peer_enabled(0,dim))
#endif
      {
        comm_start(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
      }

#ifdef P2P_COMMS
      comm_barrier(); // Sledgehammer synchronization, but okay for testing purposes

      hipStream_t *copy_stream = (stream_p) ? stream_p : stream + dir;

      if(comm_dslash_peer2peer_enabled(1,dim)) {
        // begin the copy from the forward processor
	void *ghost_dst = ghost_field + precision*ghostOffset[dim][1];

	hipMemcpyAsync(ghost_dst, 
			(void*)((char*)(backGhostFaceSrcBuffer[bufferIndex][dim]) 
			+ backGhostBufferOffset[bufferIndex][dim]),
			ghost_face_bytes[dim],
			hipMemcpyDeviceToDevice,
			*copy_stream); // copy from forward processor


	hipEventRecord(ipcCopyEvent[1][dim], *copy_stream);
      }
#endif
    } else { // sending forwards 
#ifdef P2P_COMMS 
      if(!comm_dslash_peer2peer_enabled(0,dim))
#endif
      {
        comm_start(mh_recv_back[bufferIndex][nFace-1][dim]);
      }
#ifdef P2P_COMMS
      if(!comm_dslash_peer2peer_enabled(1,dim))
#endif
      {
	comm_start(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
      }

#ifdef P2P_COMMS
      comm_barrier(); // Sledgehammer synchronization, but okay for testing purposes
      // Copy data from the processor in the backward direction
      // This means I should send to the forward direction if peer-to-peer comms are enabled there
      if(comm_dslash_peer2peer_enabled(1,dim)) {
	// send a signal to the forward direction
      }

      if (comm_dslash_peer2peer_enabled(0,dim)) {
	// wait for a signal from the backward direction
      }



      hipStream_t *copy_stream = (stream_p) ? stream_p : stream + dir;
      if(comm_dslash_peer2peer_enabled(0,dim)) {
	// copy from backward processor
	void *ghost_dst = ghost_field + precision*ghostOffset[dim][0];
	hipMemcpyAsync(ghost_dst, 
			(void*)((char*)(fwdGhostFaceSrcBuffer[bufferIndex][dim]) 
			+ fwdGhostBufferOffset[bufferIndex][dim]),
			ghost_face_bytes[dim],
			hipMemcpyDeviceToDevice,
			*copy_stream); // copy from backward processor

	hipEventRecord(ipcCopyEvent[0][dim],*copy_stream);
      }
#endif
    }

  }


#ifdef P2P_COMMS
  int cudaColorSpinorField::ipcCopyComplete(int dir, int dim){
    if(hipSuccess == hipEventQuery(ipcCopyEvent[dir][dim])){
      return 1;
    }
    return 0;
  }

  int cudaColorSpinorField::ipcRemoteCopyComplete(int dir, int dim){
    if(hipSuccess == hipEventQuery(ipcRemoteCopyEvent[dir][dim])){
      return 1;
    }
    return 0;
  }
#endif

  int cudaColorSpinorField::commsQuery(int nFace, int dir, int dagger, hipStream_t *stream_p) {

    int dim = dir/2;
    if(!commDimPartitioned(dim)) return 0;


    if(!commDimPartitioned(dim)) return 0;

    int receive_complete=0;
    int send_complete=0;

#ifdef P2P_COMMS
    comm_barrier(); // FIXME sledgehammer
#endif

    if(dir%2==0){

#ifdef P2P_COMMS
      if(comm_dslash_peer2peer_enabled(1,dim)){

	receive_complete = (comm_query(mh_recv_p2p_fwd[dim])
			    && ipcRemoteCopyComplete(1,dim));

      } else 
#endif
      {
	receive_complete = comm_query(mh_recv_fwd[bufferIndex][nFace-1][dim]);
      }

#ifdef P2P_COMMS
      if(comm_dslash_peer2peer_enabled(0,dim)){
	send_complete = ipcCopyComplete(0,dim);
      } else 
#endif
      {
	send_complete = comm_query(mh_send_back[bufferIndex][nFace-1][2*dim+dagger]);
      }

    } else { // dir%2 == 1
#ifdef P2P_COMMS
      if(comm_dslash_peer2peer_enabled(0,dim)){
        receive_complete = (comm_query(mh_recv_p2p_back[dim])
			    && ipcRemoteCopyComplete(0, dim));
      } else 
#endif
      {
	receive_complete = comm_query(mh_recv_back[bufferIndex][nFace-1][dim]);
      }

#ifdef P2P_COMMS
      if(comm_dslash_peer2peer_enabled(1,dim)){
	send_complete = ipcCopyComplete(1,dim);
      } else 
#endif
      {
	send_complete = comm_query(mh_send_fwd[bufferIndex][nFace-1][2*dim+dagger]);
      }

    }
    if(receive_complete && send_complete) return 1;
    return 0;
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;

    // both scattering occurances now go through the same stream
    if (dir%2==0) {// receive from forwards
#ifdef P2P_COMMS
      if (comm_dslash_peer2peer_enabled(1,dim)) return;
#endif
      unpackGhost(from_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, stream_p);
    } else { // receive from backwards
#ifdef P2P_COMMS
      if (comm_dslash_peer2peer_enabled(0,dim)) return;
#endif
      unpackGhost(from_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, stream_p);
    }
  }



  void cudaColorSpinorField::scatter(int nFace, int dagger, int dir)
  {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;

    // both scattering occurances now go through the same stream
    if (dir%2==0) {// receive from forwards
#ifdef P2P_COMMS
      if (comm_dslash_peer2peer_enabled(1,dim)) return;
#endif
      unpackGhost(from_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/]);
    } else { // receive from backwards
#ifdef P2P_COMMS
      if (comm_dslash_peer2peer_enabled(0,dim)) return;
#endif
      unpackGhost(from_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, &stream[2*dim/*+1*/]);
    }
  }

  
  void cudaColorSpinorField::scatterExtended(int nFace, int parity, int dagger, int dir)
  {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;
    if (dir%2==0) {// receive from forwards
      unpackGhostExtended(from_fwd_face[bufferIndex][dim], nFace, static_cast<QudaParity>(parity), dim, QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/]);
    } else { // receive from backwards
      unpackGhostExtended(from_back_face[bufferIndex][dim], nFace, static_cast<QudaParity>(parity),  dim, QUDA_BACKWARDS, dagger, &stream[2*dim/*+1*/]);
    }
  }
 

  // Return the location of the field
  QudaFieldLocation cudaColorSpinorField::Location() const { return QUDA_CUDA_FIELD_LOCATION; }

  std::ostream& operator<<(std::ostream &out, const cudaColorSpinorField &a) {
    out << (const ColorSpinorField&)a;
    out << "v = " << a.v << std::endl;
    out << "norm = " << a.norm << std::endl;
    out << "alloc = " << a.alloc << std::endl;
    out << "init = " << a.init << std::endl;
    return out;
  }

//! for deflated solvers:
  cudaColorSpinorField& cudaColorSpinorField::Eigenvec(const int idx) const {
    
    if (siteSubset == QUDA_PARITY_SITE_SUBSET && this->EigvId() == -1) {
      if (idx < this->EigvDim()) {//setup eigenvector form the set
        return *(dynamic_cast<cudaColorSpinorField*>(eigenvectors[idx])); 
      }
      else{
        errorQuda("Incorrect eigenvector index...");
      }
    }
    errorQuda("Eigenvector must be a parity spinor");
    exit(-1);
  }

//copyCuda currently cannot not work with set of spinor fields..
  void cudaColorSpinorField::CopyEigenvecSubset(cudaColorSpinorField &dst, const int range, const int first_element) const{
#if 0
    if(first_element < 0) errorQuda("\nError: trying to set negative first element.\n");
    if (siteSubset == QUDA_PARITY_SITE_SUBSET && this->EigvId() == -1) {
      if (first_element == 0 && range == this->EigvDim())
      {
        if(range != dst.EigvDim())errorQuda("\nError: eigenvector range to big.\n");
        checkField(dst, *this);
        copyCuda(dst, *this);
      }
      else if ((first_element+range) < this->EigvDim()) 
      {//setup eigenvector subset

        cudaColorSpinorField *eigv_subset;

        ColorSpinorParam param;

        param.nColor = nColor;
        param.nSpin = nSpin;
        param.twistFlavor = twistFlavor;
        param.precision = precision;
        param.nDim = nDim;
        param.pad = pad;
        param.siteSubset = siteSubset;
        param.siteOrder = siteOrder;
        param.fieldOrder = fieldOrder;
        param.gammaBasis = gammaBasis;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
 
        param.eigv_dim  = range;
        param.eigv_id   = -1;
        param.v = (void*)((char*)v + first_element*eigv_bytes);
        param.norm = (void*)((char*)norm + first_element*eigv_norm_bytes);

        eigv_subset = new cudaColorSpinorField(param);

        //Not really needed:
        eigv_subset->eigenvectors.reserve(param.eigv_dim);
        for(int id = first_element; id < (first_element+range); id++)
        {
            param.eigv_id = id;
            eigv_subset->eigenvectors.push_back(new cudaColorSpinorField(*this, param));
        }
        checkField(dst, *eigv_subset);
        copyCuda(dst, *eigv_subset);

        delete eigv_subset;
      }
      else{
        errorQuda("Incorrect eigenvector dimension...");
      }
    }
    else{  
      errorQuda("Eigenvector must be a parity spinor");
      exit(-1);
    }
#endif
  }

  void cudaColorSpinorField::getTexObjectInfo() const
  {
#ifdef USE_TEXTURE_OBJECTS
    printfQuda("\nPrint texture info for the field:\n");
    std::cout << *this;
    hipResourceDesc resDesc;
    //memset(&resDesc, 0, sizeof(resDesc));
    hipGetTextureObjectResourceDesc(&resDesc, this->Tex());
    printfQuda("\nDevice pointer: %p\n", resDesc.res.linear.devPtr);
    printfQuda("\nVolume (in bytes): %d\n", resDesc.res.linear.sizeInBytes);
    if (resDesc.resType == hipResourceTypeLinear) printfQuda("\nResource type: linear \n");
#endif
  }

} // namespace quda
