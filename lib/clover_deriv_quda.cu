#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <quda_internal.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <quda_matrix.h>
#include <index_helper.cuh>
#include <cassert>

namespace quda {
  
#ifdef GPU_CLOVER_DIRAC
  
  template<class Float, typename Force, typename Gauge, typename Oprod>
  struct CloverDerivArg
  {
    int X[4];
    int E[4];
    int border[4];
    Float coeff;
    int parity;
    int volumeCB;

    Force force;
    Gauge gauge;
    Oprod oprod;

    bool conjugate;      

    CloverDerivArg(const Force& force, const Gauge& gauge, const Oprod& oprod,
		   const int *X_, const int *E_,
		   double coeff, int parity, bool conjugate) :
      coeff(coeff), parity(parity), volumeCB(force.volumeCB),
      force(force), gauge(gauge), oprod(oprod), conjugate(conjugate)
    {
      for(int dir=0; dir<4; ++dir) {
	this->X[dir] = X_[dir];
	this->E[dir] = E_[dir];
	this->border[dir] = (E_[dir] - X_[dir])/2;
      }
    }
  };


#define SHARED_ACCUMULATOR

#ifdef SHARED_ACCUMULATOR

#define DECLARE_FORCE							\
  extern __shared__ int s[];						\
  real *force = (real*)s;						\
  {									\
    const int tid = (threadIdx.z*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x; \
    const int block = blockDim.x * blockDim.y * blockDim.z;		\
    for (int i=0; i<18; i++) force[i*block + tid] = 0.0;		\
  }

  template <typename real, typename Link>
  __device__ inline void axpy(real a, const real *x, Link &y) {
    const int tid = (threadIdx.z*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x;
    const int block = blockDim.x * blockDim.y * blockDim.z;
#pragma unroll
    for (int i=0; i<9; i++) {
      y.data[i] += a * complex<real>(x[(2*i+0)*block + tid], x[(2*i+1)*block + tid]);
    }
  }

  template <typename real, typename Link>
  __device__ inline void add(real *y, const Link &x) {
    const int tid = (threadIdx.z*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x;
    const int block = blockDim.x * blockDim.y * blockDim.z;
#pragma unroll
    for (int i=0; i<9; i++) {
      y[(2*i+0)*block + tid] += x.data[i].real();
      y[(2*i+1)*block + tid] += x.data[i].imag();
    }
  }

  template <typename real, typename Link>
  __device__ inline void sub(real *y, const Link &x) {
    const int tid = (threadIdx.z*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x;
    const int block = blockDim.x * blockDim.y * blockDim.z;
#pragma unroll
    for (int i=0; i<9; i++) {
      y[(2*i+0)*block + tid] -= x.data[i].real();
      y[(2*i+1)*block + tid] -= x.data[i].imag();
    }
  }

#else

  template <typename real, typename Link>
  __device__ inline void axpy(real a, const Link &x, Link &y) { y += a*x;  }

  template <typename Link>
  __device__ inline void add(Link &y, const Link &x) { y += x; }

  template <typename Link>
  __device__ inline void sub(Link &y, const Link &x) { y -= x; }

#endif

#ifdef SHARED_ACCUMULATOR
  template <typename real, bool isConjugate, typename Arg, int mu, int nu, typename Link>
  __device__ __forceinline__ void computeForce(real *force, Arg &arg, int xIndex, int yIndex) {
#else
  template <typename real, bool isConjugate, typename Arg, int mu, int nu, typename Link>
  __device__ __forceinline__ void computeForce(Link &force, Arg &arg, int xIndex, int yIndex) {
#endif

    int otherparity = (1-arg.parity);

    constexpr int tidx = mu > nu ? (mu-1)*mu/2 + nu : (nu-1)*nu/2 + mu;

    if (yIndex == 0) { // do "this" force

      int x[4] = {0, 0, 0, 0};
      getCoordsExtended(x, xIndex, arg.X, arg.parity, arg.border);

      // U[mu](x) U[nu](x+mu) U[*mu](x+nu) U[*nu](x) Oprod(x)
      {
	int d[4] = {0, 0, 0, 0};
	Link U1, U2, U3, U4, Oprod1, Oprod2;

	// load U(x)_(+mu)
	arg.gauge.load((real*)(U1.data), linkIndexShift(x, d, arg.E), mu, arg.parity);

	// load U(x+mu)_(+nu)
	d[mu]++;
	arg.gauge.load((real*)(U2.data), linkIndexShift(x, d, arg.E), nu, otherparity);
	d[mu]--;

	// load U(x+nu)_(+mu)
	d[nu]++;
	arg.gauge.load((real*)(U3.data), linkIndexShift(x, d, arg.E), mu, otherparity);
	d[nu]--;
      
	// load U(x)_(+nu)
	arg.gauge.load((real*)(U4.data), linkIndexShift(x, d, arg.E), nu, arg.parity);

	// load Oprod
	arg.oprod.load((real*)(Oprod1.data), linkIndexShift(x, d, arg.E), tidx, arg.parity);

	if (isConjugate) Oprod1 -= conj(Oprod1);

        if (nu < mu) sub(force, U1*U2*conj(U3)*conj(U4)*Oprod1);
	else   	     add(force, U1*U2*conj(U3)*conj(U4)*Oprod1);

	d[mu]++; d[nu]++;
	arg.oprod.load((real*)(Oprod2.data), linkIndexShift(x, d, arg.E), tidx, arg.parity);
	d[mu]--; d[nu]--;

	if (isConjugate) Oprod2 -= conj(Oprod2);

        if (nu < mu) sub(force, U1*U2*Oprod2*conj(U3)*conj(U4));
	else         add(force, U1*U2*Oprod2*conj(U3)*conj(U4));
      }
 
      {
	int d[4] = {0, 0, 0, 0};
	Link U1, U2, U3, U4, Oprod1, Oprod4;

	// load U(x-nu)(+nu)
	d[nu]--;
	arg.gauge.load((real*)(U1.data), linkIndexShift(x, d, arg.E), nu, otherparity);
	d[nu]++;

	// load U(x-nu)(+mu)
	d[nu]--;
	arg.gauge.load((real*)(U2.data), linkIndexShift(x, d, arg.E), mu, otherparity);
	d[nu]++;

	// load U(x+mu-nu)(nu)
	d[mu]++; d[nu]--;
	arg.gauge.load((real*)(U3.data), linkIndexShift(x, d, arg.E), nu, arg.parity);
	d[mu]--; d[nu]++;

	// load U(x)_(+mu)
	arg.gauge.load((real*)(U4.data), linkIndexShift(x, d, arg.E), mu, arg.parity);

	d[mu]++; d[nu]--;
	arg.oprod.load((real*)(Oprod1.data), linkIndexShift(x, d, arg.E), tidx, arg.parity);
	d[mu]--; d[nu]++;

	if (isConjugate) Oprod1 -= conj(Oprod1);

        if (nu < mu) add(force, conj(U1)*U2*Oprod1*U3*conj(U4));
	else         sub(force, conj(U1)*U2*Oprod1*U3*conj(U4));

	arg.oprod.load((real*)(Oprod4.data), linkIndexShift(x, d, arg.E), tidx, arg.parity);

	if (isConjugate) Oprod4 -= conj(Oprod4);

        if (nu < mu) add(force, Oprod4*conj(U1)*U2*U3*conj(U4));
	else         sub(force, Oprod4*conj(U1)*U2*U3*conj(U4));
      }

    } else { // else do other force

      int y[4] = {0, 0, 0, 0};
      getCoordsExtended(y, xIndex, arg.X, otherparity, arg.border);

      {
	int d[4] = {0, 0, 0, 0};
	Link U1, U2, U3, U4, Oprod3, Oprod4;

	// load U(x)_(+mu)
	arg.gauge.load((real*)(U1.data), linkIndexShift(y, d, arg.E), mu, otherparity);

	// load U(x+mu)_(+nu)
	d[mu]++;
	arg.gauge.load((real*)(U2.data), linkIndexShift(y, d, arg.E), nu, arg.parity);
	d[mu]--;

	// load U(x+nu)_(+mu)
	d[nu]++;
	arg.gauge.load((real*)(U3.data), linkIndexShift(y, d, arg.E), mu, arg.parity);
	d[nu]--;

	// load U(x)_(+nu)
	arg.gauge.load((real*)(U4.data), linkIndexShift(y, d, arg.E), nu, otherparity);

	// load opposite parity Oprod
	d[nu]++;
	arg.oprod.load((real*)(Oprod3.data), linkIndexShift(y, d, arg.E), tidx, arg.parity);
	d[nu]--;

	if (isConjugate) Oprod3 -= conj(Oprod3);

	if (nu < mu) sub(force, U1*U2*conj(U3)*Oprod3*conj(U4));
	else         add(force, U1*U2*conj(U3)*Oprod3*conj(U4));

	// load Oprod(x+mu)
	d[mu]++;
	arg.oprod.load((real*)(Oprod4.data), linkIndexShift(y, d, arg.E), tidx, arg.parity);
	d[mu]--;

	if (isConjugate) Oprod4 -= conj(Oprod4);

	if (nu < mu) sub(force, U1*Oprod4*U2*conj(U3)*conj(U4));
	else         add(force, U1*Oprod4*U2*conj(U3)*conj(U4));
      }

      // Lower leaf
      // U[nu*](x-nu) U[mu](x-nu) U[nu](x+mu-nu) Oprod(x+mu) U[*mu](x)
      {
	int d[4] = {0, 0, 0, 0};
	Link U1, U2, U3, U4, Oprod1, Oprod2;

	// load U(x-nu)(+nu)
	d[nu]--;
	arg.gauge.load((real*)(U1.data), linkIndexShift(y, d, arg.E), nu, arg.parity);
	d[nu]++;

	// load U(x-nu)(+mu)
	d[nu]--;
	arg.gauge.load((real*)(U2.data), linkIndexShift(y, d, arg.E), mu, arg.parity);
	d[nu]++;

	// load U(x+mu-nu)(nu)
	d[mu]++; d[nu]--;
	arg.gauge.load((real*)(U3.data), linkIndexShift(y, d, arg.E), nu, otherparity);
	d[mu]--; d[nu]++;

	// load U(x)_(+mu)
	arg.gauge.load((real*)(U4.data), linkIndexShift(y, d, arg.E), mu, otherparity);

	// load Oprod(x+mu)
	d[mu]++;
	arg.oprod.load((real*)(Oprod1.data), linkIndexShift(y, d, arg.E), tidx, arg.parity);
	d[mu]--;

	if (isConjugate) Oprod1 -= conj(Oprod1);

	if (nu < mu) add(force, conj(U1)*U2*U3*Oprod1*conj(U4));
	else         sub(force, conj(U1)*U2*U3*Oprod1*conj(U4));

	d[nu]--;
	arg.oprod.load((real*)(Oprod2.data), linkIndexShift(y, d, arg.E), tidx, arg.parity);
	d[nu]++;

	if (isConjugate) Oprod2 -= conj(Oprod2);

	if (nu < mu) add(force, conj(U1)*Oprod2*U2*U3*conj(U4));
	else         sub(force, conj(U1)*Oprod2*U2*U3*conj(U4));
      }

    }

  }


  template<typename real, bool isConjugate, typename Arg>
  __global__ void cloverDerivativeKernel(Arg arg)
  {
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if (index >= arg.volumeCB) return;

    // y index determines whether we're updating arg.parity or (1-arg.parity)
    int yIndex = threadIdx.y + blockIdx.y*blockDim.y;
    if (yIndex >= 2) return;

    // mu index is mapped from z thread index
    int mu = threadIdx.z + blockIdx.z*blockDim.z;

    typedef complex<real> Complex;
    typedef Matrix<Complex,3> Link;

    DECLARE_FORCE;

    switch(mu) {
    case 0:
      computeForce<real,isConjugate,Arg,0,1,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,0,2,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,0,3,Link>(force, arg, index, yIndex);
      break;
    case 1:
      computeForce<real,isConjugate,Arg,1,0,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,1,3,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,1,2,Link>(force, arg, index, yIndex);
      break;
    case 2:
      computeForce<real,isConjugate,Arg,2,3,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,2,0,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,2,1,Link>(force, arg, index, yIndex);
      break;
    case 3:
      computeForce<real,isConjugate,Arg,3,2,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,3,1,Link>(force, arg, index, yIndex);
      computeForce<real,isConjugate,Arg,3,0,Link>(force, arg, index, yIndex);
      break;
    }

    // Write to array
    Link F;
    arg.force.load((real*)(F.data), index, mu, yIndex == 0 ? arg.parity : 1-arg.parity);
    axpy(arg.coeff, force, F);
    arg.force.save((real*)(F.data), index, mu, yIndex == 0 ? arg.parity : 1-arg.parity);

    return;
  } // cloverDerivativeKernel
  
  
  template<typename Float, typename Arg>
  class CloverDerivative : public TunableVectorY {
    
  private:
    Arg arg;
    const GaugeField &meta;

    unsigned int sharedBytesPerThread() const { return 18*sizeof(Float); }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

    unsigned int minThreads() const { return arg.volumeCB; }
    bool tuneGridDim() const { return false; }

  public:
    CloverDerivative(const Arg &arg, const GaugeField &meta) : TunableVectorY(2), arg(arg), meta(meta) {
      writeAuxString("conj=%d,threads=%d,prec=%lu,fstride=%d,gstride=%d,ostride=%d",
		     arg.conjugate,arg.volumeCB,sizeof(Float),arg.force.stride,
		     arg.gauge.stride,arg.oprod.stride);
    }
    virtual ~CloverDerivative() {}

    void apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (arg.conjugate) {
	cloverDerivativeKernel<Float,true><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
      } else {
	cloverDerivativeKernel<Float,false><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
      }
    } // apply

    bool advanceBlockDim(TuneParam &param) const {
      dim3 block = param.block;
      dim3 grid = param.grid;
      bool rtn = TunableVectorY::advanceBlockDim(param);
      param.block.z = block.z;
      param.grid.z = grid.z;

      if (!rtn) {
	if (param.block.z < 4) {
	  param.block.z++;
	  param.grid.z = (4 + param.block.z - 1) / param.block.z;
	  rtn = true;
	} else {
	  param.block.z = 1;
	  param.grid.z = 4;
	  rtn = false;
	}
      }
      return rtn;
    }

    void initTuneParam(TuneParam &param) const {
      TunableVectorY::initTuneParam(param);
      param.block.y = 1;
      param.block.z = 1;
      param.grid.y = 2;
      param.grid.z = 4;
    }

    void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

    // The force field is updated so we must preserve its initial state
    void preTune() { arg.force.save(); } 
    void postTune(){ arg.force.load(); } 

    long long flops() const { return 16 * 198 * 3 * 4 * 2 * (long long)arg.volumeCB; }
    long long bytes() const { return ((8*arg.gauge.Bytes() + 4*arg.oprod.Bytes())*3 + 2*arg.force.Bytes()) * 4 * 2 * arg.volumeCB; }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };

  
  template<typename Float>
  void cloverDerivative(cudaGaugeField &force,
			cudaGaugeField &gauge,
			cudaGaugeField &oprod,
			double coeff, int parity,
			int conjugate) {
 
    if (oprod.Reconstruct() != QUDA_RECONSTRUCT_NO) 
      errorQuda("Force field does not support reconstruction");
    
    if (force.Order() != oprod.Order()) 
      errorQuda("Force and Oprod orders must match");
    
    if (force.Reconstruct() != QUDA_RECONSTRUCT_NO) 
      errorQuda("Force field does not support reconstruction");

    if (force.Order() == QUDA_FLOAT2_GAUGE_ORDER){
      typedef gauge::FloatNOrder<Float, 18, 2, 18> F;
      typedef gauge::FloatNOrder<Float, 18, 2, 18> O;

      if (gauge.isNative()) {
	if (gauge.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type G;
	  typedef CloverDerivArg<Float,F,G,O> Arg;
	  Arg arg(F(force), G(gauge), O(oprod), force.X(), oprod.X(), coeff, parity, conjugate);
	  CloverDerivative<Float, Arg> deriv(arg, gauge);
	  deriv.apply(0);
#if 0
	} else if (gauge.Reconstruct() == QUDA_RECONSTRUCT_12) {
	  typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type G;
	  typedef CloverDerivArg<Float,F,G,O> Arg;
	  Arg arg(F(force), G(gauge), O(oprod), force.X(), oprod.X(), coeff, parity, conjugate);
	  CloverDerivative<Float, Arg> deriv(arg, gauge);
	  deriv.apply(0);
#endif
	} else {
	  errorQuda("Reconstruction type %d not supported",gauge.Reconstruct());
	}
      } else {
	errorQuda("Gauge order %d not supported", gauge.Order());
      }
    } else {
      errorQuda("Force order %d not supported", force.Order());
    } // force / oprod order

    hipDeviceSynchronize();
  }
#endif // GPU_CLOVER

void cloverDerivative(cudaGaugeField &force,   
		      cudaGaugeField &gauge,
		      cudaGaugeField &oprod,
		      double coeff, QudaParity parity, int conjugate)
{
#ifdef GPU_CLOVER_DIRAC
  assert(oprod.Geometry() == QUDA_TENSOR_GEOMETRY);
  assert(force.Geometry() == QUDA_VECTOR_GEOMETRY);

  for (int d=0; d<4; d++) {
    if (oprod.X()[d] != gauge.X()[d])
      errorQuda("Incompatible extended dimensions d=%d gauge=%d oprod=%d", d, gauge.X()[d], oprod.X()[d]);
  }

  int device_parity = (parity == QUDA_EVEN_PARITY) ? 0 : 1;

  if(force.Precision() == QUDA_DOUBLE_PRECISION){
    cloverDerivative<double>(force, gauge, oprod, coeff, device_parity, conjugate);
#if 0
  } else if (force.Precision() == QUDA_SINGLE_PRECISION){
    cloverDerivative<float>(force, gauge, oprod, coeff, device_parity, conjugate);
#endif
  } else {
    errorQuda("Precision %d not supported", force.Precision());
  }

  return;
#else
  errorQuda("Clover has not been built");
#endif
}              


} // namespace quda
