#include "hip/hip_runtime.h"
#include <transfer.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <clover_field_order.h>
#include <complex_quda.h>

namespace quda {

  //A simple Euclidean gamma matrix class for use with the Wilson projectors.
  template <typename ValueType, QudaGammaBasis basis, int dir>
  class Gamma {
  private:
    const int ndim;

  protected:


    //Which gamma matrix (dir = 0,4)
    //dir = 0: gamma^1, dir = 1: gamma^2, dir = 2: gamma^3, dir = 3: gamma^4, dir =4: gamma^5
    //int dir;

    //The basis to be used.
    //QUDA_DEGRAND_ROSSI_GAMMA_BASIS is the chiral basis
    //QUDA_UKQCD_GAMMA_BASIS is the non-relativistic basis.
    //QudaGammaBasis basis;

    //The column with the non-zero element for each row
    int coupling[4];
    //The value of the matrix element, for each row
    complex<ValueType> elem[4];

  public:

    Gamma() : ndim(4) {
      complex<ValueType> I(0,1);
      if((dir==0) || (dir==1)) {
	coupling[0] = 3;
	coupling[1] = 2;
	coupling[2] = 1;
	coupling[3] = 0;
      } else if (dir == 2) {
	coupling[0] = 2;
	coupling[1] = 3;
	coupling[2] = 0;
	coupling[3] = 1;
      } else if ((dir == 3) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	coupling[0] = 2;
	coupling[1] = 3;
	coupling[2] = 0;
	coupling[3] = 1;
      } else if ((dir == 3) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	coupling[0] = 0;
	coupling[1] = 1;
	coupling[2] = 2;
	coupling[3] = 3;
      } else if ((dir == 4) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	coupling[0] = 0;
	coupling[1] = 1;
	coupling[2] = 2;
	coupling[3] = 3;
      } else if ((dir == 4) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	coupling[0] = 2;
	coupling[1] = 3;
	coupling[2] = 0;
	coupling[3] = 1;
      } else {
	printfQuda("Warning: Gamma matrix not defined for dir = %d and basis = %d\n", dir, basis);
	coupling[0] = 0;
	coupling[1] = 0;
	coupling[2] = 0;
	coupling[3] = 0;
      }


      if((dir==0)) {
	elem[0] = I;
	elem[1] = I;
	elem[2] = -I;
	elem[3] = -I;
      } else if((dir==1) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	elem[0] = -1;
	elem[1] = 1;
	elem[2] = 1;
	elem[3] = -1;
      } else if((dir==1) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = -1;
	elem[2] = -1;
	elem[3] = 1;
      } else if((dir==2)) {
	elem[0] = I;
	elem[1] = -I;
	elem[2] = -I;
	elem[3] = I;
      } else if((dir==3) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = 1;
	elem[2] = 1;
	elem[3] = 1;
      } else if((dir==3) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = 1;
	elem[2] = -1;
	elem[3] = -1;
      } else if((dir==4) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	elem[0] = -1;
	elem[1] = -1;
	elem[2] = 1;
	elem[3] = 1;
      } else if((dir==4) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = 1;
	elem[2] = 1;
	elem[3] = 1;
      } else {
	elem[0] = 0;
	elem[1] = 0;
	elem[2] = 0;
	elem[3] = 0;
      }
    } 

    Gamma(const Gamma &g) : ndim(4) {
      for(int i = 0; i < ndim+1; i++) {
	coupling[i] = g.coupling[i];
	elem[i] = g.elem[i];
      }
    }

    ~Gamma() {}

    //Returns the matrix element.
    __device__ __host__ inline complex<ValueType> getelem(int row, int col) const {
      return coupling[row] == col ? elem[row] : 0;
    }

    //Like getelem, but one only needs to specify the row.
    //The column of the non-zero component is returned via the "col" reference
    __host__ __device__ inline complex<ValueType> getrowelem(int row, int &col) const {
      col = coupling[row];
      return elem[row];
    }

    //Returns the type of Gamma matrix
    inline int Dir() const {
      return dir;
    }
  };

  //Returns the non parity-blocked integer index for a lattice site.  Also calculates the parity index of a site.
  int gauge_offset_index(const int *x, const int *x_size, int ndim, int& parity) {
    parity = 0;
    int gauge_index = 0;
    for(int d = ndim-1; d >= 0; d--) {
      parity += x[d];
      gauge_index *= x_size[d];
      gauge_index += x[d];
    }
    parity = parity%2;
    return gauge_index;
  }

  //Calculates the matrix UV^{s,c'}_mu(x) = \sum_c U^{c}_mu(x) * V^{s,c}_mu(x+mu)
  //Where:
  //mu = dir
  //s = fine spin
  //c' = coarse color
  //c = fine color
  //FIXME: N.B. Only works if color-spin field and gauge field are parity ordered in the same way.  Need LatticeIndex function for generic ordering
  template<typename Float, int dir, typename F, typename fineGauge>
  void computeUV(F &UV, const F &V, const fineGauge &G, int ndim, const int *x_size) {
	
    int coord[QUDA_MAX_DIM];
    for (int parity=0; parity<2; parity++) {
      int x_cb = 0;
      for (coord[3]=0; coord[3]<x_size[3]; coord[3]++) {
	for (coord[2]=0; coord[2]<x_size[2]; coord[2]++) {
	  for (coord[1]=0; coord[1]<x_size[1]; coord[1]++) {
	    for (coord[0]=0; coord[0]<x_size[0]/2; coord[0]++) {
	      int coord_tmp = coord[dir];

	      //Shift the V field w/respect to G (must be on full field coords)
	      int oddBit = (parity + coord[1] + coord[2] + coord[3])&1;
	      if (dir==0) coord[0] = 2*coord[0] + oddBit;
	      coord[dir] = (coord[dir]+1)%x_size[dir];
	      if (dir==0) coord[0] /= 2;
	      int y_cb = ((coord[3]*x_size[2]+coord[2])*x_size[1]+coord[1])*(x_size[0]/2) + coord[0];

              for(int s = 0; s < V.Nspin(); s++) {  //Fine Spin
		for(int ic_c = 0; ic_c < V.Nvec(); ic_c++) {  //Coarse Color
                  for(int ic = 0; ic < G.Ncolor(); ic++) { //Fine Color rows of gauge field
		    for(int jc = 0; jc < G.Ncolor(); jc++) {  //Fine Color columns of gauge field
		      UV(parity, x_cb, s, ic, ic_c) += G(dir, parity, x_cb, ic, jc) * V((parity+1)&1, y_cb, s, jc, ic_c);
		    }  //Fine color columns
		  }  //Fine color rows
		}  //Coarse color
	      }  //Fine Spin

	      coord[dir] = coord_tmp; //restore
	      x_cb++;
	    }
	  }
	}
      }
    } // parity

  }  //UV

  template<typename Float, int dir, typename F, typename coarseGauge, typename fineGauge, typename Gamma>
  void computeVUV(coarseGauge &Y, coarseGauge &X, const F &UV, const F &V, 
		  const Gamma &gamma, const fineGauge &G, const int *x_size, 
		  const int *xc_size, const int *geo_bs, int spin_bs) {

    const int nDim = 4;
    const Float half = 0.5;
    int coarse_size = 1;
    for(int d = 0; d<nDim; d++) coarse_size *= xc_size[d];
    int coord[QUDA_MAX_DIM];
    int coord_coarse[QUDA_MAX_DIM];

    // paralleling this requires care with respect to race conditions
    // on CPU, parallelize over dimension not parity

    //#pragma omp parallel for 
    for (int parity=0; parity<2; parity++) {
      int x_cb = 0;
      for (coord[3]=0; coord[3]<x_size[3]; coord[3]++) {
	for (coord[2]=0; coord[2]<x_size[2]; coord[2]++) {
	  for (coord[1]=0; coord[1]<x_size[1]; coord[1]++) {
	    for (coord[0]=0; coord[0]<x_size[0]/2; coord[0]++) {

	      int oddBit = (parity + coord[1] + coord[2] + coord[3])&1;
	      coord[0] = 2*coord[0] + oddBit;
	      for(int d = 0; d < nDim; d++) coord_coarse[d] = coord[d]/geo_bs[d];

	      //Check to see if we are on the edge of a block, i.e.
	      //if this color matrix connects adjacent blocks.  If
	      //adjacent site is in same block, M = X, else M = Y
	      const bool isDiagonal = ((coord[dir]+1)%x_size[dir])/geo_bs[dir] == coord_coarse[dir] ? true : false;
	      coarseGauge &M =  isDiagonal ? X : Y;
	      const int dim_index = isDiagonal ? 0 : dir;
	      
	      //printf("dir = %d (%d,%d,%d,%d)=(%d,%d) (%d,%d,%d,%d)=", dir, 
	      //   coord[0], coord[1], coord[2], coord[3], x_cb, parity,
	      //   coord_coarse[0], coord_coarse[1], coord_coarse[2], coord_coarse[3]);

	      int coarse_parity = 0;
	      for (int d=0; d<nDim; d++) coarse_parity += coord_coarse[d];
	      coarse_parity &= 1;
	      coord_coarse[0] /= 2;
	      int coarse_x_cb = ((coord_coarse[3]*xc_size[2]+coord_coarse[2])*xc_size[1]+coord_coarse[1])*(xc_size[0]/2) + coord_coarse[0];
	      
	      //printf("(%d,%d)\n", coarse_x_cb, coarse_parity);

	      coord[0] /= 2;

  	        for(int s = 0; s < V.Nspin(); s++) { //Loop over fine spin
		  //Spin part of the color matrix.  Will always consist
		  //of two terms - diagonal and off-diagonal part of
		  //P_mu = (1+\gamma_mu)
		
		  int s_c_row = s/spin_bs; //Coarse spin row index
	
		  //Use Gamma to calculate off-diagonal coupling and
		  //column index.  Diagonal coupling is always 1.
		  int s_col;
		  complex<Float> coupling = gamma.getrowelem(s, s_col);
		  int s_c_col = s_col/spin_bs;

		  for(int ic_c = 0; ic_c < Y.NcolorCoarse(); ic_c++) { //Coarse Color row
		    for(int jc_c = 0; jc_c < Y.NcolorCoarse(); jc_c++) { //Coarse Color column
		      for(int ic = 0; ic < G.Ncolor(); ic++) { //Sum over fine color
		        //Diagonal Spin
		        M(dim_index,coarse_parity,coarse_x_cb,s_c_row,s_c_row,ic_c,jc_c) += 
			  half * conj(V(parity, x_cb, s, ic, ic_c)) * UV(parity, x_cb, s, ic, jc_c); 
		      
		        //Off-diagonal Spin
		        M(dim_index,coarse_parity,coarse_x_cb,s_c_row,s_c_col,ic_c,jc_c) += 
			  half * coupling * conj(V(parity, x_cb, s, ic, ic_c)) * UV(parity, x_cb, s_col, ic, jc_c);
		      } //Fine color
		    } //Coarse Color column
		  } //Coarse Color row

	        } //Fine spin

	      x_cb++;
	    } // coord[0]
	  } // coord[1]
	} // coord[2]
      } // coord[3]
    } // parity

  }


  //Adds the identity matrix to the coarse local term.
  template<typename Float, typename Gauge>
  void addCoarseDiagonal(Gauge &X, int ndim, const int *xc_size) {
    const int nColor = X.NcolorCoarse();
    const int nSpin = X.NspinCoarse();
    complex<Float> *Xlocal = new complex<Float>[nSpin*nSpin*nColor*nColor];

    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<X.Volume()/2; x_cb++) {
        for(int s = 0; s < nSpin; s++) { //Spin
         for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color
            X(0,parity,x_cb,s,s,ic_c,ic_c) += 1.0;
         } //Color
        } //Spin
      } // x_cb
    } //parity
   }


  //Adds the reverse links to the coarse local term, which is just
  //the conjugate of the existing coarse local term but with
  //plus/minus signs for off-diagonal spin components
  //Also multiply by the appropriate factor of -2*kappa
  template<typename Float, typename Gauge>
  void createCoarseLocal(Gauge &X, int ndim, const int *xc_size, double kappa) {
    const int nColor = X.NcolorCoarse();
    const int nSpin = X.NspinCoarse();
    Float kap = (Float) kappa;
    complex<Float> *Xlocal = new complex<Float>[nSpin*nSpin*nColor*nColor];
	
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<X.Volume()/2; x_cb++) {

	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
	    
	    //Copy the Hermitian conjugate term to temp location 
	    for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Flip s_col, s_row on the rhs because of Hermitian conjugation.  Color part left untransposed.
		Xlocal[((nSpin*s_col+s_row)*nColor+ic_c)*nColor+jc_c] = X(0,parity,x_cb,s_row, s_col, ic_c, jc_c);
	      }	
	    }
	  }
	}
	      
	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
	    
	    const Float sign = (s_row == s_col) ? 1.0 : -1.0;
		  
	    for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Transpose color part
		X(0,parity,x_cb,s_row,s_col,ic_c,jc_c) =  
		  -2*kap*(sign*X(0,parity,x_cb,s_row,s_col,ic_c,jc_c)+conj(Xlocal[((nSpin*s_row+s_col)*nColor+jc_c)*nColor+ic_c]));
	      } //Color column
	    } //Color row
	  } //Spin column
	} //Spin row

      } // x_cb
    } //parity

    delete []Xlocal;

  }

  //Zero out a field, using the accessor.
  template<typename Float, typename F>
  void setZero(F &f) {
    for(int parity = 0; parity < 2; parity++) {
      for(int x_cb = 0; x_cb < f.Volume()/2; x_cb++) {
	for(int s = 0; s < f.Nspin(); s++) {
	  for(int c = 0; c < f.Ncolor(); c++) {
	    for(int v = 0; v < f.Nvec(); v++) {
	      f(parity,x_cb,s,c,v) = (Float) 0.0;
	    }
	  }
	}
      }
    }
  }

  template<typename Float, typename coarseGauge, typename F, typename clover>
  void createCoarseClover(coarseGauge &X, F &V,  clover &C, int ndim, const int *x_size, const int *xc_size, const int *geo_bs, int spin_bs)  {

    const int nDim = 4;
    const Float half = 0.5;
    int coord[QUDA_MAX_DIM];
    int coord_coarse[QUDA_MAX_DIM];
    int coarse_size = 1;
    for(int d = 0; d<nDim; d++) coarse_size *= xc_size[d];

    for (int parity=0; parity<2; parity++) {
      int x_cb = 0;
      for (coord[3]=0; coord[3]<x_size[3]; coord[3]++) {
        for (coord[2]=0; coord[2]<x_size[2]; coord[2]++) {
          for (coord[1]=0; coord[1]<x_size[1]; coord[1]++) {
            for (coord[0]=0; coord[0]<x_size[0]/2; coord[0]++) {

              int oddBit = (parity + coord[1] + coord[2] + coord[3])&1;
              coord[0] = 2*coord[0] + oddBit;
              for(int d = 0; d < nDim; d++) coord_coarse[d] = coord[d]/geo_bs[d];
              int coarse_parity = 0;
              for (int d=0; d<nDim; d++) coarse_parity += coord_coarse[d];
              coarse_parity &= 1;
              coord_coarse[0] /= 2;
              int coarse_x_cb = ((coord_coarse[3]*xc_size[2]+coord_coarse[2])*xc_size[1]+coord_coarse[1])*(xc_size[0]/2) + coord_coarse[0];

              coord[0] /= 2;

	      int s_c = 0;

              //If Nspin = 4, then the clover term has structure C_{\mu\nu} = \gamma_{\mu\nu}C^{\mu\nu}

                //printf("C.Ncolor() = %d C.NcolorCoarse() = %d\n",C.Ncolor(), C.NcolorCoarse());
                for(int s = 0; s < V.Nspin(); s++) { //Loop over fine spin row
		  s_c = s/spin_bs;
		  //On the fine lattice, the clover field is chirally blocked, so loop over rows/columns
		  //in the same chiral block.
                  for(int s_col = s_c*spin_bs; s_col < (s_c+1)*spin_bs; s_col++) { //Loop over fine spin column
                    for(int ic_c = 0; ic_c < X.NcolorCoarse(); ic_c++) { //Coarse Color row
                      for(int jc_c = 0; jc_c < X.NcolorCoarse(); jc_c++) { //Coarse Color column

                        for(int ic = 0; ic < C.Ncolor(); ic++) { //Sum over fine color row
                          for(int jc = 0; jc < C.Ncolor(); jc++) {  //Sum over fine color column
			    X(0,coarse_parity,coarse_x_cb,s_c,s_c,ic_c,jc_c) += conj(V(parity, x_cb, s, ic, ic_c)) * C(0, parity, x_cb, s, s_col, ic, jc) * V(parity, x_cb, s_col, jc, jc_c);
                          } //Fine color column
                        }  //Fine color row
                      } //Coarse Color column
                    } //Coarse Color row
                  }  //Fine spin column
                } //Fine spin


              x_cb++;
            } // coord[0]
          } // coord[1]
        } // coord[2]
      } // coord[3]
    } // parity

  }

  //Calculates the coarse gauge field
  template<typename Float, typename F, typename coarseGauge, typename fineGauge, typename fineClover>
  void calculateY(coarseGauge &Y, coarseGauge &X, F &UV, F &V, fineGauge &G, fineClover *C, const int *x_size, const int *xc_size, double kappa) {
    if (UV.GammaBasis() != QUDA_DEGRAND_ROSSI_GAMMA_BASIS) errorQuda("Gamma basis not supported");
    const QudaGammaBasis basis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;

    if (G.Ndim() != 4) errorQuda("Number of dimensions not supported");
    const int nDim = 4;

    int geo_bs[QUDA_MAX_DIM]; 
    for(int d = 0; d < nDim; d++) geo_bs[d] = x_size[d]/xc_size[d];
    int spin_bs = V.Nspin()/Y.NspinCoarse();

    for(int d = 0; d < nDim; d++) {
      //First calculate UV
      setZero<Float,F>(UV);

      printfQuda("Computing %d UV and VUV\n", d);
      //Calculate UV and then VUV for this direction, accumulating directly into the coarse gauge field Y
      if (d==0) {
        computeUV<Float,0>(UV, V, G, nDim, x_size);
        Gamma<Float, basis, 0> gamma;
        computeVUV<Float,0>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      } else if (d==1) {
        computeUV<Float,1>(UV, V, G, nDim, x_size);
        Gamma<Float, basis, 1> gamma;
        computeVUV<Float,1>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      } else if (d==2) {
        computeUV<Float,2>(UV, V, G, nDim, x_size);
        Gamma<Float, basis, 2> gamma;
        computeVUV<Float,2>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      } else {
        computeUV<Float,3>(UV, V, G, nDim, x_size);
        Gamma<Float, basis, 3> gamma;
        computeVUV<Float,3>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      }

      printf("UV2[%d] = %e\n", d, UV.norm2());
      printf("Y2[%d] = %e\n", d, Y.norm2(d));
    }
    printf("X2 = %e\n", X.norm2(0));
    printfQuda("Computing coarse diagonal\n");
    createCoarseLocal<Float>(X, nDim, xc_size, kappa);

    //If C!=NULL we have to coarsen the fine clover term and add it in.
    if (C != NULL) {
      printfQuda("Computing fine->coarse clover term\n");
      createCoarseClover<Float>(X, V, *C, nDim, x_size, xc_size, geo_bs, spin_bs);
      printf("X2 = %e\n", X.norm2(0));
    }
    //Otherwise, we have a fine Wilson operator.  The "clover" term for the Wilson operator
    //is just the identity matrix.
    else {
      addCoarseDiagonal<Float>(X, nDim, xc_size);
    }
    printf("X2 = %e\n", X.norm2(0));

}


  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder,
            int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
    typedef typename gauge::FieldOrder<Float,fineColor,1,gOrder> gFine;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> gCoarse;
    typedef typename clover::FieldOrder<Float,fineColor,fineSpin,clOrder> cFine;

    F vAccessor(const_cast<ColorSpinorField&>(T.Vectors()));
    F uvAccessor(const_cast<ColorSpinorField&>(uv));
    gFine gAccessor(const_cast<GaugeField&>(g));
    gCoarse yAccessor(const_cast<GaugeField&>(Y));
    gCoarse xAccessor(const_cast<GaugeField&>(X));

    if(c != NULL) {
      cFine cAccessor(const_cast<CloverField&>(*c));

      calculateY<Float>(yAccessor, xAccessor, uvAccessor, vAccessor, gAccessor, &cAccessor, g.X(), Y.X(), kappa);
    }
    else {
      cFine *cAccessor = NULL;
      calculateY<Float>(yAccessor, xAccessor, uvAccessor, vAccessor, gAccessor, cAccessor, g.X(), Y.X(), kappa);
    }    
  }

  // template on the number of coarse degrees of freedom
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder, int fineColor, int fineSpin>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (T.Vectors().Nspin()/T.Spin_bs() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 2) {
      calculateY<Float,csOrder,gOrder,clOrder,fineColor,fineSpin,2,coarseSpin>(Y, X, uv, T, g, c, kappa);
    } else if (coarseColor == 24) {
      calculateY<Float,csOrder,gOrder,clOrder,fineColor,fineSpin,24,coarseSpin>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder, int fineColor>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (uv.Nspin() == 4) {
      calculateY<Float,csOrder,gOrder,clOrder,fineColor,4>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported number of spins %d\n", uv.Nspin());
    }
  }

  // template on fine colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (g.Ncolor() == 3) {
      calculateY<Float,csOrder,gOrder,clOrder,3>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported number of colors %d\n", g.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    //If c == NULL, then this is standard Wilson.  csOrder is dummy and will not matter      
    if (c==NULL || c->Order() == QUDA_PACKED_CLOVER_ORDER) {
      calculateY<Float,csOrder,gOrder,QUDA_PACKED_CLOVER_ORDER>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", c->Order());
    }
  }

  template <typename Float, QudaFieldOrder csOrder>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (g.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      calculateY<Float,csOrder,QUDA_QDP_GAUGE_ORDER>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", g.FieldOrder());
    }
  }

 template <typename Float>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (T.Vectors().FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      calculateY<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", T.Vectors().FieldOrder());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (X.Precision() != Y.Precision() || Y.Precision() != uv.Precision() ||
        Y.Precision() != T.Vectors().Precision() || Y.Precision() != g.Precision())
      errorQuda("Unsupported precision mix");

    printfQuda("Computing Y field......\n");
    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      calculateY<double>(Y, X, uv, T, g, c, kappa);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      calculateY<float>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    printfQuda("....done computing Y field\n");
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseOp(const Transfer &T, GaugeField &Y, GaugeField &X, const cudaGaugeField &gauge, const cudaCloverField *clover, double kappa) {
    QudaPrecision precision = Y.Precision();
    //First make a cpu gauge field from the cuda gauge field

    int pad = 0;
    GaugeFieldParam gf_param(gauge.X(), precision, gauge.Reconstruct(), pad, gauge.Geometry());
    gf_param.order = QUDA_QDP_GAUGE_ORDER;
    gf_param.fixed = gauge.GaugeFixed();
    gf_param.link_type = gauge.LinkType();
    gf_param.t_boundary = gauge.TBoundary();
    gf_param.anisotropy = gauge.Anisotropy();
    gf_param.gauge = NULL;
    gf_param.create = QUDA_NULL_FIELD_CREATE;
    gf_param.siteSubset = QUDA_FULL_SITE_SUBSET;

    cpuGaugeField g(gf_param);

    //Copy the cuda gauge field to the cpu
    gauge.saveCPUField(g, QUDA_CPU_FIELD_LOCATION);

    //Create a field UV which holds U*V.  Has the same structure as V.
    ColorSpinorParam UVparam(T.Vectors());
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    cpuColorSpinorField uv(UVparam);

    //If the fine lattice operator is the clover operator, copy the cudaCloverField to cpuCloverField
    if(clover != NULL) {
      //Create a cpuCloverField from the cudaCloverField
      CloverFieldParam cf_param;
      cf_param.nDim = 4;
      cf_param.pad = pad;
      cf_param.precision = clover->Precision();
      for(int i = 0; i < cf_param.nDim; i++) {
        cf_param.x[i] = clover->X()[i];
      }

      cf_param.order = QUDA_PACKED_CLOVER_ORDER;
      cf_param.direct = true;
      cf_param.inverse = true;
      cf_param.clover = NULL;
      cf_param.norm = 0;
      cf_param.cloverInv = NULL;
      cf_param.invNorm = 0;
      cf_param.create = QUDA_NULL_FIELD_CREATE;
      cf_param.siteSubset = QUDA_FULL_SITE_SUBSET;

      cpuCloverField c(cf_param);
      clover->saveCPUField(c);

      calculateY(Y, X, uv, T, g, &c, kappa);
    }
    else {
      calculateY(Y, X, uv, T, g, NULL, kappa);
    }
  }

  //Adds the reverse links to the coarse local term, which is just
  //the conjugate of the existing coarse local term but with
  //plus/minus signs for off-diagonal spin components
  //Also multiply by the appropriate factor of -2*kappa
  template<typename Float, typename Gauge>
  void createKSCoarseLocal(Gauge &X, int ndim, const int *xc_size, double k) {
    const int nColor = X.NcolorCoarse();
    const int nSpin = X.NspinCoarse();
    if (nSpin != 2) errorQuda("\nWrong coarse spin degrees.\n");

    Float kap = (Float) k;//mass term
    complex<Float> *Xlocal = new complex<Float>[nSpin*nSpin*nColor*nColor];
	
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<X.Volume()/2; x_cb++) {

	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
	    
	    //Copy the Hermitian conjugate term to temp location 
	    for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Flip s_col, s_row on the rhs because of Hermitian conjugation.  Color part left untransposed.
		Xlocal[((nSpin*s_col+s_row)*nColor+ic_c)*nColor+jc_c] = X(0,parity,x_cb,s_row, s_col, ic_c, jc_c);
	      }	
	    }
	  }
	}
	      
	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
            for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
//!
	      if(s_row == s_col){
                X(0,parity,x_cb, parity, parity, ic_c,ic_c) += (parity == 0) ? +1.0 : -1.0;
                continue;
              }
//!
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Transpose color part
		X(0,parity,x_cb,s_row,s_col,ic_c,jc_c) = 2*kap*(+X(0,parity,x_cb,s_row,s_col,ic_c,jc_c)-conj(Xlocal[((nSpin*s_row+s_col)*nColor+jc_c)*nColor+ic_c]));//always minus sign?
	      } //Color column
	    } //Color row
	  } //Spin column
	} //Spin row

      } // x_cb
    } //parity

    delete[] Xlocal;

    return;
  }

  //added HISQ links
  template<typename Float, int dir, typename F, typename fineGauge>
  void computeKSUV(F &UV, const F &V, const fineGauge *FL, const fineGauge *LL, int ndim, 
                   const int *x_size) {
	
    int coord[QUDA_MAX_DIM];
     
    for (int parity=0; parity<2; parity++) {
      int x_cb = 0;
      for (coord[3]=0; coord[3]<x_size[3]; coord[3]++) {
	for (coord[2]=0; coord[2]<x_size[2]; coord[2]++) {
	  for (coord[1]=0; coord[1]<x_size[1]; coord[1]++) {
	    for (coord[0]=0; coord[0]<x_size[0]/2; coord[0]++) {
	      int coord_tmp  = coord[dir];
              int coord_3[4] = {coord[0], coord[1], coord[2], coord[3]} ; 

	      //Shift the V field w/respect to G (must be on full field coords)
	      int oddBit = (parity + coord[1] + coord[2] + coord[3]) & 1;
	      if (dir==0) coord[0] = 2*coord[0] + oddBit;
              //!
 	      coord[dir]   = (coord[dir]+1)%x_size[dir];
              if(LL != NULL) coord_3[dir] = (coord_3[dir]+3)%x_size[dir];

	      if (dir==0) {coord[0] /= 2; coord_3[0] /= 2;}

	      int y_cb = ((coord[3]*x_size[2]+coord[2])*x_size[1]+coord[1])*(x_size[0]/2) + coord[0];
              int y3_cb = (LL != NULL) ? (((coord_3[3]*x_size[2]+coord_3[2])*x_size[1]+coord_3[1])*(x_size[0]/2) + coord_3[0]) : 0;

	      for(int ic_c = 0; ic_c < V.Nvec(); ic_c++) {  //Coarse Color
                for(int ic = 0; ic < FL->Ncolor(); ic++) { //Fine Color rows of gauge field
		   for(int jc = 0; jc < FL->Ncolor(); jc++) {  //Fine Color columns of gauge field
		      UV(parity, x_cb, 0, ic, ic_c) += (*FL)(dir, parity, x_cb, ic, jc) * V((parity+1)&1, y_cb, 0, jc, ic_c);//mind transformation to the opposite parity field: in UVU operation.
                      if(LL != NULL) UV(parity, x_cb, 0, ic, ic_c) += (*LL)(dir, parity, x_cb, ic, jc) * V((parity+1)&1, y3_cb, 0, jc, ic_c);
		   }  //Fine color columns
		}  //Fine color rows
	      }  //Coarse color

	      coord[dir] = coord_tmp; //restore
	      x_cb++;
	    }
	  }
	}
      }
    } // parity

  }  //UV

//KS (also HISQ) operator:
  template<typename Float, int dir, typename F, typename coarseGauge>
  void computeKSVUV(coarseGauge &Y, coarseGauge &X, const F &UV, const F &V, const int nfinecolors,
		  const int *x_size, const int *xc_size, const int *geo_bs) {

    const int nDim = 4;
    Float half = 0.5;
    int coarse_size = 1;
    for(int d = 0; d<nDim; d++) coarse_size *= xc_size[d];
    int coord[QUDA_MAX_DIM];
    int coord_coarse[QUDA_MAX_DIM];

    // paralleling this requires care with respect to race conditions
    // on CPU, parallelize over dimension not parity
    Float eta = 1.0;

    //#pragma omp parallel for 
    for (int parity=0; parity<2; parity++) {
      int x_cb = 0;
      for (coord[3]=0; coord[3]<x_size[3]; coord[3]++) {
        if(dir == 3) eta *= -1.0;
	for (coord[2]=0; coord[2]<x_size[2]; coord[2]++) {
          if(dir >= 2) eta *= -1.0;
	  for (coord[1]=0; coord[1]<x_size[1]; coord[1]++) {
            if(dir >= 1) eta *= -1.0;
	    for (coord[0]=0; coord[0]<x_size[0]/2; coord[0]++) {

	      int oddBit = (parity + coord[1] + coord[2] + coord[3])&1;
	      coord[0] = 2*coord[0] + oddBit;
	      for(int d = 0; d < nDim; d++) coord_coarse[d] = coord[d]/geo_bs[d];

	      //Check to see if we are on the edge of a block, i.e.
	      //if this color matrix connects adjacent blocks.  If
	      //adjacent site is in same block, M = X, else M = Y
	      bool isDiagonal = (((coord[dir]+1)%x_size[dir])/geo_bs[dir] == coord_coarse[dir]) || (((coord[dir]+3)%x_size[dir])/geo_bs[dir] == coord_coarse[dir]) ? true : false;

	      coarseGauge &M =  isDiagonal ? X : Y;
	      const int dim_index = isDiagonal ? 0 : dir;

	      int coarse_parity = 0;
	      for (int d=0; d<nDim; d++) coarse_parity += coord_coarse[d];
	      coarse_parity &= 1;
	      coord_coarse[0] /= 2;
	      int coarse_x_cb = ((coord_coarse[3]*xc_size[2]+coord_coarse[2])*xc_size[1]+coord_coarse[1])*(xc_size[0]/2) + coord_coarse[0];
	      
	      //printf("(%d,%d)\n", coarse_x_cb, coarse_parity);

	      coord[0] /= 2;

              int coarse_spin_row = parity == 0 ? 0 : 1  ;
              int coarse_spin_col = (1 - coarse_spin_row); 

              half *= eta; //multiply by sing factor 

              for(int ic_c = 0; ic_c < Y.NcolorCoarse(); ic_c++) { //Coarse Color row
		for(int jc_c = 0; jc_c < Y.NcolorCoarse(); jc_c++) { //Coarse Color column
		  for(int ic = 0; ic < nfinecolors; ic++) { //Sum over fine color
		      M(dir,coarse_parity,coarse_x_cb,coarse_spin_row, coarse_spin_col,ic_c,jc_c) += half*conj(V(parity, x_cb, 0, ic, ic_c)) * UV(parity, x_cb, 0, ic, jc_c);
		  } //Fine color
		} //Coarse Color column
	      } //Coarse Color row
	      x_cb++;
	    } // coord[0]
	  } // coord[1]
	} // coord[2]
      } // coord[3]
    } // parity
    
    return;
  }

 //Calculates the coarse gauge field: separated from coarseSpin = 2 computations:
  template<typename Float, typename F, typename coarseGauge, typename fineGauge>
  void calculateKSY(coarseGauge &Y, coarseGauge &X, F &UV, F &V, fineGauge *FL, fineGauge *LL, const int *x_size, double k) {

    if (FL->Ndim() != 4) errorQuda("Number of dimensions not supported");
    const int nDim = 4;

    const int *xc_size = Y.Field().X();
    int geo_bs[QUDA_MAX_DIM]; 
    for(int d = 0; d < nDim; d++) geo_bs[d] = x_size[d]/xc_size[d];

    for(int d = 0; d < nDim; d++) 
    {
      //First calculate UV
      setZero<Float,F>(UV);

      printfQuda("Computing %d UV and VUV\n", d);
      //Calculate UV and then VUV for this direction, accumulating directly into the coarse gauge field Y
      if (d==0) {
        computeKSUV<Float,0>(UV, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,0>(Y, X, UV, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      } else if (d==1) {
        computeKSUV<Float,1>(UV, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,1>(Y, X, UV, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      } else if (d==2) {
        computeKSUV<Float,2>(UV, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,2>(Y, X, UV, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      } else {
        computeKSUV<Float,3>(UV, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,3>(Y, X, UV, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      }

      printf("UV2[%d] = %e\n", d, UV.norm2());
      printf("Y2[%d] = %e\n", d, Y.norm2(d));
    }

    printf("X2 = %e\n", X.norm2(0));
    printfQuda("Computing coarse diagonal\n");
    createKSCoarseLocal<Float>(X, nDim, xc_size, k);

    printf("X2 = %e\n", X.norm2(0));

  }



  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int fineColor, int coarseColor, int coarseSpin>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField *f, GaugeField *l, double k) {

    const int fineSpin = 1;

    typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
    typedef typename gauge::FieldOrder<Float,fineColor,1,gOrder> gFine;
    typedef typename gauge::FieldOrder<Float,coarseSpin*coarseColor,1,gOrder> gCoarse;

    F vAccessor(const_cast<ColorSpinorField&>(T.Vectors()));
    F uvAccessor(const_cast<ColorSpinorField&>(uv));
    gFine fAccessor(const_cast<GaugeField&>(*f));
    gCoarse yAccessor(const_cast<GaugeField&>(Y));
    gCoarse xAccessor(const_cast<GaugeField&>(X));

    if(l != NULL) {
      gFine lAccessor(const_cast<GaugeField&>(*l));
      calculateKSY<Float>(yAccessor, xAccessor, uvAccessor, vAccessor, &fAccessor, &lAccessor, f->X(), k);
    }
    else {
      gFine *lAccessor = NULL;
      calculateKSY<Float>(yAccessor, xAccessor, uvAccessor, vAccessor, &fAccessor, lAccessor, f->X(), k);
    }    
  }

  // template on the number of coarse degrees of freedom
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int fineColor>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField *f, GaugeField *l, double k) {

    if ((T.Vectors().Nspin() != 1) && (T.Vectors().Nspin()/T.Spin_bs() != 2))  errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 2) {
      calculateKSY<Float,csOrder,gOrder,fineColor,2, coarseSpin>(Y, X, uv, T, f, l, k);
    } else if (coarseColor == 24) {
      calculateKSY<Float,csOrder,gOrder,fineColor,24, coarseSpin>(Y, X, uv, T, f, l, k);
    } else if (coarseColor == 48) {
      calculateKSY<Float,csOrder,gOrder,fineColor,48, coarseSpin>(Y, X, uv, T, f, l, k);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }


  // template on fine colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (f->Ncolor() == 3) {
      if( !l ) if( f->Ncolor() != l->Ncolor() ) errorQuda("Unsupported number of colors %d\n", l->Ncolor());
      calculateKSY<Float,csOrder,gOrder, 3>(Y, X, uv, T, f, l, k);
    } else {
      errorQuda("Unsupported number of colors %d\n", f->Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder csOrder>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (f->FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      if( !l ) if( l->FieldOrder() != QUDA_QDP_GAUGE_ORDER ) errorQuda("Unsupported field order for long links %d\n", l->FieldOrder());
      calculateKSY<Float,csOrder,QUDA_QDP_GAUGE_ORDER>(Y, X, uv, T, f, l, k);
    } else {
      errorQuda("Unsupported field order %d\n", f->FieldOrder());
    }
  }

 template <typename Float>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (T.Vectors().FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      calculateKSY<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(Y, X, uv, T, f, l, k);
    } else {
      errorQuda("Unsupported field order %d\n", T.Vectors().FieldOrder());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (X.Precision() != Y.Precision() || Y.Precision() != uv.Precision() ||
        Y.Precision() != T.Vectors().Precision() || Y.Precision() != f->Precision())
    {
      errorQuda("Unsupported precision mix");
      if(l != NULL) if(Y.Precision() != l->Precision()) errorQuda("Unsupported precision mix for long links.");
    }

    printfQuda("Computing Y field......\n");
    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      calculateKSY<double>(Y, X, uv, T, f, l, k);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      calculateKSY<float>(Y, X, uv, T, f, l, k);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    if(l)
     printfQuda("....done computing Y field for improved staggered operator\n");  
    else 
     printfQuda("....done computing Y field for naive staggered operator\n");
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseKSOp(const Transfer &T, GaugeField &Y, GaugeField &X, const cudaGaugeField *fat_links, const cudaGaugeField *long_links,  double k) {
    QudaPrecision precision = Y.Precision();
    //First make a cpu gauge field from the cuda gauge field

    int pad = 0;
    GaugeFieldParam fat_param(fat_links->X(), precision, fat_links->Reconstruct(), pad, fat_links->Geometry());
    fat_param.order = QUDA_QDP_GAUGE_ORDER;
    fat_param.fixed = fat_links->GaugeFixed();
    fat_param.link_type = fat_links->LinkType();
    fat_param.t_boundary = fat_links->TBoundary();
    fat_param.anisotropy = fat_links->Anisotropy();
    fat_param.gauge = NULL;
    fat_param.create = QUDA_NULL_FIELD_CREATE;
    fat_param.siteSubset = QUDA_FULL_SITE_SUBSET;

    cpuGaugeField *f = new cpuGaugeField(fat_param);
    cpuGaugeField *l = NULL;

    //Copy the cuda gauge field to the cpu
    fat_links->saveCPUField(*f, QUDA_CPU_FIELD_LOCATION);

    if(long_links)
    {
      GaugeFieldParam long_param(long_links->X(), precision, long_links->Reconstruct(), pad, long_links->Geometry());
      long_param.order = QUDA_QDP_GAUGE_ORDER;
      long_param.fixed = long_links->GaugeFixed();
      long_param.link_type = long_links->LinkType();
      long_param.t_boundary = long_links->TBoundary();
      long_param.anisotropy = long_links->Anisotropy();
      long_param.gauge = NULL;
      long_param.create = QUDA_NULL_FIELD_CREATE;
      long_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      //Copy the cuda gauge field to the cpu
      long_links->saveCPUField(*l, QUDA_CPU_FIELD_LOCATION);
    }



    //Create a field UV which holds U*V.  Has the same structure as V.
    ColorSpinorParam UVparam(T.Vectors());
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    cpuColorSpinorField uv(UVparam);

    //If the fine lattice operator is the clover operator, copy the cudaCloverField to cpuCloverField
    calculateKSY(Y, X, uv, T, f, l, k);

    delete f;
    if(long_links) delete l;

  }



  //Apply the coarse KS Dslash to a vector:
  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template<typename Float, int nDim, typename F, typename Gauge>
  void coarseKSDslash(F &out, F &in, Gauge &Y, Gauge &X, Float k) {
    const int Nc = in.Ncolor();
    int x_size[QUDA_MAX_DIM];
    for(int d = 0; d < nDim; d++) x_size[d] = in.X(d);

    //#pragma omp parallel for 
    for (int parity=0; parity<2; parity++) {
      for(int x_cb = 0; x_cb < in.Volume()/2; x_cb++) { //Volume
	int coord[QUDA_MAX_DIM];
	in.LatticeIndex(coord,parity*in.Volume()/2+x_cb);

	for(int c = 0; c < Nc; c++) out(parity, x_cb, 0, c) = (Float)0.0; 

	for(int d = 0; d < nDim; d++) { //Ndim
	  //Forward link - compute fwd offset for spinor fetch
	  int coordTmp = coord[d];
	  coord[d] = (coord[d] + 1)%x_size[d];
	  int fwd_idx = 0;
	  for(int dim = nDim-1; dim >= 0; dim--) fwd_idx = x_size[dim] * fwd_idx + coord[dim];
	  coord[d] = coordTmp;

          for(int c_row = 0; c_row < Nc; c_row++) { //Color row
  	    for(int c_col = 0; c_col < Nc; c_col++) { //Color column
	        out(parity, x_cb, 0, c_row) += Y(d, parity, x_cb, 0, 1, c_row, c_col) * in((parity+1)&1, fwd_idx/2, 1, c_col);
	        out(parity, x_cb, 1, c_row) += Y(d, parity, x_cb, 1, 0, c_row, c_col) * in((parity+1)&1, fwd_idx/2, 0, c_col);
	    } //Color column
	  } //Color row

	  //Backward link - compute back offset for spinor and gauge fetch
	  int back_idx = 0;
	  coord[d] = (coordTmp - 1 + x_size[d])%x_size[d];
	  for(int dim = nDim-1; dim >= 0; dim--) back_idx = x_size[dim] * back_idx + coord[dim];
	  coord[d] = coordTmp;

          for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	     for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		  out(parity, x_cb, 0, c_row) += - conj(Y(d,(parity+1)&1, back_idx/2, 0, 1, c_col, c_row))* in((parity+1)&1, back_idx/2, 1, c_col);//(Remark: note the minus sign.)
		  out(parity, x_cb, 1, c_row) += - conj(Y(d,(parity+1)&1, back_idx/2, 1, 0, c_col, c_row))* in((parity+1)&1, back_idx/2, 0, c_col);//(Remark: note the minus sign.)
	     } //Color column
	  } //Color row
	} //nDim

	// apply mass term
	for (int c=0; c<Nc; c++) out(parity, x_cb, 0, c) *= -(Float)2.0*k;

	// apply clover term
        for(int c = 0; c < Nc; c++) { //Color out
           for(int c_col = 0; c_col < Nc; c_col++) { //Color in
                out(parity,x_cb,0,c) += X(0, parity, x_cb, 0, 0, c, c_col)*in(parity,x_cb,0,c_col);
                out(parity,x_cb,1,c) += X(0, parity, x_cb, 1, 0, c, c_col)*in(parity,x_cb,0,c_col);
                out(parity,x_cb,0,c) += X(0, parity, x_cb, 0, 1, c, c_col)*in(parity,x_cb,1,c_col);
                out(parity,x_cb,1,c) += X(0, parity, x_cb, 1, 1, c, c_col)*in(parity,x_cb,1,c_col);
	   } //Color in
        } //Color out
      }//VolumeCB
    } // parity

    return;    
  }

  //Multiply a field by a real constant
  template<typename Float, typename F>
  void F_eq_rF(F &f, Float r) {
    for(int i = 0; i < f.Volume(); i++) {
      for(int s = 0; s < f.Nspin(); s++) {
        for(int c = 0; c < f.Ncolor(); c++) {
          f(i,s,c) *= r;
        }
      }
    }
  }

 //Apply the coarse Dslash to a vector:
  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template<typename Float, int nDim, typename F, typename Gauge>
  void coarseDslash(F &out, F &in, Gauge &Y, Gauge &X, Float kappa) {
    const int Nc = in.Ncolor();
    const int Ns = in.Nspin();
    int x_size[QUDA_MAX_DIM];
    for(int d = 0; d < nDim; d++) x_size[d] = in.X(d);

    //#pragma omp parallel for 
    for (int parity=0; parity<2; parity++) {
      for(int x_cb = 0; x_cb < in.Volume()/2; x_cb++) { //Volume
	int coord[QUDA_MAX_DIM];
	in.LatticeIndex(coord,parity*in.Volume()/2+x_cb);

	for(int s = 0; s < Ns; s++) for(int c = 0; c < Nc; c++) out(parity, x_cb, s, c) = (Float)0.0; 

	for(int d = 0; d < nDim; d++) { //Ndim
	  //Forward link - compute fwd offset for spinor fetch
	  int coordTmp = coord[d];
	  coord[d] = (coord[d] + 1)%x_size[d];
	  int fwd_idx = 0;
	  for(int dim = nDim-1; dim >= 0; dim--) fwd_idx = x_size[dim] * fwd_idx + coord[dim];
	  coord[d] = coordTmp;

	  for(int s_row = 0; s_row < Ns; s_row++) { //Spin row
	    for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	      for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
		Float sign = (s_row == s_col) ? 1.0 : -1.0;    
		for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		  out(parity, x_cb, s_row, c_row) += sign*Y(d, parity, x_cb, s_row, s_col, c_row, c_col)
		    * in((parity+1)&1, fwd_idx/2, s_col, c_col);
		} //Color column
	      } //Spin column
	    } //Color row
	  } //Spin row 

	  //Backward link - compute back offset for spinor and gauge fetch
	  int back_idx = 0;
	  coord[d] = (coordTmp - 1 + x_size[d])%x_size[d];
	  for(int dim = nDim-1; dim >= 0; dim--) back_idx = x_size[dim] * back_idx + coord[dim];
	  coord[d] = coordTmp;

	  for(int s_row = 0; s_row < Ns; s_row++) { //Spin row
	    for(int c_row = 0; c_row < Nc; c_row++) { //Color row
	      for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
		for(int c_col = 0; c_col < Nc; c_col++) { //Color column
		  out(parity, x_cb, s_row, c_row) += conj(Y(d,(parity+1)&1, back_idx/2, s_col, s_row, c_col, c_row))
		    * in((parity+1)&1, back_idx/2, s_col, c_col);
		} //Color column
	      } //Spin column
	    } //Color row
	  } //Spin row 
	} //nDim

	// apply kappa
	for (int s=0; s<Ns; s++) for (int c=0; c<Nc; c++) out(parity, x_cb, s, c) *= -(Float)2.0*kappa;

	// apply clover term
	for(int s = 0; s < Ns; s++) { //Spin out
	  for(int c = 0; c < Nc; c++) { //Color out
	    //This term is now incorporated into the matrix X.
	    //out(parity,x_cb,s,c) += in(parity,x_cb,s,c);
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin in
	      for(int c_col = 0; c_col < Nc; c_col++) { //Color in
	        //Factor of 2*kappa now incorporated in X
		//out(parity,x_cb,s,c) -= 2*kappa*X(0, parity, x_cb, s, s_col, c, c_col)*in(parity,x_cb,s_col,c_col);
                out(parity,x_cb,s,c) += X(0, parity, x_cb, s, s_col, c, c_col)*in(parity,x_cb,s_col,c_col);
	      } //Color in
	    } //Spin in
	  } //Color out
	} //Spin out

      }//VolumeCB
    } // parity
    
  }

  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseColor, int coarseSpin>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &Y, const GaugeField &X, double kappa) {
    typedef typename colorspinor::FieldOrderCB<Float,coarseSpin,coarseColor,1,csOrder> F;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> G;
    F outAccessor(const_cast<ColorSpinorField&>(out));
    F inAccessor(const_cast<ColorSpinorField&>(in));
    G yAccessor(const_cast<GaugeField&>(Y));
    G xAccessor(const_cast<GaugeField&>(X));
    if(coarseSpin  == 2)
      coarseDslash<Float,4,F,G>(outAccessor, inAccessor, yAccessor, xAccessor, (Float)kappa);
    else if(coarseSpin  == 1)
      coarseKSDslash<Float,4,F,G>(outAccessor, inAccessor, yAccessor, xAccessor, (Float)kappa);
  }

  // template on the number of coarse colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int coarseSpin>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &Y, const GaugeField &X, double kappa) {

    if (in.Ncolor() == 2) { 
      ApplyCoarse<Float,csOrder,gOrder,2,coarseSpin>(out, in, Y, X, kappa);
    } else if (in.Ncolor() == 24) { 
      ApplyCoarse<Float,csOrder,gOrder,24,coarseSpin>(out, in, Y, X, kappa);
    } else if (in.Ncolor() == 48) { 
      ApplyCoarse<Float,csOrder,gOrder,48,coarseSpin>(out, in, Y, X, kappa);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }


  // template on the number of coarse colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &Y, const GaugeField &X, double kappa) {
    if (in.Nspin() > 2) errorQuda("Unsupported number of coarse spins %d\n",in.Nspin());

    if (in.Nspin() == 2) { 
      ApplyCoarse<Float,csOrder,gOrder,2>(out, in, Y, X, kappa);
    } else if (in.Ncolor() == 1) { 
      ApplyCoarse<Float,csOrder,gOrder,1>(out, in, Y, X, kappa);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", in.Nspin());
    }
  }

  template <typename Float, QudaFieldOrder fOrder>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &Y, const GaugeField &X, double kappa) {
    if (Y.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      ApplyCoarse<Float,fOrder,QUDA_QDP_GAUGE_ORDER>(out, in, Y, X, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", Y.FieldOrder());
    }
  }

  template <typename Float>
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &Y, const GaugeField &X, double kappa) {
    if (in.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      ApplyCoarse<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(out, in, Y, X, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", in.FieldOrder());
    }
  }


  //Apply the coarse Dirac matrix to a coarse grid vector
  //out(x) = M*in = X*in - 2*kappa*\sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  //Note factor of 2*kappa compensates for the factor of 1/2 already
  //absorbed into the Y matrices.
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &Y, const GaugeField &X, double kappa) {
    if (Y.Precision() != in.Precision() || X.Precision() != Y.Precision() || Y.Precision() != out.Precision())
      errorQuda("Unsupported precision mix");

    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (out.Precision() != in.Precision() ||
	Y.Precision() != in.Precision() ||
	X.Precision() != in.Precision()) 
      errorQuda("Precision mismatch out=%d in=%d Y=%d X=%d", 
		out.Precision(), in.Precision(), Y.Precision(), X.Precision());

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyCoarse<double>(out, in, Y, X, kappa);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyCoarse<float>(out, in, Y, X, kappa);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
  }//ApplyCoarse

} //namespace quda
