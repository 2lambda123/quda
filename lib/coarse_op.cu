#include "hip/hip_runtime.h"
#include <transfer.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <clover_field_order.h>
#include <complex_quda.h>
#include <index_helper.cuh>

namespace quda {

  //A simple Euclidean gamma matrix class for use with the Wilson projectors.
  template <typename ValueType, QudaGammaBasis basis, int dir>
  class Gamma {
  private:
    const int ndim;

  protected:


    //Which gamma matrix (dir = 0,4)
    //dir = 0: gamma^1, dir = 1: gamma^2, dir = 2: gamma^3, dir = 3: gamma^4, dir =4: gamma^5
    //int dir;

    //The basis to be used.
    //QUDA_DEGRAND_ROSSI_GAMMA_BASIS is the chiral basis
    //QUDA_UKQCD_GAMMA_BASIS is the non-relativistic basis.
    //QudaGammaBasis basis;

    //The column with the non-zero element for each row
    int coupling[4];
    //The value of the matrix element, for each row
    complex<ValueType> elem[4];

  public:

    Gamma() : ndim(4) {
      complex<ValueType> I(0,1);
      if((dir==0) || (dir==1)) {
	coupling[0] = 3;
	coupling[1] = 2;
	coupling[2] = 1;
	coupling[3] = 0;
      } else if (dir == 2) {
	coupling[0] = 2;
	coupling[1] = 3;
	coupling[2] = 0;
	coupling[3] = 1;
      } else if ((dir == 3) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	coupling[0] = 2;
	coupling[1] = 3;
	coupling[2] = 0;
	coupling[3] = 1;
      } else if ((dir == 3) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	coupling[0] = 0;
	coupling[1] = 1;
	coupling[2] = 2;
	coupling[3] = 3;
      } else if ((dir == 4) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	coupling[0] = 0;
	coupling[1] = 1;
	coupling[2] = 2;
	coupling[3] = 3;
      } else if ((dir == 4) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	coupling[0] = 2;
	coupling[1] = 3;
	coupling[2] = 0;
	coupling[3] = 1;
      } else {
	printfQuda("Warning: Gamma matrix not defined for dir = %d and basis = %d\n", dir, basis);
	coupling[0] = 0;
	coupling[1] = 0;
	coupling[2] = 0;
	coupling[3] = 0;
      }


      if((dir==0)) {
	elem[0] = I;
	elem[1] = I;
	elem[2] = -I;
	elem[3] = -I;
      } else if((dir==1) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	elem[0] = -1;
	elem[1] = 1;
	elem[2] = 1;
	elem[3] = -1;
      } else if((dir==1) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = -1;
	elem[2] = -1;
	elem[3] = 1;
      } else if((dir==2)) {
	elem[0] = I;
	elem[1] = -I;
	elem[2] = -I;
	elem[3] = I;
      } else if((dir==3) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = 1;
	elem[2] = 1;
	elem[3] = 1;
      } else if((dir==3) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = 1;
	elem[2] = -1;
	elem[3] = -1;
      } else if((dir==4) && (basis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS)) {
	elem[0] = -1;
	elem[1] = -1;
	elem[2] = 1;
	elem[3] = 1;
      } else if((dir==4) && (basis == QUDA_UKQCD_GAMMA_BASIS)) {
	elem[0] = 1;
	elem[1] = 1;
	elem[2] = 1;
	elem[3] = 1;
      } else {
	elem[0] = 0;
	elem[1] = 0;
	elem[2] = 0;
	elem[3] = 0;
      }
    } 

    Gamma(const Gamma &g) : ndim(4) {
      for(int i = 0; i < ndim+1; i++) {
	coupling[i] = g.coupling[i];
	elem[i] = g.elem[i];
      }
    }

    ~Gamma() {}

    //Returns the matrix element.
    __device__ __host__ inline complex<ValueType> getelem(int row, int col) const {
      return coupling[row] == col ? elem[row] : 0;
    }

    //Like getelem, but one only needs to specify the row.
    //The column of the non-zero component is returned via the "col" reference
    __host__ __device__ inline complex<ValueType> getrowelem(int row, int &col) const {
      col = coupling[row];
      return elem[row];
    }

    //Returns the type of Gamma matrix
    inline int Dir() const {
      return dir;
    }
  };

  //Calculates the matrix UV^{s,c'}_mu(x) = \sum_c U^{c}_mu(x) * V^{s,c}_mu(x+mu)
  //Where:
  //mu = dir
  //s = fine spin
  //c' = coarse color
  //c = fine color
  template<typename Float, int dim, typename F, typename fineGauge>
  void computeUV(F &UV, const F &V, const fineGauge &G, int ndim, const int *x_size, const int *comm_dim) {

    int coord[5];
    coord[4] = 0;
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<V.VolumeCB(); x_cb++) {
	getCoords(coord, x_cb, x_size, parity);

	if ( comm_dim[dim] && (coord[dim] + 1 >= x_size[dim]) ) {
	  int nFace = 1;
	  int ghost_idx = ghostFaceIndex<1>(coord, x_size, dim, nFace);

	  for(int s = 0; s < V.Nspin(); s++) {  //Fine Spin
	    for(int ic_c = 0; ic_c < V.Nvec(); ic_c++) {  //Coarse Color
	      for(int ic = 0; ic < G.Ncolor(); ic++) { //Fine Color rows of gauge field
		for(int jc = 0; jc < G.Ncolor(); jc++) {  //Fine Color columns of gauge field
		  UV(parity, x_cb, s, ic, ic_c) += G(dim, parity, x_cb, ic, jc) * V.Ghost(dim, 1, (parity+1)&1, ghost_idx, s, jc, ic_c);
		  int y_cb = linkIndexP1(coord, x_size, dim);
		  complex<Float> ghost = V.Ghost(dim, 1, (parity+1)&1, ghost_idx, s, jc, ic_c);
		  complex<Float> bulk = V((parity+1)&1, y_cb, s, jc, ic_c);

		  if (ghost != bulk) {
		    printf("s=%d ic_c=%d ic=%d jc=%d bulk = %e %e ghost = %e %e\n",
			   s, ic_c, ic, jc, bulk.real(), bulk.imag(), ghost.real(), ghost.imag());
		  }

		}  //Fine color columns
	      }  //Fine color rows
	    }  //Coarse color
	  }  //Fine Spin

	} else {
	  int y_cb = linkIndexP1(coord, x_size, dim);

	  for(int s = 0; s < V.Nspin(); s++) {  //Fine Spin
	    for(int ic_c = 0; ic_c < V.Nvec(); ic_c++) {  //Coarse Color
	      for(int ic = 0; ic < G.Ncolor(); ic++) { //Fine Color rows of gauge field
		for(int jc = 0; jc < G.Ncolor(); jc++) {  //Fine Color columns of gauge field
		  UV(parity, x_cb, s, ic, ic_c) += G(dim, parity, x_cb, ic, jc) * V((parity+1)&1, y_cb, s, jc, ic_c);
		}  //Fine color columns
	      }  //Fine color rows
	    }  //Coarse color
	  }  //Fine Spin

	}

      } // c/b volume
    } // parity

  }  //UV

  template<typename Float, int dir, typename F, typename coarseGauge, typename fineGauge, typename Gamma>
  void computeVUV(coarseGauge &Y, coarseGauge &X, const F &UV, const F &V, 
		  const Gamma &gamma, const fineGauge &G, const int *x_size, 
		  const int *xc_size, const int *geo_bs, int spin_bs) {

    const int nDim = 4;
    const Float half = 0.5;
    int coarse_size = 1;
    for(int d = 0; d<nDim; d++) coarse_size *= xc_size[d];
    int coord[QUDA_MAX_DIM];
    int coord_coarse[QUDA_MAX_DIM];

    // paralleling this requires care with respect to race conditions
    // on CPU, parallelize over dimension not parity

    //#pragma omp parallel for 
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<UV.VolumeCB(); x_cb++) {
	getCoords(coord, x_cb, x_size, parity);
	for(int d = 0; d < nDim; d++) coord_coarse[d] = coord[d]/geo_bs[d];

	//Check to see if we are on the edge of a block, i.e.
	//if this color matrix connects adjacent blocks.  If
	//adjacent site is in same block, M = X, else M = Y
	const bool isDiagonal = ((coord[dir]+1)%x_size[dir])/geo_bs[dir] == coord_coarse[dir] ? true : false;
	coarseGauge &M =  isDiagonal ? X : Y;
	const int dim_index = isDiagonal ? 0 : dir;
	
	int coarse_parity = 0;
	for (int d=0; d<nDim; d++) coarse_parity += coord_coarse[d];
	coarse_parity &= 1;
	coord_coarse[0] /= 2;
	int coarse_x_cb = ((coord_coarse[3]*xc_size[2]+coord_coarse[2])*xc_size[1]+coord_coarse[1])*(xc_size[0]/2) + coord_coarse[0];

	coord[0] /= 2;

	for(int s = 0; s < V.Nspin(); s++) { //Loop over fine spin
	  //Spin part of the color matrix.  Will always consist
	  //of two terms - diagonal and off-diagonal part of
	  //P_mu = (1+\gamma_mu)

	  int s_c_row = s/spin_bs; //Coarse spin row index

	  //Use Gamma to calculate off-diagonal coupling and
	  //column index.  Diagonal coupling is always 1.
	  int s_col;
	  complex<Float> coupling = gamma.getrowelem(s, s_col);
	  int s_c_col = s_col/spin_bs;

	  for(int ic_c = 0; ic_c < Y.NcolorCoarse(); ic_c++) { //Coarse Color row
	    for(int jc_c = 0; jc_c < Y.NcolorCoarse(); jc_c++) { //Coarse Color column
	      for(int ic = 0; ic < G.Ncolor(); ic++) { //Sum over fine color
		//Diagonal Spin
		M(dim_index,coarse_parity,coarse_x_cb,s_c_row,s_c_row,ic_c,jc_c) +=
		  half * conj(V(parity, x_cb, s, ic, ic_c)) * UV(parity, x_cb, s, ic, jc_c);

		//Off-diagonal Spin
		M(dim_index,coarse_parity,coarse_x_cb,s_c_row,s_c_col,ic_c,jc_c) +=
		  half * coupling * conj(V(parity, x_cb, s, ic, ic_c)) * UV(parity, x_cb, s_col, ic, jc_c);
	      } //Fine color
	    } //Coarse Color column
	  } //Coarse Color row

	} //Fine spin
      } // c/b volume
    } // parity

  }


  //Adds the identity matrix to the coarse local term.
  template<typename Float, typename Gauge>
  void addCoarseDiagonal(Gauge &X) {
    const int nColor = X.NcolorCoarse();
    const int nSpin = X.NspinCoarse();
    complex<Float> *Xlocal = new complex<Float>[nSpin*nSpin*nColor*nColor];

    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<X.VolumeCB(); x_cb++) {
        for(int s = 0; s < nSpin; s++) { //Spin
         for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color
            X(0,parity,x_cb,s,s,ic_c,ic_c) += 1.0;
         } //Color
        } //Spin
      } // x_cb
    } //parity
   }


  //Adds the reverse links to the coarse local term, which is just
  //the conjugate of the existing coarse local term but with
  //plus/minus signs for off-diagonal spin components
  //Also multiply by the appropriate factor of -2*kappa
  template<typename Float, typename Gauge>
  void createCoarseLocal(Gauge &X, double kappa) {
    const int nColor = X.NcolorCoarse();
    const int nSpin = X.NspinCoarse();
    Float kap = (Float) kappa;
    complex<Float> Xlocal[nSpin*nSpin*nColor*nColor];
	
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<X.VolumeCB(); x_cb++) {

	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
	    
	    //Copy the Hermitian conjugate term to temp location 
	    for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Flip s_col, s_row on the rhs because of Hermitian conjugation.  Color part left untransposed.
		Xlocal[((nSpin*s_col+s_row)*nColor+ic_c)*nColor+jc_c] = X(0,parity,x_cb,s_row, s_col, ic_c, jc_c);
	      }	
	    }
	  }
	}
	      
	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
	    
	    const Float sign = (s_row == s_col) ? 1.0 : -1.0;
		  
	    for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Transpose color part
		X(0,parity,x_cb,s_row,s_col,ic_c,jc_c) =  
		  -2*kap*(sign*X(0,parity,x_cb,s_row,s_col,ic_c,jc_c)+conj(Xlocal[((nSpin*s_row+s_col)*nColor+jc_c)*nColor+ic_c]));
	      } //Color column
	    } //Color row
	  } //Spin column
	} //Spin row

      } // x_cb
    } //parity

  }

  //Zero out a field, using the accessor.
  template<typename Float, typename F>
  void setZero(F &f) {
    for(int parity = 0; parity < 2; parity++) {
      for(int x_cb = 0; x_cb < f.VolumeCB(); x_cb++) {
	for(int s = 0; s < f.Nspin(); s++) {
	  for(int c = 0; c < f.Ncolor(); c++) {
	    for(int v = 0; v < f.Nvec(); v++) {
	      f(parity,x_cb,s,c,v) = (Float) 0.0;
	    }
	  }
	}
      }
    }
  }

  template<typename Float, int nDim, typename coarseGauge, typename F, typename clover>
  void createCoarseClover(coarseGauge &X, F &V, clover &C, const int *x_size, const int *xc_size, const int *geo_bs, int spin_bs)  {

    int coord[QUDA_MAX_DIM];
    int coord_coarse[QUDA_MAX_DIM];
    int coarse_size = 1;
    for(int d = 0; d<nDim; d++) coarse_size *= xc_size[d];

    for (int parity=0; parity<2; parity++) {
      int x_cb = 0;
      for (coord[3]=0; coord[3]<x_size[3]; coord[3]++) {
        for (coord[2]=0; coord[2]<x_size[2]; coord[2]++) {
          for (coord[1]=0; coord[1]<x_size[1]; coord[1]++) {
            for (coord[0]=0; coord[0]<x_size[0]/2; coord[0]++) {

              int oddBit = (parity + coord[1] + coord[2] + coord[3])&1;
              coord[0] = 2*coord[0] + oddBit;
              for(int d = 0; d < nDim; d++) coord_coarse[d] = coord[d]/geo_bs[d];
              int coarse_parity = 0;
              for (int d=0; d<nDim; d++) coarse_parity += coord_coarse[d];
              coarse_parity &= 1;
              coord_coarse[0] /= 2;
              int coarse_x_cb = ((coord_coarse[3]*xc_size[2]+coord_coarse[2])*xc_size[1]+coord_coarse[1])*(xc_size[0]/2) + coord_coarse[0];

              coord[0] /= 2;

	      int s_c = 0;

              //If Nspin = 4, then the clover term has structure C_{\mu\nu} = \gamma_{\mu\nu}C^{\mu\nu}

                //printf("C.Ncolor() = %d C.NcolorCoarse() = %d\n",C.Ncolor(), C.NcolorCoarse());
                for(int s = 0; s < V.Nspin(); s++) { //Loop over fine spin row
		  s_c = s/spin_bs;
		  //On the fine lattice, the clover field is chirally blocked, so loop over rows/columns
		  //in the same chiral block.
                  for(int s_col = s_c*spin_bs; s_col < (s_c+1)*spin_bs; s_col++) { //Loop over fine spin column
                    for(int ic_c = 0; ic_c < X.NcolorCoarse(); ic_c++) { //Coarse Color row
                      for(int jc_c = 0; jc_c < X.NcolorCoarse(); jc_c++) { //Coarse Color column

                        for(int ic = 0; ic < C.Ncolor(); ic++) { //Sum over fine color row
                          for(int jc = 0; jc < C.Ncolor(); jc++) {  //Sum over fine color column
			    X(0,coarse_parity,coarse_x_cb,s_c,s_c,ic_c,jc_c) += conj(V(parity, x_cb, s, ic, ic_c)) * C(parity, x_cb, s, s_col, ic, jc) * V(parity, x_cb, s_col, jc, jc_c);
                          } //Fine color column
                        }  //Fine color row
                      } //Coarse Color column
                    } //Coarse Color row
                  }  //Fine spin column
                } //Fine spin


              x_cb++;
            } // coord[0]
          } // coord[1]
        } // coord[2]
      } // coord[3]
    } // parity

  }

  //Calculates the coarse gauge field
  template<typename Float, typename F, typename coarseGauge, typename fineGauge, typename fineClover>
  void calculateY(coarseGauge &Y, coarseGauge &X, F &UV, F &V, fineGauge &G, fineClover *C,
		  const int *xx_size, const int *xc_size, double kappa) {
    if (UV.GammaBasis() != QUDA_DEGRAND_ROSSI_GAMMA_BASIS) errorQuda("Gamma basis not supported");
    const QudaGammaBasis basis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;

    if (G.Ndim() != 4) errorQuda("Number of dimensions not supported");
    const int nDim = 4;

    int x_size[5];
    for (int i=0; i<4; i++) x_size[i] = xx_size[i];
    x_size[4] = 1;

    int comm_dim[nDim];
    for (int i=0; i<nDim; i++) comm_dim[i] = comm_dim_partitioned(i);

    int geo_bs[QUDA_MAX_DIM]; 
    for(int d = 0; d < nDim; d++) geo_bs[d] = x_size[d]/xc_size[d];
    int spin_bs = V.Nspin()/Y.NspinCoarse();

    for(int d = 0; d < nDim; d++) {
      //First calculate UV
      setZero<Float,F>(UV);

      printfQuda("Computing %d UV and VUV\n", d);
      //Calculate UV and then VUV for this direction, accumulating directly into the coarse gauge field Y
      if (d==0) {
        computeUV<Float,0>(UV, V, G, nDim, x_size, comm_dim);
        Gamma<Float, basis, 0> gamma;
        computeVUV<Float,0>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      } else if (d==1) {
        computeUV<Float,1>(UV, V, G, nDim, x_size, comm_dim);
        Gamma<Float, basis, 1> gamma;
        computeVUV<Float,1>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      } else if (d==2) {
        computeUV<Float,2>(UV, V, G, nDim, x_size, comm_dim);
        Gamma<Float, basis, 2> gamma;
        computeVUV<Float,2>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      } else {
        computeUV<Float,3>(UV, V, G, nDim, x_size, comm_dim);
        Gamma<Float, basis, 3> gamma;
        computeVUV<Float,3>(Y, X, UV, V, gamma, G, x_size, xc_size, geo_bs, spin_bs);
      }

      printfQuda("UV2[%d] = %e\n", d, UV.norm2());
      printfQuda("Y2[%d] = %e\n", d, Y.norm2(d));
    }
    printfQuda("X2 = %e\n", X.norm2(0));
    printfQuda("Computing coarse diagonal\n");
    createCoarseLocal<Float>(X, kappa);

    //If C!=NULL we have to coarsen the fine clover term and add it in.
    if (C != NULL) {
      printfQuda("Computing fine->coarse clover term\n");
      createCoarseClover<Float,nDim>(X, V, *C, x_size, xc_size, geo_bs, spin_bs);
      printfQuda("X2 = %e\n", X.norm2(0));
    }
    //Otherwise, we have a fine Wilson operator.  The "clover" term for the Wilson operator
    //is just the identity matrix.
    else {
      addCoarseDiagonal<Float>(X);
    }
    printfQuda("X2 = %e\n", X.norm2(0));
  }


  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder,
            int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
    typedef typename gauge::FieldOrder<Float,fineColor,1,gOrder> gFine;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> gCoarse;
    typedef typename clover::FieldOrder<Float,fineColor,fineSpin,clOrder> cFine;

    const ColorSpinorField &v = T.Vectors();
    int dummy = 0;
    v.exchangeGhost(QUDA_INVALID_PARITY, dummy);

    F vAccessor(const_cast<ColorSpinorField&>(v));
    F uvAccessor(const_cast<ColorSpinorField&>(uv));
    gFine gAccessor(const_cast<GaugeField&>(g));
    gCoarse yAccessor(const_cast<GaugeField&>(Y));
    gCoarse xAccessor(const_cast<GaugeField&>(X));

    if(c != NULL) {
      cFine cAccessor(const_cast<CloverField&>(*c));
      calculateY<Float>(yAccessor, xAccessor, uvAccessor, vAccessor, gAccessor, &cAccessor, g.X(), Y.X(), kappa);
    }
    else {
      cFine *cAccessor = NULL;
      calculateY<Float>(yAccessor, xAccessor, uvAccessor, vAccessor, gAccessor, cAccessor, g.X(), Y.X(), kappa);
    }    
  }

  // template on the number of coarse degrees of freedom
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder, int fineColor, int fineSpin>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (T.Vectors().Nspin()/T.Spin_bs() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 2) {
      calculateY<Float,csOrder,gOrder,clOrder,fineColor,fineSpin,2,coarseSpin>(Y, X, uv, T, g, c, kappa);
    } else if (coarseColor == 24) {
      calculateY<Float,csOrder,gOrder,clOrder,fineColor,fineSpin,24,coarseSpin>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder, int fineColor>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (uv.Nspin() == 4) {
      calculateY<Float,csOrder,gOrder,clOrder,fineColor,4>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported number of spins %d\n", uv.Nspin());
    }
  }

  // template on fine colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, QudaCloverFieldOrder clOrder>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (g.Ncolor() == 3) {
      calculateY<Float,csOrder,gOrder,clOrder,3>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported number of colors %d\n", g.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    //If c == NULL, then this is standard Wilson.  csOrder is dummy and will not matter      
    if (c==NULL || c->Order() == QUDA_PACKED_CLOVER_ORDER) {
      calculateY<Float,csOrder,gOrder,QUDA_PACKED_CLOVER_ORDER>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", c->Order());
    }
  }

  template <typename Float, QudaFieldOrder csOrder>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (g.FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      calculateY<Float,csOrder,QUDA_QDP_GAUGE_ORDER>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", g.FieldOrder());
    }
  }

 template <typename Float>
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (T.Vectors().FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      calculateY<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported field order %d\n", T.Vectors().FieldOrder());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateY(GaugeField &Y, GaugeField &X, ColorSpinorField &uv, const Transfer &T, GaugeField &g, CloverField *c, double kappa) {
    if (X.Precision() != Y.Precision() || Y.Precision() != uv.Precision() ||
        Y.Precision() != T.Vectors().Precision() || Y.Precision() != g.Precision())
      errorQuda("Unsupported precision mix");

    printfQuda("Computing Y field......\n");

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      calculateY<double>(Y, X, uv, T, g, c, kappa);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      calculateY<float>(Y, X, uv, T, g, c, kappa);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    printfQuda("....done computing Y field\n");
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseOp(const Transfer &T, GaugeField &Y, GaugeField &X, const cudaGaugeField &gauge, const cudaCloverField *clover, double kappa) {
    QudaPrecision precision = Y.Precision();
    //First make a cpu gauge field from the cuda gauge field

    int pad = 0;
    GaugeFieldParam gf_param(gauge.X(), precision, gauge.Reconstruct(), pad, gauge.Geometry());
    gf_param.order = QUDA_QDP_GAUGE_ORDER;
    gf_param.fixed = gauge.GaugeFixed();
    gf_param.link_type = gauge.LinkType();
    gf_param.t_boundary = gauge.TBoundary();
    gf_param.anisotropy = gauge.Anisotropy();
    gf_param.gauge = NULL;
    gf_param.create = QUDA_NULL_FIELD_CREATE;
    gf_param.siteSubset = QUDA_FULL_SITE_SUBSET;

    cpuGaugeField g(gf_param);

    //Copy the cuda gauge field to the cpu
    gauge.saveCPUField(g, QUDA_CPU_FIELD_LOCATION);

    //Create a field UV which holds U*V.  Has the same structure as V.
    ColorSpinorParam UVparam(T.Vectors());
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    cpuColorSpinorField uv(UVparam);

    //If the fine lattice operator is the clover operator, copy the cudaCloverField to cpuCloverField
    if(clover != NULL) {
      //Create a cpuCloverField from the cudaCloverField
      CloverFieldParam cf_param;
      cf_param.nDim = 4;
      cf_param.pad = pad;
      cf_param.precision = clover->Precision();
      for(int i = 0; i < cf_param.nDim; i++) {
        cf_param.x[i] = clover->X()[i];
      }

      cf_param.order = QUDA_PACKED_CLOVER_ORDER;
      cf_param.direct = true;
      cf_param.inverse = true;
      cf_param.clover = NULL;
      cf_param.norm = 0;
      cf_param.cloverInv = NULL;
      cf_param.invNorm = 0;
      cf_param.create = QUDA_NULL_FIELD_CREATE;
      cf_param.siteSubset = QUDA_FULL_SITE_SUBSET;

      cpuCloverField c(cf_param);
      clover->saveCPUField(c);

      calculateY(Y, X, uv, T, g, &c, kappa);
    }
    else {
      calculateY(Y, X, uv, T, g, NULL, kappa);
    }

    // now exchange Y halos for multi-process dslash
    Y.exchangeGhost();

  }

  /**** Begin staggered version of the above ****/

  template<typename Float, typename Gauge>
  void createKSCoarseLocal(Gauge &X, int ndim, const int *xc_size, double m) {
    const int nColor = X.NcolorCoarse();
    const int nSpin = X.NspinCoarse();
    if (nSpin != 2) errorQuda("\nWrong coarse spin degrees.\n");

    Float kap = (Float) m;//mass term
    complex<Float> *Xlocal = new complex<Float>[nSpin*nSpin*nColor*nColor];
	
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<X.VolumeCB(); x_cb++) {

	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
	    
	    //Copy the Hermitian conjugate term to temp location 
	    for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Flip s_col, s_row on the rhs because of Hermitian conjugation.  Color part left untransposed.
		Xlocal[((nSpin*s_col+s_row)*nColor+ic_c)*nColor+jc_c] = X(0,parity,x_cb,s_row, s_col, ic_c, jc_c);
	      }	
	    }
	  }
	}
	      
	for(int s_row = 0; s_row < nSpin; s_row++) { //Spin row
	  for(int s_col = 0; s_col < nSpin; s_col++) { //Spin column
            for(int ic_c = 0; ic_c < nColor; ic_c++) { //Color row
              //diagonal elements
	      if(s_row == s_col){
                X(0,parity,x_cb, parity, parity, ic_c,ic_c) += (parity == 0) ? +m : -m;//dioganal mass term
                continue;
              }
              //off-diagonal elements
	      for(int jc_c = 0; jc_c < nColor; jc_c++) { //Color column
		//Transpose color part
		X(0,parity,x_cb,s_row,s_col,ic_c,jc_c) = (+X(0,parity,x_cb,s_row,s_col,ic_c,jc_c)-conj(Xlocal[((nSpin*s_row+s_col)*nColor+jc_c)*nColor+ic_c]));//always minus sign?
	      } //Color column
	    } //Color row
	  } //Spin column
	} //Spin row

      } // x_cb
    } //parity

    delete[] Xlocal;

    return;
  }

  //added HISQ links (single GPU support)
  template<typename Float, int dir, typename F, typename fineGauge>
  void computeKSUV(F *UV, F *UVL, const F &V, const fineGauge *FL, const fineGauge *LL, int ndim, const int *x_size) 
  {
    int coord[QUDA_MAX_DIM] = {0};

    const int stag_sp = 0;
     
    for (int parity=0; parity<2; parity++) {
      for( int x_cb = 0; x_cb < V.VolumeCB(); x_cb++){
         getCoords(coord, x_cb, x_size, parity);

         int y_cb  = linkIndexP1(coord, x_size, dir);
         int y3_cb = (LL != NULL) ? linkIndexP3(coord, x_size, dir) : 0;

	 for(int ic_c = 0; ic_c < V.Nvec(); ic_c++) {  //Coarse Color
             for(int ic = 0; ic < FL->Ncolor(); ic++) { //Fine Color rows of gauge field
		 for(int jc = 0; jc < FL->Ncolor(); jc++) {  //Fine Color columns of gauge field
		    (*UV)(parity, x_cb, stag_sp, ic, ic_c) += (*FL)(dir, parity, x_cb, ic, jc) * V((parity+1)&1, y_cb, stag_sp, jc, ic_c);//mind transformation to the opposite parity field: in UVU operation.
                    if(LL != NULL) (*UVL)(parity, x_cb, stag_sp, ic, ic_c) += (*LL)(dir, parity, x_cb, ic, jc) * V((parity+1)&1, y3_cb, stag_sp, jc, ic_c);
		 }  //Fine color columns
	      }  //Fine color rows
	  }
       }// x_cb
    } // parity

    return;
  }  //UV

  //KS (also HISQ) operator:
  template<typename Float, int dir, typename F, typename coarseGauge>
  void computeKSVUV(coarseGauge &Y, coarseGauge &X, const F *UV, const F *UVL, const F &V, const int nfinecolors,
		  const int *x_size, const int *xc_size, const int *geo_bs) {

    const int nDim = 4;
    Float half = -0.5;
    int coarse_size = 1;

    for(int d = 0; d<nDim; d++) coarse_size *= xc_size[d];

    int coord[QUDA_MAX_DIM];
    int coord_coarse[QUDA_MAX_DIM];

    // paralleling this requires care with respect to race conditions
    // on CPU, parallelize over dimension not parity
    Float eta = dir == 0 ? 1.0 : -1.0;

    const int stag_sp = 0;

    //#pragma omp parallel for 
    for (int parity=0; parity<2; parity++) {
      for( int x_cb = 0; x_cb < UV->VolumeCB(); x_cb++){
         getCoords(coord, x_cb, x_size, parity);

         for(int d = 0; d < nDim; d++) coord_coarse[d] = coord[d]/geo_bs[d];

	 //Check to see if we are on the edge of a block, i.e.
	 //if this color matrix connects adjacent blocks.  If
	 //adjacent site is in same block, M = X, else M = Y
	 bool isDiagonal = (((coord[dir]+1)%x_size[dir])/geo_bs[dir] == coord_coarse[dir]) ? true : false;
         //
	 bool isDiagonal_long = (UVL == NULL) ? false : (((coord[dir]+3)%x_size[dir])/geo_bs[dir] == coord_coarse[dir]) ? true : false;

	 coarseGauge *M =  isDiagonal ? &X : &Y;
         coarseGauge *M_L = (UVL == NULL) ? NULL : (isDiagonal_long ? &X : &Y);
	      
         const int dim_index      = isDiagonal ? 0 : dir;
         const int dim_index_long = isDiagonal_long ? 0 : dir;

	 int coarse_parity = 0;
	 for (int d=0; d<nDim; d++) coarse_parity += coord_coarse[d];
	 coarse_parity &= 1;
	 coord_coarse[0] /= 2;
	 int coarse_x_cb = ((coord_coarse[3]*xc_size[2]+coord_coarse[2])*xc_size[1]+coord_coarse[1])*(xc_size[0]/2) + coord_coarse[0];
	      
	 //printf("(%d,%d)\n", coarse_x_cb, coarse_parity);
	 coord[0] /= 2;

         int coarse_spin_row = parity == 0 ? 0 : 1  ;
         int coarse_spin_col = (1 - coarse_spin_row); 

         half *= eta; //multiply by sing factor 

         for(int ic_c = 0; ic_c < Y.NcolorCoarse(); ic_c++) { //Coarse Color row
           for(int jc_c = 0; jc_c < Y.NcolorCoarse(); jc_c++) { //Coarse Color column
	     for(int ic = 0; ic < nfinecolors; ic++) { //Sum over fine color
		(*M)(dim_index,coarse_parity,coarse_x_cb,coarse_spin_row, coarse_spin_col,ic_c,jc_c) += half*conj(V(parity, x_cb, stag_sp, ic, ic_c)) * (*UV)(parity, x_cb, stag_sp, ic, jc_c);
                 if(UVL != NULL) (*M_L)(dim_index_long,coarse_parity,coarse_x_cb,coarse_spin_row, coarse_spin_col,ic_c,jc_c) += half*conj(V(parity, x_cb, stag_sp, ic, ic_c)) * (*UVL)(parity, x_cb, stag_sp, ic, jc_c);
	     } //Fine color
	   } //Coarse Color column
	 } //Coarse Color row
      } // x_cb
    } // parity
    
    return;
  }

 //Calculates the coarse gauge field: separated from coarseSpin = 2 computations:
  template<typename Float, typename F, typename coarseGauge, typename fineGauge>
  void calculateKSY(coarseGauge &Y, coarseGauge &X, F *UV, F *UVL, F &V, fineGauge *FL, fineGauge *LL, const int *x_size, const int *xc_size,  double k) {

    if (FL->Ndim() != 4) errorQuda("Number of dimensions not supported");

    if ( LL ) if(LL->Ndim() != 4) errorQuda("Number of long links dimensions not supported");

    const int nDim = 4;

    int geo_bs[QUDA_MAX_DIM]; 
    for(int d = 0; d < nDim; d++) geo_bs[d] = x_size[d]/xc_size[d];

    for(int d = 0; d < nDim; d++) 
    {
      //First calculate UV
      setZero<Float,F>(*UV);

      printfQuda("Computing %d UV and VUV\n", d);
      //Calculate UV and then VUV for this direction, accumulating directly into the coarse gauge field Y
      if (d==0) {
        computeKSUV<Float,0>(UV, UVL, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,0>(Y, X, UV, UVL, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      } else if (d==1) {
        computeKSUV<Float,1>(UV, UVL, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,1>(Y, X, UV, UVL, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      } else if (d==2) {
        computeKSUV<Float,2>(UV, UVL, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,2>(Y, X, UV, UVL, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      } else {
        computeKSUV<Float,3>(UV, UVL, V, FL, LL, nDim, x_size);
        computeKSVUV<Float,3>(Y, X, UV, UVL, V, FL->Ncolor(), x_size, xc_size, geo_bs);
      }

      printf("UV2[%d] = %e\n", d, UV->norm2());
      printf("Y2[%d] = %e\n", d, Y.norm2(d));
    }

    printf("X2 = %e\n", X.norm2(0));
    printfQuda("Computing coarse diagonal\n");
    createKSCoarseLocal<Float>(X, nDim, xc_size, k);

    printf("X2 = %e\n", X.norm2(0));

  }



  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int fineColor, int coarseColor, int coarseSpin>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField *uv, ColorSpinorField *uv_long, const Transfer &T, GaugeField *f, GaugeField *l, double k) {

    const int fineSpin = 1;

    typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
    typedef typename gauge::FieldOrder<Float,fineColor,1,gOrder> gFine;
    typedef typename gauge::FieldOrder<Float,coarseSpin*coarseColor,coarseSpin,gOrder> gCoarse;

    const ColorSpinorField &v = T.Vectors();
    int dummy = 0;
    v.exchangeGhost(QUDA_INVALID_PARITY, dummy);

    F vAccessor(const_cast<ColorSpinorField&>(v));
    F uvAccessor(const_cast<ColorSpinorField&>(*uv));
    gFine fAccessor(const_cast<GaugeField&>(*f));
    gCoarse yAccessor(const_cast<GaugeField&>(Y));
    gCoarse xAccessor(const_cast<GaugeField&>(X));

    if(l != NULL) {
      gFine lAccessor(const_cast<GaugeField&>(*l));
      F uvlAccessor(const_cast<ColorSpinorField&>(*uv_long));
      calculateKSY<Float>(yAccessor, xAccessor, &uvAccessor, &uvlAccessor, vAccessor, &fAccessor, &lAccessor, f->X(), Y.X(), k);
    }
    else {
      gFine *lAccessor = NULL;
      F *uvlAccessor = NULL;
      calculateKSY<Float>(yAccessor, xAccessor, &uvAccessor, uvlAccessor, vAccessor, &fAccessor, lAccessor, f->X(),Y.X(), k);
    }    
  }

  // template on the number of coarse degrees of freedom
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder, int fineColor>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField *uv, ColorSpinorField *uv_long, const Transfer &T, GaugeField *f, GaugeField *l, double k) {

    if ((T.Vectors().Nspin() != 1) && (T.Vectors().Nspin()/T.Spin_bs() != 2))  errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 2) {
      calculateKSY<Float,csOrder,gOrder,fineColor,2, coarseSpin>(Y, X, uv, uv_long, T, f, l, k);
    } else if (coarseColor == 24) {
      calculateKSY<Float,csOrder,gOrder,fineColor,24, coarseSpin>(Y, X, uv, uv_long, T, f, l, k);
    } else if (coarseColor == 48) {
      calculateKSY<Float,csOrder,gOrder,fineColor,48, coarseSpin>(Y, X, uv, uv_long, T, f, l, k);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }


  // template on fine colors
  template <typename Float, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField *uv, ColorSpinorField *uv_long, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (f->Ncolor() == 3) {
      if( l ) if( f->Ncolor() != l->Ncolor() ) errorQuda("Unsupported number of colors %d\n", l->Ncolor());
      calculateKSY<Float,csOrder,gOrder, 3>(Y, X, uv, uv_long, T, f, l, k);
    } else {
      errorQuda("Unsupported number of colors %d\n", f->Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder csOrder>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField *uv, ColorSpinorField *uv_long, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (f->FieldOrder() == QUDA_QDP_GAUGE_ORDER) {
      if( l ) if( l->FieldOrder() != QUDA_QDP_GAUGE_ORDER ) errorQuda("Unsupported field order for long links %d\n", l->FieldOrder());
      calculateKSY<Float,csOrder,QUDA_QDP_GAUGE_ORDER>(Y, X, uv, uv_long, T, f, l, k);
    } else {
      errorQuda("Unsupported field order %d\n", f->FieldOrder());
    }
  }

 template <typename Float>
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField *uv, ColorSpinorField *uv_long, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (T.Vectors().FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      calculateKSY<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(Y, X, uv, uv_long, T, f, l, k);
    } else {
      errorQuda("Unsupported field order %d\n", T.Vectors().FieldOrder());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateKSY(GaugeField &Y, GaugeField &X, ColorSpinorField *uv, ColorSpinorField *uv_long, const Transfer &T, GaugeField *f, GaugeField *l, double k) {
    if (X.Precision() != Y.Precision() || Y.Precision() != uv->Precision() ||
        Y.Precision() != T.Vectors().Precision() || Y.Precision() != f->Precision())
    {
      errorQuda("Unsupported precision mix");
    }

    if( l )
    { 
      if(Y.Precision() != l->Precision() || Y.Precision() != uv_long->Precision()) errorQuda("Unsupported precision mix for long links.");
    }

    printfQuda("Computing Y field......\n");
    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
      calculateKSY<double>(Y, X, uv, uv_long, T, f, l, k);
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      calculateKSY<float>(Y, X, uv, uv_long, T, f, l, k);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    if(l)
     printfQuda("....done computing Y field for improved staggered operator\n");  
    else 
     printfQuda("....done computing Y field for naive staggered operator\n");
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseKSOp(const Transfer &T, GaugeField &Y, GaugeField &X, const cudaGaugeField *fat_links, const cudaGaugeField *long_links,  double k) {
    QudaPrecision precision = Y.Precision();
    //First make a cpu gauge field from the cuda gauge field

    int pad = 0;
    GaugeFieldParam fat_param(fat_links->X(), precision, fat_links->Reconstruct(), pad, fat_links->Geometry());
    fat_param.order = QUDA_QDP_GAUGE_ORDER;
    fat_param.fixed = fat_links->GaugeFixed();
    fat_param.link_type = fat_links->LinkType();
    fat_param.t_boundary = fat_links->TBoundary();
    fat_param.anisotropy = fat_links->Anisotropy();
    fat_param.gauge = NULL;
    fat_param.create = QUDA_NULL_FIELD_CREATE;
    fat_param.siteSubset = QUDA_FULL_SITE_SUBSET;

    cpuGaugeField *f = new cpuGaugeField(fat_param);
    cpuGaugeField *l = NULL;

    //Copy the cuda gauge field to the cpu
    fat_links->saveCPUField(*f, QUDA_CPU_FIELD_LOCATION);

    //Create a field UV which holds U*V.  Has the same structure as V.
    ColorSpinorParam UVparam(T.Vectors());
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    cpuColorSpinorField *uv = new cpuColorSpinorField(UVparam);

    cpuColorSpinorField *uv_long = NULL;

    if(long_links)
    {
      GaugeFieldParam long_param(long_links->X(), precision, long_links->Reconstruct(), pad, long_links->Geometry());
      long_param.order = QUDA_QDP_GAUGE_ORDER;
      long_param.fixed = long_links->GaugeFixed();
      long_param.link_type = long_links->LinkType();
      long_param.t_boundary = long_links->TBoundary();
      long_param.anisotropy = long_links->Anisotropy();
      long_param.gauge = NULL;
      long_param.create = QUDA_NULL_FIELD_CREATE;
      long_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      //
      l = new cpuGaugeField(fat_param);

      cpuColorSpinorField *uv_long = new cpuColorSpinorField(UVparam);
      //Copy the cuda gauge field to the cpu
      long_links->saveCPUField(*l, QUDA_CPU_FIELD_LOCATION);
    }

    //If the fine lattice operator is the clover operator, copy the cudaCloverField to cpuCloverField
    calculateKSY(Y, X, uv, uv_long, T, f, l, k);

    // now exchange Y halos for multi-process dslash
    Y.exchangeGhost();

    delete uv; 

    delete f;

    if(l)
    { 
      delete l;
      delete uv_long;
    }

  }

/**** End staggered version  ****/

} //namespace quda
