#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <su3_project.cuh>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <index_helper.cuh>

#define  DOUBLE_TOL	1e-15
#define  SINGLE_TOL	2e-6

namespace quda {

#ifdef GPU_GAUGE_TOOLS

  template <typename Float, typename GaugeOr, typename GaugeDs>
  struct GaugeSTOUTArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4]; 
#endif
    GaugeOr origin;
    const Float rho;
    const Float tolerance;
    
    GaugeDs dest;

    GaugeSTOUTArg(GaugeOr &origin, GaugeDs &dest, const GaugeField &data, const Float rho, const Float tolerance) 
      : origin(origin), dest(dest), rho(rho), tolerance(tolerance) {
#ifdef MULTI_GPU
      for ( int dir = 0; dir < 4; ++dir ) {
        border[dir] = data.R()[dir];
        X[dir] = data.X()[dir] - border[dir] * 2;
      } 
#else
        for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir];
#endif
	threads = X[0]*X[1]*X[2]*X[3];
    }
  };


  template <typename Float, typename GaugeOr, typename GaugeDs, typename Float2>
  __host__ __device__ void computeStaple(GaugeSTOUTArg<Float,GaugeOr,GaugeDs>& arg, int idx, int parity, int dir, Matrix<Float2,3> &staple) {

    typedef typename ComplexTypeId<Float>::Type Cmplx;
      // compute spacetime dimensions and parity

    int X[4]; 
    for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

    int x[4];
    getCoords(x, idx, X, parity);
#ifdef MULTI_GPU
    for(int dr=0; dr<4; ++dr) {
         x[dr] += arg.border[dr];
         X[dr] += 2*arg.border[dr];
    }
#endif

    setZero(&staple);

    for (int mu=0; mu<3; mu++) {  // I believe most users won't want to include time staples in smearing
      if (mu == dir) {
        continue;
      }

      int nu = dir;

      {
        int dx[4] = {0, 0, 0, 0};
        Matrix<Cmplx,3> U1;
        arg.origin.load((Float*)(U1.data),linkIndexShift(x,dx,X), mu, parity); 

        Matrix<Cmplx,3> U2;
        dx[mu]++;
        arg.origin.load((Float*)(U2.data),linkIndexShift(x,dx,X), nu, 1-parity); 

        Matrix<Cmplx,3> U3;
        dx[mu]--;
        dx[nu]++;
        arg.origin.load((Float*)(U3.data),linkIndexShift(x,dx,X), mu, 1-parity); 
   
        Matrix<Cmplx,3> tmpS;

        tmpS	= U1 * U2;
	tmpS	= tmpS * conj(U3);

	staple = staple + tmpS;

        dx[mu]--;
        dx[nu]--;
        arg.origin.load((Float*)(U1.data),linkIndexShift(x,dx,X), mu, 1-parity); 
        arg.origin.load((Float*)(U2.data),linkIndexShift(x,dx,X), nu, 1-parity); 

        dx[nu]++;
        arg.origin.load((Float*)(U3.data),linkIndexShift(x,dx,X), mu, parity); 

        tmpS	= conj(U1);
	tmpS	= tmpS * U2;
	tmpS	= tmpS * U3;

	staple = staple + tmpS;
      }
    }
  }

  template<typename Float, typename GaugeOr, typename GaugeDs>
    __global__ void computeSTOUTStep(GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg){
      int idx = threadIdx.x + blockIdx.x*blockDim.x;
      if(idx >= arg.threads) return;
      typedef typename ComplexTypeId<Float>::Type Cmplx;

      int parity = 0;
      if(idx >= arg.threads/2) {
        parity = 1;
        idx -= arg.threads/2;
      }

      int X[4]; 
      for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
#ifdef MULTI_GPU
      for(int dr=0; dr<4; ++dr) {
           x[dr] += arg.border[dr];
           X[dr] += 2*arg.border[dr];
      }
#endif

      int dx[4] = {0, 0, 0, 0};
      for (int dir=0; dir < 3; dir++) {				//Only spatial dimensions are smeared
        Matrix<Cmplx,3> U, UDag, Stap, Omega, OmegaDag, OmegaDiff, ODT, Q,
	  exp_iQ, tmp1, tmp2;
	Cmplx OmegaDiffTr;
	Cmplx i_2 = makeComplex<Cmplx>(0,0.5);

	//This function gets stap = S_{mu,nu} i.e., the staple of length 3,
        computeStaple<Float,GaugeOr,GaugeDs,Cmplx>(arg,idx,parity,dir,Stap);
	//
	// |- > -|
	// ^     v
	// |     |
	//          +  |     |
	//             v     ^
	//             |- > -|

	// Get link U
        arg.origin.load((Float*)(U.data),linkIndexShift(x,dx,X),dir,parity);

	//Compute Omega_{mu}=[Sum_{mu neq nu}rho_{mu,nu}C_{mu,nu}]*U_{mu}^dag

	//Get U^{\dagger}
	computeMatrixInverse(U,&UDag);
	
	//Compute \Omega = \rho * S * U^{\dagger}
	tmp1 = arg.rho * Stap;
	Omega = tmp1 * UDag;

	//Compute \Q_{mu} = i/2[Omega_{mu}^dag - Omega_{mu} 
	//                      - 1/3 Tr(Omega_{mu}^dag - Omega_{mu})]

	OmegaDag = conj(Omega);
	OmegaDiff = OmegaDag - Omega;

	Q = OmegaDiff;
	OmegaDiffTr = getTrace(OmegaDiff);
	OmegaDiffTr =  1.0/3.0 * OmegaDiffTr;

	//Matrix proportional to OmegaDiffTr
	setIdentity(&ODT);
	tmp1 = OmegaDiffTr * ODT;

	Q = Q - tmp1;
	Q = i_2 * Q;
	//Q is now defined.

	exponentiate_iQ(Q,&exp_iQ);
	U = exp_iQ * U;

	//No need to project back down to SU(3)
        //polarSu3<Cmplx,Float>(&U, arg.tolerance);
        arg.dest.save((Float*)(U.data),linkIndexShift(x,dx,X), dir, parity); 
    }
  }

  template<typename Float, typename GaugeOr, typename GaugeDs>
    class GaugeSTOUT : Tunable {
      GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg;
      const QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool tuneSharedBytes() const { return false; } // Don't tune shared memory
      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.threads; }

      public:
      GaugeSTOUT(GaugeSTOUTArg<Float,GaugeOr, GaugeDs> &arg, QudaFieldLocation location)
        : arg(arg), location(location) {}
      virtual ~GaugeSTOUT () {}

      void apply(const hipStream_t &stream){
        if (location == QUDA_CUDA_FIELD_LOCATION) {
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
          computeSTOUTStep<<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
        } else {
          errorQuda("CPU not supported yet\n");
          //computeSTOUTStepCPU(arg);
        }
      }

      TuneKey tuneKey() const {
        std::stringstream vol, aux;
        vol << arg.X[0] << "x";
        vol << arg.X[1] << "x";
        vol << arg.X[2] << "x";
        vol << arg.X[3];
        aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
        return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
      }


      std::string paramString(const TuneParam &param) const {
        std::stringstream ps;
        ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
        ps << "shared=" << param.shared_bytes;
        return ps.str();
      }

      void preTune(){}
      void postTune(){}
      long long flops() const { return (1)*6*arg.threads; }
      long long bytes() const { return (1)*6*arg.threads*sizeof(Float); } // Only correct if there is no link reconstruction

    }; // GaugeSTOUT

  template<typename Float,typename GaugeOr, typename GaugeDs>
    void STOUTStep(GaugeOr origin, GaugeDs dest, const GaugeField& dataOr, Float rho, QudaFieldLocation location) {
      if (dataOr.Precision() == QUDA_DOUBLE_PRECISION) {
        GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg(origin, dest, dataOr, rho, DOUBLE_TOL);
        GaugeSTOUT<Float,GaugeOr,GaugeDs> gaugeSTOUT(arg, location);
        gaugeSTOUT.apply(0);
      } else {
        GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg(origin, dest, dataOr, rho, SINGLE_TOL);
        GaugeSTOUT<Float,GaugeOr,GaugeDs> gaugeSTOUT(arg, location);
        gaugeSTOUT.apply(0);
      }
      hipDeviceSynchronize();
    }

  template<typename Float>
    void STOUTStep(GaugeField &dataDs, const GaugeField& dataOr, Float rho, QudaFieldLocation location) {

    if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GDs;

      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
      }
    } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GDs;
      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
      }
    } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GDs;
      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	STOUTStep(GOr(dataOr), GDs(dataDs), dataOr, rho, location);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
    } else {
      errorQuda("Reconstruction type %d of destination gauge field not supported", dataDs.Reconstruct());
    }

  }

#endif

  void STOUTStep(GaugeField &dataDs, const GaugeField& dataOr, double rho, QudaFieldLocation location) {

#ifdef GPU_GAUGE_TOOLS

    if(dataOr.Precision() != dataDs.Precision()) {
      errorQuda("Oriign and destination fields must have the same precision\n");
    }

    if(dataDs.Precision() == QUDA_HALF_PRECISION){
      errorQuda("Half precision not supported\n");
    }

    if (!dataOr.isNative())
      errorQuda("Order %d with %d reconstruct not supported", dataOr.Order(), dataOr.Reconstruct());

    if (!dataDs.isNative())
      errorQuda("Order %d with %d reconstruct not supported", dataDs.Order(), dataDs.Reconstruct());

    if (dataDs.Precision() == QUDA_SINGLE_PRECISION){
      STOUTStep<float>(dataDs, dataOr, (float) rho, location);
    } else if(dataDs.Precision() == QUDA_DOUBLE_PRECISION) {
      STOUTStep<double>(dataDs, dataOr, rho, location);
    } else {
      errorQuda("Precision %d not supported", dataDs.Precision());
    }
    return;
#else
  errorQuda("Gauge tools are not build");
#endif
  }

}
