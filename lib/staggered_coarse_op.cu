#include "hip/hip_runtime.h"
#include <memory>

#include <transfer.h>
#include <gauge_field.h>
#include <tunable_nd.h>

// For naive Kahler-Dirac coarsening
#include <kernels/staggered_coarse_op_kernel.cuh>

// This define controls which kernels get compiled in `coarse_op.cuh`.
// This ensures only kernels relevant for coarsening the staggered
// operator get built, saving compile time.
#define STAGGEREDCOARSE
#include <coarse_op.cuh>

namespace quda {

  template <typename Float, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  class CalculateStaggeredY : public TunableKernel3D {

    Arg &arg;
    const GaugeField &meta;
    GaugeField &Y;
    GaugeField &X;

    long long flops() const { return arg.coarseVolumeCB*coarseSpin*coarseColor; }

    long long bytes() const
    {
      // 2 from forwards / backwards contributions, Y and X are sparse - only needs to write non-zero elements, 2nd term is mass term
      return meta.Bytes() + (2 * meta.Bytes() * Y.Precision()) / meta.Precision() + 2 * 2 * coarseSpin * coarseColor * arg.coarseVolumeCB * X.Precision();
    }

    unsigned int minThreads() const { return arg.fineVolumeCB; }
    bool tuneSharedBytes() const { return false; } // don't tune the grid dimension

  public:
    CalculateStaggeredY(Arg &arg, const GaugeField &meta, GaugeField &Y, GaugeField &X) :
      TunableKernel3D(meta, fineColor*fineColor, 2),
      arg(arg),
      meta(meta),
      Y(Y),
      X(X)
    {
      strcat(aux,comm_dim_partitioned_string());
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) strcat(aux, getOmpThreadStr());
      strcat(aux,",computeStaggeredVUV");
      strcat(aux, (meta.Location()==QUDA_CUDA_FIELD_LOCATION && Y.MemType() == QUDA_MEMORY_MAPPED) ? ",GPU-mapped," :
             meta.Location()==QUDA_CUDA_FIELD_LOCATION ? ",GPU-device," : ",CPU,");
      strcat(aux,"coarse_vol=");
      strcat(aux,X.VolString());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
        launch_host<ComputeStaggeredVUV>(tp, stream, arg);
      } else {
        launch_device<ComputeStaggeredVUV>(tp, stream, arg);
      }
    }

    bool advanceTuneParam(TuneParam &param) const {
      // only do autotuning if we have device fields
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION && Y.MemType() == QUDA_MEMORY_DEVICE) return Tunable::advanceTuneParam(param);
      else return false;
    }
  };

  /**
     @brief Calculate the coarse-link field, including the coarse clover field.

     @param Y[out] Coarse (fat-)link field accessor
     @param X[out] Coarse clover field accessor
     @param G[in] Fine grid link / gauge field accessor
     @param Y_[out] Coarse link field
     @param X_[out] Coarse clover field
     @param X_[out] Coarse clover inverese field (used as temporary here)
     @param v[in] Packed null-space vectors
     @param G_[in] Fine gauge field
     @param mass[in] Kappa parameter
     @param matpc[in] The type of preconditioning of the source fine-grid operator
   */
  template<typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor,
	   typename coarseGauge, typename fineGauge>
  void calculateStaggeredY(coarseGauge &Y, coarseGauge &X, fineGauge &G, GaugeField &Y_, GaugeField &X_,
                           const GaugeField &G_, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    // sanity checks
    if (matpc == QUDA_MATPC_EVEN_EVEN_ASYMMETRIC || matpc == QUDA_MATPC_ODD_ODD_ASYMMETRIC)
      errorQuda("Unsupported coarsening of matpc = %d", matpc);

    // This is the last time we use fineSpin, since this file only coarsens
    // staggered-type ops, not wilson-type AND coarse-type.
    if (fineSpin != 1)
      errorQuda("Input Dirac operator %d should have nSpin=1, not nSpin=%d\n", dirac, fineSpin);
    if (fineColor != 3)
      errorQuda("Input Dirac operator %d should have nColor=3, not nColor=%d\n", dirac, fineColor);

    if (G.Ndim() != 4) errorQuda("Number of dimensions not supported");
    const int nDim = 4;

    int x_size[QUDA_MAX_DIM] = { };
    for (int i=0; i<4; i++) x_size[i] = G_.X()[i];
    x_size[4] = 1;

    int xc_size[QUDA_MAX_DIM] = { };
    for (int i=0; i<4; i++) xc_size[i] = X_.X()[i];
    xc_size[4] = 1;

    int geo_bs[QUDA_MAX_DIM] = { };
    for(int d = 0; d < nDim; d++) geo_bs[d] = x_size[d]/xc_size[d];

    // Calculate VUV in one pass (due to KD-transform) for each dimension,
    // accumulating directly into the coarse gauge field Y

    using Arg = CalculateStaggeredYArg<Float,coarseSpin,fineColor,coarseColor,coarseGauge,fineGauge>;
    Arg arg(Y, X, G, mass, x_size, xc_size, geo_bs);
    CalculateStaggeredY<Float, fineColor, coarseSpin, coarseColor, Arg> y(arg, G_, Y_, X_);

    QudaFieldLocation location = checkLocation(Y_, X_, G_);
    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("Running link coarsening on the %s\n", location == QUDA_CUDA_FIELD_LOCATION ? "GPU" : "CPU");

    // We know exactly what the scale should be: the max of all of the (fat) links.
    double max_scale = G_.abs_max();
    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("Global U_max = %e\n", max_scale);

    if (coarseGauge::fixedPoint()) {
      arg.Y.resetScale(max_scale);
      arg.X.resetScale(max_scale > 2.0*mass ? max_scale : 2.0*mass); // To be safe
      Y_.Scale(max_scale);
      X_.Scale(max_scale > 2.0*mass ? max_scale : 2.0*mass); // To be safe
    }

    // We can technically do a uni-directional build, but becauase
    // the coarse link builds are just permutations plus lots of zeros,
    // it's faster to skip the flip!

    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("Computing VUV\n");
    y.apply(device::get_default_stream());

    if (getVerbosity() >= QUDA_VERBOSE) {
      for (int d = 0; d < nDim; d++) printfQuda("Y2[%d] = %e\n", 4+d, Y_.norm2( 4+d ));
      for (int d = 0; d < nDim; d++) printfQuda("Y2[%d] = %e\n", d, Y_.norm2( d ));
    }

    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("X2 = %e\n", X_.norm2(0));
  }

  template <typename Float, typename vFloat, int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g,
                           double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    QudaFieldLocation location = Y.Location();

    if (location == QUDA_CPU_FIELD_LOCATION) {

      constexpr QudaFieldOrder csOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_QDP_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using gFine = typename gauge::FieldOrder<Float,fineColor,1,gOrder>;
      using gCoarse = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat>;

      gFine gAccessor(const_cast<GaugeField&>(g));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));

      calculateStaggeredY<Float,fineSpin,fineColor,coarseSpin,coarseColor>
        (yAccessor, xAccessor, gAccessor, Y, X, g, mass, dirac, matpc);
    } else {

      constexpr QudaFieldOrder csOrder = QUDA_FLOAT2_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_FLOAT2_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using gFine = typename gauge::FieldOrder<Float,fineColor,1,gOrder,true,Float>;
      using gCoarse = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat>;

      gFine gAccessor(const_cast<GaugeField&>(g));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));

      calculateStaggeredY<Float,fineSpin,fineColor,coarseSpin,coarseColor>
        (yAccessor, xAccessor, gAccessor, Y, X, g, mass, dirac, matpc);
    }

  }

  template <typename Float, typename vFloat, int fineColor, int fineSpin, int coarseColor, int coarseSpin, int uvSpin>
  void aggregateStaggeredY(GaugeField &Y, GaugeField &X,
                        const Transfer &T, const GaugeField &g, const GaugeField &l, const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    // Actually create the temporaries like UV, etc.
    auto location = Y.Location();

    // Create a field UV which holds U*V.  Has roughly the same structure as V,
    // though we need to double the spin for the KD operator to keep track of from even vs from odd.
    ColorSpinorParam UVparam(T.Vectors(location));
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    UVparam.location = location;
    UVparam.nSpin = uvSpin;
    UVparam.setPrecision(T.Vectors(location).Precision());
    UVparam.mem_type = Y.MemType(); // allocate temporaries to match coarse-grid link field

    ColorSpinorField *uv = ColorSpinorField::Create(UVparam);

    ColorSpinorField *av = (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) ? ColorSpinorField::Create(UVparam) : &const_cast<ColorSpinorField&>(T.Vectors(location));

    GaugeField *Yatomic = &Y;
    GaugeField *Xatomic = &X;

    if (Y.Precision() < QUDA_SINGLE_PRECISION) {
      // we need to coarsen into single precision fields (float or int), so we allocate temporaries for this purpose
      // else we can just coarsen directly into the original fields
      GaugeFieldParam param(X); // use X since we want scalar geometry
      param.location = location;
      param.setPrecision(QUDA_SINGLE_PRECISION, location == QUDA_CUDA_FIELD_LOCATION ? true : false);

      Yatomic = GaugeField::Create(param);
      Xatomic = GaugeField::Create(param);
    }

    // Moving along to the build

    const double kappa = -1.; // cancels a minus sign factor for kappa w/in the dslash application
    const double mu_dummy = 0.; 
    const double mu_factor_dummy = 0.;
    constexpr bool use_mma = false;
    
    bool need_bidirectional = false;
    if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) need_bidirectional = true;

    // need to exchange with depth 3 b/c of long links
    const int nFace = (dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADPC_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) ? 3 : 1;

    if (Y.Location() == QUDA_CPU_FIELD_LOCATION) {

      constexpr QudaFieldOrder csOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_QDP_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using V = typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder,vFloat>;
      using F = typename colorspinor::FieldOrderCB<Float,uvSpin,fineColor,coarseColor,csOrder,vFloat>;
      using gFine = typename gauge::FieldOrder<Float,fineColor,1,gOrder>;
      using gCoarse = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat>;
      using gCoarseAtomic = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,storeType>;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      V vAccessor(const_cast<ColorSpinorField&>(v), nFace);
      F uvAccessor(*uv, nFace);
      F avAccessor(*av, nFace);
      gFine gAccessor(const_cast<GaugeField&>(g));
      gFine lAccessor(const_cast<GaugeField&>(l));
      gFine xinvAccessor(const_cast<GaugeField&>(XinvKD));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarseAtomic yAccessorAtomic(*Yatomic);
      gCoarseAtomic xAccessorAtomic(*Xatomic);
      
      // the repeated xinvAccessor is intentional
      calculateY<use_mma, QUDA_CPU_FIELD_LOCATION, false, Float, fineSpin, fineColor, coarseSpin, coarseColor>(
        yAccessor, xAccessor, yAccessorAtomic, xAccessorAtomic, uvAccessor, avAccessor, vAccessor, gAccessor,
        lAccessor, xinvAccessor, xinvAccessor, Y, X, *Yatomic, *Xatomic, *uv, *av, v,
        kappa, mass, mu_dummy, mu_factor_dummy, dirac, matpc, need_bidirectional, T.fineToCoarse(Y.Location()),
        T.coarseToFine(Y.Location()));
    } else {

      constexpr QudaFieldOrder csOrder = QUDA_FLOAT2_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_FLOAT2_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using V = typename colorspinor::FieldOrderCB<Float, fineSpin, fineColor, coarseColor, csOrder, vFloat, vFloat, false, false>;
      using F = typename colorspinor::FieldOrderCB<Float, uvSpin, fineColor, coarseColor, csOrder, vFloat, vFloat, false, false>;
      using gFine =  typename gauge::FieldOrder<Float,fineColor,1,gOrder,true,Float>;
      using gCoarse = typename gauge::FieldOrder<Float, coarseColor * coarseSpin, coarseSpin, gOrder, true, vFloat>;
      using gCoarseAtomic = typename gauge::FieldOrder<Float, coarseColor * coarseSpin, coarseSpin, gOrder, true, storeType>;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      V vAccessor(const_cast<ColorSpinorField &>(v), nFace);
      F uvAccessor(*uv, nFace);
      F avAccessor(*av, nFace);
      gFine gAccessor(const_cast<GaugeField &>(g));
      gFine lAccessor(const_cast<GaugeField &>(l));
      gFine xinvAccessor(const_cast<GaugeField&>(XinvKD));
      gCoarse yAccessor(const_cast<GaugeField &>(Y));
      gCoarse xAccessor(const_cast<GaugeField &>(X));
      gCoarseAtomic yAccessorAtomic(*Yatomic);
      gCoarseAtomic xAccessorAtomic(*Xatomic);

      // create a dummy clover field to allow us to call the external clover reduction routines elsewhere
      calculateY<use_mma, QUDA_CUDA_FIELD_LOCATION, false, Float, fineSpin, fineColor, coarseSpin, coarseColor>(
        yAccessor, xAccessor, yAccessorAtomic, xAccessorAtomic, uvAccessor, avAccessor, vAccessor, gAccessor,
        lAccessor, xinvAccessor, xinvAccessor, Y, X, *Yatomic, *Xatomic, *uv, *av, v,
        kappa, mass, mu_dummy, mu_factor_dummy, dirac, matpc, need_bidirectional, T.fineToCoarse(Y.Location()),
        T.coarseToFine(Y.Location()));
    }

    // Clean up
    if (Yatomic != &Y) delete Yatomic;
    if (Xatomic != &X) delete Xatomic;

    if (av != nullptr && &T.Vectors(location) != av) delete av;
    if (uv != nullptr) delete uv;

  }

  // template on UV spin, which can be 1 for the non-KD ops but needs to be 2 for the KD op
  template <typename Float, typename vFloat, int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void aggregateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    if (dirac == QUDA_STAGGERED_DIRAC || dirac == QUDA_STAGGEREDPC_DIRAC || dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADPC_DIRAC) {
      // uvSpin == 1
      aggregateStaggeredY<Float, vFloat, fineColor, fineSpin, coarseColor, coarseSpin, 1>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) {
      // uvSpin == 2
      aggregateStaggeredY<Float, vFloat, fineColor, fineSpin, coarseColor, coarseSpin, 2>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else {
      errorQuda("Unexpected dirac type %d\n", dirac);
    }
  }

  // template on the number of coarse degrees of freedom, branch between naive K-D 
  // and actual aggregation
  template <typename Float, typename vFloat, int fineColor, int fineSpin>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 24) {
      if (T.getTransferType() == QUDA_TRANSFER_COARSE_KD)
        // the permutation routines don't need Yatomic, Xatomic, uv, av, l
        calculateStaggeredY<Float,vFloat,fineColor,fineSpin,24,coarseSpin>(Y, X, T, g, mass, dirac, matpc);
      else {
        // free field aggregation
        aggregateStaggeredY<Float,vFloat,fineColor,fineSpin,24,coarseSpin>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      }
    } else if (coarseColor == 64) {
      aggregateStaggeredY<Float,vFloat,fineColor,fineSpin,64,coarseSpin>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else if (coarseColor == 96) {
      aggregateStaggeredY<Float,vFloat,fineColor,fineSpin,96,coarseSpin>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else { // note --- may revisit 3 -> 96 in the future
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, typename vFloat, int fineColor>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    if (T.Vectors().Nspin() == 1) {
      calculateStaggeredY<Float,vFloat,fineColor,1>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else {
      errorQuda("Unsupported number of spins %d\n", T.Vectors(X.Location()).Nspin());
    }
  }

  // template on fine colors
  template <typename Float, typename vFloat>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    if (g.Ncolor() == 3 && l.Ncolor() == 3) {
      calculateStaggeredY<Float,vFloat,3>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else {
      errorQuda("Unsupported number of colors %d %d\n", g.Ncolor(), l.Ncolor());
    }
  }

#if defined(GPU_MULTIGRID) && defined(GPU_STAGGERED_DIRAC)
  //Does the heavy lifting of creating the coarse color matrices Y
  // FIXME: the empty GaugeField& is a placeholder for the long links for now
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    checkPrecision(T.Vectors(X.Location()), X, Y);

    if (getVerbosity() >= QUDA_SUMMARIZE) printfQuda("Computing Y field......\n");

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      if (T.Vectors(X.Location()).Precision() == QUDA_DOUBLE_PRECISION) {
        calculateStaggeredY<double,double>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      } else {
        errorQuda("Unsupported precision %d\n", Y.Precision());
      }
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else 
#if QUDA_PRECISION & 4
    if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      if (T.Vectors(X.Location()).Precision() == QUDA_SINGLE_PRECISION) {
        calculateStaggeredY<float,float>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      } else {
        errorQuda("Unsupported precision %d\n", T.Vectors(X.Location()).Precision());
      }
    } else 
#endif
#if QUDA_PRECISION & 2
    if (Y.Precision() == QUDA_HALF_PRECISION) {
      if (T.Vectors(X.Location()).Precision() == QUDA_HALF_PRECISION) {
        calculateStaggeredY<float,short>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      } else {
        errorQuda("Unsupported precision %d\n", T.Vectors(X.Location()).Precision());
      }
    } else
#endif
    {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    if (getVerbosity() >= QUDA_SUMMARIZE) printfQuda("....done computing Y field\n");
  }
#else
  void calculateStaggeredY(GaugeField &, GaugeField &, const Transfer &, const GaugeField &, const GaugeField&,
                           const GaugeField &, double, QudaDiracType, QudaMatPCType)
  {
    errorQuda("Staggered multigrid has not been built");
  }
#endif

  //Calculates the coarse color matrix and puts the result in Y.
  // N.B. Assumes Y, X have been allocated.
  void StaggeredCoarseOp(GaugeField &Y, GaugeField &X, const Transfer &T, const cudaGaugeField &gauge, const cudaGaugeField &longGauge,
                         const GaugeField& XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    QudaPrecision precision = Y.Precision();
    QudaFieldLocation location = checkLocation(Y, X);

    // sanity check long link coarsening
    if ((dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && &gauge == &longGauge)
      errorQuda("Dirac type is %d but fat and long gauge links alias", dirac);

    // sanity check KD op coarsening
    if ((dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && &gauge == &XinvKD)
      errorQuda("Dirac type is %d but fat links and KD inverse fields alias", dirac);

    if (dirac == QUDA_ASQTADKD_DIRAC && &longGauge == &XinvKD)
      errorQuda("Dirac type is %d but long links and KD inverse fields alias", dirac);

    if ((dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && XinvKD.Reconstruct() != QUDA_RECONSTRUCT_NO)
      errorQuda("Invalid reconstruct %d for KD inverse field", XinvKD.Reconstruct());

    std::unique_ptr<GaugeField> tmp_U(nullptr);
    std::unique_ptr<GaugeField> tmp_L(nullptr);
    std::unique_ptr<GaugeField> tmp_Xinv(nullptr);

    bool need_tmp_U = false;
    bool need_tmp_L = false;
    bool need_tmp_Xinv = false;

    if (location == QUDA_CPU_FIELD_LOCATION) {
      //First make a cpu gauge field from the cuda gauge field
      int pad = 0;
      GaugeFieldParam gf_param(gauge.X(), precision, QUDA_RECONSTRUCT_NO, pad, gauge.Geometry());
      gf_param.order = QUDA_QDP_GAUGE_ORDER;
      gf_param.fixed = gauge.GaugeFixed();
      gf_param.link_type = gauge.LinkType();
      gf_param.t_boundary = gauge.TBoundary();
      gf_param.anisotropy = gauge.Anisotropy();
      gf_param.gauge = nullptr;
      gf_param.create = QUDA_NULL_FIELD_CREATE;
      gf_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      gf_param.nFace = 1;
      gf_param.ghostExchange = QUDA_GHOST_EXCHANGE_PAD;

      tmp_U = std::make_unique<cpuGaugeField>(gf_param);
      need_tmp_U = true;

      //Copy the cuda gauge field to the cpu
      gauge.saveCPUField(reinterpret_cast<cpuGaugeField&>(*tmp_U));

            // Create either a real or a dummy L field
      GaugeFieldParam lgf_param(longGauge.X(), precision, QUDA_RECONSTRUCT_NO, pad, longGauge.Geometry());
      if (!(dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC))
        for (int i = 0; i < lgf_param.nDim; i++) lgf_param.x[i] = 0;
      lgf_param.order = QUDA_QDP_GAUGE_ORDER;
      lgf_param.fixed = longGauge.GaugeFixed();
      lgf_param.link_type = longGauge.LinkType();
      lgf_param.t_boundary = longGauge.TBoundary();
      lgf_param.anisotropy = longGauge.Anisotropy();
      lgf_param.gauge = nullptr;
      lgf_param.create = QUDA_NULL_FIELD_CREATE;
      lgf_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      lgf_param.nFace = 3;
      lgf_param.ghostExchange = QUDA_GHOST_EXCHANGE_PAD;

      tmp_L = std::make_unique<cpuGaugeField>(lgf_param);
      need_tmp_L = true;

      //Copy the cuda gauge field to the cpu
      if (dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)
        longGauge.saveCPUField(reinterpret_cast<cpuGaugeField&>(*tmp_L));

      // Create either a real or a dummy Xinv field
      GaugeFieldParam xgf_param(XinvKD.X(), precision, QUDA_RECONSTRUCT_NO, pad, XinvKD.Geometry());
      if (!(dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC))
        for (int i = 0; i < xgf_param.nDim; i++) xgf_param.x[i] = 0;
      xgf_param.order = QUDA_QDP_GAUGE_ORDER;
      xgf_param.fixed = XinvKD.GaugeFixed();
      xgf_param.link_type = XinvKD.LinkType();
      xgf_param.t_boundary = XinvKD.TBoundary();
      xgf_param.anisotropy = XinvKD.Anisotropy();
      if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) {
        xgf_param.create = QUDA_COPY_FIELD_CREATE;
      } else {
        xgf_param.gauge = nullptr;
        xgf_param.create = QUDA_NULL_FIELD_CREATE;
      }
      xgf_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      xgf_param.nFace = 0;
      xgf_param.ghostExchange = QUDA_GHOST_EXCHANGE_NO;

      tmp_Xinv = std::make_unique<cpuGaugeField>(xgf_param);
      need_tmp_Xinv = true;

      //Copy the cuda gauge field to the cpu
      //if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)
      //  XinvKD.saveCPUField(*static_cast<cpuGaugeField*>(Xinv));
    } else if (location == QUDA_CUDA_FIELD_LOCATION) {

      int pad = 0;

      // create some dummy fields if need be
      if (!(dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)) {
        // create a dummy field
        GaugeFieldParam lgf_param(longGauge);
        for (int i = 0; i < lgf_param.nDim; i++) lgf_param.x[i] = 0;
        lgf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        lgf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        lgf_param.setPrecision(lgf_param.Precision());
        lgf_param.create = QUDA_NULL_FIELD_CREATE;
        tmp_L = std::make_unique<cudaGaugeField>(lgf_param);
        need_tmp_L = true;
      } else if ((dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && longGauge.Reconstruct() != QUDA_RECONSTRUCT_NO) {
        // create a copy of the gauge field with no reconstruction
        GaugeFieldParam lgf_param(longGauge);
        lgf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        lgf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        lgf_param.setPrecision(lgf_param.Precision());
        tmp_L = std::make_unique<cudaGaugeField>(lgf_param);

        tmp_L->copy(longGauge);
        need_tmp_L = true;
      }

      if (!(dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)) {
        // Create a dummy field
        GaugeFieldParam xgf_param(XinvKD.X(), precision, QUDA_RECONSTRUCT_NO, pad, XinvKD.Geometry());
        for (int i = 0; i < xgf_param.nDim; i++) xgf_param.x[i] = 0;
        xgf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        xgf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        xgf_param.setPrecision(xgf_param.Precision());
        xgf_param.create = QUDA_NULL_FIELD_CREATE;
        tmp_Xinv = std::make_unique<cudaGaugeField>(xgf_param);
        need_tmp_Xinv = true;
      }
      // no need to worry about XinvKD's reconstruct

      if (gauge.Reconstruct() != QUDA_RECONSTRUCT_NO) {
        //Create a copy of the gauge field with no reconstruction, required for fine-grained access
        GaugeFieldParam gf_param(gauge);
        gf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        gf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        gf_param.setPrecision(gf_param.Precision());
        tmp_U = std::make_unique<cudaGaugeField>(gf_param);
        need_tmp_U = true;

        tmp_U->copy(gauge);
      }
    }

    const GaugeField& U = need_tmp_U ? *tmp_U : reinterpret_cast<const GaugeField&>(gauge);
    const GaugeField& L = need_tmp_L ? *tmp_L : reinterpret_cast<const GaugeField&>(longGauge);
    const GaugeField& Xinv = need_tmp_Xinv ? *tmp_Xinv : reinterpret_cast<const GaugeField&>(XinvKD);

    calculateStaggeredY(Y, X, T, U, L, Xinv, mass, dirac, matpc);

  }

} //namespace quda
