#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cstring> // needed for memset



#include <tune_quda.h>
#include <typeinfo>

#include <quda_internal.h>
#include <float_vector.h>
#include <blas_quda.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <face_quda.h> // this is where the MPI / QMP depdendent code is

#define checkSpinor(a, b)						\
  {									\
    if (a.Precision() != b.Precision())					\
      errorQuda("precisions do not match: %d %d", a.Precision(), b.Precision()); \
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

#define checkLength(a, b)						\
  {									\
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

namespace quda {

  namespace blas {

#define BLAS_SPINOR // do not include ghost functions in Spinor class to reduce parameter space overhead
#include <texture.h>

    unsigned long long flops;
    unsigned long long bytes;

    void zero(ColorSpinorField &a) {
      if (typeid(a) == typeid(cudaColorSpinorField)) {
	static_cast<cudaColorSpinorField&>(a).zero();
      } else {
	static_cast<cpuColorSpinorField&>(a).zero();
      }
    }

    static hipStream_t *blasStream;

    static struct {
      const char *vol_str;
      const char *aux_str;
      char aux_tmp[TuneKey::aux_n];
    } blasStrings;

    void initReduce();
    void endReduce();

    void init()
    {
      blasStream = &streams[Nstream-1];
      initReduce();
    }

    void end(void)
    {
      endReduce();
    }

    hipStream_t* getStream() { return blasStream; }

#include <blas_core.cuh>

#include <blas_core.h>
#include <blas_mixed_core.h>
#include <multi_blas_core.cuh>
#include <multi_blas_core.h>
#include <multi_blas_mixed_core.h>


    template <typename Float2, typename FloatN>
    struct BlasFunctor {

      //! pre-computation routine before the main loop
      virtual __device__ __host__ void init() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) = 0;
    };

    template <int NXZ, typename Float2, typename FloatN>
    struct MultiBlasFunctor {

      //! pre-computation routine before the main loop
      virtual __device__ __host__ void init() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j) = 0;
    };

    /**
       Functor to perform the operation y = a*x + b*y
    */
    template <typename Float2, typename FloatN>
    struct axpby_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      axpby_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y = a.x*x + b.x*y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 3; } //! flops per element
    };

    void axpby(const double &a, ColorSpinorField &x, const double &b, ColorSpinorField &y) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpby_,0,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
				       x, y, x, x);
      } else {
	blasCuda<axpby_,0,1,0,0>(make_double2(a, 0.0), make_double2(b, 0.0), make_double2(0.0, 0.0),
				 x, y, x, x);
      }
    }

    /**
       Functor to perform the operation y += x
    */
    template <typename Float2, typename FloatN>
    struct xpy_ : public BlasFunctor<Float2,FloatN> {
      xpy_(const Float2 &a, const Float2 &b, const Float2 &c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y += x ; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void xpy(ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<xpy_,0,1,0,0>(make_double2(1.0, 0.0), make_double2(1.0, 0.0),
			     make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to perform the operation y += a*x
    */
    template <typename Float2, typename FloatN>
    struct axpy_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      axpy_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y = a.x*x + y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    void axpy(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpy_,0,1,0,0>(make_double2(a,0.0), make_double2(1.0,0.0), make_double2(0.0,0.0),
				       x, y, x, x);
      } else {
	blasCuda<axpy_,0,1,0,0>(make_double2(a, 0.0), make_double2(1.0, 0.0), make_double2(0.0, 0.0),
			       x, y, x, x);
      }
    }

    /**
       Functor to perform the operation y = x + a*y
    */
    template <typename Float2, typename FloatN>
    struct xpay_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      xpay_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y = x + a.x*y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    void xpay(ColorSpinorField &x, const double &a, ColorSpinorField &y) {
      blasCuda<xpay_,0,1,0,0>(make_double2(a,0.0), make_double2(0.0, 0.0), make_double2(0.0, 0.0),
			     x, y, x, x);
    }

    /**
       Functor to perform the operation y -= x;
    */
    template <typename Float2, typename FloatN>
    struct mxpy_ : public BlasFunctor<Float2,FloatN> {
      mxpy_(const Float2 &a, const Float2 &b, const Float2 &c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y -= x; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void mxpy(ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<mxpy_,0,1,0,0>(make_double2(1.0, 0.0), make_double2(1.0, 0.0),
			     make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to perform the operation x *= a
    */
    template <typename Float2, typename FloatN>
    struct ax_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      ax_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { x *= a.x; }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void ax(const double &a, ColorSpinorField &x) {
      blasCuda<ax_,1,0,0,0>(make_double2(a, 0.0), make_double2(0.0, 0.0),
			   make_double2(0.0, 0.0), x, x, x, x);
    }

    /**
       Functor to perform the operation y += a * x  (real-valued)
    */

    template<int NXZ, typename Float2, typename FloatN>
    struct multiaxpy_ : public MultiBlasFunctor<NXZ, Float2, FloatN> { 
      const int NYW;
      // ignore parameter arrays since we place them in constant memory
      multiaxpy_(const coeff_array<double> &a, const coeff_array<double> &b,
		  const coeff_array<double> &c, int NYW) : NYW(NYW) { }

      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
      {
#ifdef __CUDA_ARCH__
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_d); // fetch coefficient matrix from constant memory
        y = a[MAX_MULTI_BLAS_N*j+i].x*x + y;
#else
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_h);
        y = a[NYW*j+i].x*x + y;
#endif
      }

      int streams() { return NYW + NXZ*NYW; } //! total number of input and output streams
      int flops() { return 2*NXZ*NYW; } //! flops per real element
    };



    void axpy(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {

      // mark true since we will copy the "a" matrix into constant memory
      coeff_array<double> a(a_, true), b, c;

      switch (x.size()) {
      case 1:
	multiblasCuda<1,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 2:
	multiblasCuda<2,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 3:
	multiblasCuda<3,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 4:
	multiblasCuda<4,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 5:
	multiblasCuda<5,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 6:
	multiblasCuda<6,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 7:
	multiblasCuda<7,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 8:
	multiblasCuda<8,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 9:
	multiblasCuda<9,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 10:
	multiblasCuda<10,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 11:
	multiblasCuda<11,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 12:
	multiblasCuda<12,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 13:
	multiblasCuda<13,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 14:
	multiblasCuda<14,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 15:
	multiblasCuda<15,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 16:
	multiblasCuda<16,multiaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      default:
	// split the problem in half and recurse
	const double *a0 = &a_[0];
	const double *a1 = &a_[x.size()*y.size()/2];

	std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
	std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());

	axpy(a0, x0, y);
	axpy(a1, x1, y);
      }
    }


    /**
       Functor to perform the operation y += a * x  (complex-valued)
    */

    __device__ __host__ void _caxpy(const float2 &a, const float4 &x, float4 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
      y.z += a.x*x.z; y.z -= a.y*x.w;
      y.w += a.y*x.z; y.w += a.x*x.w;
    }

    __device__ __host__ void _caxpy(const float2 &a, const float2 &x, float2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }

    __device__ __host__ void _caxpy(const double2 &a, const double2 &x, double2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }

    template <typename Float2, typename FloatN>
    struct caxpy_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      caxpy_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    void caxpy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<caxpy_,0,1,0,0>(make_double2(real(a),imag(a)), make_double2(0.0, 0.0),
			       make_double2(0.0, 0.0), x, y, x, x);
    }

    template<int NXZ, typename Float2, typename FloatN>
    struct multicaxpy_ : public MultiBlasFunctor<NXZ, Float2, FloatN> { 
      const int NYW;
      // ignore parameter arrays since we place them in constant memory
      multicaxpy_(const coeff_array<Complex> &a, const coeff_array<Complex> &b,
		  const coeff_array<Complex> &c, int NYW) : NYW(NYW) { }

      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
      {
#ifdef __CUDA_ARCH__
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_d); // fetch coefficient matrix from constant memory
	_caxpy(a[MAX_MULTI_BLAS_N*j+i], x, y);
#else
	Float2 *a = reinterpret_cast<Float2*>(Amatrix_h);
	_caxpy(a[NYW*j+i], x, y);
#endif
      }

      int streams() { return 2*NYW + NXZ*NYW; } //! total number of input and output streams
      int flops() { return 4*NXZ*NYW; } //! flops per real element
    };

    void caxpy(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {

      // mark true since we will copy the "a" matrix into constant memory
      coeff_array<Complex> a(a_, true), b, c;

      switch (x.size()) {
      case 1:
	multiblasCuda<1,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 2:
	multiblasCuda<2,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 3:
	multiblasCuda<3,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 4:
	multiblasCuda<4,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 5:
	multiblasCuda<5,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 6:
	multiblasCuda<6,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 7:
	multiblasCuda<7,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 8:
	multiblasCuda<8,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 9:
	multiblasCuda<9,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 10:
	multiblasCuda<10,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 11:
	multiblasCuda<11,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 12:
	multiblasCuda<12,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 13:
	multiblasCuda<13,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 14:
	multiblasCuda<14,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 15:
	multiblasCuda<15,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      case 16:
	multiblasCuda<16,multicaxpy_,0,1,0,0>(a, b, c, x, y, x, y);
        break;
      default:
	// split the problem in half and recurse
	const Complex *a0 = &a_[0];
	const Complex *a1 = &a_[x.size()*y.size()/2];

	std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
	std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());

	caxpy(a0, x0, y);
	caxpy(a1, x1, y);
      }
    }

    void caxpy(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy(a, x.Components(), y.Components()); }

    /**
       Functor to perform the operation y = a*x + b*y  (complex-valued)
    */

    __device__ __host__ void _caxpby(const float2 &a, const float4 &x, const float2 &b, float4 &y)
    { float4 yy;
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;
      yy.z = a.x*x.z; yy.z -= a.y*x.w; yy.z += b.x*y.z; yy.z -= b.y*y.w;
      yy.w = a.y*x.z; yy.w += a.x*x.w; yy.w += b.y*y.z; yy.w += b.x*y.w;
      y = yy; }

    __device__ __host__ void _caxpby(const float2 &a, const float2 &x, const float2 &b, float2 &y)
    { float2 yy;
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;
      y = yy; }

    __device__ __host__ void _caxpby(const double2 &a, const double2 &x, const double2 &b, double2 &y)
    { double2 yy;
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;
      y = yy; }

    template <typename Float2, typename FloatN>
    struct caxpby_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpby_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpby(a, x, b, y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 7; } //! flops per element
    };

    void caxpby(const Complex &a, ColorSpinorField &x, const Complex &b, ColorSpinorField &y) {
      blasCuda<caxpby_,0,1,0,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)),
			       make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to performs the operation z[i] = x[i] + a*y[i] + b*z[i]
    */

    __device__ __host__ void _cxpaypbz(const float4 &x, const float2 &a, const float4 &y, const float2 &b, float4 &z) {
      float4 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      zz.z = x.z + a.x*y.z; zz.z -= a.y*y.w; zz.z += b.x*z.z; zz.z -= b.y*z.w;
      zz.w = x.w + a.y*y.z; zz.w += a.x*y.w; zz.w += b.y*z.z; zz.w += b.x*z.w;
      z = zz;
    }

    __device__ __host__ void _cxpaypbz(const float2 &x, const float2 &a, const float2 &y, const float2 &b, float2 &z) {
      float2 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      z = zz;
    }

    __device__ __host__ void _cxpaypbz(const double2 &x, const double2 &a, const double2 &y, const double2 &b, double2 &z) {
      double2 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      z = zz;
    }

    template <typename Float2, typename FloatN>
    struct cxpaypbz_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      cxpaypbz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _cxpaypbz(x, a, y, b, z); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void cxpaypbz(ColorSpinorField &x, const Complex &a, ColorSpinorField &y,
		  const Complex &b, ColorSpinorField &z) {
      blasCuda<cxpaypbz_,0,0,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)),
				 make_double2(0.0, 0.0), x, y, z, z);
    }

    /**
       Functor performing the operations: y[i] = a*x[i] + y[i]; x[i] = b*z[i] + c*x[i]
    */
    template <typename Float2, typename FloatN>
    struct axpyBzpcx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      const Float2 c;
      axpyBzpcx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b), c(c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*x; x = b.x*z + c.x*x; }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    void axpyBzpcx(const double &a, ColorSpinorField& x, ColorSpinorField& y, const double &b,
		   ColorSpinorField& z, const double &c) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpyBzpcx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0),
					    make_double2(c,0.0), x, y, z, x);
      } else {
	// swap arguments around
	blasCuda<axpyBzpcx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0),
				     make_double2(c,0.0), x, y, z, x);
      }
    }


    template<int NXZ, typename Float2, typename FloatN>
    struct multi_axpyBzpcx_ : public MultiBlasFunctor<NXZ, Float2, FloatN> {
      typedef typename scalar<Float2>::type real;
      const int NYW;
      real a[MAX_MULTI_BLAS_N], b[MAX_MULTI_BLAS_N], c[MAX_MULTI_BLAS_N];

      multi_axpyBzpcx_(const coeff_array<double> &a, const coeff_array<double> &b, const coeff_array<double> &c, int NYW) : NYW(NYW){
	// copy arguments into the functor
	for (int i=0; i<NYW; i++) { this->a[i] = a.data[i]; this->b[i] = b.data[i]; this->c[i] = c.data[i]; }
      }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
      {
	y += a[i] * w;
	w = b[i] * x + c[i] * w;
      }
      int streams() { return 4*NYW + NXZ; } //! total number of input and output streams
      int flops() { return 5*NXZ*NYW; } //! flops per real element
    };

    void axpyBzpcx(const double *a_, std::vector<ColorSpinorField*> &x_, std::vector<ColorSpinorField*> &y_,
		   const double *b_, ColorSpinorField &z_, const double *c_) {

      if (y_.size() <= MAX_MULTI_BLAS_N) {
	// swizzle order since we are writing to x_ and y_, but the
	// multi-blas only allow writing to y and w, and moreover the
	// block width of y and w must match, and x and z must match.
	std::vector<ColorSpinorField*> &y = y_;
	std::vector<ColorSpinorField*> &w = x_;

	// wrap a container around the third solo vector
	std::vector<ColorSpinorField*> x;
	x.push_back(&z_);

	// we will curry the parameter arrays into the functor
	coeff_array<double> a(a_,false), b(b_,false), c(c_,false);

	if (x[0]->Precision() != y[0]->Precision() ) {
	  mixed::multiblasCuda<1,multi_axpyBzpcx_,0,1,0,1>(a, b, c, x, y, x, w);
	} else {
	  multiblasCuda<1,multi_axpyBzpcx_,0,1,0,1>(a, b, c, x, y, x, w);
	}
      } else {
	// split the problem in half and recurse
	const double *a0 = &a_[0];
	const double *b0 = &b_[0];
	const double *c0 = &c_[0];

	std::vector<ColorSpinorField*> x0(x_.begin(), x_.begin() + x_.size()/2);
	std::vector<ColorSpinorField*> y0(y_.begin(), y_.begin() + y_.size()/2);

	axpyBzpcx(a0, x0, y0, b0, z_, c0);

	const double *a1 = &a_[y_.size()/2];
	const double *b1 = &b_[y_.size()/2];
	const double *c1 = &c_[y_.size()/2];

	std::vector<ColorSpinorField*> x1(x_.begin() + x_.size()/2, x_.end());
	std::vector<ColorSpinorField*> y1(y_.begin() + y_.size()/2, y_.end());

	axpyBzpcx(a1, x1, y1, b1, z_, c1);
      }
    }


    /**
       Functor performing the operations: y[i] = a*x[i] + y[i]; x[i] = z[i] + b*x[i]
    */
    template <typename Float2, typename FloatN>
    struct axpyZpbx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      axpyZpbx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*x; x = z + b.x*x; }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    void axpyZpbx(const double &a, ColorSpinorField& x, ColorSpinorField& y,
		  ColorSpinorField& z, const double &b) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpyZpbx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
					   x, y, z, x);
      } else {
	// swap arguments around
	blasCuda<axpyZpbx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
				    x, y, z, x);
      }
    }

    /**
       Functor performing the operations z[i] = a*x[i] + b*y[i] + z[i] and y[i] -= b*w[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypzYmbw_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpbypzYmbw_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, z); _caxpy(b, y, z); _caxpy(-b, w, y); }

      static int streams() { return 6; } //! total number of input and output streams
      static int flops() { return 12; } //! flops per element
    };

    void caxpbypzYmbw(const Complex &a, ColorSpinorField &x, const Complex &b,
		      ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w) {
      blasCuda<caxpbypzYmbw_,0,1,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)),
				     make_double2(0.0,0.0), x, y, z, w);
    }

    /**
       Functor performing the operation y[i] += a*b*x[i], x[i] *= a
    */
    template <typename Float2, typename FloatN>
    struct cabxpyAx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      cabxpyAx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { x *= a.x; _caxpy(b, x, y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    void cabxpyAx(const double &a, const Complex &b,
		  ColorSpinorField &x, ColorSpinorField &y) {
      // swap arguments around
      blasCuda<cabxpyAx_,1,1,0,0>(make_double2(a,0.0), make_double2(REAL(b),IMAG(b)),
				  make_double2(0.0,0.0), x, y, x, x);
    }

    /**
       Functor performing the operation z[i] = a*x[i] + b*y[i] + z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypz_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpbypz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, z); _caxpy(b, y, z); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpbypz(const Complex &a, ColorSpinorField &x, const Complex &b,
		  ColorSpinorField &y, ColorSpinorField &z) {
      blasCuda<caxpbypz_,0,0,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b),IMAG(b)),
				  make_double2(0.0,0.0), x, y, z, z);
    }

    /**
       Functor Performing the operation w[i] = a*x[i] + b*y[i] + c*z[i] + w[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypczpw_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      const Float2 c;
      caxpbypczpw_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b), c(c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, w); _caxpy(b, y, w); _caxpy(c, z, w); }

      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 12; } //! flops per element
    };

    void caxpbypczpw(const Complex &a, ColorSpinorField &x, const Complex &b,
		     ColorSpinorField &y, const Complex &c, ColorSpinorField &z,
		     ColorSpinorField &w) {
      blasCuda<caxpbypczpw_,0,0,0,1>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b),IMAG(b)),
				     make_double2(REAL(c),IMAG(c)), x, y, z, w);
    }

    /**
       double caxpyXmaz(c a, V x, V y, V z){}

       First performs the operation y[i] += a*x[i]
       Second performs the operator x[i] -= a*z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpyxmaz_ : public BlasFunctor<Float2,FloatN> {
      Float2 a;
      caxpyxmaz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); _caxpy(-a, z, x); }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpyXmaz(const Complex &a, ColorSpinorField &x,
		   ColorSpinorField &y, ColorSpinorField &z) {
      blasCuda<caxpyxmaz_,1,1,0,0>(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0),
				   make_double2(0.0, 0.0), x, y, z, x);
    }

    /**
       double caxpyXmazMR(c a, V x, V y, V z){}

       First performs the operation y[i] += a*x[i]
       Second performs the operator x[i] -= a*z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpyxmazMR_ : public BlasFunctor<Float2,FloatN> {
      Float2 a;
      double3 *Ar3;
      caxpyxmazMR_(const Float2 &a, const Float2 &b, const Float2 &c)
	: a(a), Ar3(static_cast<double3*>(blas::getDeviceReduceBuffer())) { ; }

      inline __device__ __host__ void init() {
#ifdef __CUDA_ARCH__
	typedef decltype(a.x) real;
	double3 result = __ldg(Ar3);
	a.y = a.x * (real)(result.y) * ((real)1.0 / (real)result.z);
	a.x = a.x * (real)(result.x) * ((real)1.0 / (real)result.z);
#endif
      }

      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); _caxpy(-a, z, x); }

      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpyXmazMR(const Complex &a, ColorSpinorField &x,
		     ColorSpinorField &y, ColorSpinorField &z) {
      if (!commAsyncReduction())
	errorQuda("This kernel requires asynchronous reductions to be set");
      if (x.Location() == QUDA_CPU_FIELD_LOCATION)
	errorQuda("This kernel cannot be run on CPU fields");

      blasCuda<caxpyxmazMR_,1,1,0,0>(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0),
				     make_double2(0.0, 0.0), x, y, z, x);
    }

    /**
       double tripleCGUpdate(d a, d b, V x, V y, V z, V w){}

       First performs the operation y[i] = y[i] + a*w[i]
       Second performs the operation z[i] = z[i] - a*x[i]
       Third performs the operation w[i] = z[i] + b*w[i]
    */
    template <typename Float2, typename FloatN>
    struct tripleCGUpdate_ : public BlasFunctor<Float2,FloatN> {
      Float2 a, b;
      tripleCGUpdate_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*w; z -= a.x*x; w = z + b.x*w; }
      static int streams() { return 7; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    void tripleCGUpdate(const double &a, const double &b, ColorSpinorField &x,
			ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w) {
      if (x.Precision() != y.Precision()) {
      // call hacked mixed precision kernel
	mixed::blasCuda<tripleCGUpdate_,0,1,1,1>(make_double2(a,0.0), make_double2(b,0.0),
						 make_double2(0.0,0.0), x, y, z, w);
      } else {
	blasCuda<tripleCGUpdate_,0,1,1,1>(make_double2(a, 0.0), make_double2(b, 0.0),
					  make_double2(0.0, 0.0), x, y, z, w);
      }
    }

  } // namespace blas

} // namespace quda
