#include "hip/hip_runtime.h"
namespace quda
{
	#include	"covDev.h"	//Covariant derivative definitions

	#define MORE_GENERIC_COVDEV(FUNC, dir, DAG, kernel_type, gridDim, blockDim, shared, stream, param,  ...)			\
		if		(reconstruct == QUDA_RECONSTRUCT_NO) {									\
			switch	(dir) {													\
				case 0:													\
				FUNC ## 018 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
				case 1:													\
				FUNC ## 118 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
				case 2:													\
				FUNC ## 218 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
				case 3:													\
				FUNC ## 318 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
			}														\
		} else if	(reconstruct == QUDA_RECONSTRUCT_12) {									\
			switch	(dir) {													\
				case 0:													\
				FUNC ## 012 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
				case 1:													\
				FUNC ## 112 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
				case 2:													\
				FUNC ## 212 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
				case 3:													\
				FUNC ## 312 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);\
				break;													\
			}														\
		} else if	(reconstruct == QUDA_RECONSTRUCT_8) {									\
			switch	(dir) {													\
				case 0:													\
				FUNC ## 08 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);	\
				break;													\
				case 1:													\
				FUNC ## 18 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);	\
				break;													\
				case 2:													\
				FUNC ## 28 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);	\
				break;													\
				case 3:													\
				FUNC ## 38 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);	\
				break;													\
			}														\
		}

	#define GENERIC_COVDEV(FUNC, dir, DAG, gridDim, blockDim, shared, stream, param,  ...) \
		switch(param.kernel_type) {						\
			case INTERIOR_KERNEL:							\
			MORE_GENERIC_COVDEV(FUNC, dir, DAG, INTERIOR_KERNEL,   gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
			break;			     				\
			case EXTERIOR_KERNEL_X:	     				\
			MORE_GENERIC_COVDEV(FUNC, dir, DAG, EXTERIOR_KERNEL_X, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
			break;			     				\
			case EXTERIOR_KERNEL_Y:	     				\
			MORE_GENERIC_COVDEV(FUNC, dir, DAG, EXTERIOR_KERNEL_Y, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
			break;			     				\
			case EXTERIOR_KERNEL_Z:	     				\
			MORE_GENERIC_COVDEV(FUNC, dir, DAG, EXTERIOR_KERNEL_Z, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
			break;			     				\
			case EXTERIOR_KERNEL_T:	     				\
			MORE_GENERIC_COVDEV(FUNC, dir, DAG, EXTERIOR_KERNEL_T, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
			break;								\
		}

	#define COVDEV(FUNC, mu, gridDim, blockDim, shared, stream, param, ...)	\
		if (mu < 4) {							\
			GENERIC_COVDEV(FUNC, mu, , gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
		} else {								\
			int nMu = mu - 4;								\
			GENERIC_COVDEV(FUNC, nMu, Dagger, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
		}

	void covDevCuda(DslashCuda &dslash, const size_t regSize, const int mu, TimeProfile &profile)
	{
		profile.Start(QUDA_PROFILE_TOTAL);

		const int	dir = mu%4;

		dslashParam.kernel_type = INTERIOR_KERNEL;
//		dslashParam.threads	= dslashConstants.VolumeCB();

		PROFILE(dslash.apply(streams[Nstream-1]), profile, QUDA_PROFILE_DSLASH_KERNEL);

		checkCudaError	();

		#ifdef MULTI_GPU
			if	(comm_dim(dir) > 1)
			{
				dslashParam.kernel_type	= static_cast<KernelType>(dir);
				dslashParam.ghostDim[dir]		= commDimPartitioned(dir); // determines whether to use regular or ghost indexing at boundary
				dslashParam.commDim[dir]		= commDimPartitioned(dir); // switch off comms if override = 0
			
				PROFILE(dslash.apply(streams[Nstream-1]), profile, QUDA_PROFILE_DSLASH_KERNEL);

				checkCudaError	();

				dslashParam.ghostDim[dir]		= 0; // not sure whether neccessary 
				dslashParam.commDim[dir]		= 0; 
			}
		#endif // MULTI_GPU

		profile.Stop(QUDA_PROFILE_TOTAL);
	}

	template <typename Float, typename Float2>
	class CovDevCuda : public SharedDslashCuda//public SharedDslashCuda
	{
		private:
			const cudaGaugeField *gauge;
//			const QudaReconstructType reconstruct;
			const int dagger;
			const int mu;
			const int dir;
			const int parity;

			void *gauge0, *gauge1;

			bool binded;
			
			#ifdef MULTI_GPU
				Float	*ghostBuffer;
				int	ghostVolume;
				int	ghostBytes;
				int	offset;
				int	Npad;
				int	Nvec;
				int	Nint;
			#endif

			#ifdef USE_TEXTURE_OBJECTS
				hipTextureObject_t tex;
			#endif
		protected:
			unsigned int minThreads			() const { if (dslashParam.kernel_type == INTERIOR_KERNEL) { return in->Volume(); } else { return ghostVolume; } }

			unsigned int sharedBytesPerThread	() const
			{
				return	0;
			}

			dim3 createGrid		(const dim3 &block) const
			{
				unsigned int	gx	= (dslashParam.threads + block.x - 1) / block.x;
				unsigned int	gy	= 1;
				unsigned int	gz	= 1;
				return	dim3(gx, gy, gz);
			}

			/** Advance 1-d block size, accounting for the differences of the covariant derivative (I could not make the 3-d block work reliably) */
			bool advanceBlockDim	(TuneParam &param) const
			{
//				if	(dslashParam.kernel_type != INTERIOR_KERNEL) return DslashCuda::advanceBlockDim(param);
 				const unsigned int min_threads = 2;
				const unsigned int max_threads = 512; // FIXME: use deviceProp.maxThreadsDim[0];
    
				param.block.x	+= 2;	  
				param.block.y	= 1;	  
				param.block.z	= 1;	  
				param.grid	= createGrid(param.block);

				if	((param.block.x > min_threads) && (param.block.x < max_threads))
					return	true;
				else
					return	false;
			}

			void allocateGhosts	()
			{
				if	(hipMalloc(&ghostBuffer, ghostBytes) != hipSuccess)
				{
					printf	("Error in rank %d: Unable to allocate %d bytes for GPU ghosts\n", comm_rank(), ghostBytes);
					exit	(-1);
				}
			}

			void exchangeGhosts	()
			{
				const int	rel = (mu < 4) ? 1 : -1;

				void	*send	= 0;
				void	*recv	= 0;

				//send buffers:
				if	(hipHostAlloc(&send, ghostBytes, 0) != hipSuccess)
				{
					printf	("Error in rank %d: Unable to allocate %d bytes for MPI requests (send)\n", comm_rank(), ghostBytes);
					exit	(-1);
				}

				//recv buffers in t-dir:
				if	(hipHostAlloc(&recv, ghostBytes, 0) != hipSuccess)
				{
					printf	("Error in rank %d: Unable to allocate %d bytes for MPI requests (recv)\n", comm_rank(), ghostBytes);
					exit	(-1);
				}

				switch	(mu)
				{
					default:
					break;

					case 0:
					{
						void	*sendFacePtr	= (char*) in->V();
						size_t	len		= Nvec*sizeof(Float);
						size_t	skip		= len*in->X(0);
						size_t	dpitch		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);

						for	(int t=0;t<ghostVolume;t++)
						{
							hipMemcpy2DAsync((void*) (((char*)send)+len*t), dpitch, (void*) (((char*)sendFacePtr)+skip*t),
									  spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
							hipStreamSynchronize(streams[0]);
						}
					}

					case 1:
					{
						void	*sendFacePtr	= (char*)in->V();
						size_t	len		= in->X(0)*Nvec*sizeof(Float);
						size_t	skip		= len*in->X(1);
						size_t	dpitch		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);

						for	(int tz=0;tz<(in->X(2)*in->X(3));tz++)
						{
							hipMemcpy2DAsync((void*) (((char*)send)+len*tz), dpitch, (void*) (((char*)sendFacePtr)+skip*tz),
									  spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
							hipStreamSynchronize(streams[0]);
						}
					}

					case 2:
					{
						void	*sendFacePtr	= (char*) in->V();
						size_t	len		= ghostVolume*Nvec*sizeof(Float)/in->X(3);
						size_t	skip		= len*in->X(2);
						size_t	dpitch		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);

						for	(int t=0;t<in->X(3);t++)
						{
							hipMemcpy2DAsync((void*) (((char*)send)+len*t), dpitch, (void*) (((char*)sendFacePtr)+skip*t),
									   spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
							hipStreamSynchronize(streams[0]);
						}
					}
					break;

					case 3:
					{
						void	*sendFacePtr	= (char*)in->V();
						size_t	len		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);
						hipMemcpy2DAsync(send, len, sendFacePtr, spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
						hipStreamSynchronize(streams[0]);
					}
					break;

					case 4:
					{
						void	*sendFacePtr	= (char*) in->V() + offset*Nvec*sizeof(Float);
						size_t	len		= Nvec*sizeof(Float);
						size_t	skip		= len*in->X(0);
						size_t	dpitch		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);

						for	(int t=0;t<ghostVolume;t++)
						{
							hipMemcpy2DAsync((void*) (((char*)send)+len*t), dpitch, (void*) (((char*)sendFacePtr)+skip*t),
									  spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
							hipStreamSynchronize(streams[0]);
						}
					}

					case 5:
					{
						void	*sendFacePtr	= ((char*) in->V()) + offset*Nvec*sizeof(Float);
						size_t	len		= in->X(0)*Nvec*sizeof(Float);
						size_t	skip		= len*in->X(1);
						size_t	dpitch		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);

						for	(int tz=0;tz<(in->X(2)*in->X(3));tz++)
						{
							hipMemcpy2DAsync((void*) (((char*)send)+len*tz), dpitch, (void*) (((char*)sendFacePtr)+skip*tz),
									  spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
							hipStreamSynchronize(streams[0]);
						}
					}
					break;

					case 6:
					{
						void	*sendFacePtr	= (((char*)in->V()) + offset*Nvec*sizeof(Float));
						size_t	len		= ghostVolume*Nvec*sizeof(Float)/in->X(3);
						size_t	skip		= len*in->X(2);
						size_t	dpitch		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);

						for	(int t=0;t<in->X(3);t++)
						{
							hipMemcpy2DAsync((void*) (((char*)send)+len*t), dpitch, (void*) (((char*)sendFacePtr)+skip*t),
									  spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
							hipStreamSynchronize(streams[0]);
						}
					}
					break;

					case 7:
					{
						void	*sendFacePtr	= (char*)in->V() + offset*Nvec*sizeof(Float);
						size_t	len		= ghostVolume*Nvec*sizeof(Float);
						size_t	spitch		= in->Stride()*Nvec*sizeof(Float);

						hipMemcpy2DAsync(send, len, sendFacePtr, spitch, len, Npad, hipMemcpyDeviceToHost, streams[0]);
						hipStreamSynchronize(streams[0]);
					}
					break;
				}

				//Send buffers to neighbors:

				MsgHandle	*mh_send;
				MsgHandle	*mh_from;

				mh_send		= comm_declare_send_relative	(send, dir, rel,      ghostBytes);
				mh_from		= comm_declare_receive_relative	(recv, dir, rel*(-1), ghostBytes);
				comm_start	(mh_send);
				comm_start	(mh_from);
				comm_wait	(mh_send);
				comm_wait	(mh_from);
				comm_free	(mh_send);
				comm_free	(mh_from);

				//Send buffers to GPU:
				hipMemcpy(ghostBuffer, recv, ghostBytes, hipMemcpyHostToDevice);
				hipDeviceSynchronize();

				hipHostFree(send);
				hipHostFree(recv);
			}


			void freeGhosts		() { hipFree(ghostBuffer); }

			void bindGhosts		()
			{
				if	(binded == false)	// bind only once
				{
					#ifdef USE_TEXTURE_OBJECTS
						hipChannelFormatDesc desc;
						memset(&desc, 0, sizeof(hipChannelFormatDesc));
						if (in->Precision() == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
						else desc.f = hipChannelFormatKindSigned; // half is short, double is int2

						// staggered fields in half and single are always two component
						if (in->Nspin() == 1 && (in->Precision() == QUDA_SINGLE_PRECISION))
						{
							desc.x = 8*in->Precision();
							desc.y = 8*in->Precision();
							desc.z = 0;
							desc.w = 0;
						} else { // all others are four component
							desc.x = (in->Precision() == QUDA_DOUBLE_PRECISION) ? 32 : 8*in->Precision();
							desc.y = (in->Precision() == QUDA_DOUBLE_PRECISION) ? 32 : 8*in->Precision();
							desc.z = (in->Precision() == QUDA_DOUBLE_PRECISION) ? 32 : 8*in->Precision();
							desc.w = (in->Precision() == QUDA_DOUBLE_PRECISION) ? 32 : 8*in->Precision();
						}

						hipResourceDesc resDesc;
						memset(&resDesc, 0, sizeof(resDesc));
						resDesc.resType = hipResourceTypeLinear;
						resDesc.res.linear.devPtr = ghostBuffer;
						resDesc.res.linear.desc = desc;
						resDesc.res.linear.sizeInBytes = Nint * ghostVolume * sizeof(Float);

						hipTextureDesc texDesc;
						memset(&texDesc, 0, sizeof(texDesc));
						texDesc.readMode = hipReadModeElementType;

						hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

						dslashParam.inTex = tex;
					#else
						if	(in->Precision() == QUDA_DOUBLE_PRECISION)
							hipBindTexture		(0, spinorTexDouble, (Float2*)ghostBuffer, ghostBytes);
						else if	(in->Precision() == QUDA_SINGLE_PRECISION)
							hipBindTexture		(0, spinorTexSingle, (Float2*)ghostBuffer, ghostBytes);
						else
							errorQuda		("Half precision for covariant derivative not supported.");
					#endif
					checkCudaError();
					binded	= true;
				}
			}
			void unbindGhosts	()
			{
				if	(binded == true)
				{
					#ifdef USE_TEXTURE_OBJECTS
						hipDestroyTextureObject(tex);
/*					#else
						if	(in->Precision() == QUDA_DOUBLE_PRECISION)
							hipUnbindTexture	(spinorTexDouble);
						else
							hipUnbindTexture	(spinorTexSingle);
*/
					#endif
					checkCudaError		();
					binded	= false;
				}
			}

			void unbindGauge		()
			{
				unbindGaugeTex		(*gauge);
				checkCudaError		();
			}

	
		public:
			CovDevCuda(cudaColorSpinorField *out, const cudaGaugeField *gauge, const cudaColorSpinorField *in, const int parity, const int mu)
			: SharedDslashCuda(out, in, 0, gauge->Reconstruct()), gauge(gauge), parity(parity), mu(mu), dir(mu%4), dagger(mu<4 ? 0 : 1), binded(false)
			{ 
				bindSpinorTex<Float2>	(in, out); 
				bindGaugeTex		(*gauge, parity, &gauge0, &gauge1);

				#ifdef MULTI_GPU
					if	(comm_dim(dir) > 1)
					{
						Nvec	= sizeof(Float2)/sizeof(Float);
						Nint	= in->Ncolor()*in->Nspin()*Nvec;
						Npad	= Nint/Nvec;

						switch	(dir)
						{
							case 0:
							ghostVolume	= in->X(1)*in->X(2)*in->X(3)/2;
							offset		= in->X(0) - 1;
							break;

							case 1:
							ghostVolume	= in->X(0)*in->X(2)*in->X(3);
							offset		= in->X(0)*(in->X(1) - 1);
							break;

							case 2:
							ghostVolume	= in->X(0)*in->X(1)*in->X(3);
							offset		= in->X(0)*in->X(1)*(in->X(2) - 1);
							break;

							case 3:
							ghostVolume	= in->X(0)*in->X(1)*in->X(2);
							offset		= in->Volume() - ghostVolume;
							break;
						}	

						ghostBytes	= ghostVolume*Nint*sizeof(Float);
						allocateGhosts	();
					}
				#endif
			}
	
			virtual ~CovDevCuda()
			{
				#ifdef MULTI_GPU
					if	(comm_dim(dir) > 1)
					{
						unbindGhosts	();
						freeGhosts	();
					}
				#endif
				unbindGauge();
			 }
/*	
			TuneKey tuneKey() const
			{
				TuneKey key = DslashCuda::tuneKey();
				std::stringstream recon;
				recon << reconstruct;
				key.aux += ",reconstruct=" + recon.str();
				return key;
			}
*/	
			void	apply(const hipStream_t &stream)
			{
				#ifdef SHARED_WILSON_DSLASH
					if	(dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
						errorQuda("Shared dslash (covariant derivative) does not yet support X-dimension partitioning");
				#endif
				if	((dslashParam.kernel_type == EXTERIOR_KERNEL_X) || (dslashParam.kernel_type == EXTERIOR_KERNEL_Y))
					errorQuda("Covariant derivative does not yet support X or Y-dimension partitioning");

				dslashParam.parity				= parity;

				for	(int i=0; i<4; i++)
				{
					dslashParam.ghostDim[i]			= 0;
					dslashParam.ghostOffset[i]		= 0;
					dslashParam.ghostNormOffset[i]		= 0;
					dslashParam.commDim[i]			= 0;
				}

				if	(dslashParam.kernel_type != INTERIOR_KERNEL)
				{
					dslashParam.threads			= ghostVolume;
					exchangeGhosts	();
					bindGhosts	();
					TuneParam tp				= tuneLaunch(*this, getTuning(), getVerbosity());
					COVDEV		(covDevM, mu, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam, (Float2*)out->V(), (Float2*)gauge0, (Float2*)gauge1, (Float2*)in->V());
				} else {
					dslashParam.threads			= in->Volume();
					TuneParam tp				= tuneLaunch(*this, getTuning(), getVerbosity());
					COVDEV		(covDevM, mu, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam, (Float2*)out->V(), (Float2*)gauge0, (Float2*)gauge1, (Float2*)in->V());
				}
			}

			long long flops() const { return 144 * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
	};

	void	covDev		(cudaColorSpinorField *out, const cudaGaugeField &gauge, const cudaColorSpinorField *in, const int parity, const int mu, TimeProfile &profile)
	{
		DslashCuda	*covdev	= 0;
		size_t		regSize	= sizeof(float);

		#ifdef	GPU_CONTRACT
			if	(in->Precision	() == QUDA_HALF_PRECISION)
				errorQuda	("Error: Half precision not supported");

			if	(in->Precision() != gauge.Precision())
				errorQuda("Mixing gauge %d and spinor %d precision not supported", gauge.Precision(), in->Precision());

			if	(in->Precision	() == QUDA_SINGLE_PRECISION)
				covdev	= new CovDevCuda<float, float4>(out, &gauge, in, parity, mu);
			else if	(in->Precision	() == QUDA_DOUBLE_PRECISION)
			{
				#if (__COMPUTE_CAPABILITY__ >= 130)
					covdev	= new CovDevCuda<double, double2>(out, &gauge, in, parity, mu);
					regSize = sizeof(double);
				#else
					errorQuda	("Error: Double precision not supported by hardware");
				#endif
			}

			covDevCuda(*covdev, regSize, mu, profile);

    			delete covdev;
			checkCudaError();
		#else
			errorQuda("Contraction kernels have not been built");
		#endif
	}

}
