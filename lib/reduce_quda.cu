#include "hip/hip_runtime.h"
#include <atomic>

#include <blas_quda.h>
#include <tune_quda.h>
#include <color_spinor_field_order.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/reduce_core.cuh>

// These are used for reduction kernels
static device_reduce_t *d_reduce = nullptr;
static device_reduce_t *h_reduce = nullptr;
static device_reduce_t *hd_reduce = nullptr;
static hipEvent_t reduceEnd;
static bool fast_reduce_enabled = false;

namespace quda {

  namespace blas {

    qudaStream_t* getStream();

    void* getDeviceReduceBuffer() { return d_reduce; }
    void* getMappedHostReduceBuffer() { return hd_reduce; }
    void* getHostReduceBuffer() { return h_reduce; }
    hipEvent_t* getReduceEvent() { return &reduceEnd; }
    bool getFastReduce() { return fast_reduce_enabled; }

    void initFastReduce(int32_t words)
    {
      // initialize the reduction values in 32-bit increments to INT_MIN
      for (int32_t i = 0; i < words; i++) {
        reinterpret_cast<int32_t *>(h_reduce)[i] = std::numeric_limits<int32_t>::min();
      }

      // ensure that the host memory write is complete before we launch the kernel
      atomic_thread_fence(std::memory_order_release);
    }

    void completeFastReduce(int32_t words)
    {
      volatile int32_t *check = reinterpret_cast<int32_t *>(h_reduce);
      int count = 0;
      int complete = 0;
      while (complete < words) {
        // ensure visiblity to any changes in memory
        atomic_thread_fence(std::memory_order_acquire);

        complete = 0;
        for (int32_t i = 0; i < words; i++) {
          // spin-wait until all values have been updated
          if (check[i] != std::numeric_limits<int32_t>::min()) complete++;
        }
        if (count++ % 10000 == 0) { // check error every 10000 iterations
          // if there is an error in the kernel then we need to exit the spin-wait
          if (hipSuccess != hipPeekAtLastError()) break;
        }
      }
    }

    void initReduce()
    {
      /* we have these different reductions to cater for:

         - regular reductions (reduce_quda.cu) where are reducing to a
           single vector type (max length 4 presently), with possibly
           parity dimension, and a grid-stride loop with max number of
           blocks = 2 x SM count

         - multi-reductions where we are reducing to a matrix of size
           of size QUDA_MAX_MULTI_REDUCE of vectors (max length 4), with
           possible parity dimension, and a grid-stride loop with
           maximum number of blocks = 2 x SM count
      */

      const int reduce_size = 4 * sizeof(device_reduce_t);
      const int max_reduce_blocks = 2*deviceProp.multiProcessorCount;

      const int max_reduce = 2 * max_reduce_blocks * reduce_size;
      const int max_multi_reduce = 2 * QUDA_MAX_MULTI_REDUCE * max_reduce_blocks * reduce_size;

      // reduction buffer size
      size_t bytes = max_reduce > max_multi_reduce ? max_reduce : max_multi_reduce;

      if (!d_reduce) d_reduce = (device_reduce_t *) device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be device_reduce_t x 3)

      // if the device supports host-mapped memory then use a host-mapped array for the reduction
      if (!h_reduce) {
	// only use zero copy reductions when using 64-bit
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
	if(deviceProp.canMapHostMemory) {
	  h_reduce = (device_reduce_t *) mapped_malloc(bytes);
	  hipHostGetDevicePointer(&hd_reduce, h_reduce, 0); // set the matching device pointer
	} else
#endif
	  {
	    h_reduce = (device_reduce_t *) pinned_malloc(bytes);
	    hd_reduce = d_reduce;
	  }
	memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
      }

      hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);

      // enable fast reductions with CPU spin waiting as opposed to using CUDA events
      char *fast_reduce_env = getenv("QUDA_ENABLE_FAST_REDUCE");
      if (fast_reduce_env && strcmp(fast_reduce_env,"1") == 0) {
        warningQuda("Experimental fast reductions enabled");
        fast_reduce_enabled = true;
      }

      checkCudaError();
    }

    void endReduce(void)
    {
      if (d_reduce) {
	device_free(d_reduce);
	d_reduce = 0;
      }
      if (h_reduce) {
	host_free(h_reduce);
	h_reduce = 0;
      }
      hd_reduce = 0;

      hipEventDestroy(reduceEnd);
    }

    /**
       Generic reduction kernel launcher
    */
    template <typename host_reduce_t, typename real, int len, typename Arg>
    auto reduceLaunch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream, Tunable &tunable)
    {
      using device_reduce_t = typename Arg::Reducer::reduce_t;
      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

      const int32_t words = tp.grid.y * sizeof(device_reduce_t) / sizeof(int32_t);
      if (getFastReduce() && !commAsyncReduction()) initFastReduce(words);

#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::reduceKernel")
                                  .instantiate((int)tp.block.x, Type<real>(), len, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
#else
      LAUNCH_KERNEL(reduceKernel, tunable, tp, stream, arg, real, len);
#endif

      if (!commAsyncReduction()) {
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
        if (deviceProp.canMapHostMemory) {
          if (getFastReduce()) {
            completeFastReduce(words);
          } else {
            qudaEventRecord(reduceEnd, stream);
            while (hipSuccess != qudaEventQuery(reduceEnd)) { ; }
          }
        } else
#endif
        {
          qudaMemcpy(h_reduce, hd_reduce, sizeof(device_reduce_t), hipMemcpyDeviceToHost);
        }
      }

      host_reduce_t cpu_sum = set(((device_reduce_t *)h_reduce)[0]);
      if (tp.grid.y == 2) sum(cpu_sum, ((device_reduce_t *)h_reduce)[1]); // add other parity if needed
      return cpu_sum;
    }

    template <template <typename ReducerType, typename real> class Reducer,
              typename store_t, typename y_store_t, int nSpin, typename coeff_t>
    class Reduce : public Tunable
    {
      using real = typename mapper<y_store_t>::type;
      using host_reduce_t = typename Reducer<double, real>::reduce_t;
      Reducer<device_reduce_t, real> r;
      const int nParity; // for composite fields this includes the number of composites
      host_reduce_t &result;

      const coeff_t &a, &b;
      ColorSpinorField &x, &y, &z, &w, &v;
      QudaFieldLocation location;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
            sharedBytesPerThread() * nthreads :
            sharedBytesPerBlock(param);
        return false;
      }

    public:
      Reduce(const coeff_t &a, const coeff_t &b, const coeff_t &c, ColorSpinorField &x, ColorSpinorField &y,
             ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v, host_reduce_t &result) :
        r(a, b),
        nParity((x.IsComposite() ? x.CompositeDim() : 1) * (x.SiteSubset())),
        a(a),
        b(b),
        x(x),
        y(y),
        z(z),
        w(w),
        v(v),
        result(result),
        location(checkLocation(x, y, z, w, v))
      {
        checkLength(x, y, z, w, v);
        auto x_prec = checkPrecision(x, z, w);
        auto y_prec = checkPrecision(y, v);
        auto x_order = checkOrder(x, z, w);
        auto y_order = checkOrder(y, v);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        strcpy(aux, x.AuxString());
        if (x.Precision() != z.Precision()) {
          strcat(aux, ",");
          strcat(aux, z.AuxString());
        }
        if (location == QUDA_CPU_FIELD_LOCATION) strcat(aux, ",CPU");
        else if (getFastReduce()) strcat(aux, ",fast_reduce");

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/reduce_core.cuh");
#endif

        apply(*(blas::getStream()));
        checkCudaError();

        blas::bytes += bytes();
        blas::flops += flops();

        const int Nreduce = sizeof(host_reduce_t) / sizeof(double);
        reduceDoubleArray((double *)&result, Nreduce);
      }

      TuneKey tuneKey() const { return TuneKey(x.VolString(), typeid(r).name(), aux); }

      void apply(const qudaStream_t &stream)
      {
        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value || decltype(r)::site_unroll;
        if (site_unroll_check && (x.Ncolor() != 3 || x.Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x.Nspin(), x.Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(x, y, z, w, v); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Reducer<device_reduce_t, device_real_t> r_(a, b);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value || decltype(r)::site_unroll;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x.Length() / (nParity * M);

          ReductionArg<device_store_t, N, device_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, v, r_, length, nParity);
          result = reduceLaunch<host_reduce_t, device_real_t, M>(arg, tp, stream, *this);
        } else {
          if (checkOrder(x, y, z, w, v) != QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
            warningQuda("CPU Blas functions expect AoS field order");
            return;
          }

          using host_store_t = typename host_type_mapper<store_t>::type;
          using host_y_store_t = typename host_type_mapper<y_store_t>::type;
          using host_real_t = typename mapper<host_y_store_t>::type;
          Reducer<double, host_real_t> r_(a, b);

          // redefine site_unroll with host_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<host_store_t, host_y_store_t>::value || isFixed<host_store_t>::value || decltype(r)::site_unroll;
          constexpr int N = n_vector<host_store_t, false, nSpin, site_unroll>();
          constexpr int Ny = n_vector<host_y_store_t, false, nSpin, site_unroll>();
          constexpr int M = N; // if site unrolling then M=N will be 24/6, e.g., full AoS
          const int length = x.Length() / (nParity * M);

          ReductionArg<host_store_t, N, host_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, v, r_, length, nParity);
          result = reduceCPU<host_real_t, M>(arg);
        }
      }

      void preTune()
      {
        if (r.write.X) x.backup();
        if (r.write.Y) y.backup();
        if (r.write.Z) z.backup();
        if (r.write.W) w.backup();
        if (r.write.V) v.backup();
      }

      void postTune()
      {
        if (r.write.X) x.restore();
        if (r.write.Y) y.restore();
        if (r.write.Z) z.restore();
        if (r.write.W) w.restore();
        if (r.write.V) v.restore();
      }

      bool advanceTuneParam(TuneParam &param) const
      {
        return location == QUDA_CPU_FIELD_LOCATION ? false : Tunable::advanceTuneParam(param);
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.grid.y = nParity;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
        param.grid.y = nParity;
      }

      long long flops() const { return r.flops() * x.Length(); }

      long long bytes() const
      {
        // the factor two here assumes we are reading and writing to the high precision vector
        // this will evaluate correctly for non-mixed kernels since the +2/-2 will cancel out
        return (r.streams() - 2) * x.Bytes() + 2 * z.Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <template <typename reduce_t, typename real> class Functor, bool mixed, typename... Args>
    auto instantiateReduce(Args &&... args)
    {
      using host_reduce_t = typename Functor<double, double>::reduce_t;
      host_reduce_t value;
      ::quda::zero(value); // no default constructor so we need to explicitly zero
      instantiate<Functor, Reduce, mixed>(args..., value);
      return value;
    }

    double norm1(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x); // FIXME
      return instantiateReduce<Norm1, false>(0.0, 0.0, 0.0, y, y, y, y, y);
    }

    double norm2(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x);
      return instantiateReduce<Norm2, false>(0.0, 0.0, 0.0, y, y, y, y, y);
    }

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<Dot, false>(0.0, 0.0, 0.0, x, y, x, x, x);
    }

    double axpbyzNorm(double a, ColorSpinorField &x, double b, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<axpbyzNorm2, false>(a, b, 0.0, x, y, z, x, x);
    }

    double axpyReDot(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<AxpyReDot, false>(a, 0.0, 0.0, x, y, x, x, x);
    }

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<caxpyNorm2, false>(a, Complex(0.0), Complex(0.0), x, y, x, x, x);
    }

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<caxpyxmaznormx, false>(a, Complex(0.0), Complex(0.0), x, y, z, x, x);
    }

    double cabxpyzAxNorm(double a, const Complex &b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<cabxpyzaxnorm, false>(Complex(a), b, Complex(0.0), x, y, z, x, x);
    }

    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      auto cdot = instantiateReduce<Cdot, false>(0.0, 0.0, 0.0, x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }

    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      auto cdot = instantiateReduce<caxpydotzy, false>(a, Complex(0.0), Complex(0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<CdotNormA, false>(0.0, 0.0, 0.0, x, y, x, x, x);
    }

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x, const Complex &b, ColorSpinorField &y,
                                           ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &u)
    {
      return instantiateReduce<caxpbypzYmbwcDotProductUYNormY_, true>(a, b, Complex(0.0), x, z, y, w, u);
    }

    Complex axpyCGNorm(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      double2 cg_norm = instantiateReduce<axpyCGNorm2, true>(a, 0.0, 0.0, x, y, x, x, x);
      return Complex(cg_norm.x, cg_norm.y);
    }

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor() != 3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = instantiateReduce<HeavyQuarkResidualNorm_, false>(0.0, 0.0, 0.0, x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = instantiateReduce<xpyHeavyQuarkResidualNorm_, false>(0.0, 0.0, 0.0, x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<tripleCGReduction_, false>(0.0, 0.0, 0.0, x, y, z, x, x);
    }

    double4 quadrupleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<quadrupleCGReduction_, false>(0.0, 0.0, 0.0, x, y, z, x, x);
    }

    double quadrupleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y,
                                ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      return instantiateReduce<quadrupleCG3InitNorm_, false>(a, 0.0, 0.0, x, y, z, w, v);
    }

    double quadrupleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y,
                                  ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      return instantiateReduce<quadrupleCG3UpdateNorm_, false>(a, b, 0.0, x, y, z, w, v);
    }

  } // namespace blas

} // namespace quda
