#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>

//#define QUAD_SUM
#ifdef QUAD_SUM
#include <dbldbl.h>
#endif

#include <cub_helper.cuh>

template<typename> struct ScalarType { };
template<> struct ScalarType<double> { typedef double type; };
template<> struct ScalarType<double2> { typedef double type; };
template<> struct ScalarType<double3> { typedef double type; };
template<> struct ScalarType<double4> { typedef double type; };

template<typename> struct Vec2Type { };
template<> struct Vec2Type<double> { typedef double2 type; };

#ifdef QUAD_SUM
#define QudaSumFloat doubledouble
#define QudaSumFloat2 doubledouble2
#define QudaSumFloat3 doubledouble3
template<> struct ScalarType<doubledouble> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble2> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble3> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble4> { typedef doubledouble type; };
template<> struct Vec2Type<doubledouble> { typedef doubledouble2 type; };
#else
#define QudaSumFloat double
#define QudaSumFloat2 double2
#define QudaSumFloat3 double3
#define QudaSumFloat4 double4
#endif


void checkSpinor(const ColorSpinorField &a, const ColorSpinorField &b) {
  if (a.Precision() != b.Precision())
    errorQuda("precisions do not match: %d %d", a.Precision(), b.Precision());
  if (a.Length() != b.Length())
    errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length());
  if (a.Stride() != b.Stride())
    errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());
}

void checkLength(const ColorSpinorField &a, ColorSpinorField &b) {									\
  if (a.Length() != b.Length())
    errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length());
  if (a.Stride() != b.Stride())
    errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());
}

static struct {
  const char *vol_str;
  const char *aux_str;
  char aux_tmp[quda::TuneKey::aux_n];
} blasStrings;

// These are used for reduction kernels
static QudaSumFloat *d_reduce=0;
static QudaSumFloat *h_reduce=0;
static QudaSumFloat *hd_reduce=0;
static hipEvent_t reduceEnd;
static bool fast_reduce_enabled = false;

namespace quda {
  namespace blas {

    hipStream_t* getStream();

    void* getDeviceReduceBuffer() { return d_reduce; }
    void* getMappedHostReduceBuffer() { return hd_reduce; }
    void* getHostReduceBuffer() { return h_reduce; }
    hipEvent_t* getReduceEvent() { return &reduceEnd; }
    bool getFastReduce() { return fast_reduce_enabled; }

    void initReduce()
    {
      /* we have these different reductions to cater for:

	 - regular reductions (reduce_quda.cu) where are reducing to a
           single vector type (max length 4 presently), with possibly
           parity dimension, and a grid-stride loop with max number of
           blocks = 2 x SM count
A.S. edit: extended to 16 for CA solvers
	 - multi-reductions where we are reducing to a matrix of size
	   of size MAX_MULTI_BLAS_N^2 of vectors (max length 4), with
	   possible parity dimension, and a grid-stride loop with
	   maximum number of blocks = 2 x SM count
      */

      const int reduce_size = 16 * sizeof(QudaSumFloat); //A.S. extended from 4 to 16 for CA solvers
      const int max_reduce_blocks = 2*deviceProp.multiProcessorCount;

      const int max_reduce = 2 * max_reduce_blocks * reduce_size;
      const int max_multi_reduce = 2 * MAX_MULTI_BLAS_N * MAX_MULTI_BLAS_N * max_reduce_blocks * 4 * sizeof(QudaSumFloat);

      // reduction buffer size
      size_t bytes = max_reduce > max_multi_reduce ? max_reduce : max_multi_reduce;

      if (!d_reduce) d_reduce = (QudaSumFloat *) device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be QudaSumFloat3)

      // if the device supports host-mapped memory then use a host-mapped array for the reduction
      if (!h_reduce) {
	// only use zero copy reductions when using 64-bit
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
	if(deviceProp.canMapHostMemory) {
	  h_reduce = (QudaSumFloat *) mapped_malloc(bytes);
	  hipHostGetDevicePointer(&hd_reduce, h_reduce, 0); // set the matching device pointer
	} else
#endif
	  {
	    h_reduce = (QudaSumFloat *) pinned_malloc(bytes);
	    hd_reduce = d_reduce;
	  }
	memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
      }

      hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);

      // enable fast reductions with CPU spin waiting as opposed to using CUDA events
      char *fast_reduce_env = getenv("QUDA_ENABLE_FAST_REDUCE");
      if (fast_reduce_env && strcmp(fast_reduce_env,"1") == 0) {
        warningQuda("Experimental fast reductions enabled");
        fast_reduce_enabled = true;
      }

      checkCudaError();
    }

    void endReduce(void)
    {
      if (d_reduce) {
	device_free(d_reduce);
	d_reduce = 0;
      }
      if (h_reduce) {
	host_free(h_reduce);
	h_reduce = 0;
      }
      hd_reduce = 0;

      hipEventDestroy(reduceEnd);
    }

    namespace reduce {

#include <texture.h>
#include <reduce_core.cuh>
#include <reduce_core.h>
#include <reduce_mixed_core.h>
#include <exp_reduce_core.h>

    } // namespace reduce

    /**
       Base class from which all reduction functors should derive.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct ReduceFunctor {

      //! pre-computation routine called before the "M-loop"
      virtual __device__ __host__ void pre() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y,
							   FloatN &z, FloatN &w, FloatN &v) = 0;

      //! post-computation routine called after the "M-loop"
      virtual __device__ __host__ void post(ReduceType &sum) { ; }

    };

    /**
       Return the L1 norm of x
    */
    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const double2 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y);
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const float2 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y);
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const float4 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y) + (ReduceType)fabs(a.z) + (ReduceType)fabs(a.w);
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct Norm1 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Norm1(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v)
      { sum += norm1_<ReduceType>(x); }
      static int streams() { return 1; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double norm1(const ColorSpinorField &x) {
      ColorSpinorField &y = const_cast<ColorSpinorField&>(x); // FIXME
      return reduce::reduceCuda<double,QudaSumFloat,Norm1,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
    }

    /**
       Return the L2 norm of x
    */
    template<typename ReduceType> __device__ __host__ void norm2_(ReduceType &sum, const double2 &a) {
      sum += (ReduceType)a.x*(ReduceType)a.x;
      sum += (ReduceType)a.y*(ReduceType)a.y;
    }

    template<typename ReduceType> __device__ __host__ void norm2_(ReduceType &sum, const float2 &a) {
      sum += (ReduceType)a.x*(ReduceType)a.x;
      sum += (ReduceType)a.y*(ReduceType)a.y;
    }

    template<typename ReduceType> __device__ __host__ void norm2_(ReduceType &sum, const float4 &a) {
      sum += (ReduceType)a.x*(ReduceType)a.x;
      sum += (ReduceType)a.y*(ReduceType)a.y;
      sum += (ReduceType)a.z*(ReduceType)a.z;
      sum += (ReduceType)a.w*(ReduceType)a.w;
    }


    template <typename ReduceType, typename Float2, typename FloatN>
      struct Norm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Norm2(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v)
      { norm2_<ReduceType>(sum,x); }
      static int streams() { return 1; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double norm2(const ColorSpinorField &x) {
      ColorSpinorField &y = const_cast<ColorSpinorField&>(x);
      return reduce::reduceCuda<double,QudaSumFloat,Norm2,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
    }


    /**
       Return the real dot product of x and y
    */
    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const double2 &a, const double2 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const float2 &a, const float2 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const float4 &a, const float4 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
      sum += (ReduceType)a.z*(ReduceType)b.z;
      sum += (ReduceType)a.w*(ReduceType)b.w;
    }

   template <typename ReduceType, typename Float2, typename FloatN>
    struct Dot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Dot(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
     { dot_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,Dot,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First performs the operation z[i] = a*x[i] + b*y[i]
       Return the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct axpbyzNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      axpbyzNorm2(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	z = a.x*x + b.x*y; norm2_<ReduceType>(sum,z); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    double axpbyzNorm(double a, ColorSpinorField &x, double b, ColorSpinorField &y,
                      ColorSpinorField &z) {
      return reduce::reduceCuda<double,QudaSumFloat,axpbyzNorm2,0,0,1,0,0,false>
	(make_double2(a, 0.0), make_double2(b, 0.0), x, y, z, x, x);
    }


    /**
       First performs the operation y[i] += a*x[i]
       Return real dot product (x,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct AxpyReDot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      AxpyReDot(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	y += a.x*x; dot_<ReduceType>(sum,x,y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    double axpyReDot(double a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,AxpyReDot,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       Functor to perform the operation y += a * x  (complex-valued)
    */
    __device__ __host__ void Caxpy_(const double2 &a, const double2 &x, double2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }
    __device__ __host__ void Caxpy_(const float2 &a, const float2 &x, float2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }
    __device__ __host__ void Caxpy_(const float2 &a, const float4 &x, float4 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
      y.z += a.x*x.z; y.z -= a.y*x.w;
      y.w += a.y*x.z; y.w += a.x*x.w;
    }

    /**
       First performs the operation y[i] = a*x[i] + y[i] (complex-valued)
       Second returns the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpyNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	Caxpy_(a, x, y); norm2_<ReduceType>(sum,y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,caxpyNorm2,0,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       double caxpyXmayNormCuda(float a, float *x, float *y, n){}
       First performs the operation y[i] = a*x[i] + y[i]
       Second performs the operator x[i] -= a*z[i]
       Third returns the norm of x
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpyxmaznormx : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpyxmaznormx(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { Caxpy_(a, x, y); Caxpy_(-a,z,x); norm2_<ReduceType>(sum,x); }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 10; } //! flops per element
    };

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x,
			  ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double,QudaSumFloat,caxpyxmaznormx,1,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
    }


    /**
       double cabxpyzAxNorm(float a, complex b, float *x, float *y, float *z){}
       First performs the operation z[i] = y[i] + a*b*x[i]
       Second performs x[i] *= a
       Third returns the norm of x
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct cabxpyzaxnorm : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      cabxpyzaxnorm(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { x *= a.x; Caxpy_(b, x, y); z = y; norm2_<ReduceType>(sum,z); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 10; } //! flops per element
    };


    double cabxpyzAxNorm(double a, const Complex &b,
			ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double,QudaSumFloat,cabxpyzaxnorm,1,0,1,0,0,false>
	(make_double2(a, 0.0), make_double2(REAL(b), IMAG(b)), x, y, z, x, x);
    }


    /**
       Returns complex-valued dot product of x and y
    */
    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const double2 &a, const double2 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
    }

    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const float2 &a, const float2 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
    }

    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const float4 &a, const float4 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.x += (scalar)a.z*(scalar)b.z;
      sum.x += (scalar)a.w*(scalar)b.w;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
      sum.y += (scalar)a.z*(scalar)b.w;
      sum.y -= (scalar)a.w*(scalar)b.z;
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct Cdot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Cdot(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { cdot_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,Cdot,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }


    /**
       double caxpyDotzyCuda(float a, float *x, float *y, float *z, n){}
       First performs the operation y[i] = a*x[i] + y[i]
       Second returns the dot product (z,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpydotzy : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpydotzy(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { Caxpy_(a, x, y); cdot_<ReduceType>(sum,z,y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,caxpydotzy,0,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }


    /**
       First returns the dot product (x,y)
       Returns the norm of x
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ void cdotNormA_(ReduceType &sum, const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      typedef typename Vec2Type<scalar>::type vec2;
      cdot_<ReduceType>(sum,a,b);
      norm2_<scalar>(sum.z,a);
    }

    /**
       First returns the dot product (x,y)
       Returns the norm of y
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ void cdotNormB_(ReduceType &sum, const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      typedef typename Vec2Type<scalar>::type vec2;
      cdot_<ReduceType>(sum,a,b);
      norm2_<scalar>(sum.z,b);
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct CdotNormA : public ReduceFunctor<ReduceType, Float2, FloatN> {
      CdotNormA(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { cdotNormA_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double3,QudaSumFloat3,CdotNormA,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       This convoluted kernel does the following:
       z += a*x + b*y, y -= b*w, norm = (y,y), dot = (u, y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpbypzYmbwcDotProductUYNormY_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      caxpbypzYmbwcDotProductUYNormY_(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { Caxpy_(a, x, z); Caxpy_(b, y, z); Caxpy_(-b, w, y); cdotNormB_<ReduceType>(sum,v,y); }
      static int streams() { return 7; } //! total number of input and output streams
      static int flops() { return 18; } //! flops per element
    };

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x,
					   const Complex &b, ColorSpinorField &y,
					   ColorSpinorField &z, ColorSpinorField &w,
					   ColorSpinorField &u) {
      if (x.Precision() != z.Precision()) {
	return reduce::mixed::reduceCuda<double3,QudaSumFloat3,caxpbypzYmbwcDotProductUYNormY_,0,1,1,0,0,false>
	  (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      } else {
	return reduce::reduceCuda<double3,QudaSumFloat3,caxpbypzYmbwcDotProductUYNormY_,0,1,1,0,0,false>
	  (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      }
    }


    /**
       Specialized kernel for the modified CG norm computation for
       computing beta.  Computes y = y + a*x and returns norm(y) and
       dot(y, delta(y)) where delta(y) is the difference between the
       input and out y vector.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct axpyCGNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      axpyCGNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	FloatN z_new = z + a.x*x;
	norm2_<scalar>(sum.x,z_new);
	dot_<scalar>(sum.y,z_new,z_new-z);
	z = z_new;
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per real element
    };

    Complex axpyCGNorm(double a, ColorSpinorField &x, ColorSpinorField &y) {
      // swizzle since mixed is on z
      double2 cg_norm ;
      if (x.Precision() != y.Precision()) {
	cg_norm = reduce::mixed::reduceCuda<double2,QudaSumFloat2,axpyCGNorm2,0,0,1,0,0,false>
	  (make_double2(a, 0.0), make_double2(0.0, 0.0), x, x, y, x, x);
      } else {
	cg_norm = reduce::reduceCuda<double2,QudaSumFloat2,axpyCGNorm2,0,0,1,0,0,false>
	  (make_double2(a, 0.0), make_double2(0.0, 0.0), x, x, y, x, x);
      }
      return Complex(cg_norm.x, cg_norm.y);
    }


    /**
       This kernel returns (x, x) and (r,r) and also returns the so-called
       heavy quark norm as used by MILC: 1 / N * \sum_i (r, r)_i / (x, x)_i, where
       i is site index and N is the number of sites.
       When this kernel is launched, we must enforce that the parameter M
       in the launcher corresponds to the number of FloatN fields used to
       represent the spinor, e.g., M=6 for Wilson and M=3 for staggered.
       This is only the case for half-precision kernels by default.  To
       enable this, the siteUnroll template parameter must be set true
       when reduceCuda is instantiated.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct HeavyQuarkResidualNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      typedef typename scalar<ReduceType>::type real;
      Float2 a;
      Float2 b;
      ReduceType aux;
      HeavyQuarkResidualNorm_(const Float2 &a, const Float2 &b) : a(a), b(b), aux{ } { ; }

      __device__ __host__ void pre() { aux.x = 0; aux.y = 0; }

      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	norm2_<real>(aux.x,x); norm2_<real>(aux.y,y);
      }

      //! sum the solution and residual norms, and compute the heavy-quark norm
      __device__ __host__ void post(ReduceType &sum)
      {
	sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : static_cast<real>(1.0);
      }

      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! undercounts since it excludes the per-site division
    };

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r) {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,HeavyQuarkResidualNorm_,0,0,0,0,0,true>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }


    /**
      Variant of the HeavyQuarkResidualNorm kernel: this takes three
      arguments, the first two are summed together to form the
      solution, with the third being the residual vector.  This removes
      the need an additional xpy call in the solvers, impriving
      performance.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct xpyHeavyQuarkResidualNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
	typedef typename scalar<ReduceType>::type real;
      Float2 a;
      Float2 b;
      ReduceType aux;
      xpyHeavyQuarkResidualNorm_(const Float2 &a, const Float2 &b) : a(a), b(b), aux{ } { ; }

      __device__ __host__ void pre() { aux.x = 0; aux.y = 0; }

      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	norm2_<real>(aux.x,x + y); norm2_<real>(aux.y,z);
      }

      //! sum the solution and residual norms, and compute the heavy-quark norm
      __device__ __host__ void post(ReduceType &sum)
      {
	sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : static_cast<real>(1.0);
      }

      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 5; }
    };

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y,
				      ColorSpinorField &r) {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,xpyHeavyQuarkResidualNorm_,0,0,0,0,0,true>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    /**
       double3 tripleCGReduction(V x, V y, V z){}
       First performs the operation norm2(x)
       Second performs the operatio norm2(y)
       Third performs the operation dotPropduct(y,z)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct tripleCGReduction_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      tripleCGReduction_(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	norm2_<scalar>(sum.x,x); norm2_<scalar>(sum.y,y); dot_<scalar>(sum.z,y,z);
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double3,QudaSumFloat3,tripleCGReduction_,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    }

    /**
       double4 quadrupleCGReduction(V x, V y, V z){}
       First performs the operation norm2(x)
       Second performs the operatio norm2(y)
       Third performs the operation dotPropduct(y,z)
       Fourth performs the operation norm(z)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct quadrupleCGReduction_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      quadrupleCGReduction_(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
        typedef typename ScalarType<ReduceType>::type scalar;
        norm2_<scalar>(sum.x,x); norm2_<scalar>(sum.y,y); dot_<scalar>(sum.z,y,z); norm2_<scalar>(sum.w,w);
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    double4 quadrupleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double4,QudaSumFloat4,quadrupleCGReduction_,0,0,0,0,0,false>
        (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    }

    /**
       double quadrupleCG3InitNorm(d a, d b, V x, V y, V z, V w, V v){}
        z = x;
        w = y;
        x += a*y;
        y -= a*v;
        norm2(y);
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct quadrupleCG3InitNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      quadrupleCG3InitNorm_(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
        z = x;
        w = y;
        x += a.x*y;
        y -= a.x*v;
        norm2_<ReduceType>(sum,y);
      }
      static int streams() { return 6; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element check if it's right
    };

    double quadrupleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v) {
      return reduce::reduceCuda<double,QudaSumFloat,quadrupleCG3InitNorm_,1,1,1,1,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, z, w, v);
    }


    /**
       double quadrupleCG3UpdateNorm(d gamma, d rho, V x, V y, V z, V w, V v){}
        tmpx = x;
        tmpy = y;
        x = b*(x + a*y) + (1-b)*z;
        y = b*(y + a*v) + (1-b)*w;
        z = tmpx;
        w = tmpy;
        norm2(y);
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct quadrupleCG3UpdateNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a,b;
      quadrupleCG3UpdateNorm_(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      FloatN tmpx{}, tmpy{};
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
        tmpx = x;
        tmpy = y;
        x = b.x*(x + a.x*y) + b.y*z;
        y = b.x*(y - a.x*v) + b.y*w;
        z = tmpx;
        w = tmpy;
        norm2_<ReduceType>(sum,y);
      }
      static int streams() { return 7; } //! total number of input and output streams
      static int flops() { return 16; } //! flops per element check if it's right
    };

    double quadrupleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v) {
      return reduce::reduceCuda<double,QudaSumFloat,quadrupleCG3UpdateNorm_,1,1,1,1,0,false>
	(make_double2(a, 0.0), make_double2(b, 1.-b), x, y, z, w, v);
    }

    /**
       void doubleCG3InitNorm(d a, V x, V y, V z){}
        y = x;
        x -= a*z;
        norm2(x);
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct doubleCG3InitNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      doubleCG3InitNorm_(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
        y = x;
        x -= a.x*z;
        norm2_<ReduceType>(sum,x);
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    double doubleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double,QudaSumFloat,doubleCG3InitNorm_,1,1,0,0,0,false>
        (make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, z, z, z);
    }

    /**
       void doubleCG3UpdateNorm(d a, d b, V x, V y, V z){}
        tmp = x;
        x = b*(x-a*z) + (1-b)*y;
        y = tmp;
        norm2(x);
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct doubleCG3UpdateNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a, b;
      doubleCG3UpdateNorm_(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      FloatN tmp{};
      __device__ __host__ void operator()(ReduceType &sum,FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
        tmp = x;
        x = b.x*(x-a.x*z) + b.y*y;
        y = tmp;
        norm2_<ReduceType>(sum,x);
      }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 9; } //! flops per element
    };

    double doubleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double,QudaSumFloat,doubleCG3UpdateNorm_,1,1,0,0,0,false>
        (make_double2(a, 0.0), make_double2(b, 1.0-b), x, y, z, z, z);
    }


/*
    Reduction routines for a number of pipelined methods
*/


    template<typename ReduceType>
    __device__ __host__ void hp_axpby_reduce(ReduceType &sum, const double &a, const double &b, double2 &x, double2 &p, double2 &u, double2 &r, double2 &s, double2 &m, double2 &q, double2 &w, double2 &n, double2 &z){

#if defined( __CUDA_ARCH__)

      //the first component
      //z = n + b*z;
      z.x  = __fma_rn (b, z.x, n.x);
      //s = w + b*s;
      s.x  = __fma_rn (b, s.x, w.x);
      //q = m + b*q;
      //q.x  = __fma_rn (b, q.x, m.x);
      q.x = s.x;
      //p = u/r + b*p;
      p.x  = __fma_rn (b, p.x, u.x);

      //x = x + a*p;
      x.x  = __fma_rn (a, p.x, x.x);
      //r = r - a*s;
      r.x  = __fma_rn (-a, s.x, r.x);
      //u = u - a*q;
      //u.x  = __fma_rn (-a, q.x, u.x);
      u.x  = r.x;
      //w = w - a*z;
      w.x  = __fma_rn (-a, z.x, w.x);

      //the second component:
      //z = n + b*z;
      z.y  = __fma_rn (b, z.y, n.y);
      //s = w + b*s;
      s.y  = __fma_rn (b, s.y, w.y);
      //q = m + b*q;
      //q.y  = __fma_rn (b, q.y, m.y);
      q.y = s.y;
      //p = u/r + b*p;
      p.y  = __fma_rn (b, p.y, u.y);

      //x = x + a*p;
      x.y  = __fma_rn (a, p.y, x.y);
      //r = r - a*s;
      r.y  = __fma_rn (-a, s.y, r.y);
      //u = u - a*q;
      //u.y  = __fma_rn (-a, q.y, u.y);
      u.y  = r.y;
      //w = w - a*z;
      w.y  = __fma_rn (-a, z.y, w.y);

      double sum_0_x = (double) sum.x;
      double sum_0_y = (double) sum.y;
      double sum_0_z = (double) sum.z;

      sum_0_x  = __fma_rn (r.x, u.x, sum_0_x);
      sum_0_x  = __fma_rn (r.y, u.y, sum_0_x);

      sum_0_y  = __fma_rn (w.x, u.x, sum_0_y);
      sum_0_y  = __fma_rn (w.y, u.y, sum_0_y);

      sum_0_z  = __fma_rn (r.x, r.x, sum_0_z);
      sum_0_z  = __fma_rn (r.y, r.y, sum_0_z);

      sum.x = sum_0_x;
      sum.y = sum_0_y;
      sum.z = sum_0_z;
      sum.w = 0.0;
#else
//cpu code
#endif

      return;
    }


    template<typename ReduceType>
    __device__ __host__  void hp_axpby_reduce(ReduceType &sum, const double &a, const double &b, float2 &x, float2 &p, float2 &u, float2 &r, float2 &s, float2 &m, float2 &q, float2 &w, float2 &n, float2 &z){

#if defined( __CUDA_ARCH__)
      double s_, p_, z_, r_x, r_y, x_, w_x, w_y, q_, u_x, u_y, m_, n_;
      //the first component
      //z = n + b*z;
      z_ = z.x, n_ = n.x;
      z_ = __fma_rn (b, z_, n_);
      z.x= z_;
      //s = w + b*s;
      s_ = s.x, w_x = w.x;
      s_ = __fma_rn (b, s_, w_x);
      s.x= s_;
      //q = m + b*q;
      //q_ = q.x, m_ = m.x;
      //q_ = __fma_rn (b, q_, m_);
      //q.x = q_;
      q.x = s.x;
      //p = u/r + b*p;
      p_ = p.x, u_x = u.x;
      p_ = __fma_rn (b, p_, u_x);
      p.x = p_;

      //x = x + a*p;
      x_ = x.x;
      x_ = __fma_rn (a, p_, x_);
      x.x = x_;
      //r = r - a*s;
      r_x = r.x;
      r_x = __fma_rn (-a, s_, r_x);
      r.x = r_x;
      //u = u - a*q;
      //u_x = __fma_rn (-a, q_, u_x);
      //u.x = u_x;
      u_x = r_x;
      u.x = r.x;
      //w = w - a*z;
      w_x = __fma_rn (-a, z_, w_x);
      w.x = w_x;

      //the second component:
      //z = n + b*z;
      z_ = z.y, n_ = n.y;
      z_ = __fma_rn (b, z_, n_);
      z.y= z_;
      //s = w + b*s;
      s_ = s.y, w_y = w.y;
      s_ = __fma_rn (b, s_, w_y);
      s.y= s_;
      //q = m + b*q;
      //q_ = q.y, m_ = m.y;
      //q_ = __fma_rn (b, q_, m_);
      //q.y = q_;
      q.y = s.y;
      //p = u/r + b*p;
      p_ = p.y, u_y = u.y;
      p_ = __fma_rn (b, p_, u_x);
      p.y = p_;

      //x = x + a*p;
      x_ = x.y;
      x_ = __fma_rn (a, p_, x_);
      x.y = x_;
      //r = r - a*s;
      r_y = r.y;
      r_y = __fma_rn (-a, s_, r_y);
      r.y = r_y;
      //u = u - a*q;
      //u_y = __fma_rn (-a, q_, u_y);
      //u.y = u_y;
      u_y = r_y;
      u.y  = r.y;
      //w = w - a*z;
      w_y = __fma_rn (-a, z_, w_y);
      w.y = w_y;

      double sum_0_x = (double) sum.x;
      double sum_0_y = (double) sum.y;
      double sum_0_z = (double) sum.z;

      sum_0_x  = __fma_rn (r_x, u_x, sum_0_x);
      sum_0_x  = __fma_rn (r_y, u_y, sum_0_x);

      sum_0_y  = __fma_rn (w_x, u_x, sum_0_y);
      sum_0_y  = __fma_rn (w_y, u_y, sum_0_y);

      sum_0_z  = __fma_rn (r_x, r_x, sum_0_z);
      sum_0_z  = __fma_rn (r_y, r_y, sum_0_z);

      sum.x = sum_0_x;
      sum.y = sum_0_y;
      sum.z = sum_0_z;
      sum.w = 0.0;
#else
//cpu code
#endif

      return;
    }

    template<typename ReduceType>
    __device__ __host__ void hp_axpby_reduce(ReduceType &sum, const double &a, const double &b, float4 &x, float4 &p, float4 &u, float4 &r, float4 &s, float4 &m, float4 &q, float4 &w, float4 &n, float4 &z){

#if defined( __CUDA_ARCH__)
      double s_, p_, z_, r_x, r_y, r_z, r_w, x_, w_x, w_y, w_z, w_w, q_, u_x, u_y, u_z, u_w, m_, n_;
      //the first component
      //z = n + b*z;
      z_ = z.x, n_ = n.x;
      z_ = __fma_rn (b, z_, n_);
      z.x= z_;
      //s = w + b*s;
      s_ = s.x, w_x = w.x;
      s_ = __fma_rn (b, s_, w_x);
      s.x= s_;
      //q = m + b*q;
      //q_ = q.x, m_ = m.x;
      //q_ = __fma_rn (b, q_, m_);
      //q.x = q_;
      q.x = s.x;
      //p = u/r + b*p;
      p_ = p.x, u_x = u.x;
      p_ = __fma_rn (b, p_, u_x);
      p.x = p_;

      //x = x + a*p;
      x_ = x.x;
      x_ = __fma_rn (a, p_, x_);
      x.x = x_;
      //r = r - a*s;
      r_x = r.x;
      r_x = __fma_rn (-a, s_, r_x);
      r.x = r_x;
      //u = u - a*q;
      //u_x = __fma_rn (-a, q_, u_x);
      //u.x = u_x;
      u_x = r_x;
      u.x = r.x;
      //w = w - a*z;
      w_x = __fma_rn (-a, z_, w_x);
      w.x = w_x;

      //the second component:
      //z = n + b*z;
      z_ = z.y, n_ = n.y;
      z_ = __fma_rn (b, z_, n_);
      z.y= z_;
      //s = w + b*s;
      s_ = s.y, w_y = w.y;
      s_ = __fma_rn (b, s_, w_y);
      s.y= s_;
      //q = m + b*q;
      //q_ = q.y, m_ = m.y;
      //q_ = __fma_rn (b, q_, m_);
      //q.y = q_;
      q.y = s.y;
      //p = u/r + b*p;
      p_ = p.y, u_y = u.y;
      p_ = __fma_rn (b, p_, u_x);
      p.y = p_;

      //x = x + a*p;
      x_ = x.y;
      x_ = __fma_rn (a, p_, x_);
      x.y = x_;
      //r = r - a*s;
      r_y = r.y;
      r_y = __fma_rn (-a, s_, r_y);
      r.y = r_y;
      //u = u - a*q;
      //u_y = __fma_rn (-a, q_, u_y);
      //u.y = u_y;
      u_y = r_y;
      u.y  = r.y;
      //w = w - a*z;
      w_y = __fma_rn (-a, z_, w_y);
      w.y = w_y;

      //the third component:
      //z = n + b*z;
      z_ = z.z, n_ = n.z;
      z_ = __fma_rn (b, z_, n_);
      z.z= z_;
      //s = w + b*s;
      s_ = s.z, w_z = w.z;
      s_ = __fma_rn (b, s_, w_z);
      s.z= s_;
      //q = m + b*q;
      //q_ = q.z, m_ = m.z;
      //q_ = __fma_rn (b, q_, m_);
      //q.z = q_;
      q.z = s.z;
      //p = u/r + b*p;
      p_ = p.z, u_z = u.z;
      p_ = __fma_rn (b, p_, u_z);
      p.z = p_;

      //x = x + a*p;
      x_ = x.z;
      x_ = __fma_rn (a, p_, x_);
      x.z = x_;
      //r = r - a*s;
      r_z = r.z;
      r_z = __fma_rn (-a, s_, r_z);
      r.z = r_z;
      //u = u - a*q;
      //u_z = __fma_rn (-a, q_, u_z);
      //u.z = u_z;
      u_z = r_z;
      u.z  = r.z;
      //w = w - a*z;
      w_z = __fma_rn (-a, z_, w_z);
      w.z = w_z;

      //the fourth component:
      //z = n + b*z;
      z_ = z.w, n_ = n.w;
      z_ = __fma_rn (b, z_, n_);
      z.w= z_;
      //s = w + b*s;
      s_ = s.w, w_w = w.w;
      s_ = __fma_rn (b, s_, w_w);
      s.w= s_;
      //q = m + b*q;
      //q_ = q.w, m_ = m.w;
      //q_ = __fma_rn (b, q_, m_);
      //q.w = q_;
      q.w = s.w;
      //p = u/r + b*p;
      p_ = p.w, u_w = u.w;
      p_ = __fma_rn (b, p_, u_w);
      p.w = p_;

      //x = x + a*p;
      x_ = x.w;
      x_ = __fma_rn (a, p_, x_);
      x.w = x_;
      //r = r - a*s;
      r_w = r.w;
      r_w = __fma_rn (-a, s_, r_w);
      r.w = r_w;
      //u = u - a*q;
      //u_w = __fma_rn (-a, q_, u_w);
      //u.w = u_w;
      u_w = r_w;
      u.w  = r.w;
      //w = w - a*z;
      w_w = __fma_rn (-a, z_, w_w);
      w.w = w_w;



      double sum_0_x = (double) sum.x;
      double sum_0_y = (double) sum.y;
      double sum_0_z = (double) sum.z;

      sum_0_x  = __fma_rn (r_x, u_x, sum_0_x);
      sum_0_x  = __fma_rn (r_y, u_y, sum_0_x);
      sum_0_x  = __fma_rn (r_z, u_z, sum_0_x);
      sum_0_x  = __fma_rn (r_w, u_w, sum_0_x);

      sum_0_y  = __fma_rn (w_x, u_x, sum_0_y);
      sum_0_y  = __fma_rn (w_y, u_y, sum_0_y);
      sum_0_y  = __fma_rn (w_z, u_z, sum_0_y);
      sum_0_y  = __fma_rn (w_w, u_w, sum_0_y);

      sum_0_z  = __fma_rn (r_x, r_x, sum_0_z);
      sum_0_z  = __fma_rn (r_y, r_y, sum_0_z);
      sum_0_z  = __fma_rn (r_z, r_z, sum_0_z);
      sum_0_z  = __fma_rn (r_w, r_w, sum_0_z);

      sum.x = sum_0_x;
      sum.y = sum_0_y;
      sum.z = sum_0_z;
      sum.w = 0.0;
#else
//cpu code
#endif

      return;
    }

///for the preconditioned system:
//     y = x + a*y;
//     z = z + b*y;


    __device__ __host__ void hp_xpaybz(const double &a, const double2 &x, double2 &y, const double &b, double2 &z){

      _fma2(a, y, x);
      _fma3(b, y, z);

    }

    __device__ __host__ void hp_xpaybz(const double &a, const float2 &x, float2 &y, const double &b, float2 &z){

      double2 x_ = make_FloatN(x);
      double2 y_ = make_FloatN(y);
      double2 z_ = make_FloatN(z);

      _fma2(a, y_, x_);
      _fma3(b, y_, z_);

      y = make_FloatN(y_);
      z = make_FloatN(z_);

    }


    __device__ __host__ void hp_xpaybz(const double &a, const float4 &x, float4 &y, const double &b, float4 &z){

      double4 x_ = make_FloatN(x);
      double4 y_ = make_FloatN(y);
      double4 z_ = make_FloatN(z);

      _fma2(a, y_, x_);
      _fma3(b, y_, z_);

      y = make_FloatN(y_);
      z = make_FloatN(z_);

    }


//q = m + a*q;
//u = u + b*q;

//(r,u)

//s = w + a*s;
//r = r + b*s;

//z = n + a*z;
//w = w + b*z;

// (r, u);
// (w, u);
// norm2_(r);

    __device__ __host__ void dot_fma(double &sum, const double2 &a, const double2 &b) {
#if defined( __CUDA_ARCH__)
      sum  = __fma_rn (a.x, b.x, sum);
      sum  = __fma_rn (a.y, b.y, sum);
#else
      sum += a.x*b.x;
      sum += a.y*b.y;
#endif
    }

    __device__ __host__ void dot_fma(double &sum, const double4 &a, const double4 &b) {
#if defined( __CUDA_ARCH__)
      sum  = __fma_rn (a.x, b.x, sum);
      sum  = __fma_rn (a.y, b.y, sum);
      sum  = __fma_rn (a.z, b.z, sum);
      sum  = __fma_rn (a.w, b.w, sum);
#else
      sum += a.x*b.x;
      sum += a.y*b.y;
      sum += a.z*b.z;
      sum += a.w*b.w;
#endif
    }


    template<typename ReduceType>
    __device__ __host__ void hp_xpaybz_combo_reduce(ReduceType &sum, const double &a, const double2 &m, double2 &q, const double &b, double2 &u, double2 &w, double2 &s, double2 &r, const double2 &n, double2 &z){

      _fma2(a, q, m);
      _fma3(b, q, u);

      double& sum_w = static_cast<double&> (sum.w);

      dot_fma (sum_w, r, u);

      _fma2(a, s, w);
      _fma3(b, s, r);

      double& sum_x = static_cast<double&> (sum.x);
      dot_fma ( sum_x, r, u);
      double& sum_z = static_cast<double&> (sum.z);
      dot_fma (sum_z, r, r);

      _fma2(a, z, n);
      _fma3(b, z, w);
      double& sum_y = static_cast<double&> (sum.y);
      dot_fma (sum_y, w, u);
    }

    template<typename ReduceType>
    __device__ __host__ void hp_xpaybz_combo_reduce(ReduceType &sum, const double &a, const float2 &m, float2 &q, const double &b, float2 &u, float2 &w, float2 &s, float2 &r, const float2 &n, float2 &z){

      double2 x_ = make_FloatN(q);
      double2 y_ = make_FloatN(m);
      double2 u_ = make_FloatN(u);
      double2 w_ = make_FloatN(w);

      _fma2(a, x_, y_);
      _fma3(b, x_, u_);

      q = make_FloatN(x_);
      u = make_FloatN(u_);

      x_ = make_FloatN(s);
      y_ = make_FloatN(r);

      double& sum_w = static_cast<double&> (sum.w);
      dot_fma (sum_w, y_, u_);

      _fma2(a, x_, w_);
      _fma3(b, x_, y_);

      s = make_FloatN(x_);
      r = make_FloatN(y_);

      double& sum_x = static_cast<double&> (sum.x);
      dot_fma ( sum_x, y_, u_);
      double& sum_z = static_cast<double&> (sum.z);
      dot_fma (sum_z, y_, y_);

      x_ = make_FloatN(n);
      y_ = make_FloatN(z);

      _fma2(a, y_, x_);
      _fma3(b, y_, w_);

      z = make_FloatN(y_);
      w = make_FloatN(w_);

      double& sum_y = static_cast<double&> (sum.y);
      dot_fma (sum_y, w_, u_);

    }


    template<typename ReduceType>
    __device__ __host__ void hp_xpaybz_combo_reduce(ReduceType &sum, const double &a, const float4 &m, float4 &q, const double &b, float4 &u, float4 &w, float4 &s, float4 &r, const float4 &n, float4 &z){

      double4 x_ = make_FloatN(q);
      double4 y_ = make_FloatN(m);
      double4 u_ = make_FloatN(u);
      double4 w_ = make_FloatN(w);

      _fma2(a, x_, y_);
      _fma3(b, x_, u_);

      q = make_FloatN(x_);
      u = make_FloatN(u_);

      x_ = make_FloatN(s);
      y_ = make_FloatN(r);

      double& sum_w = static_cast<double&> (sum.w);
      dot_fma (sum_w, y_, u_);

      _fma2(a, x_, w_);
      _fma3(b, x_, y_);

      s = make_FloatN(x_);
      r = make_FloatN(y_);

      double& sum_x = static_cast<double&> (sum.x);
      dot_fma ( sum_x, y_, u_);
      double& sum_z = static_cast<double&> (sum.z);
      dot_fma (sum_z, y_, y_);

      x_ = make_FloatN(n);
      y_ = make_FloatN(z);

      _fma2(a, y_, x_);
      _fma3(b, y_, w_);

      z = make_FloatN(y_);
      w = make_FloatN(w_);

      double& sum_y = static_cast<double&> (sum.y);
      dot_fma (sum_y, w_, u_);

    }



//////////////////////////////////////////////////////////////////////////////////////////
//#define ERROR_CONTROL

    template <int Nreduce, typename ReduceType, typename Float2, typename FloatN>
    struct ReduceFunctorExp {

      //! pre-computation routine called before the "M-loop"
      virtual __device__ __host__ void pre() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x, FloatN &p, FloatN &u,FloatN &r,
						  FloatN &s, FloatN &m, FloatN &q, FloatN &w, FloatN &n, FloatN &z) = 0;

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x1, FloatN &r1, FloatN &w1,FloatN &q1,
						  FloatN &d1, FloatN &h1, FloatN &z1, FloatN &p1, FloatN &u1, FloatN &g1,
                                                  FloatN &x2, FloatN &r2, FloatN &w2,FloatN &q2, FloatN &d2, FloatN &h2,
                                                  FloatN &z2, FloatN &p2, FloatN &u2, FloatN &g2) = 0;

      //! post-computation routine called after the "M-loop"
      virtual __device__ __host__ void post(ReduceType sum[Nreduce]) { ; }

    };
    /**
       This convoluted kernel does the following:
//Gr1
       x += a*p,
       p = u + a*p,
       r -= a*s,
//Gr2
       u -= a*q,
       q = m + b*q,
//Gr3
       s = w + b*s,
       w -= a*z,
       z = n + b*z,
//Gr4
       norm = (u,u),
       rdot = (w,u),
       rdot = (r,u),
    */
    template <int Nreduce, typename ReduceType, typename Float2, typename FloatN>
    struct pipePCGRRMergedOp_ : public ReduceFunctorExp<Nreduce, ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      pipePCGRRMergedOp_(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x, FloatN &p, FloatN &u, FloatN &r, FloatN &s, FloatN &m, FloatN &q, FloatN &w, FloatN &n, FloatN &z) {

	typedef typename ScalarType<ReduceType>::type scalar;
#ifndef ERROR_CONTROL
         norm2_<scalar> (sum[1].x, p);
         norm2_<scalar> (sum[1].y, s);
         norm2_<scalar> (sum[1].z, q);
         norm2_<scalar> (sum[1].w, z);

         z = n + b.x*z;
         q = m + b.x*q;
         s = w + b.x*s;
         p = u + b.x*p;

         norm2_<scalar> (sum[1].x, p);
         norm2_<scalar> (sum[1].y, s);
         norm2_<scalar> (sum[1].z, q);
         norm2_<scalar> (sum[1].w, z);

         norm2_<scalar> (sum[2].x, x);
         norm2_<scalar> (sum[2].y, u);
         norm2_<scalar> (sum[2].z, w);
         norm2_<scalar> (sum[2].w, m);

         x = x + a.x*p;
         u = u - a.x*q;

         sum[0].w = 0.0;

         r = r - a.x*s;
         w = w - a.x*z;

         dot_<scalar>   (sum[0].x, r, u);
         dot_<scalar>   (sum[0].y, w, u);
         norm2_<scalar> (sum[0].z, r);
#else
         double a_ = a.x;
         double b_ = b.x;

         hp_axpby_reduce<ReduceType>(sum[0], a_, b_, x, p, u, r, s, m, q, w, n, z);
#endif
    }

     __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x1, FloatN &r1, FloatN &w1,FloatN &q1,
						  FloatN &d1, FloatN &h1, FloatN &z1, FloatN &p1, FloatN &u1, FloatN &g1,
                                                  FloatN &x2, FloatN &r2, FloatN &w2,FloatN &q2, FloatN &d2, FloatN &h2,
                                                  FloatN &z2, FloatN &p2, FloatN &u2, FloatN &g2) {}

      static int streams() { return 18; } //! total number of input and output streams
      static int flops() { return (16+6); } //! flops per real element
    };

    void pipePCGRRMergedOp(double4 *buffer, const int buffer_size, ColorSpinorField &x, const double &a, ColorSpinorField &p, ColorSpinorField &u, ColorSpinorField &r, ColorSpinorField &s,
                                ColorSpinorField &m, const double &b, ColorSpinorField &q,
			        ColorSpinorField &w, ColorSpinorField &n, ColorSpinorField &z) {
      if (x.Precision() != p.Precision()) {
         errorQuda("\nMixed blas is not implemented.\n");
      }
      if(buffer_size != 3) errorQuda("Incorrect buffer size. \n");

      reduce::reduceCudaExp<3, double4, QudaSumFloat4,pipePCGRRMergedOp_,1,1,1,1,1,0,1,1,0,1,false>
	  (buffer, make_double2(a, 0.0), make_double2(b, 0.0), x, p, u, r, s, m, q, w, n, z);
      return;
    }

    template <int Nreduce, typename ReduceType, typename Float2, typename FloatN>
    struct pipePCGRRFletcherReevesMergedOp_ : public ReduceFunctorExp<Nreduce, ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      pipePCGRRFletcherReevesMergedOp_(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x, FloatN &p, FloatN &u, FloatN &r, FloatN &s, FloatN &m, FloatN &q, FloatN &w, FloatN &n, FloatN &z) {

	 typedef typename ScalarType<ReduceType>::type scalar;
#ifndef ERROR_CONTROL

         if(Nreduce == 3) {
           norm2_<scalar> (sum[2].x, x);
           norm2_<scalar> (sum[2].y, u);
           norm2_<scalar> (sum[2].z, w);
           norm2_<scalar> (sum[2].w, m);
         }

//         z = n + b.x*z;
         _fma2(b.x, z, n);
//         q = m + b.x*q;
         _fma2(b.x, q, m);
//         s = w + b.x*s;
         _fma2(b.x, s, w);
//         p = u + b.x*p;
         _fma2(b.x, p, u);

//         x = x + a.x*p;
         _fma3(+a.x, p, x);
//         u = u - a.x*q;
         _fma3(-a.x, q, u);

         dot_<scalar>   (sum[0].w, r, u);

//         r = r - a.x*s;
         _fma3(-a.x, s, r);
//         w = w - a.x*z;
         _fma3(-a.x, z, w);

         dot_<scalar>   (sum[0].x, r, u);
         dot_<scalar>   (sum[0].y, w, u);
         dot_<scalar>   (sum[0].z, r, r);

         if(Nreduce == 3) {
           norm2_<scalar> (sum[1].x, p);
           norm2_<scalar> (sum[1].y, s);
           norm2_<scalar> (sum[1].z, q);
           norm2_<scalar> (sum[1].w, z);
         }

#else
         if(Nreduce == 3) {
           norm2_<scalar> (sum[2].x, x);
           norm2_<scalar> (sum[2].y, u);
           norm2_<scalar> (sum[2].z, w);
           norm2_<scalar> (sum[2].w, m);
         }

         //p = u + b.x*p;
         //x = x + a.x*p;
         hp_xpaybz(b.x, u, p, a.x, x);

         //q = m + b.x*q;
         //u = u - a.x*q;
         //<r,u>
         //s = w + b.x*s;
         //r = r - a.x*s;
         //z = n + b.x*z;
         //w = w - a.x*z;
         //<r,u>
         //<w,u>
         //<r,r>
         hp_xpaybz_combo_reduce<ReduceType>(sum[0], b.x, m, q, -a.x, u, w, s, r, n, z);

         if(Nreduce == 3) {
           norm2_<scalar> (sum[1].x, p);
           norm2_<scalar> (sum[1].y, s);
           norm2_<scalar> (sum[1].z, q);
           norm2_<scalar> (sum[1].w, z);
         }
#endif
      }

      __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x1, FloatN &r1, FloatN &w1,FloatN &q1,
						  FloatN &d1, FloatN &h1, FloatN &z1, FloatN &p1, FloatN &u1, FloatN &g1,
                                                  FloatN &x2, FloatN &r2, FloatN &w2,FloatN &q2, FloatN &d2, FloatN &h2,
                                                  FloatN &z2, FloatN &p2, FloatN &u2, FloatN &g2) {}
      static int streams() { return 18; } //! total number of input and output streams
      static int flops() { return (16+6); } //! flops per real element
    };


    void pipePCGRRFletcherReevesMergedOp(double4 *buffer, const int buffer_size,  ColorSpinorField &x, const double &a, ColorSpinorField &p, ColorSpinorField &u,
                                ColorSpinorField &r, ColorSpinorField &s,
                                ColorSpinorField &m, const double &b, ColorSpinorField &q,
			        ColorSpinorField &w, ColorSpinorField &n, ColorSpinorField &z) {
      if (x.Precision() != p.Precision()) {
         errorQuda("\nMixed blas is not implemented.\n");
      }
#if 0
      if( buffer_size == 3 ) {
         reduce::reduceCudaExp<3, double4, QudaSumFloat4,pipePCGRRFletcherReevesMergedOp_,1,1,1,1,1,0,1,1,0,1,false>
	  (buffer, make_double2(a, 0.0), make_double2(b, 0.0), x, p, u, r, s, m, q, w, n, z);
      } else if ( buffer_size == 1 ) {
         reduce::reduceCudaExp<1, double4, QudaSumFloat4,pipePCGRRFletcherReevesMergedOp_,1,1,1,1,1,0,1,1,0,1,false>
          (buffer, make_double2(a, 0.0), make_double2(b, 0.0), x, p, u, r, s, m, q, w, n, z);
      } else {
         errorQuda("Buffer size is not implemented. \n");
      }
#endif
      return;
    }

    template <int Nreduce, typename ReduceType, typename Float2, typename FloatN>
    struct pipe2PCGMergedOp_ : public ReduceFunctorExp<Nreduce, ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      Float2 c;

      Float2 a2;
      Float2 b2;
      Float2 c2;

      pipe2PCGMergedOp_(const Float2 &a, const Float2 &b, const Float2 &c, const Float2 &a2, const Float2 &b2, const Float2 &c2) : a(a), b(b), c(c), a2(a2), b2(b2), c2(c2) { ; }
      __device__ __host__ void operator()(ReduceType sum[Nreduce],FloatN &x1, FloatN &r1, FloatN &w1,FloatN &q1,FloatN &d1, FloatN &h1, FloatN &z1, FloatN &p1, FloatN &u1, FloatN &g1,
                                          FloatN &x2, FloatN &r2, FloatN &w2,FloatN &q2, FloatN &d2, FloatN &h2,FloatN &z2, FloatN &p2, FloatN &u2, FloatN &g2) {
	 typedef typename ScalarType<ReduceType>::type scalar;

         x2 = a.x*x1 + b.x*z1 + c.x*x2;
         r2 = a.x*r1 - b.x*w1 + c.x*r2;
         w2 = a.x*w1 - b.x*q1 + c.x*w2;
         q2 = a.x*q1 - b.x*d1 + c.x*q2;
         d2 = a.x*d1 - b.x*h1 + c.x*d2;
         z2 = a.x*z1 - b.x*p1 + c.x*z2;
         p2 = a.x*p1 - b.x*u1 + c.x*p2;
         u2 = a.x*u1 - b.x*g1 + c.x*u2;

         x1 = a2.x*x2 + b2.x*z2 + c2.x*x1;
         r1 = a2.x*r2 - b2.x*w2 + c2.x*r1;
         w1 = a2.x*w2 - b2.x*q2 + c2.x*w1;
         q1 = a2.x*q2 - b2.x*d2 + c2.x*q1;
         z1 = a2.x*z2 - b2.x*p2 + c2.x*z1;
         p1 = a2.x*p2 - b2.x*d2 + c2.x*p1;

         dot_<scalar> (sum[0].x, z1, w1);//l0
         dot_<scalar> (sum[0].y, z1, q1);//l1
         dot_<scalar> (sum[0].w, z1, w2);//l2
         dot_<scalar> (sum[0].z, p1, q1);//l3

         dot_<scalar> (sum[1].x, p1, w2);//l4
         dot_<scalar> (sum[1].y, z2, w2);//l5
         dot_<scalar> (sum[1].w, z1, r1);//l6
         dot_<scalar> (sum[1].z, z1, r2);//l7

         dot_<scalar>   (sum[2].x, z2, r2);//l8
         norm2_<scalar> (sum[2].y, z1);//l9

         sum[2].w = 0.0;
         sum[2].z = 0.0;

      }

      __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x, FloatN &p, FloatN &u,FloatN &r,
						  FloatN &s, FloatN &m, FloatN &q, FloatN &w, FloatN &n, FloatN &z) {}

      static int streams() { return 18; } //! total number of input and output streams
      static int flops() { return (16+6); } //! flops per real element
    };

    void pipe2PCGMergedOp(double4 *buffer, const double &a, const double &b, const double &c, const double &a2, const double &b2, const double &c2,
                                ColorSpinorField &x1, ColorSpinorField &r1, ColorSpinorField &w1,
                                ColorSpinorField &q1, ColorSpinorField &d1, ColorSpinorField &h1, ColorSpinorField &z1,
                                ColorSpinorField &p1, ColorSpinorField &u1, ColorSpinorField &g1,
                                ColorSpinorField &x2, ColorSpinorField &r2, ColorSpinorField &w2,
                                ColorSpinorField &q2, ColorSpinorField &d2, ColorSpinorField &h2, ColorSpinorField &z2,
			        ColorSpinorField &p2, ColorSpinorField &u2, ColorSpinorField &g2) {

      if (x1.Precision() != p1.Precision()) {
         errorQuda("\nMixed blas is not implemented.\n");
      }
#if 0
      reduce::reduceComponentwiseCudaExp<3, double4, QudaSumFloat4,pipe2PCGMergedOp_,1,1,1,1,1,0,1,1,1,0,false>
	  (buffer, make_double2(a, 0.0), make_double2(b, 0.0), make_double2(c, 0.0), make_double2(a2, 0.0), make_double2(b2, 0.0), make_double2(c2, 0.0),  x1, r1, w1, q1, d1, h1, z1, p1, u1, g1, x2, r2, w2, q2, d2, h2, z2, p2, u2, g2);
#endif
       return;
    }

    template <int Nreduce, typename ReduceType, typename Float2, typename FloatN>
    struct pipe2CGMergedOp_ : public ReduceFunctorExp<Nreduce, ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      Float2 c;

      Float2 a2;
      Float2 b2;
      Float2 c2;

      pipe2CGMergedOp_(const Float2 &a, const Float2 &b, const Float2 &c, const Float2 &a2, const Float2 &b2, const Float2 &c2) : a(a), b(b), c(c), a2(a2), b2(b2), c2(c2) { ; }
      __device__ __host__ void operator()(ReduceType sum[Nreduce],FloatN &x1, FloatN &r1, FloatN &w1,FloatN &q1,FloatN &d1, FloatN &h1, FloatN &z1, FloatN &p1, FloatN &u1, FloatN &g1,
                                          FloatN &x2, FloatN &r2, FloatN &w2,FloatN &q2, FloatN &d2, FloatN &h2,FloatN &z2, FloatN &p2, FloatN &u2, FloatN &g2) {
	 typedef typename ScalarType<ReduceType>::type scalar;

         x2 = a.x*x1 + b.x*r1 + c.x*x2;
         r2 = a.x*r1 - b.x*w1 + c.x*r2;
         w2 = a.x*w1 - b.x*q1 + c.x*w2;
         q2 = a.x*q1 - b.x*d1 + c.x*q2;
         d2 = a.x*d1 - b.x*h1 + c.x*d2;

         x1 = a2.x*x2 + b2.x*r2 + c2.x*x1;
         r1 = a2.x*r2 - b2.x*w2 + c2.x*r1;
         w1 = a2.x*w2 - b2.x*q2 + c2.x*w1;
         q1 = a2.x*q2 - b2.x*d2 + c2.x*q1;

         dot_<scalar> (sum[0].x, r1, w1);//l0
         dot_<scalar> (sum[0].y, r1, q1);//l1
         dot_<scalar> (sum[0].w, r1, w2);//l2
         dot_<scalar> (sum[0].z, w1, q1);//l3

         dot_<scalar> (sum[1].x, w1, w2);//l4
         dot_<scalar> (sum[1].y, r2, w2);//l5
         norm2_<scalar> (sum[1].w, r1);
         dot_<scalar> (sum[1].z, r1, r2);//l7
         norm2_<scalar> (sum[2].x, r2);


         sum[2].y = sum[1].w;
         sum[2].w = 0.0;
         sum[2].z = 0.0;
      }

      __device__ __host__ void operator()(ReduceType sum[Nreduce], FloatN &x, FloatN &p, FloatN &u,FloatN &r,
						  FloatN &s, FloatN &m, FloatN &q, FloatN &w, FloatN &n, FloatN &z) {}

      static int streams() { return 18; } //! total number of input and output streams
      static int flops() { return (16+6); } //! flops per real element
    };

    void pipe2CGMergedOp(double4 *buffer, const double &a, const double &b, const double &c, const double &a2, const double &b2, const double &c2,
                                ColorSpinorField &x1, ColorSpinorField &r1, ColorSpinorField &w1,
                                ColorSpinorField &q1, ColorSpinorField &d1, ColorSpinorField &h1, ColorSpinorField &z1,
                                ColorSpinorField &p1, ColorSpinorField &u1, ColorSpinorField &g1,
                                ColorSpinorField &x2, ColorSpinorField &r2, ColorSpinorField &w2,
                                ColorSpinorField &q2, ColorSpinorField &d2, ColorSpinorField &h2, ColorSpinorField &z2,
			        ColorSpinorField &p2, ColorSpinorField &u2, ColorSpinorField &g2) {

      if (x1.Precision() != p1.Precision()) {
         errorQuda("\nMixed blas is not implemented.\n");
      }
#if 0
      reduce::reduceComponentwiseCudaExp<3, double4, QudaSumFloat4,pipe2CGMergedOp_,1,1,1,1,1,0,0,0,0,0,false>
	  (buffer, make_double2(a, 0.0), make_double2(b, 0.0), make_double2(c, 0.0), make_double2(a2, 0.0), make_double2(b2, 0.0), make_double2(c2, 0.0),  x1, r1, w1, q1, d1, h1, z1, p1, u1, g1, x2, r2, w2, q2, d2, h2, z2, p2, u2, g2);
#endif
       return;
    }

  } // namespace blas

} // namespace quda
