#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <color_spinor_field_order.h>
#include <jitify_helper.cuh>
#include <kernels/reduce_core.cuh>

namespace quda {

  namespace blas {

    qudaStream_t* getStream();

    template <int block_size, typename real, int len, typename Arg>
    typename std::enable_if<block_size!=32, hipError_t>::type launch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream)
    {
      void *args[] = {&arg};
      if (tp.block.x == block_size)
        return qudaLaunchKernel((const void*)reduceKernel<block_size, real, len, Arg>, tp.grid, tp.block, args, tp.shared_bytes, stream);
      else
        return launch<block_size - 32, real, len>(arg, tp, stream);
    }

    template <int block_size, typename real, int len, typename Arg>
    typename std::enable_if<block_size==32, hipError_t>::type launch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream)
    {
      void *args[] = {&arg};
      return qudaLaunchKernel((const void*)reduceKernel<block_size, real, len, Arg>, tp.grid, tp.block, args, tp.shared_bytes, stream);
    }

#ifdef QUDA_FAST_COMPILE_REDUCE
    constexpr unsigned int max_block_size() { return 32; }
#else
    constexpr unsigned int max_block_size() { return 1024; }
#endif

   /**
       Generic reduction kernel launcher
    */
    template <typename host_reduce_t, typename real, int len, typename Arg>
    auto reduceLaunch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream, Tunable &tunable)
    {
      using device_reduce_t = typename Arg::Reducer::reduce_t;
      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::reduceKernel")
                                  .instantiate((int)tp.block.x, Type<real>(), len, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
#else
      if (tp.block.x <= max_block_size()) {
        auto error = launch<max_block_size(), real, len>(arg, tp, stream);
        // flag any failures when tuning so we don't try and complete which could hang
        if (activeTuning() && error != hipSuccess) tunable.jitifyError() = hipErrorInvalidValue;
      } else {
        tunable.jitifyError() = hipErrorInvalidValue;
        if (!activeTuning()) errorQuda("block size %d not instantiated", tp.block.x);
      }
#endif

      if (!commAsyncReduction()) {
        if (tunable.jitifyError() != hipErrorInvalidValue) arg.complete(stream);
      }

      host_reduce_t cpu_sum = set(((device_reduce_t *)reducer::get_host_buffer())[0]);
      if (tp.grid.y == 2) sum(cpu_sum, ((device_reduce_t *)reducer::get_host_buffer())[1]); // add other parity if needed
      return cpu_sum;
    }

    template <template <typename ReducerType, typename real> class Reducer,
              typename store_t, typename y_store_t, int nSpin, typename coeff_t>
    class Reduce : public Tunable
    {
      using real = typename mapper<y_store_t>::type;
      using host_reduce_t = typename Reducer<double, real>::reduce_t;
      Reducer<device_reduce_t, real> r;
      const int nParity; // for composite fields this includes the number of composites
      host_reduce_t &result;

      const coeff_t &a, &b;
      ColorSpinorField &x, &y, &z, &w, &v;
      QudaFieldLocation location;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
            sharedBytesPerThread() * nthreads :
            sharedBytesPerBlock(param);
        return false;
      }

    public:
      Reduce(const coeff_t &a, const coeff_t &b, const coeff_t &c, ColorSpinorField &x, ColorSpinorField &y,
             ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v, host_reduce_t &result) :
        r(a, b),
        nParity((x.IsComposite() ? x.CompositeDim() : 1) * (x.SiteSubset())),
        a(a),
        b(b),
        x(x),
        y(y),
        z(z),
        w(w),
        v(v),
        result(result),
        location(checkLocation(x, y, z, w, v))
      {
        checkLength(x, y, z, w, v);
        auto x_prec = checkPrecision(x, z, w);
        auto y_prec = checkPrecision(y, v);
        auto x_order = checkOrder(x, z, w);
        auto y_order = checkOrder(y, v);
        if (sizeof(store_t) != x_prec) errorQuda("Expected precision %lu but received %d", sizeof(store_t), x_prec);
        if (sizeof(y_store_t) != y_prec) errorQuda("Expected precision %lu but received %d", sizeof(y_store_t), y_prec);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        strcpy(aux, x.AuxString());
        if (x_prec != y_prec) {
          strcat(aux, ",");
          strcat(aux, y.AuxString());
        }
        if (location == QUDA_CPU_FIELD_LOCATION) strcat(aux, ",CPU");
#ifdef FAST_REDUCE
        if (location == QUDA_CUDA_FIELD_LOCATION) strcat(aux, ",fast_reduce");
#endif

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/reduce_core.cuh");
#endif

        apply(*(blas::getStream()));
        checkCudaError();

        blas::bytes += bytes();
        blas::flops += flops();

        const int Nreduce = sizeof(host_reduce_t) / sizeof(double);
        reduceDoubleArray((double *)&result, Nreduce);
      }

      TuneKey tuneKey() const { return TuneKey(x.VolString(), typeid(r).name(), aux); }

      void apply(const qudaStream_t &stream)
      {
        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value || decltype(r)::site_unroll;
        if (site_unroll_check && (x.Ncolor() != 3 || x.Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x.Nspin(), x.Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(x, y, z, w, v); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Reducer<device_reduce_t, device_real_t> r_(a, b);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value || decltype(r)::site_unroll;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x.Length() / (nParity * M);

          ReductionArg<device_store_t, N, device_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, v, r_, length, nParity);
          result = reduceLaunch<host_reduce_t, device_real_t, M>(arg, tp, stream, *this);
        } else {
          if (checkOrder(x, y, z, w, v) != QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
            warningQuda("CPU Blas functions expect AoS field order");
            return;
          }

          using host_store_t = typename host_type_mapper<store_t>::type;
          using host_y_store_t = typename host_type_mapper<y_store_t>::type;
          using host_real_t = typename mapper<host_y_store_t>::type;
          Reducer<double, host_real_t> r_(a, b);

          // redefine site_unroll with host_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<host_store_t, host_y_store_t>::value || isFixed<host_store_t>::value || decltype(r)::site_unroll;
          constexpr int N = n_vector<host_store_t, false, nSpin, site_unroll>();
          constexpr int Ny = n_vector<host_y_store_t, false, nSpin, site_unroll>();
          constexpr int M = N; // if site unrolling then M=N will be 24/6, e.g., full AoS
          const int length = x.Length() / (nParity * M);

          ReductionArg<host_store_t, N, host_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, v, r_, length, nParity);
          result = reduceCPU<host_real_t, M>(arg);
        }
      }

      void preTune()
      {
        if (r.write.X) x.backup();
        if (r.write.Y) y.backup();
        if (r.write.Z) z.backup();
        if (r.write.W) w.backup();
        if (r.write.V) v.backup();
      }

      void postTune()
      {
        if (r.write.X) x.restore();
        if (r.write.Y) y.restore();
        if (r.write.Z) z.restore();
        if (r.write.W) w.restore();
        if (r.write.V) v.restore();
      }

      bool advanceTuneParam(TuneParam &param) const
      {
        return location == QUDA_CPU_FIELD_LOCATION ? false : Tunable::advanceTuneParam(param);
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.grid.y = nParity;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
        param.grid.y = nParity;
      }

      long long flops() const { return r.flops() * x.Length(); }

      long long bytes() const
      {
        // the factor two here assumes we are reading and writing to the high precision vector
        // this will evaluate correctly for non-mixed kernels since the +2/-2 will cancel out
        return (r.streams() - 2) * x.Bytes() + 2 * z.Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <template <typename reduce_t, typename real> class Functor, bool mixed, typename... Args>
    auto instantiateReduce(Args &&... args)
    {
      using host_reduce_t = typename Functor<double, double>::reduce_t;
      host_reduce_t value;
      ::quda::zero(value); // no default constructor so we need to explicitly zero
      instantiate<Functor, Reduce, mixed>(args..., value);
      return value;
    }

    double norm1(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x); // FIXME
      return instantiateReduce<Norm1, false>(0.0, 0.0, 0.0, y, y, y, y, y);
    }

    double norm2(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x);
      return instantiateReduce<Norm2, false>(0.0, 0.0, 0.0, y, y, y, y, y);
    }

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<Dot, false>(0.0, 0.0, 0.0, x, y, x, x, x);
    }

    double axpbyzNorm(double a, ColorSpinorField &x, double b, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<axpbyzNorm2, false>(a, b, 0.0, x, y, z, x, x);
    }

    double axpyReDot(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<AxpyReDot, false>(a, 0.0, 0.0, x, y, x, x, x);
    }

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<caxpyNorm2, false>(a, Complex(0.0), Complex(0.0), x, y, x, x, x);
    }

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<caxpyxmaznormx, false>(a, Complex(0.0), Complex(0.0), x, y, z, x, x);
    }

    double cabxpyzAxNorm(double a, const Complex &b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<cabxpyzaxnorm, false>(Complex(a), b, Complex(0.0), x, y, z, x, x);
    }

    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      auto cdot = instantiateReduce<Cdot, false>(0.0, 0.0, 0.0, x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }

    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      auto cdot = instantiateReduce<caxpydotzy, false>(a, Complex(0.0), Complex(0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<CdotNormA, false>(0.0, 0.0, 0.0, x, y, x, x, x);
    }

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x, const Complex &b, ColorSpinorField &y,
                                           ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &u)
    {
      return instantiateReduce<caxpbypzYmbwcDotProductUYNormY_, true>(a, b, Complex(0.0), x, z, y, w, u);
    }

    Complex axpyCGNorm(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      double2 cg_norm = instantiateReduce<axpyCGNorm2, true>(a, 0.0, 0.0, x, y, x, x, y);
      return Complex(cg_norm.x, cg_norm.y);
    }

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor() != 3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = instantiateReduce<HeavyQuarkResidualNorm_, false>(0.0, 0.0, 0.0, x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = instantiateReduce<xpyHeavyQuarkResidualNorm_, false>(0.0, 0.0, 0.0, x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<tripleCGReduction_, false>(0.0, 0.0, 0.0, x, y, z, x, x);
    }

    double4 quadrupleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<quadrupleCGReduction_, false>(0.0, 0.0, 0.0, x, y, z, x, x);
    }

    double quadrupleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y,
                                ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      return instantiateReduce<quadrupleCG3InitNorm_, false>(a, 0.0, 0.0, x, y, z, w, v);
    }

    double quadrupleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y,
                                  ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      return instantiateReduce<quadrupleCG3UpdateNorm_, false>(a, b, 0.0, x, y, z, w, v);
    }

  } // namespace blas

} // namespace quda
