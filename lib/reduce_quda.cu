#include "hip/hip_runtime.h"
#include <atomic>
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/reduce_core.cuh>

// These are used for reduction kernels
static device_reduce_t *d_reduce = nullptr;
static device_reduce_t *h_reduce = nullptr;
static device_reduce_t *hd_reduce = nullptr;
static hipEvent_t reduceEnd;
static bool fast_reduce_enabled = false;

namespace quda {

  namespace blas {

#include <generic_reduce.cuh>

    qudaStream_t* getStream();

    void* getDeviceReduceBuffer() { return d_reduce; }
    void* getMappedHostReduceBuffer() { return hd_reduce; }
    void* getHostReduceBuffer() { return h_reduce; }
    hipEvent_t* getReduceEvent() { return &reduceEnd; }
    bool getFastReduce() { return fast_reduce_enabled; }

    void initFastReduce(int32_t words)
    {
      // initialize the reduction values in 32-bit increments to INT_MIN
      for (int32_t i = 0; i < words; i++) {
        reinterpret_cast<int32_t *>(h_reduce)[i] = std::numeric_limits<int32_t>::min();
      }

      // ensure that the host memory write is complete before we launch the kernel
      atomic_thread_fence(std::memory_order_release);
    }

    void completeFastReduce(int32_t words)
    {
      volatile int32_t *check = reinterpret_cast<int32_t *>(h_reduce);
      int count = 0;
      int complete = 0;
      while (complete < words) {
        // ensure visiblity to any changes in memory
        atomic_thread_fence(std::memory_order_acquire);

        complete = 0;
        for (int32_t i = 0; i < words; i++) {
          // spin-wait until all values have been updated
          if (check[i] != std::numeric_limits<int32_t>::min()) complete++;
        }
        if (count++ % 10000 == 0) { // check error every 10000 iterations
          // if there is an error in the kernel then we need to exit the spin-wait
          if (hipSuccess != hipPeekAtLastError()) break;
        }
      }
    }

    void initReduce()
    {
      /* we have these different reductions to cater for:

         - regular reductions (reduce_quda.cu) where are reducing to a
           single vector type (max length 4 presently), with possibly
           parity dimension, and a grid-stride loop with max number of
           blocks = 2 x SM count

         - multi-reductions where we are reducing to a matrix of size
           of size QUDA_MAX_MULTI_REDUCE of vectors (max length 4), with
           possible parity dimension, and a grid-stride loop with
           maximum number of blocks = 2 x SM count
      */

      const int reduce_size = 4 * sizeof(device_reduce_t);
      const int max_reduce_blocks = 2*deviceProp.multiProcessorCount;

      const int max_reduce = 2 * max_reduce_blocks * reduce_size;
      const int max_multi_reduce = 2 * QUDA_MAX_MULTI_REDUCE * max_reduce_blocks * reduce_size;

      // reduction buffer size
      size_t bytes = max_reduce > max_multi_reduce ? max_reduce : max_multi_reduce;

      if (!d_reduce) d_reduce = (device_reduce_t *) device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be device_reduce_t x 3)

      // if the device supports host-mapped memory then use a host-mapped array for the reduction
      if (!h_reduce) {
	// only use zero copy reductions when using 64-bit
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
	if(deviceProp.canMapHostMemory) {
	  h_reduce = (device_reduce_t *) mapped_malloc(bytes);
	  hipHostGetDevicePointer(&hd_reduce, h_reduce, 0); // set the matching device pointer
	} else
#endif
	  {
	    h_reduce = (device_reduce_t *) pinned_malloc(bytes);
	    hd_reduce = d_reduce;
	  }
	memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
      }

      hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);

      // enable fast reductions with CPU spin waiting as opposed to using CUDA events
      char *fast_reduce_env = getenv("QUDA_ENABLE_FAST_REDUCE");
      if (fast_reduce_env && strcmp(fast_reduce_env,"1") == 0) {
        warningQuda("Experimental fast reductions enabled");
        fast_reduce_enabled = true;
      }

      checkCudaError();
    }

    void endReduce(void)
    {
      if (d_reduce) {
	device_free(d_reduce);
	d_reduce = 0;
      }
      if (h_reduce) {
	host_free(h_reduce);
	h_reduce = 0;
      }
      hd_reduce = 0;

      hipEventDestroy(reduceEnd);
    }

    /**
       Generic reduction kernel launcher
    */
    template <typename host_reduce_t, typename real, int len, typename Arg>
    auto reduceLaunch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream, Tunable &tunable)
    {
      using device_reduce_t = typename Arg::Reducer::reduce_t;
      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

      const int32_t words = tp.grid.y * sizeof(device_reduce_t) / sizeof(int32_t);
      if (getFastReduce() && !commAsyncReduction()) initFastReduce(words);

#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::reduceKernel")
                                  .instantiate((int)tp.block.x, Type<real>(), len, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
#else
      LAUNCH_KERNEL(reduceKernel, tunable, tp, stream, arg, real, len);
#endif

      if (!commAsyncReduction()) {
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
        if (deviceProp.canMapHostMemory) {
          if (getFastReduce()) {
            completeFastReduce(words);
          } else {
            qudaEventRecord(reduceEnd, stream);
            while (hipSuccess != qudaEventQuery(reduceEnd)) { ; }
          }
        } else
#endif
        {
          qudaMemcpy(h_reduce, hd_reduce, sizeof(device_reduce_t), hipMemcpyDeviceToHost);
        }
      }

      host_reduce_t cpu_sum = set(((device_reduce_t *)h_reduce)[0]);
      if (tp.grid.y == 2) sum(cpu_sum, ((device_reduce_t *)h_reduce)[1]); // add other parity if needed
      return cpu_sum;
    }

    template <typename host_reduce_t, typename real, int len, typename SpinorX, typename SpinorY,
              typename SpinorZ, typename SpinorW, typename SpinorV, typename Reducer>
    class Reduce : public Tunable
    {
      const int nParity; // for composite fields this includes the number of composites
      mutable ReductionArg<SpinorX, SpinorY, SpinorZ, SpinorW, SpinorV, Reducer> arg;
      host_reduce_t &result;

      const ColorSpinorField &x, &y, &z, &w, &v;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
            sharedBytesPerThread() * nthreads :
            sharedBytesPerBlock(param);
        return false;
      }

  public:
      Reduce(host_reduce_t &result, SpinorX &X, SpinorY &Y, SpinorZ &Z, SpinorW &W, SpinorV &V, Reducer &r,
             ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v,
          int length) :
          nParity((x.IsComposite() ? x.CompositeDim() : 1) * (x.SiteSubset())),
          arg(X, Y, Z, W, V, r, length / nParity),
          x(x),
          y(y),
          z(z),
          w(w),
          v(v),
          result(result)
      {
        strcpy(aux, x.AuxString());
        if (x.Precision() != z.Precision()) {
          strcat(aux, ",");
          strcat(aux, z.AuxString());
        }
        if (getFastReduce()) strcat(aux, ",fast_reduce");

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/reduce_core.cuh");
#endif
      }
      virtual ~Reduce() {}

      inline TuneKey tuneKey() const { return TuneKey(x.VolString(), typeid(arg.r).name(), aux); }

      void apply(const qudaStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        result = reduceLaunch<host_reduce_t, real, len>(arg, tp, stream, *this);
      }

      void preTune()
      {
        if (arg.r.write.X) x.backup();
        if (arg.r.write.Y) y.backup();
        if (arg.r.write.Z) z.backup();
        if (arg.r.write.W) w.backup();
        if (arg.r.write.V) v.backup();
      }

      void postTune()
      {
        if (arg.r.write.X) x.restore();
        if (arg.r.write.Y) y.restore();
        if (arg.r.write.Z) z.restore();
        if (arg.r.write.W) w.restore();
        if (arg.r.write.V) v.restore();
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.grid.y = nParity;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
        param.grid.y = nParity;
      }

      long long flops() const { return arg.r.flops() * x.Length(); }

      long long bytes() const
      {
        // the factor two here assumes we are reading and writing to the high precision vector
        // this will evaluate correctly for non-mixed kernels since the +2/-2 will cancel out
        return (arg.r.streams() - 2) * x.Bytes() + 2 * z.Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <template <typename ReducerType, typename real> class Reducer, typename real,
              typename store_t, int len, int N, typename z_store_t = store_t, int Nz = N, typename coeff_t>
    auto nativeReduce(const coeff_t &a, const coeff_t &b, ColorSpinorField &x, ColorSpinorField &y,
                      ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v, int length)
    {
      checkLength(x, y);
      checkLength(x, z);
      checkLength(x, w);
      checkLength(x, v);

      using host_reduce_t = typename Reducer<double, real>::reduce_t;
      Reducer<device_reduce_t, real> r(a, b);

      Spinor<store_t, N> X(x);
      Spinor<store_t, N> Y(y);
      Spinor<z_store_t, Nz> Z(z);
      Spinor<store_t, N> W(w);
      Spinor<store_t, N> V(v);

      host_reduce_t value;
      Reduce<host_reduce_t, real, len, decltype(X), decltype(Y), decltype(Z), decltype(W), decltype(V), decltype(r)>
        reduce(value, X, Y, Z, W, V, r, x, y, z, w, v, length);
      reduce.apply(*(blas::getStream()));

      blas::bytes += reduce.bytes();
      blas::flops += reduce.flops();
      checkCudaError();
      return value;
    }

    /*
      Wilson
      double double2 M = 1/12
      single float4  M = 1/6
      half   short4  M = 6/6

      Staggered
      double double2 M = 1/3
      single float2  M = 1/3
      half   short2  M = 3/3
    */

    /**
       Driver for generic reduction routine with five loads.
       @param ReduceType
    */
    template <template <typename reduce_t, typename real> class Reducer, typename coeff_t>
    auto uni_reduce(const coeff_t &a, const coeff_t &b, ColorSpinorField &x, ColorSpinorField &y,
                    ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      checkPrecision(x, y, z, w, v);

      constexpr bool siteUnroll = Reducer<double, double>::site_unroll;
      using host_reduce_t = typename Reducer<double, double>::reduce_t;
      host_reduce_t value;

      if (checkLocation(x, y, z, w, v) == QUDA_CUDA_FIELD_LOCATION) {

        if (!x.isNative() && x.FieldOrder() != QUDA_FLOAT8_FIELD_ORDER) {
          warningQuda("Device reductions on non-native fields is not supported (prec = %d, order = %d)", x.Precision(),
                      x.FieldOrder());
          host_reduce_t value;
          ::quda::zero(value);
          return value;
        }

        // cannot do site unrolling for arbitrary color (needs JIT)
        if (siteUnroll && x.Ncolor() != 3) errorQuda("Not supported");

        int reduce_length = siteUnroll ? x.RealLength() : x.Length();

        if (x.Precision() == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
          if (x.Nspin() == 4 || x.Nspin() == 2) { // wilson
#if defined(NSPIN4) || defined(NSPIN2)
            const int M = siteUnroll ? 24 : 2; // determines how much work per thread to do
            if (x.Nspin() == 2 && siteUnroll) errorQuda("siteUnroll not supported for nSpin==2");
            value = nativeReduce<Reducer, double, double, M, 2>(a, b, x, y, z, w, v, reduce_length / M);
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
            const int M = siteUnroll ? 6 : 2; // determines how much work per thread to do
            value = nativeReduce<Reducer, double, double, M, 2>(a, b, x, y, z, w, v, reduce_length / M);
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else if (x.Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
          if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT4_FIELD_ORDER) { // wilson
#if defined(NSPIN4)
            const int M = siteUnroll ? 24 : 4; // determines how much work per thread to do
            value = nativeReduce<Reducer, float, float, M, 4>(a, b, x, y, z, w, v, reduce_length / M);
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1 || x.Nspin() == 2 || (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER)) {
#if defined(NSPIN1) || defined(NSPIN2) || defined(GPU_MULTIGRID)
            const int M = siteUnroll ? 6 : 2; // determines how much work per thread to do
            if ((x.Nspin() == 2 || x.Nspin() == 4) && siteUnroll) errorQuda("siteUnroll not supported here for nSpin=%d", x.Nspin());
            value = nativeReduce<Reducer, float, float, M, 2>(a, b, x, y, z, w, v, reduce_length / M);
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else if (x.Precision() == QUDA_HALF_PRECISION) { // half precision

#if QUDA_PRECISION & 2
          if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT4_FIELD_ORDER) { // wilson
#if defined(NSPIN4)
            const int M = 24; // determines how much work per thread to do
            value = nativeReduce<Reducer, float, short, M, 4>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT8_FIELD_ORDER) { // wilson
#if defined(NSPIN4) && defined(FLOAT8)
            const int M = 24; // determines how much work per thread to do
            value = nativeReduce<Reducer, float, short, M, 8>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
            const int M = 6; // determines how much work per thread to do
            value = nativeReduce<Reducer, float, short, M, 2>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else if (x.Precision() == QUDA_QUARTER_PRECISION) { // quarter precision

#if QUDA_PRECISION & 1
          if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT4_FIELD_ORDER) { // wilson
#if defined(NSPIN4)
            const int M = 24; // determines how much work per thread to do
            value = nativeReduce<Reducer, float, char, M, 4>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT8_FIELD_ORDER) { // wilson
#if defined(NSPIN4) && defined(FLOAT8)
            const int M = 24;
            value = nativeReduce<Reducer, float, char, M, 8>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1) { // staggered
#ifdef NSPIN1
            const int M = 3; // determines how much work per thread to do
            value = nativeReduce<Reducer, float, char, M, 2>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else {
          errorQuda("precision=%d is not supported\n", x.Precision());
        }
      } else { // fields are on the CPU
        if (x.Precision() == QUDA_DOUBLE_PRECISION) {
          Reducer<double, double> r(a, b);
          value = genericReduce<host_reduce_t, double, double, decltype(r)>(x, y, z, w, v, r);
        } else if (x.Precision() == QUDA_SINGLE_PRECISION) {
          Reducer<double, float> r(a, b);
          value = genericReduce<host_reduce_t, float, float, decltype(r)>(x, y, z, w, v, r);
        } else {
          errorQuda("Precision %d not implemented", x.Precision());
        }
      }

      const int Nreduce = sizeof(host_reduce_t) / sizeof(double);
      reduceDoubleArray((double *)&value, Nreduce);

      return value;
    }

    template <template <typename ReducerType, typename real> class Reducer, typename coeff_t>
    auto mixed_reduce(const coeff_t &a, const coeff_t &b, ColorSpinorField &x, ColorSpinorField &y,
                      ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      checkPrecision(x, y, w, v);

      using host_reduce_t = typename Reducer<double, double>::reduce_t;
      host_reduce_t value;

      if (checkLocation(x, y, z, w, v) == QUDA_CUDA_FIELD_LOCATION) {

        if (!x.isNative()) {
          warningQuda("Device reductions on non-native fields is not supported (prec = %d, order = %d)", x.Precision(),
                      x.FieldOrder());
          host_reduce_t value;
          ::quda::zero(value);
          return value;
        }

        // cannot do site unrolling for arbitrary color (needs JIT)
        if (x.Ncolor() != 3) errorQuda("Not supported");

        if (z.Precision() == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
          if (x.Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 24; // determines how much work per thread to do
              value = nativeReduce<Reducer, double, float, M, 4, double, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 6; // determines how much work per thread to do
              value = nativeReduce<Reducer, double, float, M, 2, double, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else if (x.Precision() == QUDA_HALF_PRECISION) {

#if QUDA_PRECISION & 2
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 24; // determines how much work per thread to do
              value = nativeReduce<Reducer, double, short, M, 4, double, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 6; // determines how much work per thread to do
              value = nativeReduce<Reducer, double, short, M, 2, double, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else if (x.Precision() == QUDA_QUARTER_PRECISION) {

#if QUDA_PRECISION & 1
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 24; // determines how much work per thread to do
              value = nativeReduce<Reducer, double, char, M, 4, double, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 6; // determines how much work per thread to do
              value = nativeReduce<Reducer, double, char, M, 2, double, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else {
            errorQuda("Not implemented for this precision combination %d %d", x.Precision(), z.Precision());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, z.Precision());
#endif

        } else if (z.Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
          if (x.Precision() == QUDA_HALF_PRECISION) {

#if QUDA_PRECISION & 2
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 24;
              value = nativeReduce<Reducer, float, short, M, 4, float, 4>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 6;
              value = nativeReduce<Reducer, float, short, M, 2, float, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else if (x.Precision() == QUDA_QUARTER_PRECISION) {
#if QUDA_PRECISION & 1
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 24;
              value = nativeReduce<Reducer, float, char, M, 4, float, 4>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 6;
              value = nativeReduce<Reducer, float, char, M, 2, float, 2>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif
          } else {
            errorQuda("Not implemented for this precision combination %d %d", x.Precision(), z.Precision());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else {
          errorQuda("Not implemented for this precision combination %d %d", x.Precision(), z.Precision());
        }

      } else {
        if (x.Precision() == QUDA_SINGLE_PRECISION && z.Precision() == QUDA_DOUBLE_PRECISION) {
          Reducer<double, double> r(a, b);
          value = genericReduce<host_reduce_t, float, double, decltype(r)>(x, y, z, w, v, r);
        } else {
          errorQuda("Precision %d not implemented", x.Precision());
        }
      }

      const int Nreduce = sizeof(host_reduce_t) / sizeof(double);
      reduceDoubleArray((double *)&value, Nreduce);

      return value;
    }

    double norm1(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x); // FIXME
      return uni_reduce<Norm1>(0.0, 0.0, y, y, y, y, y);
    }

    double norm2(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x);
      return uni_reduce<Norm2>(0.0, 0.0, y, y, y, y, y);
    }

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      return uni_reduce<Dot>(0.0, 0.0, x, y, x, x, x);
    }

    double axpbyzNorm(double a, ColorSpinorField &x, double b, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<axpbyzNorm2>(a, b, x, y, z, x, x);
    }

    double axpyReDot(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return uni_reduce<AxpyReDot>(a, 0.0, x, y, x, x, x);
    }

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return uni_reduce<caxpyNorm2>(a, Complex(0.0), x, y, x, x, x);
    }

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<caxpyxmaznormx>(a, Complex(0.0), x, y, z, x, x);
    }

    double cabxpyzAxNorm(double a, const Complex &b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<cabxpyzaxnorm>(Complex(a), b, x, y, z, x, x);
    }

    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      auto cdot = uni_reduce<Cdot>(0.0, 0.0, x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }

    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      double2 cdot = uni_reduce<caxpydotzy>(a, Complex(0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y)
    {
      return uni_reduce<CdotNormA>(0.0, 0.0, x, y, x, x, x);
    }

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x, const Complex &b, ColorSpinorField &y,
                                           ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &u)
    {
      if (x.Precision() != z.Precision()) {
        return mixed_reduce<caxpbypzYmbwcDotProductUYNormY_>(a, b, x, y, z, w, u);
      } else {
        return uni_reduce<caxpbypzYmbwcDotProductUYNormY_>(a, b, x, y, z, w, u);
      }
    }

    Complex axpyCGNorm(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      // swizzle since mixed is on z
      double2 cg_norm ;
      if (x.Precision() != y.Precision()) {
        cg_norm = mixed_reduce<axpyCGNorm2>(a, 0.0, x, x, y, x, x);
      } else {
        cg_norm = uni_reduce<axpyCGNorm2>(a, 0.0, x, x, y, x, x);
      }
      return Complex(cg_norm.x, cg_norm.y);
    }

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = uni_reduce<HeavyQuarkResidualNorm_>(0.0, 0.0, x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = uni_reduce<xpyHeavyQuarkResidualNorm_>(0.0, 0.0, x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<tripleCGReduction_>(0.0, 0.0, x, y, z, x, x);
    }

    double4 quadrupleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<quadrupleCGReduction_>(0.0, 0.0, x, y, z, x, x);
    }

    double quadrupleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v) {
      return uni_reduce<quadrupleCG3InitNorm_>(a, 0.0, x, y, z, w, v);
    }

    double quadrupleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y,
                                  ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      return uni_reduce<quadrupleCG3UpdateNorm_>(a, b, x, y, z, w, v);
    }

    double doubleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<doubleCG3InitNorm_>(a, 0.0, x, y, z, z, z);
    }

    double doubleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<doubleCG3UpdateNorm_>(a, b, x, y, z, z, z);
    }

  } // namespace blas

} // namespace quda
