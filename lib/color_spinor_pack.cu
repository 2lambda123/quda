#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <tune_quda.h>

namespace quda {

  template <typename Field>
  struct PackGhostArg {

    Field field;
    void **ghost;
    const void *v;
    int X[QUDA_MAX_DIM];
    const int volumeCB;
    const int nDim;
    const int nFace;
    const int parity;
    const int nParity;
    const int dagger;
    const QudaDWFPCType pc_type;

    PackGhostArg(Field field, void **ghost, const ColorSpinorField &a, int parity, int dagger)
      : field(field),
	ghost(ghost),
	v(a.V()),
	volumeCB(a.VolumeCB()),
	nDim(a.Ndim()),
	nFace(a.Nspin() == 1 ? 3 : 1),
	parity(parity),
	nParity(a.SiteSubset()),
	dagger(dagger),
	pc_type(a.DWFPCtype())
    {
      for (int d=0; d<nDim; d++) X[d] = a.X(d);
      X[0] *= (nParity == 1) ? 2 : 1; // set to full lattice dimensions
      X[4] = (nDim == 5) ? a.X(4) : 1; // set fifth dimension correctly
    }
  };

  template <typename Float, int Ns, int Nc, typename Arg>
  __device__ __host__ inline void packGhost(Arg &arg, int cb_idx, int parity, int spinor_parity) {
    typedef typename mapper<Float>::type RegType;

    const int *X = arg.X;
    int x[5] = { };
    if (arg.nDim == 5)  getCoords5(x, cb_idx, X, parity, arg.pc_type);
    else getCoords(x, cb_idx, X, parity);

    RegType tmp[2*Ns*Nc];
    // FIXME make partitioning optional

#pragma unroll
    for (int dim=0; dim<4; dim++) {
      if (x[dim] < arg.nFace){
	arg.field.load(tmp, cb_idx, spinor_parity);
	arg.field.saveGhost(tmp, ghostFaceIndex<0>(x,arg.X,dim,arg.nFace), dim, 0, spinor_parity);
      }
      
      if (x[dim] >= X[dim] - arg.nFace){
	arg.field.load(tmp, cb_idx, spinor_parity);
	arg.field.saveGhost(tmp, ghostFaceIndex<1>(x,arg.X,dim,arg.nFace), dim, 1, spinor_parity);
      }
    }
  }

  template <typename Float, int Ns, int Nc, typename Arg>
  void GenericPackGhost(Arg &arg) {
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;
      const int spinor_parity = (arg.nParity == 2) ? parity : 0;
      for (int i=0; i<arg.volumeCB; i++) packGhost<Float,Ns,Nc>(arg, i, parity, spinor_parity);
    }
  }

  template <typename Float, int Ns, int Nc, typename Arg>
  __global__ void GenericPackGhostKernel(Arg arg) {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    if (x_cb >= arg.volumeCB) return;
    const int parity = (blockDim.y == 2) ? threadIdx.y : arg.parity;
    const int spinor_parity = (blockDim.y == 2) ? parity : 0;
    packGhost<Float,Ns,Nc>(arg, x_cb, parity, spinor_parity);
  }

  template <typename Float, int Ns, int Nc, typename Arg>
  class GenericPackGhostLauncher : public Tunable {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const { return 0; }
    long long bytes() const {
      // FIXME take into account paritioning
      size_t totalBytes = 0;
      for (int d=0; d<4; d++) {
	totalBytes += 2*arg.nFace*2*Ns*Nc*meta.SurfaceCB(d)*meta.Precision();
      }
      return totalBytes;
    }

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

    bool advanceTuneParam(TuneParam &param) const
    {
      bool rtn = Tunable::advanceTuneParam(param);
      param.block.y = arg.nParity;
      return rtn;
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.block.y = arg.nParity;
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.block.y = arg.nParity;
    }

  public:
    GenericPackGhostLauncher(Arg &arg, const ColorSpinorField &meta) : arg(arg), meta(meta) { }
    virtual ~GenericPackGhostLauncher() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	GenericPackGhost<Float,Ns,Nc,Arg>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	GenericPackGhostKernel<Float,Ns,Nc,Arg> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString());
    }
  };

  template <typename Float, QudaFieldOrder order, int Ns, int Nc>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    typedef typename colorspinor_order_mapper<Float,order,Ns,Nc>::type Q;
    Q field(a, (Float*)0, (float*)0, (Float**)ghost);
    PackGhostArg<Q> arg(field, ghost, a, parity, dagger);
    GenericPackGhostLauncher<Float,Ns,Nc,PackGhostArg<Q> > launch(arg, a);
    launch.apply(0);
  }

  template <typename Float, QudaFieldOrder order, int Ns>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {
    
    if (a.Ncolor() == 2) {
      genericPackGhost<Float,order,Ns,2>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 3) {
      genericPackGhost<Float,order,Ns,3>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 4) {
      genericPackGhost<Float,order,Ns,3>(ghost, a, parity, dagger);
    } else if (a.Ncolor() == 6) {
      genericPackGhost<Float,order,Ns,3>(ghost, a, parity, dagger);
    } else {
      errorQuda("Unsupported nColor = %d", a.Ncolor());
    }

  }

  template <typename Float, QudaFieldOrder order>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    if (a.Nspin() == 4) {
      genericPackGhost<Float,order,4>(ghost, a, parity, dagger);
    } else if (a.Nspin() == 2) {
      genericPackGhost<Float,order,2>(ghost, a, parity, dagger);
    } else if (a.Nspin() == 1) {
      genericPackGhost<Float,order,1>(ghost, a, parity, dagger);
    } else {
      errorQuda("Unsupported nSpin = %d", a.Nspin());
    }

  }

  template <typename Float>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    if (a.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      genericPackGhost<Float,QUDA_FLOAT2_FIELD_ORDER>(ghost, a, parity, dagger);
    } else if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      genericPackGhost<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(ghost, a, parity, dagger);
    } else {
      errorQuda("Unsupported field order = %d", a.FieldOrder());
    }

  }

  void genericPackGhost(void **ghost, const ColorSpinorField &a, const QudaParity parity, const int dagger) {

    if (a.FieldOrder() == QUDA_QOP_DOMAIN_WALL_FIELD_ORDER) {
      errorQuda("Field order %d not supported", a.FieldOrder());
    }

    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      genericPackGhost<double>(ghost, a, parity, dagger);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      genericPackGhost<float>(ghost, a, parity, dagger);
    } else {
      errorQuda("Unsupported precision %d", a.Precision());
    }

  }

} // namespace quda
