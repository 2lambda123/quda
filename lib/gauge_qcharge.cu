#include <quda_internal.h>
#include <tune_quda.h>
#include <gauge_field.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/gauge_qcharge.cuh>


namespace quda
{

#ifdef GPU_GAUGE_TOOLS

  template <typename Float, typename Arg> class QChargeCompute : TunableLocalParity
  {

    Arg &arg;
    const GaugeField &meta;

private:
    bool tuneGridDim() const { return true; }
    unsigned int minThreads() const { return arg.threads; }

public:
    QChargeCompute(Arg &arg, const GaugeField &meta) : arg(arg), meta(meta) {}
#ifdef JITIFY
    create_jitify_program("kernels/gauge_qcharge.cuh");
#endif
    virtual ~QChargeCompute() {}

    void apply(const hipStream_t &stream)
    {
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
        arg.result_h[0] = 0.;
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
#ifdef JITIFY
        using namespace jitify::reflection;
        jitify_error = program->kernel("quda::qChargeComputeKernel")
                         .instantiate(Type<Float>(), Type<Arg>())
                         .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                         .launch(arg);
#else
	LAUNCH_KERNEL(qChargeComputeKernel, tp, stream, arg, Float);
#endif
        qudaDeviceSynchronize();
      } else { // run the CPU code
        errorQuda("qChargeComputeKernel not supported on CPU");
      }
    }

    TuneKey tuneKey() const
    {
      std::stringstream aux;
      aux << "threads=" << arg.threads << ",prec=" << sizeof(Float);
      return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
    }

    long long flops() const { return 2 * arg.threads * (3 * 198 + 9); }
    long long bytes() const { return 2 * arg.threads * (6 * 18) * sizeof(Float); }
  }; // QChargeCompute

  template <typename Float, typename Gauge> void computeQCharge(const Gauge data, const GaugeField &Fmunu, Float &qChg)
  {
    QChargeArg<Float, Gauge> arg(data, Fmunu);
    QChargeCompute<Float, QChargeArg<Float, Gauge>> qChargeCompute(arg, Fmunu);
    qChargeCompute.apply(0);
    checkCudaError();
    comm_allreduce((double *)arg.result_h);
    qChg = arg.result_h[0];
  }

  template <typename Float> Float computeQCharge(const GaugeField &Fmunu)
  {

    Float qChg = 0.0;

    if (!Fmunu.isNative()) errorQuda("Topological charge computation only supported on native ordered fields");

    if (Fmunu.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_NO>::type Gauge;
      computeQCharge<Float>(Gauge(Fmunu), Fmunu, qChg);
    } else if (Fmunu.Reconstruct() == QUDA_RECONSTRUCT_12) {
      typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_12>::type Gauge;
      computeQCharge<Float>(Gauge(Fmunu), Fmunu, qChg);
    } else if (Fmunu.Reconstruct() == QUDA_RECONSTRUCT_8) {
      typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_8>::type Gauge;
      computeQCharge<Float>(Gauge(Fmunu), Fmunu, qChg);
    } else {
      errorQuda("Reconstruction type %d of gauge field not supported", Fmunu.Reconstruct());
    }

    return qChg;
  }

#endif // GPU_GAUGE_TOOLS

  double computeQCharge(const GaugeField &Fmunu)
  {

    double qChg = 0.0;
#ifdef GPU_GAUGE_TOOLS

    if (!Fmunu.isNative()) errorQuda("Order %d with %d reconstruct not supported", Fmunu.Order(), Fmunu.Reconstruct());

    if (Fmunu.Precision() == QUDA_SINGLE_PRECISION) {
      qChg = computeQCharge<float>(Fmunu);
    } else if (Fmunu.Precision() == QUDA_DOUBLE_PRECISION) {
      qChg = computeQCharge<double>(Fmunu);
    } else {
      errorQuda("Precision %d not supported", Fmunu.Precision());
    }
#else
    errorQuda("Gauge tools are not built");
#endif // GPU_GAUGE_TOOLS
    return qChg;
  }
} // namespace quda
