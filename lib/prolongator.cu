#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <typeinfo>

#include <multigrid_helper.cuh>

namespace quda {

#ifdef GPU_MULTIGRID
  using namespace quda::colorspinor;
  
  /** 
      Kernel argument struct
  */
  template <typename Out, typename In, typename Rotator, int fineSpin, int coarseSpin>
  struct ProlongateArg {
    Out out;
    const In in;
    const Rotator V;
    const int *geo_map;  // need to make a device copy of this
    const spin_mapper<fineSpin,coarseSpin> spin_map;
    ProlongateArg(Out &out, const In &in, const Rotator &V, 
		  const int *geo_map) : 
      out(out), in(in), V(V), geo_map(geo_map), spin_map()
    { }

    ProlongateArg(const ProlongateArg<Out,In,Rotator,fineSpin,coarseSpin> &arg) :
      out(arg.out), in(arg.in), V(arg.V), geo_map(arg.geo_map), spin_map() {
    }
  };

  /**
     Applies the grid prolongation operator (coarse to the top level fine grid)
  */

  template <typename Float, int coarseSpin, int coarseColor, class Coarse>
  __device__ __host__ inline void prolongate2TopLevelStaggered(complex<Float> out[coarseSpin*coarseColor], const Coarse &in, 
					     int parity_coarse, int x_coarse_cb) {
    for (int p = 0; p < coarseSpin; p++) { //coarse-grid spin is transformed into the fine-grid parity index
      for (int c = 0; c < coarseColor; c++) {
        int staggered_coarse_spin = p;
        out[p*coarseColor+c] = in(parity_coarse, x_coarse_cb, staggered_coarse_spin, c); 
      }
    }
    return;
  }

  /**
     Applies the grid prolongation operator (coarse to fine spin dof)
  */

  template <typename Float, int fineSpin, int coarseColor, class Coarse, typename S>
  __device__ __host__ inline void prolongate(complex<Float> out[fineSpin*coarseColor], const Coarse &in, 
					     int parity_coarse, int x_coarse_cb, const S& spin_map) {
    for (int s=0; s<fineSpin; s++) {
      for (int c=0; c<coarseColor; c++) {
	out[s*coarseColor+c] = in(parity_coarse, x_coarse_cb, spin_map(s), c);
      }
    }
    return;
  }


  /**
     Rotates from the coarse-color basis into the fine-color basis.  This
     is the second step of applying the prolongator (only for the prolongation to the top level grid!).
  */
  template <typename Float, int coarseSpin, int fineColor, int coarseColor, class FineColor, class Rotator>
  __device__ __host__ inline void rotateFineColorTopLevelStaggered(FineColor &out, const complex<Float> in[coarseSpin*coarseColor],
						  const Rotator &V, int parity, int x_cb) {
    for (int i=0; i<out.Ncolor(); i++) out(parity, x_cb, 0, i) = 0.0;

    int staggered_coarse_spin = parity;

    for (int i=0; i<fineColor; i++) {
      for (int j=0; j<coarseColor; j++) { 
	// V is a ColorMatrixField with internal dimensions Ns * Nc * Nvec
 	out(parity, x_cb, 0, i) += V(parity, x_cb, 0, i, j) * in[staggered_coarse_spin*coarseColor + j];
      }
    }
  }

  template <typename Float, int fineSpin, int fineColor, int coarseColor, class FineColor, class Rotator>
  __device__ __host__ inline void rotateFineColor(FineColor &out, const complex<Float> in[fineSpin*coarseColor],
						  const Rotator &V, int parity, int x_cb) {
    for (int s=0; s<out.Nspin(); s++) 
      for (int i=0; i<out.Ncolor(); i++) out(parity, x_cb, s, i) = 0.0;
    
    for (int i=0; i<fineColor; i++) {
      for (int s=0; s<fineSpin; s++) {
	for (int j=0; j<coarseColor; j++) { 
	  // V is a ColorMatrixField with internal dimensions Ns * Nc * Nvec
	  out(parity, x_cb, s, i) += V(parity, x_cb, s, i, j) * in[s*coarseColor + j];
	}
      }
    }
  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  void Prolongate(Arg &arg) {
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<arg.out.Volume()/2; x_cb++) {

        int x = parity*arg.out.Volume()/2 + x_cb;
        int x_coarse = arg.geo_map[x];
        int parity_coarse = (x_coarse >= arg.in.Volume()/2) ? 1 : 0;
        int x_coarse_cb = x_coarse - parity_coarse*arg.in.Volume()/2;

        if(fineSpin == 1)//staggered top level
        {
          //if(coarseSpin != 2) errorQuda("\nIncorrect coarse spin number\n"); 
          complex<Float> tmp[coarseSpin*coarseColor];
	  prolongate2TopLevelStaggered<Float,coarseSpin,coarseColor>(tmp, arg.in, parity_coarse, x_coarse_cb);
	  rotateFineColorTopLevelStaggered<Float,coarseSpin,fineColor,coarseColor>(arg.out, tmp, arg.V, parity, x_cb);
        }
        else//also for staggered if the fine grid is NOT a top level grid.
        {
          complex<Float> tmp[fineSpin*coarseColor];
	  prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity_coarse, x_coarse_cb, arg.spin_map);
	  rotateFineColor<Float,fineSpin,fineColor,coarseColor>(arg.out, tmp, arg.V, parity, x_cb);
        }
      }
    }
  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  __global__ void ProlongateKernel(Arg arg) {

    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity=threadIdx.y; //parity is within the block
    if (x_cb >= arg.out.Volume()/2) return;

    int x = parity*arg.out.Volume()/2 + x_cb;
    int x_coarse = arg.geo_map[x];
    int parity_coarse = (x_coarse >= arg.in.Volume()/2) ? 1 : 0;
    int x_coarse_cb = x_coarse - parity_coarse*arg.in.Volume()/2;
    if(fineSpin == 1)
    {
      complex<Float> tmp[2*coarseColor];
      prolongate2TopLevelStaggered<Float,coarseSpin,coarseColor>(tmp, arg.in, parity_coarse, x_coarse_cb);
      rotateFineColorTopLevelStaggered<Float,coarseSpin,fineColor,coarseColor>(arg.out, tmp, arg.V, parity, x_cb);
    }
    else
    {
      complex<Float> tmp[fineSpin*coarseColor];
      prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity_coarse, x_coarse_cb, arg.spin_map);
      rotateFineColor<Float,fineSpin,fineColor,coarseColor>(arg.out, tmp, arg.V, parity, x_cb);
    }
  }
  
  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  class ProlongateLaunch : public Tunable {

  protected:
    Arg &arg;
    QudaFieldLocation location;
    char vol[TuneKey::volume_n];

    long long flops() const { return 0; }
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.out.Volume()/2; } // fine parity is the block y dimension

  public:
    ProlongateLaunch(Arg &arg, const ColorSpinorField &fine, const ColorSpinorField &coarse, 
		     const QudaFieldLocation location) : arg(arg), location(location) { 
      strcpy(vol, fine.VolString());
      strcat(vol, ",");
      strcat(vol, coarse.VolString());

      strcpy(aux, fine.AuxString());
      strcat(aux, ",");
      strcat(aux, coarse.AuxString());
    }

    virtual ~ProlongateLaunch() { }

    void apply(const hipStream_t &stream) {
      if (location == QUDA_CPU_FIELD_LOCATION) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,coarseColor>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	tp.block.y = 2; // need factor of two for parity with in the block
	ProlongateKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,Arg> 
	  <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(vol, typeid(*this).name(), aux);
    }

    void initTuneParam(TuneParam &param) const {
      Tunable::initTuneParam(param);
      param.grid = dim3( ((arg.out.Volume()/2)+param.block.x-1) / param.block.x, 1, 1);
    }

    /** sets default values for when tuning is disabled */
    void defaultTuneParam(TuneParam &param) const {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( ((arg.out.Volume()/2)+param.block.x-1) / param.block.x, 1, 1);
    }

    long long bytes() const {
      return arg.in.Bytes() + arg.out.Bytes() + arg.V.Bytes() + arg.out.Volume()*sizeof(int);
    }

  };

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  const int *fine_to_coarse) {

    typedef FieldOrderCB<Float,fineSpin,fineColor,1,order> fineSpinor;
    typedef FieldOrderCB<Float,coarseSpin,coarseColor,1,order> coarseSpinor;
    typedef FieldOrderCB<Float,fineSpin,fineColor,coarseColor,order> packedSpinor;
    typedef ProlongateArg<fineSpinor,coarseSpinor,packedSpinor,fineSpin,coarseSpin> Arg;

    fineSpinor   Out(const_cast<ColorSpinorField&>(out));
    coarseSpinor In(const_cast<ColorSpinorField&>(in));
    packedSpinor V(const_cast<ColorSpinorField&>(v));

    Arg arg(Out, In, V, fine_to_coarse);
    ProlongateLaunch<Float, fineSpin, fineColor, coarseSpin, coarseColor, Arg> prolongator(arg, out, in, Location(out, in, v));
    prolongator.apply(0);

    if (Location(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
  }


  template <typename Float, int fineSpin, int fineColor, int coarseSpin, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int nVec, const int *fine_to_coarse, const int *spin_map) {
    // first check that the spin_map matches the spin_mapper  
    if(spin_map != NULL)
    {
      spin_mapper<fineSpin,coarseSpin> mapper;
      for (int s=0; s<fineSpin; s++) 
        if (mapper(s) != spin_map[s]) errorQuda("Spin map does not match spin_mapper");
    }

    if (nVec == 2) {
      Prolongate<Float,fineSpin,fineColor,coarseSpin,2,order>(out, in, v, fine_to_coarse);
    } else if (nVec == 24) {
      Prolongate<Float,fineSpin,fineColor,coarseSpin,24,order>(out, in, v, fine_to_coarse);
    } else if (nVec == 48) {
      Prolongate<Float,fineSpin,fineColor,coarseSpin,48,order>(out, in, v, fine_to_coarse);
    } else {
      errorQuda("Unsupported nVec %d", nVec);
    }
  }

  template <typename Float, int fineSpin, int fineColor, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
                  int nVec, const int *fine_to_coarse, const int *spin_map) {

    if (in.Nspin() == 2) {
      Prolongate<Float,fineSpin,fineColor,2,order>(out, in, v, nVec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Coarse spin != 2 is not supported (%d)", in.Nspin());
    }
  }

  template <typename Float, int fineSpin, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {

    if (out.Ncolor() == 3) {
      Prolongate<Float,fineSpin,3,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Ncolor() == 2) {
      Prolongate<Float,fineSpin,2,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Ncolor() == 24) {
      Prolongate<Float,fineSpin,24,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Ncolor() == 48) {
      Prolongate<Float,fineSpin,48,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported nColor %d", out.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {

    if (out.Nspin() == 4) {
      Prolongate<Float,4,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Nspin() == 2) {
      Prolongate<Float,2,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Nspin() == 1) {
      Prolongate<Float,1,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported nSpin %d", out.Nspin());
    }
  }

  template <typename Float>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {

    if (out.FieldOrder() != in.FieldOrder() || out.FieldOrder() != v.FieldOrder())
      errorQuda("Field orders do not match (out=%d, in=%d, v=%d)", 
		out.FieldOrder(), in.FieldOrder(), v.FieldOrder());

    if (out.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      Prolongate<Float,QUDA_FLOAT2_FIELD_ORDER>
	(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      Prolongate<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>
	(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported field type %d", out.FieldOrder());
    }
  }
#endif // GPU_MULTIGRID

  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {
#ifdef GPU_MULTIGRID
    if (out.Precision() != in.Precision() || v.Precision() != in.Precision()) 
      errorQuda("Precision mismatch out=%d in=%d v=%d", out.Precision(), in.Precision(), v.Precision());

    if (out.Precision() == QUDA_DOUBLE_PRECISION) {
      Prolongate<double>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Precision() == QUDA_SINGLE_PRECISION) {
      Prolongate<float>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported precision %d", out.Precision());
    }

    if (Location(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
#else
    errorQuda("Multigrid has not been built");
#endif
  }

} // end namespace quda
