#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <typeinfo>

namespace quda {

  using namespace quda::colorspinor;
  
  /** 
      Kernel argument struct
  */
  template <typename Out, typename In, typename Rotator, int fineSpin>
  struct ProlongateArg {
    Out out;
    const In in;
    const Rotator V;
    const int *geo_map;  // need to make a device copy of this
    int spin_map[fineSpin];
    ProlongateArg(Out &out, const In &in, const Rotator &V, 
		  const int *geo_map, const int *spin_map) : 
      out(out), in(in), V(V), geo_map(geo_map)  {
      for (int s=0; s<fineSpin; s++) this->spin_map[s] = spin_map[s];
    }

    ProlongateArg(const ProlongateArg<Out,In,Rotator,fineSpin> &arg) :
      out(arg.out), in(arg.in), V(arg.V), geo_map(arg.geo_map) {
      for (int s=0; s<fineSpin; s++) this->spin_map[s] = arg.spin_map[s];
    }
  };

  /**
     Applies the grid prolongation operator (coarse to fine)
  */
  template <typename Float, int fineSpin, int coarseColor, class Coarse>
  __device__ __host__ inline void prolongate(complex<Float> out[fineSpin*coarseColor], const Coarse &in, 
					     int parity, int x_cb, const int *geo_map, const int *spin_map, int fineVolume) {
    int x = parity*fineVolume/2 + x_cb;
    int x_coarse = geo_map[x];
    int parity_coarse = (x_coarse >= in.Volume()/2) ? 1 : 0;
    int x_coarse_cb = x_coarse - parity_coarse*in.Volume()/2;

    for (int s=0; s<fineSpin; s++) {
      for (int c=0; c<coarseColor; c++) {
	out[s*coarseColor+c] = in(parity_coarse, x_coarse_cb, spin_map[s], c);
      }
    }
  }

  /**
     Rotates from the coarse-color basis into the fine-color basis.  This
     is the second step of applying the prolongator.
  */
  template <typename Float, int fineSpin, int fineColor, int coarseColor, class FineColor, class Rotator>
  __device__ __host__ inline void rotateFineColor(FineColor &out, const complex<Float> in[fineSpin*coarseColor],
						  const Rotator &V, int parity, int x_cb) {
    for (int s=0; s<out.Nspin(); s++) 
      for (int i=0; i<out.Ncolor(); i++) out(parity, x_cb, s, i) = 0.0;
    
    for (int i=0; i<fineColor; i++) {
      for (int s=0; s<fineSpin; s++) {
	for (int j=0; j<coarseColor; j++) { 
	  // V is a ColorMatrixField with internal dimensions Ns * Nc * Nvec
	  out(parity, x_cb, s, i) += V(parity, x_cb, s, i, j) * in[s*coarseColor + j];
	}
      }
    }

  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  void Prolongate(Arg &arg) {
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<arg.out.Volume()/2; x_cb++) {
	complex<Float> tmp[fineSpin*coarseColor];
	prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity, x_cb, arg.geo_map, arg.spin_map, arg.out.Volume());
	rotateFineColor<Float,fineSpin,fineColor,coarseColor>(arg.out, tmp, arg.V, parity, x_cb);
      }
    }
  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  __global__ void ProlongateKernel(Arg arg) {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity=threadIdx.y; //parity is within the block
    if (x_cb >= arg.out.Volume()/2) return;

    complex<Float> tmp[fineSpin*coarseColor];
    prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity, x_cb, arg.geo_map, arg.spin_map, arg.out.Volume());
    rotateFineColor<Float,fineSpin,fineColor,coarseColor>(arg.out, tmp, arg.V, parity, x_cb);
  }
  
  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  class ProlongateLaunch : public Tunable {

  protected:
    Arg &arg;
    QudaFieldLocation location;

    long long flops() const { return 0; }
    long long bytes() const { return 0; }
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.out.Volume()/2; } // fine parity is the block y dimension

  public:
    ProlongateLaunch(Arg &arg, const QudaFieldLocation location) 
      : arg(arg), location(location) { }
    virtual ~ProlongateLaunch() { }

    void apply(const hipStream_t &stream) {
      if (location == QUDA_CPU_FIELD_LOCATION) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,coarseColor>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	tp.block.y = 2; // need factor of two for parity with in the block
	ProlongateKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,Arg> 
	  <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
      }
    }

    TuneKey tuneKey() const {
      std::stringstream vol, aux;
      vol << arg.out.Volume(); 
      // FIXME should use stride here
      aux << "out_stride=" << arg.out.Volume() << ",in_stride=" << arg.in.Volume();
      //return TuneKey(vol.str(), typeid(*this).name(), aux.str());
      return TuneKey("fixme", typeid(*this).name(), "fixme");
    }

    void initTuneParam(TuneParam &param) const {
      Tunable::initTuneParam(param);
      param.grid = dim3( ((arg.out.Volume()/2)+param.block.x-1) / param.block.x, 1, 1);
    }

    /** sets default values for when tuning is disabled */
    void defaultTuneParam(TuneParam &param) const {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( ((arg.out.Volume()/2)+param.block.x-1) / param.block.x, 1, 1);
    }

  };

  template <typename Float, int fineSpin, int fineColor, int coarseColor, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  const int *fine_to_coarse, const int *spin_map) {
    if (in.Nspin() != 2) errorQuda("coarseSpin != 2 not supported");
    const int coarseSpin = 2;

    typedef FieldOrderCB<Float,fineSpin,fineColor,1,order> fineSpinor;
    typedef FieldOrderCB<Float,coarseSpin,coarseColor,1,order> coarseSpinor;
    typedef FieldOrderCB<Float,fineSpin,fineColor,coarseColor,order> packedSpinor;
    typedef ProlongateArg<fineSpinor,coarseSpinor,packedSpinor,fineSpin> Arg;

    fineSpinor   Out(const_cast<ColorSpinorField&>(out));
    coarseSpinor In(const_cast<ColorSpinorField&>(in));
    packedSpinor V(const_cast<ColorSpinorField&>(v));

    Arg arg(Out, In, V, fine_to_coarse,spin_map);
    ProlongateLaunch<Float, fineSpin, fineColor, coarseSpin, coarseColor, Arg> prolongator(arg, Location(out, in, v));
    prolongator.apply(0);

    if (Location(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
  }


  template <typename Float, int fineSpin, int fineColor, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int nVec, const int *fine_to_coarse, const int *spin_map) {

    if (nVec == 2) {
      Prolongate<Float,fineSpin,fineColor,2,order>(out, in, v, fine_to_coarse, spin_map);
    } else if (nVec == 24) {
      Prolongate<Float,fineSpin,fineColor,24,order>(out, in, v, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported nVec %d", nVec);
    }
  }

  template <typename Float, int fineSpin, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {

    if (out.Ncolor() == 3) {
      Prolongate<Float,fineSpin,3,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Ncolor() == 2) {
      Prolongate<Float,fineSpin,2,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Ncolor() == 24) {
      Prolongate<Float,fineSpin,24,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported nColor %d", out.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder order>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {

    if (out.Nspin() == 4) {
      Prolongate<Float,4,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Nspin() == 2) {
      Prolongate<Float,2,order>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported nSpin %d", out.Nspin());
    }
  }

  template <typename Float>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {

    if (out.FieldOrder() != in.FieldOrder() || out.FieldOrder() != v.FieldOrder())
      errorQuda("Field orders do not match (out=%d, in=%d, v=%d)", 
		out.FieldOrder(), in.FieldOrder(), v.FieldOrder());

    if (out.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      Prolongate<Float,QUDA_FLOAT2_FIELD_ORDER>
	(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      Prolongate<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>
	(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported field type %d", out.FieldOrder());
    }
  }

  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map) {
    if (out.Precision() != in.Precision() || v.Precision() != in.Precision()) 
      errorQuda("Precision mismatch out=%d in=%d v=%d", out.Precision(), in.Precision(), v.Precision());

    if (out.Precision() == QUDA_DOUBLE_PRECISION) {
      Prolongate<double>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else if (out.Precision() == QUDA_SINGLE_PRECISION) {
      Prolongate<float>(out, in, v, Nvec, fine_to_coarse, spin_map);
    } else {
      errorQuda("Unsupported precision %d", out.Precision());
    }

    if (Location(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
  }

} // end namespace quda
