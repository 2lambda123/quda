#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <typeinfo>

namespace quda {

  using namespace quda::colorspinor;

  // Applies the grid prolongation operator (coarse to fine)
  template <class FineSpinor, class CoarseSpinor>
  void prolongate(FineSpinor &out, const CoarseSpinor &in, const int *geo_map, const int *spin_map) {

    for (int x=0; x<out.Volume(); x++) {
      for (int s=0; s<out.Nspin(); s++) {
	for (int c=0; c<out.Ncolor(); c++) {
	  out(x, s, c) = in(geo_map[x], spin_map[s], c);
	}
      }
    }

  }

  /*
  // Applies the grid prolongation operator (coarse to fine)
  template <class FineSpinor, class CoarseSpinor>
  __global__ void prolongate(FineSpinor &out, const CoarseSpinor &in, const int *geo_map, const int *spin_map) {

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    for (int s=0; s<out.Nspin(); s++) {
      for (int c=0; c<out.Ncolor(); c++) {
	out(x, s, c) = in(geo_map[x], spin_map[s], c);
      }
    }

    }*/
  
  /*
    Rotates from the coarse-color basis into the fine-color basis.  This
    is the second step of applying the prolongator.
  */
  template <class FineColor, class CoarseColor, class Rotator>
  void rotateFineColor(FineColor &out, const CoarseColor &in, const Rotator &V) {

    for(int x=0; x<in.Volume(); x++) {

      for (int s=0; s<out.Nspin(); s++) for (int i=0; i<out.Ncolor(); i++) out(x, s, i) = 0.0;

      for (int i=0; i<out.Ncolor(); i++) {
	for (int s=0; s<in.Nspin(); s++) {
	  for (int j=0; j<in.Ncolor(); j++) { 
	    // V is a ColorMatrixField with internal dimensions Ns * Nc * Nvec
	    out(x, s, i) += V(x, s, i, j) * in(x, s, j);
	  }
	}
      }
      
    }

  }

  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  ColorSpinorField &tmp, int Nvec, const int *geo_map, const int *spin_map) {

    if (out.Precision() == QUDA_DOUBLE_PRECISION) {
      FieldOrder<double> *outOrder = createOrder<double>(out);
      FieldOrder<double> *inOrder = createOrder<double>(in);
      FieldOrder<double> *vOrder = createOrder<double>(v, Nvec);
      FieldOrder<double> *tmpOrder = createOrder<double>(tmp);
      prolongate(*tmpOrder, *inOrder, geo_map, spin_map);
      rotateFineColor(*outOrder, *tmpOrder, *vOrder);
      delete outOrder;
      delete inOrder;
      delete vOrder;
      delete tmpOrder;
    } else {
      FieldOrder<float> *outOrder = createOrder<float>(out);
      FieldOrder<float> *inOrder = createOrder<float>(in);
      FieldOrder<float> *vOrder = createOrder<float>(v, Nvec);
      FieldOrder<float> *tmpOrder = createOrder<float>(tmp);
      prolongate(*tmpOrder, *inOrder, geo_map, spin_map);
      rotateFineColor(*outOrder, *tmpOrder, *vOrder);
      delete outOrder;
      delete inOrder;
      delete vOrder;
      delete tmpOrder;
    }

  }

} // end namespace quda
