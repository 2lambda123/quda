#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>

namespace quda {

  using namespace colorspinor;

  // Random number insertion over all field elements
  template <class T>
  void random(T &t) {
    for (int x=0; x<t.Volume(); x++) {
      for (int s=0; s<t.Nspin(); s++) {
	for (int c=0; c<t.Ncolor(); c++) {
	  t(x,s,c).real(comm_drand());
	  t(x,s,c).imag(comm_drand());
	}
      }
    }
  }

  // Create a point source at spacetime point x, spin s and colour c
  template <class T>
  void point(T &t, int x, int s, int c) { t(x, s, c, 0) = 1.0; }

  void genericSource(cpuColorSpinorField &a, QudaSourceType sourceType, int x, int s, int c) {

    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      FieldOrder<double> *A = createOrder<double>(a);
      if (sourceType == QUDA_RANDOM_SOURCE) random(*A);
      else if (sourceType == QUDA_POINT_SOURCE) point(*A, x, s, c);
      else errorQuda("Unsupported source type %d", sourceType);
      delete A;
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      FieldOrder<float> *A = createOrder<float>(a);
      if (sourceType == QUDA_RANDOM_SOURCE) random(*A);
      else if (sourceType == QUDA_POINT_SOURCE) point(*A, x, s, c);
      else errorQuda("Unsupported source type %d", sourceType);
      delete A;
    } else {
      errorQuda("Precision not supported");
    }

  }


  template <class U, class V>
  int compareSpinor(const U &u, const V &v, const int tol) {
    int fail_check = 16*tol;
    int *fail = new int[fail_check];
    for (int f=0; f<fail_check; f++) fail[f] = 0;

    int N = 2*u.Nspin()*u.Ncolor();
    int *iter = new int[N];
    for (int i=0; i<N; i++) iter[i] = 0;

    for (int x=0; x<u.Volume(); x++) {
      //int test[u.Nspin()*u.Ncolor()*2];
      
      //printf("x = %d (", x);
      for (int s=0; s<u.Nspin(); s++) {
	for (int c=0; c<u.Ncolor(); c++) {
	  for (int z=0; z<2; z++) {
	    int j = (s*u.Ncolor() + c)*2+z;
	    //test[j] = 0;

	    double diff = z==0 ? fabs(u(x,s,c,z).real() - v(x,s,c,z).real()) :
	      fabs(u(x,s,c).imag() - v(x,s,c).imag());

	    for (int f=0; f<fail_check; f++) {
	      if (diff > pow(10.0,-(f+1)/(double)tol)) {
		fail[f]++;
	      }
	    }

	    if (diff > 1e-3) {
	      iter[j]++;
	      //printf("%d %d %e %e\n", x, j, u(x,s,c,z), v(x,s,c,z));
	      //test[j] = 1;
	    }
	    //printf("%d ", test[j]);

	  }
	}
      }
      //      printf(")\n");
    }

    for (int i=0; i<N; i++) printfQuda("%d fails = %d\n", i, iter[i]);
    
    int accuracy_level =0;
    for (int f=0; f<fail_check; f++) {
      if (fail[f] == 0) accuracy_level = f+1;
    }

    for (int f=0; f<fail_check; f++) {
      printfQuda("%e Failures: %d / %d  = %e\n", pow(10.0,-(f+1)/(double)tol), 
		 fail[f], u.Volume()*N, fail[f] / (double)(u.Volume()*N));
    }
  
    delete []iter;
    delete []fail;
  
    return accuracy_level;
  }

  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      FieldOrder<double> *A = createOrder<double>(a);
      if (b.Precision() == QUDA_DOUBLE_PRECISION) {
	FieldOrder<double> *B = createOrder<double>(b);
	ret = compareSpinor(*A, *B, tol);
	delete B;
      } else {
	FieldOrder<float> *B = createOrder<float>(b);
	ret = compareSpinor(*A, *B, tol);
	delete B;
      }
      delete A;
    } else {
      FieldOrder<float> *A = createOrder<float>(a);
      if (b.Precision() == QUDA_DOUBLE_PRECISION) {
	FieldOrder<double> *B = createOrder<double>(b);
	ret = compareSpinor(*A, *B, tol);
	delete B;
      } else {
	FieldOrder<float> *B = createOrder<float>(b);
	ret = compareSpinor(*A, *B, tol);
	delete B;
      }
      delete A;
    }
    return ret;
  }


  template <class Order>
  void print_vector(const Order &o, unsigned int x) {

    for (int s=0; s<o.Nspin(); s++) {
      std::cout << "x = " << x << ", s = " << s << ", { ";
      for (int c=0; c<o.Ncolor(); c++) {
	std::cout << " ( " << o(x, s, c, 0) << " , " ;
	if (c<o.Ncolor()-1) std::cout << o(x, s, c, 1) << " ) ," ;
	else std::cout << o(x, s, c, 1) << " ) " ;
      }
      std::cout << " } " << std::endl;
    }

  }

  // print out the vector at volume point x
  void genericPrintVector(cpuColorSpinorField &a, unsigned int x) {
  
    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      FieldOrder<double> *A = createOrder<double>(a);
      print_vector(*A, x);
      delete A;
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      FieldOrder<float> *A = createOrder<float>(a);
      print_vector(*A, x);
      delete A;
    } else {
      errorQuda("Precision %d not implemented", a.Precision()); 
    }
    
  }



} // namespace quda
