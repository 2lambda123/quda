#include <gauge_field.h>
#include <gauge_field_order.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <clover_field.h>
#include <clover_field_order.h>
#include <dslash_helper.cuh>
#include <index_helper.cuh>
#include <dslash_quda.h>
#include <color_spinor.h>
#include <worker.h>

namespace quda {
#include <dslash_events.cuh>
#include <dslash_policy.cuh>
}

#include <kernels/dslash_wilson_clover.cuh>

/**
   This is the Wilson-clover preconditioned linear operator
*/

namespace quda {

  template <typename Float, int nDim, int nColor, typename Arg>
  class WilsonClover : public Dslash<Float> {

  protected:
    Arg &arg;
    const ColorSpinorField &in;
    using Dslash<Float>::setParam;
    using Dslash<Float>::launch;

  public:

    WilsonClover(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in)
      : Dslash<Float>(arg, out, in), arg(arg), in(in)
    {  }

    virtual ~WilsonClover() { }

    template <bool dagger, bool xpay>
    inline void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

      if (in.Location() == QUDA_CPU_FIELD_LOCATION) {
        errorQuda("Not implemented");
      } else {
        switch(arg.kernel_type) {
        case INTERIOR_KERNEL:
          launch(wilsonCloverGPU<Float,nDim,nColor,dagger,xpay,INTERIOR_KERNEL,Arg>, tp, arg, stream); break;
        case EXTERIOR_KERNEL_X:
          launch(wilsonCloverGPU<Float,nDim,nColor,dagger,xpay,EXTERIOR_KERNEL_X,Arg>, tp, arg, stream); break;
        case EXTERIOR_KERNEL_Y:
          launch(wilsonCloverGPU<Float,nDim,nColor,dagger,xpay,EXTERIOR_KERNEL_Y,Arg>, tp, arg, stream); break;
        case EXTERIOR_KERNEL_Z:
          launch(wilsonCloverGPU<Float,nDim,nColor,dagger,xpay,EXTERIOR_KERNEL_Z,Arg>, tp, arg, stream); break;
        case EXTERIOR_KERNEL_T:
          launch(wilsonCloverGPU<Float,nDim,nColor,dagger,xpay,EXTERIOR_KERNEL_T,Arg>, tp, arg, stream); break;
        case EXTERIOR_KERNEL_ALL:
          launch(wilsonCloverGPU<Float,nDim,nColor,dagger,xpay,EXTERIOR_KERNEL_ALL,Arg>, tp, arg, stream); break;
        default: errorQuda("Unexpected kernel type %d", arg.kernel_type);
        }
      }
    }

    void apply(const hipStream_t &stream) {
      setParam(arg);
      if (arg.xpay) arg.dagger ? apply<true, true>(stream) : apply<false, true>(stream);
      else          arg.dagger ? apply<true,false>(stream) : apply<false,false>(stream);
    }

    long long flops() const {
      int clover_flops = 504;
      long long flops = Dslash<Float>::flops();
      switch(arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	flops += clover_flops * in.GhostFace()[arg.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	flops += clover_flops * 2 * (in.GhostFace()[0]+in.GhostFace()[1]+in.GhostFace()[2]+in.GhostFace()[3]);
	break;
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
	flops += clover_flops * in.Volume();	  

	if (arg.kernel_type == KERNEL_POLICY) break;
	// now correct for flops done by exterior kernel
	long long ghost_sites = 0;
	for (int d=0; d<4; d++) if (arg.commDim[d]) ghost_sites += 2 * in.GhostFace()[d];
	flops -= clover_flops * ghost_sites;
	
	break;
      }
      return flops;
    }

    long long bytes() const {
      bool isFixed = (in.Precision() == sizeof(short) || in.Precision() == sizeof(char)) ? true : false;
      int clover_bytes = 72 * in.Precision() + (isFixed ? 2*sizeof(float) : 0);

      long long bytes = Dslash<Float>::bytes();
      switch(arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	bytes += clover_bytes * 2 * in.GhostFace()[arg.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	bytes += clover_bytes * 2 * (in.GhostFace()[0]+in.GhostFace()[1]+in.GhostFace()[2]+in.GhostFace()[3]);
	break;
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
	bytes += clover_bytes*in.Volume();

	if (arg.kernel_type == KERNEL_POLICY) break;
	// now correct for bytes done by exterior kernel
	long long ghost_sites = 0;
	for (int d=0; d<4; d++) if (arg.commDim[d]) ghost_sites += 2*in.GhostFace()[d];
	bytes -= clover_bytes * ghost_sites;
	
	break;
      }

      return bytes;
    }

    TuneKey tuneKey() const { return TuneKey(in.VolString(), typeid(*this).name(), Dslash<Float>::aux[arg.kernel_type]); }
  };

  template <typename Float, int nColor, QudaReconstructType recon>
  void ApplyWilsonClover(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const CloverField &A,
			 double kappa, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override)
  {
    constexpr int nDim = 4;
    constexpr bool dynamic_clover = false;
    WilsonCloverArg<Float,nColor,recon,dynamic_clover> arg(out, in, U, A, kappa, x, parity, dagger, comm_override);
    WilsonClover<Float,nDim,nColor,WilsonCloverArg<Float,nColor,recon,dynamic_clover> > wilson(arg, out, in);

    TimeProfile profile("dummy");
    DslashPolicyTune<decltype(wilson)> policy(wilson, const_cast<cudaColorSpinorField*>(static_cast<const cudaColorSpinorField*>(&in)),
                                              in.VolumeCB(), in.GhostFaceCB(), profile);
    policy.apply(0);

    checkCudaError();
  }

  // template on the gauge reconstruction
  template <typename Float, int nColor>
  void ApplyWilsonClover(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const CloverField &A,
			 double kappa, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override)
  {
    if (U.Reconstruct()== QUDA_RECONSTRUCT_NO) {
      ApplyWilsonClover<Float,nColor,QUDA_RECONSTRUCT_NO>(out, in, U, A, kappa, x, parity, dagger, comm_override);
#if 0
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_12) {
      ApplyWilsonClover<Float,nColor,QUDA_RECONSTRUCT_12>(out, in, U, A, kappa, x, parity, dagger, comm_override);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_8) {
      ApplyWilsonClover<Float,nColor,QUDA_RECONSTRUCT_8>(out, in, U, A, kappa, x, parity, dagger, comm_override);
#endif
    } else {
      errorQuda("Unsupported reconstruct type %d\n", U.Reconstruct());
    }
  }

  // template on the number of colors
  template <typename Float>
  void ApplyWilsonClover(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const CloverField &A,
			 double kappa, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override)
  {
    if (in.Ncolor() == 3) {
      ApplyWilsonClover<Float,3>(out, in, U, A, kappa, x, parity, dagger, comm_override);
    } else {
      errorQuda("Unsupported number of colors %d\n", U.Ncolor());
    }
  }

  // Apply the Wilson-clover operator
  // out(x) = M*in = A(x)^{-1} (-kappa*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu))
  // Uses the kappa normalization for the Wilson operator.
  void ApplyWilsonClover(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
			 const CloverField &A, double kappa, const ColorSpinorField &x, int parity, bool dagger,
			 const int *comm_override)
  {
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());
    
    // check all precisions match
    checkPrecision(out, in, U, A);

    // check all locations match
    checkLocation(out, in, U, A);

    if (U.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyWilsonClover<double>(out, in, U, A, kappa, x, parity, dagger, comm_override);
    } else if (U.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyWilsonClover<float>(out, in, U, A, kappa, x, parity, dagger, comm_override);
    } else if (U.Precision() == QUDA_HALF_PRECISION) {
      ApplyWilsonClover<short>(out, in, U, A, kappa, x, parity, dagger, comm_override);
    } else if (U.Precision() == QUDA_QUARTER_PRECISION) {
      ApplyWilsonClover<char>(out, in, U, A, kappa, x, parity, dagger, comm_override);
    } else {
      errorQuda("Unsupported precision %d\n", U.Precision());
    }

    in.bufferIndex = (1 - in.bufferIndex);
  }


} // namespace quda
