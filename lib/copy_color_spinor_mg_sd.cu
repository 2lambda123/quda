#define BUILD_LIMITED_COPY
#include <copy_color_spinor.cuh>

namespace quda {
  
  void copyGenericColorSpinorMGSD(ColorSpinorField &dst, const ColorSpinorField &src, 
				  QudaFieldLocation location, void *Dst, void *Src, 
				  void *dstNorm, void *srcNorm) {

    float *dst_ptr = static_cast<float*>(Dst);
    double *src_ptr = static_cast<double*>(Src);

    INSTANTIATE_COLOR;

  }

} // namespace quda
