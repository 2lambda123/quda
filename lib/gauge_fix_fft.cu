#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <unitarization_links.h>
#include <atomic.cuh>
#include <index_helper.cuh>

#include <hipfft/hipfft.h>
#include <CUFFT_Plans.h>
#include <instantiate.h>

#include <tunable_reduction.h>
#include <kernels/gauge_fix_fft.cuh>

namespace quda {

//UNCOMMENT THIS IF YOU WAN'T TO USE LESS MEMORY
#define GAUGEFIXING_DONT_USE_GX
//Without using the precalculation of g(x),
//we loose some performance, because Delta(x) is written in normal lattice coordinates need for the FFTs
//and the gauge array in even/odd format

#ifdef HOST_DEBUG
#ifdef GAUGEFIXING_DONT_USE_GX
#warning Not using precalculated g(x)
#else
#warning Using precalculated g(x)
#endif
#endif

#ifndef FL_UNITARIZE_PI
#define FL_UNITARIZE_PI 3.14159265358979323846
#endif

  template <typename Float>
  struct GaugeFixFFTRotateArg {
    int threads;     // number of active threads required
    int X[4];     // grid dimensions
    complex<Float> *tmp0;
    complex<Float> *tmp1;
    GaugeFixFFTRotateArg(const GaugeField &data){
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
      threads = X[0] * X[1] * X[2] * X[3];
      tmp0 = 0;
      tmp1 = 0;
    }
  };

  template <int direction, typename Arg>
  __global__ void fft_rotate_kernel_2D2D(Arg arg){ //Cmplx *data_in, Cmplx *data_out){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= arg.threads ) return;
    if ( direction == 0 ) {
      int x3 = id / (arg.X[0] * arg.X[1] * arg.X[2]);
      int x2 = (id / (arg.X[0] * arg.X[1])) % arg.X[2];
      int x1 = (id / arg.X[0]) % arg.X[1];
      int x0 = id % arg.X[0];

      int id  =  x0 + (x1 + (x2 + x3 * arg.X[2]) * arg.X[1]) * arg.X[0];
      int id_out =  x2 + (x3 +  (x0 + x1 * arg.X[0]) * arg.X[3]) * arg.X[2];
      arg.tmp1[id_out] = arg.tmp0[id];
      //data_out[id_out] = data_in[id];
    }
    if ( direction == 1 ) {

      int x1 = id / (arg.X[2] * arg.X[3] * arg.X[0]);
      int x0 = (id / (arg.X[2] * arg.X[3])) % arg.X[0];
      int x3 = (id / arg.X[2]) % arg.X[3];
      int x2 = id % arg.X[2];

      int id  =  x2 + (x3 +  (x0 + x1 * arg.X[0]) * arg.X[3]) * arg.X[2];
      int id_out =  x0 + (x1 + (x2 + x3 * arg.X[2]) * arg.X[1]) * arg.X[0];
      arg.tmp1[id_out] = arg.tmp0[id];
      //data_out[id_out] = data_in[id];
    }
  }

  template <typename Float, typename Arg>
  class GaugeFixFFTRotate : Tunable {
    Arg &arg;
    const GaugeField &meta;
    int direction;
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

    public:
    GaugeFixFFTRotate(Arg &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta)
    {
      direction = 0;
    }

    void setDirection(int dir, complex<Float> *data_in, complex<Float> *data_out){
      direction = dir;
      arg.tmp0 = data_in;
      arg.tmp1 = data_out;
    }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if ( direction == 0 )      qudaLaunchKernel(fft_rotate_kernel_2D2D<0, Arg>, tp, stream, arg);
      else if ( direction == 1 ) qudaLaunchKernel(fft_rotate_kernel_2D2D<1, Arg>, tp, stream, arg);
      else                       errorQuda("Error in GaugeFixFFTRotate option.\n");
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }
    long long flops() const { return 0; }
    long long bytes() const { return 4LL * sizeof(Float) * arg.threads; }
  };

  template <typename Arg>
  class GaugeFixQuality : TunableReduction2D<> {
    Arg &arg;
    const GaugeField &meta;

  public:
    GaugeFixQuality(Arg &arg, const GaugeField &meta) :
      TunableReduction2D(meta),
      arg(arg),
      meta(meta) {}

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      launch<FixQualityFFT>(tp, stream, arg);
      auto reset = true; // apply called multiple times with the same arg so need to reset
      arg.complete(arg.result, stream, reset);
      if (!activeTuning()) {
        arg.result.x /= (double)(3 * Arg::gauge_dir * 2 * arg.threads.x);
        arg.result.y /= (double)(3 * 2 * arg.threads.x);
      }
    }

    long long flops() const { return (36LL * Arg::gauge_dir + 65LL) * 2 * arg.threads.x; }
    long long bytes() const
    { return (Arg::gauge_dir * meta.Bytes() / 4) + 12 * meta.Volume() * meta.Precision(); }
  };

  template <typename store_t, QudaReconstructType recon>
  struct GaugeFixArg {
    using Float = typename mapper<store_t>::type;
    using Gauge = typename gauge_mapper<store_t, recon>::type;
    Gauge data;
    int threads;     // number of active threads required
    int X[4];     // grid dimensions
    Float *invpsq;
    complex<Float> *delta;
    complex<Float> *gx;

    GaugeFixArg(GaugeField &data, const int Elems) :
      data(data),
      threads(data.Volume())
    {
      for (int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
      invpsq = (Float*)device_malloc(sizeof(Float) * threads);
      delta = (complex<Float>*)device_malloc(sizeof(complex<Float>) * threads * 6);
#ifdef GAUGEFIXING_DONT_USE_GX
      gx = (complex<Float>*)device_malloc(sizeof(complex<Float>) * threads);
#else
      gx = (complex<Float>*)device_malloc(sizeof(complex<Float>) * threads * Elems);
#endif
    }
    void free() {
      device_free(invpsq);
      device_free(delta);
      device_free(gx);
    }
  };

  template <typename Arg> __global__ void kernel_gauge_set_invpsq(Arg arg)
  {
    using Float = typename Arg::Float;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= arg.threads ) return;
    int x1 = id / (arg.X[2] * arg.X[3] * arg.X[0]);
    int x0 = (id / (arg.X[2] * arg.X[3])) % arg.X[0];
    int x3 = (id / arg.X[2]) % arg.X[3];
    int x2 = id % arg.X[2];
    //id  =  x2 + (x3 +  (x0 + x1 * arg.X[0]) * arg.X[3]) * arg.X[2];
    Float sx = sin( (Float)x0 * FL_UNITARIZE_PI / (Float)arg.X[0]);
    Float sy = sin( (Float)x1 * FL_UNITARIZE_PI / (Float)arg.X[1]);
    Float sz = sin( (Float)x2 * FL_UNITARIZE_PI / (Float)arg.X[2]);
    Float st = sin( (Float)x3 * FL_UNITARIZE_PI / (Float)arg.X[3]);
    Float sinsq = sx * sx + sy * sy + sz * sz + st * st;
    Float prcfact = 0.0;
    //The FFT normalization is done here
    if ( sinsq > 0.00001 ) prcfact = 4.0 / (sinsq * (Float)arg.threads);
    arg.invpsq[id] = prcfact;
  }

  template <typename Arg>
  class GaugeFixSETINVPSP : Tunable {
    Arg arg;
    const GaugeField &meta;
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneSharedBytes() const { return false; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

  public:
    GaugeFixSETINVPSP(Arg &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta) { }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      qudaLaunchKernel(kernel_gauge_set_invpsq<Arg>, tp, stream, arg);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }
    long long flops() const { return 21 * arg.threads; }
    long long bytes() const { return sizeof(typename Arg::Float) * arg.threads; }
  };

  template <typename Arg> __global__ void kernel_gauge_mult_norm_2D(Arg arg)
  {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < arg.threads) arg.gx[id] = arg.gx[id] * arg.invpsq[id];
  }

  template <typename Arg>
  class GaugeFixINVPSP : Tunable {
    Arg &arg;
    const GaugeField &meta;
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

  public:
    GaugeFixINVPSP(Arg &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta)
    { }

    void apply(const qudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      qudaLaunchKernel(kernel_gauge_mult_norm_2D<Arg>, tp, stream, arg);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }

    //since delta contents are irrelevant at this point, we can swap gx with delta
    void preTune() { std::swap(arg.gx, arg.delta); }
    void postTune() { std::swap(arg.gx, arg.delta); }
    long long flops() const { return 2LL * arg.threads; }
    long long bytes() const { return 5LL * sizeof(typename Arg::Float) * arg.threads; }
  };

  template <typename Float>
  __host__ __device__ inline void reunit_link( Matrix<complex<Float>,3> &U ){

    complex<Float> t2((Float)0.0, (Float)0.0);
    Float t1 = 0.0;
    //first normalize first row
    //sum of squares of row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t1 += norm(U(0,c));
    t1 = (Float)1.0 / sqrt(t1);
    //14
    //used to normalize row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(0,c) *= t1;
    //6
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t2 += conj(U(0,c)) * U(1,c);
    //24
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(1,c) -= t2 * U(0,c);
    //24
    //normalize second row
    //sum of squares of row
    t1 = 0.0;
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t1 += norm(U(1,c));
    t1 = (Float)1.0 / sqrt(t1);
    //14
    //used to normalize row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(1, c) *= t1;
    //6
    //Reconstruct lat row
    U(2,0) = conj(U(0,1) * U(1,2) - U(0,2) * U(1,1));
    U(2,1) = conj(U(0,2) * U(1,0) - U(0,0) * U(1,2));
    U(2,2) = conj(U(0,0) * U(1,1) - U(0,1) * U(1,0));
    //42
    //T=130
  }

#ifdef GAUGEFIXING_DONT_USE_GX

  template <typename Arg> __global__ void kernel_gauge_fix_U_EO_NEW(Arg arg, typename Arg::Float half_alpha)
  {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int parity = threadIdx.y + blockIdx.y * blockDim.y;
    if (id >= arg.threads/2) return;

    using Float = typename Arg::Float;
    using complex = complex<Float>;
    using matrix = Matrix<complex, 3>;

    int x[4];
    getCoords(x, id, arg.X, parity);
    int idx = ((x[3] * arg.X[2] + x[2]) * arg.X[1] + x[1]) * arg.X[0] + x[0];
    matrix de;
    //Read Delta
    de(0,0) = arg.delta[idx + 0 * arg.threads];
    de(0,1) = arg.delta[idx + 1 * arg.threads];
    de(0,2) = arg.delta[idx + 2 * arg.threads];
    de(1,1) = arg.delta[idx + 3 * arg.threads];
    de(1,2) = arg.delta[idx + 4 * arg.threads];
    de(2,2) = arg.delta[idx + 5 * arg.threads];

    de(1,0) = complex(-de(0,1).real(), de(0,1).imag());
    de(2,0) = complex(-de(0,2).real(), de(0,2).imag());
    de(2,1) = complex(-de(1,2).real(), de(1,2).imag());

    matrix g;
    setIdentity(&g);
    g += de * half_alpha;
    //36
    reunit_link<Float>( g );
    //130

    for ( int mu = 0; mu < 4; mu++ ) {
      matrix U = arg.data(mu, id, parity);

      matrix g0;
      U = g * U;
      //198
      idx = linkNormalIndexP1(x,arg.X,mu);
      //Read Delta
      de(0,0) = arg.delta[idx + 0 * arg.threads];
      de(0,1) = arg.delta[idx + 1 * arg.threads];
      de(0,2) = arg.delta[idx + 2 * arg.threads];
      de(1,1) = arg.delta[idx + 3 * arg.threads];
      de(1,2) = arg.delta[idx + 4 * arg.threads];
      de(2,2) = arg.delta[idx + 5 * arg.threads];

      de(1,0) = complex(-de(0,1).real(), de(0,1).imag());
      de(2,0) = complex(-de(0,2).real(), de(0,2).imag());
      de(2,1) = complex(-de(1,2).real(), de(1,2).imag());

      setIdentity(&g0);
      g0 += de * half_alpha;
      //36
      reunit_link<Float>( g0 );
      //130

      U = U * conj(g0);
      //198
      arg.data(mu, id, parity) = U;
    }
  }

  template <typename Arg>
  class GaugeFixNEW : TunableVectorY {
    Arg &arg;
    const GaugeField &meta;
    double half_alpha;

    bool tuneGridDim() const { return false; }
    // since GaugeFixArg is used by other kernels that don't keep
    // parity separate, arg.threads stores Volume and not VolumeCB so
    // we need to divide by two
    unsigned int minThreads() const { return arg.threads/2; }

  public:
    GaugeFixNEW(Arg &arg, double alpha, const GaugeField &meta) :
      TunableVectorY(2),
      arg(arg),
      meta(meta)
    {
      half_alpha = alpha * 0.5;
    }

    void setAlpha(double alpha) { half_alpha = alpha * 0.5; }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      qudaLaunchKernel(kernel_gauge_fix_U_EO_NEW<Arg>, tp, stream, arg, (typename Arg::Float)half_alpha);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }
    void preTune() { meta.backup(); }
    void postTune() { meta.restore(); }
    long long flops() const { return 2414LL * arg.threads; }
    long long bytes() const { return meta.Bytes() + (5 * 12LL * sizeof(typename Arg::Float)) * arg.threads; }
  };

#else

  template <int Elems, typename Arg>
  __global__ void kernel_gauge_GX(Arg arg, typename Arg::Float half_alpha)
  {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= arg.threads) return;

    using Float = typename Arg::Float;
    using complex = complex<Float>;

    Matrix<complex,3> de;
    //Read Delta
    de(0,0) = arg.delta[id];
    de(0,1) = arg.delta[id + arg.threads];
    de(0,2) = arg.delta[id + 2 * arg.threads];
    de(1,1) = arg.delta[id + 3 * arg.threads];
    de(1,2) = arg.delta[id + 4 * arg.threads];
    de(2,2) = arg.delta[id + 5 * arg.threads];

    de(1,0) = complex(-de(0,1).x, de(0,1).y);
    de(2,0) = complex(-de(0,2).x, de(0,2).y);
    de(2,1) = complex(-de(1,2).x, de(1,2).y);

    Matrix<complex, 3> g;
    setIdentity(&g);
    g += de * half_alpha;
    //36
    reunit_link<Float>( g );
    //130
    //gx is represented in even/odd order
    //normal lattice index to even/odd index
    int x3 = id / (arg.X[0] * arg.X[1] * arg.X[2]);
    int x2 = (id / (arg.X[0] * arg.X[1])) % arg.X[2];
    int x1 = (id / arg.X[0]) % arg.X[1];
    int x0 = id % arg.X[0];
    id  =  (x0 + (x1 + (x2 + x3 * arg.X[2]) * arg.X[1]) * arg.X[0]) >> 1;
    id += ((x0 + x1 + x2 + x3) & 1 ) * arg.threads / 2;

    for ( int i = 0; i < Elems; i++ ) arg.gx[id + i * arg.threads] = g.data[i];
    //T=166 for Elems 9
    //T=208 for Elems 6
  }

  template<int Elems, typename Arg>
  class GaugeFix_GX : Tunable {
    Arg arg;
    const GaugeField &meta;
    double half_alpha;
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

    public:
    GaugeFix_GX(Arg &arg, double alpha, const GaugeField &meta) :
      arg(arg),
      meta(meta)
    {
      half_alpha = alpha * 0.5;
    }

    void setAlpha(double alpha) { half_alpha = alpha * 0.5; }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      qudaLaunchKernel(kernel_gauge_GX<Elems, Arg>, tp, stream, arg, half_alpha);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }

    long long flops() const {
      if ( Elems == 6 ) return 208LL * arg.threads;
      else return 166LL * arg.threads;
    }
    long long bytes() const { return 4LL * Elems * meta.Precision() * arg.threads; }
  };

  template <int Elems, typename Arg>
  __global__ void kernel_gauge_fix_U_EO(Arg arg)
  {
    int idd = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idd >= arg.threads ) return;

    int parity = 0;
    int id = idd;
    if ( idd >= arg.threads / 2 ) {
      parity = 1;
      id -= arg.threads / 2;
    }
    typedef complex<typename Arg::Float> Cmplx;

    Matrix<Cmplx,3> g;
    //for(int i = 0; i < Elems; i++) g.data[i] = arg.gx[idd + i * arg.threads];
    for ( int i = 0; i < Elems; i++ ) {
      g.data[i] = arg.gx[idd + i * arg.threads];
    }
    if ( Elems == 6 ) {
      g(2,0) = conj(g(0,1) * g(1,2) - g(0,2) * g(1,1));
      g(2,1) = conj(g(0,2) * g(1,0) - g(0,0) * g(1,2));
      g(2,2) = conj(g(0,0) * g(1,1) - g(0,1) * g(1,0));
      //42
    }
    int x[4];
    getCoords(x, id, arg.X, parity);
    for ( int mu = 0; mu < 4; mu++ ) {
      Matrix<Cmplx,3> U = arg.data(mu, id, parity);
      Matrix<Cmplx,3> g0;
      U = g * U;
      //198
      int idm1 = linkIndexP1(x,arg.X,mu);
      idm1 += (1 - parity) * arg.threads / 2;
      //for(int i = 0; i < Elems; i++) g0.data[i] = arg.gx[idm1 + i * arg.threads];
      for ( int i = 0; i < Elems; i++ ) {
        g0.data[i] = arg.gx[idm1 + i * arg.threads];
      }
      if ( Elems == 6 ) {
        g0(2,0) = conj(g0(0,1) * g0(1,2) - g0(0,2) * g0(1,1));
        g0(2,1) = conj(g0(0,2) * g0(1,0) - g0(0,0) * g0(1,2));
        g0(2,2) = conj(g0(0,0) * g0(1,1) - g0(0,1) * g0(1,0));
        //42
      }
      U = U * conj(g0);
      //198
      arg.data(mu, id, parity) = U;
    }
    //T=42+4*(198*2+42) Elems=6
    //T=4*(198*2) Elems=9
    //Not accounting here the reconstruction of the gauge if 12 or 8!!!!!!
  }

  template<int Elems, typename Arg>
  class GaugeFix : Tunable {
    Arg arg;
    const GaugeField &meta;
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

  public:
    GaugeFix(Arg &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta)
    { }

    void apply(const qudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      qudaLaunchKernel(kernel_gauge_fix_U_EO<Elems, Arg>, tp, stream, arg);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }

    void preTune() { meta.backup(); }
    void postTune() { meta.restore(); }
    long long flops() const {
      if ( Elems == 6 ) return 1794LL * arg.threads;
      else return 1536LL * arg.threads;
    }
    long long bytes() const { return 26LL * Elems * meta.Precision() * arg.threads; }
  };
#endif
//GAUGEFIXING_DONT_USE_GX

  template <typename Float, QudaReconstructType recon, int gauge_dir>
  void gaugefixingFFT(GaugeField& data, const int Nsteps, const int verbose_interval,
                      const Float alpha0, const int autotune, const double tolerance, const int stopWtheta)
  {
    TimeProfile profileInternalGaugeFixFFT("InternalGaugeFixQudaFFT", false);

    profileInternalGaugeFixFFT.TPSTART(QUDA_PROFILE_COMPUTE);

    Float alpha = alpha0;
    std::cout << "\tAlpha parameter of the Steepest Descent Method: " << alpha << std::endl;
    if ( autotune ) std::cout << "\tAuto tune active: yes" << std::endl;
    else std::cout << "\tAuto tune active: no" << std::endl;
    std::cout << "\tStop criterium: " << tolerance << std::endl;
    if ( stopWtheta ) std::cout << "\tStop criterium method: theta" << std::endl;
    else std::cout << "\tStop criterium method: Delta" << std::endl;
    std::cout << "\tMaximum number of iterations: " << Nsteps << std::endl;
    std::cout << "\tPrint convergence results at every " << verbose_interval << " steps" << std::endl;

    unsigned int delta_pad = data.X()[0] * data.X()[1] * data.X()[2] * data.X()[3];
    int4 size = make_int4( data.X()[0], data.X()[1], data.X()[2], data.X()[3] );
    hipfftHandle plan_xy;
    hipfftHandle plan_zt;

    constexpr int Elems = recon / 2; // number of complex elements used to store g(x) and Delta(x)
    GaugeFixArg<Float, recon> arg(data, Elems);
    SetPlanFFT2DMany( plan_zt, size, 0, arg.delta);     //for space and time ZT
    SetPlanFFT2DMany( plan_xy, size, 1, arg.delta);    //with space only XY

    GaugeFixFFTRotateArg<Float> arg_rotate(data);
    GaugeFixFFTRotate<Float, decltype(arg_rotate)> GFRotate(arg_rotate, data);

    GaugeFixSETINVPSP<decltype(arg)> setinvpsp(arg, data);
    setinvpsp.apply(0);
    GaugeFixINVPSP<decltype(arg)> invpsp(arg, data);

#ifdef GAUGEFIXING_DONT_USE_GX
    //without using GX, gx will be created only for plane rotation but with less size
    GaugeFixNEW<decltype(arg)> gfixNew(arg, alpha, data);
#else
    //using GX
    GaugeFix_GX<Elems, decltype(arg)> calcGX(arg, alpha, data);
    GaugeFix<Elems, decltype(arg)> gfix(arg, data);
#endif

    GaugeFixQualityFFTArg<Float, recon, gauge_dir> argQ(data, arg.delta);
    GaugeFixQuality<decltype(argQ)> gfixquality(argQ, data);
    gfixquality.apply(0);
    double action0 = argQ.getAction();
    printf("Step: %d\tAction: %.16e\ttheta: %.16e\n", 0, argQ.getAction(), argQ.getTheta());

    double diff = 0.0;
    int iter = 0;
    for (iter = 0; iter < Nsteps; iter++) {
      for (int k = 0; k < 6; k++) {
        //------------------------------------------------------------------------
        // Set a pointer do the element k in lattice volume
        // each element is stored with stride lattice volume
        // it uses gx as temporary array!!!!!!
        //------------------------------------------------------------------------
        complex<Float> *_array = arg.delta + k * delta_pad;
        //////  2D FFT + 2D FFT
        //------------------------------------------------------------------------
        // Perform FFT on xy plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_xy, _array, arg.gx, HIPFFT_FORWARD);
        //------------------------------------------------------------------------
        // Rotate hypercube, xyzt -> ztxy
        //------------------------------------------------------------------------
        GFRotate.setDirection(0, arg.gx, _array);
        GFRotate.apply(0);
        //------------------------------------------------------------------------
        // Perform FFT on zt plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_zt, _array, arg.gx, HIPFFT_FORWARD);
        //------------------------------------------------------------------------
        // Normalize FFT and apply pmax^2/p^2
        //------------------------------------------------------------------------
        invpsp.apply(0);
        //------------------------------------------------------------------------
        // Perform IFFT on zt plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_zt, arg.gx, _array, HIPFFT_BACKWARD);
        //------------------------------------------------------------------------
        // Rotate hypercube, ztxy -> xyzt
        //------------------------------------------------------------------------
        GFRotate.setDirection(1, _array, arg.gx);
        GFRotate.apply(0);
        //------------------------------------------------------------------------
        // Perform IFFT on xy plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_xy, arg.gx, _array, HIPFFT_BACKWARD);
      }

      gfixquality.apply(0);
      printfQuda("Debug %e\n", argQ.getAction());
#ifdef GAUGEFIXING_DONT_USE_GX
      //------------------------------------------------------------------------
      // Apply gauge fix to current gauge field
      //------------------------------------------------------------------------
      gfixNew.apply(0); // something wrong here, perhaps quality is messing up delta?
#else
      //------------------------------------------------------------------------
      // Calculate g(x)
      //------------------------------------------------------------------------
      calcGX.apply(0);
      //------------------------------------------------------------------------
      // Apply gauge fix to current gauge field
      //------------------------------------------------------------------------
      gfix.apply(0);
#endif
      gfixquality.apply(0);
      //------------------------------------------------------------------------
      // Measure gauge quality and recalculate new Delta(x)
      //------------------------------------------------------------------------
      gfixquality.apply(0);
      double action = argQ.getAction();
      diff = abs(action0 - action);
      if ((iter % verbose_interval) == (verbose_interval - 1))
        printf("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter + 1, argQ.getAction(), argQ.getTheta(), diff);
      if ( autotune && ((action - action0) < -1e-14) ) {
        if ( alpha > 0.01 ) {
          alpha = 0.95 * alpha;
#ifdef GAUGEFIXING_DONT_USE_GX
          gfixNew.setAlpha(alpha);
#else
          calcGX.setAlpha(alpha);
#endif
          printf(">>>>>>>>>>>>>> Warning: changing alpha down -> %.4e\n", alpha );
        }
      }
      //------------------------------------------------------------------------
      // Check gauge fix quality criterium
      //------------------------------------------------------------------------
      if ( stopWtheta ) {   if ( argQ.getTheta() < tolerance ) break; }
      else { if ( diff < tolerance ) break; }

      action0 = action;
    }
    if ((iter % verbose_interval) != 0 )
      printf("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter, argQ.getAction(), argQ.getTheta(), diff);

    // Reunitarize at end
    const double unitarize_eps = 1e-14;
    const double max_error = 1e-10;
    const int reunit_allow_svd = 1;
    const int reunit_svd_only  = 0;
    const double svd_rel_error = 1e-6;
    const double svd_abs_error = 1e-6;
    setUnitarizeLinksConstants(unitarize_eps, max_error,
                               reunit_allow_svd, reunit_svd_only,
                               svd_rel_error, svd_abs_error);
    int num_failures = 0;
    int* num_failures_dev = static_cast<int*>(pool_device_malloc(sizeof(int)));
    qudaMemset(num_failures_dev, 0, sizeof(int));
    unitarizeLinks(data, data, num_failures_dev);
    qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);

    pool_device_free(num_failures_dev);
    if ( num_failures > 0 ) {
      errorQuda("Error in the unitarization\n");
      exit(1);
    }
    // end reunitarize

    arg.free();
    CUFFT_SAFE_CALL(hipfftDestroy(plan_zt));
    CUFFT_SAFE_CALL(hipfftDestroy(plan_xy));
    qudaDeviceSynchronize();
    profileInternalGaugeFixFFT.TPSTOP(QUDA_PROFILE_COMPUTE);

    if (getVerbosity() > QUDA_SUMMARIZE){
      double secs = profileInternalGaugeFixFFT.Last(QUDA_PROFILE_COMPUTE);
      double fftflop = 5.0 * (log2((double)( data.X()[0] * data.X()[1]) ) + log2( (double)(data.X()[2] * data.X()[3] )));
      fftflop *= (double)( data.X()[0] * data.X()[1] * data.X()[2] * data.X()[3] );
      double gflops = setinvpsp.flops() + gfixquality.flops();
      double gbytes = setinvpsp.bytes() + gfixquality.bytes();
      double flop = invpsp.flops() * Elems;
      double byte = invpsp.bytes() * Elems;
      flop += (GFRotate.flops() + fftflop) * Elems * 2;
      byte += GFRotate.bytes() * Elems * 4;     //includes FFT reads, assuming 1 read and 1 write per site
      #ifdef GAUGEFIXING_DONT_USE_GX
      flop += gfixNew.flops();
      byte += gfixNew.bytes();
      #else
      flop += calcGX.flops();
      byte += calcGX.bytes();
      flop += gfix.flops();
      byte += gfix.bytes();
      #endif
      flop += gfixquality.flops();
      byte += gfixquality.bytes();
      gflops += flop * iter;
      gbytes += byte * iter;
      gflops += 4588.0 * data.Volume(); //Reunitarize at end
      gbytes += 2 * data.Bytes(); //Reunitarize at end

      gflops = (gflops * 1e-9) / (secs);
      gbytes = gbytes / (secs * 1e9);
      printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops, gbytes);
    }
  }

  template<typename Float, int nColors, QudaReconstructType recon> struct GaugeFixingFFT {
    GaugeFixingFFT(GaugeField& data, const int gauge_dir, const int Nsteps, const int verbose_interval,
                   const Float alpha, const int autotune, const double tolerance, const int stopWtheta)
    {
      if (gauge_dir != 3) {
        printfQuda("Starting Landau gauge fixing with FFTs...\n");
        gaugefixingFFT<Float, recon, 4>(data, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);
      } else {
        printfQuda("Starting Coulomb gauge fixing with FFTs...\n");
        gaugefixingFFT<Float, recon, 3>(data, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);
      }
    }
  };

  /**
   * @brief Gauge fixing with Steepest descent method with FFTs with support for single GPU only.
   * @param[in,out] data, quda gauge field
   * @param[in] gauge_dir, 3 for Coulomb gauge fixing, other for Landau gauge fixing
   * @param[in] Nsteps, maximum number of steps to perform gauge fixing
   * @param[in] verbose_interval, print gauge fixing info when iteration count is a multiple of this
   * @param[in] alpha, gauge fixing parameter of the method, most common value is 0.08
   * @param[in] autotune, 1 to autotune the method, i.e., if the Fg inverts its tendency we decrease the alpha value
   * @param[in] tolerance, torelance value to stop the method, if this value is zero then the method stops when iteration reachs the maximum number of steps defined by Nsteps
   * @param[in] stopWtheta, 0 for MILC criterium and 1 to use the theta value
   */
  void gaugeFixingFFT(GaugeField& data, const int gauge_dir, const int Nsteps, const int verbose_interval, const double alpha,
                      const int autotune, const double tolerance, const int stopWtheta)
  {
#ifdef GPU_GAUGE_ALG
#ifdef MULTI_GPU
    if (comm_dim_partitioned(0) || comm_dim_partitioned(1) || comm_dim_partitioned(2) || comm_dim_partitioned(3))
      errorQuda("Gauge Fixing with FFTs in multi-GPU support NOT implemented yet!\n");
#endif
    instantiate<GaugeFixingFFT>(data, gauge_dir, Nsteps, verbose_interval, (float)alpha, autotune, tolerance, stopWtheta);
#else
    errorQuda("Gauge fixing has bot been built");
#endif
  }

}
