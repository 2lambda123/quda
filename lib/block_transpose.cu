#include "hip/hip_runtime.h"
#include <instantiate.h>
#include <multigrid.h>
#include <tunable_nd.h>
#include <reference_wrapper_helper.h>
#include <kernels/block_transpose.cuh>

namespace quda {

  namespace impl {

  using namespace quda::colorspinor;

  template <class v_t, class b_t, typename vFloat, typename bFloat, int nSpin, int nColor, int nVec>
  class BlockTranspose : public TunableKernel3D {

    using real = typename mapper<vFloat>::type;
    template <bool is_device, typename vOrder, typename bOrder> using Arg =
      BlockTransposeArg<v_t, b_t, is_device, vFloat, vOrder, bFloat, bOrder, nSpin, nColor, nVec>;

    v_t &V;
    cvector_ref<b_t> &B;

  public:
    BlockTranspose(v_t &V, cvector_ref<b_t> &B) :
      TunableKernel3D(V, V.SiteSubset(), V.Nvec()),
      V(V),
      B(B)
    {

#ifdef QUDA_FAST_COMPILE_REDUCE
      strcat(aux, ",fast_compile");
#endif

      apply(device::get_default_stream());
    }

#if 0
    template <typename Rotator, typename Vector, std::size_t... S>
    void launch_host_(const TuneParam &tp, const qudaStream_t &stream,
                     const std::vector<ColorSpinorField*> &B, std::index_sequence<S...>)
    {
      Arg<false, Rotator, Vector> arg(V, fine_to_coarse, coarse_to_fine, QUDA_INVALID_PARITY, geo_bs, n_block_ortho, V, B[S]...);
      launch_host<BlockOrtho_, OrthoAggregates>(tp, stream, arg);
      if (two_pass && iter == 0 && V.Precision() < QUDA_SINGLE_PRECISION && !activeTuning()) max = Rotator(V).abs_max(V);
    }
#endif

    template <typename vAccessor, typename bAccessor, std::size_t... S>
    void launch_device_(const TuneParam &tp, const qudaStream_t &stream, std::index_sequence<S...>)
    {
      Arg<true, vAccessor, bAccessor> arg(V, B[S]...);
      launch_device<BlockTransposeKernel>(tp, stream, arg);
    }

    void apply(const qudaStream_t &stream)
    {
      constexpr bool disable_ghost = true;
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (V.Location() == QUDA_CPU_FIELD_LOCATION) {
#if 0
        if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER && B[0]->FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
          typedef FieldOrderCB<real,nSpin,nColor,nVec,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,vFloat,vFloat,disable_ghost> Rotator;
          typedef FieldOrderCB<real,nSpin,nColor,1,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,bFloat,bFloat,disable_ghost> Vector;
          launch_host_<Rotator, Vector>(tp, stream, B, std::make_index_sequence<nVec>());
        } else {
          errorQuda("Unsupported field order %d", V.FieldOrder());
        }
#endif
      } else {
        constexpr auto vOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
        constexpr auto bOrder = colorspinor::getNative<bFloat>(nSpin);
        if (V.FieldOrder() == vOrder && B[0].FieldOrder() == bOrder) {
          typedef FieldOrderCB<real, nSpin, nColor, nVec, vOrder, vFloat, vFloat, disable_ghost> vAccessor;
          typedef FieldOrderCB<real, nSpin, nColor, 1, bOrder, bFloat, bFloat, disable_ghost> bAccessor;
          if constexpr (std::is_const_v<v_t>) {
            launch_device_<const vAccessor, bAccessor>(tp, stream, std::make_index_sequence<nVec>());
          } else {
            launch_device_<vAccessor, const bAccessor>(tp, stream, std::make_index_sequence<nVec>());
          }
        } else {
          errorQuda("Unsupported field order V=%d B=%d", V.FieldOrder(), B[0].FieldOrder());
        }
      }
    }

    virtual unsigned int minThreads() const {
      return V.VolumeCB();
    }

    long long flops() const
    {
      return 0;
    }

    long long bytes() const
    {
      return V.Bytes() + nVec * B[0].Bytes();
    }
  };

  } // namespace impl

  template <class v_t, class b_t, typename vFloat, typename bFloat, int nSpin, int nColor>
  void block_transpose(v_t &V, cvector_ref<b_t> &B)
  {
    if (V.Nvec() != static_cast<int>(B.size())) { errorQuda("V.Nvec() (=%d) != B.size() (=%d)", V.Nvec(), static_cast<int>(B.size())); }

    if (V.Nvec() == 8) {
      impl::BlockTranspose<v_t, b_t, vFloat, bFloat, nSpin, nColor, 8> tranpose(V, B);
    } else if (V.Nvec() == 16) {
      impl::BlockTranspose<v_t, b_t, vFloat, bFloat, nSpin, nColor, 16> tranpose(V, B);
    } else {
      errorQuda("Unexpected nVec = %d", V.Nvec());
    }
  }

  template <class v_t, class b_t, typename vFloat, typename bFloat, int nSpin>
  void block_transpose(v_t &V, cvector_ref<b_t> &B)
  {
    if (V.Ncolor() / V.Nvec() != B[0].Ncolor()) { errorQuda("V.Ncolor() / V.Nvec() (=%d) != B.Ncolor() (=%d)", V.Ncolor() / V.Nvec(), B[0].Ncolor()); }

    if (B[0].Ncolor() == 24) {
      block_transpose<v_t, b_t, vFloat, bFloat, nSpin, 24>(V, B);
    } else if (B[0].Ncolor() == 32) {
      block_transpose<v_t, b_t, vFloat, bFloat, nSpin, 32>(V, B);
    } else {
      errorQuda("Unexpected nColor = %d", B[0].Ncolor());
    }
  }

  template <class v_t, class b_t, typename vFloat, typename bFloat>
  void block_transpose(v_t &V, cvector_ref<b_t> &B)
  {
    if (V.Nspin() != B[0].Nspin()) { errorQuda("V.Nspin() (=%d) != B.Nspin() (=%d)", V.Nspin(), B[0].Nspin()); }

    if (V.Nspin() == 2) {
      block_transpose<v_t, b_t, vFloat, bFloat, 2>(V, B);
    } else if (V.Nspin() == 4) {
      block_transpose<v_t, b_t, vFloat, bFloat, 4>(V, B);
    } else if (V.Nspin() == 1) {
      block_transpose<v_t, b_t, vFloat, bFloat, 1>(V, B);
    } else {
      errorQuda("Unexpected nSpin = %d", V.Nspin());
    }
  }

  template <class v_t, class b_t>
  void block_transpose(v_t &V, cvector_ref<b_t> &B)
  {
    if (!is_enabled(V.Precision()) || !is_enabled(B[0].Precision()))
      errorQuda("QUDA_PRECISION=%d does not enable required precision combination (V = %d B = %d)",
                QUDA_PRECISION, V.Precision(), B[0].Precision());

    if constexpr (is_enabled_multigrid()) {
      if (V.Precision() == QUDA_DOUBLE_PRECISION && B[0].Precision() == QUDA_DOUBLE_PRECISION) {
        if constexpr (is_enabled_multigrid_double())
          block_transpose<v_t, b_t, double, double>(V, B);
        else
          errorQuda("Double precision multigrid has not been enabled");
      } else if (V.Precision() == QUDA_SINGLE_PRECISION && B[0].Precision() == QUDA_SINGLE_PRECISION) {
        if constexpr (is_enabled(QUDA_SINGLE_PRECISION))
          block_transpose<v_t, b_t, float, float>(V, B);
      } else {
        errorQuda("Unsupported precision combination V=%d B=%d\n", V.Precision(), B[0].Precision());
      }
    } else {
      errorQuda("Multigrid has not been built");
    }
  }

  void BlockTransposeForward(ColorSpinorField &V, cvector_ref<const ColorSpinorField> &B) {
    block_transpose(V, B);
  }

  void BlockTransposeBackward(const ColorSpinorField &V, cvector_ref<ColorSpinorField> &B) {
    block_transpose(V, B);
  }

} // namespace quda
