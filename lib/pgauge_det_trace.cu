#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <comm_quda.h>
#include <pgauge_monte.h>
#include <reduce_helper.h>
#include <index_helper.cuh>
#include <instantiate.h>

namespace quda {

  template <typename Float, int nColor_, QudaReconstructType recon_>
  struct KernelArg : public ReduceArg<double2> {
    static constexpr int nColor = nColor_;
    static constexpr QudaReconstructType recon = recon_;
    using real = typename mapper<Float>::type;
    using Gauge = typename gauge_mapper<real, recon>::type;
    int threads; // number of active threads required
    int X[4]; // grid dimensions
    int border[4];
    Gauge dataOr;

    KernelArg(const GaugeField &data) :
      ReduceArg<double2>(),
      dataOr(data)
    {
      for (int dir=0; dir<4; ++dir) {
        border[dir] = data.R()[dir];
        X[dir] = data.X()[dir] - border[dir]*2;
      }
      threads = X[0]*X[1]*X[2]*X[3]/2;
    }
  };

  template <int blockSize, int type, typename Arg>
  __global__ void compute_Value(Arg arg)
  {
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int parity = threadIdx.y;

    complex<double> val(0.0, 0.0);
    while (idx < arg.threads) {
      int X[4];
#pragma unroll
      for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
#pragma unroll
      for(int dr=0; dr<4; ++dr) {
        x[dr] += arg.border[dr];
        X[dr] += 2*arg.border[dr];
      }
      idx = linkIndex(x,X);
#pragma unroll
      for (int mu = 0; mu < 4; mu++) {
        Matrix<complex<typename Arg::real>, Arg::nColor> U = arg.dataOr(mu, idx, parity);
        if (type == 0) val += getDeterminant(U);
        else if (type == 1) val += getTrace(U);
      }

      idx += blockDim.x*gridDim.x;
    }

    double2 sum = make_double2(val.real(), val.imag());
    reduce2d<blockSize,2>(arg, sum);
  }

  template <typename Float, int nColor, QudaReconstructType recon, int type>
  class CalcFunc : TunableLocalParity {
    double2 &result;
    const GaugeField &u;
    TuneParam tp;
    bool tuneGridDim() const { return true; }

  public:
    CalcFunc(double2 &result, const GaugeField &u) :
      result(result),
      u(u)
    {
      TimeProfile profileGenericFunc("GenericFunc", false);
      if (getVerbosity() >= QUDA_SUMMARIZE) profileGenericFunc.TPSTART(QUDA_PROFILE_COMPUTE);

      apply(0);

      if (getVerbosity() >= QUDA_SUMMARIZE){
        profileGenericFunc.TPSTOP(QUDA_PROFILE_COMPUTE);
        double secs = profileGenericFunc.Last(QUDA_PROFILE_COMPUTE);
        double gflops = flops()*1e-9/secs;
        double gbytes = bytes()/(secs*1e9);
        printfQuda("%s: %.16e, %.16e\n", type == 0 ? "Determinant" : "Trace", result.x, result.y);
        printfQuda("%s: Time = %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n",
                   type == 0 ? "Determinant" : "Trace", secs, gflops*comm_size(), gbytes*comm_size());
      }
    }

    void apply(const qudaStream_t &stream)
    {
      tp = tuneLaunch(*this, getTuning(), getVerbosity());
      KernelArg<Float, nColor, recon> arg(u);
      LAUNCH_KERNEL_LOCAL_PARITY(compute_Value, (*this), tp, stream, arg, type, decltype(arg));
      arg.complete(&result, stream);
      if (!activeTuning()) {
        comm_allreduce_array((double*)&result, 2);
        result.x /= (double)(4*2*arg.threads*comm_size());
        result.y /= (double)(4*2*arg.threads*comm_size());
      }
    }

    TuneKey tuneKey() const { return TuneKey(u.VolString(), typeid(*this).name(), u.AuxString()); }

    long long flops() const {
      if (u.Ncolor()==3 && type == 0) return 264LL*u.Volume();
      else if (type == 1) return 2*u.Geometry()*u.Ncolor()*u.Volume();
      else return 0;
    }

    long long bytes() const { return u.Bytes(); }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct computeDeterminant {
    computeDeterminant(GaugeField &data, double2 &det)
    {
      CalcFunc<Float, nColor, recon, 0>(det, data);
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct computeTrace {
    computeTrace(GaugeField &data, double2 &trace)
    {
      CalcFunc<Float, nColor, recon, 1>(trace, data);
    }
  };

  /**
   * @brief Calculate the Determinant
   *
   * @param[in] data Gauge field
   * @returns double2 complex Determinant value
   */
  double2 getLinkDeterminant(GaugeField& data)
  {
    double2 det = make_double2(0.0,0.0);
#ifdef GPU_GAUGE_ALG
    instantiate<computeDeterminant>(data, det);
#else
    errorQuda("Pure gauge code has not been built");
#endif // GPU_GAUGE_ALG
    return det;
  }

  /**
   * @brief Calculate the Trace
   *
   * @param[in] data Gauge field
   * @returns double2 complex trace value
   */
  double2 getLinkTrace(GaugeField& data)
  {
    double2 det = make_double2(0.0,0.0);
#ifdef GPU_GAUGE_ALG
    instantiate<computeTrace>(data, det);
#else
    errorQuda("Pure gauge code has not been built");
#endif // GPU_GAUGE_ALG
    return det;
  }

} // namespace quda
