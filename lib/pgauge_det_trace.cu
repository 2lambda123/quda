#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <comm_quda.h>
#include <instantiate.h>

#include <tunable_reduction.h>
#include <kernels/gauge_det_trace.cuh>

namespace quda {

  template <typename Float, int nColor, QudaReconstructType recon>
  class CalcFunc : TunableReduction2D<> {
    double2 &result;
    const GaugeField &u;
    const int type;

  public:
    CalcFunc(const GaugeField &u, double2 &result, int type) :
      TunableReduction2D(u),
      result(result),
      u(u),
      type(type)
    {
      strcat(aux, type == 0 ? ",det" : ",trace");
      apply(device::get_default_stream());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (type == 0) {
        KernelArg<Float, nColor, recon, 0> arg(u);
        launch<DetTrace>(result, tp, stream, arg);
      } else {
        KernelArg<Float, nColor, recon, 1> arg(u);
        launch<DetTrace>(result, tp, stream, arg);
      }

      result.x /= (double)(4*u.LocalVolume()*comm_size());
      result.y /= (double)(4*u.LocalVolume()*comm_size());
    }

    long long flops() const {
      if (u.Ncolor()==3 && type == 0) return 264LL*u.LocalVolume();
      else if (type == 1) return 2*u.Geometry()*u.Ncolor()*u.LocalVolume();
      else return 0;
    }

    long long bytes() const { return u.Bytes(); }
  };

#ifdef GPU_GAUGE_ALG
  double2 getLinkDeterminant(GaugeField& data)
  {
    double2 det = make_double2(0.0,0.0);
    instantiate<CalcFunc>(data, det, 0);
    return det;
  }

  double2 getLinkTrace(GaugeField& data)
  {
    double2 det = make_double2(0.0,0.0);
    instantiate<CalcFunc>(data, det, 1);
    return det;
  }
#else
  double2 getLinkDeterminant(GaugeField&)
  {
    errorQuda("Pure gauge code has not been built");
    return make_double2(0.0,0.0);
  }

  double2 getLinkTrace(GaugeField&)
  {
    errorQuda("Pure gauge code has not been built");
    return make_double2(0.0,0.0);
  }
#endif

} // namespace quda
