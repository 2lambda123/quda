#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <blas_quda.h>
#include <blas_lapack.h>
#include <tunable_nd.h>
#include <instantiate.h>

#include <staggered_kd_build_xinv.h>
#include <kernels/staggered_kd_reorder_xinv_kernel.cuh>

namespace quda {

  template <typename Float, int fineColor, bool dagger_approximation>
  class CalculateStaggeredGeometryReorder : public TunableKernel3D {

    GaugeField &fineXinv;
    const GaugeField &coarseXinv;
    double scale;

    long long flops() const {
      if (dagger_approximation) {
        // rescale of all values
        return fineXinv.Volume() * fineXinv.Geometry() * fineXinv.Ncolor() * fineXinv.Ncolor() * 2ll;
      } else {
        // just a permutation
        return 0ll;
      }
    }

    long long bytes() const
    {
      // 1. Loading coarseXinv, the coarse KD inverse field
      // 2. Storing fineXinv, the reordered fine KD inverse field
      return coarseXinv.Bytes() + fineXinv.Bytes();
    }

    unsigned int minThreads() const { return fineXinv.VolumeCB(); }

  public:
    CalculateStaggeredGeometryReorder(GaugeField& fineXinv, const GaugeField& coarseXinv, const double scale) :
      TunableKernel3D(fineXinv, QUDA_KDINVERSE_GEOMETRY, 2),
      fineXinv(fineXinv),
      coarseXinv(coarseXinv),
      scale(scale)
    {
      checkPrecision(fineXinv, coarseXinv);
      checkLocation(fineXinv, coarseXinv);
      if (fineXinv.Geometry() != QUDA_KDINVERSE_GEOMETRY)
        errorQuda("Unsupported geometry %d", fineXinv.Geometry());
      if (fineXinv.Ndim() != 4)
        errorQuda("Number of dimensions %d is not supported", fineXinv.Ndim());
      if (coarseXinv.Geometry() != QUDA_SCALAR_GEOMETRY)
        errorQuda("Unsupported geometry %d", coarseXinv.Geometry());

      strcat(aux,",computeStaggeredGeometryReorder");
      if (dagger_approximation) strcat(aux, ",dagger_approximation");

      apply(device::get_default_stream());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (fineXinv.Location() == QUDA_CPU_FIELD_LOCATION) {
        constexpr QudaGaugeFieldOrder fineOrder = QUDA_QDP_GAUGE_ORDER;
        constexpr QudaGaugeFieldOrder coarseOrder = QUDA_QDP_GAUGE_ORDER;
        CalculateStaggeredGeometryReorderArg<Float,fineColor,fineOrder,coarseOrder,dagger_approximation> arg(fineXinv, coarseXinv, scale);
        launch_host<ComputeStaggeredGeometryReorder>(tp, stream, arg);

      } else if (fineXinv.Location() == QUDA_CUDA_FIELD_LOCATION) {
        constexpr QudaGaugeFieldOrder fineOrder = QUDA_FLOAT2_GAUGE_ORDER;
        constexpr QudaGaugeFieldOrder coarseOrder = QUDA_MILC_GAUGE_ORDER;
        CalculateStaggeredGeometryReorderArg<Float,fineColor,fineOrder,coarseOrder,dagger_approximation> arg(fineXinv, coarseXinv, scale);
        launch_device<ComputeStaggeredGeometryReorder>(tp, stream, arg);

      }
    }
  };

  template<typename Float, int fineColor>
  struct calculateStaggeredGeometryReorder {
    calculateStaggeredGeometryReorder(GaugeField &fineXinv, const GaugeField &coarseXinv, const bool dagger_approximation, const double mass) {
      // template on dagger approximation
      if (dagger_approximation)  {
        // approximate the inverse with the dagger: the free field for staggered, 
        // B^-1 = 1 / (4 * (d + mass^2)), where the 4 is due to the factor of 2 convention
        double scale = 1. / (4. * (fineXinv.Ndim() + mass * mass));

        // reset scales as appropriate
        if constexpr (sizeof(Float) < QUDA_SINGLE_PRECISION) {
          double max_scale = coarseXinv.abs_max() * abs(scale) * 1.01;
          if (getVerbosity() >= QUDA_VERBOSE) printfQuda("Global xInv_max = %e\n", max_scale);

          fineXinv.Scale(max_scale);
        }
        CalculateStaggeredGeometryReorder<Float,fineColor,true>(fineXinv, coarseXinv, scale);
      } else {
        double scale = 1.;
        CalculateStaggeredGeometryReorder<Float,fineColor,false>(fineXinv, coarseXinv, scale);
      }
    }
  };

#if defined(GPU_STAGGERED_DIRAC) && defined(GPU_MULTIGRID)
  /**
     @brief Reorder the staggered Kahler-Dirac inverse from a coarse scalar layout to a fine KD geometry

     @param fineXinv[out] KD inverse fine gauge in KD geometry
     @param coarseXinv[in] KD inverse coarse lattice field
     @param dagger_approximation[in] Whether or not to apply the dagger approximation
     @param mass[in] Mass of staggered fermion (used for dagger approximation only)
   */
  void ReorderStaggeredKahlerDiracInverse(GaugeField &fineXinv, const GaugeField &coarseXinv, const bool dagger_approximation, const double mass) {
    // Instantiate based on precision, number of colors
    instantiate<calculateStaggeredGeometryReorder>(fineXinv, coarseXinv, dagger_approximation, mass);
  }
#else
  void ReorderStaggeredKahlerDiracInverse(GaugeField &, const GaugeField &, const bool, const double) {
    errorQuda("Staggered fermion support has not been built");
  }
#endif

} //namespace quda
