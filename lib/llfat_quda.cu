#include "hip/hip_runtime.h"
#include <stdio.h>

#include <quda_internal.h>
#include <llfat_quda.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <read_gauge.h>
#include <gauge_quda.h>
#include <force_common.h>

#define SITE_MATRIX_LOAD_TEX 0
#define MULINK_LOAD_TEX 0
#define FATLINK_LOAD_TEX 0


#define WRITE_FAT_MATRIX(gauge, dir, idx)do {		       \
    gauge[idx + dir*9*llfat_ga_stride] = FAT0;			\
    gauge[idx + (dir*9+1) * llfat_ga_stride] = FAT1;			\
    gauge[idx + (dir*9+2) * llfat_ga_stride] = FAT2;			\
    gauge[idx + (dir*9+3) * llfat_ga_stride] = FAT3;			\
    gauge[idx + (dir*9+4) * llfat_ga_stride] = FAT4;		\
    gauge[idx + (dir*9+5) * llfat_ga_stride] = FAT5;		\
    gauge[idx + (dir*9+6) * llfat_ga_stride] = FAT6;		\
    gauge[idx + (dir*9+7) * llfat_ga_stride] = FAT7;		\
    gauge[idx + (dir*9+8) * llfat_ga_stride] = FAT8;} while(0)			


#define WRITE_STAPLE_MATRIX(gauge, idx)				\
  gauge[idx] = STAPLE0;						\
  gauge[idx + staple_stride] = STAPLE1;				\
  gauge[idx + 2*staple_stride] = STAPLE2;			\
  gauge[idx + 3*staple_stride] = STAPLE3;			\
  gauge[idx + 4*staple_stride] = STAPLE4;			\
  gauge[idx + 5*staple_stride] = STAPLE5;			\
  gauge[idx + 6*staple_stride] = STAPLE6;			\
  gauge[idx + 7*staple_stride] = STAPLE7;			\
  gauge[idx + 8*staple_stride] = STAPLE8;					
    

#define SCALAR_MULT_SU3_MATRIX(a, b, c) \
  c##00_re = a*b##00_re;		\
  c##00_im = a*b##00_im;		\
  c##01_re = a*b##01_re;		\
  c##01_im = a*b##01_im;		\
  c##02_re = a*b##02_re;		\
  c##02_im = a*b##02_im;		\
  c##10_re = a*b##10_re;		\
  c##10_im = a*b##10_im;		\
  c##11_re = a*b##11_re;		\
  c##11_im = a*b##11_im;		\
  c##12_re = a*b##12_re;		\
  c##12_im = a*b##12_im;		\
  c##20_re = a*b##20_re;		\
  c##20_im = a*b##20_im;		\
  c##21_re = a*b##21_re;		\
  c##21_im = a*b##21_im;		\
  c##22_re = a*b##22_re;		\
  c##22_im = a*b##22_im;		\
    

#define LOAD_MATRIX_18_SINGLE(gauge, dir, idx, var, stride)		\
  float2 var##0 = gauge[idx + dir*9*stride];				\
  float2 var##1 = gauge[idx + dir*9*stride + stride];			\
  float2 var##2 = gauge[idx + dir*9*stride + 2*stride];			\
  float2 var##3 = gauge[idx + dir*9*stride + 3*stride];			\
  float2 var##4 = gauge[idx + dir*9*stride + 4*stride];			\
  float2 var##5 = gauge[idx + dir*9*stride + 5*stride];			\
  float2 var##6 = gauge[idx + dir*9*stride + 6*stride];			\
  float2 var##7 = gauge[idx + dir*9*stride + 7*stride];			\
  float2 var##8 = gauge[idx + dir*9*stride + 8*stride];			

#define LOAD_MATRIX_18_SINGLE_TEX(gauge, dir, idx, var, stride)		\
  float2 var##0 = tex1Dfetch(gauge, idx + dir*9*stride);		\
  float2 var##1 = tex1Dfetch(gauge, idx + dir*9*stride + stride);	\
  float2 var##2 = tex1Dfetch(gauge, idx + dir*9*stride + 2*stride);	\
  float2 var##3 = tex1Dfetch(gauge, idx + dir*9*stride + 3*stride);	\
  float2 var##4 = tex1Dfetch(gauge, idx + dir*9*stride + 4*stride);	\
  float2 var##5 = tex1Dfetch(gauge, idx + dir*9*stride + 5*stride);	\
  float2 var##6 = tex1Dfetch(gauge, idx + dir*9*stride + 6*stride);	\
  float2 var##7 = tex1Dfetch(gauge, idx + dir*9*stride + 7*stride);	\
  float2 var##8 = tex1Dfetch(gauge, idx + dir*9*stride + 8*stride);	

#define LOAD_MATRIX_18_DOUBLE(gauge, dir, idx, var, stride)		\
  double2 var##0 = gauge[idx + dir*9*stride];				\
  double2 var##1 = gauge[idx + dir*9*stride + stride];			\
  double2 var##2 = gauge[idx + dir*9*stride + 2*stride];		\
  double2 var##3 = gauge[idx + dir*9*stride + 3*stride];		\
  double2 var##4 = gauge[idx + dir*9*stride + 4*stride];		\
  double2 var##5 = gauge[idx + dir*9*stride + 5*stride];		\
  double2 var##6 = gauge[idx + dir*9*stride + 6*stride];		\
  double2 var##7 = gauge[idx + dir*9*stride + 7*stride];		\
  double2 var##8 = gauge[idx + dir*9*stride + 8*stride];		

#define LOAD_MATRIX_18_DOUBLE_TEX(gauge, dir, idx, var, stride)		\
  double2 var##0 = fetch_double2(gauge, idx + dir*9*stride);		\
  double2 var##1 = fetch_double2(gauge, idx + dir*9*stride + stride);	\
  double2 var##2 = fetch_double2(gauge, idx + dir*9*stride + 2*stride);	\
  double2 var##3 = fetch_double2(gauge, idx + dir*9*stride + 3*stride);	\
  double2 var##4 = fetch_double2(gauge, idx + dir*9*stride + 4*stride);	\
  double2 var##5 = fetch_double2(gauge, idx + dir*9*stride + 5*stride);	\
  double2 var##6 = fetch_double2(gauge, idx + dir*9*stride + 6*stride);	\
  double2 var##7 = fetch_double2(gauge, idx + dir*9*stride + 7*stride);	\
  double2 var##8 = fetch_double2(gauge, idx + dir*9*stride + 8*stride);	



#define LOAD_MATRIX_12_SINGLE_DECLARE(gauge, dir, idx, var, stride)	\
  float4 var##0 = gauge[idx + dir*3*stride];				\
  float4 var##1 = gauge[idx + dir*3*stride + stride];			\
  float4 var##2 = gauge[idx + dir*3*stride + 2*stride];			\
  float4 var##3, var##4;

#define LOAD_MATRIX_12_SINGLE_TEX_DECLARE(gauge, dir, idx, var, stride)	\
  float4 var##0 = tex1Dfetch(gauge, idx + dir*3*stride);		\
  float4 var##1 = tex1Dfetch(gauge, idx + dir*3*stride + stride);	\
  float4 var##2 = tex1Dfetch(gauge, idx + dir*3*stride + 2*stride);	\
  float4 var##3, var##4;

#define LOAD_MATRIX_18_SINGLE_DECLARE(gauge, dir, idx, var, stride)	\
  float2 var##0 = gauge[idx + dir*9*stride];				\
  float2 var##1 = gauge[idx + dir*9*stride + stride];			\
  float2 var##2 = gauge[idx + dir*9*stride + 2*stride];			\
  float2 var##3 = gauge[idx + dir*9*stride + 3*stride];			\
  float2 var##4 = gauge[idx + dir*9*stride + 4*stride];			\
  float2 var##5 = gauge[idx + dir*9*stride + 5*stride];			\
  float2 var##6 = gauge[idx + dir*9*stride + 6*stride];			\
  float2 var##7 = gauge[idx + dir*9*stride + 7*stride];			\
  float2 var##8 = gauge[idx + dir*9*stride + 8*stride];			


#define LOAD_MATRIX_18_SINGLE_TEX_DECLARE(gauge, dir, idx, var, stride)	\
  float2 var##0 = tex1Dfetch(gauge, idx + dir*9*stride);		\
  float2 var##1 = tex1Dfetch(gauge, idx + dir*9*stride + stride);	\
  float2 var##2 = tex1Dfetch(gauge, idx + dir*9*stride + 2*stride);	\
  float2 var##3 = tex1Dfetch(gauge, idx + dir*9*stride + 3*stride);	\
  float2 var##4 = tex1Dfetch(gauge, idx + dir*9*stride + 4*stride);	\
  float2 var##5 = tex1Dfetch(gauge, idx + dir*9*stride + 5*stride);	\
  float2 var##6 = tex1Dfetch(gauge, idx + dir*9*stride + 6*stride);	\
  float2 var##7 = tex1Dfetch(gauge, idx + dir*9*stride + 7*stride);	\
  float2 var##8 = tex1Dfetch(gauge, idx + dir*9*stride + 8*stride);			



#define LOAD_MATRIX_18_DOUBLE_DECLARE(gauge, dir, idx, var, stride)	\
  double2 var##0 = gauge[idx + dir*9*stride];				\
  double2 var##1 = gauge[idx + dir*9*stride + stride];			\
  double2 var##2 = gauge[idx + dir*9*stride + 2*stride];		\
  double2 var##3 = gauge[idx + dir*9*stride + 3*stride];		\
  double2 var##4 = gauge[idx + dir*9*stride + 4*stride];		\
  double2 var##5 = gauge[idx + dir*9*stride + 5*stride];		\
  double2 var##6 = gauge[idx + dir*9*stride + 6*stride];		\
  double2 var##7 = gauge[idx + dir*9*stride + 7*stride];		\
  double2 var##8 = gauge[idx + dir*9*stride + 8*stride];			


#define LOAD_MATRIX_18_DOUBLE_TEX_DECLARE(gauge, dir, idx, var, stride)	\
  double2 var##0 = fetch_double2(gauge, idx + dir*9*stride);		\
  double2 var##1 = fetch_double2(gauge, idx + dir*9*stride + stride);	\
  double2 var##2 = fetch_double2(gauge, idx + dir*9*stride + 2*stride);	\
  double2 var##3 = fetch_double2(gauge, idx + dir*9*stride + 3*stride);	\
  double2 var##4 = fetch_double2(gauge, idx + dir*9*stride + 4*stride);	\
  double2 var##5 = fetch_double2(gauge, idx + dir*9*stride + 5*stride);	\
  double2 var##6 = fetch_double2(gauge, idx + dir*9*stride + 6*stride);	\
  double2 var##7 = fetch_double2(gauge, idx + dir*9*stride + 7*stride);	\
  double2 var##8 = fetch_double2(gauge, idx + dir*9*stride + 8*stride);	


#define LOAD_MATRIX_12_DOUBLE_DECLARE(gauge, dir, idx, var)		\
  double2 var##0 = gauge[idx + dir*Vhx6];				\
  double2 var##1 = gauge[idx + dir*Vhx6 + Vh];				\
  double2 var##2 = gauge[idx + dir*Vhx6 + Vhx2];			\
  double2 var##3 = gauge[idx + dir*Vhx6 + Vhx3];			\
  double2 var##4 = gauge[idx + dir*Vhx6 + Vhx4];			\
  double2 var##5 = gauge[idx + dir*Vhx6 + Vhx5];			\
  double2 var##6, var##7, var##8;


#define LOAD_MATRIX_12_DOUBLE_TEX_DECLARE(gauge, dir, idx, var)		\
  double2 var##0 = fetch_double2(gauge, idx + dir*Vhx6);		\
  double2 var##1 = fetch_double2(gauge, idx + dir*Vhx6 + Vh);		\
  double2 var##2 = fetch_double2(gauge, idx + dir*Vhx6 + Vhx2);		\
  double2 var##3 = fetch_double2(gauge, idx + dir*Vhx6 + Vhx3);		\
  double2 var##4 = fetch_double2(gauge, idx + dir*Vhx6 + Vhx4);		\
  double2 var##5 = fetch_double2(gauge, idx + dir*Vhx6 + Vhx5);		\
  double2 var##6, var##7, var##8;

#define LLFAT_ADD_SU3_MATRIX(ma, mb, mc)	\
  mc##00_re = ma##00_re + mb##00_re;		\
  mc##00_im = ma##00_im + mb##00_im;		\
  mc##01_re = ma##01_re + mb##01_re;		\
  mc##01_im = ma##01_im + mb##01_im;		\
  mc##02_re = ma##02_re + mb##02_re;		\
  mc##02_im = ma##02_im + mb##02_im;		\
  mc##10_re = ma##10_re + mb##10_re;		\
  mc##10_im = ma##10_im + mb##10_im;		\
  mc##11_re = ma##11_re + mb##11_re;		\
  mc##11_im = ma##11_im + mb##11_im;		\
  mc##12_re = ma##12_re + mb##12_re;		\
  mc##12_im = ma##12_im + mb##12_im;		\
  mc##20_re = ma##20_re + mb##20_re;		\
  mc##20_im = ma##20_im + mb##20_im;		\
  mc##21_re = ma##21_re + mb##21_re;		\
  mc##21_im = ma##21_im + mb##21_im;		\
  mc##22_re = ma##22_re + mb##22_re;		\
  mc##22_im = ma##22_im + mb##22_im;		


__constant__ int site_ga_stride;
__constant__ int staple_stride;
__constant__ int llfat_ga_stride;

void
llfat_init_cuda(QudaGaugeParam* param)
{
  static int llfat_init_cuda_flag = 0;
  if (llfat_init_cuda_flag){
    return;
  }
  
  llfat_init_cuda_flag = 1;
  
  init_kernel_cuda(param);
  int Vh = param->X[0]*param->X[1]*param->X[2]*param->X[3]/2;
  int site_ga_stride = param->site_ga_pad + Vh;
  int staple_stride = param->staple_pad + Vh;
  int llfat_ga_stride = param->llfat_ga_pad + Vh;
  
  hipMemcpyToSymbol(HIP_SYMBOL("site_ga_stride"), &site_ga_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("staple_stride"), &staple_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("llfat_ga_stride"), &llfat_ga_stride, sizeof(int));
}


#define COMPUTE_RECONSTRUCT_SIGN(sign, dir, i1,i2,i3,i4) do {	\
    sign =1;							\
    switch(dir){						\
    case XUP:							\
      if ( (i4 & 1) == 1){					\
	sign = -1;						\
      }								\
      break;							\
    case YUP:							\
      if ( ((i4+i1) & 1) == 1){					\
	sign = -1;						\
      }								\
      break;							\
    case ZUP:							\
      if ( ((i4+i1+i2) & 1) == 1){				\
	sign = -1;						\
      }								\
      break;							\
    case TUP:							\
      if (i4 == X4m1 ){						\
	sign = -1;						\
      }								\
      break;							\
    }								\
  }while (0)


#define LLFAT_CONCAT(a,b) a##b##Kernel
#define LLFAT_KERNEL(a,b) LLFAT_CONCAT(a,b)

//precision: 0 is for double, 1 is for single

//single precision, common macro
#define PRECISION 1
#define Float  float
#define LOAD_FAT_MATRIX(gauge, dir, idx) LOAD_MATRIX_18_SINGLE(gauge, dir, idx, FAT, llfat_ga_stride)
#if (MULINK_LOAD_TEX == 1)
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX(muLink0TexSingle, dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX(muLink1TexSingle, dir, idx, var, staple_stride)
#else
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE(mulink_even, dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE(mulink_odd, dir, idx, var, staple_stride)
#endif

#if (FATLINK_LOAD_TEX == 1)
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_SINGLE_TEX(fatGauge0TexSingle, dir, idx, FAT, llfat_ga_stride)
#define LOAD_ODD_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_SINGLE_TEX(fatGauge1TexSingle, dir, idx, FAT, llfat_ga_stride)
#else
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_SINGLE(fatlink_even, dir, idx, FAT, llfat_ga_stride)
#define LOAD_ODD_FAT_MATRIX(dir, idx)  LOAD_MATRIX_18_SINGLE(fatlink_odd, dir, idx, FAT, llfat_ga_stride)
#endif


//single precision, 12-reconstruct
#define SITELINK0TEX siteLink0TexSingle
#define SITELINK1TEX siteLink1TexSingle
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX_DECLARE(SITELINK0TEX, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX_DECLARE(SITELINK1TEX, dir, idx, var, site_ga_stride)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_DECLARE(sitelink_even, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_DECLARE(sitelink_odd, dir, idx, var, site_ga_stride)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_12_SINGLE_DECLARE(sitelink, dir, idx, var, site_ga_stride)

#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  RECONSTRUCT_LINK_12(dir, idx, sign, var);
#define FloatN float4
#define FloatM float2
#define RECONSTRUCT 12
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT

//single precision, 18-reconstruct
#define SITELINK0TEX siteLink0TexSingle_norecon
#define SITELINK1TEX siteLink1TexSingle_norecon
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX_DECLARE(SITELINK0TEX, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX_DECLARE(SITELINK1TEX, dir, idx, var, site_ga_stride)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_DECLARE(sitelink_even, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_DECLARE(sitelink_odd, dir, idx, var, site_ga_stride)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_18_SINGLE(sitelink, dir, idx, var, site_ga_stride)
#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  
#define FloatN float2
#define FloatM float2
#define RECONSTRUCT 18
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT


#undef PRECISION
#undef Float
#undef LOAD_FAT_MATRIX
#undef LOAD_EVEN_MULINK_MATRIX
#undef LOAD_ODD_MULINK_MATRIX
#undef LOAD_EVEN_FAT_MATRIX
#undef LOAD_ODD_FAT_MATRIX


//double precision, common macro
#define PRECISION 0
#define Float double
#define LOAD_FAT_MATRIX(gauge, dir, idx) LOAD_MATRIX_18_DOUBLE(gauge, dir, idx, FAT, llfat_ga_stride)
#if (MULINK_LOAD_TEX == 1)
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX(muLink0TexDouble, dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX(muLink1TexDouble, dir, idx, var, staple_stride)
#else
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE(mulink_even, dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE(mulink_odd, dir, idx, var, staple_stride)
#endif

#if (FATLINK_LOAD_TEX == 1)
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_DOUBLE_TEX(fatGauge0TexDouble, dir, idx, FAT, llfat_ga_stride)
#define LOAD_ODD_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_DOUBLE_TEX(fatGauge1TexDouble, dir, idx, FAT, llfat_ga_stride)
#else
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_DOUBLE(fatlink_even, dir, idx, FAT, llfat_ga_stride)
#define LOAD_ODD_FAT_MATRIX(dir, idx)  LOAD_MATRIX_18_DOUBLE(fatlink_odd, dir, idx, FAT, llfat_ga_stride)
#endif

//double precision,  18-reconstruct
#define SITELINK0TEX siteLink0TexDouble
#define SITELINK1TEX siteLink1TexDouble
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX_DECLARE(SITELINK0TEX, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX_DECLARE(SITELINK1TEX, dir, idx, var, site_ga_stride)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_DECLARE(sitelink_even, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_DECLARE(sitelink_odd, dir, idx, var, site_ga_stride)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_18_DOUBLE(sitelink, dir, idx, var, site_ga_stride)
#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  
#define FloatN double2
#define FloatM double2
#define RECONSTRUCT 18
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT

#if 1
//double precision, 12-reconstruct
#define SITELINK0TEX siteLink0TexDouble
#define SITELINK1TEX siteLink1TexDouble
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_TEX_DECLARE(SITELINK0TEX, dir, idx, var)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_TEX_DECLARE(SITELINK1TEX, dir, idx, var)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_DECLARE(sitelink_even, dir, idx, var)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_DECLARE(sitelink_odd, dir, idx, var)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_12_DOUBLE_DECLARE(sitelink, dir, idx, var)
#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  RECONSTRUCT_LINK_12(dir, idx, sign, var);
#define FloatN double2
#define FloatM double2
#define RECONSTRUCT 12
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT
#endif

#undef PRECISION
#undef Float
#undef LOAD_FAT_MATRIX
#undef LOAD_EVEN_MULINK_MATRIX
#undef LOAD_ODD_MULINK_MATRIX
#undef LOAD_EVEN_FAT_MATRIX
#undef LOAD_ODD_FAT_MATRIX

#undef LLFAT_CONCAT
#undef LLFAT_KERNEL

#define UNBIND_ALL_TEXTURE do{						\
    if(prec ==QUDA_DOUBLE_PRECISION){					\
      hipUnbindTexture(siteLink0TexDouble);				\
      hipUnbindTexture(siteLink1TexDouble);				\
      hipUnbindTexture(fatGauge0TexDouble);				\
      hipUnbindTexture(fatGauge1TexDouble);				\
      hipUnbindTexture(muLink0TexDouble);				\
      hipUnbindTexture(muLink1TexDouble);				\
    }else{								\
      if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
	hipUnbindTexture(siteLink0TexSingle_norecon);			\
	hipUnbindTexture(siteLink1TexSingle_norecon);			\
      }else{								\
	hipUnbindTexture(siteLink0TexSingle);				\
	hipUnbindTexture(siteLink1TexSingle);				\
      }									\
      hipUnbindTexture(fatGauge0TexSingle);				\
      hipUnbindTexture(fatGauge1TexSingle);				\
      hipUnbindTexture(muLink0TexSingle);				\
      hipUnbindTexture(muLink1TexSingle);				\
    }									\
  }while(0)

#define UNBIND_SITE_AND_FAT_LINK do{					\
    if(prec == QUDA_DOUBLE_PRECISION){					\
      hipUnbindTexture(siteLink0TexDouble);				\
      hipUnbindTexture(siteLink1TexDouble);				\
      hipUnbindTexture(fatGauge0TexDouble);				\
      hipUnbindTexture(fatGauge1TexDouble);				\
    }else {								\
      if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
	hipUnbindTexture(siteLink0TexSingle_norecon);			\
	hipUnbindTexture(siteLink1TexSingle_norecon);			\
      }else{								\
	hipUnbindTexture(siteLink0TexSingle);				\
	hipUnbindTexture(siteLink1TexSingle);				\
      }									\
      hipUnbindTexture(fatGauge0TexSingle);				\
      hipUnbindTexture(fatGauge1TexSingle);				\
    }									\
  }while(0)

#define BIND_SITE_AND_FAT_LINK do {					\
  if(prec == QUDA_DOUBLE_PRECISION){					\
    hipBindTexture(0, siteLink0TexDouble, cudaSiteLink.even, cudaSiteLink.bytes); \
    hipBindTexture(0, siteLink1TexDouble, cudaSiteLink.odd, cudaSiteLink.bytes); \
    hipBindTexture(0, fatGauge0TexDouble, cudaFatLink.even, cudaFatLink.bytes); \
    hipBindTexture(0, fatGauge1TexDouble, cudaFatLink.odd,  cudaFatLink.bytes); \
  }else{								\
    if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
      hipBindTexture(0, siteLink0TexSingle_norecon, cudaSiteLink.even, cudaSiteLink.bytes); \
      hipBindTexture(0, siteLink1TexSingle_norecon, cudaSiteLink.odd, cudaSiteLink.bytes); \
    }else{								\
      hipBindTexture(0, siteLink0TexSingle, cudaSiteLink.even, cudaSiteLink.bytes); \
      hipBindTexture(0, siteLink1TexSingle, cudaSiteLink.odd, cudaSiteLink.bytes); \
    }									\
    hipBindTexture(0, fatGauge0TexSingle, cudaFatLink.even, cudaFatLink.bytes); \
    hipBindTexture(0, fatGauge1TexSingle, cudaFatLink.odd,  cudaFatLink.bytes); \
    }									\
  }while(0)

#define BIND_SITE_AND_FAT_LINK_REVERSE do {				\
    if(prec == QUDA_DOUBLE_PRECISION){					\
      hipBindTexture(0, siteLink1TexDouble, cudaSiteLink.even, cudaSiteLink.bytes); \
      hipBindTexture(0, siteLink0TexDouble, cudaSiteLink.odd, cudaSiteLink.bytes); \
      hipBindTexture(0, fatGauge1TexDouble, cudaFatLink.even, cudaFatLink.bytes); \
      hipBindTexture(0, fatGauge0TexDouble, cudaFatLink.odd,  cudaFatLink.bytes); \
    }else{								\
      if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
	hipBindTexture(0, siteLink1TexSingle_norecon, cudaSiteLink.even, cudaSiteLink.bytes); \
	hipBindTexture(0, siteLink0TexSingle_norecon, cudaSiteLink.odd, cudaSiteLink.bytes); \
      }else{								\
	hipBindTexture(0, siteLink1TexSingle, cudaSiteLink.even, cudaSiteLink.bytes); \
	hipBindTexture(0, siteLink0TexSingle, cudaSiteLink.odd, cudaSiteLink.bytes); \
      }									\
      hipBindTexture(0, fatGauge1TexSingle, cudaFatLink.even, cudaFatLink.bytes); \
      hipBindTexture(0, fatGauge0TexSingle, cudaFatLink.odd,  cudaFatLink.bytes); \
    }									\
  }while(0)



#define ENUMERATE_FUNCS(mu,nu,odd_bit)	switch(mu) {			\
  case 0:								\
    switch(nu){								\
    case 0:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    case 1:								\
      if (!odd_bit) { CALL_FUNCTION(0,1,0); }				\
      else {CALL_FUNCTION(0,1,1); }					\
      break;								\
    case 2:								\
      if (!odd_bit) { CALL_FUNCTION(0,2,0); }				\
      else {CALL_FUNCTION(0,2,1); }					\
      break;								\
    case 3:								\
      if (!odd_bit) { CALL_FUNCTION(0,3,0); }				\
      else {CALL_FUNCTION(0,3,1); }					\
      break;								\
    }									\
    break;								\
  case 1:								\
    switch(nu){								\
    case 0:								\
      if (!odd_bit) { CALL_FUNCTION(1,0,0); }				\
      else {CALL_FUNCTION(1,0,1); }					\
      break;								\
    case 1:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    case 2:								\
      if (!odd_bit) { CALL_FUNCTION(1,2,0); }				\
      else {CALL_FUNCTION(1,2,1); }					\
      break;								\
    case 3:								\
      if (!odd_bit) { CALL_FUNCTION(1,3,0); }				\
      else {CALL_FUNCTION(1,3,1); }					\
      break;								\
    }									\
    break;								\
  case 2:								\
    switch(nu){								\
    case 0:								\
      if (!odd_bit) { CALL_FUNCTION(2,0,0); }				\
      else {CALL_FUNCTION(2,0,1); }					\
      break;								\
    case 1:								\
      if (!odd_bit) { CALL_FUNCTION(2,1,0); }				\
      else {CALL_FUNCTION(2,1,1); }					\
      break;								\
    case 2:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    case 3:								\
      if (!odd_bit) { CALL_FUNCTION(2,3,0); }				\
      else {CALL_FUNCTION(2,3,1); }					\
      break;								\
    }									\
    break;								\
  case 3:								\
    switch(nu){								\
    case 0:								\
      if (!odd_bit) { CALL_FUNCTION(3,0,0); }				\
      else {CALL_FUNCTION(3,0,1); }					\
      break;								\
    case 1:								\
      if (!odd_bit) { CALL_FUNCTION(3,1,0); }				\
      else {CALL_FUNCTION(3,1,1); }					\
      break;								\
    case 2:								\
      if (!odd_bit) { CALL_FUNCTION(3,2,0); }				\
      else {CALL_FUNCTION(3,2,1); }					\
      break;								\
    case 3:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    }									\
    break;								\
  }

#define ENUMERATE_FUNCS_SAVE(mu,nu,odd_bit, save_staple) if(save_staple){ \
    switch(mu) {							\
    case 0:								\
      switch(nu){							\
      case 0:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 1:								\
	if (!odd_bit) { CALL_FUNCTION(0,1,0,1); }			\
	else {CALL_FUNCTION(0,1,1,1); }					\
	break;								\
      case 2:								\
	if (!odd_bit) { CALL_FUNCTION(0,2,0,1); }			\
	else {CALL_FUNCTION(0,2,1,1); }					\
	break;								\
      case 3:								\
	if (!odd_bit) { CALL_FUNCTION(0,3,0,1); }			\
	else {CALL_FUNCTION(0,3,1,1); }					\
	break;								\
      }									\
      break;								\
    case 1:								\
      switch(nu){							\
      case 0:								\
	if (!odd_bit) { CALL_FUNCTION(1,0,0,1); }			\
	else {CALL_FUNCTION(1,0,1,1); }					\
	break;								\
      case 1:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 2:								\
	if (!odd_bit) { CALL_FUNCTION(1,2,0,1); }			\
	else {CALL_FUNCTION(1,2,1,1); }					\
	break;								\
      case 3:								\
	if (!odd_bit) { CALL_FUNCTION(1,3,0,1); }			\
	else {CALL_FUNCTION(1,3,1,1); }					\
	break;								\
      }									\
      break;								\
    case 2:								\
      switch(nu){							\
      case 0:								\
	if (!odd_bit) { CALL_FUNCTION(2,0,0,1); }			\
	else {CALL_FUNCTION(2,0,1,1); }					\
	break;								\
      case 1:								\
	if (!odd_bit) { CALL_FUNCTION(2,1,0,1); }			\
	else {CALL_FUNCTION(2,1,1,1); }					\
	break;								\
      case 2:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 3:								\
	if (!odd_bit) { CALL_FUNCTION(2,3,0,1); }			\
	else {CALL_FUNCTION(2,3,1,1); }					\
	break;								\
      }									\
      break;								\
    case 3:								\
      switch(nu){							\
      case 0:								\
	if (!odd_bit) { CALL_FUNCTION(3,0,0,1); }			\
	else {CALL_FUNCTION(3,0,1,1); }					\
	break;								\
      case 1:								\
	if (!odd_bit) { CALL_FUNCTION(3,1,0,1); }			\
	else {CALL_FUNCTION(3,1,1,1); }					\
	break;								\
      case 2:								\
	if (!odd_bit) { CALL_FUNCTION(3,2,0,1); }			\
	else {CALL_FUNCTION(3,2,1,1); }					\
	break;								\
      case 3:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      }									\
      break;								\
    }									\
  }else{								\
    switch(mu) {							\
    case 0:								\
      switch(nu){							\
      case 0:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 1:								\
	if (!odd_bit) { CALL_FUNCTION(0,1,0,0); }			\
	else {CALL_FUNCTION(0,1,1,0); }					\
	break;								\
      case 2:								\
	if (!odd_bit) { CALL_FUNCTION(0,2,0,0); }			\
	else {CALL_FUNCTION(0,2,1,0); }					\
	break;								\
      case 3:								\
	if (!odd_bit) { CALL_FUNCTION(0,3,0,0); }			\
	else {CALL_FUNCTION(0,3,1,0); }					\
	break;								\
      }									\
      break;								\
    case 1:								\
      switch(nu){							\
      case 0:								\
	if (!odd_bit) { CALL_FUNCTION(1,0,0,0); }			\
	else {CALL_FUNCTION(1,0,1,0); }					\
	break;								\
      case 1:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 2:								\
	if (!odd_bit) { CALL_FUNCTION(1,2,0,0); }			\
	else {CALL_FUNCTION(1,2,1,0); }					\
	break;								\
      case 3:								\
	if (!odd_bit) { CALL_FUNCTION(1,3,0,0); }			\
	else {CALL_FUNCTION(1,3,1,0); }					\
	break;								\
      }									\
      break;								\
    case 2:								\
      switch(nu){							\
      case 0:								\
	if (!odd_bit) { CALL_FUNCTION(2,0,0,0); }			\
	else {CALL_FUNCTION(2,0,1,0); }					\
	break;								\
      case 1:								\
	if (!odd_bit) { CALL_FUNCTION(2,1,0,0); }			\
	else {CALL_FUNCTION(2,1,1,0); }					\
	break;								\
      case 2:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 3:								\
	if (!odd_bit) { CALL_FUNCTION(2,3,0,0); }			\
	else {CALL_FUNCTION(2,3,1,0); }					\
	break;								\
      }									\
      break;								\
    case 3:								\
      switch(nu){							\
      case 0:								\
	if (!odd_bit) { CALL_FUNCTION(3,0,0,0); }			\
	else {CALL_FUNCTION(3,0,1,0); }					\
	break;								\
      case 1:								\
	if (!odd_bit) { CALL_FUNCTION(3,1,0,0); }			\
	else {CALL_FUNCTION(3,1,1,0); }					\
	break;								\
      case 2:								\
	if (!odd_bit) { CALL_FUNCTION(3,2,0,0); }			\
	else {CALL_FUNCTION(3,2,1,0); }					\
	break;								\
      case 3:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      }									\
      break;								\
    }									\
  }

void siteComputeGenStapleParityKernel(void* staple_even, void* staple_odd, 
				      void* sitelink_even, void* sitelink_odd, 
				      void* fatlink_even, void* fatlink_odd,	
				      int mu, int nu,int odd_bit,
				      double mycoeff,
				      QudaReconstructType recon, QudaPrecision prec,
				      int2 tloc, dim3 halfGridDim, 
				      hipStream_t* stream)
{

  
#define  CALL_FUNCTION(mu, nu, odd_bit)					\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel<mu,nu, odd_bit>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double)mycoeff, tloc);	\
    }else{								\
      do_siteComputeGenStapleParity12Kernel<mu,nu, odd_bit>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double)mycoeff, tloc);	\
    }									\
  }else {								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel<mu,nu, odd_bit>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float2*)sitelink_even, (float2*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float)mycoeff, tloc);	\
    }else{								\
      do_siteComputeGenStapleParity12Kernel<mu,nu, odd_bit>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float4*)sitelink_even, (float4*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float)mycoeff, tloc);	\
    }									\
  }
  

  dim3 blockDim(BLOCK_DIM , 1, 1);  
  ENUMERATE_FUNCS(mu,nu,odd_bit);  

#undef CALL_FUNCTION
    
    
}


void
computeGenStapleFieldParityKernel(void* staple_even, void* staple_odd, 
				  void* sitelink_even, void* sitelink_odd,
				  void* fatlink_even, void* fatlink_odd,			    
				  void* mulink_even, void* mulink_odd, 
				  int mu, int nu, int odd_bit, int save_staple,
				  double mycoeff,
				  QudaReconstructType recon, QudaPrecision prec,
				  int2 tloc, dim3 halfGridDim, 
				  hipStream_t* stream)
{

#define  CALL_FUNCTION(mu, nu, odd_bit, save_staple)			\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel<mu,nu, odd_bit, save_staple> \
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double2*)mulink_even, (double2*)mulink_odd, \
						(double)mycoeff, tloc);	\
    }else{								\
      do_computeGenStapleFieldParity12Kernel<mu,nu, odd_bit, save_staple> \
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double2*)mulink_even, (double2*)mulink_odd, \
						(double)mycoeff, tloc);	\
    }									\
  }else{								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel<mu,nu, odd_bit, save_staple> \
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float2*)sitelink_even, (float2*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float2*)mulink_even, (float2*)mulink_odd, \
						(float)mycoeff, tloc);	\
    }else{								\
      do_computeGenStapleFieldParity12Kernel<mu,nu, odd_bit, save_staple> \
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float4*)sitelink_even, (float4*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float2*)mulink_even, (float2*)mulink_odd, \
						(float)mycoeff, tloc);	\
    }									\
  }
  
  dim3 blockDim(BLOCK_DIM , 1, 1);
  ENUMERATE_FUNCS_SAVE(mu,nu,odd_bit, save_staple);


#undef CALL_FUNCTION 
    
}



void llfatOneLinkKernel(FullGauge cudaFatLink, FullGauge cudaSiteLink,
           FullStaple cudaStaple, FullStaple cudaStaple1,
           QudaGaugeParam* param, double* act_path_coeff)
{  
  QudaPrecision prec = cudaSiteLink.precision;
  QudaReconstructType recon = cudaSiteLink.reconstruct;
  
  int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];  
  dim3 gridDim(volume/BLOCK_DIM,1,1);
  dim3 blockDim(BLOCK_DIM , 1, 1);

  if(prec == QUDA_DOUBLE_PRECISION){
    if(recon == QUDA_RECONSTRUCT_NO){
      llfatOneLink18Kernel<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						  (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						  (double)act_path_coeff[0], (double)act_path_coeff[5]);    
    }else{
      
      llfatOneLink12Kernel<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						  (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						  (double)act_path_coeff[0], (double)act_path_coeff[5]);    
      
    }
  }else{ //single precision
    if(recon == QUDA_RECONSTRUCT_NO){    
      llfatOneLink18Kernel<<<gridDim, blockDim>>>((float2*)cudaSiteLink.even, (float2*)cudaSiteLink.odd,
						  (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						  (float)act_path_coeff[0], (float)act_path_coeff[5]);    						  
    }else{
      llfatOneLink12Kernel<<<gridDim, blockDim>>>((float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd,
						  (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						  (float)act_path_coeff[0], (float)act_path_coeff[5]);    
    }
  }
}
