#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include "exchange_face.h"

#define BLOCK_DIM 64

//these are access control for staggered action
#if (__CUDA_ARCH__ >= 200)
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
#else
#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
#endif

#include <quda_internal.h>
#include <dslash_quda.h>

struct DslashParam {
  int tOffset; // offset into the T dimension (multi gpu only)
  int tMul;    // spatial volume distance between the T faces being updated (multi gpu only)
  int threads; // the desired number of active threads
  int parity;  // Even-Odd or Odd-Even
};

DslashParam dslashParam;

// these are set in initDslashConst
int Vspatial;
#ifdef MULTI_GPU
static const int Nstream = 3;
#else
static const int Nstream = 1;
#endif
static hipStream_t streams[Nstream];
FaceBuffer *face;
int stride;

#include <dslash_textures.h>
#include <dslash_constants.h>

#include <staggered_dslash_def.h> // staggered Dslash kernels
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#include <dw_dslash_def.h>        // Domain Wall kernels
#include <tm_dslash_def.h>        // Twisted Mass kernels
#include <dslash_core/tm_core.h>  // solo twisted mass kernel
#include <clover_def.h>           // kernels for applying the clover term alone

#ifndef SHARED_FLOATS_PER_THREAD
#define SHARED_FLOATS_PER_THREAD 0
#endif

#include <blas_quda.h>
#include <face_quda.h>


// dslashTuning = QUDA_TUNE_YES turns off error checking
static QudaTune dslashTuning = QUDA_TUNE_NO;

void setDslashTuning(QudaTune tune)
{
  dslashTuning = tune;
}

__global__ void dummyKernel() {
  // do nothing
}

void initCache() {

#if (__CUDA_ARCH__ >= 200)

  static int firsttime = 1;
  if (firsttime){	
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dummyKernel), hipFuncCachePreferL1);
    dummyKernel<<<1,1>>>();
    firsttime=0;
  }

#endif

}

void setFace(const FaceBuffer &Face, const int Stride) {
  face = (FaceBuffer*)&Face; // nasty
  stride = Stride;
}

template <int spinorN, typename spinorFloat, typename gaugeFloat>
void dslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat *gauge0, const gaugeFloat *gauge1, 
		const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
		const int dagger, const spinorFloat *x, const float *xNorm, const double &a, 
		const int volume, const size_t bytes, const size_t norm_bytes, hipStream_t &stream,
		const int shared_bytes, const dim3 blockDim) 
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  if (x==0) { // not doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	dslash18Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam);
      } else {
	dslash18DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	dslash12Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam);
      } else {
	dslash12DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam);
      }
    } else {
      if (!dagger) {
	dslash8Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam);
      } else {
	dslash8DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam);
      }
    }
  } else { // doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	dslash18XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, x, xNorm, a);
      } else {
	dslash18DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, x, xNorm, a);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	dslash12XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, x, xNorm, a);
      } else {
	dslash12DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, x, xNorm, a);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {
      if (!dagger) {
	dslash8XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, x, xNorm, a);
      } else {
	dslash8DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, x, xNorm, a);
      }
    }
  }
 
}

template <int spinorN, typename spinorFloat, typename gaugeFloat>
void dslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat *gauge0, const gaugeFloat *gauge1, 
		const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
		const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
		const double &a, const int volume, const size_t bytes, const size_t norm_bytes,
		const dim3 block, const dim3 blockFace) {

  int shared_bytes = block.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<spinorN>(bytes, norm_bytes, in, inNorm, x, xNorm);

  dslashParam.parity = parity;

#ifndef MULTI_GPU
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = volume;

  dslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, in, inNorm, 
		      dagger, x, xNorm, a, volume, bytes, norm_bytes, streams[0], shared_bytes, block);
#else

  // Gather from source spinor
  face->exchangeFacesStart((void*)in, (void*)inNorm, stride, dagger, streams);
  
#ifdef OVERLAP_COMMS // do body
  dslashParam.tOffset = 1;
  dslashParam.tMul = 1;
  dslashParam.threads = volume - 2*Vspatial;
  dslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, in, inNorm, 
		      dagger, x, xNorm, a, volume, bytes, norm_bytes, streams[Nstream-1], shared_bytes, block);    
#endif // OVERLAP_COMMS

  // Finish gather and start comms
  face->exchangeFacesComms();

  // Wait for comms to finish, and scatter into the end zone
  face->exchangeFacesWait((void*)in, (void*)inNorm, stride, dagger);

  dslashParam.tOffset = 0;
#ifdef OVERLAP_COMMS // do faces
  dslashParam.tMul = volume/Vspatial - 1; // hacky way to get Nt
  dslashParam.threads = 2*Vspatial;
#else // do all
  dslashParam.tMul = 1;
  dslashParam.threads = volume;
#endif // OVERLAP_COMMS
  shared_bytes = blockFace.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<spinorN>(bytes, norm_bytes, in, inNorm, x, xNorm);
  dslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, in, inNorm, 
		      dagger, x, xNorm, a, volume, bytes, norm_bytes, streams[Nstream-2], shared_bytes, blockFace);    

#endif // MULTI_GPU

  // texture unbinding is not asynchronous so we don't need to cudaThreadSynchonize()
  unbindSpinorTex<spinorN>(in, inNorm, x, xNorm); 
}

// Wilson wrappers
void dslashCuda(void *out, void *outNorm, const FullGauge gauge, const void *in, const void *inNorm, 
		const int parity, const int dagger, const void *x, const void *xNorm, 
		const double k, const int volume, const size_t bytes, const size_t norm_bytes, 
		const QudaPrecision precision, const dim3 block, const dim3 blockFace) {

#ifdef GPU_WILSON_DIRAC
  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslashCuda<2>((double2*)out, (float*)outNorm, (double2*)gauge0, (double2*)gauge1, 
		  gauge.reconstruct, (double2*)in, (float*)inNorm, parity, dagger, 
		  (double2*)x, (float*)xNorm, k, volume, bytes, norm_bytes, block, blockFace);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    dslashCuda<4>((float4*)out, (float*)outNorm, (float4*)gauge0, (float4*)gauge1,
    		  gauge.reconstruct, (float4*)in, (float*)inNorm, parity, dagger, 
		  (float4*)x, (float*)xNorm, k, volume, bytes, norm_bytes, block, blockFace);
  } else if (precision == QUDA_HALF_PRECISION) {
    dslashCuda<4>((short4*)out, (float*)outNorm, (short4*)gauge0, (short4*)gauge1,
		  gauge.reconstruct, (short4*)in, (float*)inNorm, parity, dagger, 
		  (short4*)x, (float*)xNorm, k, volume, bytes, norm_bytes, block, blockFace);
  }
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC

}

// FIXME: cloverCuda cannot be issued asynchronously because of texture unbinding and checkCudaError
template <int N, typename spinorFloat, typename cloverFloat>
void cloverCuda(spinorFloat *out, float *outNorm, const cloverFloat *clover,
		const float *cloverNorm, const spinorFloat *in, const float *inNorm, 
		const size_t bytes, const size_t norm_bytes, const dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<N>(bytes, norm_bytes, in, inNorm);
  cloverKernel<<<gridDim, blockDim, shared_bytes>>> 
    (out, outNorm, clover, cloverNorm, in, inNorm, dslashParam);
  unbindSpinorTex<N>(in, inNorm);
}

void cloverCuda(void *out, void *outNorm, const FullGauge gauge, const FullClover clover, 
		const void *in, const void *inNorm, const int parity, const int volume,
		const size_t bytes, const size_t norm_bytes, const QudaPrecision precision,
		const dim3 blockDim) {

  dslashParam.parity = parity;
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = volume;

#ifdef GPU_WILSON_DIRAC
  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(clover, parity, &cloverP, &cloverNormP);

  if (precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    cloverCuda<2>((double2*)out, (float*)outNorm, (double2*)cloverP, 
		  (float*)cloverNormP, (double2*)in, 
		  (float*)inNorm, bytes, norm_bytes, blockDim);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    cloverCuda<4>((float4*)out, (float*)outNorm, (float4*)cloverP, 
		  (float*)cloverNormP, (float4*)in, 
		  (float*)inNorm, bytes, norm_bytes, blockDim);
  } else if (precision == QUDA_HALF_PRECISION) {
    cloverCuda<4>((short4*)out, (float*)outNorm, (short4*)cloverP, 
		  (float*)cloverNormP, (short4*)in,
		  (float*)inNorm, bytes, norm_bytes, blockDim);
  }
  unbindCloverTex(clover);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}

// Clover wrappers
template <int N, typename spinorFloat, typename cloverFloat, typename gaugeFloat>
void cloverDslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat gauge0, 
		      const gaugeFloat gauge1, const QudaReconstructType reconstruct, 
		      const cloverFloat *clover, const float *cloverNorm, const spinorFloat *in, 
		      const float* inNorm, const int dagger, const spinorFloat *x, 
		      const float* xNorm, const double &a, const size_t bytes, 
		      const size_t norm_bytes, hipStream_t &stream, const int shared_bytes, 
		      const dim3 blockDim) 
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  if (x==0) { // not xpay
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	cloverDslash18Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam);
      } else {
	cloverDslash18DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	cloverDslash12Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam);
      } else {
	cloverDslash12DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam);
      }
    } else {
      if (!dagger) {
	cloverDslash8Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 	
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam);
      } else {
	cloverDslash8DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam);
      }
    }
  } else { // doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	cloverDslash18XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam, x, xNorm, a);
      } else {
	cloverDslash18DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam, x, xNorm, a);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	cloverDslash12XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam, x, xNorm, a);
      } else {
	cloverDslash12DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam, x, xNorm, a);
      }
    } else {
      if (!dagger) {
	cloverDslash8XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 	
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam, x, xNorm, a);
      } else {
	cloverDslash8DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, dslashParam, x, xNorm, a);
      }
    }
  }
}

template <int spinorN, typename spinorFloat, typename cloverFloat, typename gaugeFloat>
void cloverDslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat *gauge0, const gaugeFloat *gauge1, 
		      const QudaReconstructType reconstruct, const cloverFloat *clover, const float *cloverNorm, 
		      const spinorFloat *in, const float *inNorm, const int parity, const int dagger, 
		      const spinorFloat *x, const float *xNorm, const double &a, const int volume, 
		      const size_t bytes, const size_t norm_bytes, const dim3 block, const dim3 blockFace) {

  int shared_bytes = block.x*SHARED_FLOATS_PER_THREAD*
    bindSpinorTex<spinorN>(bytes, norm_bytes, in, inNorm, x, xNorm);

  dslashParam.parity = parity;

#ifndef MULTI_GPU
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = volume;

  cloverDslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, clover, cloverNorm, in, inNorm, 
			    dagger, x, xNorm, a, bytes, norm_bytes, streams[0], shared_bytes, block);
#else

  // Gather from source spinor
  face->exchangeFacesStart((void*)in, (void*)inNorm, stride, dagger, streams);
  
#ifdef OVERLAP_COMMS // do body
  dslashParam.tOffset = 1;
  dslashParam.tMul = 1;
  dslashParam.threads = volume - 2*Vspatial;
  cloverDslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, clover, cloverNorm, in, inNorm, 
			    dagger, x, xNorm, a, bytes, norm_bytes, streams[Nstream-1], shared_bytes,
			    block);
#endif // OVERLAP_COMMS

  // Finish gather and start comms
  face->exchangeFacesComms();

  // Wait for comms to finish, and scatter into the end zone
  face->exchangeFacesWait((void*)in, (void*)inNorm, stride, dagger);

  dslashParam.tOffset = 0;
#ifdef OVERLAP_COMMS // do faces
  dslashParam.tMul = volume/Vspatial - 1; // hacky way to get Nt
  dslashParam.threads = 2*Vspatial;
#else // do all
  dslashParam.tMul = 1;
  dslashParam.threads = volume;
#endif // OVERLAP_COMMS
  shared_bytes = blockFace.x*SHARED_FLOATS_PER_THREAD*
    bindSpinorTex<spinorN>(bytes, norm_bytes, in, inNorm, x, xNorm);
  cloverDslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, clover, cloverNorm, in, inNorm, 
			    dagger, x, xNorm, a, bytes, norm_bytes, streams[Nstream-2], shared_bytes,
			    blockFace);

#endif // MULTI_GPU

  // texture unbinding is not asynchronous so we don't need to cudaThreadSynchonize()
  unbindSpinorTex<spinorN>(in, inNorm, x, xNorm); 
}

void cloverDslashCuda(void *out, void *outNorm, const FullGauge gauge, const FullClover cloverInv,
		      const void *in, const void *inNorm, const int parity, const int dagger, 
		      const void *x, const void *xNorm, const double a, const int volume, 
		      const size_t bytes, const size_t norm_bytes, const QudaPrecision precision,
		      const dim3 block, const dim3 blockFace) {

#ifdef GPU_WILSON_DIRAC
  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    cloverDslashCuda<2>((double2*)out, (float*)outNorm, (double2*)gauge0, (double2*)gauge1, 
			gauge.reconstruct, (double2*)cloverP, (float*)cloverNormP, (double2*)in, 
			(float*)inNorm, parity, dagger, (double2*)x, (float*)xNorm, a, volume, bytes, 
			norm_bytes, block, blockFace);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    cloverDslashCuda<4>((float4*)out, (float*)outNorm, (float4*)gauge0, (float4*)gauge1, 
			gauge.reconstruct, (float4*)cloverP, (float*)cloverNormP, (float4*)in, 
			(float*)inNorm, parity, dagger, (float4*)x, (float*)xNorm, a, volume, bytes, 
			norm_bytes, block, blockFace);
  } else if (precision == QUDA_HALF_PRECISION) {
    cloverDslashCuda<4>((short4*)out, (float*)outNorm, (short4*)gauge0, (short4*)gauge1, 
			gauge.reconstruct, (short4*)cloverP, (float*)cloverNormP, (short4*)in,
			(float*)inNorm, parity, dagger, (short4*)x, (float*)xNorm, a, volume, bytes, 
			norm_bytes, block, blockFace);
  }

  unbindGaugeTex(gauge);
  unbindCloverTex(cloverInv);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}

// Domain wall wrappers
template <int N, typename spinorFloat, typename gaugeFloat>
void domainWallDslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat gauge0, 
			  const gaugeFloat gauge1, const QudaReconstructType reconstruct, 
			  const spinorFloat *in, const float* inNorm, 
			  const int dagger, const spinorFloat *x, const float* xNorm, 
			  const double &m_f, const double &k2, const size_t bytes, 
			  const size_t norm_bytes, const dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<N>(bytes, norm_bytes, in, inNorm, x, xNorm);

  if (x==0) { // not xpay
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	domainWallDslash18Kernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f);
      } else {
	domainWallDslash18DaggerKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	domainWallDslash12Kernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f);
      } else {
	domainWallDslash12DaggerKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f);
      }
    } else {
      if (!dagger) {
	domainWallDslash8Kernel <<<gridDim, blockDim, shared_bytes>>> 	
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f);
      } else {
	domainWallDslash8DaggerKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f);
      }
    }
  } else { // doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	domainWallDslash18XpayKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f, x, xNorm, k2);
      } else {
	domainWallDslash18DaggerXpayKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f, x, xNorm, k2);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	domainWallDslash12XpayKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f, x, xNorm, k2);
      } else {
	domainWallDslash12DaggerXpayKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f, x, xNorm, k2);
      }
    } else {
      if (!dagger) {
	domainWallDslash8XpayKernel <<<gridDim, blockDim, shared_bytes>>> 	
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f, x, xNorm, k2);
      } else {
	domainWallDslash8DaggerXpayKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, m_f, x, xNorm, k2);
      }
    }
  }

  unbindSpinorTex<N>(in, inNorm, x, xNorm);
}

void domainWallDslashCuda(void *out, void *outNorm, const FullGauge gauge, 
			  const void *in, const void *inNorm, const int parity, const int dagger, 
			  const void *x, const void *xNorm, const double m_f, const double k2, const int volume5d, 
			  const size_t bytes, const size_t norm_bytes, const QudaPrecision precision,
			  const dim3 block, const dim3 blockFace) {

#ifdef MULTI_GPU
  errorQuda("Multi-GPU domain wall not implemented\n");
#endif

  dslashParam.parity = parity;
  dslashParam.threads = volume5d;

#ifdef GPU_DOMAIN_WALL_DIRAC
  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    domainWallDslashCuda<2>((double2*)out, (float*)outNorm, (double2*)gauge0, (double2*)gauge1, 
			    gauge.reconstruct, (double2*)in, (float*)inNorm, dagger, 
			    (double2*)x, (float*)xNorm, m_f, k2, bytes, norm_bytes, block);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    domainWallDslashCuda<4>((float4*)out, (float*)outNorm, (float4*)gauge0, (float4*)gauge1, 
			    gauge.reconstruct, (float4*)in, (float*)inNorm, dagger, 
			    (float4*)x, (float*)xNorm, m_f, k2, bytes, norm_bytes, block);
  } else if (precision == QUDA_HALF_PRECISION) {
    domainWallDslashCuda<4>((short4*)out, (float*)outNorm, (short4*)gauge0, (short4*)gauge1, 
			    gauge.reconstruct, (short4*)in, (float*)inNorm, dagger, 
			    (short4*)x, (float*)xNorm, m_f, k2, bytes, norm_bytes, block);
  }

  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Domain wall dslash has not been built");
#endif

}


#define INTERIOR_KERNEL 0
#define EXTERIOR_KERNEL 1


static void
initTLocation(int toffset, int tmul, int threads) 
{
  short2 tLocate = make_short2((short)toffset, (short)tmul);
  hipMemcpyToSymbol(HIP_SYMBOL("tLocate"), &(tLocate), sizeof(short2));
  hipMemcpyToSymbol(HIP_SYMBOL("threads"), &(threads), sizeof(threads));

}

template <int spinorN, typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
  void staggeredDslashCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
			   const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
			   const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
			   const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
			   const double &a, const int volume, const int Vsh, const int tdim,
			   const int length, const int ghost_length,
			   cudaColorSpinorField* inSpinor, dim3 blockDim) {
    
  dim3 interiorGridDim( (dslashParam.threads + blockDim.x -1)/blockDim.x, 1, 1);
  dim3 exteriorGridDim( (6*Vsh + blockDim.x -1)/blockDim.x, 1, 1);

  int shared_bytes = blockDim.x*6*bindSpinorTex_mg<spinorN>(length, ghost_length, in, inNorm, x, xNorm); CUERR;

  initTLocation(0, INTERIOR_KERNEL, volume);  CUERR;
  if (x==0) { // not doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12Kernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											 longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      } else {
	staggeredDslash12DaggerKernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
												 longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8){
	  
      if (!dagger) {
	staggeredDslash8Kernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											 longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      } else {
	staggeredDslash8DaggerKernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											      longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value(%d) in function %s\n", reconstruct, __FUNCTION__);
    }
  } else { // doing xpay
    
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12AxpyKernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      } else {
	staggeredDslash12DaggerAxpyKernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												   longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {
      if (!dagger) {
	staggeredDslash8AxpyKernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											    longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      } else {
	staggeredDslash8DaggerAxpyKernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
												  longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value in function %s\n", __FUNCTION__);	  
    }    
  }


  exchange_gpu_spinor_start(inSpinor, &streams[1]); CUERR;
  exchange_gpu_spinor_wait(inSpinor, &streams[1]); CUERR;
  hipStreamSynchronize(streams[0]); CUERR;

  initTLocation(tdim-6,EXTERIOR_KERNEL , 6*Vsh);  
  if (x==0) { // not doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12Kernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
											 longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      } else {
	staggeredDslash12DaggerKernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1,
												longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8){
      
      if (!dagger) {
	staggeredDslash8Kernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											 longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      } else {
	staggeredDslash8DaggerKernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, dslashParam); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value(%d) in function %s\n", reconstruct, __FUNCTION__);
    }
  } else { // doing xpay
    
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12AxpyKernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      } else {
	staggeredDslash12DaggerAxpyKernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												   longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {
      if (!dagger) {
	staggeredDslash8AxpyKernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											    longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      } else {
	staggeredDslash8DaggerAxpyKernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
												  longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value in function %s\n", __FUNCTION__);	  
    }    
  }
  hipDeviceSynchronize(); CUERR;    
}

//This function is a special case for 18(no) reconstruct long link
//The reason is to make the type match easier(e.g float2 instead of float4)

template <int spinorN, typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
  void staggeredDslashNoReconCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
				  const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
				  const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
				  const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
				  const double &a, const int volume, const int Vsh, const int tdim,
				  const int length, const int ghost_length,
				  cudaColorSpinorField* inSpinor, dim3 blockDim) 
{
  
  
  dim3 interiorGridDim( (dslashParam.threads + blockDim.x -1)/blockDim.x, 1, 1);
  dim3 exteriorGridDim( (6*Vsh + blockDim.x -1)/blockDim.x, 1, 1);
  
  int shared_bytes = blockDim.x*6*bindSpinorTex_mg<spinorN>(length, ghost_length, in, inNorm, x, xNorm);
  
  initTLocation(0, INTERIOR_KERNEL, volume);  
  if (x==0) { // not doing xpay
    if (!dagger) {
      staggeredDslash18Kernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											       longGauge0, longGauge1, in, inNorm, dslashParam);CUERR;
    } else {
      staggeredDslash18DaggerKernel <<<interiorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
												      longGauge0, longGauge1, in, inNorm, dslashParam);CUERR;
    }    
  } else { // doing xpay
    
    if (!dagger) {
      staggeredDslash18AxpyKernel<<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												  longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
    } else {
      staggeredDslash18DaggerAxpyKernel<<<interiorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
													longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
    }          
  }
  exchange_gpu_spinor_start(inSpinor, &streams[1]);   
  exchange_gpu_spinor_wait(inSpinor, &streams[1]); 
  
  initTLocation(tdim-6,EXTERIOR_KERNEL , 6*Vsh);  
  if (x==0) { // not doing xpay
    if (!dagger) {
      staggeredDslash18Kernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
										       longGauge0, longGauge1, in, inNorm, dslashParam);CUERR;
    } else {
      staggeredDslash18DaggerKernel <<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
												      longGauge0, longGauge1, in, inNorm, dslashParam);CUERR;
    }    
  } else { // doing xpay
    
    if (!dagger) {
      staggeredDslash18AxpyKernel<<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,      
												 longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
    } else {
      staggeredDslash18AxpyKernel<<<exteriorGridDim, blockDim, shared_bytes, streams[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												  longGauge0, longGauge1, in, inNorm, dslashParam, x, xNorm, a); CUERR;
    }          
  }
  
  hipDeviceSynchronize();
}



void staggeredDslashCuda(void *out, void *outNorm, const FullGauge fatGauge, const FullGauge longGauge, 
			 cudaColorSpinorField *in,
			 const int parity, const int dagger, const void *x, const void *xNorm, 
			 const double k, const int volume, const int Vsh, const int tdim, 
			 const int length, const int ghost_length, const QudaPrecision precision,
			 const dim3 block, const dim3 blockFace)
{
  

#ifdef GPU_STAGGERED_DIRAC

#ifdef MULTI_GPU
  //errorQuda("Multi-GPU staggered not implemented\n");
#endif

  for(int i=0;i < 2 ;i ++){
    hipStreamCreate(&streams[i]); CUERR;
  }

  dslashParam.parity = parity;
  dslashParam.threads = volume;

  void *fatGauge0, *fatGauge1;
  void* longGauge0, *longGauge1;
  bindFatGaugeTex(fatGauge, parity, &fatGauge0, &fatGauge1);
  bindLongGaugeTex(longGauge, parity, &longGauge0, &longGauge1);
    
  if (precision != fatGauge.precision || precision != longGauge.precision){
    errorQuda("Mixing gauge and spinor precision not supported");
  }
    


  if (precision == QUDA_DOUBLE_PRECISION) {
    if (longGauge.reconstruct == QUDA_RECONSTRUCT_NO){
      staggeredDslashNoReconCuda<2>((double2*)out, (float*)outNorm, (double2*)fatGauge0, (double2*)fatGauge1, 			       
				    (double2*)longGauge0, (double2*)longGauge1,
				    longGauge.reconstruct, (double2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
				    (double2*)x, (float*)xNorm, k, volume, Vsh, tdim, length, ghost_length, in, block);
    }else{
      staggeredDslashCuda<2>((double2*)out, (float*)outNorm, (double2*)fatGauge0, (double2*)fatGauge1, 			       
			     (double2*)longGauge0, (double2*)longGauge1,
			     longGauge.reconstruct, (double2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
			     (double2*)x, (float*)xNorm, k, volume, Vsh, tdim, length, ghost_length, in, block);
    }
    
  } else if (precision == QUDA_SINGLE_PRECISION) {
    if (longGauge.reconstruct == QUDA_RECONSTRUCT_NO){
      staggeredDslashNoReconCuda<2>((float2*)out, (float*)outNorm, (float2*)fatGauge0, (float2*)fatGauge1,
				    (float2*)longGauge0, (float2*)longGauge1,
				    longGauge.reconstruct, (float2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
				    (float2*)x, (float*)xNorm, k, volume, Vsh, tdim, length, ghost_length, in, block);
    }else{
      staggeredDslashCuda<2>((float2*)out, (float*)outNorm, (float2*)fatGauge0, (float2*)fatGauge1,
			     (float4*)longGauge0, (float4*)longGauge1,
			     longGauge.reconstruct, (float2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
			     (float2*)x, (float*)xNorm, k, volume, Vsh, tdim, length, ghost_length, in, block);
    }
  } else if (precision == QUDA_HALF_PRECISION) {	
    if (longGauge.reconstruct == QUDA_RECONSTRUCT_NO){
      staggeredDslashNoReconCuda<2>((short2*)out, (float*)outNorm, (short2*)fatGauge0, (short2*)fatGauge1,
				    (short2*)longGauge0, (short2*)longGauge1,
				    longGauge.reconstruct, (short2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
				    (short2*)x, (float*)xNorm, k, volume, Vsh, tdim, length, ghost_length, in, block);
    }else{
      staggeredDslashCuda<2>((short2*)out, (float*)outNorm, (short2*)fatGauge0, (short2*)fatGauge1,
			     (short4*)longGauge0, (short4*)longGauge1,
			     longGauge.reconstruct, (short2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
			     (short2*)x, (float*)xNorm, k, volume, Vsh, tdim, length, ghost_length, in, block);
    }
  }

  for (int i = 0; i < 2; i++) {
    hipStreamDestroy(streams[i]);
  }
  
  if (!dslashTuning) checkCudaError();
  
#else
  errorQuda("Staggered dslash has not been built");
#endif  
}





void setTwistParam(double &a, double &b, const double &kappa, const double &mu, 
		   const int dagger, const QudaTwistGamma5Type twist) {
  if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
    a = 2.0 * kappa * mu;
    b = 1.0;
  } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
    a = -2.0 * kappa * mu;
    b = 1.0 / (1.0 + a*a);
  } else {
    errorQuda("Twist type %d not defined\n", twist);
  }
  if (dagger) a *= -1.0;

}

// FIXME: twist kernel cannot be issued asynchronously because of texture unbinding
template <int N, typename spinorFloat>
void twistGamma5Cuda(spinorFloat *out, float *outNorm, const spinorFloat *in, 
		     const float *inNorm, const int dagger, const double &kappa, 
		     const double &mu, const size_t bytes, const size_t norm_bytes, 
		     const QudaTwistGamma5Type twist, dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  double a=0.0, b=0.0;
  setTwistParam(a, b, kappa, mu, dagger, twist);

  bindSpinorTex<N>(bytes, norm_bytes, in, inNorm);
  twistGamma5Kernel<<<gridDim, blockDim, 0>>> (out, outNorm, a, b, dslashParam);
  unbindSpinorTex<N>(in, inNorm);
}

void twistGamma5Cuda(void *out, void *outNorm, const void *in, const void *inNorm,
		     const int dagger, const double kappa, const double mu, const int volume, 
		     const size_t bytes, const size_t norm_bytes, const QudaPrecision precision, 
		     const QudaTwistGamma5Type twist, const dim3 block) {

  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = volume;

#ifdef GPU_TWISTED_MASS_DIRAC
  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    twistGamma5Cuda<2>((double2*)out, (float*)outNorm, (double2*)in, (float*)inNorm, 
		       dagger, kappa, mu, bytes, norm_bytes, twist, block);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    twistGamma5Cuda<4>((float4*)out, (float*)outNorm, (float4*)in, (float*)inNorm, 
		       dagger, kappa, mu, bytes, norm_bytes, twist, block);
  } else if (precision == QUDA_HALF_PRECISION) {
    twistGamma5Cuda<4>((short4*)out, (float*)outNorm, (short4*)in, (float*)inNorm, 
		       dagger, kappa, mu, bytes, norm_bytes, twist, block);
  }
  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC

}

// Twisted mass wrappers
template <int N, typename spinorFloat, typename gaugeFloat>
void twistedMassDslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat gauge0, 
			   const gaugeFloat gauge1, const QudaReconstructType reconstruct, 
			   const spinorFloat *in, const float* inNorm, const int dagger, 
			   const spinorFloat *x, const float* xNorm, const double &kappa, 
			   const double &mu, const double &k, const int volume, 
			   const size_t bytes, const size_t norm_bytes, hipStream_t &stream, 
			   const int shared_bytes, const dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  double a=0.0, b=0.0;
  setTwistParam(a, b, kappa, mu, dagger, QUDA_TWIST_GAMMA5_INVERSE);

  if (x==0) { // not xpay
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	twistedMassDslash18Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b);
      } else {
	twistedMassDslash18DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	twistedMassDslash12Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b);
      } else {
	twistedMassDslash12DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b);
      }
    } else {
      if (!dagger) {
	twistedMassDslash8Kernel <<<gridDim, blockDim, shared_bytes, stream>>> 	
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b);
      } else {
	twistedMassDslash8DaggerKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b);
      }
    }
  } else { // doing xpay
    b *= k;
    if (reconstruct == QUDA_RECONSTRUCT_NO) {
      if (!dagger) {
	twistedMassDslash18XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b, x, xNorm);
      } else {
	twistedMassDslash18DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b, x, xNorm);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	twistedMassDslash12XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b, x, xNorm);
      } else {
	twistedMassDslash12DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b, x, xNorm);
      }
    } else {
      if (!dagger) {
	twistedMassDslash8XpayKernel <<<gridDim, blockDim, shared_bytes, stream>>> 	
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b, x, xNorm);
      } else {
	twistedMassDslash8DaggerXpayKernel <<<gridDim, blockDim, shared_bytes, stream>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, dslashParam, a, b, x, xNorm);
      }
    }
  }
  
  unbindSpinorTex<N>(in, inNorm, x, xNorm);
}

template <int spinorN, typename spinorFloat, typename gaugeFloat>
void twistedMassDslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat *gauge0, const gaugeFloat *gauge1, 
			   const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
			   const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
			   const double &kappa, const double &mu, const double &a, const int volume, 
			   const size_t bytes, const size_t norm_bytes, const dim3 block, const dim3 blockFace) {
  
  int shared_bytes = block.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<spinorN>(bytes, norm_bytes, in, inNorm, x, xNorm);

  dslashParam.parity = parity;

#ifndef MULTI_GPU
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = volume;

  twistedMassDslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, in, inNorm, 
				 dagger, x, xNorm, kappa, mu, a, volume, bytes, norm_bytes, 
				 streams[0], shared_bytes, block);
#else

  // Gather from source spinor
  face->exchangeFacesStart((void*)in, (void*)inNorm, stride, dagger, streams);
  
#ifdef OVERLAP_COMMS // do body
  dslashParam.tOffset = 1;
  dslashParam.tMul = 1;
  dslashParam.threads = volume - 2*Vspatial;
  twistedMassDslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, in, inNorm, 
				 dagger, x, xNorm, kappa, mu, a, volume, bytes, norm_bytes, 
				 streams[Nstream-1], shared_bytes, block);    
#endif // OVERLAP_COMMS

  // Finish gather and start comms
  face->exchangeFacesComms();

  // Wait for comms to finish, and scatter into the end zone
  face->exchangeFacesWait((void*)in, (void*)inNorm, stride, dagger);

  dslashParam.tOffset = 0;
#ifdef OVERLAP_COMMS // do faces
  dslashParam.tMul = volume/Vspatial - 1; // hacky way to get Nt
  dslashParam.threads = 2*Vspatial;
#else // do all
  dslashParam.tMul = 1;
  dslashParam.threads = volume;
#endif // OVERLAP_COMMS
  shared_bytes = blockFace.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<spinorN>(bytes, norm_bytes, in, inNorm, x, xNorm);
  twistedMassDslashCuda<spinorN>(out, outNorm, gauge0, gauge1, reconstruct, in, inNorm, 
				 dagger, x, xNorm, kappa, mu, a, volume, bytes, norm_bytes, 
				 streams[Nstream-2], shared_bytes, blockFace);    

#endif // MULTI_GPU

  // texture unbinding is not asynchronous so we don't need to cudaThreadSynchonize()
  unbindSpinorTex<spinorN>(in, inNorm, x, xNorm); 
}

void twistedMassDslashCuda(void *out, void *outNorm, const FullGauge gauge, 
			   const void *in, const void *inNorm, const int parity, const int dagger, 
			   const void *x, const void *xNorm, const double kappa, const double mu, 
			   const double a, const int volume, const size_t bytes, const size_t norm_bytes, 
			   const QudaPrecision precision, const dim3 block, const dim3 blockFace) {

#ifdef GPU_TWISTED_MASS_DIRAC
  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    twistedMassDslashCuda<2>((double2*)out, (float*)outNorm, (double2*)gauge0, (double2*)gauge1, 
			     gauge.reconstruct, (double2*)in, (float*)inNorm, parity, dagger, 
			     (double2*)x, (float*)xNorm, kappa, mu, a, volume, bytes, norm_bytes,
			     block, blockFace);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    twistedMassDslashCuda<4>((float4*)out, (float*)outNorm, (float4*)gauge0, (float4*)gauge1, 
			     gauge.reconstruct, (float4*)in, (float*)inNorm, parity, dagger, 
			     (float4*)x, (float*)xNorm, kappa, mu, a, volume, bytes, norm_bytes,
			     block, blockFace);
  } else if (precision == QUDA_HALF_PRECISION) {
    twistedMassDslashCuda<4>((short4*)out, (float*)outNorm, (short4*)gauge0, (short4*)gauge1, 
			     gauge.reconstruct, (short4*)in, (float*)inNorm, parity, dagger, 
			     (short4*)x, (float*)xNorm, kappa, mu, a, volume, bytes, norm_bytes,
			     block, blockFace);
  }

  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif

}


#if defined(GPU_FATLINK)||defined(GPU_GAUGE_FORCE)|| defined(GPU_FERMION_FORCE)
#include <force_common.h>
#include "force_kernel_common.cu"
#endif

#ifdef GPU_FATLINK
#include "misc_helpers.cu"
#include "llfat_quda.cu"
#endif

#ifdef GPU_GAUGE_FORCE
#include "gauge_force_quda.cu"
#endif

#ifdef GPU_FERMION_FORCE
#include "fermion_force_quda.cu"
#endif
