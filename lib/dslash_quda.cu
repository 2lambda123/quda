#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include <color_spinor_field.h>

#define BLOCK_DIM 64

// these control the Wilson-type actions
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR

//these are access control for staggered action
#if (__CUDA_ARCH__ >= 200)
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
#else
#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
#endif

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>

struct DslashParam {
  int tOffset; // offset into the T dimension (multi gpu only)
  int tMul;    // spatial volume distance between the T faces being updated (multi gpu only)
  int threads; // the desired number of active threads
  int parity;  // Even-Odd or Odd-Even
  int ghostDim[QUDA_MAX_DIM];
  int ghostOffset[QUDA_MAX_DIM];
};

DslashParam dslashParam;

// these are set in initDslashConst
int Vspatial;
#ifdef MULTI_GPU
static const int Nstream = 9;
#else
static const int Nstream = 1;
#endif
static hipStream_t streams[Nstream];

FaceBuffer *face;
cudaColorSpinorField *inSpinor;

#include <dslash_textures.h>
#include <dslash_constants.h>

#define SHORT_LENGTH 65536
#define SCALE_FLOAT ((SHORT_LENGTH-1) * 0.5) // 32767.5
#define SHIFT_FLOAT (-1.f / (SHORT_LENGTH-1)) // 1.5259021897e-5

__device__ short float2short(float c, float a) {
  //return (short)(a*MAX_SHORT);
  short rtn = (short)((a+SHIFT_FLOAT)*SCALE_FLOAT*c);
  return rtn;
}

__device__ float short2float(short a) {
  return (float)a/SCALE_FLOAT - SHIFT_FLOAT;
}

__device__ short4 float42short4(float c, float4 a) {
  return make_short4(float2short(c, a.x), float2short(c, a.y), float2short(c, a.z), float2short(c, a.w));
}

__device__ float4 short42float4(short4 a) {
  return make_float4(short2float(a.x), short2float(a.y), short2float(a.z), short2float(a.w));
}

__device__ float2 short22float2(short2 a) {
  return make_float2(short2float(a.x), short2float(a.y));
}


#include <staggered_dslash_def.h> // staggered Dslash kernels
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#include <dw_dslash_def.h>        // Domain Wall kernels
#include <tm_dslash_def.h>        // Twisted Mass kernels
#include <tm_core.h>              // solo twisted mass kernel
#include <clover_def.h>           // kernels for applying the clover term alone

#ifdef MULTI_GPU
#include <pack_face_def.h>        // kernels for packing the ghost zones
#endif

#ifndef SHARED_FLOATS_PER_THREAD
#define SHARED_FLOATS_PER_THREAD 0
#endif

#include <blas_quda.h>
#include <face_quda.h>


// dslashTuning = QUDA_TUNE_YES turns off error checking
static QudaTune dslashTuning = QUDA_TUNE_NO;

void setDslashTuning(QudaTune tune)
{
  dslashTuning = tune;
}

__global__ void dummyKernel() {
  // do nothing
}

void initCache() {

#if (__CUDA_ARCH__ >= 200)

  static int firsttime = 1;
  if (firsttime){	
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dummyKernel), hipFuncCachePreferL1);
    dummyKernel<<<1,1>>>();
    firsttime=0;
  }

#endif

}

void setFace(const FaceBuffer &Face) {
  face = (FaceBuffer*)&Face; // nasty
}

// Use an abstract class interface to drive the different CUDA dslash
// kernels.  All parameters are curried into the derived classes to
// allow a simple interface.
class DslashCuda {
public:
  DslashCuda() { ; }
  virtual ~DslashCuda() { ; }
  virtual void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) = 0;
};

// Use this macro for all dslash types
#define DSLASH(FUNC, X, gridDim, blockDim, shared, stream, ...)		\
if (x==0) {								\
  if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
    if (!dagger) {							\
      FUNC ## 18Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    } else {								\
      FUNC ## 18DaggerKernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    }									\
  } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
    if (!dagger) {							\
      FUNC ## 12Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    } else {								\
      FUNC ## 12DaggerKernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    }									\
  } else {								\
    if (!dagger) {							\
      FUNC ## 8Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    } else {								\
      FUNC ## 8DaggerKernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    }									\
  }									\
 } else {								\
  if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
    if (!dagger) {							\
      FUNC ## 18 ## X ## Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    } else {								\
      FUNC ## 18Dagger ## X ## Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    }									\
  } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
    if (!dagger) {							\
      FUNC ## 12 ## X ## Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    } else {								\
      FUNC ## 12Dagger ## X ## Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    }									\
  } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
    if (!dagger) {							\
      FUNC ## 8 ## X ## Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    } else {								\
      FUNC ## 8Dagger ## X ## Kernel <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ ); \
    }									\
  }									\
 }									
  

template <typename sFloat, typename gFloat>
class WilsonDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  const double a;

public:
  WilsonDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		   const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		   const sFloat *x, const float *xNorm, const double a,
		   const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), in(in), 
    inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, x, xNorm); 
  }
  virtual ~WilsonDslashCuda() { unbindSpinorTex(in, inNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(dslash, Xpay, gridDim, blockDim, shared_bytes, stream, out, outNorm, 
	   gauge0, gauge1, in, inNorm, x, xNorm, a, dslashParam);
  }

};

template <typename sFloat, typename gFloat, typename cFloat>
class CloverDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const cFloat *clover;
  const float *cloverNorm;
  const int dagger;
  const double a;

public:
  CloverDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		   const QudaReconstructType reconstruct, const cFloat *clover, 
		   const float *cloverNorm, const sFloat *in, const float *inNorm,
		   const sFloat *x, const float *xNorm, const double a,
		   const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    clover(clover), cloverNorm(cloverNorm), in(in), inNorm(inNorm), 
    reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, x, xNorm); 
  }
  virtual ~CloverDslashCuda() { unbindSpinorTex(in, inNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(cloverDslash, Xpay, gridDim, blockDim, shared_bytes, stream, out, outNorm, 
	   gauge0, gauge1, clover, cloverNorm, in, inNorm, x, xNorm, a, dslashParam);
  }

};

void setTwistParam(double &a, double &b, const double &kappa, const double &mu, 
		   const int dagger, const QudaTwistGamma5Type twist) {
  if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
    a = 2.0 * kappa * mu;
    b = 1.0;
  } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
    a = -2.0 * kappa * mu;
    b = 1.0 / (1.0 + a*a);
  } else {
    errorQuda("Twist type %d not defined\n", twist);
  }
  if (dagger) a *= -1.0;

}

template <typename sFloat, typename gFloat>
class TwistedDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  double a;
  double b;

public:
  TwistedDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		    const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		    const sFloat *x, const float *xNorm, const double kappa, const double mu,
		    const double k, const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    in(in), inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, x, xNorm); 
    setTwistParam(a, b, kappa, mu, dagger, QUDA_TWIST_GAMMA5_INVERSE);
    if (x) b *= k;
  }
  virtual ~TwistedDslashCuda() { unbindSpinorTex(in, inNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(twistedMassDslash, Xpay, gridDim, blockDim, shared_bytes, stream, out, outNorm, 
	   gauge0, gauge1, in, inNorm, a, b, x, xNorm, dslashParam);
  }

};

template <typename sFloat, typename gFloat>
class DomainWallDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  const double mferm;
  const double a;

public:
  DomainWallDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		       const QudaReconstructType reconstruct, const sFloat *in, 
		       const float *inNorm, const sFloat *x, const float *xNorm, const double mferm, 
		       const double a, const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    in(in), inNorm(inNorm), mferm(mferm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, x, xNorm); 
  }
  virtual ~DomainWallDslashCuda() { unbindSpinorTex(in, inNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const int shared_bytes, const hipStream_t &stream) {
    dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(domainWallDslash, Xpay, gridDim, blockDim, shared_bytes, stream, out, outNorm, 
	   gauge0, gauge1, in, inNorm, mferm, x, xNorm, a, dslashParam);
  }

};

void dslashCuda(DslashCuda &dslash, const size_t regSize, const int parity, const int dagger, 
		const int volume, const dim3 block, const dim3 blockFace) {
  int shared_bytes = block.x*SHARED_FLOATS_PER_THREAD*regSize;

  dslashParam.parity = parity;

#ifndef MULTI_GPU
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = volume;

  dslash.apply(block, shared_bytes, streams[0]);
#else

  // Gather from source spinor
  face->exchangeFacesStart(*inSpinor, 1-parity, dagger, streams);
  
#ifdef OVERLAP_COMMS // do body
  dslashParam.tOffset = 1;
  dslashParam.tMul = 1;
  dslashParam.threads = volume - 2*Vspatial;

  dslash.apply(block, shared_bytes, streams[Nstream-1]);
#endif // OVERLAP_COMMS

  // Finish gather and start comms
  face->exchangeFacesComms(3);

  // Wait for comms to finish, and scatter into the end zone
  face->exchangeFacesWait(*inSpinor, dagger,3);

  dslashParam.tOffset = 0;
#ifdef OVERLAP_COMMS // do faces
  dslashParam.tMul = volume/Vspatial - 1; // hacky way to get Nt
  dslashParam.threads = 2*Vspatial;
#else // do all
  dslashParam.tMul = 1;
  dslashParam.threads = volume;
#endif // OVERLAP_COMMS
  shared_bytes = blockFace.x*SHARED_FLOATS_PER_THREAD*regSize;
  dslash.apply(blockFace, shared_bytes, streams[Nstream-2]);

#endif // MULTI_GPU
}

// Wilson wrappers
void dslashCuda(cudaColorSpinorField *out, const FullGauge gauge, const cudaColorSpinorField *in,
		const int parity, const int dagger, const cudaColorSpinorField *x,
		const double &k, const dim3 &block, const dim3 &blockFace) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_WILSON_DIRAC
  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  void *xv = (x ? x->v : 0);
  void *xn = (x ? x->norm : 0);

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);
  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new WilsonDslashCuda<double2, double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, (double2*)gauge1, 
						    gauge.reconstruct, (double2*)in->v, (float*)in->norm, 
						    (double2*)xv, (float*)xn, k, dagger, in->bytes, in->norm_bytes);
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new WilsonDslashCuda<float4, float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, (float4*)gauge1,
						  gauge.reconstruct, (float4*)in->v, (float*)in->norm, 
						  (float4*)xv, (float*)xn, k, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new WilsonDslashCuda<short4, short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, (short4*)gauge1,
						  gauge.reconstruct, (short4*)in->v, (float*)in->norm,
						  (short4*)xv, (float*)xn, k, dagger, in->bytes, in->norm_bytes);
  }
  dslashCuda(*dslash, regSize, parity, dagger, in->volume, block, blockFace);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC

}

void cloverDslashCuda(cudaColorSpinorField *out, const FullGauge gauge, const FullClover cloverInv,
		      const cudaColorSpinorField *in, const int parity, const int dagger, 
		      const cudaColorSpinorField *x, const double &a,
		      const dim3 &block, const dim3 &blockFace) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_WILSON_DIRAC
  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (in->precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  void *xv = x ? x->v : 0;
  void *xn = x ? x->norm : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new CloverDslashCuda<double2, double2, double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, 
							     (double2*)gauge1, gauge.reconstruct, (double2*)cloverP, 
							     (float*)cloverNormP, (double2*)in->v, (float*)in->norm,
							     (double2*)xv, (float*)xn, a, dagger, in->bytes, in->norm_bytes);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new CloverDslashCuda<float4, float4, float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, 
							  (float4*)gauge1, gauge.reconstruct, (float4*)cloverP, 
							  (float*)cloverNormP, (float4*)in->v, (float*)in->norm, 
							  (float4*)xv, (float*)xn, a, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new CloverDslashCuda<short4, short4, short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, 
							  (short4*)gauge1, gauge.reconstruct, (short4*)cloverP, 
							  (float*)cloverNormP, (short4*)in->v, (float*)in->norm, 
							  (short4*)xv, (float*)xn, a, dagger, in->bytes, in->norm_bytes);
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->volume, block, blockFace);

  delete dslash;
  unbindGaugeTex(gauge);
  unbindCloverTex(cloverInv);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}


void twistedMassDslashCuda(cudaColorSpinorField *out, const FullGauge gauge, 
			   const cudaColorSpinorField *in, const int parity, const int dagger, 
			   const cudaColorSpinorField *x, const double &kappa, const double &mu, 
			   const double &a, const dim3 &block, const dim3 &blockFace) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_TWISTED_MASS_DIRAC
  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  void *xv = x ? x->v : 0;
  void *xn = x ? x->norm : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new TwistedDslashCuda<double2,double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, 
						    (double2*)gauge1, gauge.reconstruct, (double2*)in->v, 
						    (float*)in->norm, (double2*)xv, (float*)xn, 
						    kappa, mu, a, dagger, in->bytes, in->norm_bytes);
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new TwistedDslashCuda<float4,float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, (float4*)gauge1, 
						  gauge.reconstruct, (float4*)in->v, (float*)in->norm, 
						  (float4*)xv, (float*)xn, kappa, mu, a, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new TwistedDslashCuda<short4,short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, (short4*)gauge1, 
						  gauge.reconstruct, (short4*)in->v, (float*)in->norm, 
						  (short4*)xv, (float*)xn, kappa, mu, a, dagger, in->bytes, in->norm_bytes);
    
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->volume, block, blockFace);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif

}

void domainWallDslashCuda(cudaColorSpinorField *out, const FullGauge gauge, 
			  const cudaColorSpinorField *in, const int parity, const int dagger, 
			  const cudaColorSpinorField *x, const double &m_f, const double &k2,
			  const dim3 &block, const dim3 &blockFace) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef MULTI_GPU
  errorQuda("Multi-GPU domain wall not implemented\n");
#endif

  dslashParam.parity = parity;
  dslashParam.threads = in->volume;

#ifdef GPU_DOMAIN_WALL_DIRAC
  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  void *xv = x ? x->v : 0;
  void *xn = x ? x->norm : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new DomainWallDslashCuda<double2,double2>((double2*)out->v, (float*)out->norm, (double2*)gauge0, (double2*)gauge1, 
						       gauge.reconstruct, (double2*)in->v, (float*)in->norm, (double2*)xv, 
						       (float*)xn, m_f, k2, dagger, in->bytes, in->norm_bytes);
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    dslash = new DomainWallDslashCuda<float4,float4>((float4*)out->v, (float*)out->norm, (float4*)gauge0, (float4*)gauge1, 
						     gauge.reconstruct, (float4*)in->v, (float*)in->norm, (float4*)xv, 
						     (float*)xn, m_f, k2, dagger, in->bytes, in->norm_bytes);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    dslash = new DomainWallDslashCuda<short4,short4>((short4*)out->v, (float*)out->norm, (short4*)gauge0, (short4*)gauge1, 
						     gauge.reconstruct, (short4*)in->v, (float*)in->norm, (short4*)xv, 
						     (float*)xn, m_f, k2, dagger, in->bytes, in->norm_bytes);
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->volume, block, blockFace);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Domain wall dslash has not been built");
#endif

}

#define INTERIOR_KERNEL 0
#define EXTERIOR_KERNEL_X 1
#define EXTERIOR_KERNEL_Y 2
#define EXTERIOR_KERNEL_Z 3
#define EXTERIOR_KERNEL_T 4


void
initTLocation(int toffset, int tmul, int threads) 
{
  short2 tLocate = make_short2((short)toffset, (short)tmul);
  hipMemcpyToSymbol(HIP_SYMBOL("tLocate"), &(tLocate), sizeof(short2));
  hipMemcpyToSymbol(HIP_SYMBOL("threads"), &(threads), sizeof(threads));

}

template <typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
  void staggeredDslashCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
			   const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
			   const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
			   const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
			   const double &a, const int volume, const int* Vsh, const int* dims,
			   const int length, const int ghost_length, dim3 blockDim) {
    
  dim3 interiorGridDim( (dslashParam.threads + blockDim.x -1)/blockDim.x, 1, 1);
  dim3 exteriorGridDim[4]  = {
    dim3((6*Vsh[0] + blockDim.x -1)/blockDim.x, 1, 1),
    dim3((6*Vsh[1] + blockDim.x -1)/blockDim.x, 1, 1),
    dim3((6*Vsh[2] + blockDim.x -1)/blockDim.x, 1, 1),
    dim3((6*Vsh[3] + blockDim.x -1)/blockDim.x, 1, 1)
  };
    
  int shared_bytes = blockDim.x*6*bindSpinorTex_mg(length, ghost_length, in, inNorm, x, xNorm); CUERR;

#ifdef DSLASH_PROFILE  
  hipEvent_t interior_start, interior_stop;
  hipEvent_t exterior_start[4], exterior_stop[4];
  struct timeval comm_start[4], comm_stop[4];
  struct timeval dslash_start, dslash_stop;
  hipEventCreate(&interior_start);
  hipEventCreate(&interior_stop);
  for(int i=0;i < 4;i++){
    hipEventCreate(&exterior_start[i]);
    hipEventCreate(&exterior_stop[i]);
  }
  hipDeviceSynchronize();
  gettimeofday(&dslash_start, NULL);  
#endif
  
  initTLocation(0, INTERIOR_KERNEL, volume);  CUERR;
  
#ifdef MULTI_GPU
  // Gather from source spinor
  face->exchangeFacesStart(*inSpinor, 1-parity, dagger, streams);
#endif

#ifdef DSLASH_PROFILE  
  hipEventRecord(interior_start, streams[Nstream-1]);
#endif

  DSLASH(staggeredDslash, Axpy, interiorGridDim, blockDim, shared_bytes, streams[Nstream-1], out, outNorm, 
	 fatGauge0, fatGauge1, longGauge0, longGauge1, in, inNorm, x, xNorm, a, dslashParam); CUERR;
#ifdef DSLASH_PROFILE  
  hipEventRecord(interior_stop, streams[Nstream-1]);
#endif

#ifdef MULTI_GPU


  int exterior_kernel_flag[4]={
    EXTERIOR_KERNEL_X, EXTERIOR_KERNEL_Y, EXTERIOR_KERNEL_Z, EXTERIOR_KERNEL_T
  };
  for(int i=0 ;i < 4;i++){
    if(!commDimPartitioned(i)){
      continue;
    }
#ifdef DSLASH_PROFILE  
    gettimeofday(&comm_start[i], NULL);
#endif
    // Finish gather and start comms
    face->exchangeFacesComms(i);
    // Wait for comms to finish, and scatter into the end zone
    face->exchangeFacesWait(*inSpinor, dagger,i);    
    
#ifdef DSLASH_PROFILE  
    gettimeofday(&comm_stop[i], NULL);
#endif

    initTLocation(dims[i]-6, exterior_kernel_flag[i] , 6*Vsh[i]);  

#ifdef DSLASH_PROFILE  
    hipEventRecord(exterior_start[i], streams[Nstream-1]);
#endif
    DSLASH(staggeredDslash, Axpy, exteriorGridDim[i], blockDim, shared_bytes, streams[Nstream-1], out, outNorm, 
	   fatGauge0, fatGauge1, longGauge0, longGauge1, in, inNorm, x, xNorm, a, dslashParam); CUERR;
#ifdef DSLASH_PROFILE  
    hipEventRecord(exterior_stop[i], streams[Nstream-1]);
#endif
  }

#ifdef DSLASH_PROFILE  
  hipDeviceSynchronize();
  gettimeofday(&dslash_stop, NULL);
  float interior_time, exterior_time[4], comm_time[4], dslash_time;
  hipEventElapsedTime(&interior_time, interior_start, interior_stop);
  dslash_time = (dslash_stop.tv_sec - dslash_start.tv_sec)*1e+3
    + (dslash_stop.tv_usec - dslash_start.tv_usec)*1e-3;
  printfQuda("Interior kernel: %.2f ms, overall dslash time=%.2f ms\n", interior_time, dslash_time); 
  for(int i=0;i < 4;i++){
    if(commDimPartitioned(i)){
      hipEventElapsedTime(&exterior_time[i], exterior_start[i], exterior_stop[i]);
#define TDIFF(a,b) ((a.tv_sec - b.tv_sec)*1e+3 + (a.tv_usec - b.tv_usec)*1e-3)
      comm_time[i] = TDIFF(comm_stop[i], comm_start[i]);
      printfQuda("dir=%d, comm=%.2f ms, exterior kernel=%.2f ms\n", i, comm_time[i], exterior_time[i]); 
    }
  }

  
  hipEventDestroy(interior_start);
  hipEventDestroy(interior_stop);
  for(int i=0;i < 4;i++){
    hipEventDestroy(exterior_start[i]);
    hipEventDestroy(exterior_stop[i]);
  }

#endif

#endif
}

void staggeredDslashCuda(cudaColorSpinorField *out, const FullGauge fatGauge, 
			 const FullGauge longGauge, const cudaColorSpinorField *in,
			 const int parity, const int dagger, const cudaColorSpinorField *x,
			 const double &k, const dim3 &block, const dim3 &blockFace)
{
  
  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_STAGGERED_DIRAC

  dslashParam.parity = parity;
  dslashParam.threads = in->volume;
  for(int i=0;i < 4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i);
    dslashParam.ghostOffset[i] = in->ghostOffset[i];
  }
  void *fatGauge0, *fatGauge1;
  void* longGauge0, *longGauge1;
  bindFatGaugeTex(fatGauge, parity, &fatGauge0, &fatGauge1);
  bindLongGaugeTex(longGauge, parity, &longGauge0, &longGauge1);
    
  if (in->precision != fatGauge.precision || in->precision != longGauge.precision){
    errorQuda("Mixing gauge and spinor precision not supported"
	      "(precision=%d, fatlinkGauge.precision=%d, longGauge.precision=%d",
	      in->precision, fatGauge.precision, longGauge.precision);
  }
    
  int Vsh[] = {
    in->x[1]*in->x[2]*in->x[3]/2,
    in->x[0]*in->x[2]*in->x[3],
    in->x[0]*in->x[1]*in->x[3],
    in->x[0]*in->x[1]*in->x[2]};

  void *xv = x ? x->v : 0;
  void *xn = x ? x->norm : 0;

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    staggeredDslashCuda((double2*)out->v, (float*)out->norm, (double2*)fatGauge0, (double2*)fatGauge1,
			(double2*)longGauge0, (double2*)longGauge1, longGauge.reconstruct, 
			(double2*)in->v, (float*)in->norm, parity, dagger, 
			(double2*)xv, (float*)x, k, in->volume, Vsh, 
			in->x, in->length, in->ghost_length, block);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    staggeredDslashCuda((float2*)out->v, (float*)out->norm, (float2*)fatGauge0, (float2*)fatGauge1,
			(float4*)longGauge0, (float4*)longGauge1, longGauge.reconstruct, 
			(float2*)in->v, (float*)in->norm, parity, dagger, 
			(float2*)xv, (float*)xn, k, in->volume, Vsh, 
			in->x, in->length, in->ghost_length, block);
  } else if (in->precision == QUDA_HALF_PRECISION) {	
    staggeredDslashCuda((short2*)out->v, (float*)out->norm, (short2*)fatGauge0, (short2*)fatGauge1,
			(short4*)longGauge0, (short4*)longGauge1, longGauge.reconstruct, 
			(short2*)in->v, (float*)in->norm, parity, dagger, 
			(short2*)xv, (float*)xn, k, in->volume, Vsh, 
			in->x, in->length, in->ghost_length, block);
  }

  if (!dslashTuning) checkCudaError();
  
#else
  errorQuda("Staggered dslash has not been built");
#endif  
}


template <typename spinorFloat, typename cloverFloat>
void cloverCuda(spinorFloat *out, float *outNorm, const cloverFloat *clover,
		const float *cloverNorm, const spinorFloat *in, const float *inNorm, 
		const size_t bytes, const size_t norm_bytes, const dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex(bytes, norm_bytes, in, inNorm);
  cloverKernel<<<gridDim, blockDim, shared_bytes>>> 
    (out, outNorm, clover, cloverNorm, in, inNorm, dslashParam);
  unbindSpinorTex(in, inNorm);
}

void cloverCuda(cudaColorSpinorField *out, const FullGauge gauge, const FullClover clover, 
		const cudaColorSpinorField *in, const int parity, const dim3 &blockDim) {

  dslashParam.parity = parity;
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = in->volume;

#ifdef GPU_WILSON_DIRAC
  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(clover, parity, &cloverP, &cloverNormP);

  if (in->precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    cloverCuda((double2*)out->v, (float*)out->norm, (double2*)cloverP, 
	       (float*)cloverNormP, (double2*)in->v, (float*)in->norm, 
	       in->bytes, in->norm_bytes, blockDim);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    cloverCuda((float4*)out->v, (float*)out->norm, (float4*)cloverP, 
	       (float*)cloverNormP, (float4*)in->v, (float*)in->norm,
	       in->bytes, in->norm_bytes, blockDim);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    cloverCuda((short4*)out->v, (float*)out->norm, (short4*)cloverP, 
	       (float*)cloverNormP, (short4*)in->v, (float*)in->norm, 
	       in->bytes, in->norm_bytes, blockDim);
  }
  unbindCloverTex(clover);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}
// FIXME: twist kernel cannot be issued asynchronously because of texture unbinding
template <typename spinorFloat>
void twistGamma5Cuda(spinorFloat *out, float *outNorm, const spinorFloat *in, 
		     const float *inNorm, const int dagger, const double &kappa, 
		     const double &mu, const size_t bytes, const size_t norm_bytes, 
		     const QudaTwistGamma5Type twist, dim3 blockDim)
{
  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  double a=0.0, b=0.0;
  setTwistParam(a, b, kappa, mu, dagger, twist);

  bindSpinorTex(bytes, norm_bytes, in, inNorm);
  twistGamma5Kernel<<<gridDim, blockDim, 0>>> (out, outNorm, a, b, dslashParam);
  unbindSpinorTex(in, inNorm);
}

void twistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		     const int dagger, const double &kappa, const double &mu,
		     const QudaTwistGamma5Type twist, const dim3 &block) {

  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = in->Volume();

#ifdef GPU_TWISTED_MASS_DIRAC
  if (in->precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    twistGamma5Cuda((double2*)out->v, (float*)out->norm, 
		    (double2*)in->v, (float*)in->norm, 
		    dagger, kappa, mu, in->bytes, 
		    in->norm_bytes, twist, block);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->precision == QUDA_SINGLE_PRECISION) {
    twistGamma5Cuda((float4*)out->v, (float*)out->norm,
		    (float4*)in->v, (float*)in->norm, 
		    dagger, kappa, mu, in->bytes, 
		    in->norm_bytes, twist, block);
  } else if (in->precision == QUDA_HALF_PRECISION) {
    twistGamma5Cuda((short4*)out->v, (float*)out->norm,
		    (short4*)in->v, (float*)in->norm, 
		    dagger, kappa, mu, in->bytes, 
		    in->norm_bytes, twist, block);
  }
  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
}


#include "misc_helpers.cu"


#if defined(GPU_FATLINK)||defined(GPU_GAUGE_FORCE)|| defined(GPU_FERMION_FORCE)
#include <force_common.h>
#include "force_kernel_common.cu"
#endif

#ifdef GPU_FATLINK
#include "llfat_quda.cu"
#endif

#ifdef GPU_GAUGE_FORCE
#include "gauge_force_quda.cu"
#endif

#ifdef GPU_FERMION_FORCE
#include "fermion_force_quda.cu"
#endif
