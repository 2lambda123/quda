#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
//#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

//these are access control for staggered action
#ifdef GPU_STAGGERED_DIRAC
#if (__COMPUTE_CAPABILITY__ >= 300) // Kepler works best with texture loads only
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#else // fermi
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#endif
#endif // GPU_STAGGERED_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace dslash_aux {
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>
#include <io_spinor.h>

//#include <tm_core.h>              // solo twisted mass kernel
//#include <tmc_core.h>              // solo twisted mass kernel
//#include <clover_def.h>           // kernels for applying the clover term alone
  }

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

  // these should not be namespaced!!
  // determines whether the temporal ghost zones are packed with a gather kernel,
  // as opposed to multiple calls to hipMemcpy()
  static bool kernelPackT = false;

  void setKernelPackT(bool packT) { kernelPackT = packT; }

  bool getKernelPackT() { return kernelPackT; }

  namespace dslash {
    int it = 0;

#ifdef PTHREADS
    hipEvent_t interiorDslashEnd;
#endif
    hipEvent_t packEnd[Nstream];
    hipEvent_t gatherStart[Nstream];
    hipEvent_t gatherEnd[Nstream];
    hipEvent_t scatterStart[Nstream];
    hipEvent_t scatterEnd[Nstream];
    hipEvent_t dslashStart;
    hipEvent_t dslashEnd;

    // FIX this is a hack from hell
    // Auxiliary work that can be done while waiting on comms to finis
    Worker *aux_worker;

#if CUDA_VERSION >= 8000
    cuuint32_t *commsEnd_h;
    hipDeviceptr_t commsEnd_d[Nstream];
#endif
  }

  void createDslashEvents()
  {
    using namespace dslash;
    // add hipEventDisableTiming for lower sync overhead
    for (int i=0; i<Nstream; i++) {
      hipEventCreate(&packEnd[i], hipEventDisableTiming);
      hipEventCreate(&gatherStart[i], hipEventDisableTiming);
      hipEventCreate(&gatherEnd[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterStart[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterEnd[i], hipEventDisableTiming);
    }
    hipEventCreateWithFlags(&dslashStart, hipEventDisableTiming);
    hipEventCreateWithFlags(&dslashEnd, hipEventDisableTiming);
#ifdef PTHREADS
    hipEventCreateWithFlags(&interiorDslashEnd, hipEventDisableTiming);
#endif

    aux_worker = NULL;

#if CUDA_VERSION >= 8000
    commsEnd_h = static_cast<cuuint32_t*>(mapped_malloc(Nstream*sizeof(int)));
    for (int i=0; i<Nstream; i++) {
      hipHostGetDevicePointer((void**)&commsEnd_d[i], commsEnd_h+i, 0);
      commsEnd_h[i] = 0;
    }
#endif

    checkCudaError();
  }


  void destroyDslashEvents()
  {
    using namespace dslash;

#if CUDA_VERSION >= 8000
    host_free(commsEnd_h);
    commsEnd_h = 0;
#endif

    for (int i=0; i<Nstream; i++) {
      hipEventDestroy(packEnd[i]);
      hipEventDestroy(gatherStart[i]);
      hipEventDestroy(gatherEnd[i]);
      hipEventDestroy(scatterStart[i]);
      hipEventDestroy(scatterEnd[i]);
    }

    hipEventDestroy(dslashStart);
    hipEventDestroy(dslashEnd);
#ifdef PTHREADS
    hipEventDestroy(interiorDslashEnd);
#endif

    checkCudaError();
  }

  using namespace dslash_aux;

} // namespace quda

#include "contract.cu"
