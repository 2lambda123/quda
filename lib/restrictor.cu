#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <hipcub/hipcub.hpp>
#include <typeinfo>
#include <multigrid_helper.cuh>

namespace quda {

#ifdef GPU_MULTIGRID

  using namespace quda::colorspinor;

  /** 
      Kernel argument struct
  */
  template <typename Out, typename In, typename Rotator, int fineSpin, int coarseSpin>
  struct RestrictArg {
    Out out;
    const In in;
    const Rotator V;
    const int *fine_to_coarse;
    const int *coarse_to_fine;
    const spin_mapper<fineSpin,coarseSpin> spin_map;

    RestrictArg(Out &out, const In &in, const Rotator &V,
		const int *fine_to_coarse, const int *coarse_to_fine) : 
      out(out), in(in), V(V), fine_to_coarse(fine_to_coarse), coarse_to_fine(coarse_to_fine), spin_map()
    { }

    RestrictArg(const RestrictArg<Out,In,Rotator,fineSpin,coarseSpin> &arg) :
      out(arg.out), in(arg.in), V(arg.V), 
      fine_to_coarse(arg.fine_to_coarse), coarse_to_fine(arg.coarse_to_fine), spin_map()
    { }
  };


  /**
     Rotates from the fine-color basis into the coarse-color basis.
     A.S.: also works for staggered (fineSpin = 1)
  */
  template <typename Float, int fineSpin, int fineColor, int coarseColor, class FineColor, class Rotator>
  __device__ __host__ inline void rotateCoarseColor(complex<Float> out[fineSpin*coarseColor],
						    const FineColor &in, const Rotator &V, int parity, int x_cb) {
    for (int s=0; s<fineSpin; s++)
      for (int i=0; i<coarseColor; i++) out[s*coarseColor+i] = 0.0;

    for (int i=0; i<coarseColor; i++) { // coarse color
      for (int s=0; s<fineSpin; s++) {
	for (int j=0; j<fineColor; j++) {
	  out[s*coarseColor + i] += conj(V(parity, x_cb, s, j, i)) * in(parity, x_cb, s, j);
	}
      }
    }
  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  void Restrict(Arg arg) {
    for (int parity_coarse=0; parity_coarse<2; parity_coarse++) 
      for (int x_coarse_cb=0; x_coarse_cb<arg.out.VolumeCB(); x_coarse_cb++)
	for (int s=0; s<coarseSpin; s++) 
	  for (int c=0; c<coarseColor; c++)
	    arg.out(parity_coarse, x_coarse_cb, s, c) = 0.0;

    // loop over fine degrees of freedom
    for (int parity=0; parity<2; parity++) {
      for (int x_cb=0; x_cb<arg.in.VolumeCB(); x_cb++) {
	complex<Float> tmp[fineSpin*coarseColor];
	rotateCoarseColor<Float,fineSpin,fineColor,coarseColor>(tmp, arg.in, arg.V, parity, x_cb);

	int x = parity*arg.in.VolumeCB() + x_cb;
	int x_coarse = arg.fine_to_coarse[x];
	int parity_coarse = (x_coarse >= arg.out.VolumeCB()) ? 1 : 0;
	int x_coarse_cb = x_coarse - parity_coarse*arg.out.VolumeCB();
	
        if(fineSpin == 1)
        {
           int staggered_coarse_spin = parity; //0 if fine parity even, 1 otherwise
           for (int c=0; c<coarseColor; c++)
	      arg.out(parity_coarse,x_coarse_cb,staggered_coarse_spin,c) += tmp[c];
        }
        else
        {
	  for (int s=0; s<fineSpin; s++) 
	    for (int c=0; c<coarseColor; c++)
	      arg.out(parity_coarse,x_coarse_cb,arg.spin_map(s),c) += tmp[s*coarseColor+c];
        }
      }
    }

    return;
  }

  /**
     Here, we ensure that each thread block maps exactly to a
     geometric block.  Each thread block corresponds to one geometric
     block, with number of threads equal to the number of fine grid
     points per aggregate, so each thread represents a fine-grid
     point.  The look up table coarse_to_fine is the mapping to the
     each fine grid point.
  */
  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, typename Arg, int block_size>
  __global__ void RestrictKernel(Arg arg) {
    int x_coarse = blockIdx.x;
    int parity_coarse = x_coarse >= arg.out.VolumeCB() ? 1 : 0;
    int x_coarse_cb = x_coarse - parity_coarse*arg.out.VolumeCB();

    // obtain fine index from this look up table
    // since both parities map to the same block, each thread block must do both parities

    // threadIdx.x - fine checkboard offset
    // threadIdx.y - fine parity offset
    // blockIdx.x  - which coarse block are we working on
    // assume that coarse_to_fine look up map is ordered as (coarse-block-id + fine-point-id)
    // and that fine-point-id is parity ordered
    int x_fine = arg.coarse_to_fine[ (blockIdx.x*blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
    int parity = threadIdx.y;
    int x_fine_cb = x_fine - parity*arg.in.VolumeCB();

    complex<Float> tmp[fineSpin*coarseColor];
    rotateCoarseColor<Float,fineSpin,fineColor,coarseColor>(tmp, arg.in, arg.V, parity, x_fine_cb);

    complex<Float> reduced[coarseSpin * coarseColor];
    for (int i=0; i<coarseSpin*coarseColor; i++) reduced[i] = 0.0;//Why the class constructor does not initialize it to zero?

    if(fineSpin != 1)
    {
      // first lets coarsen spin locally
      for (int s=0; s<fineSpin; s++) {
        for (int v=0; v<coarseColor; v++) {
	  reduced[arg.spin_map(s)*coarseColor+v] += tmp[s*coarseColor+v];
        }
      }

      // now lets coarse geometry across threads
      typedef hipcub::BlockReduce<complex<Float>, block_size, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 2> BlockReduce;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      for (int s=0; s<coarseSpin; s++) {
        for (int v=0; v<coarseColor; v++) {
	  reduced[s*coarseColor+v] = BlockReduce(temp_storage).Sum( reduced[s*coarseColor+v] );
	  __syncthreads();
        }
      }
    }
    else//staggered block (temporary hack)
    {
      
      for (int s=0; s<coarseSpin; s++) {
        for (int v=0; v<coarseColor; v++) {
	  reduced[s*coarseColor+v] += (s == parity) ? tmp[v] : 0.0;
        }
      }

      // now lets coarse geometry across threads
      typedef hipcub::BlockReduce<complex<Float>, block_size, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 2> BlockReduce;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      for (int s=0; s<coarseSpin; s++) {
        for (int v=0; v<coarseColor; v++) {
	  reduced[s*coarseColor+v] = BlockReduce(temp_storage).Sum( reduced[s*coarseColor+v] );
	  __syncthreads();
        }
      }
    }

    if (threadIdx.x==0 && threadIdx.y == 0) {
      for (int s=0; s<coarseSpin; s++) { // hard code coarse spin to 2 for now
	for (int v=0; v<coarseColor; v++) {
	  arg.out(parity_coarse, x_coarse_cb, s, v) = reduced[s*coarseColor+v];
	}
      }

    }

    return;

  }

  template <typename Float, typename Arg, int fineSpin, int fineColor, int coarseSpin, int coarseColor>
  class RestrictLaunch : public Tunable {

  protected:
    Arg &arg;
    QudaFieldLocation location;
    const int block_size;
    char vol[TuneKey::volume_n];

    long long flops() const { return 0; }
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.in.VolumeCB(); } // fine parity is the block y dimension

  public:
    RestrictLaunch(Arg &arg, const ColorSpinorField &coarse, const ColorSpinorField &fine, 
		   const QudaFieldLocation location) : arg(arg), location(location), 
	block_size((arg.in.VolumeCB())/arg.out.Volume()) {
      strcpy(vol, coarse.VolString());
      strcat(vol, ",");
      strcat(vol, fine.VolString());

      strcpy(aux, coarse.AuxString());
      strcat(aux, ",");
      strcat(aux, fine.AuxString());
    } // block size is checkerboard fine length / full coarse length
    virtual ~RestrictLaunch() { }

    void apply(const hipStream_t &stream) {
      if (location == QUDA_CPU_FIELD_LOCATION) {
	Restrict<Float,fineSpin,fineColor,coarseSpin,coarseColor>(arg);
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	tp.block.y = 2; // need factor of two for fine parity with in the block

	if (block_size == 8) {
	  RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,Arg,8>
	    <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	} else if (block_size == 16) {
	  RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,Arg,16>
	    <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	} else if (block_size == 128) {
	  RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,Arg,128>
	  <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	} else {
	  errorQuda("Block size %d not instantiated", block_size);
	}
      }
    }

    // only tune shared memory per thread since grid and block sizes are fixed
    bool advanceTuneParam(TuneParam &param) const { return advanceSharedBytes(param); }

    TuneKey tuneKey() const {
      return TuneKey(vol, typeid(*this).name(), aux);
    }

    void initTuneParam(TuneParam &param) const { defaultTuneParam(param); }

    /** sets default values for when tuning is disabled */
    void defaultTuneParam(TuneParam &param) const {
      param.block = dim3(block_size, 1, 1);
      param.grid = dim3( (minThreads()+param.block.x-1) / param.block.x, 1, 1);
      param.shared_bytes = 0;
    }

    long long bytes() const {
      return arg.in.Bytes() + arg.out.Bytes() + arg.V.Bytes() + arg.in.Volume()*sizeof(int);
    }

  };

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, QudaFieldOrder order>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		const int *fine_to_coarse, const int *coarse_to_fine) {

    typedef FieldOrderCB<Float,fineSpin,fineColor,1,order> fineSpinor;
    typedef FieldOrderCB<Float,coarseSpin,coarseColor,1,order> coarseSpinor;
    typedef FieldOrderCB<Float,fineSpin,fineColor,coarseColor,order> packedSpinor;
    typedef RestrictArg<coarseSpinor,fineSpinor,packedSpinor,fineSpin,coarseSpin> Arg;

    coarseSpinor Out(const_cast<ColorSpinorField&>(out));
    fineSpinor   In(const_cast<ColorSpinorField&>(in));
    packedSpinor V(const_cast<ColorSpinorField&>(v));

    Arg arg(Out, In, V, fine_to_coarse,coarse_to_fine);
    RestrictLaunch<Float, Arg, fineSpin, fineColor, coarseSpin, coarseColor> restrictor(arg, out, in, Location(out, in, v));
    restrictor.apply(0);

    if (Location(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, QudaFieldOrder order>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int nVec, const int *fine_to_coarse, const int *coarse_to_fine, const int *spin_map) {

    // first check that the spin_map matches the spin_mapper
    if(spin_map != NULL) //spin_map is undefined for the top level staggered fermions.
    {
      spin_mapper<fineSpin,coarseSpin> mapper;
      for (int s=0; s<fineSpin; s++) 
        if (mapper(s) != spin_map[s]) errorQuda("Spin map does not match spin_mapper");
    }
 
    if (nVec == 2) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,2,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 4) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,4,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 8) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,8,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 12) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,12,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 16) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,16,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 20) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,20,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 24) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,24,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 48) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,48,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else if (nVec == 96) {
      Restrict<Float,fineSpin,fineColor,coarseSpin,96,order>(out, in, v, fine_to_coarse, coarse_to_fine);
    } else {
      errorQuda("Unsupported nVec %d", nVec);
    }
  }

  template <typename Float, int fineSpin, QudaFieldOrder order>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int Nvec, const int *fine_to_coarse, const int *coarse_to_fine, const int *spin_map) {
    if (out.Nspin() != 2) errorQuda("coarseSpin is not supported");

    if (out.Nspin() != 2) errorQuda("Unsupported nSpin %d", out.Nspin());

    if (in.Ncolor() == 3) {
      Restrict<Float,fineSpin,3, 2,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (in.Ncolor() == 2) {
      Restrict<Float,fineSpin,2, 2,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (in.Ncolor() == 8) {
      Restrict<Float,fineSpin,8, 2,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (in.Ncolor() == 16) {
      Restrict<Float,fineSpin,16, 2,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (in.Ncolor() == 24) {
      Restrict<Float,fineSpin,24, 2,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (in.Ncolor() == 48) {
      Restrict<Float,fineSpin,48, 2,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else {
      errorQuda("Unsupported nColor %d", in.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder order>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int Nvec, const int *fine_to_coarse, const int *coarse_to_fine, const int *spin_map) {

    if (in.Nspin() == 4) {
      Restrict<Float,4,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (in.Nspin() == 2) {
      Restrict<Float,2,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
#if GPU_STAGGERED_DIRAC
    } else if (in.Nspin() == 1) {
      Restrict<Float,1,order>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
#endif
    } else {
      errorQuda("Unsupported nSpin %d", in.Nspin());
    }
  }


  template <typename Float>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int Nvec, const int *fine_to_coarse, const int *coarse_to_fine, const int *spin_map) {

    if (out.FieldOrder() != in.FieldOrder() ||	out.FieldOrder() != v.FieldOrder())
      errorQuda("Field orders do not match (out=%d, in=%d, v=%d)", 
		out.FieldOrder(), in.FieldOrder(), v.FieldOrder());

    if (out.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      Restrict<Float,QUDA_FLOAT2_FIELD_ORDER>
	(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (out.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      Restrict<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>
	(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else {
      errorQuda("Unsupported field type %d", out.FieldOrder());
    }
  }

#endif // GPU_MULTIGRID

  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int Nvec, const int *fine_to_coarse, const int *coarse_to_fine, const int *spin_map) {

#ifdef GPU_MULTIGRID
    if (out.Precision() != in.Precision() || v.Precision() != in.Precision())
      errorQuda("Precision mismatch out=%d in=%d v=%d", out.Precision(), in.Precision(), v.Precision());

    if (out.Precision() == QUDA_DOUBLE_PRECISION) {
      Restrict<double>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else if (out.Precision() == QUDA_SINGLE_PRECISION) {
      Restrict<float>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map);
    } else {
      errorQuda("Unsupported precision %d", out.Precision());
    }
#else
    errorQuda("Multigrid has not been built");
#endif
  }

} // namespace quda
