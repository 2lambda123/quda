#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>
#include <uint_to_char.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/multi_reduce_core.cuh>

namespace quda {

  namespace blas {

    hipStream_t* getStream();
    hipEvent_t* getReduceEvent();
    bool getFastReduce();

    template <int writeX, int writeY, int writeZ, int writeW>
    struct write {
      static constexpr int X = writeX;
      static constexpr int Y = writeY;
      static constexpr int Z = writeZ;
      static constexpr int W = writeW;
    };

    template <typename doubleN, typename ReduceType, typename FloatN, int M, int NXZ, typename Arg>
    void multiReduceLaunch(doubleN result[], Arg &arg, const TuneParam &tp, const hipStream_t &stream, Tunable &tunable)
    {

      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

      if (getFastReduce() && !commAsyncReduction()) {
        // initialize the reduction values in 32-bit increments to INT_MIN
        constexpr int32_t words = sizeof(ReduceType) / sizeof(int32_t);
        void *h_reduce = getHostReduceBuffer();
        for (unsigned int i = 0; i < tp.grid.z * NXZ * arg.NYW * words; i++) {
          reinterpret_cast<int32_t *>(h_reduce)[i] = std::numeric_limits<int32_t>::min();
        }
      }

#ifdef WARP_MULTI_REDUCE
#error "Untested - should be reverified"
      // multiReduceKernel<ReduceType,FloatN,M,NXZ><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
#else
#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::multiReduceKernel")
                                  .instantiate((int)tp.block.x, Type<ReduceType>(), Type<FloatN>(), M, NXZ, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
#else
#ifdef CONSTANT_ARG
      hipMemcpyToSymbolAsync(HIP_SYMBOL(arg_buffer), reinterpret_cast<char *>(&arg), sizeof(arg), 0, hipMemcpyHostToDevice, stream);
      LAUNCH_KERNEL_REDUCE(multiReduceKernel, tp, stream,    , ReduceType, FloatN, M, NXZ, Arg);
#else
      LAUNCH_KERNEL_REDUCE(multiReduceKernel, tp, stream, arg, ReduceType, FloatN, M, NXZ, Arg);
#endif
#endif
#endif

      if (!commAsyncReduction()) {
#if (defined(_MSC_VER) && defined(_WIN64) || defined(__LP64__))
        if (deviceProp.canMapHostMemory) {
          if (getFastReduce()) {
            constexpr int32_t words = sizeof(ReduceType) / sizeof(int32_t);
            volatile int32_t *check = reinterpret_cast<int32_t *>(getHostReduceBuffer());
            int count = 0;
            for (unsigned int i = 0; i < tp.grid.z * NXZ * arg.NYW * words; i++) {
              // spin-wait until all values have been updated
              while (check[i] == std::numeric_limits<int32_t>::min()) {
                if (count++ % 10000 == 0) { // check error every 10000 iterations
                  // if there is an error in the kernel then we need to exit the spin-wait
                  if (hipSuccess != hipPeekAtLastError()) break;
                }
              }
            }
          } else {
            qudaEventRecord(*getReduceEvent(), stream);
            while (hipSuccess != qudaEventQuery(*getReduceEvent())) {}
          }
        } else
#endif
        {
          qudaMemcpy(getHostReduceBuffer(), getMappedHostReduceBuffer(), tp.grid.z * sizeof(ReduceType) * NXZ * arg.NYW,
              hipMemcpyDeviceToHost);
        }
      }

      // need to transpose for same order with vector thread reduction
      for (int i = 0; i < NXZ; i++) {
        for (int j = 0; j < arg.NYW; j++) {
          result[i * arg.NYW + j] = set(((ReduceType *)getHostReduceBuffer())[j * NXZ + i]);
          if (tp.grid.z == 2)
            sum(result[i * arg.NYW + j], ((ReduceType *)getHostReduceBuffer())[NXZ * arg.NYW + j * NXZ + i]);
        }
      }
    }

    namespace detail
    {
      template <unsigned... digits> struct to_chars {
        static const char value[];
      };

      template <unsigned... digits> const char to_chars<digits...>::value[] = {('0' + digits)..., 0};

      template <unsigned rem, unsigned... digits> struct explode : explode<rem / 10, rem % 10, digits...> {
      };

      template <unsigned... digits> struct explode<0, digits...> : to_chars<digits...> {
      };
    } // namespace detail

    template <unsigned num> struct num_to_string : detail::explode<num / 10, num % 10> {
    };

    template <int NXZ, typename doubleN, typename ReduceType, typename FloatN, int M, typename SpinorX,
        typename SpinorY, typename SpinorZ, typename SpinorW, typename Reducer>
    class MultiReduce : public Tunable
    {

  private:
      typedef typename scalar<FloatN>::type Float;
      typedef typename vector<Float, 2>::type Float2;
      static constexpr int NYW_max = max_YW_size<NXZ, typename SpinorX::StoreType, typename SpinorY::StoreType, Reducer>();
      const int NYW;
      int nParity;
      MultiReduceArg<NXZ, ReduceType, SpinorX, SpinorY, SpinorZ, SpinorW, Reducer> arg;
      doubleN *result;

      std::vector<ColorSpinorField *> &x, &y, &z, &w;

      // don't curry into the Spinors to minimize parameter size
      char *Y_h[NYW_max], *W_h[NYW_max], *Ynorm_h[NYW_max], *Wnorm_h[NYW_max];

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
            sharedBytesPerThread() * nthreads :
            sharedBytesPerBlock(param);
        return false;
      }

      // we only launch thread blocks up to size 512 since the autotuner
      // tuner favours smaller blocks and this helps with compile time
      unsigned int maxBlockSize(const TuneParam &param) const { return 128; }//deviceProp.maxThreadsPerBlock / 2; }

  public:
      MultiReduce(doubleN result[], SpinorX X[], SpinorY Y[], SpinorZ Z[], SpinorW W[], Reducer &r,
          std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y, std::vector<ColorSpinorField *> &z,
          std::vector<ColorSpinorField *> &w, int NYW, int length) :
          NYW(NYW),
          nParity(x[0]->SiteSubset()),
          arg(X, Y, Z, W, r, NYW, length / nParity),
          x(x),
          y(y),
          z(z),
          w(w),
          result(result),
          Y_h(),
          W_h(),
          Ynorm_h(),
          Wnorm_h()
      {
        if (sizeof(arg) > MAX_MATRIX_SIZE) errorQuda("Kernel argument size %lu greater than maximum %d", sizeof(arg), MAX_MATRIX_SIZE);

        strcpy(aux, "policy_kernel,");
        strcat(aux, x[0]->AuxString());

        // since block dot product and block norm use the same functors, we need to distinguish them
        bool is_norm = false;
        if (NXZ == NYW) {
          is_norm = true;
          for (int i = 0; i < NXZ; i++) {
            if (x[i]->V() != y[i]->V() || x[i]->V() != z[i]->V() || x[i]->V() != w[i]->V()) {
              is_norm = false;
              break;
            }
          }
        }
        if (is_norm) strcat(aux, ",norm");

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/multi_reduce_core.cuh");
#endif
      }

      inline TuneKey tuneKey() const
      {
        char name[TuneKey::name_n];
        strcpy(name, num_to_string<NXZ>::value);
        strcat(name, std::to_string(NYW).c_str());
        strcat(name, typeid(arg.r).name());
        return TuneKey(x[0]->VolString(), name, aux);
      }

      void apply(const hipStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        multiReduceLaunch<doubleN, ReduceType, FloatN, M, NXZ>(result, arg, tp, stream, *this);
      }

      // Should these be NYW?
#ifdef WARP_MULTI_REDUCE
      /**
         @brief This is a specialized variant of the reducer that only
         assigns an individial warp within a thread block to a given row
         of the reduction.  It's typically slower than CTA-wide reductions
         and spreading the y dimension across blocks rather then within
         the blocks so left disabled.
      */
      bool advanceBlockDim(TuneParam &param) const
      {
        if (param.block.y < NYW) {
          param.block.y++;
          param.grid.y = (NYW + param.block.y - 1) / param.block.y;
          return true;
        } else {
          param.block.y = 1;
          param.grid.y = NYW;
          return false;
        }
      }
#endif

      bool advanceGridDim(TuneParam &param) const
      {
        bool rtn = Tunable::advanceGridDim(param);
        if (NYW > deviceProp.maxGridSize[1]) errorQuda("N=%d is greater than the maximum support grid size", NYW);
        return rtn;
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.block.y = 1;
        param.grid.y = NYW;
        param.grid.z = nParity;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
        param.block.y = 1;
        param.grid.y = NYW;
        param.grid.z = nParity;
      }

      void preTune()
      {
        for (int i = 0; i < NYW; ++i) {
          arg.Y[i].backup(&Y_h[i], &Ynorm_h[i], y[i]->Bytes(), y[i]->NormBytes());
          arg.W[i].backup(&W_h[i], &Wnorm_h[i], w[i]->Bytes(), w[i]->NormBytes());
        }
      }

      void postTune()
      {
        for (int i = 0; i < NYW; ++i) {
          arg.Y[i].restore(&Y_h[i], &Ynorm_h[i], y[i]->Bytes(), y[i]->NormBytes());
          arg.W[i].restore(&W_h[i], &Wnorm_h[i], w[i]->Bytes(), w[i]->NormBytes());
        }
      }

      long long flops() const
      {
        return NYW * NXZ * arg.r.flops() * vec_length<FloatN>::value * (long long)arg.length * nParity * M;
      }

      long long bytes() const
      {
        // this will be wrong when mixed precision is added
        return NYW * NXZ * arg.r.streams() * x[0]->Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <typename doubleN, typename ReduceType, typename RegType, typename StoreType, typename yType, int M, int NXZ,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class Reducer, typename write, typename T>
    void multiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y, std::vector<ColorSpinorField *> &z,
        std::vector<ColorSpinorField *> &w, int length)
    {
      typedef typename scalar<RegType>::type Float;
      typedef typename vector<Float, 2>::type Float2;
      typedef vector<Float, 2> vec2;

      const int NYW = y.size();
      Reducer<NXZ, ReduceType, Float2, RegType> r(a, b, c, NYW);
      constexpr int NYW_max = max_YW_size<NXZ, StoreType, yType, decltype(r)>();

      memset(result, 0, NXZ * NYW * sizeof(doubleN));

      const int NYW_max_check = max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), 2*y[0]->Precision(),
                                            r.use_z, r.use_w, true);

      if (NYW_max != NYW_max_check) errorQuda("Runtime %d and compile time %d limits disagree", NYW_max, NYW_max_check);
      if (NXZ * NYW > QUDA_MAX_MULTI_REDUCE) errorQuda("NXZ * NYW = %d exceeds maximum number of reductions %d", NXZ * NYW, QUDA_MAX_MULTI_REDUCE);
      if (!is_valid_NXZ(NXZ)) errorQuda("NXZ=%d is not a valid size ( MAX_MULTI_BLAS_N %d)", NXZ, MAX_MULTI_BLAS_N);
      if (NYW > NYW_max) errorQuda("NYW exceeds max size (%d > %d)", NYW, NYW_max);
      if (NXZ * NYW * sizeof(Float2) > MAX_MATRIX_SIZE)
        errorQuda("Coefficient  matrix exceeds max size (%lu > %d)", NXZ * NYW * sizeof(Float2), MAX_MATRIX_SIZE);

      const int N_MIN = NXZ < NYW ? NXZ : NYW;
      for (int i = 0; i < N_MIN; i++) {
        checkSpinor(*x[i], *y[i]);
        checkSpinor(*x[i], *z[i]);
        checkSpinor(*x[i], *w[i]);
        if (!x[i]->isNative()) {
          warningQuda("Reductions on non-native fields are not supported\n");
          return;
        }
      }

#ifdef JITIFY
      // need to get constants pointer from jitify instance
      if (a.data || b.data || c.data) errorQuda("Constant memory buffer support not enabled with jitify yet");
#endif

      // FIXME - if NXZ=1 no need to copy entire array
      // FIXME - do we really need strided access here?
      if (a.data) {
        Float2 A[MAX_MATRIX_SIZE / sizeof(Float2)];
        // since the kernel doesn't know the width of them matrix at compile
        // time we stride it and copy the padded matrix to GPU
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++)
            A[NYW * i + j] = make_Float2<Float2>(Complex(a.data[NYW * i + j]));

        hipMemcpyToSymbolAsync(HIP_SYMBOL(Amatrix_d), A, MAX_MATRIX_SIZE, 0, hipMemcpyHostToDevice, *getStream());
        Amatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(a.data));
      }

      if (b.data) {
        Float2 B[MAX_MATRIX_SIZE / sizeof(Float2)];
        // since the kernel doesn't know the width of them matrix at compile
        // time we stride it and copy the padded matrix to GPU
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++)
            B[NYW * i + j] = make_Float2<Float2>(Complex(b.data[NYW * i + j]));

        hipMemcpyToSymbolAsync(HIP_SYMBOL(Bmatrix_d), B, MAX_MATRIX_SIZE, 0, hipMemcpyHostToDevice, *getStream());
        Bmatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(b.data));
      }

      if (c.data) {
        Float2 C[MAX_MATRIX_SIZE / sizeof(Float2)];
        // since the kernel doesn't know the width of them matrix at compile
        // time we stride it and copy the padded matrix to GPU
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++)
            C[NYW * i + j] = make_Float2<Float2>(Complex(c.data[NYW * i + j]));

        hipMemcpyToSymbolAsync(HIP_SYMBOL(Cmatrix_d), C, MAX_MATRIX_SIZE, 0, hipMemcpyHostToDevice, *getStream());
        Cmatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(c.data));
      }

      SpinorTexture<RegType, StoreType, M> X[NXZ];
      Spinor<RegType, yType, M, write::Y> Y[NYW_max];
      SpinorTexture<RegType, StoreType, M> Z[NXZ];
      Spinor<RegType, StoreType, M, write::W> W[NYW_max];

      for (int i = 0; i < NXZ; i++) {
        X[i].set(*dynamic_cast<cudaColorSpinorField *>(x[i]));
        Z[i].set(*dynamic_cast<cudaColorSpinorField *>(z[i]));
      }
      for (int i = 0; i < NYW; i++) {
        Y[i].set(*dynamic_cast<cudaColorSpinorField *>(y[i]));
        W[i].set(*dynamic_cast<cudaColorSpinorField *>(w[i]));
      }

      MultiReduce<NXZ, doubleN, ReduceType, RegType, M, SpinorTexture<RegType, StoreType, M>,
                  Spinor<RegType, yType, M, write::Y>, SpinorTexture<RegType, StoreType, M>,
                  Spinor<RegType, StoreType, M, write::W>, decltype(r)>
        reduce(result, X, Y, Z, W, r, x, y, z, w, NYW, length);
      reduce.apply(*blas::getStream());

      blas::bytes += reduce.bytes();
      blas::flops += reduce.flops();

      checkCudaError();
    }

    /**
       Driver for multi-reduce with up to four vectors
    */
    template <int NXZ, typename doubleN, typename ReduceType,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class Reducer, typename write,
        bool siteUnroll, typename T>
    void uniMultiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
        CompositeColorSpinorField &w)
    {
      int reduce_length = siteUnroll ? x[0]->RealLength() : x[0]->Length();

      QudaPrecision precision = checkPrecision(*x[0], *y[0], *z[0], *w[0]);

      if (precision == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
        if (x[0]->Nspin() == 4 || x[0]->Nspin() == 2) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC) || defined(GPU_MULTIGRID)
          const int M = siteUnroll ? 12 : 1; // determines how much work per thread to do
          if (x[0]->Nspin() == 2 && siteUnroll) errorQuda("siteUnroll not supported for nSpin==2");
          multiReduce<doubleN, ReduceType, double2, double2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) {
#ifdef GPU_STAGGERED_DIRAC
          const int M = siteUnroll ? 3 : 1; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, double2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d field", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif

      } else if (precision == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = siteUnroll ? 6 : 1; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, float4, float4, float4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (4 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1 || x[0]->Nspin() == 2) { // staggered
#if defined(GPU_STAGGERED_DIRAC) || defined(GPU_MULTIGRID)
          const int M = siteUnroll ? 3 : 1;
          if (x[0]->Nspin() == 2 && siteUnroll) errorQuda("siteUnroll not supported for nSpin==2");
          multiReduce<doubleN, ReduceType, float2, float2, float2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif

      } else if (precision == QUDA_HALF_PRECISION) { // half precision

#if QUDA_PRECISION & 2
        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6;
          multiReduce<doubleN, ReduceType, float4, short4, short4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) { // staggered
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3;
          multiReduce<doubleN, ReduceType, float2, short2, short2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif

      } else if (precision == QUDA_QUARTER_PRECISION) { // quarter precision

#if QUDA_PRECISION & 1
        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6;
          multiReduce<doubleN, ReduceType, float4, char4, char4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) { // staggered
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3;
          multiReduce<doubleN, ReduceType, float2, char2, char2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif
      } else {
        errorQuda("Precision %d not supported\n", precision);
      }
    }

    /**
       Driver for multi-reduce with up to five vectors
    */
    template <int NXZ, typename doubleN, typename ReduceType,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class Reducer, typename write,
        bool siteUnroll, typename T>
    void mixedMultiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
        CompositeColorSpinorField &w)
    {
      checkPrecision(*x[0], *z[0]);
      checkPrecision(*y[0], *w[0]);

      assert(siteUnroll == true);
      int reduce_length = siteUnroll ? x[0]->RealLength() : x[0]->Length();

      if (y[0]->Precision() == QUDA_DOUBLE_PRECISION && x[0]->Precision() == QUDA_SINGLE_PRECISION) {

        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 12; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, float4, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) {
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, float2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d field", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }

      } else if (y[0]->Precision() == QUDA_DOUBLE_PRECISION && x[0]->Precision() == QUDA_HALF_PRECISION) {

        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, short4, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (4 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1 || x[0]->Nspin() == 2) { // staggered
#if defined(GPU_STAGGERED_DIRAC)
          const int M = 3;
          multiReduce<doubleN, ReduceType, double2, short2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }

      } else if (y[0]->Precision() == QUDA_SINGLE_PRECISION && x[0]->Precision() == QUDA_HALF_PRECISION) {

        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6;
          multiReduce<doubleN, ReduceType, float4, short4, float4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) { // staggered
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3;
          multiReduce<doubleN, ReduceType, float2, short2, float2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }

      } else {
        errorQuda("Precision combination x=%d y=%d not supported\n", x[0]->Precision(), y[0]->Precision());
      }
    }

    template <int NXZ, typename doubleN, typename ReduceType,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal, typename writeDiagonal,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal,
        typename writeOffDiagonal, bool siteUnroll, typename T>
    void multiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
        CompositeColorSpinorField &w, int i, int j)
    {

      if (x[0]->Precision() == y[0]->Precision()) {
        if (i == j) { // we are on the diagonal so invoke the diagonal reducer
          uniMultiReduce<NXZ, doubleN, ReduceType, ReducerDiagonal, writeDiagonal, siteUnroll, T>(
              result, a, b, c, x, y, z, w);
        } else { // we are on the diagonal so invoke the off-diagonal reducer
          uniMultiReduce<NXZ, doubleN, ReduceType, ReducerOffDiagonal, writeOffDiagonal, siteUnroll, T>(
              result, a, b, c, x, y, z, w);
        }
      } else {
        if (i == j) { // we are on the diagonal so invoke the diagonal reducer
          mixedMultiReduce<NXZ, doubleN, ReduceType, ReducerDiagonal, writeDiagonal, true, T>(
              result, a, b, c, x, y, z, w);
        } else { // we are on the diagonal so invoke the off-diagonal reducer
          mixedMultiReduce<NXZ, doubleN, ReduceType, ReducerOffDiagonal, writeOffDiagonal, true, T>(
              result, a, b, c, x, y, z, w);
        }
      }
    }

    void reDotProduct(double* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
#ifndef SSTEP
    errorQuda("S-step code not built\n");
#else
    switch(x.size()){
      case 1:
        multiReduce<1, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 2:
        multiReduce<2, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 3:
        multiReduce<3, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 4:
        multiReduce<4, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 5:
        multiReduce<5, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 6:
        multiReduce<6, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 7:
        multiReduce<7, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 8:
        multiReduce<8, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      /*case 9:
        multiReduce<9,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 10:
        multiReduce<10,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 11:
        multiReduce<11,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 12:
        multiReduce<12,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 13:
        multiReduce<13,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 14:
        multiReduce<14,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 15:
        multiReduce<15,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 16:
        multiReduce<16,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;*/
      default:
        errorQuda("Unsupported vector size");
        break;
    }
#endif // SSTEP
    // do a single multi-node reduction only once we have computed all local dot products
    const int Nreduce = x.size()*y.size();
    reduceDoubleArray((double*)result, Nreduce);
  }


    // This function does the outer product of dot products... in column major.
    // There's a function below called 'cDotProduct' that flips it to row major.
    template <template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal, typename writeDiagonal,
	      template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal, typename writeOffDiagonal>
    void multiReduce_recurse(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y,
			     std::vector<ColorSpinorField*>&z, std::vector<ColorSpinorField*>&w, int i_idx, int j_idx, bool hermitian, int2 tile_size) {

      if (y.size() > tile_size.y) // if greater than max single-kernel size, split and recurse
      {
        // Do the recurse first.
        Complex* result0 = &result[0];
        Complex* result1 = &result[x.size()*(y.size()/2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());
        std::vector<ColorSpinorField*> w0(w.begin(), w.begin() + w.size()/2);
        std::vector<ColorSpinorField*> w1(w.begin() + w.size()/2, w.end());
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result0, x, y0, z, w0, i_idx, 2*j_idx+0, hermitian, tile_size);
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result1, x, y1, z, w1, i_idx, 2*j_idx+1, hermitian, tile_size);
      }
      else
      {
        double2* cdot = new double2[x.size()*y.size()];

	// if at bottom of recursion, return if on lower left
	if (x.size() <= tile_size.x && is_valid_NXZ(x.size()) && hermitian) {
	  if (j_idx < i_idx) { return; }
	}

        coeff_array<Complex> a, b, c;

        if (x.size() <= tile_size.x && is_valid_NXZ(x.size())) {
        switch(x.size()){ // COMMENT HERE FOR COMPILE TIME
        case 1:
          multiReduce<1, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
        case 2:
          multiReduce<2, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
        case 4:
          multiReduce<4, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
        case 8:
          multiReduce<8, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
        case 16:
          multiReduce<16, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
        case 32:
          multiReduce<32, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
        case 64:
          multiReduce<64, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 3
        case 3:
          multiReduce<3, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 5
        case 5:
          multiReduce<5, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 6
        case 6:
          multiReduce<6, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 7
        case 7:
          multiReduce<7, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 9
	case 9:
          multiReduce<9, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 10
        case 10:
          multiReduce<10, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 11
        case 11:
          multiReduce<11, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 12
        case 12:
          multiReduce<12, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 13
        case 13:
          multiReduce<13, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 14
        case 14:
          multiReduce<14, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 15
        case 15:
          multiReduce<15, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 17
        case 17:
          multiReduce<17, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 18
        case 18:
          multiReduce<18, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 19
	case 19:
          multiReduce<19, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 20
        case 20:
          multiReduce<20, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 21
        case 21:
          multiReduce<21, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 22
        case 22:
          multiReduce<22, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 23
        case 23:
          multiReduce<23, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 24
        case 24:
          multiReduce<24, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 25
        case 25:
          multiReduce<25, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 26
        case 26:
          multiReduce<26, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 27
        case 27:
          multiReduce<27, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 28
        case 28:
          multiReduce<28, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 29
        case 29:
          multiReduce<29, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 30
        case 30:
          multiReduce<30, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 31
        case 31:
          multiReduce<31, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#endif //31
#endif //30
#endif //29
#endif //28
#endif //27
#endif //26
#endif //25
#endif //24
#endif //23
#endif //22
#endif //21
#endif //20
#endif //19
#endif //18
#endif //17
#endif //15
#endif //14
#endif //13
#endif //12
#endif //11
#endif //10
#endif // 9
#endif // 7
#endif // 6
#endif // 5
#endif // 3
        default:
          errorQuda("x.size %lu invalid (MAX_MULTI_BLAS_N = %d)", x.size(), MAX_MULTI_BLAS_N);
	}
	} else {
          // split the problem and recurse. Splitting in x requires
          // memory reshuffling (unless y = 1).
          // Use a few temporary variables.

          Complex* tmpmajor = new Complex[x.size()*y.size()];
          Complex* result0 = &tmpmajor[0];
          Complex* result1 = &tmpmajor[(x.size()/2)*y.size()];
          std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
          std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());
          std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
          std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());

          multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result0, x0, y, z0, w, 2*i_idx+0, j_idx, hermitian, tile_size);
          multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result1, x1, y, z1, w, 2*i_idx+1, j_idx, hermitian, tile_size);

          const unsigned int xlen0 = x.size()/2;
          const unsigned int xlen1 = x.size() - xlen0;
          const unsigned int ylen = y.size();

          // Copy back into result.
          int count = 0, count0 = 0, count1 = 0;
          for (unsigned int i = 0; i < ylen; i++)
          {
            for (unsigned int j = 0; j < xlen0; j++)
              result[count++] = result0[count0++];
            for (unsigned int j = 0; j < xlen1; j++)
              result[count++] = result1[count1++];
          }

          delete[] tmpmajor;
        }

	// we are at the leaf of the binary tree (e.g., we ran the kernel): perform the row-to-column-major transpose here.
        if (x.size() <= tile_size.x && is_valid_NXZ(x.size()))
        {
          const unsigned int xlen = x.size();
          const unsigned int ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++)
              result[i*xlen+j] = Complex(cdot[j*ylen + i].x, cdot[j*ylen+i].y);
        }
        delete[] cdot;
      }
    }


    template <template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal,
	      typename writeDiagonal,
	      template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal,
	      typename writeOffDiagonal>
    class TileSizeTune : public Tunable {
      typedef std::vector<ColorSpinorField*> vec;
      Complex *result;
      vec &x, &y, &z, &w;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      int NYW_max;
      int2 max_tile_size;

    public:
      TileSizeTune(Complex *result, vec &x, vec &y, vec &z, vec &w, bool hermitian, bool Anorm = false,
                   bool nested_policy = false)
	: result(result), x(x), y(y), z(z), w(w), hermitian(hermitian), Anorm(Anorm)
      {
        NYW_max = max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), 2*y[0]->Precision(), false, false, true);
        max_tile_size = make_int2(1,1);

        strcpy(aux, nested_policy ? "nested_policy," : "policy,");
      	strcat(aux, x[0]->AuxString());
      	strcat(aux, ",");
      	strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
	strcat(aux,",n=");
	char size[8];
	u64toa(size, x.size());
	strcat(aux,size);
	strcat(aux,",m=");
	u64toa(size, y.size());
	strcat(aux,size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        // FIXME this will break if the kernels are destructive - which they aren't here
        if (!tuned()) {
          if (!nested_policy) disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          if (x.size() == 1) { // 1-d reduction
            max_tile_size = make_int2(1, std::min(NYW_max, (int)y.size()));

            // Make sure constituents are tuned.
            int2 tile_size = make_int2(1,1);
	    for ( tile_size.y=1; tile_size.y <= max_tile_size.y; tile_size.y++) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, tile_size);
	    }

          } else if (y.size() == 1) { // 1-d reduction
            max_tile_size = make_int2(std::min(max_NXZ_power2(), (int)x.size()), 1);

            // Make sure constituents are tuned.
            int2 tile_size = make_int2(1,1);
	    for ( tile_size.x=1; tile_size.x <= max_tile_size.x; tile_size.x++) {
              if (is_valid_NXZ(tile_size.x)) {
                multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
                  (result, x, y, z, w, 0, 0, hermitian, tile_size);
              }
	    }

          } else { // 2-d reduction

#if 0
            // max_tile_size should be set to the largest power of 2,
            // since we have a requirement that the tile size is a
            // power of 2.
            // FIXME - we only do simple square tiling here
            unsigned int max_count = 0;
	    unsigned int tile_size_tmp = MAX_MULTI_BLAS_N
	    while (tile_size_tmp != 1) { tile_size_tmp = tile_size_tmp >> 1; max_count++; }
	    tile_size_tmp = 1;
	    for (unsigned int i = 0; i < max_count; i++) { tile_size_tmp = tile_size_tmp << 1; }
	    max_tile_size = make_int2(tile_size_tmp, tile_size_tmp);
#endif
            max_tile_size = make_int2(max_NXZ_power2(), max_NXZ_power2());;

	    // Make sure constituents are tuned.
	    for ( unsigned int tile_size=1; tile_size <= max_tile_size.x && tile_size <= x.size() &&
		    (tile_size <= y.size() || y.size()==1) ; tile_size*=2) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, make_int2(tile_size, tile_size));
	    }

            // also test case using a single kernel if both dimensions are less than max
            if ( is_valid_NXZ(x.size()) && y.size() <= NYW_max) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, make_int2(x.size(), y.size()));
            }
          }

          if (!nested_policy) enableProfileCount();
          setPolicyTuning(true);
        }
      }

      virtual ~TileSizeTune() { setPolicyTuning(false); }

      void apply(const hipStream_t &stream) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        // tp.aux.x is where the tile size is stored. "tp" is the tuning struct.
        // it contains blocksize, grid size, etc. Since we're only tuning
        // a policy, we don't care about those sizes. That's why we only
        // tune "aux.x", which is the tile size.
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
          (result, x, y, z, w, 0, 0, hermitian, make_int2(tp.aux.x, tp.aux.y));
      }

      // aux.x is the tile size
      bool advanceAux(TuneParam &param) const
      {
	if ( x.size()==1 ) { // 1-d reduction

	  param.aux.y++;
	  if ( (unsigned int)param.aux.y <= max_tile_size.y ) {
	    return true;
	  } else {
	    param.aux.y = 1;
	    return false;
	  }

        } else if ( y.size()==1 ) { // 1-d reduction

	  do {
            param.aux.x++;
          } while (!is_valid_NXZ(param.aux.x) && param.aux.x <= max_tile_size.x);

	  if ( (unsigned int)param.aux.x <= max_tile_size.x ) {
	    return true;
	  } else {
	    param.aux.x = 1;
	    return false;
	  }

	} else { // 2-d reduction

	  if ( (unsigned int)(2*param.aux.x) <= max_tile_size.x &&
               (unsigned int)(2*param.aux.y) <= max_tile_size.y &&
               (unsigned int)(2*param.aux.x) <= x.size() &&
	       (unsigned int)(2*param.aux.y) <= y.size() ) {
            // only tune powers of two
            param.aux.x *= 2;
            param.aux.y *= 2;
	    return true;
	  } else if ( is_valid_NXZ(x.size()) && y.size() <= NYW_max &&
                     (param.aux.x != x.size() || param.aux.y != y.size()) ) {
            // we've run out of power of two tiles to try, but before
            // we finish, try a single kernel if it fits
            param.aux.x = x.size();
            param.aux.y = y.size();
            return true;
          } else {
            // reset to the beginning (which we'd need for multi-dimensional tuning)
            param.aux.x = 1;
            param.aux.y = 1;
	    return false;
	  }

	}
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const  {
        Tunable::initTuneParam(param);
        param.aux.x = 1; param.aux.y = 1; param.aux.z = 0; param.aux.w = 0;
      }

      void defaultTuneParam(TuneParam &param) const  {
        Tunable::defaultTuneParam(param); // default is max tile size
        param.aux.x = max_tile_size.x; param.aux.y = max_tile_size.y; param.aux.z = 0; param.aux.w = 0;
      }

      TuneKey tuneKey() const {
        return TuneKey(x[0]->VolString(), typeid(*this).name(), aux);
      }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() { } // FIXME - use write to determine what needs to be saved
      void postTune() { } // FIXME - use write to determine what needs to be saved
    };

    template <template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal,
	      typename writeDiagonal,
	      template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal,
	      typename writeOffDiagonal>
    class TransposeTune : public Tunable {
      using TileTuner = TileSizeTune<ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal>;
      typedef std::vector<ColorSpinorField*> vec;
      Complex *result;
      vec &x, &y, &z, &w;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

    public:
      TransposeTune(Complex *result, vec &x, vec &y, vec &z, vec &w, bool hermitian, bool Anorm = false)
	: result(result), x(x), y(y), z(z), w(w), hermitian(hermitian), Anorm(Anorm)
      {
        strcpy(aux, "policy,");
        strcat(aux, x[0]->AuxString());
        strcat(aux, ",");
        strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
	strcat(aux,",n=");
	char size[8];
	u64toa(size, x.size());
	strcat(aux,size);
	strcat(aux,",m=");
	u64toa(size, y.size());
	strcat(aux,size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        if (!tuned()) {
          disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          { // tune regular inner product
            TileTuner tile(result, x, y, z, w, hermitian, Anorm, true);
            tile.apply(0);
          }

          { // tune transpose inner product
            TileTuner tile(result, y, z, w, z, hermitian, Anorm, true);
            tile.apply(0);
          }

          enableProfileCount();
          setPolicyTuning(true);
        }
      }

      virtual ~TransposeTune() { setPolicyTuning(false); }

      void apply(const hipStream_t &stream) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (tp.aux.x == 0) {
          TileTuner tile(result, x, y, z, w, hermitian, Anorm, true);
          tile.apply(stream);
        } else if (tp.aux.x == 1){
          Complex *result_trans = new Complex[x.size()*y.size()];

          // swap (x<->y and w<-z> when doing transpose calculation)
          TileTuner tile(result_trans, y, x, w, z, hermitian, Anorm, true);
          tile.apply(stream);

          // tranpose the result if we are doing the transpose calculation
          const auto xlen = x.size();
          const auto ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++)
              result[i*xlen+j] = conj(result_trans[j*ylen+i]);

          delete []result_trans;
        } else {
          errorQuda("Unexpected transpose parameter %d", tp.aux.x);
        }
      }

      // aux.x is the tile size
      bool advanceAux(TuneParam &param) const
      {
        if (param.aux.x == 0) {
          param.aux.x = 1;
          return true;
        } else {
          param.aux.x = 0;
          return false;
        }
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const  {
        Tunable::initTuneParam(param);
        param.aux = make_int4(0, 0, 0, 0);
      }

      void defaultTuneParam(TuneParam &param) const  {
        Tunable::defaultTuneParam(param);
        param.aux = make_int4(0, 0, 0, 0); // default is not to transpose
      }

      TuneKey tuneKey() const {
        return TuneKey(x[0]->VolString(), typeid(*this).name(), aux);
      }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() { } // FIXME - use write to determine what needs to be saved
      void postTune() { } // FIXME - use write to determine what needs to be saved
    };

    void cDotProduct(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      // cDotProduct_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      TransposeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > trans(result_tmp, x, y, x, y, false);
      trans.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
    }

    void hDotProduct(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true, false); // last false is b/c L2 norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
	}

      delete[] result_tmp;
    }

    // for (p, Ap) norms in CG which are Hermitian.
    void hDotProduct_Anorm(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block A-norm dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true, true); // last true is b/c A norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
  }

      delete[] result_tmp;
    }

    // takes the outer product of inner products between and y and copies y into z
    void cDotProductCopy(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y,
			 std::vector<ColorSpinorField*>&z){

#if 0
      // FIXME - if this is enabled we need to ensure that use_w is enabled above
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (y.size() != z.size()) errorQuda("Cannot copy input y of size %lu into z of size %lu\n", y.size(), z.size());

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      // When recursing, only the diagonal tiles will do the copy, the rest just do the outer product
      TileSizeTune<CdotCopy,write<0,0,0,1>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true);
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
#else
      errorQuda("cDotProductCopy not enabled");
#endif
    }

   } // namespace blas

} // namespace quda
