#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <color_spinor_field_order.h>
#include <uint_to_char.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/multi_reduce_core.cuh>

namespace quda {

  namespace blas {

    qudaStream_t* getStream();
    hipEvent_t* getReduceEvent();
    bool getFastReduce();
    void initFastReduce(int words);
    void completeFastReduce(int32_t words);

    template <typename real, int M, int NXZ, typename Arg, typename T>
    void multiReduceLaunch(T result[], Arg &arg, const TuneParam &tp, const qudaStream_t &stream, Tunable &tunable)
    {
      using reduce_t = typename Arg::Reducer::reduce_t;
      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

      const int32_t words = tp.grid.z * NXZ * arg.NYW * sizeof(reduce_t) / sizeof(int32_t);
      if (getFastReduce() && !commAsyncReduction()) initFastReduce(words);

#ifdef WARP_MULTI_REDUCE
#error "Untested - should be reverified"
      // multiReduceKernel<FloatN,M,NXZ><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
#else
#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::multiReduceKernel")
                                  .instantiate((int)tp.block.x, Type<real>(), M, NXZ, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
#else
      LAUNCH_KERNEL_REDUCE(multiReduceKernel, tunable, tp, stream, arg, real, M, NXZ, Arg);
#endif
#endif

      if (!commAsyncReduction()) {
#if (defined(_MSC_VER) && defined(_WIN64) || defined(__LP64__))
        if (deviceProp.canMapHostMemory) {
          if (getFastReduce()) {
            completeFastReduce(words);
          } else {
            qudaEventRecord(*getReduceEvent(), stream);
            while (hipSuccess != qudaEventQuery(*getReduceEvent())) {}
          }
        } else
#endif
        {
          qudaMemcpy(getHostReduceBuffer(), getMappedHostReduceBuffer(), tp.grid.z * sizeof(reduce_t) * NXZ * arg.NYW,
                     hipMemcpyDeviceToHost);
        }
      }

      // need to transpose for same order with vector thread reduction
      auto buffer = (reduce_t *)getHostReduceBuffer();
      for (int i = 0; i < NXZ; i++) {
        for (int j = 0; j < arg.NYW; j++) {
          result[i * arg.NYW + j] = static_cast<T>(buffer[j * NXZ + i]);
          if (tp.grid.z == 2) result[i * arg.NYW + j] += static_cast<T>(buffer[NXZ * arg.NYW + j * NXZ + i]);
        }
      }
    }

    template <template <typename ...> class Reducer, typename store_t, typename y_store_t,
              int nSpin, typename T, int NXZ = 1>
    class MultiReduce : public Tunable
    {
      using real = typename mapper<y_store_t>::type;
      using host_reduce_t = typename Reducer<double, real>::reduce_t;
      const int NYW;
      Reducer<device_reduce_t, real> r;
      static constexpr int NYW_max = max_YW_size<NXZ, store_t, y_store_t, decltype(r)>();
      const int nParity;
      const T &a, &b, &c;
      std::vector<ColorSpinorField *> &x, &y, &z, &w;
      host_reduce_t *result;
      QudaFieldLocation location;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
          sharedBytesPerThread() * nthreads : sharedBytesPerBlock(param);
        return false;
      }

      // we only launch thread blocks up to size 512 since the autotuner
      // tuner favours smaller blocks and this helps with compile time
      unsigned int maxBlockSize(const TuneParam &param) const { return 128; } // deviceProp.maxThreadsPerBlock / 2; }

    public:
      MultiReduce(const T &a, const T &b, const T &c, const ColorSpinorField &x_meta, const ColorSpinorField &y_meta,
                  std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                  std::vector<ColorSpinorField *> &z, std::vector<ColorSpinorField *> &w,
                  host_reduce_t *result) :
        NYW(y.size()),
        r(NXZ, NYW),
        nParity(x[0]->SiteSubset()),
        a(a),
        b(b),
        c(c),
        x(x),
        y(y),
        z(z),
        w(w),
        result(result),
        location(checkLocation(*x[0], *y[0], *z[0], *w[0]))
      {
        checkLength(*x[0], *y[0], *z[0], *w[0]);
        auto x_prec = checkPrecision(*x[0], *z[0]);
        auto y_prec = checkPrecision(*y[0], *w[0]);
        auto x_order = checkOrder(*x[0], *z[0]);
        auto y_order = checkOrder(*y[0], *w[0]);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        // check sizes are valid
        constexpr int NYW_max = max_YW_size<NXZ, store_t, y_store_t, decltype(r)>();
        constexpr int scalar_width = decltype(r)::coeff_mul ? sizeof(typename decltype(r)::coeff_t) / sizeof(real) : 0;
        const int NYW_max_check = max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), r.use_z, r.use_w, scalar_width, true);

        if (!is_valid_NXZ(NXZ, true))
          errorQuda("NXZ=%d is not a valid size ( MAX_MULTI_BLAS_N %d)", NXZ, MAX_MULTI_BLAS_N);
        if (NYW_max != NYW_max_check) errorQuda("Compile-time %d and run-time %d limits disagree", NYW_max, NYW_max_check);
        if (NXZ * NYW > QUDA_MAX_MULTI_REDUCE)
          errorQuda("NXZ * NYW = %d exceeds maximum number of reductions %d * %d > %d",
                    NXZ * NYW, NXZ, NYW, QUDA_MAX_MULTI_REDUCE);
        if (NYW > NYW_max) errorQuda("NYW exceeds max size (%d > %d)", NYW, NYW_max);
        if (NXZ * NYW * scalar_width > MAX_MATRIX_SIZE)
          errorQuda("Coefficient matrix exceeds max size (%d > %d)", NXZ * NYW * scalar_width, MAX_MATRIX_SIZE);

        strcpy(aux, "policy_kernel,");
        strcat(aux, x[0]->AuxString());
        if (getFastReduce()) strcat(aux, ",fast_reduce");

        // since block dot product and block norm use the same functors, we need to distinguish them
        bool is_norm = false;
        if (NXZ == NYW) {
          is_norm = true;
          for (int i = 0; i < NXZ; i++) {
            if (x[i]->V() != y[i]->V() || x[i]->V() != z[i]->V() || x[i]->V() != w[i]->V()) {
              is_norm = false;
              break;
            }
          }
        }
        if (is_norm) strcat(aux, ",norm");

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/multi_reduce_core.cuh");
#endif

        apply(*blas::getStream());
        checkCudaError();

        blas::bytes += bytes();
        blas::flops += flops();
      }

      TuneKey tuneKey() const
      {
        char name[TuneKey::name_n];
        strcpy(name, num_to_string<NXZ>::value);
        strcat(name, std::to_string(NYW).c_str());
        strcat(name, typeid(r).name());
        return TuneKey(x[0]->VolString(), name, aux);
      }

      template <typename buffer_t>
      void set_param(buffer_t &d, const T &h, const qudaStream_t &stream)
      {
        using coeff_t = typename decltype(r)::coeff_t;
        constexpr size_t n_coeff = MAX_MATRIX_SIZE / sizeof(coeff_t);

        coeff_t tmp[n_coeff];
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++) tmp[NYW * i + j] = coeff_t(h.data[NYW * i + j]);
        hipMemcpyToSymbolAsync(HIP_SYMBOL(d), tmp, NXZ * NYW * sizeof(decltype(tmp[0])), 0, hipMemcpyHostToDevice, stream);
        //hipMemcpyHtoDAsync(d, tmp, NXZ * NYW * sizeof(decltype(tmp[0])), stream);
      }

      void apply(const qudaStream_t &stream)
      {
        constexpr bool site_unroll = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value;
        if (site_unroll && (x[0]->Ncolor() != 3 || x[0]->Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x[0]->Nspin(), x[0]->Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (location == QUDA_CUDA_FIELD_LOCATION) {
          // need to add native check here
          constexpr int N = n_vector<store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x[0]->Length() / (nParity * M);

          MultiReduceArg<NXZ, store_t, N, y_store_t, Ny, decltype(r)> arg(x, y, z, w, r, NYW, length);

#ifdef JITIFY
          // need to get constants pointer from jitify instance
          if (a.data || b.data || c.data) errorQuda("Constant memory buffer support not enabled with jitify yet");
#else
          if (a.data) set_param(Amatrix_d, a, stream);
          if (b.data) set_param(Bmatrix_d, b, stream);
          if (c.data) set_param(Cmatrix_d, c, stream);
#endif
          multiReduceLaunch<real, M, NXZ>(result, arg, tp, stream, *this);
        } else {
          errorQuda("Only implemented for GPU fields");
        }
      }

      // Should these be NYW?
#ifdef WARP_MULTI_REDUCE
      /**
         @brief This is a specialized variant of the reducer that only
         assigns an individial warp within a thread block to a given row
         of the reduction.  It's typically slower than CTA-wide reductions
         and spreading the y dimension across blocks rather then within
         the blocks so left disabled.
      */
      bool advanceBlockDim(TuneParam &param) const
      {
        if (param.block.y < NYW) {
          param.block.y++;
          param.grid.y = (NYW + param.block.y - 1) / param.block.y;
          return true;
        } else {
          param.block.y = 1;
          param.grid.y = NYW;
          return false;
        }
      }
#endif

      bool advanceGridDim(TuneParam &param) const
      {
        bool rtn = Tunable::advanceGridDim(param);
        if (NYW > deviceProp.maxGridSize[1]) errorQuda("N=%d is greater than the maximum support grid size", NYW);
        return rtn;
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.block.y = 1;
        param.grid.y = NYW;
        param.grid.z = nParity;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
        param.block.y = 1;
        param.grid.y = NYW;
        param.grid.z = nParity;
      }

      void preTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (r.write.Y) y[i]->backup();
          if (r.write.W) w[i]->backup();
        }
      }

      void postTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (r.write.Y) y[i]->restore();
          if (r.write.W) w[i]->restore();
        }
      }

      long long flops() const
      {
        return NYW * NXZ * r.flops() * x[0]->Length();
      }

      long long bytes() const
      {
        // this will be wrong when mixed precision is added
        return NYW * NXZ * r.streams() * x[0]->Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <int NXZ, template <typename ...> class ReducerDiagonal, template <typename ...> class ReducerOffDiagonal, typename T>
    void multiReduce(T result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                     CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
                     CompositeColorSpinorField &w, int i, int j)
    {
      if (i == j) { // we are on the diagonal so invoke the diagonal reducer
        using host_reduce_t = typename ReducerDiagonal<double, double>::reduce_t;
        instantiate<ReducerDiagonal, MultiReduce, true, NXZ>(a, b, c, *x[0], *y[0], x, y, z, w, (host_reduce_t*)result);
      } else { // we are on the diagonal so invoke the off-diagonal reducer
        using host_reduce_t = typename ReducerOffDiagonal<double, double>::reduce_t;
        instantiate<ReducerOffDiagonal, MultiReduce, true, NXZ>(a, b, c, *x[0], *y[0], x, y, z, w, (host_reduce_t*)result);
      }
    }

    template <template <typename ...> class ReducerDiagonal, template <typename ...> class ReducerOffDiagonal,
              int n, typename T>
    typename std::enable_if<n!=1, void>::type
    multiReduce(T result [], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
                CompositeColorSpinorField &w, int i, int j)
    {
      if (x.size() == n) multiReduce<n, ReducerDiagonal, ReducerOffDiagonal>(result, a, b, c, x, y, z, w, i, j);
      else multiReduce<ReducerDiagonal, ReducerOffDiagonal, n-1>(result, a, b, c, x, y, z, w, i, j);
    }

    template <template <typename ...> class ReducerDiagonal, template <typename ...> class ReducerOffDiagonal,
              int n, typename T>
      typename std::enable_if<n==1, void>::type
    multiReduce(T result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
                CompositeColorSpinorField &w, int i, int j)
    {
      multiReduce<n, ReducerDiagonal, ReducerOffDiagonal>(result, a, b, c, x, y, z, w, i, j);
    }

    // This function does the outer product of dot products... in column major.
    // There's a function below called 'cDotProduct' that flips it to row major.
    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T>
    void multiReduce_recurse(T *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                             std::vector<ColorSpinorField *> &z, std::vector<ColorSpinorField *> &w, int i_idx,
                             int j_idx, bool hermitian, uint2 tile_size)
    {
      if (y.size() > tile_size.y) { // if greater than max single-kernel size, split and recurse
        // Do the recurse first.
        T* result0 = &result[0];
        T* result1 = &result[x.size()*(y.size()/2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());
        std::vector<ColorSpinorField*> w0(w.begin(), w.begin() + w.size()/2);
        std::vector<ColorSpinorField*> w1(w.begin() + w.size()/2, w.end());
        multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result0, x, y0, z, w0, i_idx, 2*j_idx+0, hermitian, tile_size);
        multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result1, x, y1, z, w1, i_idx, 2*j_idx+1, hermitian, tile_size);
      } else {
        T* tmp_dot = new T[x.size()*y.size()];

	// if at bottom of recursion, return if on lower left
        if (x.size() <= tile_size.x && is_valid_NXZ(x.size(), true) && hermitian) {
          if (j_idx < i_idx) { return; }
        }

        coeff_array<T> a, b, c;

        if (x.size() <= tile_size.x && is_valid_NXZ(x.size(), true)) {
          switch (x.size()) {
            // by default all powers of two <= 16 are instantiated
          case 1:
            multiReduce<1, ReducerDiagonal, ReducerOffDiagonal>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);
            break;
          case 2:
            multiReduce<2, ReducerDiagonal, ReducerOffDiagonal>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);
            break;
          case 4:
            multiReduce<4, ReducerDiagonal, ReducerOffDiagonal>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);
            break;
          case 8:
            multiReduce<8, ReducerDiagonal, ReducerOffDiagonal>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);
            break;
          case 16:
            multiReduce<16, ReducerDiagonal, ReducerOffDiagonal>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);
            break;
          default:
            if (x.size() <= MAX_MULTI_BLAS_N)
              multiReduce<ReducerDiagonal, ReducerOffDiagonal, MAX_MULTI_BLAS_N>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);
            else errorQuda("x.size %lu invalid (MAX_MULTI_BLAS_N = %d)", x.size(), MAX_MULTI_BLAS_N);
          }
        } else {
          // split the problem and recurse. Splitting in x requires
          // memory reshuffling (unless y = 1).
          // Use a few temporary variables.

          T* tmpmajor = new T[x.size()*y.size()];
          T* result0 = &tmpmajor[0];
          T* result1 = &tmpmajor[(x.size()/2)*y.size()];
          std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
          std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());
          std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
          std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());

          multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result0, x0, y, z0, w, 2*i_idx+0, j_idx, hermitian, tile_size);
          multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result1, x1, y, z1, w, 2*i_idx+1, j_idx, hermitian, tile_size);

          const unsigned int xlen0 = x.size()/2;
          const unsigned int xlen1 = x.size() - xlen0;
          const unsigned int ylen = y.size();

          // Copy back into result.
          int count = 0, count0 = 0, count1 = 0;
          for (unsigned int i = 0; i < ylen; i++)
          {
            for (unsigned int j = 0; j < xlen0; j++)
              result[count++] = result0[count0++];
            for (unsigned int j = 0; j < xlen1; j++)
              result[count++] = result1[count1++];
          }

          delete[] tmpmajor;
        }

        // we are at the leaf of the binary tree (e.g., we ran the kernel): perform the row-to-column-major transpose here.
        if (x.size() <= tile_size.x && is_valid_NXZ(x.size(), true)) {
          const unsigned int xlen = x.size();
          const unsigned int ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++)
              result[i*xlen+j] = tmp_dot[j*ylen + i];
        }
        delete[] tmp_dot;
      }
    }

    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T>
    class TileSizeTune : public Tunable
    {
      typedef std::vector<ColorSpinorField*> vec;
      T *result;
      vec &x, &y, &z, &w;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      int NYW_max;
      uint2 max_tile_size;

    public:
      TileSizeTune(T *result, vec &x, vec &y, vec &z, vec &w, int coeff_width, bool hermitian, bool Anorm = false,
                   bool nested_policy = false) :
        result(result),
        x(x),
        y(y),
        z(z),
        w(w),
        hermitian(hermitian),
        Anorm(Anorm)
      {
        NYW_max = max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), false, false, coeff_width, true);
        max_tile_size = make_uint2(1, 1);

        strcpy(aux, nested_policy ? "nested_policy," : "policy,");
        strcat(aux, x[0]->AuxString());
      	strcat(aux, ",");
      	strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
	strcat(aux,",n=");
	char size[8];
	u64toa(size, x.size());
	strcat(aux,size);
	strcat(aux,",m=");
	u64toa(size, y.size());
	strcat(aux,size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        // FIXME this will break if the kernels are destructive - which they aren't here
        if (!tuned()) {
          if (!nested_policy) disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          // note the 1-d tuning is all redundent now that we call
          // multiReduce_recurse directly now for 1-d multi
          // reductions, but I'll keep this code here for now
          if (x.size() == 1) { // 1-d reduction

            max_tile_size = make_uint2(1, std::min(NYW_max, (int)y.size()));
            multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, max_tile_size);

          } else if (y.size() == 1) { // 1-d reduction

            max_tile_size = make_uint2(std::min((size_t)max_NXZ_power2(true), x.size()), 1);
            multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, max_tile_size);

          } else { // 2-d reduction

            // max_tile_size should be set to the largest power of 2,
            // since we have a requirement that the tile size is a
            // power of 2.
            // FIXME - we only do simple square tiling here
            max_tile_size = make_uint2(max_NXZ_power2(true), max_NXZ_power2(true));

            // Make sure constituents are tuned.
            for (unsigned int tile_size = 1;
                 tile_size <= max_tile_size.x && tile_size <= x.size() && (tile_size <= y.size() || y.size() == 1);
                 tile_size *= 2) {
              multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(tile_size, tile_size));
            }

            // also test case using a single kernel if both dimensions are less than max
            if (is_valid_NXZ(x.size(), true) && y.size() <= (unsigned int)NYW_max) {
              multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(x.size(), y.size()));
            }
          }

          if (!nested_policy) enableProfileCount();
          setPolicyTuning(true);
        }
      }

      virtual ~TileSizeTune() { setPolicyTuning(false); }

      void apply(const qudaStream_t &stream) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        // tp.aux.x is where the tile size is stored. "tp" is the tuning struct.
        // it contains blocksize, grid size, etc. Since we're only tuning
        // a policy, we don't care about those sizes. That's why we only
        // tune "aux.x", which is the tile size.
        multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(tp.aux.x, tp.aux.y));
      }

      // aux.x is the tile size
      bool advanceAux(TuneParam &param) const
      {
        // for 1-d reductions we don't do any tuning and just use the largest tile
        if (x.size() == 1 || y.size() == 1) {
          return false;
        } else { // 2-d reduction

          if ((unsigned int)(2 * param.aux.x) <= max_tile_size.x && (unsigned int)(2 * param.aux.y) <= max_tile_size.y
              && (unsigned int)(2 * param.aux.x) <= x.size() && (unsigned int)(2 * param.aux.y) <= y.size()) {
            // only tune powers of two
            param.aux.x *= 2;
            param.aux.y *= 2;
            return true;
          } else if (is_valid_NXZ(x.size(), true) && y.size() <= (size_t)NYW_max
                     && ((size_t)param.aux.x != x.size() || (size_t)param.aux.y != y.size())) {
            // we've run out of power of two tiles to try, but before
            // we finish, try a single kernel if it fits
            param.aux.x = x.size();
            param.aux.y = y.size();
            return true;
          } else {
            // reset to the beginning (which we'd need for multi-dimensional tuning)
            param.aux.x = 1;
            param.aux.y = 1;
            return false;
          }
        }
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const  {
        Tunable::initTuneParam(param);
        if (x.size() == 1 || y.size() == 1) {
          param.aux.x = max_tile_size.x;
          param.aux.y = max_tile_size.y;
        } else { // only do non-trivial tuning for 2-d reductions
          param.aux.x = 1;
          param.aux.y = 1;
        }
        param.aux.z = 0;
        param.aux.w = 0;
      }

      void defaultTuneParam(TuneParam &param) const  {
        Tunable::defaultTuneParam(param); // default is max tile size
        param.aux.x = max_tile_size.x;
        param.aux.y = max_tile_size.y;
        param.aux.z = 0;
        param.aux.w = 0;
      }

      TuneKey tuneKey() const {
        return TuneKey(x[0]->VolString(), typeid(*this).name(), aux);
      }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() { } // FIXME - use write to determine what needs to be saved
      void postTune() { } // FIXME - use write to determine what needs to be saved
    };

    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T>
    class TransposeTune : public Tunable
    {
      using TileTuner = TileSizeTune<ReducerDiagonal, ReducerOffDiagonal, T>;
      using vec = std::vector<ColorSpinorField *>;
      T *result;
      vec &x, &y, &z, &w;
      int coeff_width;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

    public:
      TransposeTune(T *result, vec &x, vec &y, vec &z, vec &w, int coeff_width, bool hermitian, bool Anorm = false) :
        result(result),
        x(x),
        y(y),
        z(z),
        w(w),
        coeff_width(coeff_width),
        hermitian(hermitian),
        Anorm(Anorm)
      {
        strcpy(aux, "policy,");
        strcat(aux, x[0]->AuxString());
        strcat(aux, ",");
        strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
        strcat(aux, ",n=");
        char size[8];
        u64toa(size, x.size());
        strcat(aux, size);
        strcat(aux, ",m=");
        u64toa(size, y.size());
        strcat(aux, size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        if (!tuned()) {
          disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          // note the 1-d tuning is all redundent now that we call
          // multiReduce_recurse directly now for 1-d multi
          // reductions, but I'll keep this code here for now
          if (x.size() == 1) {
            TileTuner tile(result, x, y, z, w, coeff_width, hermitian, Anorm, true);
            tile.apply(0);
          } else if (y.size() == 1) {
            TileTuner tile(result, y, z, w, z, coeff_width, hermitian, Anorm, true);
            tile.apply(0);
          } else {

            { // tune regular inner product
              TileTuner tile(result, x, y, z, w, coeff_width, hermitian, Anorm, true);
              tile.apply(0);
            }

            { // tune transpose inner product
              TileTuner tile(result, y, z, w, z, coeff_width, hermitian, Anorm, true);
              tile.apply(0);
            }
          }

          enableProfileCount();
          setPolicyTuning(true);
        }
      }

      virtual ~TransposeTune() { setPolicyTuning(false); }

      void apply(const qudaStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (tp.aux.x == 0) {
          TileTuner tile(result, x, y, z, w, coeff_width, hermitian, Anorm, true);
          tile.apply(stream);
        } else if (tp.aux.x == 1) {
          T *result_trans = new T[x.size() * y.size()];

          // swap (x<->y and w<-z> when doing transpose calculation)
          TileTuner tile(result_trans, y, x, w, z, coeff_width, hermitian, Anorm, true);
          tile.apply(stream);

          // tranpose the result if we are doing the transpose calculation
          const auto xlen = x.size();
          const auto ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++) result[i * xlen + j] = conj(result_trans[j * ylen + i]);

          delete[] result_trans;
        } else {
          errorQuda("Unexpected transpose parameter %d", tp.aux.x);
        }
      }

      bool advanceAux(TuneParam &param) const
      {
        if (x.size() == 1 || y.size() == 1) {
          return false;
        } else {
          if (param.aux.x == 0) {
            param.aux.x = 1;
            return true;
          } else {
            param.aux.x = 0;
            return false;
          }
        }
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        if (x.size() == 1)
          param.aux = make_int4(0, 0, 0, 0);
        else if (y.size() == 1)
          param.aux = make_int4(1, 0, 0, 0);
        else
          param.aux = make_int4(0, 0, 0, 0); // default is not to transpose
      }

      void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

      TuneKey tuneKey() const { return TuneKey(x[0]->VolString(), typeid(*this).name(), aux); }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() {}  // FIXME - use write to determine what needs to be saved
      void postTune() {} // FIXME - use write to determine what needs to be saved
    };

    void reDotProduct(double *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      double *result_tmp = new double[x.size() * y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;
      int coeff_width = 0;

      if (x.size() == 1) {
        int NYW_max = max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), false, false, coeff_width, true);
        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NYW_max, (int)y.size(), x[0]->Ncolor() == 3 ? 32 : NYW_max} ));
        multiReduce_recurse<Dot, Dot>(result_tmp, x, y, x, y, 0, 0, false, max_tile_size);
      } else if (y.size() == 1) {

        double *result_trans = new double[x.size() * y.size()];

        // swap (x<->y and w<-z> when doing transpose calculation)
        int NXZ_max = max_YW_size(y.size(), y[0]->Precision(), x[0]->Precision(), false, false, coeff_width, true);
        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NXZ_max, (int)x.size(), x[0]->Ncolor() == 3 ? 32 : NXZ_max} ));
        multiReduce_recurse<Dot, Dot>(result_trans, y, x, y, x, 0, 0, false, max_tile_size);

        // transpose the result if we are doing the transpose calculation
        const auto xlen = x.size();
        const auto ylen = y.size();
        for (unsigned int j = 0; j < xlen; j++)
          for (unsigned int i = 0; i < ylen; i++) result_tmp[i * xlen + j] = result_trans[j * ylen + i];

        delete[] result_trans;

      } else {
        TransposeTune<Dot, Dot, double> tile(result_tmp, x, y, x, y, coeff_width, false);
        tile.apply(0);
      }

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = x.size() * y.size();
      reduceDoubleArray(result_tmp, Nreduce);

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++) result[j * ylen + i] = result_tmp[i * xlen + j];

      delete[] result_tmp;
    }

    void cDotProduct(Complex *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      Complex *result_tmp = new Complex[x.size() * y.size()];
      for (unsigned int i = 0; i < x.size() * y.size(); i++) result_tmp[i] = 0.0;
      int coeff_width = 0;

      if (x.size() == 1) {
        int NYW_max = max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), false, false, coeff_width, true);
        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NYW_max, (int)y.size(), x[0]->Ncolor() == 3 ? 32 : NYW_max} ));
        multiReduce_recurse<Cdot, Cdot>(result_tmp, x, y, x, y, 0, 0, false, max_tile_size);
      } else if (y.size() == 1) {

        Complex *result_trans = new Complex[x.size() * y.size()];

        // swap (x<->y and w<-z> when doing transpose calculation)
        int NXZ_max = max_YW_size(y.size(), y[0]->Precision(), x[0]->Precision(), false, false, coeff_width, true);
        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NXZ_max, (int)x.size(), x[0]->Ncolor() == 3 ? 32 : NXZ_max} ));
        multiReduce_recurse<Cdot, Cdot>(result_trans, y, x, y, x, 0, 0, false, max_tile_size);

        // transpose the result if we are doing the transpose calculation
        const auto xlen = x.size();
        const auto ylen = y.size();
        for (unsigned int j = 0; j < xlen; j++)
          for (unsigned int i = 0; i < ylen; i++) result_tmp[i * xlen + j] = conj(result_trans[j * ylen + i]);

        delete[] result_trans;

      } else {
        TransposeTune<Cdot, Cdot, Complex> trans(result_tmp, x, y, x, y, coeff_width, false);
        trans.apply(0);
      }

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
    }

    void hDotProduct(Complex *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      int coeff_width = 0;
      TileSizeTune<Cdot, Cdot, Complex> tile(result_tmp, x, y, x, y, coeff_width, true, false); // last false is b/c L2 norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
	}

      delete[] result_tmp;
    }

    // for (p, Ap) norms in CG which are Hermitian.
    void hDotProduct_Anorm(Complex *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block A-norm dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      int coeff_width = 0;
      TileSizeTune<Cdot, Cdot, Complex> tile(result_tmp, x, y, x, y, coeff_width, true, true); // last true is b/c A norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
        }

      delete[] result_tmp;
    }

    // takes the outer product of inner products between and y and copies y into z
    void cDotProductCopy(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y,
			 std::vector<ColorSpinorField*>&z){

#if 0
      // FIXME - if this is enabled we need to ensure that use_w is
      // enabled above.  Also, I think this might break if the diagonal
      // write is different from the off-diagonal write
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (y.size() != z.size()) errorQuda("Cannot copy input y of size %lu into z of size %lu\n", y.size(), z.size());

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      int coeff_width = 0;
      // When recursing, only the diagonal tiles will do the copy, the rest just do the outer product
      TileSizeTune<double2, typename vector<device_reduce_t,2>::type,CdotCopy,Cdot,Complex> tile(result_tmp, x, y, x, y, coeff_width, true);
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
#else
      errorQuda("cDotProductCopy not enabled");
#endif
    }

   } // namespace blas

} // namespace quda
