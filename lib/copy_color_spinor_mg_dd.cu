#define BUILD_LIMITED_COPY
#include <copy_color_spinor.cuh>

namespace quda {
  
  void copyGenericColorSpinorMGDD(ColorSpinorField &dst, const ColorSpinorField &src, 
				  QudaFieldLocation location, void *Dst, void *Src, 
				  void *dstNorm, void *srcNorm) {

    double *dst_ptr = static_cast<double*>(Dst);
    double *src_ptr = static_cast<double*>(Src);

    INSTANTIATE_COLOR;

  }

} // namespace quda
