#include "hip/hip_runtime.h"
#include <transfer.h>
#include <quda_internal.h>
#include <quda_matrix.h>
#include <index_helper.cuh>
#include <color_spinor.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <mpi.h>
#include <interface_qlua_internal.h>
#include <qlua_contract_kernels.cuh>

namespace quda {
  
  struct QluaContractArg {

    typedef typename colorspinor_mapper<QC_REAL,QC_Ns,QC_Nc>::type Propagator;

    Propagator prop[QUDA_PROP_NVEC];  // Input propagator 1

    const int parity;                 // only use this for single parity fields
    const int nParity;                // number of parities we're working on
    const int nFace;                  // hard code to 1 for now
    const int dim[5];                 // full lattice dimensions
    const int commDim[4];             // whether a given dimension is partitioned or not
    const int lL[4];      	      // 4-d local lattice dimensions
    const int volumeCB;               // checkerboarded volume
    const int volume;                 // full-site local volume

  QluaContractArg(ColorSpinorField **propIn, int parity)
  :   parity(parity), nParity(propIn[0]->SiteSubset()), nFace(1),
      dim{ (3-nParity) * propIn[0]->X(0), propIn[0]->X(1), propIn[0]->X(2), propIn[0]->X(3), 1 },
      commDim{comm_dim_partitioned(0), comm_dim_partitioned(1), comm_dim_partitioned(2), comm_dim_partitioned(3)},
      lL{propIn[0]->X(0), propIn[0]->X(1), propIn[0]->X(2), propIn[0]->X(3)},
      volumeCB(propIn[0]->VolumeCB()),volume(propIn[0]->Volume())
    {
      
      for(int ivec=0;ivec<QUDA_PROP_NVEC;ivec++){
        prop[ivec].init(*propIn[ivec]);
      }      
      
    }//-- constructor    
  };//-- Structure definition
  //---------------------------------------------------------------------------
  
  /**
     When copying ColorSpinorFields to GPU, Quda rotates the fields to another basis using a rotation matrix.
     This function is required in order to rotate the ColorSpinorFields between the Quda and the QDP bases.
     The rotation matrix is ( with a factor sqrt(0.5) ):
              ( 0 -1  0 -1)
          M = ( 1  0  1  0)
              ( 0 -1  0  1)
              ( 1  0 -1  0)

     Before the calculation the ColorSpinorFields must be rotated as F <- M F  (quda2qdp).
     After the calculation the result must be rotated back to the Quda basis R <- M^T R (qdp2quda),
     so that when Quda copies back to the CPU the result is again rotated to the QDP convention.
   */
  __device__ __host__ inline void rotatePropBasis(ColorSpinor<QC_REAL,QC_Nc,QC_Ns> *prop, RotateType rType){

    const int Ns = QC_Ns;
    const int Nc = QC_Nc;

    typedef ColorSpinor<QC_REAL,Nc,Ns> Vector;
    Vector res[QUDA_PROP_NVEC];

    complex<QC_REAL> zro = complex<QC_REAL>{0,0};
    complex<QC_REAL> val = complex<QC_REAL>{sqrt(0.5),0};
    complex<QC_REAL> M[Ns][Ns] = { { zro, -val,  zro, -val},
				   { val,  zro,  val,  zro},
				   { zro, -val,  zro,  val},
				   { val,  zro, -val,  zro} };

    complex<QC_REAL> M_Trans[Ns][Ns];
    for(int i=0;i<Ns;i++){
      for(int j=0;j<Ns;j++){
        M_Trans[i][j] = M[j][i];
      }
    }

    complex<QC_REAL> (*A)[Ns] = NULL;
    if      (rType == QLUA_quda2qdp) A = M;
    else if (rType == QLUA_qdp2quda) A = M_Trans;

    for(int ic = 0; ic < Nc; ic++){
      for(int jc = 0; jc < Nc; jc++){
        for(int is = 0; is < Ns; is++){
          for(int js = 0; js < Ns; js++){
            int iv = js + Ns*jc;
            int id = ic + Nc*is;

            res[iv].data[id] = 0.0;
            for(int a=0;a<Ns;a++){
              int as = ic + Nc*a;

              res[iv].data[id] += A[is][a] * prop[iv].data[as];
            }
          }}}
    }

    for(int v = 0; v<QUDA_PROP_NVEC; v++)
      prop[v] = res[v];

  }
  //---------------------------------------------------------------------------

  
  __global__ void QluaCoordCheck_kernel(QluaUtilArg *utilArg){

    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;    
    int pty  = blockIdx.y*blockDim.y + threadIdx.y;

    if (x_cb >= utilArg->volumeCB) return;
    if (pty >= utilArg->nParity) return;

    int crd[5];
    getCoords(crd, x_cb, utilArg->lL, pty);  //-- Get local coordinates crd[] at given x_cb and pty
    crd[4] = 0;  

    int idx_cb = linkIndex(crd, utilArg->lL); //-- Checkerboard index, MUST be equal to x_cb
    
    int i_rlex = crd[0] + utilArg->lL[0]*(crd[1] + utilArg->lL[1]*(crd[2] + utilArg->lL[2]*(crd[3])));  //-- Full lattice site index
    int i_par = (crd[0] + crd[1] + crd[2] + crd[3]) & 1;

    if( (i_rlex/2 != x_cb) || (pty != i_par) || (idx_cb != x_cb) ){
      d_crdChkVal = -1;
      printf("coordCheck - ERROR: x_cb = %d, pty = %d: Site order mismatch!\n", x_cb, pty);    
    }
    else d_crdChkVal = 0;

  }//-- function

  
  int QluaCoordCheck(QluaUtilArg utilArg){
    int crdChkVal;

    QluaUtilArg *utilArg_dev;
    hipMalloc((void**)&(utilArg_dev), sizeof(QluaUtilArg));
    checkCudaErrorNoSync();
    hipMemcpy(utilArg_dev, &utilArg,  sizeof(QluaUtilArg), hipMemcpyHostToDevice);
    
    dim3 blockDim(THREADS_PER_BLOCK, utilArg.nParity, 1);
    dim3 gridDim((utilArg.volumeCB + blockDim.x -1)/blockDim.x, 1, 1);
    
    QluaCoordCheck_kernel<<<gridDim,blockDim>>>(utilArg_dev);
    checkCudaError();
    hipMemcpyFromSymbol(&crdChkVal, HIP_SYMBOL(d_crdChkVal), sizeof(crdChkVal), 0, hipMemcpyDeviceToHost);
    checkCudaErrorNoSync();
    
    hipFree(utilArg_dev);
    
    return crdChkVal;
  }//-- function
  //---------------------------------------------------------------------------

  
  __global__ void conv_siteOrder_QudaQdp_to_momproj_kernel(void *dst, const void *src, QluaUtilArg *arg){

    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int pty  = blockIdx.y*blockDim.y + threadIdx.y;

    int crd[5];
    getCoords(crd, x_cb, arg->lL, pty);
    int i_t = crd[arg->t_axis];
    int i_sp = 0;

    for (int i = 0 ; i < 4 ; i++)
      i_sp += arg->sp_stride[i] * crd[i];

    for (int i_f = 0 ; i_f < arg->nFldSrc ; i_f++){
      char *dst_i = (char*)dst + arg->rec_size * (
          i_sp + arg->sp_locvol * (
          i_f  + arg->nFldDst   * i_t));
      const char *src_i = (char*)src + arg->rec_size * (
          x_cb + arg->volumeCB * (
          pty + 2 * i_f));

      for (int j = 0 ; j < arg->rec_size ; j++)
        *dst_i++ = *src_i++;
    }//- i_f
    
  }//-- function
  
  void conv_siteOrder_QudaQdp_to_momproj(void *corrInp_dev, const void *corrQuda_dev, QluaUtilArg utilArg){

    QluaUtilArg *utilArg_dev;
    hipMalloc((void**)&(utilArg_dev), sizeof(QluaUtilArg));
    checkCudaErrorNoSync();
    hipMemcpy(utilArg_dev, &utilArg,  sizeof(QluaUtilArg), hipMemcpyHostToDevice);

    dim3 blockDim(THREADS_PER_BLOCK, utilArg.nParity, 1);
    dim3 gridDim((utilArg.volumeCB + blockDim.x -1)/blockDim.x, 1, 1);

    conv_siteOrder_QudaQdp_to_momproj_kernel<<<gridDim,blockDim>>>(corrInp_dev, corrQuda_dev, utilArg_dev);
    checkCudaError();
    
    hipFree(utilArg_dev);
  }
  //---------------------------------------------------------------------------

  
  __device__ __host__ inline void RunPropagatorTransform(complex<QC_REAL> *devProp, QluaContractArg *arg, int x_cb, int pty){

    const int Ns = QC_Ns;
    const int Nc = QC_Nc;

    typedef ColorSpinor<QC_REAL,Nc,Ns> Vector;
    Vector vec[QUDA_PROP_NVEC];

    for(int i=0;i<QUDA_PROP_NVEC;i++){
      vec[i] = arg->prop[i](x_cb, pty);
    }
    rotatePropBasis(vec,QLUA_quda2qdp); //-- Rotate basis back to the QDP conventions

    int crd[5];
    getCoords(crd, x_cb, arg->lL, pty);  //-- Get local coordinates crd[] at given x_cb and pty
    crd[4] = 0;  

    int i_QudaQdp = x_cb + pty * arg->volumeCB;
    int lV = 2*arg->volumeCB;

    for(int jc = 0; jc < Nc; jc++){
      for(int js = 0; js < Ns; js++){
	int vIdx = js + Ns*jc;     //-- vector index (which vector within propagator)
	for(int ic = 0; ic < Nc; ic++){
	  for(int is = 0; is < Ns; is++){
	    int dIdx = ic + Nc*is; //-- spin-color index within each vector

	    int pIdx = i_QudaQdp + lV*QC_QUDA_LIDX_P(ic,is,jc,js);	    
	    
	    devProp[pIdx] = vec[vIdx].data[dIdx];
	  }}}
    }
    
  }//--function
  //------------------------------------------------------------------------------------------

  
  __global__ void propagatorTransform_kernel(complex<QC_REAL> *devProp, QluaContractArg *arg){

    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;

    int pty  = blockIdx.y*blockDim.y + threadIdx.y;

    if (x_cb >= arg->volumeCB) return;
    if (pty >= arg->nParity) return;

    RunPropagatorTransform(devProp, arg, x_cb, pty);
  }
  //------------------------------------------------------------------------------------------

  
  void propagatorTransform(complex<QC_REAL> *devProp, ColorSpinorField **propIn, int parity){//, int t_axis){
    
    QluaContractArg arg(propIn, parity);//, t_axis);
    
    if(arg.nParity != 2) errorQuda("run_propagatorTransform: This function supports only Full Site Subset spinors!\n");
    
    QluaContractArg *arg_dev;
    hipMalloc((void**)&(arg_dev), sizeof(QluaContractArg) );
    checkCudaErrorNoSync();
    hipMemcpy(arg_dev, &arg, sizeof(QluaContractArg), hipMemcpyHostToDevice);

    dim3 blockDim(THREADS_PER_BLOCK, arg.nParity, 1);
    dim3 gridDim((arg.volumeCB + blockDim.x -1)/blockDim.x, 1, 1);

    propagatorTransform_kernel<<<gridDim,blockDim>>>(devProp, arg_dev);
    checkCudaError();

    hipFree(arg_dev);
  }
  //------------------------------------------------------------------------------------------

  
  //-Top-level function in GPU contractions
  void contractGPU_baryon_sigma_twopt_asymsrc_gvec(complex<QC_REAL> *corrQuda_dev,
						   ColorSpinorField **cudaProp1,
						   ColorSpinorField **cudaProp2,
						   ColorSpinorField **cudaProp3,
						   complex<QC_REAL> *S2, complex<QC_REAL> *S1,
						   momProjParam mpParam){

    char *func_name;
    asprintf(&func_name,"contractGPU_baryon_sigma_twopt_asymsrc_gvec");

    LONG_T locvol = mpParam.locvol;
    
    //-- Transform the propagators
    size_t propSizeCplx = sizeof(complex<QC_REAL>) * locvol * QUDA_Nc*QUDA_Nc * QUDA_Ns*QUDA_Ns;
    printfQuda("%s: propSizeCplx = %lld bytes\n", func_name, (LONG_T)propSizeCplx);
    
    complex<QC_REAL> *prop1_dev = NULL;
    complex<QC_REAL> *prop2_dev = NULL;
    complex<QC_REAL> *prop3_dev = NULL;
    
    hipMalloc((void**)&prop1_dev, propSizeCplx );
    hipMalloc((void**)&prop2_dev, propSizeCplx );
    hipMalloc((void**)&prop3_dev, propSizeCplx );
    checkCudaErrorNoSync();
    hipMemset(prop1_dev, 0, propSizeCplx);
    hipMemset(prop2_dev, 0, propSizeCplx);
    hipMemset(prop3_dev, 0, propSizeCplx);
    
    int parity = 0; //-- not functional for full-site fields, just set it to zero
    propagatorTransform(prop1_dev, cudaProp1, parity);
    propagatorTransform(prop2_dev, cudaProp2, parity);
    propagatorTransform(prop3_dev, cudaProp3, parity);
    printfQuda("%s: Propagators transformed\n", func_name);
    //-------------------------------------------------------------
    
    //-- allocate local volume on device
    LONG_T *locvol_dev;
    hipMalloc((void**)&locvol_dev, sizeof(LONG_T));
    checkCudaErrorNoSync();
    hipMemcpy(locvol_dev, &locvol, sizeof(LONG_T), hipMemcpyHostToDevice);

    //-- allocate S-matrices on device
    size_t SmatSize = sizeof(complex<QC_REAL>)*QUDA_LEN_G;
    complex<QC_REAL> *S2_dev, *S1_dev;
    hipMalloc((void**)&S2_dev, SmatSize);
    hipMalloc((void**)&S1_dev, SmatSize);
    checkCudaErrorNoSync();    
    hipMemcpy(S2_dev, S2, SmatSize, hipMemcpyHostToDevice);
    hipMemcpy(S1_dev, S1, SmatSize, hipMemcpyHostToDevice);

    
    //-- Call the kernel wrapper to perform contractions
    dim3 blockDim(THREADS_PER_BLOCK, 1, 1);
    dim3 gridDim((locvol + blockDim.x - 1)/blockDim.x, 1, 1);

    baryon_sigma_twopt_asymsrc_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, locvol_dev,
    								 prop1_dev, prop2_dev, prop3_dev,
    								 S2_dev, S1_dev);
    checkCudaError();

    
    //-- Clean-up
    hipFree(prop1_dev);
    hipFree(prop2_dev);
    hipFree(prop3_dev);
    hipFree(S2_dev);
    hipFree(S1_dev);
    hipFree(locvol_dev);

  }//-- function
  
} //-namespace quda



    
    /* int i_rlex = crd[0] + arg->lL[0]*( */
    /* 	         crd[1] + arg->lL[1]*( */
    /* 	         crd[2] + arg->lL[2]*( */
    /* 	         crd[3])));  //-- Full lattice site index */
    /* int i_par = (crd[0] + crd[1] + crd[2] + crd[3]) & 1; */

    /* if (i_rlex / 2 != x_cb || pty != i_par) */
    /*   printf("x_cb = %d, pty = %d: site order mismatch\n", x_cb, pty); */



    /* hipMemcpy(prop1_dev, hprop1, propSizeCplx, hipMemcpyHostToDevice); */
    /* hipMemcpy(prop2_dev, hprop2, propSizeCplx, hipMemcpyHostToDevice); */
    /* hipMemcpy(prop3_dev, hprop3, propSizeCplx, hipMemcpyHostToDevice); */


//int pIdx = is + Ns*ic + Ns*Nc*iv + Ns*Nc*lV*js + Ns*Nc*lV*Ns*jc;  //-- QDP propagator index order (spin-inside-color-inside-volume, spin-inside-color for each vector)


//	    printf("(x_cb,pty) = %d , %d , (jc,js,ic,is,iv) = %d , %d , %d , %d , %d , devProp[%d] = %+lf  %+lf * i\n",
//		   x_cb, pty, jc, js, ic, is, iv, pIdx3, devProp[pIdx3].real(), devProp[pIdx3].imag());



    /* printfQuda("propagatorTransform:\n"); */
    /* printfQuda("  arg.parity   = %d\n", arg.parity); */
    /* printfQuda("  arg.nParity  = %d\n", arg.nParity); */
    /* printfQuda("  arg.nFace    = %d\n", arg.nFace); */
    /* printfQuda("  arg.volumeCB = %d\n", arg.volumeCB); */
    /* printfQuda("  arg.volume   = %d\n", arg.volume); */
    /* printfQuda("  arg.dim      = [%d,%d,%d,%d]\n", arg.dim[0], arg.dim[1], arg.dim[2], arg.dim[3]); */
    /* printfQuda("  arg.commDim  = [%d,%d,%d,%d]\n", arg.commDim[0], arg.commDim[1], arg.commDim[2], arg.commDim[3]); */
    /* printfQuda("  arg.lL   = [%d,%d,%d,%d]\n", arg.lL[0], arg.lL[1], arg.lL[2], arg.lL[3]); */



    /* int r1 = crd / arg->lL[0]; */
    /* int r2 = r1  / arg->lL[1]; */
    /* int locx = crd - r1 * arg->lL[0]; */
    /* int locy = r1  - r2 * arg->lL[1]; */
    /* int loct = r2 / arg->lL[2]; */
    /* int locz = r2 - loct * arg->lL[2]; */

    /* int evenSiteBit = (locx + locy + locz + loct) & 1; */
    /* int oddSiteBit  = evenSiteBit ^ 1; */
    
    /* int sBit = (pty == 0) ? evenSiteBit : oddSiteBit; //-- TODO: Need to verify/check this */
