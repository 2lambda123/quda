#include "hip/hip_runtime.h"
#include <transfer.h>
#include <quda_internal.h>
#include <quda_matrix.h>
#include <index_helper.cuh>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <color_spinor.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <mpi.h>
#include <qlua_contract.h>
#include <qlua_contract_kernels.cuh>
#include <qlua_contract_shifts.cuh>

namespace quda {  

  
  __global__ void QluaSiteOrderCheck_kernel(QluaUtilArg *utilArg){

    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;    
    int pty  = blockIdx.y*blockDim.y + threadIdx.y;

    if (x_cb >= utilArg->volumeCB) return;
    if (pty >= utilArg->nParity) return;

    int crd[5];
    getCoords(crd, x_cb, utilArg->lL, pty);  //-- Get local coordinates crd[] at given x_cb and pty
    crd[4] = 0;  

    int idx_cb = linkIndex(crd, utilArg->lL); //-- Checkerboard index, MUST be equal to x_cb
    
    int i_rlex = crd[0] + utilArg->lL[0]*(crd[1] + utilArg->lL[1]*(crd[2] + utilArg->lL[2]*(crd[3])));  //-- Full lattice site index
    int i_par = (crd[0] + crd[1] + crd[2] + crd[3]) & 1;

    if( (i_rlex/2 != x_cb) || (pty != i_par) || (idx_cb != x_cb) ){
      d_crdChkVal = -1;
      printf("coordCheck - ERROR: x_cb = %d, pty = %d: Site order mismatch!\n", x_cb, pty);    
    }
    else d_crdChkVal = 0;

  }//-- function

  
  int QluaSiteOrderCheck(QluaUtilArg utilArg){
    int crdChkVal;

    QluaUtilArg *utilArg_dev;
    hipMalloc((void**)&(utilArg_dev), sizeof(QluaUtilArg));
    checkCudaErrorNoSync();
    hipMemcpy(utilArg_dev, &utilArg,  sizeof(QluaUtilArg), hipMemcpyHostToDevice);
    
    dim3 blockDim(THREADS_PER_BLOCK, utilArg.nParity, 1);
    dim3 gridDim((utilArg.volumeCB + blockDim.x -1)/blockDim.x, 1, 1);
    
    QluaSiteOrderCheck_kernel<<<gridDim,blockDim>>>(utilArg_dev);
    hipDeviceSynchronize();
    checkCudaError();
    hipMemcpyFromSymbol(&crdChkVal, HIP_SYMBOL(d_crdChkVal), sizeof(crdChkVal), 0, hipMemcpyDeviceToHost);
    checkCudaErrorNoSync();
    
    hipFree(utilArg_dev);
    
    return crdChkVal;
  }//-- function
  //---------------------------------------------------------------------------

  
  __global__ void convertSiteOrder_QudaQDP_to_momProj_kernel(void *dst, const void *src, QluaUtilArg *arg){

    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int pty  = blockIdx.y*blockDim.y + threadIdx.y;

    int crd[5];
    getCoords(crd, x_cb, arg->lL, pty);
    int i_t = crd[arg->t_axis];
    int i_sp = 0;

    for (int i = 0 ; i < 4 ; i++)
      i_sp += arg->sp_stride[i] * crd[i];

    for (int i_f = 0 ; i_f < arg->nFldSrc ; i_f++){
      char *dst_i = (char*)dst + arg->rec_size * (
          i_sp + arg->sp_locvol * (
          i_f  + arg->nFldDst   * i_t));
      const char *src_i = (char*)src + arg->rec_size * (
          x_cb + arg->volumeCB * (
          pty + 2 * i_f));

      for (int j = 0 ; j < arg->rec_size ; j++)
        *dst_i++ = *src_i++;
    }//- i_f
    
  }//-- function

  void convertSiteOrder_QudaQDP_to_momProj(void *corrInp_dev, const void *corrQuda_dev, QluaUtilArg utilArg){

    QluaUtilArg *utilArg_dev;
    hipMalloc((void**)&(utilArg_dev), sizeof(QluaUtilArg));
    checkCudaErrorNoSync();
    hipMemcpy(utilArg_dev, &utilArg,  sizeof(QluaUtilArg), hipMemcpyHostToDevice);

    dim3 blockDim(THREADS_PER_BLOCK, utilArg.nParity, 1);
    dim3 gridDim((utilArg.volumeCB + blockDim.x -1)/blockDim.x, 1, 1);

    convertSiteOrder_QudaQDP_to_momProj_kernel<<<gridDim,blockDim>>>(corrInp_dev, corrQuda_dev, utilArg_dev);
    hipDeviceSynchronize();
    checkCudaError();
    
    hipFree(utilArg_dev);
  }
  //---------------------------------------------------------------------------


  void perform_NonCovShiftPropOnAxis(qcTMD_ShiftString shfFlag, QluaContractArg *arg_dev, QluaAuxCntrArg *auxArg_dev, int vCB, int nPty){

    qcTMD_ShiftDir shfDir = qcShfDirNone;
    qcTMD_ShiftSgn shfSgn = qcShfSgnNone;

    switch(shfFlag){
    case qcShfStr_x: {
      shfDir = qcShfDir_x;
      shfSgn = qcShfSgnPlus;
    } break;
    case qcShfStr_X: {
      shfDir = qcShfDir_x;
      shfSgn = qcShfSgnMinus;
    } break;
    case qcShfStr_y: {
      shfDir = qcShfDir_y;
      shfSgn = qcShfSgnPlus;
    } break;
    case qcShfStr_Y: {
      shfDir = qcShfDir_y;
      shfSgn = qcShfSgnMinus;
    } break;
    case qcShfStr_z: {
      shfDir = qcShfDir_z;
      shfSgn = qcShfSgnPlus;
    } break;
    case qcShfStr_Z: {
      shfDir = qcShfDir_z;
      shfSgn = qcShfSgnMinus;
    } break;
    case qcShfStr_t: {
      shfDir = qcShfDir_t;
      shfSgn = qcShfSgnPlus;
    } break;
    case qcShfStr_T: {
      shfDir = qcShfDir_t;
      shfSgn = qcShfSgnMinus;
    } break;
    default: errorQuda("performNonCovShiftProp_OnAxis: Got un-supported shift flag, shfFlag = %s.\n", (shfFlag >=0 && shfFlag<20) ? qcTMD_ShiftStringArray[(int)shfFlag] : "None");
    }//-- switch    

    printfQuda("perform_NonCovShiftPropOnAxis: Got shift flag shfFlag = %s\n", qcTMD_ShiftStringArray[(int)shfFlag]);
    if( (shfSgn>=0 && shfSgn<2) && (shfDir>=0 && shfDir<4) ){
      printfQuda("perform_NonCovShiftPropOnAxis: Will perform an On-Axis non-covariant propagator shift in the %s%s direction\n", qcTMD_ShiftSgnArray[(int)shfSgn], qcTMD_ShiftDirArray[(int)shfDir]);
    }
    else{
      errorQuda("perform_NonCovShiftPropOnAxis: Got invalid shfDir and/or shfSgn.\n");
    }

    //-- Call kernel that performs non-covariant on axis propagator shift
    dim3 blockDim(THREADS_PER_BLOCK, nPty, 1);
    dim3 gridDim((vCB + blockDim.x -1)/blockDim.x, 1, 1);

    NonCovShiftPropOnAxis_kernel<<<gridDim,blockDim>>>(arg_dev, auxArg_dev, shfDir, shfSgn);
    hipDeviceSynchronize();
    checkCudaError();

  }//-- perform_NonCovShiftPropOnAxis
  //---------------------------------------------------------------------------

  void qcExchangeGhostProp(ColorSpinorField **prop){
    int nFace  = 1;
    int parity = 0; //prop[0]->SiteSubset();
    for(int i=0;i<QUDA_PROP_NVEC;i++){
      prop[i]->exchangeGhost((QudaParity)(1-parity), nFace, 0);
      hipDeviceSynchronize();
      checkCudaError();
    }
  }
  
  //-Top-level function in GPU contractions
  void QuarkContract_GPU(complex<QUDA_REAL> *corrQuda_dev,
			 ColorSpinorField **cudaProp1,
			 ColorSpinorField **cudaProp2,
			 ColorSpinorField **cudaProp3,
			 GaugeField *U,
			 complex<QUDA_REAL> *S2, complex<QUDA_REAL> *S1,
			 qudaAPI_Param paramAPI){    

    char *func_name;
    asprintf(&func_name,"QuarkContract_GPU");
    
    //-- C.K. Here we check in fact that the contractions precision (QC_REAL)
    //-- is the same as the one used throughout.
    if(typeid(QC_REAL) != typeid(QUDA_REAL)) errorQuda("%s: QUDA_REAL and QC_REAL type mismatch!\n", func_name);

    momProjParam mpParam = paramAPI.mpParam;

    //-- Take care of the ghost exchange (only for forward prop)
    if( (mpParam.cntrType == what_tmd_g_F_B) || (mpParam.cntrType == what_qpdf_g_F_B) ){
      double t7 = MPI_Wtime();
      int nFace  = 1;
      //      int parity = 0; //prop[0]->SiteSubset();
      for(int i=0;i<QUDA_PROP_NVEC;i++){
	cudaProp1[i]->exchangeGhost((QudaParity)1, nFace, 0);
	hipDeviceSynchronize();
	checkCudaError();
	// cudaProp1[i]->exchangeGhost((QudaParity)1, nFace, 0);
	// hipDeviceSynchronize();
	// checkCudaError();
      }
      //      qcExchangeGhostProp(cudaProp1);
      double t8 = MPI_Wtime();
      printfQuda("TIMING - %s: Propagator ghost exchange done in %f sec.\n", func_name, t8-t7);
    }

    //-- Define the arguments structure
    QluaContractArg arg(cudaProp1, cudaProp2, cudaProp3, U, mpParam.cntrType, paramAPI.preserveBasis); 
    if(arg.nParity != 2) errorQuda("%s: This function supports only Full Site Subset spinors!\n", func_name);
    QluaContractArg *arg_dev;
    hipMalloc((void**)&(arg_dev), sizeof(QluaContractArg) );
    checkCudaError();
    hipMemcpy(arg_dev, &arg, sizeof(QluaContractArg), hipMemcpyHostToDevice);    
    hipDeviceSynchronize();
    checkCudaError();


    //-- Call kernels that perform contractions
    dim3 blockDim(THREADS_PER_BLOCK, arg.nParity, 1);
    dim3 gridDim((arg.volumeCB + blockDim.x -1)/blockDim.x, 1, 1);


    double t5 = MPI_Wtime();
    switch(mpParam.cntrType){
    case what_baryon_sigma_UUS: {
      copySmatricesToSymbol(S2, S1);
      baryon_sigma_twopt_asymsrc_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_qbarq_g_F_B: {
      qbarq_g_P_P_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_qbarq_g_F_aB: {
      qbarq_g_P_aP_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_qbarq_g_F_hB: {
      qbarq_g_P_hP_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_meson_F_B: {
      meson_F_B_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_meson_F_aB: {
      meson_F_aB_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_meson_F_hB: {
      meson_F_hB_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_qpdf_g_F_B: {
      qpdf_g_P_P_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev);
    } break;
    case what_tmd_g_F_B: {
      //-- C.K. Define the structure containing the auxilliary propagators (and gauge fields later on...)
      //-- Use cudaProp3 as output propagator in the qpdf case
      QluaAuxCntrArg auxArg(cudaProp3, mpParam.cntrType);
      QluaAuxCntrArg *auxArg_dev;
      hipMalloc((void**)&(auxArg_dev), sizeof(QluaAuxCntrArg) );
      checkCudaError();
      hipMemcpy(auxArg_dev, &auxArg, sizeof(QluaAuxCntrArg), hipMemcpyHostToDevice);    
      hipDeviceSynchronize();
      checkCudaError();

      //-- Non-covariant on-axis shift of propagator, test case
      qcTMD_ShiftString shfFlag;
      shfFlag = qcShfStr_y;
      double t9 = MPI_Wtime();
      perform_NonCovShiftPropOnAxis(shfFlag, arg_dev, auxArg_dev, arg.volumeCB, arg.nParity);
      double t10 = MPI_Wtime();
      printfQuda("TIMING - %s: Propagator shift finished in %f sec.\n", func_name, t10-t9);

      //-- Perform contractions
      qtmd_g_P_P_gvec_kernel<<<gridDim,blockDim>>>(corrQuda_dev, arg_dev, auxArg_dev);
    } break;
    default: errorQuda("%s: Contraction type \'%s\' not supported!\n", func_name, qc_contractTypeStr[mpParam.cntrType]);
    }//-- switch
    hipDeviceSynchronize();
    checkCudaError();
    double t6 = MPI_Wtime();
    printfQuda("TIMING - %s: Contraction kernel \'%s\' finished in %f sec.\n", func_name, qc_contractTypeStr[mpParam.cntrType], t6-t5);
    
    //-- Clean-up
    hipFree(arg_dev);
    //    hipFree(auxArg_dev);
    free(func_name);
    
  }//-- function
  
} //-namespace quda
